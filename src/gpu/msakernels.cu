#include "hip/hip_runtime.h"
//#define NDEBUG

#include <gpu/kernels.hpp>
#include <hostdevicefunctions.cuh>

//#include <gpu/bestalignment.hpp>
#include <bestalignment.hpp>
#include <gpu/utility_kernels.cuh>

//#include <msa.hpp>
#include <sequence.hpp>

#include <gpu/gpumsa.cuh>


#include <hpc_helpers.cuh>
#include <config.hpp>

#include <cassert>


#include <hipcub/hipcub.hpp>

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

#include <thrust/binary_search.h>


namespace care{
namespace gpu{

    enum class MemoryType{
        Global,
        Shared
    };

    enum class SequenceLayout{
        Transposed,
        Linear
    };


    __device__ __forceinline__
    void checkBuiltMSA(
            const MSAColumnProperties* __restrict__ msaColumnProperties,
            const int* __restrict__ counts,
            const float* __restrict__ weights,
            size_t msaColumnPitchInElements,
            int subjectIndex){

        const int firstColumn_incl = msaColumnProperties->firstColumn_incl;
        const int lastColumn_excl = msaColumnProperties->lastColumn_excl;

        for(int column = firstColumn_incl + threadIdx.x; column < lastColumn_excl; column += blockDim.x){
            const int* const mycounts = counts + column;
            const float* const myweights = weights + column;
            float sumOfWeights = 0.0f;

            for(int k = 0; k < 4; k++){
                const int count = mycounts[k * msaColumnPitchInElements];
                const float weight = myweights[k * msaColumnPitchInElements];
                if(count > 0 && weight <= 0.0f){
                    printf("msa check failed! subjectIndex %d, column %d, base %d, count %d, weight %f\n",
                        subjectIndex, column, k, count, weight);
                    assert(false);
                }

                if(count <= 0 && weight > 0.0f){
                    printf("msa check failed! subjectIndex %d, column %d, base %d, count %d, weight %f\n",
                        subjectIndex, column, k, count, weight);
                    assert(false);
                }

                sumOfWeights += weight;
            }

            if(sumOfWeights == 0){
                printf("s %d c %d\n", subjectIndex, column);
                assert(sumOfWeights != 0);
            }
        }
    }

    template<int BLOCKSIZE>
    __device__ __forceinline__
    void msaInit(
            typename hipcub::BlockReduce<int, BLOCKSIZE>::TempStorage& tempReduce,
            MSAColumnProperties* __restrict__ msaColumnProperties,
            const int* __restrict__ goodCandidateIndices,
            int numGoodCandidates,
            const int* __restrict__ shifts,
            const BestAlignment_t* __restrict__ alignmentFlags,
            const int subjectLength,
            const int* __restrict__ candidateLengths
            ){

        using BlockReduceInt = hipcub::BlockReduce<int, BLOCKSIZE>;

        int startindex = 0;
        int endindex = subjectLength;

        for(int k = threadIdx.x; k < numGoodCandidates; k += BLOCKSIZE) {
            const int localCandidateIndex = goodCandidateIndices[k];

            const int shift = shifts[localCandidateIndex];
            const BestAlignment_t flag = alignmentFlags[localCandidateIndex];
            const int queryLength = candidateLengths[localCandidateIndex];

            assert(flag != BestAlignment_t::None);

            const int queryEndsAt = queryLength + shift;
            startindex = min(startindex, shift);
            endindex = max(endindex, queryEndsAt);
        }

        startindex = BlockReduceInt(tempReduce).Reduce(startindex, hipcub::Min());
        __syncthreads();

        endindex = BlockReduceInt(tempReduce).Reduce(endindex, hipcub::Max());
        __syncthreads();

        if(threadIdx.x == 0) {
            MSAColumnProperties my_columnproperties;

            my_columnproperties.subjectColumnsBegin_incl = max(-startindex, 0);
            my_columnproperties.subjectColumnsEnd_excl = my_columnproperties.subjectColumnsBegin_incl + subjectLength;
            my_columnproperties.firstColumn_incl = 0;
            my_columnproperties.lastColumn_excl = endindex - startindex;

            *msaColumnProperties = my_columnproperties;
        }
    }

    template<int BLOCKSIZE>
    __device__ __forceinline__
    void msaUpdatePropertiesAfterSequenceRemovalSingleBlock(
            MSAColumnProperties* __restrict__ myMsaColumnProperties,
            const int* __restrict__ myCoverage){

        const int firstColumn_incl = myMsaColumnProperties->firstColumn_incl;
        const int lastColumn_excl = myMsaColumnProperties->lastColumn_excl;
        const int numColumnsToCheck = lastColumn_excl - firstColumn_incl;

        int newFirstColumn_incl = -1;
        int newLastColumn_excl = -1;

        for(int i = threadIdx.x; i < numColumnsToCheck-1; i += BLOCKSIZE){
            const int column = firstColumn_incl + i;

            const int thisCoverage = myCoverage[column];
            const int nextCoverage = myCoverage[column+1];
            assert(thisCoverage >= 0);
            assert(nextCoverage >= 0);

            if(thisCoverage == 0 && nextCoverage > 0){
                newFirstColumn_incl = column+1;
            }

            if(thisCoverage > 0 && nextCoverage == 0){
                newLastColumn_excl = column+1;
            }
        }

        __syncthreads();

        __shared__ int checkcount[2];
        if(threadIdx.x == 0){
            checkcount[0] = 0;
            checkcount[1] = 0;
        }
        __syncthreads();

        //there can be at most one thread for which this is true
        if(newFirstColumn_incl != -1){    
            atomicAdd(&checkcount[0], 1);
            myMsaColumnProperties->firstColumn_incl = newFirstColumn_incl;
        }
        //there can be at most one thread for which this is true
        if(newLastColumn_excl != -1){
            atomicAdd(&checkcount[1], 1);
            myMsaColumnProperties->lastColumn_excl = newLastColumn_excl;
        }
        
        __syncthreads();
        if(threadIdx.x == 0){
            assert(checkcount[0] <= 1);
            assert(checkcount[1] <= 1);
        }
    }





    template<bool isTransposedSequence, bool doAdd>
    __device__ 
    void msaAddOrDeleteASequence2Bit(
            int* __restrict__ counts,
            float* __restrict__ weights,
            int* __restrict__ coverages,
            int msaColumnPitchInElements,
            const unsigned int* sequence, 
            int sequenceLength, 
            bool isForward,
            int columnStart,
            float overlapweight,
            const char* quality,
            bool canUseQualityScores,
            int sequenceelementoffset){

        auto getEncodedNucFromInt2Bit = [](unsigned int data, int pos){
            return ((data >> (30 - 2*pos)) & 0x00000003);
        };

        constexpr int nucleotidesPerInt2Bit = 16;
        const int fullInts = sequenceLength / nucleotidesPerInt2Bit;

        for(int intIndex = 0; intIndex < fullInts; intIndex++){
            const unsigned int currentDataInt = sequence[intIndex * sequenceelementoffset];

            for(int k = 0; k < 4; k++){
                alignas(4) char currentFourQualities[4];

                assert(size_t(&currentFourQualities[0]) % 4 == 0);

                if(canUseQualityScores){
                    *((int*)&currentFourQualities[0]) = ((const int*)quality)[intIndex * 4 + k];
                }

                for(int l = 0; l < 4; l++){
                    const int posInInt = k * 4 + l;

                    unsigned int encodedBaseAsInt = getEncodedNucFromInt2Bit(currentDataInt, posInInt);
                    if(!isForward){
                        //reverse complement
                        encodedBaseAsInt = (~encodedBaseAsInt & 0x00000003);
                    }
                    const float weight = canUseQualityScores ? getQualityWeight(currentFourQualities[l]) * overlapweight : overlapweight;

                    assert(weight != 0);
                    const int rowOffset = encodedBaseAsInt * msaColumnPitchInElements;
                    const int columnIndex = columnStart 
                            + (isForward ? (intIndex * 16 + posInInt) : sequenceLength - 1 - (intIndex * 16 + posInInt));
                    
                    atomicAdd(counts + rowOffset + columnIndex, doAdd ? 1 : -1);
                    float n = atomicAdd(weights + rowOffset + columnIndex, doAdd ? weight : -weight);
                    atomicAdd(coverages + columnIndex, doAdd ? 1 : -1);

                    // if(columnIndex == 26){
                    //     float aaa[4]{0,0,0,0};
                    //     aaa[encodedBaseAsInt] = weight;

                    //     printf("column 26 add qscore %c, qual weight %.8f overlap weight %.8f, %.10f %.10f %.10f %.10f. new %.10f\n", 
                    //         currentFourQualities[l], getQualityWeight(currentFourQualities[l]),overlapweight,aaa[0], aaa[1], aaa[2], aaa[3], n + weight);
                    // }
                }
            }
        }

        //add remaining positions
        if(sequenceLength % nucleotidesPerInt2Bit != 0){
            const unsigned int currentDataInt = sequence[fullInts * sequenceelementoffset];
            const int maxPos = sequenceLength - fullInts * 16;
            for(int posInInt = 0; posInInt < maxPos; posInInt++){
                unsigned int encodedBaseAsInt = getEncodedNucFromInt2Bit(currentDataInt, posInInt);
                if(!isForward){
                    //reverse complement
                    encodedBaseAsInt = (~encodedBaseAsInt & 0x00000003);
                }
                const float weight = canUseQualityScores ? getQualityWeight(quality[fullInts * 16 + posInInt]) * overlapweight : overlapweight;

                assert(weight != 0);
                const int rowOffset = encodedBaseAsInt * msaColumnPitchInElements;
                const int columnIndex = columnStart 
                    + (isForward ? (fullInts * 16 + posInInt) : sequenceLength - 1 - (fullInts * 16 + posInInt));
                    atomicAdd(counts + rowOffset + columnIndex, doAdd ? 1 : -1);
                    atomicAdd(weights + rowOffset + columnIndex, doAdd ? weight : -weight);
                    atomicAdd(coverages + columnIndex, doAdd ? 1 : -1);
            } 
        }
    }


    template<bool isTransposedSequence, bool doAdd, class ThreadGroup>
    __device__ 
    void msaAddOrDeleteASequence2Bit(
            ThreadGroup& threadGroup,
            int* __restrict__ counts,
            float* __restrict__ weights,
            int* __restrict__ coverages,
            int msaColumnPitchInElements,
            const unsigned int* sequence, 
            int sequenceLength, 
            bool isForward,
            int columnStart,
            float overlapweight,
            const char* quality,
            bool canUseQualityScores,
            int sequenceelementoffset){

        auto getEncodedNucFromInt2Bit = [](unsigned int data, int pos){
            return ((data >> (30 - 2*pos)) & 0x00000003);
        };

        constexpr int nucleotidesPerInt2Bit = 16;
        const int fullInts = sequenceLength / nucleotidesPerInt2Bit;

        for(int intIndex = threadGroup.thread_rank(); intIndex < fullInts; intIndex += threadGroup.size()){
        //for(int intIndex = 0; intIndex < fullInts; intIndex++){
            const unsigned int currentDataInt = sequence[intIndex * sequenceelementoffset];

            for(int k = 0; k < 4; k++){
                alignas(4) char currentFourQualities[4];

                assert(size_t(&currentFourQualities[0]) % 4 == 0);

                if(canUseQualityScores){
                    *((int*)&currentFourQualities[0]) = ((const int*)quality)[intIndex * 4 + k];
                }

                for(int l = 0; l < 4; l++){
                    const int posInInt = k * 4 + l;

                    unsigned int encodedBaseAsInt = getEncodedNucFromInt2Bit(currentDataInt, posInInt);
                    if(!isForward){
                        //reverse complement
                        encodedBaseAsInt = (~encodedBaseAsInt & 0x00000003);
                    }
                    const float weight = canUseQualityScores ? getQualityWeight(currentFourQualities[l]) * overlapweight : overlapweight;

                    assert(weight != 0);
                    const int rowOffset = encodedBaseAsInt * msaColumnPitchInElements;
                    const int columnIndex = columnStart 
                            + (isForward ? (intIndex * 16 + posInInt) : sequenceLength - 1 - (intIndex * 16 + posInInt));
                    
                    atomicAdd(counts + rowOffset + columnIndex, doAdd ? 1 : -1);
                    float n = atomicAdd(weights + rowOffset + columnIndex, doAdd ? weight : -weight);
                    atomicAdd(coverages + columnIndex, doAdd ? 1 : -1);

                    // if(columnIndex == 26){
                    //     float aaa[4]{0,0,0,0};
                    //     aaa[encodedBaseAsInt] = weight;

                    //     printf("column 26 add qscore %c, qual weight %.8f overlap weight %.8f, %.10f %.10f %.10f %.10f. new %.10f\n", 
                    //         currentFourQualities[l], getQualityWeight(currentFourQualities[l]),overlapweight,aaa[0], aaa[1], aaa[2], aaa[3], n + weight);
                    // }
                }
            }
        }

        //add remaining positions
        if(sequenceLength % nucleotidesPerInt2Bit != 0){
            const unsigned int currentDataInt = sequence[fullInts * sequenceelementoffset];
            const int maxPos = sequenceLength - fullInts * 16;
            //for(int posInInt = 0; posInInt < maxPos; posInInt++){
            for(int posInInt = threadGroup.thread_rank(); posInInt < maxPos; posInInt += threadGroup.size()){
                unsigned int encodedBaseAsInt = getEncodedNucFromInt2Bit(currentDataInt, posInInt);
                if(!isForward){
                    //reverse complement
                    encodedBaseAsInt = (~encodedBaseAsInt & 0x00000003);
                }
                const float weight = canUseQualityScores ? getQualityWeight(quality[fullInts * 16 + posInInt]) * overlapweight : overlapweight;

                assert(weight != 0);
                const int rowOffset = encodedBaseAsInt * msaColumnPitchInElements;
                const int columnIndex = columnStart 
                    + (isForward ? (fullInts * 16 + posInInt) : sequenceLength - 1 - (fullInts * 16 + posInInt));
                    atomicAdd(counts + rowOffset + columnIndex, doAdd ? 1 : -1);
                    atomicAdd(weights + rowOffset + columnIndex, doAdd ? weight : -weight);
                    atomicAdd(coverages + columnIndex, doAdd ? 1 : -1);
            } 
        }
    }



    template<int BLOCKSIZE, bool isTransposedSequence, bool doAdd>
    __device__ 
    void msaAddOrDeleteASequence2Bit_wholeblock(
            int* __restrict__ counts,
            float* __restrict__ weights,
            int* __restrict__ coverages,
            int msaColumnPitchInElements,
            const unsigned int* sequence, 
            int sequenceLength, 
            bool isForward,
            int columnStart,
            float overlapweight,
            const char* quality,
            bool canUseQualityScores,
            int sequenceelementoffset){

        auto getEncodedNucFromInt2Bit = [](unsigned int data, int pos){
            return ((data >> (30 - 2*pos)) & 0x00000003);
        };

        for(int k = threadIdx.x; k < sequenceLength; k += BLOCKSIZE){
		char qual = 'A';
            if(canUseQualityScores){
                qual = quality[k];
            }
            const unsigned int currentDataInt = sequence[(k / 16) * sequenceelementoffset];
            unsigned int encodedBase = getEncodedNucFromInt2Bit(currentDataInt, k % 16);
            if(!isForward){
                encodedBase = (~encodedBase & 0x00000003);
            }
            const float weight = canUseQualityScores ? getQualityWeight(qual) * overlapweight : overlapweight;
            assert(weight != 0);
            const int rowOffset = encodedBase * msaColumnPitchInElements;
            const int columnIndex = columnStart + (isForward ? k : sequenceLength - 1 - k);
//            counts[rowOffset + columnIndex] += (doAdd ? 1 : -1);
//            weights[rowOffset + columnIndex] += (doAdd ? weight : -weight);
//            coverages[columnIndex] += (doAdd ? 1 : -1);
            atomicAdd(counts + rowOffset + columnIndex, 1);
            atomicAdd(weights + rowOffset + columnIndex, weight);
            atomicAdd(coverages + columnIndex, 1);

       }
   }



    template<int BLOCKSIZE>
    __device__ __forceinline__
    void addSequencesToMSASingleBlock(
            int* __restrict__ inputcounts,
            float* __restrict__ inputweights,
            int* __restrict__ inputcoverages,
            const MSAColumnProperties* __restrict__ myMsaColumnProperties,
            const int* __restrict__ myShifts,
            const int* __restrict__ myOverlaps,
            const int* __restrict__ myNops,
            const BestAlignment_t* __restrict__ myAlignmentFlags,
            const unsigned int* __restrict__ myAnchorSequenceData,
            const char* __restrict__ myAnchorQualityData,
            const unsigned int* __restrict__ myTransposedCandidateSequencesData,
            const char* __restrict__ myCandidateQualities,
            const int* __restrict__ myCandidateLengths,
            const int* __restrict__ myIndices,
            int numIndices,
            size_t elementOffsetForTransposedCandidates,
            bool canUseQualityScores, 
            size_t msaColumnPitchInElements,
            size_t encodedSequencePitchInInts,
            size_t qualityPitchInBytes,
            float desiredAlignmentMaxErrorRate,
            int subjectIndex){  

        constexpr bool candidatesAreTransposed = true;

        int* const mycounts = inputcounts;
        float* const myweights = inputweights;
        int* const mycoverages = inputcoverages;        

        for(int column = threadIdx.x; column < msaColumnPitchInElements * 4; column += BLOCKSIZE){
            mycounts[column] = 0;
            myweights[column] = 0;
        }

        for(int column = threadIdx.x; column < msaColumnPitchInElements; column += BLOCKSIZE){
            mycoverages[column] = 0;
        }   
        
        __syncthreads();

        //add subject
        const int subjectColumnsBegin_incl = myMsaColumnProperties->subjectColumnsBegin_incl;
        const int subjectColumnsEnd_excl = myMsaColumnProperties->subjectColumnsEnd_excl;

        const int subjectLength = subjectColumnsEnd_excl - subjectColumnsBegin_incl;
        const unsigned int* const subject = myAnchorSequenceData;
        const char* const subjectQualityScore = myAnchorQualityData;
                        
        for(int i = threadIdx.x; i < subjectLength; i += BLOCKSIZE){
            const int columnIndex = subjectColumnsBegin_incl + i;
            const unsigned int encbase = getEncodedNuc2Bit(subject, subjectLength, i);
            const float weight = canUseQualityScores ? getQualityWeight(subjectQualityScore[i]) : 1.0f;
            const int rowOffset = int(encbase) * msaColumnPitchInElements;

            atomicAdd(mycounts + rowOffset + columnIndex, 1);
            atomicAdd(myweights + rowOffset + columnIndex, weight);
            atomicAdd(mycoverages + columnIndex, 1);
        }

        for(int indexInList = threadIdx.x; indexInList < numIndices; indexInList += BLOCKSIZE){

            const int localCandidateIndex = myIndices[indexInList];
            const int shift = myShifts[localCandidateIndex];
            const BestAlignment_t flag = myAlignmentFlags[localCandidateIndex];

            const int queryLength = myCandidateLengths[localCandidateIndex];
            const unsigned int* const query = myTransposedCandidateSequencesData + localCandidateIndex;

            const char* const queryQualityScore = myCandidateQualities + std::size_t(localCandidateIndex) * qualityPitchInBytes;

            const int query_alignment_overlap = myOverlaps[localCandidateIndex];
            const int query_alignment_nops = myNops[localCandidateIndex];

            const float overlapweight = calculateOverlapWeight(
                subjectLength, 
                query_alignment_nops, 
                query_alignment_overlap,
                desiredAlignmentMaxErrorRate
            );

            assert(overlapweight <= 1.0f);
            assert(overlapweight >= 0.0f);
            assert(flag != BestAlignment_t::None);                 // indices should only be pointing to valid alignments

            const int defaultcolumnoffset = subjectColumnsBegin_incl + shift;

            const bool isForward = flag == BestAlignment_t::Forward;

            // if(subjectIndex == 2){
            //     for(int x = 0; x < BLOCKSIZE; x++){
            //         if(threadIdx.x == x){
            //             printf("subject 2, indexInList %d, localCandidateIndex %d, isForward %d, shift %d\n",  
            //                     indexInList, localCandidateIndex, isForward, shift);
            //             for(int i = 0; i < queryLength; i += 1){
            //                 const unsigned int encbase = getEncodedNuc2Bit(query, queryLength, i, [&](auto p){return p * elementOffsetForTransposedCandidates;});
            //                 printf("%d ", encbase);
            //             }
            //             printf("\n");
            //         }
            //         __syncthreads();
            //     }
                
                
            // }

            msaAddOrDeleteASequence2Bit<candidatesAreTransposed, true>(
                mycounts,
                myweights,
                mycoverages,
                msaColumnPitchInElements,
                query, 
                queryLength, 
                isForward,
                defaultcolumnoffset,
                overlapweight,
                queryQualityScore,
                canUseQualityScores,
                elementOffsetForTransposedCandidates
            );
        }
    }



    template<int BLOCKSIZE>
    __device__ __forceinline__
    void addSequencesToMSASingleBlockNotTranposed(
            int* __restrict__ inputcounts,
            float* __restrict__ inputweights,
            int* __restrict__ inputcoverages,
            const MSAColumnProperties* __restrict__ myMsaColumnProperties,
            const int* __restrict__ myShifts,
            const int* __restrict__ myOverlaps,
            const int* __restrict__ myNops,
            const BestAlignment_t* __restrict__ myAlignmentFlags,
            const unsigned int* __restrict__ myAnchorSequenceData,
            const char* __restrict__ myAnchorQualityData,
            const unsigned int* __restrict__ myCandidateSequencesData,
            const char* __restrict__ myCandidateQualities,
            const int* __restrict__ myCandidateLengths,
            const int* __restrict__ myIndices,
            int numIndices,
            size_t elementOffsetForTransposedCandidates,
            bool canUseQualityScores, 
            size_t msaColumnPitchInElements,
            size_t encodedSequencePitchInInts,
            size_t qualityPitchInBytes,
            float desiredAlignmentMaxErrorRate,
            int subjectIndex){  

        constexpr bool candidatesAreTransposed = false;

        int* const mycounts = inputcounts;
        float* const myweights = inputweights;
        int* const mycoverages = inputcoverages;        

        for(int column = threadIdx.x; column < msaColumnPitchInElements * 4; column += BLOCKSIZE){
            mycounts[column] = 0;
            myweights[column] = 0;
        }

        for(int column = threadIdx.x; column < msaColumnPitchInElements; column += BLOCKSIZE){
            mycoverages[column] = 0;
        }   
        
        __syncthreads();

        //add subject
        const int subjectColumnsBegin_incl = myMsaColumnProperties->subjectColumnsBegin_incl;
        const int subjectColumnsEnd_excl = myMsaColumnProperties->subjectColumnsEnd_excl;

        const int subjectLength = subjectColumnsEnd_excl - subjectColumnsBegin_incl;
        const unsigned int* const subject = myAnchorSequenceData;
        const char* const subjectQualityScore = myAnchorQualityData;
                        
        for(int i = threadIdx.x; i < subjectLength; i += BLOCKSIZE){
            const int columnIndex = subjectColumnsBegin_incl + i;
            const unsigned int encbase = getEncodedNuc2Bit(subject, subjectLength, i);
            const float weight = canUseQualityScores ? getQualityWeight(subjectQualityScore[i]) : 1.0f;
            const int rowOffset = int(encbase) * msaColumnPitchInElements;

            atomicAdd(mycounts + rowOffset + columnIndex, 1);
            atomicAdd(myweights + rowOffset + columnIndex, weight);
            atomicAdd(mycoverages + columnIndex, 1);
        }

        constexpr int threadsPerSequence = 8;
        auto tile = cg::tiled_partition<threadsPerSequence>(cg::this_thread_block());
        const int tileIdInBlock = threadIdx.x / threadsPerSequence;
        const int numTilesInBlock = blockDim.x / threadsPerSequence;

        for(int indexInList = tileIdInBlock; indexInList < numIndices; indexInList += numTilesInBlock){
         //    const int indexInList = k / threadsPerSequence;
        //for(int indexInList = threadIdx.x; indexInList < numIndices; indexInList += BLOCKSIZE){


            const int localCandidateIndex = myIndices[indexInList];
            const int shift = myShifts[localCandidateIndex];
            const BestAlignment_t flag = myAlignmentFlags[localCandidateIndex];

            const int queryLength = myCandidateLengths[localCandidateIndex];
            const unsigned int* const query = myCandidateSequencesData + localCandidateIndex * encodedSequencePitchInInts;

            const char* const queryQualityScore = myCandidateQualities + std::size_t(localCandidateIndex) * qualityPitchInBytes;

            const int query_alignment_overlap = myOverlaps[localCandidateIndex];
            const int query_alignment_nops = myNops[localCandidateIndex];

            const float overlapweight = calculateOverlapWeight(
                subjectLength, 
                query_alignment_nops, 
                query_alignment_overlap,
                desiredAlignmentMaxErrorRate
            );

            assert(overlapweight <= 1.0f);
            assert(overlapweight >= 0.0f);
            assert(flag != BestAlignment_t::None);                 // indices should only be pointing to valid alignments

            const int defaultcolumnoffset = subjectColumnsBegin_incl + shift;

            const bool isForward = flag == BestAlignment_t::Forward;

            // msaAddOrDeleteASequence2Bit<candidatesAreTransposed, true>(
            //     mycounts,
            //     myweights,
            //     mycoverages,
            //     msaColumnPitchInElements,
            //     query, 
            //     queryLength, 
            //     isForward,
            //     defaultcolumnoffset,
            //     overlapweight,
            //     queryQualityScore,
            //     canUseQualityScores,
            //     1
            // );

            msaAddOrDeleteASequence2Bit<candidatesAreTransposed, true>(
                tile,
                mycounts,
                myweights,
                mycoverages,
                msaColumnPitchInElements,
                query, 
                queryLength, 
                isForward,
                defaultcolumnoffset,
                overlapweight,
                queryQualityScore,
                canUseQualityScores,
                1
            );
        }
    }







    template<int BLOCKSIZE>
    __device__ __forceinline__
    void addSequencesToMSASingleBlockLessAtomic(
            int* __restrict__ inputcounts,
            float* __restrict__ inputweights,
            int* __restrict__ inputcoverages,
            const MSAColumnProperties* __restrict__ myMsaColumnProperties,
            const int* __restrict__ myShifts,
            const int* __restrict__ myOverlaps,
            const int* __restrict__ myNops,
            const BestAlignment_t* __restrict__ myAlignmentFlags,
            const unsigned int* __restrict__ myAnchorSequenceData,
            const char* __restrict__ myAnchorQualityData,
            const unsigned int* __restrict__ myTransposedCandidateSequencesData,
            const char* __restrict__ myCandidateQualities,
            const int* __restrict__ myCandidateLengths,
            const int* __restrict__ myIndices,
            int numIndices,
            size_t elementOffsetForTransposedCandidates,
            bool canUseQualityScores, 
            size_t msaColumnPitchInElements,
            size_t encodedSequencePitchInInts,
            size_t qualityPitchInBytes,
            float desiredAlignmentMaxErrorRate,
            int subjectIndex){  

        constexpr bool candidatesAreTransposed = true;

        int* const mycounts = inputcounts;
        float* const myweights = inputweights;
        int* const mycoverages = inputcoverages;        

        for(int column = threadIdx.x; column < msaColumnPitchInElements * 4; column += BLOCKSIZE){
            mycounts[column] = 0;
            myweights[column] = 0;
        }

        for(int column = threadIdx.x; column < msaColumnPitchInElements; column += BLOCKSIZE){
            mycoverages[column] = 0;
        }   
        
        //__syncthreads();

        //add subject
        const int subjectColumnsBegin_incl = myMsaColumnProperties->subjectColumnsBegin_incl;
        const int subjectColumnsEnd_excl = myMsaColumnProperties->subjectColumnsEnd_excl;

        const int subjectLength = subjectColumnsEnd_excl - subjectColumnsBegin_incl;
        const unsigned int* const subject = myAnchorSequenceData;
        const char* const subjectQualityScore = myAnchorQualityData;
                        
        for(int i = threadIdx.x; i < subjectLength; i += BLOCKSIZE){
            const int columnIndex = subjectColumnsBegin_incl + i;
            const unsigned int encbase = getEncodedNuc2Bit(subject, subjectLength, i);
            const float weight = canUseQualityScores ? getQualityWeight(subjectQualityScore[i]) : 1.0f;
            const int rowOffset = int(encbase) * msaColumnPitchInElements;

            mycounts[rowOffset + columnIndex] = 1;
            myweights[rowOffset + columnIndex] = weight;
            mycoverages[columnIndex] = 1;
        }
        __syncthreads();

        const int firstColumn_incl = myMsaColumnProperties->firstColumn_incl;
        const int lastColumn_excl = myMsaColumnProperties->lastColumn_excl;

        //const int numOuterIters = SDIV(lastColumn_excl, 4);

        //for each column in msa
        for(int column = threadIdx.x; column < lastColumn_excl; column += BLOCKSIZE){

            if(firstColumn_incl <= column){

                int countA = 0;
                int countC = 0;
                int countG = 0;
                int countT = 0;
                float weightA = 0;
                float weightC = 0;
                float weightG = 0;
                float weightT = 0;
                int coverage = 0;

                //for each candidate
                for(int indexInList = 0; indexInList < numIndices; indexInList += 1){
                    const int localCandidateIndex = myIndices[indexInList];
                    const int shift = myShifts[localCandidateIndex];
                    const int queryLength = myCandidateLengths[localCandidateIndex];

                    //if candidate occupies the column, update column
                    if(subjectColumnsBegin_incl + shift <= column && column < subjectColumnsBegin_incl + shift + queryLength){
                    
                        const BestAlignment_t flag = myAlignmentFlags[localCandidateIndex];
            
                        const unsigned int* const query = myTransposedCandidateSequencesData + localCandidateIndex;
            
                        const char* const queryQualityScore = myCandidateQualities + std::size_t(localCandidateIndex) * qualityPitchInBytes;
            
                        const int query_alignment_overlap = myOverlaps[localCandidateIndex];
                        const int query_alignment_nops = myNops[localCandidateIndex];
            
                        const float overlapweight = calculateOverlapWeight(
                            subjectLength, 
                            query_alignment_nops, 
                            query_alignment_overlap,
                            desiredAlignmentMaxErrorRate
                        );

                        assert(overlapweight <= 1.0f);
                        assert(overlapweight >= 0.0f);
                        assert(flag != BestAlignment_t::None);                 // indices should only be pointing to valid alignments
            
                        //const int defaultcolumnoffset = subjectColumnsBegin_incl + shift;
            
                        const bool isForward = flag == BestAlignment_t::Forward;

                        
                        int positionInSequence = column - subjectColumnsBegin_incl - shift;
                        if(!isForward){
                            positionInSequence = queryLength - 1 - positionInSequence;
                        }                        

                        char quality = 'A';

                        auto getEncodedNucFromInt2Bit = [](unsigned int data, int pos){
                            return ((data >> (30 - 2*pos)) & 0x00000003);
                        };

                        const unsigned int currentDataInt = query[(positionInSequence / 16) * elementOffsetForTransposedCandidates];
                        unsigned int encodedBaseAsInt = getEncodedNucFromInt2Bit(currentDataInt, positionInSequence % 16);
                        if(!isForward){
                            //reverse complement
                            encodedBaseAsInt = (~encodedBaseAsInt & 0x00000003);
                        }
                        if(canUseQualityScores){
                            quality = queryQualityScore[positionInSequence];
                        }

                        
                        const float weight = canUseQualityScores ? getQualityWeight(quality) * overlapweight : overlapweight;

                        assert(weight != 0);

                        if(encodedBaseAsInt == 0){
                            countA++;
                            weightA += weight;
                        }else if(encodedBaseAsInt == 1){
                            countC++;
                            weightC += weight;
                        }else if(encodedBaseAsInt == 2){
                            countG++;
                            weightG += weight;
                        }else{
                            countT++;
                            weightT += weight;
                        }

                        coverage += 1;
                    }
                }

#if 0

                if(countA > 0){
                    mycounts[0 * msaColumnPitchInElements + column] += countA;
                }
                if(countC > 0){
                    mycounts[1 * msaColumnPitchInElements + column] += countC;
                }
                if(countG > 0){
                    mycounts[2 * msaColumnPitchInElements + column] += countG;
                }
                if(countT > 0){
                    mycounts[3 * msaColumnPitchInElements + column] += countT;
                }
                if(weightA > 0){
                    myweights[0 * msaColumnPitchInElements + column] += weightA;
                }
                if(weightC > 0){
                    myweights[1 * msaColumnPitchInElements + column] += weightC;
                }
                if(weightG > 0){
                    myweights[2 * msaColumnPitchInElements + column] += weightG;
                }
                if(weightT > 0){
                    myweights[3 * msaColumnPitchInElements + column] += weightT;
                }
                if(coverage > 0){
                    mycoverages[column] += coverage;
                }

#else 


                    mycounts[0 * msaColumnPitchInElements + column] += countA;
                    mycounts[1 * msaColumnPitchInElements + column] += countC;
                    mycounts[2 * msaColumnPitchInElements + column] += countG;
                    mycounts[3 * msaColumnPitchInElements + column] += countT;
                    myweights[0 * msaColumnPitchInElements + column] += weightA;
                    myweights[1 * msaColumnPitchInElements + column] += weightC;
                    myweights[2 * msaColumnPitchInElements + column] += weightG;
                    myweights[3 * msaColumnPitchInElements + column] += weightT;
                    mycoverages[column] += coverage;
#endif
            }
        }

    }








    template<int BLOCKSIZE, class Selector>
    __device__ __forceinline__
    void removeCandidatesFromMSASingleBlock(
            Selector shouldBeRemoved,
            int* __restrict__ inputcounts,
            float* __restrict__ inputweights,
            int* __restrict__ inputcoverages,
            const MSAColumnProperties* __restrict__ myMsaColumnProperties,
            const int* __restrict__ myShifts,
            const int* __restrict__ myOverlaps,
            const int* __restrict__ myNops,
            const BestAlignment_t* __restrict__ myAlignmentFlags,
            const unsigned int* __restrict__ myTransposedCandidateSequencesData,
            const char* __restrict__ myCandidateQualities,
            const int* __restrict__ myCandidateLengths,
            const int* __restrict__ myIndices,
            int numIndices,
            size_t elementOffsetForTransposedCandidates,
            bool canUseQualityScores, 
            size_t msaColumnPitchInElements,
            size_t encodedSequencePitchInInts,
            size_t qualityPitchInBytes,
            float desiredAlignmentMaxErrorRate,
            int subjectIndex){  

        constexpr bool candidatesAreTransposed = true;

        int* const mycounts = inputcounts;
        float* const myweights = inputweights;
        int* const mycoverages = inputcoverages;        

        const int subjectColumnsBegin_incl = myMsaColumnProperties->subjectColumnsBegin_incl;
        const int subjectColumnsEnd_excl = myMsaColumnProperties->subjectColumnsEnd_excl;
                        
        for(int indexInList = threadIdx.x; indexInList < numIndices; indexInList += BLOCKSIZE){

            if(shouldBeRemoved(indexInList)){

                const int localCandidateIndex = myIndices[indexInList];
                const int shift = myShifts[localCandidateIndex];
                const BestAlignment_t flag = myAlignmentFlags[localCandidateIndex];

                const int subjectLength = subjectColumnsEnd_excl - subjectColumnsBegin_incl;
                const int queryLength = myCandidateLengths[localCandidateIndex];
                const unsigned int* const query = myTransposedCandidateSequencesData + localCandidateIndex;

                const char* const queryQualityScore = myCandidateQualities + std::size_t(localCandidateIndex) * qualityPitchInBytes;

                const int query_alignment_overlap = myOverlaps[localCandidateIndex];
                const int query_alignment_nops = myNops[localCandidateIndex];

                const float overlapweight = calculateOverlapWeight(
                    subjectLength, 
                    query_alignment_nops, 
                    query_alignment_overlap,
                    desiredAlignmentMaxErrorRate
                );

                assert(overlapweight <= 1.0f);
                assert(overlapweight >= 0.0f);
                assert(flag != BestAlignment_t::None);                 // indices should only be pointing to valid alignments

                const int defaultcolumnoffset = subjectColumnsBegin_incl + shift;

                const bool isForward = flag == BestAlignment_t::Forward;

                msaAddOrDeleteASequence2Bit<candidatesAreTransposed, false>(
                    mycounts,
                    myweights,
                    mycoverages,
                    msaColumnPitchInElements,
                    query, 
                    queryLength, 
                    isForward,
                    defaultcolumnoffset,
                    overlapweight,
                    queryQualityScore,
                    canUseQualityScores,
                    elementOffsetForTransposedCandidates
                );
            }
        }
    }


    template<int BLOCKSIZE, class Selector>
    __device__ __forceinline__
    void removeCandidatesFromMSASingleBlockNotTransposed(
            Selector shouldBeRemoved,
            int* __restrict__ inputcounts,
            float* __restrict__ inputweights,
            int* __restrict__ inputcoverages,
            const MSAColumnProperties* __restrict__ myMsaColumnProperties,
            const int* __restrict__ myShifts,
            const int* __restrict__ myOverlaps,
            const int* __restrict__ myNops,
            const BestAlignment_t* __restrict__ myAlignmentFlags,
            const unsigned int* __restrict__ myCandidateSequencesData,
            const char* __restrict__ myCandidateQualities,
            const int* __restrict__ myCandidateLengths,
            const int* __restrict__ myIndices,
            int numIndices,
            size_t elementOffsetForTransposedCandidates,
            bool canUseQualityScores, 
            size_t msaColumnPitchInElements,
            size_t encodedSequencePitchInInts,
            size_t qualityPitchInBytes,
            float desiredAlignmentMaxErrorRate,
            int subjectIndex){  

        constexpr bool candidatesAreTransposed = true;

        int* const mycounts = inputcounts;
        float* const myweights = inputweights;
        int* const mycoverages = inputcoverages;        

        const int subjectColumnsBegin_incl = myMsaColumnProperties->subjectColumnsBegin_incl;
        const int subjectColumnsEnd_excl = myMsaColumnProperties->subjectColumnsEnd_excl;

        constexpr int threadsPerSequence = 8;
        auto tile = cg::tiled_partition<threadsPerSequence>(cg::this_thread_block());
        const int tileIdInBlock = threadIdx.x / threadsPerSequence;
        const int numTilesInBlock = blockDim.x / threadsPerSequence;
                        
        //for(int indexInList = threadIdx.x; indexInList < numIndices; indexInList += BLOCKSIZE){
        for(int indexInList = tileIdInBlock; indexInList < numIndices; indexInList += numTilesInBlock){

            if(shouldBeRemoved(indexInList)){

                const int localCandidateIndex = myIndices[indexInList];
                const int shift = myShifts[localCandidateIndex];
                const BestAlignment_t flag = myAlignmentFlags[localCandidateIndex];

                const int subjectLength = subjectColumnsEnd_excl - subjectColumnsBegin_incl;
                const int queryLength = myCandidateLengths[localCandidateIndex];
                const unsigned int* const query = myCandidateSequencesData + localCandidateIndex * encodedSequencePitchInInts;

                const char* const queryQualityScore = myCandidateQualities + std::size_t(localCandidateIndex) * qualityPitchInBytes;

                const int query_alignment_overlap = myOverlaps[localCandidateIndex];
                const int query_alignment_nops = myNops[localCandidateIndex];

                const float overlapweight = calculateOverlapWeight(
                    subjectLength, 
                    query_alignment_nops, 
                    query_alignment_overlap,
                    desiredAlignmentMaxErrorRate
                );

                assert(overlapweight <= 1.0f);
                assert(overlapweight >= 0.0f);
                assert(flag != BestAlignment_t::None);                 // indices should only be pointing to valid alignments

                const int defaultcolumnoffset = subjectColumnsBegin_incl + shift;

                const bool isForward = flag == BestAlignment_t::Forward;

                msaAddOrDeleteASequence2Bit<candidatesAreTransposed, false>(
                    tile,
                    mycounts,
                    myweights,
                    mycoverages,
                    msaColumnPitchInElements,
                    query, 
                    queryLength, 
                    isForward,
                    defaultcolumnoffset,
                    overlapweight,
                    queryQualityScore,
                    canUseQualityScores,
                    1
                );
            }
        }
    }







    template<int BLOCKSIZE>
    __device__ __forceinline__
    void findConsensusSingleBlock(
            float* __restrict__ my_support,
            float* __restrict__ my_orig_weights,
            int* __restrict__ my_orig_coverage,
            char* __restrict__ my_consensus,
            const MSAColumnProperties* __restrict__ myMsaColumnProperties, 
            const int* __restrict__ myCounts,
            const float* __restrict__ myWeights,  
            const unsigned int* __restrict__ myAnchorSequenceData, 
            int subjectIndex,
            int encodedSequencePitchInInts, 
            size_t msaColumnPitchInElements){

        const int subjectColumnsBegin_incl = myMsaColumnProperties->subjectColumnsBegin_incl;
        const int subjectColumnsEnd_excl = myMsaColumnProperties->subjectColumnsEnd_excl;
        const int firstColumn_incl = myMsaColumnProperties->firstColumn_incl;
        const int lastColumn_excl = myMsaColumnProperties->lastColumn_excl;

        if(lastColumn_excl > msaColumnPitchInElements){
            if(threadIdx.x == 0){
                printf("%d, %d %lu\n", subjectIndex, lastColumn_excl, msaColumnPitchInElements);
            }
            __syncthreads();
        }
        assert(lastColumn_excl <= msaColumnPitchInElements);

        const int subjectLength = subjectColumnsEnd_excl - subjectColumnsBegin_incl;
        const unsigned int* const subject = myAnchorSequenceData;

        //set columns to zero which are not in range firstColumn_incl <= column && column < lastColumn_excl

        for(int column = threadIdx.x; 
                column < firstColumn_incl; 
                column += BLOCKSIZE){

            my_support[column] = 0;
            my_orig_weights[column] = 0;
            my_orig_coverage[column] = 0;
        }

        const int leftoverRight = msaColumnPitchInElements - lastColumn_excl;

        for(int i = threadIdx.x; i < leftoverRight; i += BLOCKSIZE){
            const int column = lastColumn_excl + i;

            my_support[column] = 0;
            my_orig_weights[column] = 0;
            my_orig_coverage[column] = 0;
        }

        for(int column = threadIdx.x; 
            column < firstColumn_incl; 
            column += BLOCKSIZE){
                
            my_consensus[column] = 5;
        }

        for(int i = threadIdx.x; i < leftoverRight; i += BLOCKSIZE){
            const int column = lastColumn_excl + i;

            my_consensus[column] = 5;
        }

        const int* const myCountsA = myCounts + 0 * msaColumnPitchInElements;
        const int* const myCountsC = myCounts + 1 * msaColumnPitchInElements;
        const int* const myCountsG = myCounts + 2 * msaColumnPitchInElements;
        const int* const myCountsT = myCounts + 3 * msaColumnPitchInElements;

        const float* const my_weightsA = myWeights + 0 * msaColumnPitchInElements;
        const float* const my_weightsC = myWeights + 1 * msaColumnPitchInElements;
        const float* const my_weightsG = myWeights + 2 * msaColumnPitchInElements;
        const float* const my_weightsT = myWeights + 3 * msaColumnPitchInElements;
#if 1
        const int numOuterIters = SDIV(lastColumn_excl, 4);

        // auto getEncodedNucFromInt2Bit = [](unsigned int data, int pos){
        //     return ((data >> (30 - 2*pos)) & 0x00000003);
        // };

#if 1
        for(int outerIter = threadIdx.x; outerIter < numOuterIters; outerIter += BLOCKSIZE){

            // const int pos1 = (outerIter * 4) - subjectColumnsBegin_incl);
            // const int pos2 = (outerIter * 4 + 3) - subjectColumnsBegin_incl);
            // const int intIndex1InEncodedSequence = (max(0, pos1/ 16);
            // const int intIndex2InEncodedSequence = (max(0, pos2/ 16);
            // const unsigned int encoded1 = subject[intIndex1InEncodedSequence];
            // const unsigned int encoded2 = subject[intIndex2InEncodedSequence];
            // const unsigned int encodedBases16 = encoded1;

            alignas(4) char consensusArray[4];

            #pragma unroll 
            for(int i = 0; i < 4; i++){
                const int column = outerIter * 4 + i;

                if(firstColumn_incl <= column && column < lastColumn_excl){

                    const int ca = myCountsA[column];
                    const int cc = myCountsC[column];
                    const int cg = myCountsG[column];
                    const int ct = myCountsT[column];
                    const float wa = my_weightsA[column];
                    const float wc = my_weightsC[column];
                    const float wg = my_weightsG[column];
                    const float wt = my_weightsT[column];

                    char cons = 5;
                    float consWeight = 0.0f;
                    if(wa > consWeight){
                        cons = 0;
                        consWeight = wa;
                    }
                    if(wc > consWeight){
                        cons = 1;
                        consWeight = wc;
                    }
                    if(wg > consWeight){
                        cons = 2;
                        consWeight = wg;
                    }
                    if(wt > consWeight){
                        cons = 3;
                        consWeight = wt;
                    }

                    consensusArray[i] = cons;

                    const float columnWeight = wa + wc + wg + wt;
                    if(columnWeight == 0){
                        printf("s %d c %d\n", subjectIndex, column);
                        assert(columnWeight != 0);
                    }
                    //assert(weightPerCountSum != 0);
                    my_support[column] = consWeight / columnWeight;
                    //my_support[column] = consWeightPerCount / weightPerCountSum;


                    if(subjectColumnsBegin_incl <= column && column < subjectColumnsEnd_excl){
                        constexpr unsigned int A_enc = 0x00;
                        constexpr unsigned int C_enc = 0x01;
                        constexpr unsigned int G_enc = 0x02;
                        constexpr unsigned int T_enc = 0x03;

                        const int localIndex = column - subjectColumnsBegin_incl;
                        const unsigned int encNuc = getEncodedNuc2Bit(subject, subjectLength, localIndex);

                        // const unsigned int encNuc2 = getEncodedNucFromInt2Bit(encodedBases16, localIndex % 16);

                        // if(intIndexInEncodedSequence != localIndex / 16){
                        //     printf("outerIter %d, i %d, intIndexInEncodedSequence %d, localIndex %d, column %d, "
                        //         "subjectColumnsBegin_incl %d\n", 
                        //         outerIter, i, intIndexInEncodedSequence, localIndex, column, subjectColumnsBegin_incl);
                        // }
                        // assert(intIndexInEncodedSequence == localIndex / 16);

                        // assert(encNuc == encNuc2);

                        if(encNuc == A_enc){
                            my_orig_weights[column] = wa;
                            my_orig_coverage[column] = ca;
                        }else if(encNuc == C_enc){
                            my_orig_weights[column] = wc;
                            my_orig_coverage[column] = cc;
                        }else if(encNuc == G_enc){
                            my_orig_weights[column] = wg;
                            my_orig_coverage[column] = cg;
                        }else if(encNuc == T_enc){
                            my_orig_weights[column] = wt;
                            my_orig_coverage[column] = ct;
                        }
                    }
                }
            }

            *((char4*)(my_consensus + 4*outerIter)) = *((const char4*)(&consensusArray[0]));
        }

#else        
        for(int outerIter = threadIdx.x; outerIter < numOuterIters; outerIter += BLOCKSIZE){

            int regCountsA[4];
            int regCountsC[4];
            int regCountsG[4];
            int regCountsT[4];
            float regWeightsA[4];
            float regWeightsC[4];
            float regWeightsG[4];
            float regWeightsT[4];

            *((int4*)&regCountsA[0]) = *((const int4*)(myCountsA + 4 * outerIter));
            *((int4*)&regCountsC[0]) = *((const int4*)(myCountsC + 4 * outerIter));
            *((int4*)&regCountsG[0]) = *((const int4*)(myCountsG + 4 * outerIter));
            *((int4*)&regCountsT[0]) = *((const int4*)(myCountsT + 4 * outerIter));
            *((float4*)&regWeightsA[0]) = *((const float4*)(my_weightsA + 4 * outerIter));
            *((float4*)&regWeightsC[0]) = *((const float4*)(my_weightsC + 4 * outerIter));
            *((float4*)&regWeightsG[0]) = *((const float4*)(my_weightsG + 4 * outerIter));
            *((float4*)&regWeightsT[0]) = *((const float4*)(my_weightsT + 4 * outerIter));



            #pragma unroll 
            for(int i = 0; i < 4; i++){
                const int column = outerIter * 4 + i;

                if(firstColumn_incl <= column && column < lastColumn_excl){

                    // const int ca = myCountsA[column];
                    // const int cc = myCountsC[column];
                    // const int cg = myCountsG[column];
                    // const int ct = myCountsT[column];
                    // const float wa = my_weightsA[column];
                    // const float wc = my_weightsC[column];
                    // const float wg = my_weightsG[column];
                    // const float wt = my_weightsT[column];

                    const int ca = regCountsA[i];
                    const int cc = regCountsC[i];
                    const int cg = regCountsG[i];
                    const int ct = regCountsT[i];
                    const float wa = regWeightsA[i];
                    const float wc = regWeightsC[i];
                    const float wg = regWeightsG[i];
                    const float wt = regWeightsT[i];

                    char cons = 5;
                    float consWeight = 0.0f;
                    if(wa > consWeight){
                        cons = 0;
                        consWeight = wa;
                    }
                    if(wc > consWeight){
                        cons = 1;
                        consWeight = wc;
                    }
                    if(wg > consWeight){
                        cons = 2;
                        consWeight = wg;
                    }
                    if(wt > consWeight){
                        cons = 3;
                        consWeight = wt;
                    }
                    my_consensus[column] = cons;
                    const float columnWeight = wa + wc + wg + wt;
                    if(columnWeight == 0){
                        printf("s %d c %d\n", subjectIndex, column);
                        assert(columnWeight != 0);
                    }
                    //assert(weightPerCountSum != 0);
                    my_support[column] = consWeight / columnWeight;
                    //my_support[column] = consWeightPerCount / weightPerCountSum;


                    if(subjectColumnsBegin_incl <= column && column < subjectColumnsEnd_excl){
                        constexpr unsigned int A_enc = 0x00;
                        constexpr unsigned int C_enc = 0x01;
                        constexpr unsigned int G_enc = 0x02;
                        constexpr unsigned int T_enc = 0x03;

                        const int localIndex = column - subjectColumnsBegin_incl;
                        const unsigned int encNuc = getEncodedNuc2Bit(subject, subjectLength, localIndex);

                        if(encNuc == A_enc){
                            my_orig_weights[column] = wa;
                            my_orig_coverage[column] = ca;
                        }else if(encNuc == C_enc){
                            my_orig_weights[column] = wc;
                            my_orig_coverage[column] = cc;
                        }else if(encNuc == G_enc){
                            my_orig_weights[column] = wg;
                            my_orig_coverage[column] = cg;
                        }else if(encNuc == T_enc){
                            my_orig_weights[column] = wt;
                            my_orig_coverage[column] = ct;
                        }
                    }
                }
            }
        }

#endif

#else 

        const int numColumnsToCheck = lastColumn_excl - firstColumn_incl;

        for(int i = threadIdx.x; i < numColumnsToCheck; i += BLOCKSIZE){
            const int column = firstColumn_incl + i;

            const int ca = myCountsA[column];
            const int cc = myCountsC[column];
            const int cg = myCountsG[column];
            const int ct = myCountsT[column];
            const float wa = my_weightsA[column];
            const float wc = my_weightsC[column];
            const float wg = my_weightsG[column];
            const float wt = my_weightsT[column];

            char cons = 5;
            float consWeight = 0.0f;
            if(wa > consWeight){
                cons = 0;
                consWeight = wa;
            }
            if(wc > consWeight){
                cons = 1;
                consWeight = wc;
            }
            if(wg > consWeight){
                cons = 2;
                consWeight = wg;
            }
            if(wt > consWeight){
                cons = 3;
                consWeight = wt;
            }
            my_consensus[column] = cons;
            const float columnWeight = wa + wc + wg + wt;
            if(columnWeight == 0){
                printf("s %d c %d\n", subjectIndex, column);
                assert(columnWeight != 0);
            }
            //assert(weightPerCountSum != 0);
            my_support[column] = consWeight / columnWeight;
            //my_support[column] = consWeightPerCount / weightPerCountSum;


            if(subjectColumnsBegin_incl <= column && column < subjectColumnsEnd_excl){
                constexpr unsigned int A_enc = 0x00;
                constexpr unsigned int C_enc = 0x01;
                constexpr unsigned int G_enc = 0x02;
                constexpr unsigned int T_enc = 0x03;

                const int localIndex = column - subjectColumnsBegin_incl;
                const unsigned int encNuc = getEncodedNuc2Bit(subject, subjectLength, localIndex);

                if(encNuc == A_enc){
                    my_orig_weights[column] = wa;
                    my_orig_coverage[column] = ca;
                }else if(encNuc == C_enc){
                    my_orig_weights[column] = wc;
                    my_orig_coverage[column] = cc;
                }else if(encNuc == G_enc){
                    my_orig_weights[column] = wg;
                    my_orig_coverage[column] = cg;
                }else if(encNuc == T_enc){
                    my_orig_weights[column] = wt;
                    my_orig_coverage[column] = ct;
                }
            }
        }

#endif        
    }



    template<int BLOCKSIZE>
    __device__ __forceinline__
    void findCandidatesOfDifferentRegionSingleBlock(
            int2* smem,
            int* __restrict__ myNewIndicesPtr,
            int* __restrict__ myNewNumIndicesPerSubjectPtr,
            const MSAColumnProperties* __restrict__ myMsaColumnProperties,
            const char* __restrict__ myConsensus,
            const int* __restrict__ myCounts,
            const float* __restrict__ myWeights,
            const unsigned int* __restrict__ myAnchorSequenceData,
            const int subjectLength,
            const unsigned int* __restrict__ myCandidateSequencesData,
            const int* __restrict__ myCandidateLengths,
            const BestAlignment_t* myAlignmentFlags,
            const int* __restrict__ myShifts,
            const int* __restrict__ myNops,
            const int* __restrict__ myOverlaps,
            bool* __restrict__ myShouldBeKept,
            float desiredAlignmentMaxErrorRate,
            int subjectIndex,
            int encodedSequencePitchInInts,
            size_t msaColumnPitchInElements,
            const int* __restrict__ myIndices,
            const int myNumIndices,
            int dataset_coverage){

        auto is_significant_count = [](int count, int coverage){
            if(int(coverage * 0.3f) <= count)
                return true;
            return false;
        };        

        // auto to_nuc = [](unsigned int c){
        //     constexpr unsigned int A_enc = 0x00;
        //     constexpr unsigned int C_enc = 0x01;
        //     constexpr unsigned int G_enc = 0x02;
        //     constexpr unsigned int T_enc = 0x03;

        //     switch(c){
        //     case A_enc: return 'A';
        //     case C_enc: return 'C';
        //     case G_enc: return 'G';
        //     case T_enc: return 'T';
        //     default: return 'F';
        //     }
        // };

        using BlockReduceBool = hipcub::BlockReduce<bool, BLOCKSIZE>;
        using BlockReduceInt2 = hipcub::BlockReduce<int2, BLOCKSIZE>;

        typename BlockReduceBool::TempStorage* temp_storage_boolreduce = (typename BlockReduceBool::TempStorage*) smem;
        typename BlockReduceInt2::TempStorage* temp_storage_int2reduce = (typename BlockReduceInt2::TempStorage*) smem;

        __shared__ bool broadcastbufferbool;
        __shared__ int broadcastbufferint4[4];
        __shared__ int counts[1];

        const unsigned int* const subjectptr = myAnchorSequenceData;

        const int subjectColumnsBegin_incl = myMsaColumnProperties->subjectColumnsBegin_incl;
        const int subjectColumnsEnd_excl = myMsaColumnProperties->subjectColumnsEnd_excl;

        //check if subject and consensus differ at at least one position

        bool hasMismatchToConsensus = false;

        for(int pos = threadIdx.x; pos < subjectLength && !hasMismatchToConsensus; pos += BLOCKSIZE){
            const int column = subjectColumnsBegin_incl + pos;
            const char consbase = myConsensus[column];
            const char subjectbase = getEncodedNuc2Bit(subjectptr, subjectLength, pos);

            hasMismatchToConsensus |= (consbase != subjectbase);
        }

        hasMismatchToConsensus = 
            BlockReduceBool(*temp_storage_boolreduce)
                .Reduce(hasMismatchToConsensus, [](auto l, auto r){return l || r;});

        if(threadIdx.x == 0){
            broadcastbufferbool = hasMismatchToConsensus;
        }
        __syncthreads();

        hasMismatchToConsensus = broadcastbufferbool;

        //if subject and consensus differ at at least one position, check columns in msa

        if(hasMismatchToConsensus){
            int col = std::numeric_limits<int>::max();
            bool foundColumn = false;
            char foundBase = 'F';
            int foundBaseIndex = std::numeric_limits<int>::max();
            int consindex = std::numeric_limits<int>::max();

            const int* const myCountsA = myCounts + 0 * msaColumnPitchInElements;
            const int* const myCountsC = myCounts + 1 * msaColumnPitchInElements;
            const int* const myCountsG = myCounts + 2 * msaColumnPitchInElements;
            const int* const myCountsT = myCounts + 3 * msaColumnPitchInElements;

            for(int columnindex = subjectColumnsBegin_incl + threadIdx.x; 
                    columnindex < subjectColumnsEnd_excl && !foundColumn; 
                    columnindex += BLOCKSIZE){

                int counts[4];
                counts[0] = myCountsA[columnindex];
                counts[1] = myCountsC[columnindex];
                counts[2] = myCountsG[columnindex];
                counts[3] = myCountsT[columnindex];

                const char consbase = myConsensus[columnindex];
                consindex = consbase;

                assert(0 <= consindex && consindex < 4);

                //find out if there is a non-consensus base with significant coverage
                int significantBaseIndex = -1;

                #pragma unroll
                for(int i = 0; i < 4; i++){
                    if(i != consindex){
                        const bool significant = is_significant_count(counts[i], dataset_coverage);

                        significantBaseIndex = significant ? i : significantBaseIndex;
                    }
                }

                if(significantBaseIndex != -1){
                    foundColumn = true;
                    col = columnindex;
                    foundBaseIndex = significantBaseIndex;
                }
            }

            int2 packed{col, foundBaseIndex};
            //find packed value with smallest col
            packed = BlockReduceInt2(*temp_storage_int2reduce).Reduce(packed, [](auto l, auto r){
                if(l.x < r.x){
                    return l;
                }else{
                    return r;
                }
            });

            if(threadIdx.x == 0){
                if(packed.x != std::numeric_limits<int>::max()){
                    broadcastbufferint4[0] = 1;
                    broadcastbufferint4[1] = packed.x;
                    broadcastbufferint4[2] = packed.y;
                    broadcastbufferint4[3] = packed.y;
                }else{
                    broadcastbufferint4[0] = 0;
                }
            }

            __syncthreads();

            foundColumn = (1 == broadcastbufferint4[0]);
            col = broadcastbufferint4[1];
            foundBase = broadcastbufferint4[2];
            foundBaseIndex = broadcastbufferint4[3];

            if(foundColumn){
                
                auto discard_rows = [&](bool keepMatching){
                    
                    for(int k = threadIdx.x; k < myNumIndices; k += BLOCKSIZE){
                        const int localCandidateIndex = myIndices[k];
                        const unsigned int* const candidateptr = myCandidateSequencesData + std::size_t(localCandidateIndex) * encodedSequencePitchInInts;
                        const int candidateLength = myCandidateLengths[localCandidateIndex];
                        const int shift = myShifts[localCandidateIndex];
                        const BestAlignment_t alignmentFlag = myAlignmentFlags[localCandidateIndex];

                        //check if row is affected by column col
                        const int row_begin_incl = subjectColumnsBegin_incl + shift;
                        const int row_end_excl = row_begin_incl + candidateLength;
                        const bool notAffected = (col < row_begin_incl || row_end_excl <= col);
                        char base = 5;
                        if(!notAffected){
                            if(alignmentFlag == BestAlignment_t::Forward){
                                base = getEncodedNuc2Bit(candidateptr, candidateLength, (col - row_begin_incl));
                            }else{
                                //all candidates of MSA must not have alignmentflag None
                                assert(alignmentFlag == BestAlignment_t::ReverseComplement); 

                                const unsigned int forwardbaseEncoded = getEncodedNuc2Bit(candidateptr, candidateLength, row_end_excl-1 - col);
                                base = (~forwardbaseEncoded & 0x03);
                            }
                        }

                        if(notAffected || (!(keepMatching ^ (base == foundBase)))){
                            myShouldBeKept[k] = true; //same region
                        }else{
                            myShouldBeKept[k] = false; //different region
                        }
                    }
                    #if 1
                    //check that no candidate which should be removed has very good alignment.
                    //if there is such a candidate, none of the candidates will be removed.
                    bool veryGoodAlignment = false;
                    for(int k = threadIdx.x; k < myNumIndices && !veryGoodAlignment; k += BLOCKSIZE){
                        if(!myShouldBeKept[+ k]){
                            const int localCandidateIndex = myIndices[k];
                            const int nOps = myNops[localCandidateIndex];
                            const int overlapsize = myOverlaps[localCandidateIndex];
                            const float overlapweight = calculateOverlapWeight(
                                subjectLength, 
                                nOps, 
                                overlapsize,
                                desiredAlignmentMaxErrorRate
                            );
                            assert(overlapweight <= 1.0f);
                            assert(overlapweight >= 0.0f);

                            if(fgeq(overlapweight, 0.90f)){
                                veryGoodAlignment = true;
                            }
                        }
                    }

                    veryGoodAlignment = BlockReduceBool(*temp_storage_boolreduce)
                        .Reduce(veryGoodAlignment, [](auto l, auto r){return l || r;});

                    if(threadIdx.x == 0){
                        broadcastbufferbool = veryGoodAlignment;
                    }
                    __syncthreads();

                    veryGoodAlignment = broadcastbufferbool;

                    if(veryGoodAlignment){
                        for(int k = threadIdx.x; k < myNumIndices; k += blockDim.x){
                            myShouldBeKept[k] = true;
                        }
                    }
                    #endif

                    //select indices of candidates to keep and write them to new indices
                    if(threadIdx.x == 0){
                        counts[0] = 0;
                    }
                    __syncthreads();

                    const int limit = SDIV(myNumIndices, BLOCKSIZE) * BLOCKSIZE;
                    for(int k = threadIdx.x; k < limit; k += BLOCKSIZE){
                        bool keep = false;
                        if(k < myNumIndices){
                            keep = myShouldBeKept[k];
                        }                               
            
                        if(keep){
                            cg::coalesced_group g = cg::coalesced_threads();
                            int outputPos;
                            if (g.thread_rank() == 0) {
                                outputPos = atomicAdd(&counts[0], g.size());
                            }
                            outputPos = g.thread_rank() + g.shfl(outputPos, 0);
                            myNewIndicesPtr[outputPos] = myIndices[k];
                        }                        
                    }

                    __syncthreads();

                    if(threadIdx.x == 0){
                        *myNewNumIndicesPerSubjectPtr = counts[0];
                    }

                    __syncthreads();

                };

                //compare found base to original base
                const char originalbase = getEncodedNuc2Bit(subjectptr, subjectLength, col - subjectColumnsBegin_incl);

                if(originalbase == foundBase){
                    //discard all candidates whose base in column col differs from foundBase
                    discard_rows(true);
                }else{
                    //discard all candidates whose base in column col matches foundBase
                    discard_rows(false);
                }

            }else{
                //did not find a significant columns

                //remove no candidate
                for(int k = threadIdx.x; k < myNumIndices; k += blockDim.x){
                    myNewIndicesPtr[k] = myIndices[k];
                }
                if(threadIdx.x == 0){
                    *myNewNumIndicesPerSubjectPtr = myNumIndices;
                }
            }

        }else{
            //no mismatch between consensus and subject

            //remove no candidate
            for(int k = threadIdx.x; k < myNumIndices; k += blockDim.x){
                myShouldBeKept[k] = true;
            }

            for(int k = threadIdx.x; k < myNumIndices; k += blockDim.x){
                myNewIndicesPtr[k] = myIndices[k];
            }
            if(threadIdx.x == 0){
                *myNewNumIndicesPerSubjectPtr = myNumIndices;
            }
        }
    }

  


    #ifdef __CUDACC_DEBUG__

        #define buildMSA3Kernel_MIN_BLOCKS   1
        
    #else

        #if __CUDA_ARCH__ >= 610
            #define buildMSA3Kernel_MIN_BLOCKS   8
        #else
            #define buildMSA3Kernel_MIN_BLOCKS   4
        #endif

    #endif



    template<int BLOCKSIZE, MemoryType addSequencesMemType>
    __launch_bounds__(BLOCKSIZE, buildMSA3Kernel_MIN_BLOCKS)
    __global__
    void buildMSA3Kernel(
            MSAColumnProperties* __restrict__ msaColumnProperties,
            int* __restrict__ coverage,
            int* __restrict__ counts,
            float* __restrict__ weights,
            float* __restrict__ d_support,
            float* __restrict__ d_origWeights,
            int* __restrict__ d_origCoverages,
            char* __restrict__ d_consensus,          
            const int* __restrict__ overlaps,
            const int* __restrict__ shifts,
            const int* __restrict__ nOps,
            const BestAlignment_t* __restrict__ bestAlignmentFlags,
            const int* __restrict__ anchorLengths,
            const int* __restrict__ candidateLengths,
            const int* __restrict__ indices,
            const int* __restrict__ indices_per_subject,
            const int* __restrict__ candidatesPerSubjectPrefixSum,            
            const unsigned int* __restrict__ subjectSequencesData,
            const unsigned int* __restrict__ candidateSequencesData,
            const char* __restrict__ subjectQualities,
            const char* __restrict__ candidateQualities,
            const int* __restrict__ d_numAnchors,
            const int*,
            float desiredAlignmentMaxErrorRate,
            bool canUseQualityScores,
            int encodedSequencePitchInInts,
            size_t qualityPitchInBytes,
            size_t msaColumnPitchInElements,
            const bool* __restrict__ canExecute){

        constexpr bool useSmemForAddSequences = (addSequencesMemType == MemoryType::Shared);

        extern __shared__ float sharedmem[];
        __shared__ MSAColumnProperties shared_columnProperties;

        using BlockReduceInt = hipcub::BlockReduce<int, BLOCKSIZE>;            

        if(*canExecute){

            auto tbGroup = cg::this_thread_block();

            const int n_subjects = *d_numAnchors;

            typename BlockReduceInt::TempStorage* const cubTempStorage = (typename BlockReduceInt::TempStorage*)sharedmem;

            float* const shared_weights = sharedmem;
            int* const shared_counts = (int*)(shared_weights + 4 * msaColumnPitchInElements);
            int* const shared_coverages = (int*)(shared_counts + 4 * msaColumnPitchInElements);

            GPUMultiMSA multiMSA;

            multiMSA.numMSAs = n_subjects;
            multiMSA.columnPitchInElements = msaColumnPitchInElements;
            multiMSA.counts = counts;
            multiMSA.weights = weights;
            multiMSA.coverages = coverage;
            multiMSA.consensus = d_consensus;
            multiMSA.support = d_support;
            multiMSA.origWeights = d_origWeights;
            multiMSA.origCoverages = d_origCoverages;
            multiMSA.columnProperties = msaColumnProperties;

            for(int subjectIndex = blockIdx.x; subjectIndex < n_subjects; subjectIndex += gridDim.x){
                const int myNumGoodCandidates = indices_per_subject[subjectIndex];

                if(myNumGoodCandidates > 0){

                    tbGroup.sync(); //wait for smem of previous iteration

                    GpuSingleMSA msa = multiMSA.getSingleMSA(subjectIndex);

                    if(useSmemForAddSequences){
                        msa.counts = shared_counts;
                        msa.weights = shared_weights;
                        msa.coverages = shared_coverages;
                    }

                    const int globalCandidateOffset = candidatesPerSubjectPrefixSum[subjectIndex];

                    const int* const myOverlaps = overlaps + globalCandidateOffset;
                    const int* const myShifts = shifts + globalCandidateOffset;
                    const int* const myNops = nOps + globalCandidateOffset;
                    const BestAlignment_t* const myAlignmentFlags = bestAlignmentFlags + globalCandidateOffset;
                    const int subjectLength = anchorLengths[subjectIndex];
                    const int* const myCandidateLengths = candidateLengths + globalCandidateOffset;
                    const int* const myIndices = indices + globalCandidateOffset;

                    const unsigned int* const myAnchorSequenceData = subjectSequencesData + std::size_t(subjectIndex) * encodedSequencePitchInInts;
                    const unsigned int* const myCandidateSequencesData = candidateSequencesData + size_t(globalCandidateOffset) * encodedSequencePitchInInts;
                    const char* const myAnchorQualityData = subjectQualities + std::size_t(subjectIndex) * qualityPitchInBytes;
                    const char* const myCandidateQualities = candidateQualities + size_t(globalCandidateOffset) * qualityPitchInBytes;

                    MSAColumnProperties columnProperties;

                    msa.columnProperties = &columnProperties;

                    auto groupReduceIntMin = [&](int data){
                        data = BlockReduceInt(*cubTempStorage).Reduce(data, hipcub::Min());
                        tbGroup.sync();
                        return data;
                    };

                    auto groupReduceIntMax = [&](int data){                        
                        data = BlockReduceInt(*cubTempStorage).Reduce(data, hipcub::Max());
                        tbGroup.sync();
                        return data;
                    };

                    msa.initColumnProperties(
                        tbGroup,
                        groupReduceIntMin,
                        groupReduceIntMax,
                        myIndices,
                        myNumGoodCandidates,
                        myShifts,
                        myAlignmentFlags,
                        subjectLength,
                        myCandidateLengths
                    );

                    //only thread 0 has valid column properties. 
                    //save to global memory and broadcast to all threads in block
                    if(tbGroup.thread_rank() == 0){
                        auto* globalDest = multiMSA.getColumnPropertiesOfMSA(subjectIndex);
                        *globalDest = columnProperties;
                        shared_columnProperties = columnProperties;
                    }

                    tbGroup.sync();

                    columnProperties = shared_columnProperties;

                    const int columnsToCheck = columnProperties.lastColumn_excl;

                    assert(columnsToCheck <= msaColumnPitchInElements);

                    msa.constructFromSequences(
                        tbGroup,
                        myShifts,
                        myOverlaps,
                        myNops,
                        myAlignmentFlags,
                        myAnchorSequenceData,
                        myAnchorQualityData,
                        myCandidateSequencesData,
                        myCandidateQualities,
                        myCandidateLengths,
                        myIndices,
                        myNumGoodCandidates,
                        canUseQualityScores, 
                        encodedSequencePitchInInts,
                        qualityPitchInBytes,
                        desiredAlignmentMaxErrorRate,
                        subjectIndex
                    );

                    tbGroup.sync();
            
                    msa.checkAfterBuild(tbGroup, subjectIndex);

                    msa.findConsensus(
                        tbGroup,
                        myAnchorSequenceData, 
                        encodedSequencePitchInInts, 
                        subjectIndex
                    );

                    if(useSmemForAddSequences){
                        // copy from counts and weights and coverages from shared to global
                        int* const gmemCounts = multiMSA.getCountsOfMSA(subjectIndex);
                        float* const gmemWeights = multiMSA.getWeightsOfMSA(subjectIndex);
                        int* const gmemCoverages = multiMSA.getCoveragesOfMSA(subjectIndex);

                        for(int index = tbGroup.thread_rank(); index < columnsToCheck; index += tbGroup.size()){
                            for(int k = 0; k < 4; k++){
                                const int* const srcCounts = msa.counts + k * msaColumnPitchInElements + index;
                                int* const destCounts = gmemCounts + k * msaColumnPitchInElements + index;
            
                                const float* const srcWeights = msa.weights + k * msaColumnPitchInElements + index;
                                float* const destWeights = gmemWeights + k * msaColumnPitchInElements + index;
            
                                *destCounts = *srcCounts;
                                *destWeights = *srcWeights;
                            }
                            gmemCoverages[index] = msa.coverages[index];
                        }
                    }
                } 
            }
        }

    }





    template<int BLOCKSIZE, MemoryType addSequencesMemType>
    __global__
    void msa_findCandidatesOfDifferentRegionAndRemoveThemViaRebuild_kernel(
            int* __restrict__ d_newIndices,
            int* __restrict__ d_newNumIndicesPerSubject,
            int* __restrict__ d_newNumIndices,
            MSAColumnProperties* __restrict__ msaColumnProperties,
            char* __restrict__ consensus,
            int* __restrict__ coverage,
            int* __restrict__ counts,
            float* __restrict__ weights,
            float* __restrict__ support,
            int* __restrict__ origCoverages,
            float* __restrict__ origWeights,
            const BestAlignment_t* __restrict__ bestAlignmentFlags,
            const int* __restrict__ shifts,
            const int* __restrict__ nOps,
            const int* __restrict__ overlaps,
            const unsigned int* __restrict__ subjectSequencesData,
            const unsigned int* __restrict__ candidateSequencesData,
            const unsigned int* __restrict__ transposedCandidateSequencesData,
            const int* __restrict__ subjectSequencesLength,
            const int* __restrict__ candidateSequencesLength,
            const char* __restrict__ subjectQualities,
            const char* __restrict__ candidateQualities,
            bool* __restrict__ d_shouldBeKept,
            const int* __restrict__ d_candidates_per_subject_prefixsum,
            const int* __restrict__ d_numAnchors,
            const int*,
            float desiredAlignmentMaxErrorRate,
            bool canUseQualityScores,
            size_t encodedSequencePitchInInts,
            size_t qualityPitchInBytes,
            size_t msaColumnPitchInElements,
            const int* __restrict__ d_indices,
            const int* __restrict__ d_indices_per_subject,
            int dataset_coverage,
            const bool* __restrict__ canExecute,
            int iteration,
            bool* __restrict__ d_anchorIsFinished){

        if(*canExecute){

            using BlockReduceBool = hipcub::BlockReduce<bool, BLOCKSIZE>;
            using BlockReduceInt2 = hipcub::BlockReduce<int2, BLOCKSIZE>;
            using BlockReduceInt = hipcub::BlockReduce<int, BLOCKSIZE>;

            __shared__ union{
                typename BlockReduceBool::TempStorage boolreduce;
                typename BlockReduceInt2::TempStorage int2reduce;
                typename BlockReduceInt::TempStorage intreduce;
            } temp_storage;

            __shared__ MSAColumnProperties shared_columnProperties;

            extern __shared__ float externsharedmem[];

            constexpr bool useSmemMSA = (addSequencesMemType == MemoryType::Shared);

            float* const shared_weights = externsharedmem;
            int* const shared_counts = (int*)(shared_weights + 4 * msaColumnPitchInElements);
            int* const shared_coverages = (int*)(shared_counts + 4 * msaColumnPitchInElements);

            const int n_subjects = *d_numAnchors;

            GPUMultiMSA multiMSA;

            multiMSA.numMSAs = n_subjects;
            multiMSA.columnPitchInElements = msaColumnPitchInElements;
            multiMSA.counts = counts;
            multiMSA.weights = weights;
            multiMSA.coverages = coverage;
            multiMSA.consensus = consensus;
            multiMSA.support = support;
            multiMSA.origWeights = origWeights;
            multiMSA.origCoverages = origCoverages;
            multiMSA.columnProperties = msaColumnProperties;

            auto tbGroup = cg::this_thread_block();

            for(int subjectIndex = blockIdx.x; subjectIndex < n_subjects; subjectIndex += gridDim.x){
                const bool myAnchorIsFinished = d_anchorIsFinished[subjectIndex];
                const int myNumIndices = d_indices_per_subject[subjectIndex];

                if(myAnchorIsFinished){
                    if(threadIdx.x == 0){
                        atomicAdd(d_newNumIndices, myNumIndices);
                    }
                }else{               

                    if(myNumIndices > 0){
                        GpuSingleMSA msa = multiMSA.getSingleMSA(subjectIndex);
                        msa.columnProperties = &shared_columnProperties;

                        tbGroup.sync(); //wait for previous iteration

                        if(threadIdx.x == 0){
                            shared_columnProperties = msaColumnProperties[subjectIndex];
                        }
                        tbGroup.sync();

                        const int globalOffset = d_candidates_per_subject_prefixsum[subjectIndex];
                        const int* const myIndices = d_indices + globalOffset;
                        const int* const myNumIndicesPerSubjectPtr = d_indices_per_subject + subjectIndex;

                        int* const myNewIndicesPtr = d_newIndices + globalOffset;
                        int* const myNewNumIndicesPerSubjectPtr = d_newNumIndicesPerSubject + subjectIndex;

                        bool* const myShouldBeKept = d_shouldBeKept + globalOffset;                    

                        const BestAlignment_t* const myAlignmentFlags = bestAlignmentFlags + globalOffset;
                        const int* const myShifts = shifts + globalOffset;
                        const int* const myNops = nOps + globalOffset;
                        const int* const myOverlaps = overlaps + globalOffset;

                        const unsigned int* const myAnchorSequenceData = subjectSequencesData 
                            + std::size_t(subjectIndex) * encodedSequencePitchInInts;
                        const unsigned int* const myCandidateSequencesData = candidateSequencesData 
                            + std::size_t(globalOffset) * encodedSequencePitchInInts;

                        const char* const myAnchorQualityData = subjectQualities + std::size_t(subjectIndex) * qualityPitchInBytes;
                        const char* const myCandidateQualities = candidateQualities 
                            + size_t(globalOffset) * qualityPitchInBytes;

                        const int subjectLength = subjectSequencesLength[subjectIndex];
                        const int* const myCandidateLengths = candidateSequencesLength + globalOffset;

                        const int* const srcIndices = myIndices;
                        int* const destIndices = myNewIndicesPtr;

                        const int* const srcNumIndices = myNumIndicesPerSubjectPtr;
                        int* const destNumIndices = myNewNumIndicesPerSubjectPtr;

                        auto groupReduceBool = [&](bool b, auto comp){
                            b = BlockReduceBool(temp_storage.boolreduce).Reduce(b, comp);
                            return b;
                        };

                        auto groupReduceInt2 = [&](int2 b, auto comp){
                            b = BlockReduceInt2(temp_storage.int2reduce).Reduce(b, comp);
                            return b;
                        };

                        msa.flagCandidatesOfDifferentRegion(
                            tbGroup,
                            groupReduceBool,
                            groupReduceInt2,
                            destIndices,
                            destNumIndices,
                            myAnchorSequenceData,
                            subjectLength,
                            myCandidateSequencesData,
                            myCandidateLengths,
                            myAlignmentFlags,
                            myShifts,
                            myNops,
                            myOverlaps,
                            myShouldBeKept,
                            desiredAlignmentMaxErrorRate,
                            subjectIndex,
                            encodedSequencePitchInInts,
                            srcIndices,
                            *srcNumIndices,
                            dataset_coverage
                        );

                        tbGroup.sync();

                        const int myNewNumIndices = *destNumIndices;

                        if(tbGroup.thread_rank()== 0){
                            atomicAdd(d_newNumIndices, myNewNumIndices);
                        }

                        assert(myNewNumIndices <= myNumIndices);
                        if(myNewNumIndices > 0 && myNewNumIndices < myNumIndices){

                            if(useSmemMSA){
                                msa.counts = shared_counts;
                                msa.weights = shared_weights;
                                msa.coverages = shared_coverages;
                            }

                            auto groupReduceIntMin = [&](int data){
                                data = BlockReduceInt(temp_storage.intreduce).Reduce(data, hipcub::Min());
                                tbGroup.sync();
                                return data;
                            };
        
                            auto groupReduceIntMax = [&](int data){                        
                                data = BlockReduceInt(temp_storage.intreduce).Reduce(data, hipcub::Max());
                                tbGroup.sync();
                                return data;
                            };
        
                            msa.initColumnProperties(
                                tbGroup,
                                groupReduceIntMin,
                                groupReduceIntMax,
                                destIndices,
                                *destNumIndices,
                                myShifts,
                                myAlignmentFlags,
                                subjectLength,
                                myCandidateLengths
                            );

                            //only thread 0 has valid column properties. save to global memory and broadcast to all threads in block
                            if(tbGroup.thread_rank() == 0){
                                msaColumnProperties[subjectIndex] = shared_columnProperties;
                            }

                            tbGroup.sync();

                            const int columnsToCheck = shared_columnProperties.lastColumn_excl;

                            assert(columnsToCheck <= msaColumnPitchInElements);

                            msa.constructFromSequences(
                                tbGroup,
                                myShifts,
                                myOverlaps,
                                myNops,
                                myAlignmentFlags,
                                myAnchorSequenceData,
                                myAnchorQualityData,
                                myCandidateSequencesData,
                                myCandidateQualities,
                                myCandidateLengths,
                                destIndices,
                                *destNumIndices,
                                canUseQualityScores, 
                                encodedSequencePitchInInts,
                                qualityPitchInBytes,
                                desiredAlignmentMaxErrorRate,
                                subjectIndex
                            );

                            tbGroup.sync();
                    
                            msa.checkAfterBuild(tbGroup, subjectIndex);

                            msa.findConsensus(
                                tbGroup,
                                myAnchorSequenceData, 
                                encodedSequencePitchInInts, 
                                subjectIndex
                            );

                            tbGroup.sync();

                            if(useSmemMSA){
                                // copy from counts and weights and coverages from shared to global
                                int* const gmemCounts = multiMSA.getCountsOfMSA(subjectIndex);
                                float* const gmemWeights = multiMSA.getWeightsOfMSA(subjectIndex);
                                int* const gmemCoverages = multiMSA.getCoveragesOfMSA(subjectIndex);
        
                                for(int index = tbGroup.thread_rank(); index < columnsToCheck; index += tbGroup.size()){
                                    for(int k = 0; k < 4; k++){
                                        const int* const srcCounts = msa.counts + k * msaColumnPitchInElements + index;
                                        int* const destCounts = gmemCounts + k * msaColumnPitchInElements + index;
                    
                                        const float* const srcWeights = msa.weights + k * msaColumnPitchInElements + index;
                                        float* const destWeights = gmemWeights + k * msaColumnPitchInElements + index;
                    
                                        *destCounts = *srcCounts;
                                        *destWeights = *srcWeights;
                                    }
                                    gmemCoverages[index] = msa.coverages[index];
                                }

                                tbGroup.sync();
                            }

                            
                    
                        }else{
                            if(threadIdx.x == 0){
                                d_anchorIsFinished[subjectIndex] = true;
                            }
                        }
                    }else{
                        if(threadIdx.x == 0){
                            d_newNumIndicesPerSubject[subjectIndex] = 0;
                            d_anchorIsFinished[subjectIndex] = true;
                        }
                        ; //nothing else to do if there are no candidates in msa
                    }
                }
            }
        }
    }



    #ifdef __CUDACC_DEBUG__

        #define findCandidatesOfDifferentRegionAndRemoveThemViaDeletion2_multiiter_MIN_BLOCKS   1
        
    #else

        #if __CUDA_ARCH__ >= 610
            #define findCandidatesOfDifferentRegionAndRemoveThemViaDeletion2_multiiter_MIN_BLOCKS   4
        #else
            #define findCandidatesOfDifferentRegionAndRemoveThemViaDeletion2_multiiter_MIN_BLOCKS   4
        #endif

    #endif


    template<int BLOCKSIZE, MemoryType memoryType>
    __launch_bounds__(BLOCKSIZE, findCandidatesOfDifferentRegionAndRemoveThemViaDeletion2_multiiter_MIN_BLOCKS)
    __global__
    void msa_findCandidatesOfDifferentRegionAndRemoveThemViaDeletion2_multiiteration_kernel(
            int* __restrict__ d_newIndices,
            int* __restrict__ d_newNumIndicesPerSubject,
            int* __restrict__ d_newNumIndices,
            MSAColumnProperties* __restrict__ msaColumnProperties,
            char* __restrict__ consensus,
            int* __restrict__ coverage,
            int* __restrict__ counts,
            float* __restrict__ weights,
            float* __restrict__ support,
            int* __restrict__ origCoverages,
            float* __restrict__ origWeights,
            const BestAlignment_t* __restrict__ bestAlignmentFlags,
            const int* __restrict__ shifts,
            const int* __restrict__ nOps,
            const int* __restrict__ overlaps,
            const unsigned int* __restrict__ subjectSequencesData,
            const unsigned int* __restrict__ candidateSequencesData,
            const unsigned int* __restrict__ transposedCandidateSequencesData,
            const int* __restrict__ subjectSequencesLength,
            const int* __restrict__ candidateSequencesLength,
            const char* __restrict__ subjectQualities,
            const char* __restrict__ candidateQualities,
            bool* __restrict__ d_shouldBeKept,
            const int* __restrict__ d_candidates_per_subject_prefixsum,
            const int* __restrict__ d_numAnchors,
            const int*,
            float desiredAlignmentMaxErrorRate,
            bool canUseQualityScores,
            size_t encodedSequencePitchInInts,
            size_t qualityPitchInBytes,
            size_t msaColumnPitchInElements,
            int* __restrict__ d_indices,
            int* __restrict__ d_indices_per_subject,
            int dataset_coverage,
            const bool* __restrict__ canExecute,
            int numRefinementIterations,
            bool*){

        constexpr bool useSmemMSA = (memoryType == MemoryType::Shared);

        using BlockReduceBool = hipcub::BlockReduce<bool, BLOCKSIZE>;
        using BlockReduceInt2 = hipcub::BlockReduce<int2, BLOCKSIZE>;

        extern __shared__ float externsharedmem[];
        __shared__ MSAColumnProperties shared_columnProperties;

        __shared__ union{
            typename BlockReduceBool::TempStorage boolreduce;
            typename BlockReduceInt2::TempStorage int2reduce;
        } temp_storage;      

        if(*canExecute){

            assert(numRefinementIterations > 0);

            auto tbGroup = cg::this_thread_block();

            float* const shared_weights = externsharedmem;
            int* const shared_counts = (int*)(shared_weights + 4 * msaColumnPitchInElements);
            int* const shared_coverages = (int*)(shared_counts + 4 * msaColumnPitchInElements);            

            const int n_subjects = *d_numAnchors;

            GPUMultiMSA multiMSA;

            multiMSA.numMSAs = n_subjects;
            multiMSA.columnPitchInElements = msaColumnPitchInElements;
            multiMSA.counts = counts;
            multiMSA.weights = weights;
            multiMSA.coverages = coverage;
            multiMSA.consensus = consensus;
            multiMSA.support = support;
            multiMSA.origWeights = origWeights;
            multiMSA.origCoverages = origCoverages;
            multiMSA.columnProperties = msaColumnProperties;

            for(int subjectIndex = blockIdx.x; subjectIndex < n_subjects; subjectIndex += gridDim.x){
                int myNumIndices = d_indices_per_subject[subjectIndex];                            

                if(myNumIndices > 0){

                    tbGroup.sync();

                    if(threadIdx.x == 0){
                        shared_columnProperties = msaColumnProperties[subjectIndex];
                    }
                    tbGroup.sync();

                    const int globalOffset = d_candidates_per_subject_prefixsum[subjectIndex];
                    int* const myIndices = d_indices + globalOffset;
                    int* const myNumIndicesPerSubjectPtr = d_indices_per_subject + subjectIndex;

                    int* const myNewIndicesPtr = d_newIndices + globalOffset;
                    int* const myNewNumIndicesPerSubjectPtr = d_newNumIndicesPerSubject + subjectIndex;

                    bool* const myShouldBeKept = d_shouldBeKept + globalOffset;                    

                    GpuSingleMSA msa = multiMSA.getSingleMSA(subjectIndex);
                    msa.columnProperties = &shared_columnProperties;

                    if(useSmemMSA){
                        msa.counts = shared_counts;
                        msa.weights = shared_weights;
                        msa.coverages = shared_coverages;
                    }

                    if(useSmemMSA){
                        //load counts weights and coverages from gmem to smem

                        const int* const gmemCounts = multiMSA.getCountsOfMSA(subjectIndex);
                        const float* const gmemWeights = multiMSA.getWeightsOfMSA(subjectIndex);
                        const int* const gmemCoverages = multiMSA.getCoveragesOfMSA(subjectIndex);

                        for(int k = tbGroup.thread_rank(); k < msaColumnPitchInElements; k += tbGroup.size()){
                            for(int i = 0; i < 4; i++){
                                shared_counts[k + i * msaColumnPitchInElements] 
                                    = gmemCounts[k + i * msaColumnPitchInElements];
                                shared_weights[k + i * msaColumnPitchInElements] 
                                    = gmemWeights[k + i * msaColumnPitchInElements];
                            }
                            shared_coverages[k] = gmemCoverages[k];
                        }
                    }

                    auto storeSmemMSAToGmem = [&](){
                        int* const gmemCounts = multiMSA.getCountsOfMSA(subjectIndex);
                        float* const gmemWeights = multiMSA.getWeightsOfMSA(subjectIndex);
                        int* const gmemCoverages = multiMSA.getCoveragesOfMSA(subjectIndex);

                        for(int k = tbGroup.thread_rank(); k < msaColumnPitchInElements; k += tbGroup.size()){
                            for(int i = 0; i < 4; i++){
                                gmemCounts[k + i * msaColumnPitchInElements] 
                                    = shared_counts[k + i * msaColumnPitchInElements];
                                gmemWeights[k + i * msaColumnPitchInElements] 
                                    = shared_weights[k + i * msaColumnPitchInElements];
                            }
                            gmemCoverages[k] = shared_coverages[k];
                        }
                    };

                    const BestAlignment_t* const myAlignmentFlags = bestAlignmentFlags + globalOffset;
                    const int* const myShifts = shifts + globalOffset;
                    const int* const myNops = nOps + globalOffset;
                    const int* const myOverlaps = overlaps + globalOffset;

                    const unsigned int* const myAnchorSequenceData = subjectSequencesData 
                        + std::size_t(subjectIndex) * encodedSequencePitchInInts;
                    const unsigned int* const myCandidateSequencesData = candidateSequencesData 
                        + std::size_t(globalOffset) * encodedSequencePitchInInts;

                    const char* const myCandidateQualities = candidateQualities 
                        + std::size_t(globalOffset) * qualityPitchInBytes;

                    const int subjectLength = subjectSequencesLength[subjectIndex];
                    const int* const myCandidateLengths = candidateSequencesLength + globalOffset;

                    for(int refinementIteration = 0; 
                            refinementIteration < numRefinementIterations; 
                            refinementIteration++){

                        auto finalizeRefinement = [&](int newNumIndicesPerSubject){
                            //copy indices to correct output array
                            if(refinementIteration % 2 == 1){
                                for(int i = tbGroup.thread_rank(); i < myNumIndices; i += tbGroup.size()){
                                    myNewIndicesPtr[i] = myIndices[i];
                                }
                                if(tbGroup.thread_rank() == 0){
                                    *myNewNumIndicesPerSubjectPtr = *myNumIndicesPerSubjectPtr;
                                }
                            }

                            if(tbGroup.thread_rank() == 0){
                                atomicAdd(d_newNumIndices, newNumIndicesPerSubject);
                            }
                        };

                        int* const srcIndices = (refinementIteration % 2 == 0) ?
                                myIndices : myNewIndicesPtr;
                        int* const destIndices = (refinementIteration % 2 == 0) ?
                                myNewIndicesPtr : myIndices;

                        int* const srcNumIndices = (refinementIteration % 2 == 0) ?
                            myNumIndicesPerSubjectPtr : myNewNumIndicesPerSubjectPtr;
                        int* const destNumIndices = (refinementIteration % 2 == 0) ?
                            myNewNumIndicesPerSubjectPtr : myNumIndicesPerSubjectPtr;

                        tbGroup.sync();

                        auto groupReduceBool = [&](bool b, auto comp){
                            b = BlockReduceBool(temp_storage.boolreduce).Reduce(b, comp);
                            return b;
                        };

                        auto groupReduceInt2 = [&](int2 b, auto comp){
                            b = BlockReduceInt2(temp_storage.int2reduce).Reduce(b, comp);
                            return b;
                        };
                        

                        msa.flagCandidatesOfDifferentRegion(
                            tbGroup,
                            groupReduceBool,
                            groupReduceInt2,
                            destIndices,
                            destNumIndices,
                            myAnchorSequenceData,
                            subjectLength,
                            myCandidateSequencesData,
                            myCandidateLengths,
                            myAlignmentFlags,
                            myShifts,
                            myNops,
                            myOverlaps,
                            myShouldBeKept,
                            desiredAlignmentMaxErrorRate,
                            subjectIndex,
                            encodedSequencePitchInInts,
                            srcIndices,
                            *srcNumIndices,
                            dataset_coverage
                        );

                        tbGroup.sync();

                        const int myNewNumIndices = *destNumIndices;
                        
                        assert(myNewNumIndices <= myNumIndices);
                        if(myNewNumIndices > 0 && myNewNumIndices < myNumIndices){
                            auto selector = [&](int i){
                                return !myShouldBeKept[i];
                            };

                            msa.removeCandidates(
                                tbGroup,
                                selector,
                                myShifts,
                                myOverlaps,
                                myNops,
                                myAlignmentFlags,
                                myCandidateSequencesData, //not transposed
                                myCandidateQualities, //not transposed
                                myCandidateLengths,
                                srcIndices,
                                *srcNumIndices,
                                canUseQualityScores, 
                                encodedSequencePitchInInts,
                                qualityPitchInBytes,
                                desiredAlignmentMaxErrorRate
                            );

                            tbGroup.sync();

                            msa.updateColumnProperties(tbGroup);

                            tbGroup.sync();

                            //msa.checkAfterBuild(tbGroup, subjectIndex);

                            assert(shared_columnProperties.firstColumn_incl != -1);
                            assert(shared_columnProperties.lastColumn_excl != -1);

                            msa.findConsensus(
                                tbGroup,
                                myAnchorSequenceData, 
                                encodedSequencePitchInInts, 
                                subjectIndex
                            );

                            if(tbGroup.thread_rank() == 0){
                                msaColumnProperties[subjectIndex] = shared_columnProperties;
                            }

                            tbGroup.sync();

                            myNumIndices = myNewNumIndices;

                            if(refinementIteration == numRefinementIterations - 1){
                                //copy shared mem msa back to gmem

                                if(useSmemMSA){                                            
                                    storeSmemMSAToGmem();
                                }

                                finalizeRefinement(myNewNumIndices);
                            }

                        }else{
                            assert(myNewNumIndices == myNumIndices);

                            if(useSmemMSA){                                 
                                if(refinementIteration > 0){ // if iteration 0 fails, no changes were made
                                    storeSmemMSAToGmem();
                                }
                            }

                            finalizeRefinement(myNewNumIndices);

                            break; //stop refinement
                        }

                        
                    }
                }else{
                    if(tbGroup.thread_rank() == 0){
                        d_newNumIndicesPerSubject[subjectIndex] = 0;
                    }
                    ; //nothing else to do if there are no candidates in msa
                }                
            }
        }
    }









    //####################   KERNEL DISPATCH   ####################
    
    


    void callMsaFindCandidatesOfDifferentRegionAndRemoveThemViaDeletion2MultiIterationKernel_async(
        int* d_newIndices,
        int* d_newNumIndicesPerSubject,
        int* d_newNumIndices,
        MSAColumnProperties* d_msaColumnProperties,
        char* d_consensus,
        int* d_coverage,
        int* d_counts,
        float* d_weights,
        float* d_support,
        int* d_origCoverages,
        float* d_origWeights,
        const BestAlignment_t* d_bestAlignmentFlags,
        const int* d_shifts,
        const int* d_nOps,
        const int* d_overlaps,
        const unsigned int* d_subjectSequencesData,
        const unsigned int* d_candidateSequencesData,
        const unsigned int* d_transposedCandidateSequencesData,
        const int* d_subjectSequencesLength,
        const int* d_candidateSequencesLength,
        const char* d_subjectQualities,
        const char* d_candidateQualities,
        bool* d_shouldBeKept,
        const int* d_candidates_per_subject_prefixsum,
        const int* d_numAnchors,
        const int* d_numCandidates,
        float desiredAlignmentMaxErrorRate,
        int maxNumAnchors,
        int maxNumCandidates,
        bool canUseQualityScores,
        size_t encodedSequencePitchInInts,
        size_t qualityPitchInBytes,
        size_t msaColumnPitchInElements,
        int* d_indices,
        int* d_indices_per_subject,
        int dataset_coverage,
        const bool* d_canExecute,
        int numIteration,
        bool* d_anchorIsFinished,
        hipStream_t stream,
        KernelLaunchHandle& handle
    ){

        call_fill_kernel_async(
            d_newNumIndices,
            1,
            0,
            stream
        );

        constexpr int blocksize = 128;

        constexpr MemoryType memType = MemoryType::Shared;

        constexpr bool usesSmem = memType == MemoryType::Shared;

        const std::size_t smemAddSequences = (usesSmem ? 
                                                sizeof(float) * 4 * msaColumnPitchInElements // weights
                                                    + sizeof(int) * 4 * msaColumnPitchInElements // counts
                                                    + sizeof(int) * msaColumnPitchInElements // coverages
                                                : 0);

        const std::size_t smem = smemAddSequences;

        constexpr auto kernelId = KernelId::MSAFindCandidatesOfDifferentRegionAndRemoveThemViaDeletion;

        int max_blocks_per_device = 1;

        KernelLaunchConfig kernelLaunchConfig;
        kernelLaunchConfig.threads_per_block = blocksize;
        kernelLaunchConfig.smem = smem;

        auto iter = handle.kernelPropertiesMap.find(kernelId);
        if(iter == handle.kernelPropertiesMap.end()) {

            std::map<KernelLaunchConfig, KernelProperties> mymap;

            KernelProperties kernelProperties;
            hipOccupancyMaxActiveBlocksPerMultiprocessor(
                &kernelProperties.max_blocks_per_SM,
                msa_findCandidatesOfDifferentRegionAndRemoveThemViaDeletion2_multiiteration_kernel<blocksize, memType>,
                kernelLaunchConfig.threads_per_block, 
                kernelLaunchConfig.smem
            ); CUERR;

            mymap[kernelLaunchConfig] = kernelProperties;
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

            handle.kernelPropertiesMap[kernelId] = std::move(mymap);
        }else{
            std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
            const KernelProperties& kernelProperties = map[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
        }

        dim3 block(blocksize, 1, 1);
        //dim3 grid(maxNumAnchors, 1, 1);
        dim3 grid(max_blocks_per_device, 1, 1);


        msa_findCandidatesOfDifferentRegionAndRemoveThemViaDeletion2_multiiteration_kernel<blocksize, memType>
                <<<grid, block, smem, stream>>>(
            d_newIndices,
            d_newNumIndicesPerSubject,
            d_newNumIndices,
            d_msaColumnProperties,
            d_consensus,
            d_coverage,
            d_counts,
            d_weights,
            d_support,
            d_origCoverages,
            d_origWeights,
            d_bestAlignmentFlags,
            d_shifts,
            d_nOps,
            d_overlaps,
            d_subjectSequencesData,
            d_candidateSequencesData,
            d_transposedCandidateSequencesData,
            d_subjectSequencesLength,
            d_candidateSequencesLength,
            d_subjectQualities,
            d_candidateQualities,
            d_shouldBeKept,
            d_candidates_per_subject_prefixsum,
            d_numAnchors,
            d_numCandidates,
            desiredAlignmentMaxErrorRate,
            canUseQualityScores,
            encodedSequencePitchInInts,
            qualityPitchInBytes,
            msaColumnPitchInElements,
            d_indices,
            d_indices_per_subject,
            dataset_coverage,
            d_canExecute,
            numIteration,
            d_anchorIsFinished
        );
    }





    void callMsaFindCandidatesOfDifferentRegionAndRemoveThemViaRebuildKernel_async(
            int* d_newIndices,
            int* d_newNumIndicesPerSubject,
            int* d_newNumIndices,
            MSAColumnProperties* d_msaColumnProperties,
            char* d_consensus,
            int* d_coverage,
            int* d_counts,
            float* d_weights,
            float* d_support,
            int* d_origCoverages,
            float* d_origWeights,
            const BestAlignment_t* d_bestAlignmentFlags,
            const int* d_shifts,
            const int* d_nOps,
            const int* d_overlaps,
            const unsigned int* d_subjectSequencesData,
            const unsigned int* d_candidateSequencesData,
            const unsigned int* d_transposedCandidateSequencesData,
            const int* d_subjectSequencesLength,
            const int* d_candidateSequencesLength,
            const char* d_subjectQualities,
            const char* d_candidateQualities,
            bool* d_shouldBeKept,
            const int* d_candidates_per_subject_prefixsum,
            const int* d_numAnchors,
            const int* d_numCandidates,
            float desiredAlignmentMaxErrorRate,
            int maxNumAnchors,
            int maxNumCandidates,
            bool canUseQualityScores,
            size_t encodedSequencePitchInInts,
            size_t qualityPitchInBytes,
            size_t msaColumnPitchInElements,
            const int* d_indices,
            const int* d_indices_per_subject,
            int dataset_coverage,
            const bool* d_canExecute,
            int iteration,
            bool* d_anchorIsFinished,
            hipStream_t stream,
            KernelLaunchHandle& handle){

        call_fill_kernel_async(
            d_newNumIndices,
            1,
            0,
            stream
        );

        constexpr int blocksize = 128;

        constexpr MemoryType addSequencesMemType = MemoryType::Shared;

        constexpr bool addSequencesUsesSmem = addSequencesMemType == MemoryType::Shared;

        const std::size_t smemAddSequences = (addSequencesUsesSmem ? 
                                                sizeof(float) * 4 * msaColumnPitchInElements // weights
                                                    + sizeof(int) * 4 * msaColumnPitchInElements // counts
                                                    + sizeof(int) * msaColumnPitchInElements // coverages
                                                : 0);

        const std::size_t smem = smemAddSequences;

        constexpr auto kernelId = KernelId::MSAFindCandidatesOfDifferentRegionAndRemoveThemViaRebuild;

        int max_blocks_per_device = 1;

        KernelLaunchConfig kernelLaunchConfig;
        kernelLaunchConfig.threads_per_block = blocksize;
        kernelLaunchConfig.smem = smem;

        auto iter = handle.kernelPropertiesMap.find(kernelId);
        if(iter == handle.kernelPropertiesMap.end()) {

            std::map<KernelLaunchConfig, KernelProperties> mymap;

            KernelProperties kernelProperties;
            hipOccupancyMaxActiveBlocksPerMultiprocessor(
                &kernelProperties.max_blocks_per_SM,
                msa_findCandidatesOfDifferentRegionAndRemoveThemViaRebuild_kernel<blocksize, addSequencesMemType>,
                kernelLaunchConfig.threads_per_block, 
                kernelLaunchConfig.smem
            ); CUERR;

            mymap[kernelLaunchConfig] = kernelProperties;
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

            handle.kernelPropertiesMap[kernelId] = std::move(mymap);
        }else{
            std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
            const KernelProperties& kernelProperties = map[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
        }

        dim3 block(blocksize, 1, 1);
        //dim3 grid(maxNumAnchors, 1, 1);
        dim3 grid(max_blocks_per_device, 1, 1);


        msa_findCandidatesOfDifferentRegionAndRemoveThemViaRebuild_kernel<blocksize, addSequencesMemType><<<grid, block, smem, stream>>>(
            d_newIndices,
            d_newNumIndicesPerSubject,
            d_newNumIndices,
            d_msaColumnProperties,
            d_consensus,
            d_coverage,
            d_counts,
            d_weights,
            d_support,
            d_origCoverages,
            d_origWeights,
            d_bestAlignmentFlags,
            d_shifts,
            d_nOps,
            d_overlaps,
            d_subjectSequencesData,
            d_candidateSequencesData,
            d_transposedCandidateSequencesData,
            d_subjectSequencesLength,
            d_candidateSequencesLength,
            d_subjectQualities,
            d_candidateQualities,
            d_shouldBeKept,
            d_candidates_per_subject_prefixsum,
            d_numAnchors,
            d_numCandidates,
            desiredAlignmentMaxErrorRate,
            canUseQualityScores,
            encodedSequencePitchInInts,
            qualityPitchInBytes,
            msaColumnPitchInElements,
            d_indices,
            d_indices_per_subject,
            dataset_coverage,
            d_canExecute,
            iteration,
            d_anchorIsFinished
        );
    }





    void callBuildMSA3Kernel_async(
        MSAColumnProperties* d_msaColumnProperties,
        int* d_coverage,
        int* d_counts,
        float* d_weights,
        float* d_support,
        float* d_origWeights,
        int* d_origCoverages,
        char* d_consensus,          
        const int* d_overlaps,
        const int* d_shifts,
        const int* d_nOps,
        const BestAlignment_t* d_bestAlignmentFlags,
        const int* d_anchorLengths,
        const int* d_candidateLengths,
        const int* d_indices,
        const int* d_indices_per_subject,
        const int* d_candidatesPerSubjectPrefixSum,            
        const unsigned int* d_subjectSequencesData,
        const unsigned int* d_candidateSequencesTransposedData,
        const char* d_subjectQualities,
        const char* d_candidateQualities,
        const int* d_numAnchors,
        const int* d_numCandidates,
        float desiredAlignmentMaxErrorRate,
        int maxNumAnchors,
        int maxNumCandidates,
        bool canUseQualityScores,
        int encodedSequencePitchInInts,
        size_t qualityPitchInBytes,
        size_t msaColumnPitchInElements,
        const bool* d_canExecute,
        hipStream_t stream,
        KernelLaunchHandle& handle){
            

    constexpr MemoryType addSequencesMemType = MemoryType::Shared;
    constexpr bool addSequencesUsesSmem = addSequencesMemType == MemoryType::Shared;
    constexpr int BLOCKSIZE = 128;

    using BlockReduceInt = hipcub::BlockReduce<int, BLOCKSIZE>;
    using BlockReduceIntStorage = typename BlockReduceInt::TempStorage;

    const std::size_t smemCub = sizeof(BlockReduceIntStorage);
    const std::size_t smemAddSequences = (addSequencesUsesSmem ? 
                                            sizeof(float) * 4 * msaColumnPitchInElements // weights
                                                + sizeof(int) * 4 * msaColumnPitchInElements // counts
                                                + sizeof(int) * msaColumnPitchInElements // coverages
                                            : 0);

    const std::size_t smem = std::max(smemCub, smemAddSequences);

    int max_blocks_per_device = 1;

    KernelLaunchConfig kernelLaunchConfig;
    kernelLaunchConfig.threads_per_block = BLOCKSIZE;
    kernelLaunchConfig.smem = smem;

    auto iter = handle.kernelPropertiesMap.find(KernelId::MSABuild3);
    if(iter == handle.kernelPropertiesMap.end()) {

        std::map<KernelLaunchConfig, KernelProperties> mymap;

        KernelProperties kernelProperties;
        hipOccupancyMaxActiveBlocksPerMultiprocessor(
            &kernelProperties.max_blocks_per_SM,
            buildMSA3Kernel<BLOCKSIZE, addSequencesMemType>,
            kernelLaunchConfig.threads_per_block, 
            kernelLaunchConfig.smem
        ); CUERR;

        mymap[kernelLaunchConfig] = kernelProperties;
        max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

        handle.kernelPropertiesMap[KernelId::MSABuild3] = std::move(mymap);
    }else{
        std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
        const KernelProperties& kernelProperties = map[kernelLaunchConfig];
        max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
    }

    dim3 block(BLOCKSIZE, 1, 1);
    //dim3 grid(maxNumAnchors, 1, 1);
    dim3 grid(max_blocks_per_device, 1, 1);
    
    buildMSA3Kernel<BLOCKSIZE, addSequencesMemType><<<grid, block, smem, stream>>>(
        d_msaColumnProperties,
        d_coverage,
        d_counts,
        d_weights,
        d_support,
        d_origWeights,
        d_origCoverages,
        d_consensus,          
        d_overlaps,
        d_shifts,
        d_nOps,
        d_bestAlignmentFlags,
        d_anchorLengths,
        d_candidateLengths,
        d_indices,
        d_indices_per_subject,
        d_candidatesPerSubjectPrefixSum,            
        d_subjectSequencesData,
        d_candidateSequencesTransposedData,
        d_subjectQualities,
        d_candidateQualities,
        d_numAnchors,
        d_numCandidates,
        desiredAlignmentMaxErrorRate,
        canUseQualityScores,
        encodedSequencePitchInInts,
        qualityPitchInBytes,
        msaColumnPitchInElements,
        d_canExecute
    ); CUERR;



}





}
}
