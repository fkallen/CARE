#include <qualityscoreweights.hpp>
#include <gpu/qualityscoreweights.hpp>
#include <hpc_helpers.cuh>
#include <config.hpp>

#include <cmath>
#include <cassert>
#include <vector>

namespace care{
namespace gpu{

    #ifdef __NVCC__
    __device__ float d_qscore_to_weight[256];
    #endif

    void init_weights(const std::vector<int>& deviceIds){
        cpu::QualityScoreConversion conversion;
        #ifdef __NVCC__


            auto weights = conversion.getWeights();
            assert(weights.size() == 256);

            int oldId;
            hipGetDevice(&oldId); CUERR;

            for(auto deviceId : deviceIds){
                hipSetDevice(deviceId);
                hipMemcpyToSymbol(HIP_SYMBOL(d_qscore_to_weight), weights.data(), 256*sizeof(float)); CUERR;
            }

            hipSetDevice(oldId); CUERR;
        #endif
    }


}
}
