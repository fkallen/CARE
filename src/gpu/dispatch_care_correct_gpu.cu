#include "hip/hip_runtime.h"
#include <gpu/dispatch_care_correct_gpu.cuh>
#include <gpu/gpuminhasherconstruction.cuh>
#include <gpu/fakegpuminhasher.cuh>
#include <gpu/cudaerrorcheck.cuh>

#include <config.hpp>
#include <options.hpp>
#include <readlibraryio.hpp>
#include <memorymanagement.hpp>
#include <correctedsequence.hpp>
#include <correctionresultoutput.hpp>
#include <gpu/correct_gpu.hpp>
#include <classification.hpp>
#include <gpu/forest_gpu.cuh>

#include <gpu/multigpureadstorage.cuh>
#include <chunkedreadstorageconstruction.hpp>
#include <chunkedreadstorage.hpp>

#include <rangegenerator.hpp>
#include <sortserializedresults.hpp>


#include <algorithm>
#include <iostream>
#include <memory>
#include <vector>
#include <numeric>
#include <random>

#include <experimental/filesystem>

#include <rmm/mr/device/per_device_resource.hpp>
#include <rmm/mr/device/cuda_async_memory_resource.hpp>
#include <rmm/mr/device/logging_resource_adaptor.hpp>
#include <gpu/rmm_utilities.cuh>

namespace filesys = std::experimental::filesystem;

namespace care{

    namespace correction{

        std::vector<int> getUsableDeviceIds(std::vector<int> deviceIds){
            int nDevices;

            CUDACHECK(hipGetDeviceCount(&nDevices));

            std::vector<int> invalidIds;

            for(int id : deviceIds) {
                if(id >= nDevices) {
                    invalidIds.emplace_back(id);
                    std::cout << "Found invalid device Id: " << id << std::endl;
                }
            }

            if(invalidIds.size() > 0) {
                std::cout << "Available GPUs on your machine:" << std::endl;
                for(int j = 0; j < nDevices; j++) {
                    hipDeviceProp_t prop;
                    CUDACHECK(hipGetDeviceProperties(&prop, j));
                    std::cout << "Id " << j << " : " << prop.name << std::endl;
                }

                for(int invalidid : invalidIds) {
                    deviceIds.erase(std::find(deviceIds.begin(), deviceIds.end(), invalidid));
                }
            }

            //check gpu restrictions of remaining gpus
            invalidIds.clear();

            for(int id : deviceIds){
                hipDeviceProp_t prop;
                hipGetDeviceProperties(&prop, id);

                if(prop.major < 6){
                    invalidIds.emplace_back(id);
                    std::cerr << "Warning. Removing gpu id " << id << " because its not arch 6 or greater.\n";
                }

                if(prop.managedMemory != 1){
                    invalidIds.emplace_back(id);
                    std::cerr << "Warning. Removing gpu id " << id << " because it does not support managed memory. (may be required for hash table construction).\n";
                }
            }

            if(invalidIds.size() > 0) {
                for(int invalidid : invalidIds) {
                    deviceIds.erase(std::find(deviceIds.begin(), deviceIds.end(), invalidid));
                }
            }

            return deviceIds;
        }

    }

    

    template<class T>
    void printDataStructureMemoryUsage(const T& datastructure, const std::string& name){
    	auto toGB = [](std::size_t bytes){
    			    double gb = bytes / 1024. / 1024. / 1024.0;
    			    return gb;
    		    };

        auto memInfo = datastructure.getMemoryInfo();
        
        std::cout << name << " memory usage: " << toGB(memInfo.host) << " GB on host\n";
        for(const auto& pair : memInfo.device){
            std::cout << name << " memory usage: " << toGB(pair.second) << " GB on device " << pair.first << '\n';
        }
    }




    void performCorrection(
        ProgramOptions programOptions
    ){

        std::cout << "Running CARE GPU" << std::endl;

        if(programOptions.deviceIds.size() == 0){
            std::cout << "No device ids found. Abort!" << std::endl;
            return;
        }

        if(programOptions.correctionType == CorrectionType::Print 
            || programOptions.correctionTypeCands == CorrectionType::Print){

            std::cout << "CorrectionType Print is not supported in CARE GPU. Please use CARE CPU instead to print features. Abort!" << std::endl;
            return;
        }

        CUDACHECK(hipSetDevice(programOptions.deviceIds[0]));

        //debug buffer printf
        CUDACHECK(hipDeviceSetLimit(hipLimitPrintfFifoSize, 1024*1024*512));

        helpers::PeerAccessDebug peerAccess(programOptions.deviceIds, true);
        peerAccess.enableAllPeerAccesses();

        //set up memory pools for malloc_async
        for(auto id : programOptions.deviceIds){
            hipMemPool_t defaultMemoryPool;
            CUDACHECK(hipDeviceGetDefaultMemPool(&defaultMemoryPool, id));
            uint64_t threshold = UINT64_MAX;
            CUDACHECK(hipMemPoolSetAttribute(defaultMemoryPool, hipMemPoolAttrReleaseThreshold, &threshold));
        }

        //set up rmm resources
        std::vector<std::unique_ptr<MyRMMCudaAsyncResource>> rmmCudaAsyncResources;
        std::vector<std::unique_ptr<rmm::mr::logging_resource_adaptor<MyRMMCudaAsyncResource>>> rmmLoggingResources;
        std::vector<std::unique_ptr<std::ofstream>> logfilestreams;

        for(auto id : programOptions.deviceIds){
            hipcub::SwitchDevice sd(id);

            hipMemPool_t defaultMemoryPool;
            CUDACHECK(hipDeviceGetDefaultMemPool(&defaultMemoryPool, id));

            const bool autoflush = true;

            rmmCudaAsyncResources.push_back(std::make_unique<MyRMMCudaAsyncResource>(defaultMemoryPool));
            logfilestreams.push_back(std::make_unique<std::ofstream>("logging_device_0.txt"));
            rmmLoggingResources.push_back(std::make_unique<rmm::mr::logging_resource_adaptor<MyRMMCudaAsyncResource>>(rmmCudaAsyncResources.back().get(), *logfilestreams.back(), autoflush));

            rmm::mr::set_per_device_resource(rmm::cuda_device_id(id), rmmCudaAsyncResources.back().get());
            //rmm::mr::set_per_device_resource(rmm::cuda_device_id(id), rmmLoggingResources.back().get());
        }

        
        /*
            Step 1: 
            - load all reads from all input files into (gpu-)memory
            - construct minhash signatures of all reads and store them in hash tables
        */

        helpers::CpuTimer step1timer("STEP1");

        std::cout << "STEP 1: Database construction" << std::endl;

        helpers::CpuTimer buildReadStorageTimer("build_readstorage");

        const int numQualityBits = programOptions.qualityScoreBits;
        
        std::unique_ptr<ChunkedReadStorage> cpuReadStorage = constructChunkedReadStorageFromFiles(
            programOptions
        );

        buildReadStorageTimer.print();

        std::cout << "Determined the following read properties:\n";
        std::cout << "----------------------------------------\n";
        std::cout << "Total number of reads: " << cpuReadStorage->getNumberOfReads() << "\n";
        std::cout << "Minimum sequence length: " << cpuReadStorage->getSequenceLengthLowerBound() << "\n";
        std::cout << "Maximum sequence length: " << cpuReadStorage->getSequenceLengthUpperBound() << "\n";
        std::cout << "----------------------------------------\n";

        if(programOptions.save_binary_reads_to != ""){
            std::cout << "Saving reads to file " << programOptions.save_binary_reads_to << std::endl;
            helpers::CpuTimer timer("save_to_file");
            cpuReadStorage->saveToFile(programOptions.save_binary_reads_to);
            timer.print();
            std::cout << "Saved reads" << std::endl;
        }

        if(programOptions.autodetectKmerlength){
            const int maxlength = cpuReadStorage->getSequenceLengthUpperBound();

            auto getKmerSizeForHashing = [](int maximumReadLength){
                if(maximumReadLength < 160){
                    return 20;
                }else{
                    return 32;
                }
            };

            programOptions.kmerlength = getKmerSizeForHashing(maxlength);

            std::cout << "Will use k-mer length = " << programOptions.kmerlength << " for hashing.\n";
        }

        std::cout << "Reads with ambiguous bases: " << cpuReadStorage->getNumberOfReadsWithN() << std::endl;

        //compareMaxRssToLimit(programOptions.memoryTotalLimit, "Error memorylimit after cpureadstorage");

        std::vector<std::size_t> gpumemorylimits(programOptions.deviceIds.size(), 0);

        // gpumemorylimits.resize(2);
        // std::fill(gpumemorylimits.begin(), gpumemorylimits.end(), 512000000);

        // std::vector<int> tempids2(gpumemorylimits.size(), 0);


        gpu::MultiGpuReadStorage gpuReadStorage(
            *cpuReadStorage, 
            programOptions.deviceIds,
            //tempids2,
            gpumemorylimits,
            0,
            numQualityBits
        );

        std::vector<gpu::GpuForest> anchorForests;
        std::vector<gpu::GpuForest> candidateForests;

        {
            ClfAgent clfAgent_(programOptions);

            for(int deviceId : programOptions.deviceIds){
                hipcub::SwitchDevice sd{deviceId};
                if(programOptions.correctionType == CorrectionType::Forest){
                    anchorForests.emplace_back(*clfAgent_.classifier_anchor, deviceId);
                }

                if(programOptions.correctionTypeCands == CorrectionType::Forest){
                    candidateForests.emplace_back(*clfAgent_.classifier_cands, deviceId);
                }
            }

        }

        helpers::CpuTimer buildMinhasherTimer("build_minhasher");

        auto minhasherAndType = gpu::constructGpuMinhasherFromGpuReadStorage(
            programOptions,
            gpuReadStorage,
            gpu::GpuMinhasherType::Multi
        );

        //compareMaxRssToLimit(programOptions.memoryTotalLimit, "Error memorylimit after gpuminhasher");

        gpu::GpuMinhasher* gpuMinhasher = minhasherAndType.first.get();

        buildMinhasherTimer.print();

        std::cout << "Using minhasher type: " << to_string(minhasherAndType.second) << "\n";
        std::cout << "GpuMinhasher can use " << gpuMinhasher->getNumberOfMaps() << " maps\n";

        if(gpuMinhasher->getNumberOfMaps() <= 0){
            std::cout << "Cannot construct a single gpu hashtable. Abort!" << std::endl;
            return;
        }

        if(programOptions.mustUseAllHashfunctions 
            && programOptions.numHashFunctions != gpuMinhasher->getNumberOfMaps()){
            std::cout << "Cannot use specified number of hash functions (" 
                << programOptions.numHashFunctions <<")\n";
            std::cout << "Abort!\n";
            return;
        }

        if(minhasherAndType.second == gpu::GpuMinhasherType::Fake){

            gpu::FakeGpuMinhasher* fakeGpuMinhasher = dynamic_cast<gpu::FakeGpuMinhasher*>(gpuMinhasher);
            assert(fakeGpuMinhasher != nullptr);

            if(programOptions.save_hashtables_to != "") {
                std::cout << "Saving minhasher to file " << programOptions.save_hashtables_to << std::endl;
                std::ofstream os(programOptions.save_hashtables_to);
                assert((bool)os);
                helpers::CpuTimer timer("save_to_file");
                fakeGpuMinhasher->writeToStream(os);
                timer.print();

                std::cout << "Saved minhasher" << std::endl;
            }

        }

        printDataStructureMemoryUsage(*gpuMinhasher, "hash tables");        

        step1timer.print();

        //After minhasher is constructed, remaining gpu memory can be used to store reads

        //std::fill(gpumemorylimits.begin(), gpumemorylimits.end(), 2ull*1024ull*1024ull*1024ull);
        std::fill(gpumemorylimits.begin(), gpumemorylimits.end(), 0);
        for(int i = 0; i < int(programOptions.deviceIds.size()); i++){
            std::size_t total = 0;
            hipMemGetInfo(&gpumemorylimits[i], &total);

            std::size_t safety = 1 << 30; //leave 1 GB for correction algorithm
            if(gpumemorylimits[i] > safety){
                gpumemorylimits[i] -= safety;
            }else{
                gpumemorylimits[i] = 0;
            }
        }

        std::size_t memoryLimitHost = programOptions.memoryTotalLimit 
            - cpuReadStorage->getMemoryInfo().host
            - gpuMinhasher->getMemoryInfo().host;

        // gpumemorylimits.resize(2);
        // std::fill(gpumemorylimits.begin(), gpumemorylimits.end(), 128000000);

        // std::vector<int> tempids(gpumemorylimits.size(), 0);

        helpers::CpuTimer cpugputimer("cpu->gpu reads");
        cpugputimer.start();
        gpuReadStorage.rebuild(
            *cpuReadStorage,
            programOptions.deviceIds, 
            //tempids,
            gpumemorylimits,
            memoryLimitHost,
            numQualityBits
        );
        cpugputimer.print();

        printDataStructureMemoryUsage(gpuReadStorage, "reads");

        //compareMaxRssToLimit(programOptions.memoryTotalLimit, "Error memorylimit after gpureadstorage");

        //std::cout << "constructed gpu readstorage " << std::endl;

        

        if(gpuReadStorage.isStandalone()){
            cpuReadStorage.reset();
        }


        // {
        //     auto rshandle = gpuReadStorage.makeHandle();
        //     helpers::SimpleAllocationPinnedHost<char> d_quality_data(128 * 10);
        //     helpers::SimpleAllocationPinnedHost<read_number> readIds(10);
        //     std::iota(readIds.begin(), readIds.end(), 0);

        //     gpuReadStorage.gatherQualities(
        //         rshandle,
        //         d_quality_data,
        //         128,
        //         AsyncConstBufferWrapper<read_number>(readIds.data()),
        //         readIds.data(),
        //         10,
        //         0
        //     );
        //     CUDACHECK(hipDeviceSynchronize());

        //     for(int k = 0; k < 10; k++){
        //         for(int i = 0; i < 101; i++){
        //             std::cerr << d_quality_data[128 * k + i];
        //         }
        //         std::cerr << "\n";
        //     }
        //     std::exit(0);
        // }


        std::cout << "STEP 2: Error correction" << std::endl;

        helpers::CpuTimer step2timer("STEP2");

        auto partialResults = gpu::correct_gpu(
            programOptions,
            *gpuMinhasher, 
            gpuReadStorage,
            anchorForests,
            candidateForests
        );

        step2timer.print();

        std::cout << "Correction throughput : ~" << (gpuReadStorage.getNumberOfReads() / step2timer.elapsed()) << " reads/second.\n";
    
        std::cerr << "Constructed " << partialResults.size() << " corrections. ";
        std::cerr << "They occupy a total of " << (partialResults.dataBytes() + partialResults.offsetBytes()) << " bytes\n";

        //compareMaxRssToLimit(programOptions.memoryTotalLimit, "Error memorylimit after correction");


        minhasherAndType.first.reset();
        gpuMinhasher = nullptr;
        gpuReadStorage.destroy();
        cpuReadStorage.reset();


        //Merge corrected reads with input file to generate output file

        const std::size_t availableMemoryInBytes = getAvailableMemoryInKB() * 1024;
        const auto partialResultMemUsage = partialResults.getMemoryInfo();

        // std::cerr << "availableMemoryInBytes = " << availableMemoryInBytes << "\n";
        // std::cerr << "memoryLimitOption = " << programOptions.memoryTotalLimit << "\n";
        // std::cerr << "partialResultMemUsage = " << partialResultMemUsage.host << "\n";

        std::size_t memoryForSorting = std::min(
            availableMemoryInBytes,
            programOptions.memoryTotalLimit - partialResultMemUsage.host
        );

        if(memoryForSorting > 1*(std::size_t(1) << 30)){
            memoryForSorting = memoryForSorting - 1*(std::size_t(1) << 30);
        }
        //std::cerr << "memoryForSorting = " << memoryForSorting << "\n";        

        std::cout << "STEP 3: Constructing output file(s)" << std::endl;

        helpers::CpuTimer step3timer("STEP3");

        helpers::CpuTimer sorttimer("sort_results_by_read_id");

        sortSerializedResultsByReadIdAscending<EncodedTempCorrectedSequence>(
            partialResults,
            memoryForSorting
        );

        sorttimer.print();

        std::vector<FileFormat> formats;
        for(const auto& inputfile : programOptions.inputfiles){
            formats.emplace_back(getFileFormat(inputfile));
        }
        std::vector<std::string> outputfiles;
        for(const auto& outputfilename : programOptions.outputfilenames){
            outputfiles.emplace_back(programOptions.outputdirectory + "/" + outputfilename);
        }
        constructOutputFileFromCorrectionResults(
            programOptions.inputfiles, 
            partialResults, 
            formats[0],
            outputfiles,
            programOptions.showProgress
        );

        step3timer.print();

        //compareMaxRssToLimit(programOptions.memoryTotalLimit, "Error memorylimit after output construction");

        std::cout << "Construction of output file(s) finished." << std::endl;

    }


}
