#include "hip/hip_runtime.h"

#include <gpu/gpuminhasherconstruction.cuh>

#include <gpu/gpuminhasher.cuh>
#include <gpu/fakegpuminhasher.cuh>
#include <gpu/singlegpuminhasher.cuh>
#include <gpu/multigpuminhasher.cuh>
#include <minhasherlimit.hpp>

#include <options.hpp>

#include <memory>
#include <utility>


namespace care{
namespace gpu{

        std::string to_string(GpuMinhasherType type){
            switch(type){
                case GpuMinhasherType::Fake: return "FakeGpu";
                case GpuMinhasherType::FakeSingleHash: return "FakeGpuSingleHash";
                case GpuMinhasherType::Single: return "SingleGpu";
                case GpuMinhasherType::SingleSingleHash: return "SingleGpuSingleHash";
                case GpuMinhasherType::Multi: return "MultiGpu";
                case GpuMinhasherType::MultiSingleHash: return "MultiGpuSingleHash";
                case GpuMinhasherType::None: return "None";
                default: return "Unknown";
            }
        }

        void constructGpuMinhasherFromReadStorage(
            const ProgramOptions& programOptions,
            const GpuReadStorage& gpuReadStorage,
            GpuMinhasher* gpuMinhasher
        ){
            
            auto& readStorage = gpuReadStorage;
            const auto& deviceIds = programOptions.deviceIds;
    
            int deviceId = deviceIds[0];
    
            hipcub::SwitchDevice sd{deviceId};
    
            const int requestedNumberOfMaps = programOptions.numHashFunctions;
    
            const read_number numReads = readStorage.getNumberOfReads();
            const int maximumSequenceLength = readStorage.getSequenceLengthUpperBound();
    
            auto sequencehandle = gpuReadStorage.makeHandle();
            const std::size_t encodedSequencePitchInInts = SequenceHelpers::getEncodedNumInts2Bit(maximumSequenceLength);
    
            rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource();
    
            constexpr read_number parallelReads = 1000000;
            const int numIters = SDIV(numReads, parallelReads);
    
            const MemoryUsage memoryUsageOfReadStorage = readStorage.getMemoryInfo();
            std::size_t totalLimit = programOptions.memoryTotalLimit;
            if(totalLimit > memoryUsageOfReadStorage.host){
                totalLimit -= memoryUsageOfReadStorage.host;
            }else{
                totalLimit = 0;
            }
            if(totalLimit == 0){
                throw std::runtime_error("Not enough memory available for hash tables. Abort!");
            }
            std::size_t maxMemoryForTables = getAvailableMemoryInKB() * 1024;
            // std::cerr << "available: " << maxMemoryForTables 
            //         << ",memoryForHashtables: " << programOptions.memoryForHashtables
            //         << ", memoryTotalLimit: " << programOptions.memoryTotalLimit
            //         << ", rsHostUsage: " << memoryUsageOfReadStorage.host << "\n";
    
            maxMemoryForTables = std::min(maxMemoryForTables, 
                                    std::min(programOptions.memoryForHashtables, totalLimit));
    
            std::cerr << "maxMemoryForTables = " << maxMemoryForTables << " bytes\n";
    
            const int hashFunctionOffset = 0;
    
            
            std::vector<int> usedHashFunctionNumbers;
    
            hipStream_t stream = hipStreamPerThread;
            
            rmm::device_uvector<unsigned int> d_sequenceData(encodedSequencePitchInInts * parallelReads, stream, mr);
            rmm::device_uvector<int> d_lengths(parallelReads, stream, mr);
            rmm::device_uvector<read_number> d_indices(parallelReads, stream, mr);
            
            helpers::SimpleAllocationPinnedHost<read_number, 0> h_indices(parallelReads);
    
            
            ThreadPool tpForHashing(programOptions.threads);
            ThreadPool tpForCompacting(std::min(2,programOptions.threads));
    
            
            gpuMinhasher->setHostMemoryLimitForConstruction(maxMemoryForTables);
            gpuMinhasher->setDeviceMemoryLimitsForConstruction({1*1024*1024*1024});

            
            
            //std::size_t bytesOfCachedConstructedTables = 0;
            int remainingHashFunctions = requestedNumberOfMaps;
            bool keepGoing = true;
    
            while(remainingHashFunctions > 0 && keepGoing){
    
                gpuMinhasher->setThreadPool(&tpForHashing);

                const int alreadyExistingHashFunctions = requestedNumberOfMaps - remainingHashFunctions;
                std::vector<int> h_hashfunctionNumbers(remainingHashFunctions);
                std::iota(
                    h_hashfunctionNumbers.begin(),
                    h_hashfunctionNumbers.end(),
                    alreadyExistingHashFunctions + hashFunctionOffset
                );
    
                //Hacky way to limit gpu memory usage of hash tables
                constexpr std::size_t hackbytes = 1024 * 1024 * 1024;
                char* hackbuffer = nullptr;
                std::size_t free, total;
                CUDACHECK(hipMemGetInfo(&free, &total));
                if(free > hackbytes){
                    CUDACHECK(hipMalloc(&hackbuffer, hackbytes));
                }

                int addedHashFunctions = gpuMinhasher->addHashTables(remainingHashFunctions,h_hashfunctionNumbers.data(), stream);

                CUDACHECK(hipFree(hackbuffer));
    
                if(addedHashFunctions == 0){
                    keepGoing = false;
                    break;
                }
    
                std::cout << "Constructing maps: ";
                for(int i = 0; i < addedHashFunctions; i++){
                    std::cout << (alreadyExistingHashFunctions + i) << "(" << (hashFunctionOffset + alreadyExistingHashFunctions + i) << ") ";
                }
                std::cout << '\n';
    
                usedHashFunctionNumbers.insert(usedHashFunctionNumbers.end(), h_hashfunctionNumbers.begin(), h_hashfunctionNumbers.begin() + addedHashFunctions);
    
                for (int iter = 0; iter < numIters; iter++){
                    read_number readIdBegin = iter * parallelReads;
                    read_number readIdEnd = std::min((iter + 1) * parallelReads, numReads);
    
                    const std::size_t curBatchsize = readIdEnd - readIdBegin;
    
                    std::iota(h_indices.get(), h_indices.get() + curBatchsize, readIdBegin);
    
                    CUDACHECK(hipMemcpyAsync(d_indices.data(), h_indices, sizeof(read_number) * curBatchsize, H2D, stream));
    
                    gpuReadStorage.gatherSequences(
                        sequencehandle,
                        d_sequenceData.data(),
                        encodedSequencePitchInInts,
                        makeAsyncConstBufferWrapper(h_indices.data()),
                        d_indices.data(),
                        curBatchsize,
                        stream,
                        mr
                    );
                
                    gpuReadStorage.gatherSequenceLengths(
                        sequencehandle,
                        d_lengths.data(),
                        d_indices.data(),
                        curBatchsize,
                        stream
                    );
    
                    gpuMinhasher->insert(
                        d_sequenceData.data(),
                        curBatchsize,
                        d_lengths.data(),
                        encodedSequencePitchInInts,
                        d_indices.data(),
                        h_indices,
                        alreadyExistingHashFunctions,
                        addedHashFunctions,
                        h_hashfunctionNumbers.data(),
                        stream,
                        mr
                    );
    
                    CUDACHECK(hipStreamSynchronize(stream));
                }
    
                CUDACHECK(hipStreamSynchronize(stream));
    
                std::cerr << "Compacting\n";
                if(tpForCompacting.getConcurrency() > 1){
                    gpuMinhasher->setThreadPool(&tpForCompacting);
                }else{
                    gpuMinhasher->setThreadPool(nullptr);
                }
                
                gpuMinhasher->compact(stream);
                CUDACHECK(hipStreamSynchronize(stream));
    
                remainingHashFunctions -= addedHashFunctions;
            }
    
            gpuMinhasher->setThreadPool(nullptr); 
            
            gpuReadStorage.destroyHandle(sequencehandle);
    
            gpuMinhasher->constructionIsFinished(stream);
            CUDACHECK(hipStreamSynchronize(stream));
        }
    
    
        std::pair<std::unique_ptr<GpuMinhasher>, GpuMinhasherType>
        constructGpuMinhasherFromGpuReadStorage(
            const ProgramOptions& programOptions,
            const GpuReadStorage& gpuReadStorage,
            GpuMinhasherType requestedType
        ){
            std::unique_ptr<GpuMinhasher> gpuMinhasher;
            GpuMinhasherType gpuMinhasherType = GpuMinhasherType::None;

            auto makeFake = [&](){                
                gpuMinhasher = std::make_unique<FakeGpuMinhasher>(
                    gpuReadStorage.getNumberOfReads(),
                    calculateResultsPerMapThreshold(programOptions.estimatedCoverage),
                    programOptions.kmerlength,
                    programOptions.hashtableLoadfactor
                );

                gpuMinhasherType = GpuMinhasherType::Fake;
            };

            auto makeSingle = [&](){
                gpuMinhasher = std::make_unique<SingleGpuMinhasher>(
                    gpuReadStorage.getNumberOfReads(), 
                    calculateResultsPerMapThreshold(programOptions.estimatedCoverage), 
                    programOptions.kmerlength
                );

                gpuMinhasherType = GpuMinhasherType::Single;
            };

            auto makeMulti = [&](){
                gpuMinhasher = std::make_unique<MultiGpuMinhasher>(
                    MultiGpuMinhasher::Layout::FirstFit,
                    gpuReadStorage.getNumberOfReads(), 
                    calculateResultsPerMapThreshold(programOptions.estimatedCoverage), 
                    programOptions.kmerlength,
                    programOptions.deviceIds
                );

                gpuMinhasherType = GpuMinhasherType::Multi;
            };

            if(requestedType == GpuMinhasherType::Fake || programOptions.warpcore == 0){
                makeFake();

            #ifdef CARE_HAS_WARPCORE
            }else if(requestedType == GpuMinhasherType::Single || programOptions.deviceIds.size() < 2){
                makeSingle();
            }else if(requestedType == GpuMinhasherType::Multi){
                makeMulti();
            #endif
            }else{
                makeFake();
            }

            if(programOptions.load_hashtables_from != "" && gpuMinhasher->canLoadFromStream()){

                std::ifstream is(programOptions.load_hashtables_from);
                assert((bool)is);

                const int loadedMaps = gpuMinhasher->loadFromStream(is, programOptions.numHashFunctions);

                std::cout << "Loaded " << loadedMaps << " hash tables from " << programOptions.load_hashtables_from << std::endl;
            }else{
                constructGpuMinhasherFromReadStorage(
                    programOptions,
                    gpuReadStorage,
                    gpuMinhasher.get()
                );
            }

            return {std::move(gpuMinhasher), gpuMinhasherType};
        }
    
    
}
}