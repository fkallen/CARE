#include "hip/hip_runtime.h"
//#define NDEBUG

#include <gpu/kernels.hpp>
#include <gpu/kernellaunch.hpp>
#include <hostdevicefunctions.cuh>
#include <gpu/gpumsa.cuh>

#include <bestalignment.hpp>

#include <sequencehelpers.hpp>
#include <correctionresultprocessing.hpp>

#include <hpc_helpers.cuh>
#include <config.hpp>
#include <cassert>

#include <gpu/forest_gpu.cuh>


#include <hipcub/hipcub.hpp>

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

namespace care{
namespace gpu{



    template<int BLOCKSIZE>
    __global__
    void msaCorrectAnchorsKernel(
            char* __restrict__ correctedSubjects,
            bool* __restrict__ subjectIsCorrected,
            AnchorHighQualityFlag* __restrict__ isHighQualitySubject,
            GPUMultiMSA multiMSA,
            const unsigned int* __restrict__ subjectSequencesData,
            const unsigned int* __restrict__ candidateSequencesData,
            const int* __restrict__ candidateSequencesLength,
            const int* __restrict__ d_indices_per_subject,
            const int* __restrict__ numAnchorsPtr,
            int encodedSequencePitchInInts,
            size_t decodedSequencePitchInBytes,
            int maximumSequenceLength,
            float estimatedErrorrate,
            float desiredAlignmentMaxErrorRate,
            float avg_support_threshold,
            float min_support_threshold,
            float min_coverage_threshold,
            float max_coverage_threshold,
            int k_region){

        using BlockReduceInt = hipcub::BlockReduce<int, BLOCKSIZE>;
        using BlockReduceFloat = hipcub::BlockReduce<float, BLOCKSIZE>;

        __shared__ union {
            typename BlockReduceInt::TempStorage intreduce;
            typename BlockReduceFloat::TempStorage floatreduce;
            GpuMSAProperties msaProperties;
        } temp_storage;

        __shared__ int broadcastbuffer;

        //__shared__ int numUncorrectedPositions;
        //__shared__ int uncorrectedPositions[BLOCKSIZE];
        //__shared__ float avgCountPerWeight[4];

        auto tbGroup = cg::this_thread_block();

        auto groupReduceFloatSum = [&](float f){
            const float result = BlockReduceFloat(temp_storage.floatreduce).Sum(f);
            __syncthreads();
            return result;
        };

        auto groupReduceFloatMin = [&](float f){
            const float result = BlockReduceFloat(temp_storage.floatreduce).Reduce(f, hipcub::Min{});
            __syncthreads();
            return result;
        };

        auto groupReduceIntMin = [&](int i){
            const int result = BlockReduceInt(temp_storage.intreduce).Reduce(i, hipcub::Min{});
            __syncthreads();
            return result;
        };

        auto groupReduceIntMax = [&](int i){
            const int result = BlockReduceInt(temp_storage.intreduce).Reduce(i, hipcub::Max{});
            __syncthreads();
            return result;
        };

        const int n_subjects = *numAnchorsPtr;

        auto isGoodAvgSupport = [&](float avgsupport){
            return fgeq(avgsupport, avg_support_threshold);
        };
        auto isGoodMinSupport = [&](float minsupport){
            return fgeq(minsupport, min_support_threshold);
        };
        auto isGoodMinCoverage = [&](float mincoverage){
            return fgeq(mincoverage, min_coverage_threshold);
        };

        auto to_nuc = [](std::uint8_t c){
            return SequenceHelpers::decodeBase(c);
        };

        for(unsigned subjectIndex = blockIdx.x; subjectIndex < n_subjects; subjectIndex += gridDim.x){
            const int myNumIndices = d_indices_per_subject[subjectIndex];
            if(myNumIndices > 0){

                const GpuSingleMSA msa = multiMSA.getSingleMSA(subjectIndex);

                char* const my_corrected_subject = correctedSubjects + subjectIndex * decodedSequencePitchInBytes;

                const int subjectColumnsBegin_incl = msa.columnProperties->subjectColumnsBegin_incl;
                const int subjectColumnsEnd_excl = msa.columnProperties->subjectColumnsEnd_excl;
                //const int lastColumn_excl = msa.columnProperties->lastColumn_excl;

                GpuMSAProperties msaProperties = msa.getMSAProperties(
                    tbGroup,
                    groupReduceFloatSum,
                    groupReduceFloatMin,
                    groupReduceIntMin,
                    groupReduceIntMax,
                    subjectColumnsBegin_incl,
                    subjectColumnsEnd_excl
                );

                if(tbGroup.thread_rank() == 0){
                    subjectIsCorrected[subjectIndex] = true; //canBeCorrected;

                    const bool canBeCorrectedByConsensus = isGoodAvgSupport(msaProperties.avg_support) && isGoodMinSupport(msaProperties.min_support) && isGoodMinCoverage(msaProperties.min_coverage);
                    int flag = 0;

                    if(canBeCorrectedByConsensus){
                        int smallestErrorrateThatWouldMakeHQ = 100;

                        const int estimatedErrorratePercent = ceil(estimatedErrorrate * 100.0f);
                        for(int percent = estimatedErrorratePercent; percent >= 0; percent--){
                            const float factor = percent / 100.0f;
                            const float avg_threshold = 1.0f - 1.0f * factor;
                            const float min_threshold = 1.0f - 3.0f * factor;
                            if(fgeq(msaProperties.avg_support, avg_threshold) && fgeq(msaProperties.min_support, min_threshold)){
                                smallestErrorrateThatWouldMakeHQ = percent;
                            }
                            // if(readId == 134){
                            //     printf("avg_support %f, avg_threshold %f, msaProperties.min_support %f, min_threshold %f\n", 
                            //     avg_support, avg_threshold,msaProperties. min_support, min_threshold);
                            // }
                        }

                        const bool isHQ = isGoodMinCoverage(msaProperties.min_coverage)
                                            && fleq(smallestErrorrateThatWouldMakeHQ, estimatedErrorratePercent * 0.5f);

                        //broadcastbuffer = isHQ;
                        isHighQualitySubject[subjectIndex].hq(isHQ);

                        flag = isHQ ? 2 : 1;

                        // if(readId == 134){
                        //     printf("read 134 isHQ %d, min_coverage %d, avg_support %f, min_support %f, smallestErrorrateThatWouldMakeHQ %d, min_coverage_threshold %f\n", 
                        //         isHQ, min_coverage, avg_support, min_support, smallestErrorrateThatWouldMakeHQ, min_coverage_threshold);
                        // }
                    }else{
                        isHighQualitySubject[subjectIndex].hq(false);
                    }

                    broadcastbuffer = flag;
                }

                tbGroup.sync();

                const int flag = broadcastbuffer;

                if(flag > 0){
                    for(int i = subjectColumnsBegin_incl + tbGroup.thread_rank(); 
                            i < subjectColumnsEnd_excl; 
                            i += tbGroup.size()){

                        const std::uint8_t nuc = msa.consensus[i];
                        //assert(nuc == 'A' || nuc == 'C' || nuc == 'G' || nuc == 'T');
                        assert(0 == nuc || nuc < 4);

                        my_corrected_subject[i - subjectColumnsBegin_incl] = to_nuc(nuc);
                    }
                }else{
                    //correct only positions with high support.
                    for(int i = subjectColumnsBegin_incl + tbGroup.thread_rank(); 
                            i < subjectColumnsEnd_excl; 
                            i += tbGroup.size()){

                        
                        if(msa.support[i] > 0.90f && msa.origCoverages[i] <= 2){
                            my_corrected_subject[i - subjectColumnsBegin_incl] = to_nuc(msa.consensus[i]);
                        }else{
                            const unsigned int* const subject = subjectSequencesData + std::size_t(subjectIndex) * encodedSequencePitchInInts;
                            const std::uint8_t encodedBase = SequenceHelpers::getEncodedNuc2Bit(subject, subjectColumnsEnd_excl- subjectColumnsBegin_incl, i - subjectColumnsBegin_incl);
                            const char base = to_nuc(encodedBase);
                            assert(base == 'A' || base == 'C' || base == 'G' || base == 'T');
                            my_corrected_subject[i - subjectColumnsBegin_incl] = base;
                        }
                    }
                }
            }else{
                if(tbGroup.thread_rank() == 0){
                    isHighQualitySubject[subjectIndex].hq(false);
                    subjectIsCorrected[subjectIndex] = false;
                }
            }
        }
    }


    template<int BLOCKSIZE, class GpuClf>
    __global__
    void msaCorrectAnchorsWithForestKernel(
        char* __restrict__ correctedSubjects,
        bool* __restrict__ subjectIsCorrected,
        AnchorHighQualityFlag* __restrict__ isHighQualitySubject,
        GPUMultiMSA multiMSA,
        GpuClf gpuForest,
        float forestThreshold,
        const unsigned int* __restrict__ subjectSequencesData,
        const int* __restrict__ d_indices_per_subject,
        const int numAnchors,
        int encodedSequencePitchInInts,
        size_t decodedSequencePitchInBytes,
        int maximumSequenceLength,
        float estimatedErrorrate,
        float desiredAlignmentMaxErrorRate,
        float estimatedCoverage,
        float avg_support_threshold,
        float min_support_threshold,
        float min_coverage_threshold,
        float max_coverage_threshold
    ){

        using BlockReduceInt = hipcub::BlockReduce<int, BLOCKSIZE>;
        using BlockReduceFloat = hipcub::BlockReduce<float, BLOCKSIZE>;

        __shared__ union {
            typename BlockReduceInt::TempStorage intreduce;
            typename BlockReduceFloat::TempStorage floatreduce;
            GpuMSAProperties msaProperties;
        } temp_storage;

        __shared__ bool sharedHQ;

        auto tbGroup = cg::this_thread_block();

        auto groupReduceFloatSum = [&](float f){
            const float result = BlockReduceFloat(temp_storage.floatreduce).Sum(f);
            __syncthreads();
            return result;
        };

        auto groupReduceFloatMin = [&](float f){
            const float result = BlockReduceFloat(temp_storage.floatreduce).Reduce(f, hipcub::Min{});
            __syncthreads();
            return result;
        };

        auto groupReduceIntMin = [&](int i){
            const int result = BlockReduceInt(temp_storage.intreduce).Reduce(i, hipcub::Min{});
            __syncthreads();
            return result;
        };

        auto groupReduceIntMax = [&](int i){
            const int result = BlockReduceInt(temp_storage.intreduce).Reduce(i, hipcub::Max{});
            __syncthreads();
            return result;
        };

        auto isGoodAvgSupport = [&](float avgsupport){
            return fgeq(avgsupport, avg_support_threshold);
        };
        auto isGoodMinSupport = [&](float minsupport){
            return fgeq(minsupport, min_support_threshold);
        };
        auto isGoodMinCoverage = [&](float mincoverage){
            return fgeq(mincoverage, min_coverage_threshold);
        };

        auto to_nuc = [](std::uint8_t c){
            return SequenceHelpers::decodeBase(c);
        };

        for(unsigned subjectIndex = blockIdx.x; subjectIndex < numAnchors; subjectIndex += gridDim.x){
            const int myNumIndices = d_indices_per_subject[subjectIndex];
            if(myNumIndices > 0){

                const GpuSingleMSA msa = multiMSA.getSingleMSA(subjectIndex);

                char* const my_corrected_subject = correctedSubjects + subjectIndex * decodedSequencePitchInBytes;

                const int subjectColumnsBegin_incl = msa.columnProperties->subjectColumnsBegin_incl;
                const int subjectColumnsEnd_excl = msa.columnProperties->subjectColumnsEnd_excl;
                //const int lastColumn_excl = msa.columnProperties->lastColumn_excl;

                //only first thread in group returns valid properties
                GpuMSAProperties msaProperties = msa.getMSAProperties(
                    tbGroup,
                    groupReduceFloatSum,
                    groupReduceFloatMin,
                    groupReduceIntMin,
                    groupReduceIntMax,
                    subjectColumnsBegin_incl,
                    subjectColumnsEnd_excl
                );

                if(tbGroup.thread_rank() == 0){
                    temp_storage.msaProperties = msaProperties;

                    subjectIsCorrected[subjectIndex] = true; //canBeCorrected;

                    const bool canBeCorrectedByConsensus = isGoodAvgSupport(msaProperties.avg_support) && isGoodMinSupport(msaProperties.min_support) && isGoodMinCoverage(msaProperties.min_coverage);

                    if(canBeCorrectedByConsensus){
                        int smallestErrorrateThatWouldMakeHQ = 100;

                        const int estimatedErrorratePercent = ceil(estimatedErrorrate * 100.0f);
                        for(int percent = estimatedErrorratePercent; percent >= 0; percent--){
                            const float factor = percent / 100.0f;
                            const float avg_threshold = 1.0f - 1.0f * factor;
                            const float min_threshold = 1.0f - 3.0f * factor;
                            if(fgeq(msaProperties.avg_support, avg_threshold) && fgeq(msaProperties.min_support, min_threshold)){
                                smallestErrorrateThatWouldMakeHQ = percent;
                            }
                        }

                        const bool isHQ = isGoodMinCoverage(msaProperties.min_coverage)
                                            && fleq(smallestErrorrateThatWouldMakeHQ, estimatedErrorratePercent * 0.5f);

                        //broadcastbuffer = isHQ;
                        isHighQualitySubject[subjectIndex].hq(isHQ);
                        sharedHQ = isHQ;
                    }else{
                        isHighQualitySubject[subjectIndex].hq(false);
                        sharedHQ = false;
                    }

                }

                tbGroup.sync();

                msaProperties = temp_storage.msaProperties;
                const bool isHQ = sharedHQ;

                const int anchorLength = subjectColumnsEnd_excl - subjectColumnsBegin_incl;
                const unsigned int* const subject = subjectSequencesData + std::size_t(subjectIndex) * encodedSequencePitchInInts;

                //set corrected anchor to consensus
                for(int i = subjectColumnsBegin_incl + tbGroup.thread_rank(); 
                        i < subjectColumnsEnd_excl; 
                        i += tbGroup.size()){

                    const std::uint8_t nuc = msa.consensus[i];
                    //assert(nuc == 'A' || nuc == 'C' || nuc == 'G' || nuc == 'T');
                    assert(0 == nuc || nuc < 4);

                    my_corrected_subject[i - subjectColumnsBegin_incl] = to_nuc(nuc);
                }

                if(!isHQ){
                    //maybe revert some positions to original base                                        

                    for (int i = tbGroup.thread_rank(); i < anchorLength; i += tbGroup.size()){
                        const int msaPos = subjectColumnsBegin_incl + i;
                        const std::uint8_t origEncodedBase = SequenceHelpers::getEncodedNuc2Bit(subject, anchorLength, i);
                        const std::uint8_t consensusEncodedBase = msa.consensus[msaPos];

                        if (origEncodedBase != consensusEncodedBase){          
                            
                            const float countsACGT = msa.coverages[msaPos];
                            const int* const countsA = &msa.counts[0 * msa.columnPitchInElements];
                            const int* const countsC = &msa.counts[1 * msa.columnPitchInElements];
                            const int* const countsG = &msa.counts[2 * msa.columnPitchInElements];
                            const int* const countsT = &msa.counts[3 * msa.columnPitchInElements];

                            const float* const weightsA = &msa.weights[0 * msa.columnPitchInElements];
                            const float* const weightsC = &msa.weights[1 * msa.columnPitchInElements];
                            const float* const weightsG = &msa.weights[2 * msa.columnPitchInElements];
                            const float* const weightsT = &msa.weights[3 * msa.columnPitchInElements];

                            float features[37]{
                                float(origEncodedBase == SequenceHelpers::encodedbaseA()),
                                float(origEncodedBase == SequenceHelpers::encodedbaseC()),
                                float(origEncodedBase == SequenceHelpers::encodedbaseG()),
                                float(origEncodedBase == SequenceHelpers::encodedbaseT()),
                                float(consensusEncodedBase == SequenceHelpers::encodedbaseA()),
                                float(consensusEncodedBase == SequenceHelpers::encodedbaseC()),
                                float(consensusEncodedBase == SequenceHelpers::encodedbaseG()),
                                float(consensusEncodedBase == SequenceHelpers::encodedbaseT()),
                                origEncodedBase == SequenceHelpers::encodedbaseA() ? countsA[msaPos] / countsACGT : 0,
                                origEncodedBase == SequenceHelpers::encodedbaseC() ? countsC[msaPos] / countsACGT : 0,
                                origEncodedBase == SequenceHelpers::encodedbaseG() ? countsG[msaPos] / countsACGT : 0,
                                origEncodedBase == SequenceHelpers::encodedbaseT() ? countsT[msaPos] / countsACGT : 0,
                                origEncodedBase == SequenceHelpers::encodedbaseA() ? weightsA[msaPos]:0,
                                origEncodedBase == SequenceHelpers::encodedbaseC() ? weightsC[msaPos]:0,
                                origEncodedBase == SequenceHelpers::encodedbaseG() ? weightsG[msaPos]:0,
                                origEncodedBase == SequenceHelpers::encodedbaseT() ? weightsT[msaPos]:0,
                                consensusEncodedBase == SequenceHelpers::encodedbaseA() ? countsA[msaPos] / countsACGT : 0,
                                consensusEncodedBase == SequenceHelpers::encodedbaseC() ? countsC[msaPos] / countsACGT : 0,
                                consensusEncodedBase == SequenceHelpers::encodedbaseG() ? countsG[msaPos] / countsACGT : 0,
                                consensusEncodedBase == SequenceHelpers::encodedbaseT() ? countsT[msaPos] / countsACGT : 0,
                                consensusEncodedBase == SequenceHelpers::encodedbaseA() ? weightsA[msaPos]:0,
                                consensusEncodedBase == SequenceHelpers::encodedbaseC() ? weightsC[msaPos]:0,
                                consensusEncodedBase == SequenceHelpers::encodedbaseG() ? weightsG[msaPos]:0,
                                consensusEncodedBase == SequenceHelpers::encodedbaseT() ? weightsT[msaPos]:0,
                                weightsA[msaPos],
                                weightsC[msaPos],
                                weightsG[msaPos],
                                weightsT[msaPos],
                                countsA[msaPos] / countsACGT,
                                countsC[msaPos] / countsACGT,
                                countsG[msaPos] / countsACGT,
                                countsT[msaPos] / countsACGT,
                                msaProperties.avg_support,
                                msaProperties.min_support,
                                float(msaProperties.max_coverage) / estimatedCoverage,
                                float(msaProperties.min_coverage) / estimatedCoverage,
                                float(std::max(subjectColumnsBegin_incl - msaPos, msaPos - subjectColumnsEnd_excl)) / (subjectColumnsEnd_excl-subjectColumnsBegin_incl)
                            };

                            // if(subjectIndex == 2 && i == 95){
                            //     printf("features\n");
                            //     for(int k = 0; k < 37; k++){
                            //         printf("%f\n", features[k]);
                            //     }
                            //     printf("anchorMsaProperties.avg_support %f\n", msaProperties.avg_support);
                            //     printf("anchorMsaProperties.min_support %f\n", msaProperties.min_support);
                            //     printf("anchorMsaProperties.max_coverage %f\n", float(msaProperties.max_coverage));
                            //     printf("anchorMsaProperties.min_coverage %f\n", float(msaProperties.min_coverage));
                            //     printf("a_begin %d\n", subjectColumnsBegin_incl);
                            //     printf("pos %d\n", msaPos);
                            //     printf("a_end %d\n", subjectColumnsEnd_excl);
                            //     printf("estimatedCoverage %f\n", estimatedCoverage);
                            // }

                            const bool useConsensus = gpuForest.decide(&features[0], forestThreshold);
                            if(!useConsensus){
                                my_corrected_subject[i] = to_nuc(origEncodedBase);
                                // if(subjectIndex == 2){
                                //     printf("revert position %d\n", i);
                                // }
                            }else{
                                ; //consensus
                            }
                        }else{
                            ; //consensus
                        }
                    }
                }else{
                    ; //consensus
                }

            }else{
                if(tbGroup.thread_rank() == 0){
                    isHighQualitySubject[subjectIndex].hq(false);
                    subjectIsCorrected[subjectIndex] = false;
                }
            }

            tbGroup.sync();
        }
    }


    void callMsaCorrectAnchorsWithForestKernel(
        char* d_correctedSubjects,
        bool* d_subjectIsCorrected,
        AnchorHighQualityFlag* d_isHighQualitySubject,
        GPUMultiMSA multiMSA,
        GpuForest::Clf gpuForest,
        float forestThreshold,
        const unsigned int* d_subjectSequencesData,
        const int* d_indices_per_subject,
        const int numAnchors,
        int encodedSequencePitchInInts,
        size_t decodedSequencePitchInBytes,
        int maximumSequenceLength,
        float estimatedErrorrate,
        float desiredAlignmentMaxErrorRate,
        float estimatedCoverage,
        float avg_support_threshold,
        float min_support_threshold,
        float min_coverage_threshold,
        float max_coverage_threshold,
        hipStream_t stream,
        KernelLaunchHandle& handle
    ){
        constexpr int blocksize = 128;
        const int numBlocks = numAnchors;

        msaCorrectAnchorsWithForestKernel<blocksize><<<numBlocks, blocksize, 0, stream>>>(
            d_correctedSubjects,
            d_subjectIsCorrected,
            d_isHighQualitySubject,
            multiMSA,
            gpuForest,
            forestThreshold,
            d_subjectSequencesData,
            d_indices_per_subject,
            numAnchors,
            encodedSequencePitchInInts,
            decodedSequencePitchInBytes,
            maximumSequenceLength,
            estimatedErrorrate,
            desiredAlignmentMaxErrorRate,
            estimatedCoverage,
            avg_support_threshold,
            min_support_threshold,
            min_coverage_threshold,
            max_coverage_threshold
        );
    }



    template<int BLOCKSIZE, int groupsize, class GpuClf>
    __global__
    void msaCorrectCandidatesWithForestKernel(
        char* __restrict__ correctedCandidates,
        TempCorrectedSequence::EncodedEdit* __restrict__ d_editsPerCorrectedCandidate,
        int* __restrict__ d_numEditsPerCorrectedCandidate,
        GPUMultiMSA multiMSA,
        GpuClf gpuForest,
        float forestThreshold,
        float estimatedCoverage,
        const int* __restrict__ shifts,
        const BestAlignment_t* __restrict__ bestAlignmentFlags,
        const unsigned int* __restrict__ candidateSequencesData,
        const int* __restrict__ candidateSequencesLengths,
        const bool* __restrict__ d_candidateContainsN,
        const int* __restrict__ candidateIndicesOfCandidatesToBeCorrected,
        const int* __restrict__ numCandidatesToBeCorrected,
        const int* __restrict__ anchorIndicesOfCandidates,
        int doNotUseEditsValue,
        int numEditsThreshold,            
        int encodedSequencePitchInInts,
        size_t decodedSequencePitchInBytes,
        size_t editsPitchInBytes,
        size_t dynamicsmemSequencePitchInInts,
        const read_number* candidateReadIds
    ){

        /*
            Use groupsize threads per candidate to perform correction
        */
        static_assert(BLOCKSIZE % groupsize == 0, "BLOCKSIZE % groupsize != 0");
        constexpr int groupsPerBlock = BLOCKSIZE / groupsize;
        static_assert(groupsize == 32);

        auto tgroup = cg::tiled_partition<groupsize>(cg::this_thread_block());

        auto reverseWithGroupShfl = [](auto& group, char* sequence, int sequenceLength){

            auto reverse = [](char4 data){
                char4 s;
                s.x = data.w;
                s.y = data.z;
                s.z = data.y;
                s.w = data.x;
                return s;
            };
        
            auto shiftLeft1 = [](char4 data){
                char4 s;
                s.x = data.y;
                s.y = data.z;
                s.z = data.w;
                s.w = '\0';
                return s;
            };
        
            auto shiftLeft2 = [](char4 data){
                char4 s;
                s.x = data.z;
                s.y = data.w;
                s.z = '\0';
                s.w = '\0';
                return s;
            };
        
            auto shiftLeft3 = [](char4 data){
                char4 s;
                s.x = data.w;
                s.y = '\0';
                s.z = '\0';
                s.w = '\0';
                return s;
            };
        
            //treat [left,right] as "char8", shift to the left by one char. return leftmost 4 chars
            auto handleUnusedPositions1 = [](char4 left, char4 right){
                char4 s;
                s.x = left.y;
                s.y = left.z;
                s.z = left.w;
                s.w = right.x;
                return s;
            };
        
            //treat [left,right] as "char8", shift to the left by two chars. return leftmost 4 chars
            auto handleUnusedPositions2 = [](char4 left, char4 right){
                char4 s;
                s.x = left.z;
                s.y = left.w;
                s.z = right.x;
                s.w = right.y;
                return s;
            };
        
            //treat [left,right] as "char8", shift to the left by three chars. return leftmost 4 chars
            auto handleUnusedPositions3 = [](char4 left, char4 right){
                char4 s;
                s.x = left.w;
                s.y = right.x;
                s.z = right.y;
                s.w = right.z;
                return s;
            };
        
            if(sequenceLength <= 1) return;
        
            const int arrayLength = SDIV(sequenceLength, 4); // 4 bases per int
            const int unusedPositions = arrayLength * 4 - sequenceLength;
            char4* sequenceAsChar4 = (char4*)sequence;
        
            for(int i = group.thread_rank(); i < arrayLength/2; i += group.size()){
                const char4 fdata = ((char4*)sequence)[i];
                const char4 bdata = ((char4*)sequence)[arrayLength - 1 - i];
        
                const char4 front = reverse(fdata);
                const char4 back = reverse(bdata);
                sequenceAsChar4[i] = back;
                sequenceAsChar4[arrayLength - 1 - i] = front;
            }
        
            if(arrayLength % 2 == 1 && group.thread_rank() == 0){
                const int middleindex = arrayLength/2;
                const char4 mdata = ((char4*)sequence)[middleindex];
                sequenceAsChar4[middleindex] = reverse(mdata);
            }
        
            group.sync();
        
            if(unusedPositions > 0){
        
                char4 left;
                char4 right;
                char4 tmp;
        
                const int numIterations = SDIV(arrayLength-1, group.size());
        
                for(int iteration = 0; iteration < numIterations; iteration++){
                    const int index = iteration * group.size() + group.thread_rank();
                    if(index < arrayLength){
                        left = sequenceAsChar4[index];
                    }
                    const int index2 = (iteration+1) * group.size() + group.thread_rank();
                    if(index2 < arrayLength && group.thread_rank() == 0){
                        tmp = sequenceAsChar4[index2];
                    }
                    #if __CUDACC_VER_MAJOR__ < 11
                    //CUDA < 11 does not have shuffle api for char4
                    *((int*)(&right)) = group.shfl_down(*((const int*)(&left)), 1);
                    *((int*)(&tmp)) = group.shfl(*((const int*)(&tmp)), 0);
                    #else
                    right = group.shfl_down(left, 1);
                    tmp = group.shfl(tmp, 0);
                    #endif
                    if(group.thread_rank() == group.size() - 1){
                        right = tmp;
                    }
        
                    if(unusedPositions == 1){
                        char4 result = handleUnusedPositions1(left, right);
                        if(index < arrayLength - 1){
                            sequenceAsChar4[index] = result;
                        }
                    }else if(unusedPositions == 2){
                        char4 result = handleUnusedPositions2(left, right);
                        if(index < arrayLength - 1){
                            sequenceAsChar4[index] = result;
                        }
                    }else{
                        char4 result = handleUnusedPositions3(left, right);
                        if(index < arrayLength - 1){
                            sequenceAsChar4[index] = result;
                        }
                    }
                }
        
                group.sync();
        
                if(group.thread_rank() == 0){
                    if(unusedPositions == 1){
                        sequenceAsChar4[arrayLength-1] = shiftLeft1(sequenceAsChar4[arrayLength-1]);
                    }else if(unusedPositions == 2){
                        sequenceAsChar4[arrayLength-1] = shiftLeft2(sequenceAsChar4[arrayLength-1]);
                    }else{
                        assert(unusedPositions == 3);
                        sequenceAsChar4[arrayLength-1] = shiftLeft3(sequenceAsChar4[arrayLength-1]);
                    }
                }
            }
        };

        auto to_nuc = [](std::uint8_t c){
            return SequenceHelpers::decodeBase(c);
        };

        using WarpReduceInt = hipcub::WarpReduce<int>;
        using WarpReduceFloat = hipcub::WarpReduce<float>;

        __shared__ typename WarpReduceInt::TempStorage intreduce[groupsPerBlock];
        __shared__ typename WarpReduceFloat::TempStorage floatreduce[groupsPerBlock];
        __shared__ GpuMSAProperties sharedMsaProperties[groupsPerBlock];
        __shared__ float sharedFeatures[groupsPerBlock][42];
       

        __shared__ int shared_numEditsOfCandidate[groupsPerBlock];

        extern __shared__ int dynamicsmem[]; // for sequences


        const int numGroups = (gridDim.x * blockDim.x) / groupsize;
        const int groupId = (threadIdx.x + blockIdx.x * blockDim.x) / groupsize;
        const int groupIdInBlock = threadIdx.x / groupsize;

        auto groupReduceFloatSum = [&](float f){
            const float result = WarpReduceFloat(floatreduce[groupIdInBlock]).Sum(f);
            tgroup.sync();
            return result;
        };

        auto groupReduceFloatMin = [&](float f){
            const float result = WarpReduceFloat(floatreduce[groupIdInBlock]).Reduce(f, hipcub::Min{});
            tgroup.sync();
            return result;
        };

        auto groupReduceIntMin = [&](int i){
            const int result = WarpReduceInt(intreduce[groupIdInBlock]).Reduce(i, hipcub::Min{});
            tgroup.sync();
            return result;
        };

        auto groupReduceIntMax = [&](int i){
            const int result = WarpReduceInt(intreduce[groupIdInBlock]).Reduce(i, hipcub::Max{});
            tgroup.sync();
            return result;
        };

        const std::size_t smemPitchEditsInInts = SDIV(editsPitchInBytes, sizeof(int));

        char* const shared_correctedCandidate = (char*)(dynamicsmem + dynamicsmemSequencePitchInInts * groupIdInBlock);



        TempCorrectedSequence::EncodedEdit* const shared_Edits 
            = (TempCorrectedSequence::EncodedEdit*)((dynamicsmem + dynamicsmemSequencePitchInInts * groupsPerBlock) 
                + smemPitchEditsInInts * groupIdInBlock);

        const int loopEnd = *numCandidatesToBeCorrected;

        for(int id = groupId;
                id < loopEnd;
                id += numGroups){

            const int candidateIndex = candidateIndicesOfCandidatesToBeCorrected[id];
            const int subjectIndex = anchorIndicesOfCandidates[candidateIndex];
            const int destinationIndex = id;

            const GpuSingleMSA msa = multiMSA.getSingleMSA(subjectIndex);

            char* const my_corrected_candidate = correctedCandidates + destinationIndex * decodedSequencePitchInBytes;
            const int candidate_length = candidateSequencesLengths[candidateIndex];

            const int shift = shifts[candidateIndex];
            const int subjectColumnsBegin_incl = msa.columnProperties->subjectColumnsBegin_incl;
            const int subjectColumnsEnd_excl = msa.columnProperties->subjectColumnsEnd_excl;
            const int queryColumnsBegin_incl = subjectColumnsBegin_incl + shift;
            const int queryColumnsEnd_excl = subjectColumnsBegin_incl + shift + candidate_length;

            //const int candidateReadId = candidateReadIds[candidateIndex];

            //only first thread in group returns valid properties
            GpuMSAProperties msaProperties = msa.getMSAProperties(
                tgroup,
                groupReduceFloatSum,
                groupReduceFloatMin,
                groupReduceIntMin,
                groupReduceIntMax,
                queryColumnsBegin_incl,
                queryColumnsEnd_excl
            );

            if(tgroup.thread_rank() == 0){                        
                shared_numEditsOfCandidate[groupIdInBlock] = 0;
                sharedMsaProperties[groupIdInBlock] = msaProperties;
            }
            tgroup.sync(); 

            msaProperties = sharedMsaProperties[groupIdInBlock];

            const int copyposbegin = queryColumnsBegin_incl;
            const int copyposend = queryColumnsEnd_excl;
            assert(copyposend - copyposbegin == candidate_length);
            
            for(int i = copyposbegin + tgroup.thread_rank(); i < copyposend; i += tgroup.size()) {
                shared_correctedCandidate[i - queryColumnsBegin_incl] = to_nuc(msa.consensus[i]);
            }
            tgroup.sync();

            const BestAlignment_t bestAlignmentFlag = bestAlignmentFlags[candidateIndex];

            const unsigned int* const encUncorrectedCandidate = candidateSequencesData 
                        + std::size_t(candidateIndex) * encodedSequencePitchInInts;

            // if(candidateReadId == 38851){
            //     if(tgroup.thread_rank() == 0){
            //         printf("decodedCandidate:\n");
            //         for(int i = 0; i < candidate_length; i++){
            //             std::uint8_t origEncodedBase = 0;

            //             if(bestAlignmentFlag == BestAlignment_t::ReverseComplement){
            //                 origEncodedBase = SequenceHelpers::getEncodedNuc2Bit(
            //                     encUncorrectedCandidate,
            //                     candidate_length,
            //                     candidate_length - i - 1
            //                 );
            //                 origEncodedBase = SequenceHelpers::complementBase2Bit(origEncodedBase);
            //             }else{
            //                 origEncodedBase = SequenceHelpers::getEncodedNuc2Bit(
            //                     encUncorrectedCandidate,
            //                     candidate_length,
            //                     i
            //                 );
            //             }

            //             const char origBase = to_nuc(origEncodedBase);

            //             printf("%c", origBase);
            //         }
            //         printf("\n");

            //         printf("consensusCandidate:\n");
            //         for(int i = 0; i < candidate_length; i++){
            //             printf("%c", shared_correctedCandidate[i]);
            //         }
            //         printf("\n");
            //     }

            //     tgroup.sync();
            // }

            #if 1 // 1: use full group per position, 0: use 1 thread per position

            for(int i = 0; i < candidate_length; i += 1){
                std::uint8_t origEncodedBase = 0;

                if(bestAlignmentFlag == BestAlignment_t::ReverseComplement){
                    origEncodedBase = SequenceHelpers::getEncodedNuc2Bit(
                        encUncorrectedCandidate,
                        candidate_length,
                        candidate_length - i - 1
                    );
                    origEncodedBase = SequenceHelpers::complementBase2Bit(origEncodedBase);
                }else{
                    origEncodedBase = SequenceHelpers::getEncodedNuc2Bit(
                        encUncorrectedCandidate,
                        candidate_length,
                        i
                    );
                }

                const char origBase = to_nuc(origEncodedBase);
                const char consensusBase = shared_correctedCandidate[i];
                if(origBase != consensusBase){

                    const int* const countsA = &msa.counts[0 * msa.columnPitchInElements];
                    const int* const countsC = &msa.counts[1 * msa.columnPitchInElements];
                    const int* const countsG = &msa.counts[2 * msa.columnPitchInElements];
                    const int* const countsT = &msa.counts[3 * msa.columnPitchInElements];

                    const float* const weightsA = &msa.weights[0 * msa.columnPitchInElements];
                    const float* const weightsC = &msa.weights[1 * msa.columnPitchInElements];
                    const float* const weightsG = &msa.weights[2 * msa.columnPitchInElements];
                    const float* const weightsT = &msa.weights[3 * msa.columnPitchInElements];

                    const int a_begin = subjectColumnsBegin_incl;
                    const int a_end = subjectColumnsEnd_excl;
                    const int c_begin = queryColumnsBegin_incl;
                    const int c_end = queryColumnsEnd_excl;
                    const int msaPos = c_begin + i;
                    const char orig = origBase;
                    const float countsACGT = msa.coverages[msaPos];

                    if(tgroup.thread_rank() == 0){
                        sharedFeatures[groupIdInBlock][0] = float(orig == 'A');
                        sharedFeatures[groupIdInBlock][1] = float(orig == 'C');
                        sharedFeatures[groupIdInBlock][2] = float(orig == 'G');
                        sharedFeatures[groupIdInBlock][3] = float(orig == 'T');
                        sharedFeatures[groupIdInBlock][4] = float(consensusBase == 'A');
                        sharedFeatures[groupIdInBlock][5] = float(consensusBase == 'C');
                        sharedFeatures[groupIdInBlock][6] = float(consensusBase == 'G');
                        sharedFeatures[groupIdInBlock][7] = float(consensusBase == 'T');
                        sharedFeatures[groupIdInBlock][8] = orig == 'A'? countsA[msaPos] / countsACGT : 0;
                        sharedFeatures[groupIdInBlock][9] = orig == 'C'? countsC[msaPos] / countsACGT : 0;
                        sharedFeatures[groupIdInBlock][10] = orig == 'G'? countsG[msaPos] / countsACGT : 0;
                        sharedFeatures[groupIdInBlock][11] = orig == 'T'? countsT[msaPos] / countsACGT : 0;
                        sharedFeatures[groupIdInBlock][12] = orig == 'A'? weightsA[msaPos]:0;
                        sharedFeatures[groupIdInBlock][13] = orig == 'C'? weightsC[msaPos]:0;
                        sharedFeatures[groupIdInBlock][14] = orig == 'G'? weightsG[msaPos]:0;
                        sharedFeatures[groupIdInBlock][15] = orig == 'T'? weightsT[msaPos]:0;
                        sharedFeatures[groupIdInBlock][16] = consensusBase == 'A'? countsA[msaPos] / countsACGT : 0;
                        sharedFeatures[groupIdInBlock][17] = consensusBase == 'C'? countsC[msaPos] / countsACGT : 0;
                        sharedFeatures[groupIdInBlock][18] = consensusBase == 'G'? countsG[msaPos] / countsACGT : 0;
                        sharedFeatures[groupIdInBlock][19] = consensusBase == 'T'? countsT[msaPos] / countsACGT : 0;
                        sharedFeatures[groupIdInBlock][20] = consensusBase == 'A'? weightsA[msaPos]:0;
                        sharedFeatures[groupIdInBlock][21] = consensusBase == 'C'? weightsC[msaPos]:0;
                        sharedFeatures[groupIdInBlock][22] = consensusBase == 'G'? weightsG[msaPos]:0;
                        sharedFeatures[groupIdInBlock][23] = consensusBase == 'T'? weightsT[msaPos]:0;
                        sharedFeatures[groupIdInBlock][24] = weightsA[msaPos];
                        sharedFeatures[groupIdInBlock][25] = weightsC[msaPos];
                        sharedFeatures[groupIdInBlock][26] = weightsG[msaPos];
                        sharedFeatures[groupIdInBlock][27] = weightsT[msaPos];
                        sharedFeatures[groupIdInBlock][28] = countsA[msaPos] / countsACGT;
                        sharedFeatures[groupIdInBlock][29] = countsC[msaPos] / countsACGT;
                        sharedFeatures[groupIdInBlock][30] = countsG[msaPos] / countsACGT;
                        sharedFeatures[groupIdInBlock][31] = countsT[msaPos] / countsACGT;
                        sharedFeatures[groupIdInBlock][32] = msaProperties.avg_support;
                        sharedFeatures[groupIdInBlock][33] = msaProperties.min_support;
                        sharedFeatures[groupIdInBlock][34] = float(msaProperties.max_coverage)/estimatedCoverage;
                        sharedFeatures[groupIdInBlock][35] = float(msaProperties.min_coverage)/estimatedCoverage;
                        sharedFeatures[groupIdInBlock][36] = float(std::max(std::abs(c_begin-a_begin), std::abs(a_end-c_end)))/(c_end-c_begin); // absolute shift (compatible with differing read lengths)
                        sharedFeatures[groupIdInBlock][37] = float(std::max(std::abs(c_begin-a_begin), std::abs(a_end-c_end)))/(a_end-a_begin);
                        sharedFeatures[groupIdInBlock][38] = float(std::min(a_end, c_end)-std::max(a_begin, c_begin))/(a_end-a_begin); // relative overlap (ratio of a or c length in case of diff. read len)
                        sharedFeatures[groupIdInBlock][39] = float(std::min(a_end, c_end)-std::max(a_begin, c_begin))/(c_end-c_begin);
                        sharedFeatures[groupIdInBlock][40] = float(std::max(a_begin-msaPos, msaPos-a_end))/(a_end-a_begin);
                        sharedFeatures[groupIdInBlock][41] = float(std::max(a_begin-msaPos, msaPos-a_end))/(c_end-c_begin);
                    }

                    tgroup.sync();

                    // float features[42] {
                    //     float(orig == 'A'),
                    //     float(orig == 'C'),
                    //     float(orig == 'G'),
                    //     float(orig == 'T'),
                    //     float(consensusBase == 'A'),
                    //     float(consensusBase == 'C'),
                    //     float(consensusBase == 'G'),
                    //     float(consensusBase == 'T'),
                    //     orig == 'A'? countsA[msaPos] / countsACGT : 0,
                    //     orig == 'C'? countsC[msaPos] / countsACGT : 0,
                    //     orig == 'G'? countsG[msaPos] / countsACGT : 0,
                    //     orig == 'T'? countsT[msaPos] / countsACGT : 0,
                    //     orig == 'A'? weightsA[msaPos]:0,
                    //     orig == 'C'? weightsC[msaPos]:0,
                    //     orig == 'G'? weightsG[msaPos]:0,
                    //     orig == 'T'? weightsT[msaPos]:0,
                    //     consensusBase == 'A'? countsA[msaPos] / countsACGT : 0,
                    //     consensusBase == 'C'? countsC[msaPos] / countsACGT : 0,
                    //     consensusBase == 'G'? countsG[msaPos] / countsACGT : 0,
                    //     consensusBase == 'T'? countsT[msaPos] / countsACGT : 0,
                    //     consensusBase == 'A'? weightsA[msaPos]:0,
                    //     consensusBase == 'C'? weightsC[msaPos]:0,
                    //     consensusBase == 'G'? weightsG[msaPos]:0,
                    //     consensusBase == 'T'? weightsT[msaPos]:0,
                    //     weightsA[msaPos],
                    //     weightsC[msaPos],
                    //     weightsG[msaPos],
                    //     weightsT[msaPos],
                    //     countsA[msaPos] / countsACGT,
                    //     countsC[msaPos] / countsACGT,
                    //     countsG[msaPos] / countsACGT,
                    //     countsT[msaPos] / countsACGT,
                    //     msaProperties.avg_support,
                    //     msaProperties.min_support,
                    //     float(msaProperties.max_coverage)/estimatedCoverage,
                    //     float(msaProperties.min_coverage)/estimatedCoverage,
                    //     float(std::max(std::abs(c_begin-a_begin), std::abs(a_end-c_end)))/(c_end-c_begin), // absolute shift (compatible with differing read lengths)
                    //     float(std::max(std::abs(c_begin-a_begin), std::abs(a_end-c_end)))/(a_end-a_begin),
                    //     float(std::min(a_end, c_end)-std::max(a_begin, c_begin))/(a_end-a_begin), // relative overlap (ratio of a or c length in case of diff. read len)
                    //     float(std::min(a_end, c_end)-std::max(a_begin, c_begin))/(c_end-c_begin),
                    //     float(std::max(a_begin-msaPos, msaPos-a_end))/(a_end-a_begin),
                    //     float(std::max(a_begin-msaPos, msaPos-a_end))/(c_end-c_begin)
                    // };

                    //only thread 0 has valid result                   
                    const bool useConsensus = gpuForest.decide(tgroup, &sharedFeatures[groupIdInBlock][0], forestThreshold, groupReduceFloatSum);
                    //const bool useConsensus = gpuForest.decide(tgroup, &features[0], forestThreshold, groupReduceFloatSum);

                    if(tgroup.thread_rank() == 0){
                        if(!useConsensus){
                            shared_correctedCandidate[i] = origBase;
                        }else{
                            ; //keep consensus
                        }
                    }
                }
            }
            #else

            for(int i = tgroup.thread_rank(); i < candidate_length; i += tgroup.size()){
                std::uint8_t origEncodedBase = 0;

                if(bestAlignmentFlag == BestAlignment_t::ReverseComplement){
                    origEncodedBase = SequenceHelpers::getEncodedNuc2Bit(
                        encUncorrectedCandidate,
                        candidate_length,
                        candidate_length - i - 1
                    );
                    origEncodedBase = SequenceHelpers::complementBase2Bit(origEncodedBase);
                }else{
                    origEncodedBase = SequenceHelpers::getEncodedNuc2Bit(
                        encUncorrectedCandidate,
                        candidate_length,
                        i
                    );
                }

                const char origBase = to_nuc(origEncodedBase);
                const char consensusBase = shared_correctedCandidate[i];
                if(origBase != consensusBase){

                    const int* const countsA = &msa.counts[0 * msa.columnPitchInElements];
                    const int* const countsC = &msa.counts[1 * msa.columnPitchInElements];
                    const int* const countsG = &msa.counts[2 * msa.columnPitchInElements];
                    const int* const countsT = &msa.counts[3 * msa.columnPitchInElements];

                    const float* const weightsA = &msa.weights[0 * msa.columnPitchInElements];
                    const float* const weightsC = &msa.weights[1 * msa.columnPitchInElements];
                    const float* const weightsG = &msa.weights[2 * msa.columnPitchInElements];
                    const float* const weightsT = &msa.weights[3 * msa.columnPitchInElements];

                    const int a_begin = subjectColumnsBegin_incl;
                    const int a_end = subjectColumnsEnd_excl;
                    const int c_begin = queryColumnsBegin_incl;
                    const int c_end = queryColumnsEnd_excl;
                    const int msaPos = c_begin + i;
                    const char orig = origBase;
                    const float countsACGT = msa.coverages[msaPos];

                    float features[42] {
                        float(orig == 'A'),
                        float(orig == 'C'),
                        float(orig == 'G'),
                        float(orig == 'T'),
                        float(consensusBase == 'A'),
                        float(consensusBase == 'C'),
                        float(consensusBase == 'G'),
                        float(consensusBase == 'T'),
                        orig == 'A'? countsA[msaPos] / countsACGT : 0,
                        orig == 'C'? countsC[msaPos] / countsACGT : 0,
                        orig == 'G'? countsG[msaPos] / countsACGT : 0,
                        orig == 'T'? countsT[msaPos] / countsACGT : 0,
                        orig == 'A'? weightsA[msaPos]:0,
                        orig == 'C'? weightsC[msaPos]:0,
                        orig == 'G'? weightsG[msaPos]:0,
                        orig == 'T'? weightsT[msaPos]:0,
                        consensusBase == 'A'? countsA[msaPos] / countsACGT : 0,
                        consensusBase == 'C'? countsC[msaPos] / countsACGT : 0,
                        consensusBase == 'G'? countsG[msaPos] / countsACGT : 0,
                        consensusBase == 'T'? countsT[msaPos] / countsACGT : 0,
                        consensusBase == 'A'? weightsA[msaPos]:0,
                        consensusBase == 'C'? weightsC[msaPos]:0,
                        consensusBase == 'G'? weightsG[msaPos]:0,
                        consensusBase == 'T'? weightsT[msaPos]:0,
                        weightsA[msaPos],
                        weightsC[msaPos],
                        weightsG[msaPos],
                        weightsT[msaPos],
                        countsA[msaPos] / countsACGT,
                        countsC[msaPos] / countsACGT,
                        countsG[msaPos] / countsACGT,
                        countsT[msaPos] / countsACGT,
                        msaProperties.avg_support,
                        msaProperties.min_support,
                        float(msaProperties.max_coverage)/estimatedCoverage,
                        float(msaProperties.min_coverage)/estimatedCoverage,
                        float(std::max(std::abs(c_begin-a_begin), std::abs(a_end-c_end)))/(c_end-c_begin), // absolute shift (compatible with differing read lengths)
                        float(std::max(std::abs(c_begin-a_begin), std::abs(a_end-c_end)))/(a_end-a_begin),
                        float(std::min(a_end, c_end)-std::max(a_begin, c_begin))/(a_end-a_begin), // relative overlap (ratio of a or c length in case of diff. read len)
                        float(std::min(a_end, c_end)-std::max(a_begin, c_begin))/(c_end-c_begin),
                        float(std::max(a_begin-msaPos, msaPos-a_end))/(a_end-a_begin),
                        float(std::max(a_begin-msaPos, msaPos-a_end))/(c_end-c_begin)
                    };

                    // if(candidateReadId == 38851 && i == 5){
                    //     printf("features 38851,5\n");
                    //     for(int k = 0; k < 42; k++){
                    //         printf("%f\n", features[k]);
                    //     }
                    //     printf("msaProperties.avg_support %f\n", msaProperties.avg_support);
                    //     printf("msaProperties.min_support %f\n", msaProperties.min_support);
                    //     printf("msaProperties.max_coverage %f\n", float(msaProperties.max_coverage));
                    //     printf("msaProperties.min_coverage %f\n", float(msaProperties.min_coverage));
                    // }


                    const bool useConsensus = gpuForest.decide(&features[0], forestThreshold);

                    if(!useConsensus){
                        shared_correctedCandidate[i] = origBase;

                        // if(candidateReadId == 38851){
                        //     printf("position %d revert consensus\n", i);
                        // }
                    }else{
                        // if(candidateReadId == 38851){
                        //     printf("position %d keep consensus\n", i);
                        // }
                    }
                }
            }

            #endif

            tgroup.sync();

            //the forward strand will be returned -> make reverse complement again
            if(bestAlignmentFlag == BestAlignment_t::ReverseComplement) {
                for(int i = tgroup.thread_rank(); i < candidate_length; i += tgroup.size()) {
                    shared_correctedCandidate[i] = SequenceHelpers::reverseComplementBaseDecoded(shared_correctedCandidate[i]);
                }
                tgroup.sync(); // threads may access elements in shared memory which were written by another thread
                reverseWithGroupShfl(tgroup, shared_correctedCandidate, candidate_length);
                tgroup.sync();
            }else{
                ; //orientation ok
            }

            // if(candidateReadId == 38851){
            //     if(tgroup.thread_rank() == 0){
            //         printf("correctedCandidate:\n");
            //         for(int i = 0; i < candidate_length; i++){
            //             printf("%c", shared_correctedCandidate[i]);
            //         }
            //         printf("\n");
            //     }

            //     tgroup.sync();
            // }
            
            //copy corrected sequence from smem to global output
            const int fullInts1 = candidate_length / sizeof(int);

            for(int i = tgroup.thread_rank(); i < fullInts1; i += tgroup.size()) {
                ((int*)my_corrected_candidate)[i] = ((int*)shared_correctedCandidate)[i];
            }

            for(int i = tgroup.thread_rank(); i < candidate_length - fullInts1 * sizeof(int); i += tgroup.size()) {
                my_corrected_candidate[fullInts1 * sizeof(int) + i] 
                    = shared_correctedCandidate[fullInts1 * sizeof(int) + i];
            }       

            //compare corrected candidate with uncorrected candidate, calculate edits   
            
            
            const bool thisSequenceContainsN = d_candidateContainsN[candidateIndex];            

            if(thisSequenceContainsN){
                if(tgroup.thread_rank() == 0){
                    d_numEditsPerCorrectedCandidate[destinationIndex] = doNotUseEditsValue;
                }
            }else{
                const int maxEdits = min(candidate_length / 7, numEditsThreshold);

                auto countAndSaveEditInSmem = [&](const int posInSequence, const char correctedNuc){
                    cg::coalesced_group g = cg::coalesced_threads();
                                
                    int currentNumEdits = 0;
                    if(g.thread_rank() == 0){
                        currentNumEdits = atomicAdd(&shared_numEditsOfCandidate[groupIdInBlock], g.size());
                    }
                    currentNumEdits = g.shfl(currentNumEdits, 0);
    
                    if(currentNumEdits + g.size() <= maxEdits){
                        const int myEditOutputPos = g.thread_rank() + currentNumEdits;
                        if(myEditOutputPos < maxEdits){
                            const auto theEdit = TempCorrectedSequence::EncodedEdit{posInSequence, correctedNuc};
                            //myEdits[myEditOutputPos] = theEdit;
                            //shared_Edits[groupIdInBlock][myEditOutputPos] = theEdit;
                            shared_Edits[myEditOutputPos] = theEdit;
                        }
                    }
                };

                auto countAndSaveEditInSmem2 = [&](const int posInSequence, const char correctedNuc){
                    const int groupsPerWarp = 32 / tgroup.size();
                    if(groupsPerWarp == 1){
                        countAndSaveEditInSmem(posInSequence, correctedNuc);
                    }else{
                        const int groupIdInWarp = (threadIdx.x % 32) / tgroup.size();
                        unsigned int subwarpmask = ((1u << (tgroup.size() - 1)) | ((1u << (tgroup.size() - 1)) - 1));
                        subwarpmask <<= (tgroup.size() * groupIdInWarp);

                        unsigned int lanemask_lt;
                        asm volatile("mov.u32 %0, %%lanemask_lt;" : "=r"(lanemask_lt));
                        const unsigned int writemask = subwarpmask & __activemask();
                        const unsigned int total = __popc(writemask);
                        const unsigned int prefix = __popc(writemask & lanemask_lt);

                        const int elected_lane = __ffs(writemask) - 1;
                        int currentNumEdits = 0;
                        if (prefix == 0) {
                            currentNumEdits = atomicAdd(&shared_numEditsOfCandidate[groupIdInBlock], total);
                        }
                        currentNumEdits = __shfl_sync(writemask, currentNumEdits, elected_lane);

                        if(currentNumEdits + total <= maxEdits){
                            const int myEditOutputPos = prefix + currentNumEdits;
                            if(myEditOutputPos < maxEdits){
                                const auto theEdit = TempCorrectedSequence::EncodedEdit{posInSequence, correctedNuc};
                                //myEdits[myEditOutputPos] = theEdit;
                                //shared_Edits[groupIdInBlock][myEditOutputPos] = theEdit;
                                shared_Edits[myEditOutputPos] = theEdit;
                            }
                        }

                    }
                };

                constexpr int basesPerInt = SequenceHelpers::basesPerInt2Bit();
                const int fullInts = candidate_length / basesPerInt;   
                
                for(int i = 0; i < fullInts; i++){
                    const unsigned int encodedDataInt = encUncorrectedCandidate[i];

                    //compare with basesPerInt bases of corrected sequence

                    for(int k = tgroup.thread_rank(); k < basesPerInt; k += tgroup.size()){
                        const int posInInt = k;
                        const int posInSequence = i * basesPerInt + posInInt;
                        const std::uint8_t encodedUncorrectedNuc = SequenceHelpers::getEncodedNucFromInt2Bit(encodedDataInt, posInInt);
                        const char correctedNuc = shared_correctedCandidate[posInSequence];

                        if(correctedNuc != to_nuc(encodedUncorrectedNuc)){
                            countAndSaveEditInSmem2(posInSequence, correctedNuc);
                        }
                    }

                    tgroup.sync();

                    if(shared_numEditsOfCandidate[groupIdInBlock] > maxEdits){
                        break;
                    }
                }

                //process remaining positions
                if(shared_numEditsOfCandidate[groupIdInBlock] <= maxEdits){
                    const int remainingPositions = candidate_length - basesPerInt * fullInts;

                    if(remainingPositions > 0){
                        const unsigned int encodedDataInt = encUncorrectedCandidate[fullInts];
                        for(int posInInt = tgroup.thread_rank(); posInInt < remainingPositions; posInInt += tgroup.size()){
                            const int posInSequence = fullInts * basesPerInt + posInInt;
                            const std::uint8_t encodedUncorrectedNuc = SequenceHelpers::getEncodedNucFromInt2Bit(encodedDataInt, posInInt);
                            const char correctedNuc = shared_correctedCandidate[posInSequence];

                            if(correctedNuc != to_nuc(encodedUncorrectedNuc)){
                                countAndSaveEditInSmem2(posInSequence, correctedNuc);
                            }
                        }
                    }
                }

                tgroup.sync();

                int* const myNumEdits = d_numEditsPerCorrectedCandidate + destinationIndex;

                TempCorrectedSequence::EncodedEdit* const myEdits 
                    = (TempCorrectedSequence::EncodedEdit*)(((char*)d_editsPerCorrectedCandidate) + destinationIndex * editsPitchInBytes);

                if(shared_numEditsOfCandidate[groupIdInBlock] <= maxEdits){
                    const int numEdits = shared_numEditsOfCandidate[groupIdInBlock];

                    if(tgroup.thread_rank() == 0){ 
                        *myNumEdits = numEdits;
                    }

                    const int fullInts = (numEdits * sizeof(TempCorrectedSequence::EncodedEdit)) / sizeof(int);
                    static_assert(sizeof(TempCorrectedSequence::EncodedEdit) * 2 == sizeof(int), "");

                    for(int i = tgroup.thread_rank(); i < fullInts; i += tgroup.size()) {
                        ((int*)myEdits)[i] = ((int*)shared_Edits)[i];
                    }

                    for(int i = tgroup.thread_rank(); i < numEdits - fullInts * 2; i += tgroup.size()) {
                        myEdits[fullInts * 2 + i] = shared_Edits[fullInts * 2 + i];
                    } 
                }else{
                    if(tgroup.thread_rank() == 0){
                        *myNumEdits = doNotUseEditsValue;
                    }
                }

            }
            

            tgroup.sync(); //sync before handling next candidate
                        
        }
    }



    void callMsaCorrectCandidatesWithForestKernel(
        char* d_correctedCandidates,
        TempCorrectedSequence::EncodedEdit* d_editsPerCorrectedCandidate,
        int* d_numEditsPerCorrectedCandidate,
        GPUMultiMSA multiMSA,
        GpuForest::Clf gpuForest,
        float forestThreshold,
        float estimatedCoverage,
        const int* d_shifts,
        const BestAlignment_t* d_bestAlignmentFlags,
        const unsigned int* d_candidateSequencesData,
        const int* d_candidateSequencesLengths,
        const bool* d_candidateContainsN,
        const int* d_candidateIndicesOfCandidatesToBeCorrected,
        const int* d_numCandidatesToBeCorrected,
        const int* d_anchorIndicesOfCandidates,
        const int numCandidates,
        int doNotUseEditsValue,
        int numEditsThreshold,            
        int encodedSequencePitchInInts,
        size_t decodedSequencePitchInBytes,
        size_t editsPitchInBytes,
        int maximum_sequence_length,
        hipStream_t stream,
        KernelLaunchHandle& handle,
        const read_number* candidateReadIds
    ){

        constexpr int blocksize = 128;
        constexpr int groupsize = 32;

        const size_t dynamicsmemPitchInInts = SDIV(maximum_sequence_length, sizeof(int));
        const size_t smemPitchEditsInInts = SDIV(editsPitchInBytes, sizeof(int));

        auto calculateSmemUsage = [&](int blockDim){
            const int numGroupsPerBlock = blockDim / groupsize;
            std::size_t smem = numGroupsPerBlock * (sizeof(int) * dynamicsmemPitchInInts)
                + numGroupsPerBlock * (sizeof(int) * smemPitchEditsInInts);

            return smem;
        };

        const std::size_t smem = calculateSmemUsage(blocksize);

        dim3 block = blocksize;
        dim3 grid = 270;


        msaCorrectCandidatesWithForestKernel<blocksize, groupsize><<<grid, block, smem, stream>>>(
            d_correctedCandidates,
            d_editsPerCorrectedCandidate,
            d_numEditsPerCorrectedCandidate,
            multiMSA,
            gpuForest,
            forestThreshold,
            estimatedCoverage,
            d_shifts,
            d_bestAlignmentFlags,
            d_candidateSequencesData,
            d_candidateSequencesLengths,
            d_candidateContainsN,
            d_candidateIndicesOfCandidatesToBeCorrected,
            d_numCandidatesToBeCorrected,
            d_anchorIndicesOfCandidates,            
            doNotUseEditsValue,
            numEditsThreshold,            
            encodedSequencePitchInInts,
            decodedSequencePitchInBytes,
            editsPitchInBytes,
            dynamicsmemPitchInInts,
            candidateReadIds
        );
    }








    __device__ __forceinline__
    bool checkIfCandidateShouldBeCorrectedGlobal(
        const GpuSingleMSA msa,
        const int alignmentShift,
        const int candidateLength,
        float min_support_threshold,
        float min_coverage_threshold,
        int new_columns_to_correct
    ){

        const auto columnProperties = *msa.columnProperties;

        const int& subjectColumnsBegin_incl = columnProperties.subjectColumnsBegin_incl;
        const int& subjectColumnsEnd_excl = columnProperties.subjectColumnsEnd_excl;
        const int& lastColumn_excl = columnProperties.lastColumn_excl;

        const int shift = alignmentShift;
        const int candidate_length = candidateLength;
        const int queryColumnsBegin_incl = subjectColumnsBegin_incl + shift;
        const int queryColumnsEnd_excl = subjectColumnsBegin_incl + shift + candidate_length;

        if(subjectColumnsBegin_incl - new_columns_to_correct <= queryColumnsBegin_incl
           && queryColumnsBegin_incl <= subjectColumnsBegin_incl + new_columns_to_correct
           && queryColumnsEnd_excl <= subjectColumnsEnd_excl + new_columns_to_correct) {

            float newColMinSupport = 1.0f;
            int newColMinCov = std::numeric_limits<int>::max();
            //check new columns left of subject
            for(int columnindex = subjectColumnsBegin_incl - new_columns_to_correct;
                columnindex < subjectColumnsBegin_incl;
                columnindex++) {

                assert(columnindex < lastColumn_excl);
                if(queryColumnsBegin_incl <= columnindex) {
                    newColMinSupport = msa.support[columnindex] < newColMinSupport ? msa.support[columnindex] : newColMinSupport;
                    newColMinCov = msa.coverages[columnindex] < newColMinCov ? msa.coverages[columnindex] : newColMinCov;
                }
            }
            //check new columns right of subject
            for(int columnindex = subjectColumnsEnd_excl;
                    columnindex < subjectColumnsEnd_excl + new_columns_to_correct
                        && columnindex < lastColumn_excl;
                    columnindex++) {

                newColMinSupport = msa.support[columnindex] < newColMinSupport ? msa.support[columnindex] : newColMinSupport;
                newColMinCov = msa.coverages[columnindex] < newColMinCov ? msa.coverages[columnindex] : newColMinCov;
            }

            bool result = fgeq(newColMinSupport, min_support_threshold)
                            && fgeq(newColMinCov, min_coverage_threshold);

            //return result;
            return true;
        }else{
            return false;
        }

    }



    __global__ 
    void flagCandidatesToBeCorrectedKernel(
            bool* __restrict__ candidateCanBeCorrected,
            int* __restrict__ numCorrectedCandidatesPerAnchor,
            GPUMultiMSA multiMSA,
            const int* __restrict__ alignmentShifts,
            const int* __restrict__ candidateSequencesLengths,
            const int* __restrict__ anchorIndicesOfCandidates,
            const AnchorHighQualityFlag* __restrict__ hqflags,
            const int* __restrict__ numCandidatesPerSubjectPrefixsum,
            const int* __restrict__ localGoodCandidateIndices,
            const int* __restrict__ numLocalGoodCandidateIndicesPerSubject,
            const int* __restrict__ d_numAnchors,
            const int* __restrict__ d_numCandidates,
            float min_support_threshold,
            float min_coverage_threshold,
            int new_columns_to_correct){

        __shared__ int numAgg;

        const int n_subjects = *d_numAnchors;

        for(int anchorIndex = blockIdx.x; 
                anchorIndex < n_subjects; 
                anchorIndex += gridDim.x){

            if(threadIdx.x == 0){
                numAgg = 0;
            }
            __syncthreads();

            const GpuSingleMSA msa = multiMSA.getSingleMSA(anchorIndex);

            const bool isHighQualitySubject = hqflags[anchorIndex].hq();
            const int numGoodIndices = numLocalGoodCandidateIndicesPerSubject[anchorIndex];
            const int dataoffset = numCandidatesPerSubjectPrefixsum[anchorIndex];
            const int* myGoodIndices = localGoodCandidateIndices + dataoffset;

            if(isHighQualitySubject){

                for(int tid = threadIdx.x; tid < numGoodIndices; tid += blockDim.x){
                    const int localCandidateIndex = myGoodIndices[tid];
                    const int globalCandidateIndex = dataoffset + localCandidateIndex;

                    const bool canHandleCandidate =  checkIfCandidateShouldBeCorrectedGlobal(
                        msa,
                        alignmentShifts[globalCandidateIndex],
                        candidateSequencesLengths[globalCandidateIndex],
                        min_support_threshold,
                        min_coverage_threshold,
                        new_columns_to_correct
                    );

                    candidateCanBeCorrected[globalCandidateIndex] = canHandleCandidate;

                    if(canHandleCandidate){
                        atomicAdd(&numAgg, 1);
                        //atomicAdd(numCorrectedCandidatesPerAnchor + anchorIndex, 1);
                    }
                }

                __syncthreads();

                if(threadIdx.x == 0){
                    numCorrectedCandidatesPerAnchor[anchorIndex] = numAgg;
                }
                
            }
        }
    }


    __global__ 
    void flagCandidatesToBeCorrectedWithExcludeFlagsKernel(
            bool* __restrict__ candidateCanBeCorrected,
            int* __restrict__ numCorrectedCandidatesPerAnchor,
            GPUMultiMSA multiMSA,
            const bool* __restrict__ excludeFlags,
            const int* __restrict__ alignmentShifts,
            const int* __restrict__ candidateSequencesLengths,
            const int* __restrict__ anchorIndicesOfCandidates,
            const AnchorHighQualityFlag* __restrict__ hqflags,
            const int* __restrict__ numCandidatesPerSubjectPrefixsum,
            const int* __restrict__ localGoodCandidateIndices,
            const int* __restrict__ numLocalGoodCandidateIndicesPerSubject,
            const int* __restrict__ d_numAnchors,
            const int* __restrict__ d_numCandidates,
            float min_support_threshold,
            float min_coverage_threshold,
            int new_columns_to_correct){

        __shared__ int numAgg;

        const int n_subjects = *d_numAnchors;

        for(int anchorIndex = blockIdx.x; 
                anchorIndex < n_subjects; 
                anchorIndex += gridDim.x){

            if(threadIdx.x == 0){
                numAgg = 0;
            }
            __syncthreads();

            const GpuSingleMSA msa = multiMSA.getSingleMSA(anchorIndex);

            const bool isHighQualitySubject = hqflags[anchorIndex].hq();
            const int numGoodIndices = numLocalGoodCandidateIndicesPerSubject[anchorIndex];
            const int dataoffset = numCandidatesPerSubjectPrefixsum[anchorIndex];
            const int* myGoodIndices = localGoodCandidateIndices + dataoffset;

            if(isHighQualitySubject){

                for(int tid = threadIdx.x; tid < numGoodIndices; tid += blockDim.x){
                    const int localCandidateIndex = myGoodIndices[tid];
                    const int globalCandidateIndex = dataoffset + localCandidateIndex;

                    const bool excludeCandidate = excludeFlags[globalCandidateIndex];

                    const bool canHandleCandidate = !excludeCandidate && checkIfCandidateShouldBeCorrectedGlobal(
                        msa,
                        alignmentShifts[globalCandidateIndex],
                        candidateSequencesLengths[globalCandidateIndex],
                        min_support_threshold,
                        min_coverage_threshold,
                        new_columns_to_correct
                    );

                    candidateCanBeCorrected[globalCandidateIndex] = canHandleCandidate;

                    if(canHandleCandidate){
                        atomicAdd(&numAgg, 1);
                        //atomicAdd(numCorrectedCandidatesPerAnchor + anchorIndex, 1);
                    }
                }

                __syncthreads();

                if(threadIdx.x == 0){
                    numCorrectedCandidatesPerAnchor[anchorIndex] = numAgg;
                }
                
            }
        }
    }


    template<int BLOCKSIZE, int groupsize>
    __global__
    void msa_correct_candidates_with_group_kernel(
            char* __restrict__ correctedCandidates,
            TempCorrectedSequence::EncodedEdit* __restrict__ d_editsPerCorrectedCandidate,
            int* __restrict__ d_numEditsPerCorrectedCandidate,
            GPUMultiMSA multiMSA,
            const int* __restrict__ shifts,
            const BestAlignment_t* __restrict__ bestAlignmentFlags,
            const unsigned int* __restrict__ candidateSequencesData,
            const int* __restrict__ candidateSequencesLengths,
            const bool* __restrict__ d_candidateContainsN,
            const int* __restrict__ candidateIndicesOfCandidatesToBeCorrected,
            const int* __restrict__ numCandidatesToBeCorrected,
            const int* __restrict__ anchorIndicesOfCandidates,
            const int* __restrict__ d_numAnchors,
            const int* __restrict__ d_numCandidates,
            int doNotUseEditsValue,
            int numEditsThreshold,            
            int encodedSequencePitchInInts,
            size_t decodedSequencePitchInBytes,
            size_t editsPitchInBytes,
            size_t dynamicsmemSequencePitchInInts){

        /*
            Use groupsize threads per candidate to perform correction
        */
        static_assert(BLOCKSIZE % groupsize == 0, "BLOCKSIZE % groupsize != 0");
        constexpr int groupsPerBlock = BLOCKSIZE / groupsize;

        auto reverseWithGroupShfl = [](auto& group, char* sequence, int sequenceLength){

            auto reverse = [](char4 data){
                char4 s;
                s.x = data.w;
                s.y = data.z;
                s.z = data.y;
                s.w = data.x;
                return s;
            };
        
            auto shiftLeft1 = [](char4 data){
                char4 s;
                s.x = data.y;
                s.y = data.z;
                s.z = data.w;
                s.w = '\0';
                return s;
            };
        
            auto shiftLeft2 = [](char4 data){
                char4 s;
                s.x = data.z;
                s.y = data.w;
                s.z = '\0';
                s.w = '\0';
                return s;
            };
        
            auto shiftLeft3 = [](char4 data){
                char4 s;
                s.x = data.w;
                s.y = '\0';
                s.z = '\0';
                s.w = '\0';
                return s;
            };
        
            //treat [left,right] as "char8", shift to the left by one char. return leftmost 4 chars
            auto handleUnusedPositions1 = [](char4 left, char4 right){
                char4 s;
                s.x = left.y;
                s.y = left.z;
                s.z = left.w;
                s.w = right.x;
                return s;
            };
        
            //treat [left,right] as "char8", shift to the left by two chars. return leftmost 4 chars
            auto handleUnusedPositions2 = [](char4 left, char4 right){
                char4 s;
                s.x = left.z;
                s.y = left.w;
                s.z = right.x;
                s.w = right.y;
                return s;
            };
        
            //treat [left,right] as "char8", shift to the left by three chars. return leftmost 4 chars
            auto handleUnusedPositions3 = [](char4 left, char4 right){
                char4 s;
                s.x = left.w;
                s.y = right.x;
                s.z = right.y;
                s.w = right.z;
                return s;
            };
        
            if(sequenceLength <= 1) return;
        
            const int arrayLength = SDIV(sequenceLength, 4); // 4 bases per int
            const int unusedPositions = arrayLength * 4 - sequenceLength;
            char4* sequenceAsChar4 = (char4*)sequence;
        
            for(int i = group.thread_rank(); i < arrayLength/2; i += group.size()){
                const char4 fdata = ((char4*)sequence)[i];
                const char4 bdata = ((char4*)sequence)[arrayLength - 1 - i];
        
                const char4 front = reverse(fdata);
                const char4 back = reverse(bdata);
                sequenceAsChar4[i] = back;
                sequenceAsChar4[arrayLength - 1 - i] = front;
            }
        
            if(arrayLength % 2 == 1 && group.thread_rank() == 0){
                const int middleindex = arrayLength/2;
                const char4 mdata = ((char4*)sequence)[middleindex];
                sequenceAsChar4[middleindex] = reverse(mdata);
            }
        
            group.sync();
        
            if(unusedPositions > 0){
        
                char4 left;
                char4 right;
                char4 tmp;
        
                const int numIterations = SDIV(arrayLength-1, group.size());
        
                for(int iteration = 0; iteration < numIterations; iteration++){
                    const int index = iteration * group.size() + group.thread_rank();
                    if(index < arrayLength){
                        left = sequenceAsChar4[index];
                    }
                    const int index2 = (iteration+1) * group.size() + group.thread_rank();
                    if(index2 < arrayLength && group.thread_rank() == 0){
                        tmp = sequenceAsChar4[index2];
                    }
                    #if __CUDACC_VER_MAJOR__ < 11
                    //CUDA < 11 does not have shuffle api for char4
                    *((int*)(&right)) = group.shfl_down(*((const int*)(&left)), 1);
                    *((int*)(&tmp)) = group.shfl(*((const int*)(&tmp)), 0);
                    #else
                    right = group.shfl_down(left, 1);
                    tmp = group.shfl(tmp, 0);
                    #endif
                    if(group.thread_rank() == group.size() - 1){
                        right = tmp;
                    }
        
                    if(unusedPositions == 1){
                        char4 result = handleUnusedPositions1(left, right);
                        if(index < arrayLength - 1){
                            sequenceAsChar4[index] = result;
                        }
                    }else if(unusedPositions == 2){
                        char4 result = handleUnusedPositions2(left, right);
                        if(index < arrayLength - 1){
                            sequenceAsChar4[index] = result;
                        }
                    }else{
                        char4 result = handleUnusedPositions3(left, right);
                        if(index < arrayLength - 1){
                            sequenceAsChar4[index] = result;
                        }
                    }
                }
        
                group.sync();
        
                if(group.thread_rank() == 0){
                    if(unusedPositions == 1){
                        sequenceAsChar4[arrayLength-1] = shiftLeft1(sequenceAsChar4[arrayLength-1]);
                    }else if(unusedPositions == 2){
                        sequenceAsChar4[arrayLength-1] = shiftLeft2(sequenceAsChar4[arrayLength-1]);
                    }else{
                        assert(unusedPositions == 3);
                        sequenceAsChar4[arrayLength-1] = shiftLeft3(sequenceAsChar4[arrayLength-1]);
                    }
                }
            }
        };

        auto to_nuc = [](std::uint8_t c){
            return SequenceHelpers::decodeBase(c);
        };

        __shared__ int shared_numEditsOfCandidate[groupsPerBlock];

        extern __shared__ int dynamicsmem[]; // for sequences

        auto tgroup = cg::tiled_partition<groupsize>(cg::this_thread_block());

        const int numGroups = (gridDim.x * blockDim.x) / groupsize;
        const int groupId = (threadIdx.x + blockIdx.x * blockDim.x) / groupsize;
        const int groupIdInBlock = threadIdx.x / groupsize;

        const std::size_t smemPitchEditsInInts = SDIV(editsPitchInBytes, sizeof(int));

        char* const shared_correctedCandidate = (char*)(dynamicsmem + dynamicsmemSequencePitchInInts * groupIdInBlock);



        TempCorrectedSequence::EncodedEdit* const shared_Edits 
            = (TempCorrectedSequence::EncodedEdit*)((dynamicsmem + dynamicsmemSequencePitchInInts * groupsPerBlock) 
                + smemPitchEditsInInts * groupIdInBlock);

        const int loopEnd = *numCandidatesToBeCorrected;

        for(int id = groupId;
                id < loopEnd;
                id += numGroups){

            const int candidateIndex = candidateIndicesOfCandidatesToBeCorrected[id];
            const int subjectIndex = anchorIndicesOfCandidates[candidateIndex];
            const int destinationIndex = id;

            const GpuSingleMSA msa = multiMSA.getSingleMSA(subjectIndex);

            char* const my_corrected_candidate = correctedCandidates + destinationIndex * decodedSequencePitchInBytes;
            const int candidate_length = candidateSequencesLengths[candidateIndex];

            const int shift = shifts[candidateIndex];
            const int subjectColumnsBegin_incl = msa.columnProperties->subjectColumnsBegin_incl;
            const int queryColumnsBegin_incl = subjectColumnsBegin_incl + shift;
            const int queryColumnsEnd_excl = subjectColumnsBegin_incl + shift + candidate_length;

            const BestAlignment_t bestAlignmentFlag = bestAlignmentFlags[candidateIndex];

            if(tgroup.thread_rank() == 0){                        
                shared_numEditsOfCandidate[groupIdInBlock] = 0;
            }
            tgroup.sync();          

            const int copyposbegin = queryColumnsBegin_incl;
            const int copyposend = queryColumnsEnd_excl;
            assert(copyposend - copyposbegin == candidate_length);

            //the forward strand will be returned -> make reverse complement again
            if(bestAlignmentFlag == BestAlignment_t::ReverseComplement) {
                for(int i = copyposbegin + tgroup.thread_rank(); i < copyposend; i += tgroup.size()) {
                    shared_correctedCandidate[i - queryColumnsBegin_incl] = to_nuc(SequenceHelpers::complementBase2Bit(msa.consensus[i]));
                }
                tgroup.sync(); // threads may access elements in shared memory which were written by another thread
                reverseWithGroupShfl(tgroup, shared_correctedCandidate, candidate_length);
                tgroup.sync();
            }else{
                for(int i = copyposbegin + tgroup.thread_rank(); i < copyposend; i += tgroup.size()) {
                    shared_correctedCandidate[i - queryColumnsBegin_incl] = to_nuc(msa.consensus[i]);
                }
                tgroup.sync();
            }
            
            //copy corrected sequence from smem to global output
            const int fullInts1 = candidate_length / sizeof(int);

            for(int i = tgroup.thread_rank(); i < fullInts1; i += tgroup.size()) {
                ((int*)my_corrected_candidate)[i] = ((int*)shared_correctedCandidate)[i];
            }

            for(int i = tgroup.thread_rank(); i < candidate_length - fullInts1 * sizeof(int); i += tgroup.size()) {
                my_corrected_candidate[fullInts1 * sizeof(int) + i] 
                    = shared_correctedCandidate[fullInts1 * sizeof(int) + i];
            }       

            //compare corrected candidate with uncorrected candidate, calculate edits   
            
            const unsigned int* const encUncorrectedCandidate = candidateSequencesData 
                        + std::size_t(candidateIndex) * encodedSequencePitchInInts;
            const bool thisSequenceContainsN = d_candidateContainsN[candidateIndex];            

            if(thisSequenceContainsN){
                if(tgroup.thread_rank() == 0){
                    d_numEditsPerCorrectedCandidate[destinationIndex] = doNotUseEditsValue;
                }
            }else{
                const int maxEdits = min(candidate_length / 7, numEditsThreshold);

                auto countAndSaveEditInSmem = [&](const int posInSequence, const char correctedNuc){
                    cg::coalesced_group g = cg::coalesced_threads();
                                
                    int currentNumEdits = 0;
                    if(g.thread_rank() == 0){
                        currentNumEdits = atomicAdd(&shared_numEditsOfCandidate[groupIdInBlock], g.size());
                    }
                    currentNumEdits = g.shfl(currentNumEdits, 0);
    
                    if(currentNumEdits + g.size() <= maxEdits){
                        const int myEditOutputPos = g.thread_rank() + currentNumEdits;
                        if(myEditOutputPos < maxEdits){
                            const auto theEdit = TempCorrectedSequence::EncodedEdit{posInSequence, correctedNuc};
                            //myEdits[myEditOutputPos] = theEdit;
                            //shared_Edits[groupIdInBlock][myEditOutputPos] = theEdit;
                            shared_Edits[myEditOutputPos] = theEdit;
                        }
                    }
                };

                auto countAndSaveEditInSmem2 = [&](const int posInSequence, const char correctedNuc){
                    const int groupsPerWarp = 32 / tgroup.size();
                    if(groupsPerWarp == 1){
                        countAndSaveEditInSmem(posInSequence, correctedNuc);
                    }else{
                        const int groupIdInWarp = (threadIdx.x % 32) / tgroup.size();
                        unsigned int subwarpmask = ((1u << (tgroup.size() - 1)) | ((1u << (tgroup.size() - 1)) - 1));
                        subwarpmask <<= (tgroup.size() * groupIdInWarp);

                        unsigned int lanemask_lt;
                        asm volatile("mov.u32 %0, %%lanemask_lt;" : "=r"(lanemask_lt));
                        const unsigned int writemask = subwarpmask & __activemask();
                        const unsigned int total = __popc(writemask);
                        const unsigned int prefix = __popc(writemask & lanemask_lt);

                        const int elected_lane = __ffs(writemask) - 1;
                        int currentNumEdits = 0;
                        if (prefix == 0) {
                            currentNumEdits = atomicAdd(&shared_numEditsOfCandidate[groupIdInBlock], total);
                        }
                        currentNumEdits = __shfl_sync(writemask, currentNumEdits, elected_lane);

                        if(currentNumEdits + total <= maxEdits){
                            const int myEditOutputPos = prefix + currentNumEdits;
                            if(myEditOutputPos < maxEdits){
                                const auto theEdit = TempCorrectedSequence::EncodedEdit{posInSequence, correctedNuc};
                                //myEdits[myEditOutputPos] = theEdit;
                                //shared_Edits[groupIdInBlock][myEditOutputPos] = theEdit;
                                shared_Edits[myEditOutputPos] = theEdit;
                            }
                        }

                    }
                };

                constexpr int basesPerInt = SequenceHelpers::basesPerInt2Bit();
                const int fullInts = candidate_length / basesPerInt;   
                
                for(int i = 0; i < fullInts; i++){
                    const unsigned int encodedDataInt = encUncorrectedCandidate[i];

                    //compare with basesPerInt bases of corrected sequence

                    for(int k = tgroup.thread_rank(); k < basesPerInt; k += tgroup.size()){
                        const int posInInt = k;
                        const int posInSequence = i * basesPerInt + posInInt;
                        const std::uint8_t encodedUncorrectedNuc = SequenceHelpers::getEncodedNucFromInt2Bit(encodedDataInt, posInInt);
                        const char correctedNuc = shared_correctedCandidate[posInSequence];

                        if(correctedNuc != to_nuc(encodedUncorrectedNuc)){
                            countAndSaveEditInSmem2(posInSequence, correctedNuc);
                        }
                    }

                    tgroup.sync();

                    if(shared_numEditsOfCandidate[groupIdInBlock] > maxEdits){
                        break;
                    }
                }

                //process remaining positions
                if(shared_numEditsOfCandidate[groupIdInBlock] <= maxEdits){
                    const int remainingPositions = candidate_length - basesPerInt * fullInts;

                    if(remainingPositions > 0){
                        const unsigned int encodedDataInt = encUncorrectedCandidate[fullInts];
                        for(int posInInt = tgroup.thread_rank(); posInInt < remainingPositions; posInInt += tgroup.size()){
                            const int posInSequence = fullInts * basesPerInt + posInInt;
                            const std::uint8_t encodedUncorrectedNuc = SequenceHelpers::getEncodedNucFromInt2Bit(encodedDataInt, posInInt);
                            const char correctedNuc = shared_correctedCandidate[posInSequence];

                            if(correctedNuc != to_nuc(encodedUncorrectedNuc)){
                                countAndSaveEditInSmem2(posInSequence, correctedNuc);
                            }
                        }
                    }
                }

                tgroup.sync();

                int* const myNumEdits = d_numEditsPerCorrectedCandidate + destinationIndex;

                TempCorrectedSequence::EncodedEdit* const myEdits 
                    = (TempCorrectedSequence::EncodedEdit*)(((char*)d_editsPerCorrectedCandidate) + destinationIndex * editsPitchInBytes);

                if(shared_numEditsOfCandidate[groupIdInBlock] <= maxEdits){
                    const int numEdits = shared_numEditsOfCandidate[groupIdInBlock];

                    if(tgroup.thread_rank() == 0){ 
                        *myNumEdits = numEdits;
                    }

                    const int fullInts = (numEdits * sizeof(TempCorrectedSequence::EncodedEdit)) / sizeof(int);
                    static_assert(sizeof(TempCorrectedSequence::EncodedEdit) * 2 == sizeof(int), "");

                    for(int i = tgroup.thread_rank(); i < fullInts; i += tgroup.size()) {
                        ((int*)myEdits)[i] = ((int*)shared_Edits)[i];
                    }

                    for(int i = tgroup.thread_rank(); i < numEdits - fullInts * 2; i += tgroup.size()) {
                        myEdits[fullInts * 2 + i] = shared_Edits[fullInts * 2 + i];
                    } 
                }else{
                    if(tgroup.thread_rank() == 0){
                        *myNumEdits = doNotUseEditsValue;
                    }
                }

            }
            

            tgroup.sync(); //sync before handling next candidate
                        
        }
    }





    __global__
    void constructAnchorResultsKernel(
        TempCorrectedSequence::EncodedEdit* __restrict__ d_editsPerCorrectedSubject,
        int* __restrict__ d_numEditsPerCorrectedSubject,
        int doNotUseEditsValue,
        const int* __restrict__ d_indicesOfCorrectedSubjects,
        const int* __restrict__ d_numIndicesOfCorrectedSubjects,
        const bool* __restrict__ d_readContainsN,
        const unsigned int* __restrict__ d_uncorrectedSubjects,
        const int* __restrict__ d_subjectLengths,
        const char* __restrict__ d_correctedSubjects,
        int numEditsThreshold,
        size_t encodedSequencePitchInInts,
        size_t decodedSequencePitchInBytes,
        size_t editsPitchInBytes
    ){

        auto to_nuc = [](std::uint8_t enc){
            return SequenceHelpers::decodeBase(enc);
        };

        const int numIndicesToProcess = *d_numIndicesOfCorrectedSubjects;

        for(int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < numIndicesToProcess; tid += blockDim.x * gridDim.x){
            const int indexOfCorrectedSubject = d_indicesOfCorrectedSubjects[tid];

            const bool thisSequenceContainsN = d_readContainsN[indexOfCorrectedSubject];            
            int* const myNumEdits = d_numEditsPerCorrectedSubject + tid;

            if(thisSequenceContainsN){
                *myNumEdits = doNotUseEditsValue;
            }else{
                const int length = d_subjectLengths[indexOfCorrectedSubject];

                //find correct pointers
                const unsigned int* const encodedUncorrectedSequence = d_uncorrectedSubjects + encodedSequencePitchInInts * indexOfCorrectedSubject;
                const char* const decodedCorrectedSequence = d_correctedSubjects + decodedSequencePitchInBytes * indexOfCorrectedSubject;
    
                TempCorrectedSequence::EncodedEdit* const myEdits = (TempCorrectedSequence::EncodedEdit*)(((char*)d_editsPerCorrectedSubject) + editsPitchInBytes * tid);

                const int maxEdits = min(length / 7, numEditsThreshold);
                int edits = 0;
                
                for(int i = 0; i < length && edits <= maxEdits; i++){
                    const char correctedNuc = decodedCorrectedSequence[i];
                    const char uncorrectedNuc = to_nuc(SequenceHelpers::getEncodedNuc2Bit(encodedUncorrectedSequence, length, i));

                    if(correctedNuc != uncorrectedNuc){
                        if(edits < maxEdits){
                            myEdits[edits] = TempCorrectedSequence::EncodedEdit{i, correctedNuc};
                        }
                        edits++;
                    }
                }
                if(edits <= maxEdits){
                    *myNumEdits = edits;
                }else{
                    *myNumEdits = doNotUseEditsValue;
                }
            }
        }
    }




    //####################   KERNEL DISPATCH   ####################


    void call_msaCorrectAnchorsKernel_async(
        char* d_correctedSubjects,
        bool* d_subjectIsCorrected,
        AnchorHighQualityFlag* d_isHighQualitySubject,
        GPUMultiMSA multiMSA,
        const unsigned int* d_subjectSequencesData,
        const unsigned int* d_candidateSequencesData,
        const int* d_candidateSequencesLength,
        const int* d_indices_per_subject,
        const int* d_numAnchors,
        int maxNumAnchors,
        int encodedSequencePitchInInts,
        size_t decodedSequencePitchInBytes,
        int maximumSequenceLength,
        float estimatedErrorrate,
        float desiredAlignmentMaxErrorRate,
        float avg_support_threshold,
        float min_support_threshold,
        float min_coverage_threshold,
        float max_coverage_threshold,
        int k_region,
        int maximum_sequence_length,
        hipStream_t stream,
        KernelLaunchHandle& handle
    ){

        const int max_block_size = 256;
        const int blocksize = std::min(max_block_size, SDIV(maximum_sequence_length, 32) * 32);
        const std::size_t smem = 0;

        int max_blocks_per_device = 1;

        KernelLaunchConfig kernelLaunchConfig;
        kernelLaunchConfig.threads_per_block = blocksize;
        kernelLaunchConfig.smem = smem;

        auto iter = handle.kernelPropertiesMap.find(KernelId::MSACorrectSubjectImplicit);
        if(iter == handle.kernelPropertiesMap.end()){

            std::map<KernelLaunchConfig, KernelProperties> mymap;

            #define getProp(blocksize) { \
                KernelLaunchConfig kernelLaunchConfig; \
                kernelLaunchConfig.threads_per_block = (blocksize); \
                kernelLaunchConfig.smem = 0; \
                KernelProperties kernelProperties; \
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                    msaCorrectAnchorsKernel<(blocksize)>, \
                                                                kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
                mymap[kernelLaunchConfig] = kernelProperties; \
            }

            getProp(32);
            getProp(64);
            getProp(96);
            getProp(128);
            getProp(160);
            getProp(192);
            getProp(224);
            getProp(256);

            const auto& kernelProperties = mymap[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

            handle.kernelPropertiesMap[KernelId::MSACorrectSubjectImplicit] = std::move(mymap);

            #undef getProp
        }else{
            std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
            const KernelProperties& kernelProperties = map[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
        }

        //hipMemsetAsync(d_correctionResultPointers.isHighQualitySubject, 0, n_subjects * sizeof(AnchorHighQualityFlag), stream); CUERR;

        dim3 block(blocksize, 1, 1);
        //dim3 grid(std::min(maxNumAnchors, max_blocks_per_device));
        dim3 grid(max_blocks_per_device);

        #define mycall(blocksize) msaCorrectAnchorsKernel<(blocksize)> \
                                <<<grid, block, 0, stream>>>( \
                                    d_correctedSubjects, \
                                    d_subjectIsCorrected, \
                                    d_isHighQualitySubject, \
                                    multiMSA, \
                                    d_subjectSequencesData, \
                                    d_candidateSequencesData, \
                                    d_candidateSequencesLength, \
                                    d_indices_per_subject, \
                                    d_numAnchors, \
                                    encodedSequencePitchInInts, \
                                    decodedSequencePitchInBytes, \
                                    maximumSequenceLength, \
                                    estimatedErrorrate, \
                                    desiredAlignmentMaxErrorRate, \
                                    avg_support_threshold, \
                                    min_support_threshold, \
                                    min_coverage_threshold, \
                                    max_coverage_threshold, \
                                    k_region \
                                ); CUERR;

        assert(blocksize > 0 && blocksize <= max_block_size);

        switch(blocksize){
            case 32: mycall(32); break;
            case 64: mycall(64); break;
            case 96: mycall(96); break;
            case 128: mycall(128); break;
            case 160: mycall(160); break;
            case 192: mycall(192); break;
            case 224: mycall(224); break;
            case 256: mycall(256); break;
            default: mycall(256); break;
        }
        #undef mycall
    }



    void callFlagCandidatesToBeCorrectedKernel_async(
        bool* d_candidateCanBeCorrected,
        int* d_numCorrectedCandidatesPerAnchor,
        GPUMultiMSA multiMSA,
        const int* d_alignmentShifts,
        const int* d_candidateSequencesLengths,
        const int* d_anchorIndicesOfCandidates,
        const AnchorHighQualityFlag* d_hqflags,
        const int* d_candidatesPerSubjectPrefixsum,
        const int* d_localGoodCandidateIndices,
        const int* d_numLocalGoodCandidateIndicesPerSubject,
        const int* d_numAnchors,
        const int* d_numCandidates,
        float min_support_threshold,
        float min_coverage_threshold,
        int new_columns_to_correct,
        hipStream_t stream,
        KernelLaunchHandle& handle
    ){

        constexpr int blocksize = 256;
        const std::size_t smem = 0;

        int max_blocks_per_device = 1;

        KernelLaunchConfig kernelLaunchConfig;
        kernelLaunchConfig.threads_per_block = blocksize;
        kernelLaunchConfig.smem = smem;

        auto iter = handle.kernelPropertiesMap.find(KernelId::FlagCandidatesToBeCorrected);
        if(iter == handle.kernelPropertiesMap.end()){

            std::map<KernelLaunchConfig, KernelProperties> mymap;

            KernelProperties kernelProperties;

            hipOccupancyMaxActiveBlocksPerMultiprocessor(
                &kernelProperties.max_blocks_per_SM,
                flagCandidatesToBeCorrectedKernel,
                kernelLaunchConfig.threads_per_block, 
                kernelLaunchConfig.smem
            ); CUERR;

            mymap[kernelLaunchConfig] = kernelProperties;

            max_blocks_per_device = handle.deviceProperties.multiProcessorCount 
                                        * kernelProperties.max_blocks_per_SM;

            handle.kernelPropertiesMap[KernelId::FlagCandidatesToBeCorrected] = std::move(mymap);

            #undef getProp
        }else{
            std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
            const KernelProperties& kernelProperties = map[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount 
                                        * kernelProperties.max_blocks_per_SM;
        }



        dim3 block(blocksize);
        dim3 grid(max_blocks_per_device);

        flagCandidatesToBeCorrectedKernel<<<grid, block, 0, stream>>>(
            d_candidateCanBeCorrected,
            d_numCorrectedCandidatesPerAnchor,
            multiMSA,
            d_alignmentShifts,
            d_candidateSequencesLengths,
            d_anchorIndicesOfCandidates,
            d_hqflags,
            d_candidatesPerSubjectPrefixsum,
            d_localGoodCandidateIndices,
            d_numLocalGoodCandidateIndicesPerSubject,
            d_numAnchors,
            d_numCandidates,
            min_support_threshold,
            min_coverage_threshold,
            new_columns_to_correct
        );

        CUERR;

    }


    void callFlagCandidatesToBeCorrectedWithExcludeFlagsKernel(
        bool* d_candidateCanBeCorrected,
        int* d_numCorrectedCandidatesPerAnchor,
        GPUMultiMSA multiMSA,
        const bool* d_excludeFlags, //candidates with flag == true will not be considered
        const int* d_alignmentShifts,
        const int* d_candidateSequencesLengths,
        const int* d_anchorIndicesOfCandidates,
        const AnchorHighQualityFlag* d_hqflags,
        const int* d_candidatesPerSubjectPrefixsum,
        const int* d_localGoodCandidateIndices,
        const int* d_numLocalGoodCandidateIndicesPerSubject,
        const int* d_numAnchors,
        const int* d_numCandidates,
        float min_support_threshold,
        float min_coverage_threshold,
        int new_columns_to_correct,
        hipStream_t stream,
        KernelLaunchHandle& handle
    ){

        constexpr int blocksize = 256;
        const std::size_t smem = 0;

        int max_blocks_per_device = 1;

        KernelLaunchConfig kernelLaunchConfig;
        kernelLaunchConfig.threads_per_block = blocksize;
        kernelLaunchConfig.smem = smem;

        const auto kernelId = KernelId::FlagCandidatesToBeCorrectedWithExcludeFlags;

        auto iter = handle.kernelPropertiesMap.find(kernelId);
        if(iter == handle.kernelPropertiesMap.end()){

            std::map<KernelLaunchConfig, KernelProperties> mymap;

            KernelProperties kernelProperties;

            hipOccupancyMaxActiveBlocksPerMultiprocessor(
                &kernelProperties.max_blocks_per_SM,
                flagCandidatesToBeCorrectedWithExcludeFlagsKernel,
                kernelLaunchConfig.threads_per_block, 
                kernelLaunchConfig.smem
            ); CUERR;

            mymap[kernelLaunchConfig] = kernelProperties;

            max_blocks_per_device = handle.deviceProperties.multiProcessorCount 
                                        * kernelProperties.max_blocks_per_SM;

            handle.kernelPropertiesMap[kernelId] = std::move(mymap);

            #undef getProp
        }else{
            std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
            const KernelProperties& kernelProperties = map[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount 
                                        * kernelProperties.max_blocks_per_SM;
        }



        dim3 block(blocksize);
        dim3 grid(max_blocks_per_device);

        flagCandidatesToBeCorrectedWithExcludeFlagsKernel<<<grid, block, 0, stream>>>(
            d_candidateCanBeCorrected,
            d_numCorrectedCandidatesPerAnchor,
            multiMSA,
            d_excludeFlags,
            d_alignmentShifts,
            d_candidateSequencesLengths,
            d_anchorIndicesOfCandidates,
            d_hqflags,
            d_candidatesPerSubjectPrefixsum,
            d_localGoodCandidateIndices,
            d_numLocalGoodCandidateIndicesPerSubject,
            d_numAnchors,
            d_numCandidates,
            min_support_threshold,
            min_coverage_threshold,
            new_columns_to_correct
        );

        CUERR;

    }



    void callCorrectCandidatesKernel_async(
        char* __restrict__ correctedCandidates,
        TempCorrectedSequence::EncodedEdit* __restrict__ d_editsPerCorrectedCandidate,
        int* __restrict__ d_numEditsPerCorrectedCandidate,
        GPUMultiMSA multiMSA,
        const int* __restrict__ shifts,
        const BestAlignment_t* __restrict__ bestAlignmentFlags,
        const unsigned int* __restrict__ candidateSequencesData,
        const int* __restrict__ candidateSequencesLengths,
        const bool* __restrict__ d_candidateContainsN,
        const int* __restrict__ candidateIndicesOfCandidatesToBeCorrected,
        const int* __restrict__ numCandidatesToBeCorrected,
        const int* __restrict__ anchorIndicesOfCandidates,
        const int* d_numAnchors,
        const int* d_numCandidates,
        int doNotUseEditsValue,
        int numEditsThreshold,
        int encodedSequencePitchInInts,
        size_t decodedSequencePitchInBytes,
        size_t editsPitchInBytes,
        int maximum_sequence_length,
        hipStream_t stream,
        KernelLaunchHandle& handle
    ){

        constexpr int blocksize = 128;
        constexpr int groupsize = 32;

        const size_t dynamicsmemPitchInInts = SDIV(maximum_sequence_length, sizeof(int));
        const size_t smemPitchEditsInInts = SDIV(editsPitchInBytes, sizeof(int));

        auto calculateSmemUsage = [&](int blockDim){
            const int numGroupsPerBlock = blockDim / groupsize;
            std::size_t smem = numGroupsPerBlock * (sizeof(int) * dynamicsmemPitchInInts)
                + numGroupsPerBlock * (sizeof(int) * smemPitchEditsInInts);

            return smem;
        };

        const std::size_t smem = calculateSmemUsage(blocksize);

    	int max_blocks_per_device = 1;

    	KernelLaunchConfig kernelLaunchConfig;
    	kernelLaunchConfig.threads_per_block = blocksize;
    	kernelLaunchConfig.smem = smem;

    	auto iter = handle.kernelPropertiesMap.find(KernelId::MSACorrectCandidates);
    	if(iter == handle.kernelPropertiesMap.end()) {

    		std::map<KernelLaunchConfig, KernelProperties> mymap;

    	    #define getProp(blocksize) { \
                KernelLaunchConfig kernelLaunchConfig; \
                kernelLaunchConfig.threads_per_block = (blocksize); \
                kernelLaunchConfig.smem = calculateSmemUsage((blocksize)); \
                KernelProperties kernelProperties; \
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                    msa_correct_candidates_with_group_kernel<(blocksize), groupsize>, \
                            kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
                mymap[kernelLaunchConfig] = kernelProperties; \
            }

    		getProp(32);
    		getProp(64);
    		getProp(96);
    		getProp(128);
    		getProp(160);
    		getProp(192);
    		getProp(224);
    		getProp(256);

    		const auto& kernelProperties = mymap[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
            
            // std::cerr << "msa_correct_candidates_with_group_kernel "
            //     << "multiProcessorCount = " << handle.deviceProperties.multiProcessorCount
            //     << " max_blocks_per_SM = " << kernelProperties.max_blocks_per_SM << "\n"; 

    		handle.kernelPropertiesMap[KernelId::MSACorrectCandidates] = std::move(mymap);

    	    #undef getProp
    	}else{
    		std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
    		const KernelProperties& kernelProperties = map[kernelLaunchConfig];
    		max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
    	}

    	dim3 block(blocksize, 1, 1);
        //dim3 grid(std::min(max_blocks_per_device, n_candidates * numGroupsPerBlock));
        dim3 grid(max_blocks_per_device);
        
        assert(smem % sizeof(int) == 0);

    	#define mycall(blocksize) msa_correct_candidates_with_group_kernel<(blocksize), groupsize> \
    	        <<<grid, block, smem, stream>>>( \
                    correctedCandidates, \
                    d_editsPerCorrectedCandidate, \
                    d_numEditsPerCorrectedCandidate, \
                    multiMSA, \
                    shifts, \
                    bestAlignmentFlags, \
                    candidateSequencesData, \
                    candidateSequencesLengths, \
                    d_candidateContainsN, \
                    candidateIndicesOfCandidatesToBeCorrected, \
                    numCandidatesToBeCorrected, \
                    anchorIndicesOfCandidates, \
                    d_numAnchors, \
                    d_numCandidates, \
                    doNotUseEditsValue, \
                    numEditsThreshold, \
                    encodedSequencePitchInInts, \
                    decodedSequencePitchInBytes, \
                    editsPitchInBytes, \
                    dynamicsmemPitchInInts \
                ); CUERR;


    	switch(blocksize) {
    	case 32: mycall(32); break;
    	case 64: mycall(64); break;
    	case 96: mycall(96); break;
    	case 128: mycall(128); break;
    	case 160: mycall(160); break;
    	case 192: mycall(192); break;
    	case 224: mycall(224); break;
    	case 256: mycall(256); break;
    	default: mycall(256); break;
    	}

    		#undef mycall 
    }




    void callConstructAnchorResultsKernelAsync(
        TempCorrectedSequence::EncodedEdit* __restrict__ d_editsPerCorrectedSubject,
        int* __restrict__ d_numEditsPerCorrectedSubject,
        int doNotUseEditsValue,
        const int* __restrict__ d_indicesOfCorrectedSubjects,
        const int* __restrict__ d_numIndicesOfCorrectedSubjects,
        const bool* __restrict__ d_readContainsN,
        const unsigned int* __restrict__ d_uncorrectedSubjects,
        const int* __restrict__ d_subjectLengths,
        const char* __restrict__ d_correctedSubjects,
        int numEditsThreshold,
        size_t encodedSequencePitchInInts,
        size_t decodedSequencePitchInBytes,
        size_t editsPitchInBytes,
        const int* d_numAnchors,
        int maxNumAnchors,
        hipStream_t stream,
        KernelLaunchHandle& handle
    ){

        hipMemsetAsync(
            d_editsPerCorrectedSubject, 
            0, 
            editsPitchInBytes * maxNumAnchors, 
            stream
        ); CUERR;

        const int blocksize = 128;
        const std::size_t smem = 0;

        int max_blocks_per_device = 1;

        KernelLaunchConfig kernelLaunchConfig;
        kernelLaunchConfig.threads_per_block = blocksize;
        kernelLaunchConfig.smem = smem;

        auto iter = handle.kernelPropertiesMap.find(KernelId::ConstructAnchorResults);
        if(iter == handle.kernelPropertiesMap.end()){

            std::map<KernelLaunchConfig, KernelProperties> mymap;

            #define getProp(blocksize) { \
                KernelLaunchConfig kernelLaunchConfig; \
                kernelLaunchConfig.threads_per_block = (blocksize); \
                kernelLaunchConfig.smem = 0; \
                KernelProperties kernelProperties; \
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                    constructAnchorResultsKernel, \
                                                                kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
                mymap[kernelLaunchConfig] = kernelProperties; \
            }

            getProp(32);
            getProp(64);
            getProp(96);
            getProp(128);
            getProp(160);
            getProp(192);
            getProp(224);
            getProp(256);

            const auto& kernelProperties = mymap[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

            handle.kernelPropertiesMap[KernelId::ConstructAnchorResults] = std::move(mymap);

            #undef getProp
        }else{
            std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
            const KernelProperties& kernelProperties = map[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
        }

        dim3 block(blocksize, 1, 1);
        dim3 grid(std::min(SDIV(maxNumAnchors, blocksize), max_blocks_per_device));

        #define mycall(blocksize) constructAnchorResultsKernel \
                                <<<grid, block, 0, stream>>>( \
                                        d_editsPerCorrectedSubject, \
                                        d_numEditsPerCorrectedSubject, \
                                        doNotUseEditsValue, \
                                        d_indicesOfCorrectedSubjects, \
                                        d_numIndicesOfCorrectedSubjects, \
                                        d_readContainsN, \
                                        d_uncorrectedSubjects, \
                                        d_subjectLengths, \
                                        d_correctedSubjects, \
                                        numEditsThreshold, \
                                        encodedSequencePitchInInts, \
                                        decodedSequencePitchInBytes, \
                                        editsPitchInBytes); CUERR;

        mycall();

        // switch(blocksize){
        //     case 32: mycall(32); break;
        //     case 64: mycall(64); break;
        //     case 96: mycall(96); break;
        //     case 128: mycall(128); break;
        //     case 160: mycall(160); break;
        //     case 192: mycall(192); break;
        //     case 224: mycall(224); break;
        //     case 256: mycall(256); break;
        //     default: mycall(256); break;
        // }
         #undef mycall
    }











}
}
