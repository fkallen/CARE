#include "hip/hip_runtime.h"
//#define NDEBUG

#include <gpu/kernels.hpp>
#include <gpu/kernellaunch.hpp>
#include <hostdevicefunctions.cuh>
#include <gpu/gpumsa.cuh>

#include <bestalignment.hpp>

#include <sequencehelpers.hpp>
#include <correctionresultprocessing.hpp>

#include <hpc_helpers.cuh>
#include <config.hpp>
#include <cassert>


#include <hipcub/hipcub.hpp>

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

namespace care{
namespace gpu{



    template<int BLOCKSIZE>
    __global__
    void msaCorrectAnchorsKernel(
            char* __restrict__ correctedSubjects,
            bool* __restrict__ subjectIsCorrected,
            AnchorHighQualityFlag* __restrict__ isHighQualitySubject,
            GPUMultiMSA multiMSA,
            const unsigned int* __restrict__ subjectSequencesData,
            const unsigned int* __restrict__ candidateSequencesData,
            const int* __restrict__ candidateSequencesLength,
            const int* __restrict__ d_indices_per_subject,
            const int* __restrict__ numAnchorsPtr,
            int encodedSequencePitchInInts,
            size_t decodedSequencePitchInBytes,
            int maximumSequenceLength,
            float estimatedErrorrate,
            float desiredAlignmentMaxErrorRate,
            float avg_support_threshold,
            float min_support_threshold,
            float min_coverage_threshold,
            float max_coverage_threshold,
            int k_region){

        using BlockReduceBool = hipcub::BlockReduce<bool, BLOCKSIZE>;
        using BlockReduceInt = hipcub::BlockReduce<int, BLOCKSIZE>;
        using BlockReduceFloat = hipcub::BlockReduce<float, BLOCKSIZE>;

        __shared__ union {
            typename BlockReduceBool::TempStorage boolreduce;
            typename BlockReduceInt::TempStorage intreduce;
            typename BlockReduceFloat::TempStorage floatreduce;
        } temp_storage;

        __shared__ int broadcastbuffer;

        //__shared__ int numUncorrectedPositions;
        //__shared__ int uncorrectedPositions[BLOCKSIZE];
        //__shared__ float avgCountPerWeight[4];

        auto tbGroup = cg::this_thread_block();

        const int n_subjects = *numAnchorsPtr;

        auto isGoodAvgSupport = [&](float avgsupport){
            return fgeq(avgsupport, avg_support_threshold);
        };
        auto isGoodMinSupport = [&](float minsupport){
            return fgeq(minsupport, min_support_threshold);
        };
        auto isGoodMinCoverage = [&](float mincoverage){
            return fgeq(mincoverage, min_coverage_threshold);
        };

        auto to_nuc = [](std::uint8_t c){
            return SequenceHelpers::decodeBase(c);
        };

        for(unsigned subjectIndex = blockIdx.x; subjectIndex < n_subjects; subjectIndex += gridDim.x){
            const int myNumIndices = d_indices_per_subject[subjectIndex];
            if(myNumIndices > 0){

                const GpuSingleMSA msa = multiMSA.getSingleMSA(subjectIndex);

                char* const my_corrected_subject = correctedSubjects + subjectIndex * decodedSequencePitchInBytes;

                const int subjectColumnsBegin_incl = msa.columnProperties->subjectColumnsBegin_incl;
                const int subjectColumnsEnd_excl = msa.columnProperties->subjectColumnsEnd_excl;
                const int lastColumn_excl = msa.columnProperties->lastColumn_excl;

                float avg_support = 0;
                float min_support = 1.0f;
                //int max_coverage = 0;
                int min_coverage = std::numeric_limits<int>::max();

                for(int i = subjectColumnsBegin_incl + tbGroup.thread_rank(); 
                        i < subjectColumnsEnd_excl; 
                        i += tbGroup.size()){

                    assert(i < lastColumn_excl);

                    avg_support += msa.support[i];
                    min_support = min(msa.support[i], min_support);
                    //max_coverage = max(my_coverage[i], max_coverage);
                    min_coverage = min(msa.coverages[i], min_coverage);
                }

                avg_support = BlockReduceFloat(temp_storage.floatreduce).Sum(avg_support);
                __syncthreads();

                min_support = BlockReduceFloat(temp_storage.floatreduce).Reduce(min_support, hipcub::Min());
                __syncthreads();

                //max_coverage = BlockReduceInt(temp_storage.intreduce).Reduce(max_coverage, hipcub::Max());

                min_coverage = BlockReduceInt(temp_storage.intreduce).Reduce(min_coverage, hipcub::Min());
                __syncthreads();

                avg_support /= (subjectColumnsEnd_excl - subjectColumnsBegin_incl);


                //const float avg_support_threshold = 1.0f-1.0f*estimatedErrorrate;
        		//const float min_support_threshold = 1.0f-3.0f*estimatedErrorrate;

                if(tbGroup.thread_rank() == 0){
                    subjectIsCorrected[subjectIndex] = true; //canBeCorrected;

                    const bool canBeCorrectedByConsensus = isGoodAvgSupport(avg_support) && isGoodMinSupport(min_support) && isGoodMinCoverage(min_coverage);
                    int flag = 0;

                    if(canBeCorrectedByConsensus){
                        int smallestErrorrateThatWouldMakeHQ = 100;

                        const int estimatedErrorratePercent = ceil(estimatedErrorrate * 100.0f);
                        for(int percent = estimatedErrorratePercent; percent >= 0; percent--){
                            const float factor = percent / 100.0f;
                            const float avg_threshold = 1.0f - 1.0f * factor;
                            const float min_threshold = 1.0f - 3.0f * factor;
                            if(fgeq(avg_support, avg_threshold) && fgeq(min_support, min_threshold)){
                                smallestErrorrateThatWouldMakeHQ = percent;
                            }
                            // if(readId == 134){
                            //     printf("avg_support %f, avg_threshold %f, min_support %f, min_threshold %f\n", 
                            //     avg_support, avg_threshold, min_support, min_threshold);
                            // }
                        }

                        const bool isHQ = isGoodMinCoverage(min_coverage)
                                            && fleq(smallestErrorrateThatWouldMakeHQ, estimatedErrorratePercent * 0.5f);

                        //broadcastbuffer = isHQ;
                        isHighQualitySubject[subjectIndex].hq(isHQ);

                        flag = isHQ ? 2 : 1;

                        // if(readId == 134){
                        //     printf("read 134 isHQ %d, min_coverage %d, avg_support %f, min_support %f, smallestErrorrateThatWouldMakeHQ %d, min_coverage_threshold %f\n", 
                        //         isHQ, min_coverage, avg_support, min_support, smallestErrorrateThatWouldMakeHQ, min_coverage_threshold);
                        // }
                    }else{
                        isHighQualitySubject[subjectIndex].hq(false);
                    }

                    broadcastbuffer = flag;
                }

                tbGroup.sync();

                const int flag = broadcastbuffer;

                if(flag > 0){
                    for(int i = subjectColumnsBegin_incl + tbGroup.thread_rank(); 
                            i < subjectColumnsEnd_excl; 
                            i += tbGroup.size()){

                        const std::uint8_t nuc = msa.consensus[i];
                        //assert(nuc == 'A' || nuc == 'C' || nuc == 'G' || nuc == 'T');
                        assert(0 == nuc || nuc < 4);

                        my_corrected_subject[i - subjectColumnsBegin_incl] = to_nuc(nuc);
                    }
                }else{
                    //correct only positions with high support.
                    for(int i = subjectColumnsBegin_incl + tbGroup.thread_rank(); 
                            i < subjectColumnsEnd_excl; 
                            i += tbGroup.size()){

                        
                        if(msa.support[i] > 0.90f && msa.origCoverages[i] <= 2){
                            my_corrected_subject[i - subjectColumnsBegin_incl] = to_nuc(msa.consensus[i]);
                        }else{
                            const unsigned int* const subject = subjectSequencesData + std::size_t(subjectIndex) * encodedSequencePitchInInts;
                            const std::uint8_t encodedBase = SequenceHelpers::getEncodedNuc2Bit(subject, subjectColumnsEnd_excl- subjectColumnsBegin_incl, i - subjectColumnsBegin_incl);
                            const char base = to_nuc(encodedBase);
                            assert(base == 'A' || base == 'C' || base == 'G' || base == 'T');
                            my_corrected_subject[i - subjectColumnsBegin_incl] = base;
                        }
                    }
                }
            }else{
                if(tbGroup.thread_rank() == 0){
                    isHighQualitySubject[subjectIndex].hq(false);
                    subjectIsCorrected[subjectIndex] = false;
                }
            }
        }
    }



    __device__ __forceinline__
    bool checkIfCandidateShouldBeCorrectedGlobal(
        const GpuSingleMSA msa,
        const int alignmentShift,
        const int candidateLength,
        float min_support_threshold,
        float min_coverage_threshold,
        int new_columns_to_correct
    ){

        const auto columnProperties = *msa.columnProperties;

        const int& subjectColumnsBegin_incl = columnProperties.subjectColumnsBegin_incl;
        const int& subjectColumnsEnd_excl = columnProperties.subjectColumnsEnd_excl;
        const int& lastColumn_excl = columnProperties.lastColumn_excl;

        const int shift = alignmentShift;
        const int candidate_length = candidateLength;
        const int queryColumnsBegin_incl = subjectColumnsBegin_incl + shift;
        const int queryColumnsEnd_excl = subjectColumnsBegin_incl + shift + candidate_length;

        if(subjectColumnsBegin_incl - new_columns_to_correct <= queryColumnsBegin_incl
           && queryColumnsBegin_incl <= subjectColumnsBegin_incl + new_columns_to_correct
           && queryColumnsEnd_excl <= subjectColumnsEnd_excl + new_columns_to_correct) {

            float newColMinSupport = 1.0f;
            int newColMinCov = std::numeric_limits<int>::max();
            //check new columns left of subject
            for(int columnindex = subjectColumnsBegin_incl - new_columns_to_correct;
                columnindex < subjectColumnsBegin_incl;
                columnindex++) {

                assert(columnindex < lastColumn_excl);
                if(queryColumnsBegin_incl <= columnindex) {
                    newColMinSupport = msa.support[columnindex] < newColMinSupport ? msa.support[columnindex] : newColMinSupport;
                    newColMinCov = msa.coverages[columnindex] < newColMinCov ? msa.coverages[columnindex] : newColMinCov;
                }
            }
            //check new columns right of subject
            for(int columnindex = subjectColumnsEnd_excl;
                    columnindex < subjectColumnsEnd_excl + new_columns_to_correct
                        && columnindex < lastColumn_excl;
                    columnindex++) {

                newColMinSupport = msa.support[columnindex] < newColMinSupport ? msa.support[columnindex] : newColMinSupport;
                newColMinCov = msa.coverages[columnindex] < newColMinCov ? msa.coverages[columnindex] : newColMinCov;
            }

            bool result = fgeq(newColMinSupport, min_support_threshold)
                            && fgeq(newColMinCov, min_coverage_threshold);

            //return result;
            return true;
        }else{
            return false;
        }

    }



    __global__ 
    void flagCandidatesToBeCorrectedKernel(
            bool* __restrict__ candidateCanBeCorrected,
            int* __restrict__ numCorrectedCandidatesPerAnchor,
            GPUMultiMSA multiMSA,
            const int* __restrict__ alignmentShifts,
            const int* __restrict__ candidateSequencesLengths,
            const int* __restrict__ anchorIndicesOfCandidates,
            const AnchorHighQualityFlag* __restrict__ hqflags,
            const int* __restrict__ numCandidatesPerSubjectPrefixsum,
            const int* __restrict__ localGoodCandidateIndices,
            const int* __restrict__ numLocalGoodCandidateIndicesPerSubject,
            const int* __restrict__ d_numAnchors,
            const int* __restrict__ d_numCandidates,
            float min_support_threshold,
            float min_coverage_threshold,
            int new_columns_to_correct){

        __shared__ int numAgg;

        const int n_subjects = *d_numAnchors;

        for(int anchorIndex = blockIdx.x; 
                anchorIndex < n_subjects; 
                anchorIndex += gridDim.x){

            if(threadIdx.x == 0){
                numAgg = 0;
            }
            __syncthreads();

            const GpuSingleMSA msa = multiMSA.getSingleMSA(anchorIndex);

            const bool isHighQualitySubject = hqflags[anchorIndex].hq();
            const int numGoodIndices = numLocalGoodCandidateIndicesPerSubject[anchorIndex];
            const int dataoffset = numCandidatesPerSubjectPrefixsum[anchorIndex];
            const int* myGoodIndices = localGoodCandidateIndices + dataoffset;

            if(isHighQualitySubject){

                for(int tid = threadIdx.x; tid < numGoodIndices; tid += blockDim.x){
                    const int localCandidateIndex = myGoodIndices[tid];
                    const int globalCandidateIndex = dataoffset + localCandidateIndex;

                    const bool canHandleCandidate =  checkIfCandidateShouldBeCorrectedGlobal(
                        msa,
                        alignmentShifts[globalCandidateIndex],
                        candidateSequencesLengths[globalCandidateIndex],
                        min_support_threshold,
                        min_coverage_threshold,
                        new_columns_to_correct
                    );

                    candidateCanBeCorrected[globalCandidateIndex] = canHandleCandidate;

                    if(canHandleCandidate){
                        atomicAdd(&numAgg, 1);
                        //atomicAdd(numCorrectedCandidatesPerAnchor + anchorIndex, 1);
                    }
                }

                __syncthreads();

                if(threadIdx.x == 0){
                    numCorrectedCandidatesPerAnchor[anchorIndex] = numAgg;
                }
                
            }
        }
    }


    template<int BLOCKSIZE, int groupsize>
    __global__
    void msa_correct_candidates_with_group_kernel(
            char* __restrict__ correctedCandidates,
            TempCorrectedSequence::EncodedEdit* __restrict__ d_editsPerCorrectedCandidate,
            int* __restrict__ d_numEditsPerCorrectedCandidate,
            GPUMultiMSA multiMSA,
            const int* __restrict__ shifts,
            const BestAlignment_t* __restrict__ bestAlignmentFlags,
            const unsigned int* __restrict__ candidateSequencesData,
            const int* __restrict__ candidateSequencesLengths,
            const bool* __restrict__ d_candidateContainsN,
            const int* __restrict__ candidateIndicesOfCandidatesToBeCorrected,
            const int* __restrict__ numCandidatesToBeCorrected,
            const int* __restrict__ anchorIndicesOfCandidates,
            const int* __restrict__ d_numAnchors,
            const int* __restrict__ d_numCandidates,
            int doNotUseEditsValue,
            int numEditsThreshold,            
            int encodedSequencePitchInInts,
            size_t decodedSequencePitchInBytes,
            size_t editsPitchInBytes,
            size_t dynamicsmemSequencePitchInInts){

        /*
            Use groupsize threads per candidate to perform correction
        */
        static_assert(BLOCKSIZE % groupsize == 0, "BLOCKSIZE % groupsize != 0");
        constexpr int groupsPerBlock = BLOCKSIZE / groupsize;

        auto reverseWithGroupShfl = [](auto& group, char* sequence, int sequenceLength){

            auto reverse = [](char4 data){
                char4 s;
                s.x = data.w;
                s.y = data.z;
                s.z = data.y;
                s.w = data.x;
                return s;
            };
        
            auto shiftLeft1 = [](char4 data){
                char4 s;
                s.x = data.y;
                s.y = data.z;
                s.z = data.w;
                s.w = '\0';
                return s;
            };
        
            auto shiftLeft2 = [](char4 data){
                char4 s;
                s.x = data.z;
                s.y = data.w;
                s.z = '\0';
                s.w = '\0';
                return s;
            };
        
            auto shiftLeft3 = [](char4 data){
                char4 s;
                s.x = data.w;
                s.y = '\0';
                s.z = '\0';
                s.w = '\0';
                return s;
            };
        
            //treat [left,right] as "char8", shift to the left by one char. return leftmost 4 chars
            auto handleUnusedPositions1 = [](char4 left, char4 right){
                char4 s;
                s.x = left.y;
                s.y = left.z;
                s.z = left.w;
                s.w = right.x;
                return s;
            };
        
            //treat [left,right] as "char8", shift to the left by two chars. return leftmost 4 chars
            auto handleUnusedPositions2 = [](char4 left, char4 right){
                char4 s;
                s.x = left.z;
                s.y = left.w;
                s.z = right.x;
                s.w = right.y;
                return s;
            };
        
            //treat [left,right] as "char8", shift to the left by three chars. return leftmost 4 chars
            auto handleUnusedPositions3 = [](char4 left, char4 right){
                char4 s;
                s.x = left.w;
                s.y = right.x;
                s.z = right.y;
                s.w = right.z;
                return s;
            };
        
            if(sequenceLength <= 1) return;
        
            const int arrayLength = SDIV(sequenceLength, 4); // 4 bases per int
            const int unusedPositions = arrayLength * 4 - sequenceLength;
            char4* sequenceAsChar4 = (char4*)sequence;
        
            for(int i = group.thread_rank(); i < arrayLength/2; i += group.size()){
                const char4 fdata = ((char4*)sequence)[i];
                const char4 bdata = ((char4*)sequence)[arrayLength - 1 - i];
        
                const char4 front = reverse(fdata);
                const char4 back = reverse(bdata);
                sequenceAsChar4[i] = back;
                sequenceAsChar4[arrayLength - 1 - i] = front;
            }
        
            if(arrayLength % 2 == 1 && group.thread_rank() == 0){
                const int middleindex = arrayLength/2;
                const char4 mdata = ((char4*)sequence)[middleindex];
                sequenceAsChar4[middleindex] = reverse(mdata);
            }
        
            group.sync();
        
            if(unusedPositions > 0){
        
                char4 left;
                char4 right;
                char4 tmp;
        
                const int numIterations = SDIV(arrayLength-1, group.size());
        
                for(int iteration = 0; iteration < numIterations; iteration++){
                    const int index = iteration * group.size() + group.thread_rank();
                    if(index < arrayLength){
                        left = sequenceAsChar4[index];
                    }
                    const int index2 = (iteration+1) * group.size() + group.thread_rank();
                    if(index2 < arrayLength && group.thread_rank() == 0){
                        tmp = sequenceAsChar4[index2];
                    }
                    #if __CUDACC_VER_MAJOR__ < 11
                    //CUDA < 11 does not have shuffle api for char4
                    *((int*)(&right)) = group.shfl_down(*((const int*)(&left)), 1);
                    *((int*)(&tmp)) = group.shfl(*((const int*)(&tmp)), 0);
                    #else
                    right = group.shfl_down(left, 1);
                    tmp = group.shfl(tmp, 0);
                    #endif
                    if(group.thread_rank() == group.size() - 1){
                        right = tmp;
                    }
        
                    if(unusedPositions == 1){
                        char4 result = handleUnusedPositions1(left, right);
                        if(index < arrayLength - 1){
                            sequenceAsChar4[index] = result;
                        }
                    }else if(unusedPositions == 2){
                        char4 result = handleUnusedPositions2(left, right);
                        if(index < arrayLength - 1){
                            sequenceAsChar4[index] = result;
                        }
                    }else{
                        char4 result = handleUnusedPositions3(left, right);
                        if(index < arrayLength - 1){
                            sequenceAsChar4[index] = result;
                        }
                    }
                }
        
                group.sync();
        
                if(group.thread_rank() == 0){
                    if(unusedPositions == 1){
                        sequenceAsChar4[arrayLength-1] = shiftLeft1(sequenceAsChar4[arrayLength-1]);
                    }else if(unusedPositions == 2){
                        sequenceAsChar4[arrayLength-1] = shiftLeft2(sequenceAsChar4[arrayLength-1]);
                    }else{
                        assert(unusedPositions == 3);
                        sequenceAsChar4[arrayLength-1] = shiftLeft3(sequenceAsChar4[arrayLength-1]);
                    }
                }
            }
        };

        auto to_nuc = [](std::uint8_t c){
            return SequenceHelpers::convertIntToDNACharNoIf(c);
        };

        __shared__ int shared_numEditsOfCandidate[groupsPerBlock];

        extern __shared__ int dynamicsmem[]; // for sequences

        auto tgroup = cg::tiled_partition<groupsize>(cg::this_thread_block());

        const int numGroups = (gridDim.x * blockDim.x) / groupsize;
        const int groupId = (threadIdx.x + blockIdx.x * blockDim.x) / groupsize;
        const int groupIdInBlock = threadIdx.x / groupsize;

        const std::size_t smemPitchEditsInInts = SDIV(editsPitchInBytes, sizeof(int));

        char* const shared_correctedCandidate = (char*)(dynamicsmem + dynamicsmemSequencePitchInInts * groupIdInBlock);



        TempCorrectedSequence::EncodedEdit* const shared_Edits 
            = (TempCorrectedSequence::EncodedEdit*)((dynamicsmem + dynamicsmemSequencePitchInInts * groupsPerBlock) 
                + smemPitchEditsInInts * groupIdInBlock);

        const int loopEnd = *numCandidatesToBeCorrected;

        for(int id = groupId;
                id < loopEnd;
                id += numGroups){

            const int candidateIndex = candidateIndicesOfCandidatesToBeCorrected[id];
            const int subjectIndex = anchorIndicesOfCandidates[candidateIndex];
            const int destinationIndex = id;

            const GpuSingleMSA msa = multiMSA.getSingleMSA(subjectIndex);

            char* const my_corrected_candidate = correctedCandidates + destinationIndex * decodedSequencePitchInBytes;
            const int candidate_length = candidateSequencesLengths[candidateIndex];

            const int shift = shifts[candidateIndex];
            const int subjectColumnsBegin_incl = msa.columnProperties->subjectColumnsBegin_incl;
            const int queryColumnsBegin_incl = subjectColumnsBegin_incl + shift;
            const int queryColumnsEnd_excl = subjectColumnsBegin_incl + shift + candidate_length;

            const BestAlignment_t bestAlignmentFlag = bestAlignmentFlags[candidateIndex];

            if(tgroup.thread_rank() == 0){                        
                shared_numEditsOfCandidate[groupIdInBlock] = 0;
            }
            tgroup.sync();          

            
            

            const int copyposbegin = queryColumnsBegin_incl;
            const int copyposend = queryColumnsEnd_excl;
            assert(copyposend - copyposbegin == candidate_length);

            //the forward strand will be returned -> make reverse complement again
            if(bestAlignmentFlag == BestAlignment_t::ReverseComplement) {
                for(int i = copyposbegin + tgroup.thread_rank(); i < copyposend; i += tgroup.size()) {
                    shared_correctedCandidate[i - queryColumnsBegin_incl] = to_nuc(SequenceHelpers::complementBase2Bit(msa.consensus[i]));
                }
                tgroup.sync(); // threads may access elements in shared memory which were written by another thread
                reverseWithGroupShfl(tgroup, shared_correctedCandidate, candidate_length);
                tgroup.sync();
            }else{
                for(int i = copyposbegin + tgroup.thread_rank(); i < copyposend; i += tgroup.size()) {
                    shared_correctedCandidate[i - queryColumnsBegin_incl] = to_nuc(msa.consensus[i]);
                }
                tgroup.sync();
            }
            
            //copy corrected sequence from smem to global output
            const int fullInts1 = candidate_length / sizeof(int);

            for(int i = tgroup.thread_rank(); i < fullInts1; i += tgroup.size()) {
                ((int*)my_corrected_candidate)[i] = ((int*)shared_correctedCandidate)[i];
            }

            for(int i = tgroup.thread_rank(); i < candidate_length - fullInts1 * sizeof(int); i += tgroup.size()) {
                my_corrected_candidate[fullInts1 * sizeof(int) + i] 
                    = shared_correctedCandidate[fullInts1 * sizeof(int) + i];
            }       

            //compare corrected candidate with uncorrected candidate, calculate edits   
            
            const unsigned int* const encUncorrectedCandidate = candidateSequencesData 
                        + std::size_t(candidateIndex) * encodedSequencePitchInInts;
            const bool thisSequenceContainsN = d_candidateContainsN[candidateIndex];            

            if(thisSequenceContainsN){
                if(tgroup.thread_rank() == 0){
                    d_numEditsPerCorrectedCandidate[candidateIndex] = doNotUseEditsValue;
                }
            }else{
                const int maxEdits = min(candidate_length / 7, numEditsThreshold);

                auto countAndSaveEditInSmem = [&](const int posInSequence, const char correctedNuc){
                    cg::coalesced_group g = cg::coalesced_threads();
                                
                    int currentNumEdits = 0;
                    if(g.thread_rank() == 0){
                        currentNumEdits = atomicAdd(&shared_numEditsOfCandidate[groupIdInBlock], g.size());
                    }
                    currentNumEdits = g.shfl(currentNumEdits, 0);
    
                    if(currentNumEdits + g.size() <= maxEdits){
                        const int myEditOutputPos = g.thread_rank() + currentNumEdits;
                        if(myEditOutputPos < maxEdits){
                            const auto theEdit = TempCorrectedSequence::EncodedEdit{posInSequence, correctedNuc};
                            //myEdits[myEditOutputPos] = theEdit;
                            //shared_Edits[groupIdInBlock][myEditOutputPos] = theEdit;
                            shared_Edits[myEditOutputPos] = theEdit;
                        }
                    }
                };

                auto countAndSaveEditInSmem2 = [&](const int posInSequence, const char correctedNuc){
                    const int groupsPerWarp = 32 / tgroup.size();
                    if(groupsPerWarp == 1){
                        countAndSaveEditInSmem(posInSequence, correctedNuc);
                    }else{
                        const int groupIdInWarp = (threadIdx.x % 32) / tgroup.size();
                        unsigned int subwarpmask = ((1u << (tgroup.size() - 1)) | ((1u << (tgroup.size() - 1)) - 1));
                        subwarpmask <<= (tgroup.size() * groupIdInWarp);

                        unsigned int lanemask_lt;
                        asm volatile("mov.u32 %0, %%lanemask_lt;" : "=r"(lanemask_lt));
                        const unsigned int writemask = subwarpmask & __activemask();
                        const unsigned int total = __popc(writemask);
                        const unsigned int prefix = __popc(writemask & lanemask_lt);

                        const int elected_lane = __ffs(writemask) - 1;
                        int currentNumEdits = 0;
                        if (prefix == 0) {
                            currentNumEdits = atomicAdd(&shared_numEditsOfCandidate[groupIdInBlock], total);
                        }
                        currentNumEdits = __shfl_sync(writemask, currentNumEdits, elected_lane);

                        if(currentNumEdits + total <= maxEdits){
                            const int myEditOutputPos = prefix + currentNumEdits;
                            if(myEditOutputPos < maxEdits){
                                const auto theEdit = TempCorrectedSequence::EncodedEdit{posInSequence, correctedNuc};
                                //myEdits[myEditOutputPos] = theEdit;
                                //shared_Edits[groupIdInBlock][myEditOutputPos] = theEdit;
                                shared_Edits[myEditOutputPos] = theEdit;
                            }
                        }

                    }
                };

                constexpr int basesPerInt = SequenceHelpers::basesPerInt2Bit();
                const int fullInts = candidate_length / basesPerInt;                

                for(int i = 0; i < fullInts; i++){
                    const unsigned int encodedDataInt = encUncorrectedCandidate[i];

                    //compare with basesPerInt bases of corrected sequence

                    for(int k = tgroup.thread_rank(); k < basesPerInt; k += tgroup.size()){
                        const int posInInt = k;
                        const int posInSequence = i * basesPerInt + posInInt;
                        const std::uint8_t encodedUncorrectedNuc = SequenceHelpers::getEncodedNucFromInt2Bit(encodedDataInt, posInInt);
                        const char correctedNuc = shared_correctedCandidate[posInSequence];

                        if(correctedNuc != to_nuc(encodedUncorrectedNuc)){
                            countAndSaveEditInSmem2(posInSequence, correctedNuc);
                        }
                    }

                    tgroup.sync();

                    if(shared_numEditsOfCandidate[groupIdInBlock] > maxEdits){
                        break;
                    }
                }

                //process remaining positions
                if(shared_numEditsOfCandidate[groupIdInBlock] <= maxEdits){
                    const int remainingPositions = candidate_length - basesPerInt * fullInts;
                    if(remainingPositions > 0){
                        const unsigned int encodedDataInt = encUncorrectedCandidate[fullInts];
                        for(int posInInt = tgroup.thread_rank(); posInInt < remainingPositions; posInInt += tgroup.size()){
                            const int posInSequence = fullInts * basesPerInt + posInInt;
                            const std::uint8_t encodedUncorrectedNuc = SequenceHelpers::getEncodedNucFromInt2Bit(encodedDataInt, posInInt);
                            const char correctedNuc = shared_correctedCandidate[posInSequence];

                            if(correctedNuc != to_nuc(encodedUncorrectedNuc)){
                                countAndSaveEditInSmem2(posInSequence, correctedNuc);
                            }
                        }
                    }
                }

                tgroup.sync();

                //int* const myNumEdits = d_numEditsPerCorrectedCandidate + candidateIndex;
                int* const myNumEdits = d_numEditsPerCorrectedCandidate + destinationIndex;

                TempCorrectedSequence::EncodedEdit* const myEdits 
                    = (TempCorrectedSequence::EncodedEdit*)(((char*)d_editsPerCorrectedCandidate) + destinationIndex * editsPitchInBytes);

                if(shared_numEditsOfCandidate[groupIdInBlock] <= maxEdits){
                    const int numEdits = shared_numEditsOfCandidate[groupIdInBlock];

                    if(tgroup.thread_rank() == 0){ 
                        *myNumEdits = numEdits;
                    }

                    const int fullInts = (numEdits * sizeof(TempCorrectedSequence::EncodedEdit)) / sizeof(int);
                    static_assert(sizeof(TempCorrectedSequence::EncodedEdit) * 2 == sizeof(int), "");

                    for(int i = tgroup.thread_rank(); i < fullInts; i += tgroup.size()) {
                        ((int*)myEdits)[i] = ((int*)shared_Edits)[i];
                    }

                    for(int i = tgroup.thread_rank(); i < numEdits - fullInts * 2; i += tgroup.size()) {
                        myEdits[fullInts * 2 + i] = shared_Edits[fullInts * 2 + i];
                    } 
                }else{
                    if(tgroup.thread_rank() == 0){
                        *myNumEdits = doNotUseEditsValue;
                    }
                }
            }
            

            tgroup.sync(); //sync before handling next candidate
                        
        }
    }





    __global__
    void constructAnchorResultsKernel(
        TempCorrectedSequence::EncodedEdit* __restrict__ d_editsPerCorrectedSubject,
        int* __restrict__ d_numEditsPerCorrectedSubject,
        int doNotUseEditsValue,
        const int* __restrict__ d_indicesOfCorrectedSubjects,
        const int* __restrict__ d_numIndicesOfCorrectedSubjects,
        const bool* __restrict__ d_readContainsN,
        const unsigned int* __restrict__ d_uncorrectedSubjects,
        const int* __restrict__ d_subjectLengths,
        const char* __restrict__ d_correctedSubjects,
        int numEditsThreshold,
        size_t encodedSequencePitchInInts,
        size_t decodedSequencePitchInBytes,
        size_t editsPitchInBytes
    ){

        auto to_nuc = [](std::uint8_t enc){
            return SequenceHelpers::decodeBase(enc);
        };

        const int numIndicesToProcess = *d_numIndicesOfCorrectedSubjects;

        for(int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < numIndicesToProcess; tid += blockDim.x * gridDim.x){
            const int indexOfCorrectedSubject = d_indicesOfCorrectedSubjects[tid];

            const bool thisSequenceContainsN = d_readContainsN[indexOfCorrectedSubject];            
            int* const myNumEdits = d_numEditsPerCorrectedSubject + tid;

            if(thisSequenceContainsN){
                *myNumEdits = doNotUseEditsValue;
            }else{
                const int length = d_subjectLengths[indexOfCorrectedSubject];

                //find correct pointers
                const unsigned int* const encodedUncorrectedSequence = d_uncorrectedSubjects + encodedSequencePitchInInts * indexOfCorrectedSubject;
                const char* const decodedCorrectedSequence = d_correctedSubjects + decodedSequencePitchInBytes * indexOfCorrectedSubject;
    
                TempCorrectedSequence::EncodedEdit* const myEdits = (TempCorrectedSequence::EncodedEdit*)(((char*)d_editsPerCorrectedSubject) + editsPitchInBytes * tid);

                const int maxEdits = min(length / 7, numEditsThreshold);
                int edits = 0;
                
                for(int i = 0; i < length && edits <= maxEdits; i++){
                    const char correctedNuc = decodedCorrectedSequence[i];
                    const char uncorrectedNuc = to_nuc(SequenceHelpers::getEncodedNuc2Bit(encodedUncorrectedSequence, length, i));

                    if(correctedNuc != uncorrectedNuc){
                        if(edits < maxEdits){
                            myEdits[edits] = TempCorrectedSequence::EncodedEdit{i, correctedNuc};
                        }
                        edits++;
                    }
                }
                if(edits <= maxEdits){
                    *myNumEdits = edits;
                }else{
                    *myNumEdits = doNotUseEditsValue;
                }
            }
        }
    }




    //####################   KERNEL DISPATCH   ####################


    void call_msaCorrectAnchorsKernel_async(
        char* d_correctedSubjects,
        bool* d_subjectIsCorrected,
        AnchorHighQualityFlag* d_isHighQualitySubject,
        GPUMultiMSA multiMSA,
        const unsigned int* d_subjectSequencesData,
        const unsigned int* d_candidateSequencesData,
        const int* d_candidateSequencesLength,
        const int* d_indices_per_subject,
        const int* d_numAnchors,
        int maxNumAnchors,
        int encodedSequencePitchInInts,
        size_t decodedSequencePitchInBytes,
        int maximumSequenceLength,
        float estimatedErrorrate,
        float desiredAlignmentMaxErrorRate,
        float avg_support_threshold,
        float min_support_threshold,
        float min_coverage_threshold,
        float max_coverage_threshold,
        int k_region,
        int maximum_sequence_length,
        hipStream_t stream,
        KernelLaunchHandle& handle
    ){

        const int max_block_size = 256;
        const int blocksize = std::min(max_block_size, SDIV(maximum_sequence_length, 32) * 32);
        const std::size_t smem = 0;

        int max_blocks_per_device = 1;

        KernelLaunchConfig kernelLaunchConfig;
        kernelLaunchConfig.threads_per_block = blocksize;
        kernelLaunchConfig.smem = smem;

        auto iter = handle.kernelPropertiesMap.find(KernelId::MSACorrectSubjectImplicit);
        if(iter == handle.kernelPropertiesMap.end()){

            std::map<KernelLaunchConfig, KernelProperties> mymap;

            #define getProp(blocksize) { \
                KernelLaunchConfig kernelLaunchConfig; \
                kernelLaunchConfig.threads_per_block = (blocksize); \
                kernelLaunchConfig.smem = 0; \
                KernelProperties kernelProperties; \
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                    msaCorrectAnchorsKernel<(blocksize)>, \
                                                                kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
                mymap[kernelLaunchConfig] = kernelProperties; \
            }

            getProp(32);
            getProp(64);
            getProp(96);
            getProp(128);
            getProp(160);
            getProp(192);
            getProp(224);
            getProp(256);

            const auto& kernelProperties = mymap[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

            handle.kernelPropertiesMap[KernelId::MSACorrectSubjectImplicit] = std::move(mymap);

            #undef getProp
        }else{
            std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
            const KernelProperties& kernelProperties = map[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
        }

        //hipMemsetAsync(d_correctionResultPointers.isHighQualitySubject, 0, n_subjects * sizeof(AnchorHighQualityFlag), stream); CUERR;

        dim3 block(blocksize, 1, 1);
        //dim3 grid(std::min(maxNumAnchors, max_blocks_per_device));
        dim3 grid(max_blocks_per_device);

        #define mycall(blocksize) msaCorrectAnchorsKernel<(blocksize)> \
                                <<<grid, block, 0, stream>>>( \
                                    d_correctedSubjects, \
                                    d_subjectIsCorrected, \
                                    d_isHighQualitySubject, \
                                    multiMSA, \
                                    d_subjectSequencesData, \
                                    d_candidateSequencesData, \
                                    d_candidateSequencesLength, \
                                    d_indices_per_subject, \
                                    d_numAnchors, \
                                    encodedSequencePitchInInts, \
                                    decodedSequencePitchInBytes, \
                                    maximumSequenceLength, \
                                    estimatedErrorrate, \
                                    desiredAlignmentMaxErrorRate, \
                                    avg_support_threshold, \
                                    min_support_threshold, \
                                    min_coverage_threshold, \
                                    max_coverage_threshold, \
                                    k_region \
                                ); CUERR;

        assert(blocksize > 0 && blocksize <= max_block_size);

        switch(blocksize){
            case 32: mycall(32); break;
            case 64: mycall(64); break;
            case 96: mycall(96); break;
            case 128: mycall(128); break;
            case 160: mycall(160); break;
            case 192: mycall(192); break;
            case 224: mycall(224); break;
            case 256: mycall(256); break;
            default: mycall(256); break;
        }
        #undef mycall
    }



    void callFlagCandidatesToBeCorrectedKernel_async(
        bool* d_candidateCanBeCorrected,
        int* d_numCorrectedCandidatesPerAnchor,
        GPUMultiMSA multiMSA,
        const int* d_alignmentShifts,
        const int* d_candidateSequencesLengths,
        const int* d_anchorIndicesOfCandidates,
        const AnchorHighQualityFlag* d_hqflags,
        const int* d_candidatesPerSubjectPrefixsum,
        const int* d_localGoodCandidateIndices,
        const int* d_numLocalGoodCandidateIndicesPerSubject,
        const int* d_numAnchors,
        const int* d_numCandidates,
        float min_support_threshold,
        float min_coverage_threshold,
        int new_columns_to_correct,
        hipStream_t stream,
        KernelLaunchHandle& handle
    ){

        constexpr int blocksize = 256;
        const std::size_t smem = 0;

        int max_blocks_per_device = 1;

        KernelLaunchConfig kernelLaunchConfig;
        kernelLaunchConfig.threads_per_block = blocksize;
        kernelLaunchConfig.smem = smem;

        auto iter = handle.kernelPropertiesMap.find(KernelId::FlagCandidatesToBeCorrected);
        if(iter == handle.kernelPropertiesMap.end()){

            std::map<KernelLaunchConfig, KernelProperties> mymap;

            KernelProperties kernelProperties;

            hipOccupancyMaxActiveBlocksPerMultiprocessor(
                &kernelProperties.max_blocks_per_SM,
                flagCandidatesToBeCorrectedKernel,
                kernelLaunchConfig.threads_per_block, 
                kernelLaunchConfig.smem
            ); CUERR;

            mymap[kernelLaunchConfig] = kernelProperties;

            max_blocks_per_device = handle.deviceProperties.multiProcessorCount 
                                        * kernelProperties.max_blocks_per_SM;

            handle.kernelPropertiesMap[KernelId::FlagCandidatesToBeCorrected] = std::move(mymap);

            #undef getProp
        }else{
            std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
            const KernelProperties& kernelProperties = map[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount 
                                        * kernelProperties.max_blocks_per_SM;
        }



        dim3 block(blocksize);
        dim3 grid(max_blocks_per_device);

        flagCandidatesToBeCorrectedKernel<<<grid, block, 0, stream>>>(
            d_candidateCanBeCorrected,
            d_numCorrectedCandidatesPerAnchor,
            multiMSA,
            d_alignmentShifts,
            d_candidateSequencesLengths,
            d_anchorIndicesOfCandidates,
            d_hqflags,
            d_candidatesPerSubjectPrefixsum,
            d_localGoodCandidateIndices,
            d_numLocalGoodCandidateIndicesPerSubject,
            d_numAnchors,
            d_numCandidates,
            min_support_threshold,
            min_coverage_threshold,
            new_columns_to_correct
        );

        CUERR;

    }



    void callCorrectCandidatesKernel_async(
        char* __restrict__ correctedCandidates,
        TempCorrectedSequence::EncodedEdit* __restrict__ d_editsPerCorrectedCandidate,
        int* __restrict__ d_numEditsPerCorrectedCandidate,
        GPUMultiMSA multiMSA,
        const int* __restrict__ shifts,
        const BestAlignment_t* __restrict__ bestAlignmentFlags,
        const unsigned int* __restrict__ candidateSequencesData,
        const int* __restrict__ candidateSequencesLengths,
        const bool* __restrict__ d_candidateContainsN,
        const int* __restrict__ candidateIndicesOfCandidatesToBeCorrected,
        const int* __restrict__ numCandidatesToBeCorrected,
        const int* __restrict__ anchorIndicesOfCandidates,
        const int* d_numAnchors,
        const int* d_numCandidates,
        int doNotUseEditsValue,
        int numEditsThreshold,
        int encodedSequencePitchInInts,
        size_t decodedSequencePitchInBytes,
        size_t editsPitchInBytes,
        int maximum_sequence_length,
        hipStream_t stream,
        KernelLaunchHandle& handle
    ){

        constexpr int blocksize = 128;
        constexpr int groupsize = 32;

        const size_t dynamicsmemPitchInInts = SDIV(maximum_sequence_length, sizeof(int));
        const size_t smemPitchEditsInInts = SDIV(editsPitchInBytes, sizeof(int));

        auto calculateSmemUsage = [&](int blockDim){
            const int numGroupsPerBlock = blockDim / groupsize;
            std::size_t smem = numGroupsPerBlock * (sizeof(int) * dynamicsmemPitchInInts)
                + numGroupsPerBlock * (sizeof(int) * smemPitchEditsInInts);

            return smem;
        };

        const std::size_t smem = calculateSmemUsage(blocksize);

    	int max_blocks_per_device = 1;

    	KernelLaunchConfig kernelLaunchConfig;
    	kernelLaunchConfig.threads_per_block = blocksize;
    	kernelLaunchConfig.smem = smem;

    	auto iter = handle.kernelPropertiesMap.find(KernelId::MSACorrectCandidates);
    	if(iter == handle.kernelPropertiesMap.end()) {

    		std::map<KernelLaunchConfig, KernelProperties> mymap;

    	    #define getProp(blocksize) { \
                KernelLaunchConfig kernelLaunchConfig; \
                kernelLaunchConfig.threads_per_block = (blocksize); \
                kernelLaunchConfig.smem = calculateSmemUsage((blocksize)); \
                KernelProperties kernelProperties; \
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                    msa_correct_candidates_with_group_kernel<(blocksize), groupsize>, \
                            kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
                mymap[kernelLaunchConfig] = kernelProperties; \
            }

    		getProp(32);
    		getProp(64);
    		getProp(96);
    		getProp(128);
    		getProp(160);
    		getProp(192);
    		getProp(224);
    		getProp(256);

    		const auto& kernelProperties = mymap[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
            
            // std::cerr << "msa_correct_candidates_with_group_kernel "
            //     << "multiProcessorCount = " << handle.deviceProperties.multiProcessorCount
            //     << " max_blocks_per_SM = " << kernelProperties.max_blocks_per_SM << "\n"; 

    		handle.kernelPropertiesMap[KernelId::MSACorrectCandidates] = std::move(mymap);

    	    #undef getProp
    	}else{
    		std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
    		const KernelProperties& kernelProperties = map[kernelLaunchConfig];
    		max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
    	}

    	dim3 block(blocksize, 1, 1);
        //dim3 grid(std::min(max_blocks_per_device, n_candidates * numGroupsPerBlock));
        dim3 grid(max_blocks_per_device);
        
        assert(smem % sizeof(int) == 0);

    	#define mycall(blocksize) msa_correct_candidates_with_group_kernel<(blocksize), groupsize> \
    	        <<<grid, block, smem, stream>>>( \
                    correctedCandidates, \
                    d_editsPerCorrectedCandidate, \
                    d_numEditsPerCorrectedCandidate, \
                    multiMSA, \
                    shifts, \
                    bestAlignmentFlags, \
                    candidateSequencesData, \
                    candidateSequencesLengths, \
                    d_candidateContainsN, \
                    candidateIndicesOfCandidatesToBeCorrected, \
                    numCandidatesToBeCorrected, \
                    anchorIndicesOfCandidates, \
                    d_numAnchors, \
                    d_numCandidates, \
                    doNotUseEditsValue, \
                    numEditsThreshold, \
                    encodedSequencePitchInInts, \
                    decodedSequencePitchInBytes, \
                    editsPitchInBytes, \
                    dynamicsmemPitchInInts \
                ); CUERR;


    	switch(blocksize) {
    	case 32: mycall(32); break;
    	case 64: mycall(64); break;
    	case 96: mycall(96); break;
    	case 128: mycall(128); break;
    	case 160: mycall(160); break;
    	case 192: mycall(192); break;
    	case 224: mycall(224); break;
    	case 256: mycall(256); break;
    	default: mycall(256); break;
    	}

    		#undef mycall 
    }




    void callConstructAnchorResultsKernelAsync(
        TempCorrectedSequence::EncodedEdit* __restrict__ d_editsPerCorrectedSubject,
        int* __restrict__ d_numEditsPerCorrectedSubject,
        int doNotUseEditsValue,
        const int* __restrict__ d_indicesOfCorrectedSubjects,
        const int* __restrict__ d_numIndicesOfCorrectedSubjects,
        const bool* __restrict__ d_readContainsN,
        const unsigned int* __restrict__ d_uncorrectedSubjects,
        const int* __restrict__ d_subjectLengths,
        const char* __restrict__ d_correctedSubjects,
        int numEditsThreshold,
        size_t encodedSequencePitchInInts,
        size_t decodedSequencePitchInBytes,
        size_t editsPitchInBytes,
        const int* d_numAnchors,
        int maxNumAnchors,
        hipStream_t stream,
        KernelLaunchHandle& handle
    ){

        hipMemsetAsync(
            d_editsPerCorrectedSubject, 
            0, 
            editsPitchInBytes * maxNumAnchors, 
            stream
        ); CUERR;

        const int blocksize = 128;
        const std::size_t smem = 0;

        int max_blocks_per_device = 1;

        KernelLaunchConfig kernelLaunchConfig;
        kernelLaunchConfig.threads_per_block = blocksize;
        kernelLaunchConfig.smem = smem;

        auto iter = handle.kernelPropertiesMap.find(KernelId::ConstructAnchorResults);
        if(iter == handle.kernelPropertiesMap.end()){

            std::map<KernelLaunchConfig, KernelProperties> mymap;

            #define getProp(blocksize) { \
                KernelLaunchConfig kernelLaunchConfig; \
                kernelLaunchConfig.threads_per_block = (blocksize); \
                kernelLaunchConfig.smem = 0; \
                KernelProperties kernelProperties; \
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                    constructAnchorResultsKernel, \
                                                                kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
                mymap[kernelLaunchConfig] = kernelProperties; \
            }

            getProp(32);
            getProp(64);
            getProp(96);
            getProp(128);
            getProp(160);
            getProp(192);
            getProp(224);
            getProp(256);

            const auto& kernelProperties = mymap[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

            handle.kernelPropertiesMap[KernelId::ConstructAnchorResults] = std::move(mymap);

            #undef getProp
        }else{
            std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
            const KernelProperties& kernelProperties = map[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
        }

        dim3 block(blocksize, 1, 1);
        dim3 grid(std::min(SDIV(maxNumAnchors, blocksize), max_blocks_per_device));

        #define mycall(blocksize) constructAnchorResultsKernel \
                                <<<grid, block, 0, stream>>>( \
                                        d_editsPerCorrectedSubject, \
                                        d_numEditsPerCorrectedSubject, \
                                        doNotUseEditsValue, \
                                        d_indicesOfCorrectedSubjects, \
                                        d_numIndicesOfCorrectedSubjects, \
                                        d_readContainsN, \
                                        d_uncorrectedSubjects, \
                                        d_subjectLengths, \
                                        d_correctedSubjects, \
                                        numEditsThreshold, \
                                        encodedSequencePitchInInts, \
                                        decodedSequencePitchInBytes, \
                                        editsPitchInBytes); CUERR;

        mycall();

        // switch(blocksize){
        //     case 32: mycall(32); break;
        //     case 64: mycall(64); break;
        //     case 96: mycall(96); break;
        //     case 128: mycall(128); break;
        //     case 160: mycall(160); break;
        //     case 192: mycall(192); break;
        //     case 224: mycall(224); break;
        //     case 256: mycall(256); break;
        //     default: mycall(256); break;
        // }
         #undef mycall
    }











}
}
