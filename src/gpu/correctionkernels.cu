#include "hip/hip_runtime.h"
//#define NDEBUG

#include <gpu/kernels.hpp>
#include <gpu/kernellaunch.hpp>
#include <gpu/devicefunctionsforkernels.cuh>

// //#include <gpu/bestalignment.hpp>
// #include <bestalignment.hpp>
// #include <gpu/utility_kernels.cuh>
#include <gpu/cubcachingallocator.cuh>

// #include <msa.hpp>
#include <sequence.hpp>
// #include <correctionresultprocessing.hpp>

// #include <shiftedhammingdistance_common.hpp>

// #include <hpc_helpers.cuh>
// #include <config.hpp>

// #include <cassert>


#include <hipcub/hipcub.hpp>

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

// #include <thrust/binary_search.h>


namespace care{
namespace gpu{




    template<int BLOCKSIZE>
    __global__
    void msa_correct_subject_implicit_kernel(
                            MSAPointers msapointers,
                            AlignmentResultPointers alignmentresultpointers,
                            ReadSequencesPointers d_sequencePointers,
                            CorrectionResultPointers d_correctionResultPointers,
                            const int* __restrict__ d_indices,
                            const int* __restrict__ d_indices_per_subject,
                            const int* __restrict__ d_candidates_per_subject_prefixsum,
                            int n_subjects,
                            int encodedSequencePitchInInts,
                            size_t sequence_pitch,
                            size_t msa_pitch,
                            size_t msa_weights_pitch,
                            int maximumSequenceLength,
                            float estimatedErrorrate,
                            float desiredAlignmentMaxErrorRate,
                            float avg_support_threshold,
                            float min_support_threshold,
                            float min_coverage_threshold,
                            float max_coverage_threshold,
                            int k_region){

        using BlockReduceBool = hipcub::BlockReduce<bool, BLOCKSIZE>;
        using BlockReduceInt = hipcub::BlockReduce<int, BLOCKSIZE>;
        using BlockReduceFloat = hipcub::BlockReduce<float, BLOCKSIZE>;

        __shared__ union {
            typename BlockReduceBool::TempStorage boolreduce;
            typename BlockReduceInt::TempStorage intreduce;
            typename BlockReduceFloat::TempStorage floatreduce;
        } temp_storage;

        __shared__ bool broadcastbuffer;

        __shared__ int numUncorrectedPositions;
        __shared__ int uncorrectedPositions[BLOCKSIZE];
        __shared__ float avgCountPerWeight[4];

        auto get = [] (const char* data, int length, int index){
            //return Sequence_t::get_as_nucleotide(data, length, index);
            return getEncodedNuc2Bit((const unsigned int*)data, length, index, [](auto i){return i;});
        };

        auto getSubjectPtr = [&] (int subjectIndex){
            const unsigned int* result = d_sequencePointers.subjectSequencesData + std::size_t(subjectIndex) * encodedSequencePitchInInts;
            return result;
        };

        auto getCandidatePtr = [&] (int candidateIndex){
            const unsigned int* result = d_sequencePointers.candidateSequencesData + std::size_t(candidateIndex) * encodedSequencePitchInInts;
            return result;
        };

        auto getCandidateLength = [&](int candidateIndex){
            return d_sequencePointers.candidateSequencesLength[candidateIndex];
        };

        auto isGoodAvgSupport = [&](float avgsupport){
            return fgeq(avgsupport, avg_support_threshold);
        };
        auto isGoodMinSupport = [&](float minsupport){
            return fgeq(minsupport, min_support_threshold);
        };
        auto isGoodMinCoverage = [&](float mincoverage){
            return fgeq(mincoverage, min_coverage_threshold);
        };

        constexpr char A_enc = 0x00;
        constexpr char C_enc = 0x01;
        constexpr char G_enc = 0x02;
        constexpr char T_enc = 0x03;

        auto to_nuc = [](char c){
            switch(c){
            case A_enc: return 'A';
            case C_enc: return 'C';
            case G_enc: return 'G';
            case T_enc: return 'T';
            default: return 'F';
            }
        };

        auto saveUncorrectedPositionInSmem = [&](int pos){
            const int smemindex = atomicAdd(&numUncorrectedPositions, 1);
            uncorrectedPositions[smemindex] = pos;
        };

        const size_t msa_weights_pitch_floats = msa_weights_pitch / sizeof(float);

        for(unsigned subjectIndex = blockIdx.x; subjectIndex < n_subjects; subjectIndex += gridDim.x){
            const int myNumIndices = d_indices_per_subject[subjectIndex];
            if(myNumIndices > 0){

                const float* const my_support = msapointers.support + msa_weights_pitch_floats * subjectIndex;
                const int* const my_coverage = msapointers.coverage + msa_weights_pitch_floats * subjectIndex;
                const int* const my_orig_coverage = msapointers.origCoverages + msa_weights_pitch_floats * subjectIndex;
                const char* const my_consensus = msapointers.consensus + msa_pitch  * subjectIndex;
                char* const my_corrected_subject = d_correctionResultPointers.correctedSubjects + subjectIndex * sequence_pitch;

                const int subjectColumnsBegin_incl = msapointers.msaColumnProperties[subjectIndex].subjectColumnsBegin_incl;
                const int subjectColumnsEnd_excl = msapointers.msaColumnProperties[subjectIndex].subjectColumnsEnd_excl;
                const int lastColumn_excl = msapointers.msaColumnProperties[subjectIndex].lastColumn_excl;

                float avg_support = 0;
                float min_support = 1.0f;
                //int max_coverage = 0;
                int min_coverage = std::numeric_limits<int>::max();

                for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                    assert(i < lastColumn_excl);

                    avg_support += my_support[i];
                    min_support = min(my_support[i], min_support);
                    //max_coverage = max(my_coverage[i], max_coverage);
                    min_coverage = min(my_coverage[i], min_coverage);
                }

                avg_support = BlockReduceFloat(temp_storage.floatreduce).Sum(avg_support);
                __syncthreads();

                min_support = BlockReduceFloat(temp_storage.floatreduce).Reduce(min_support, hipcub::Min());
                __syncthreads();

                //max_coverage = BlockReduceInt(temp_storage.intreduce).Reduce(max_coverage, hipcub::Max());

                min_coverage = BlockReduceInt(temp_storage.intreduce).Reduce(min_coverage, hipcub::Min());
                __syncthreads();

                avg_support /= (subjectColumnsEnd_excl - subjectColumnsBegin_incl);

                bool isHQ = isGoodAvgSupport(avg_support) && isGoodMinSupport(min_support) && isGoodMinCoverage(min_coverage);
                //bool isHQ = true;

                if(threadIdx.x == 0){
                    broadcastbuffer = isHQ;
                    d_correctionResultPointers.isHighQualitySubject[subjectIndex].hq(isHQ);
                    //printf("%f %f %d %d\n", avg_support, min_support, min_coverage, isHQ);
                }
                __syncthreads();

                isHQ = broadcastbuffer;

                if(isHQ){
                    for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                        //assert(my_consensus[i] == 'A' || my_consensus[i] == 'C' || my_consensus[i] == 'G' || my_consensus[i] == 'T');
                        my_corrected_subject[i - subjectColumnsBegin_incl] = my_consensus[i];
                    }
                    if(threadIdx.x == 0){
                        d_correctionResultPointers.subjectIsCorrected[subjectIndex] = true;
                    }
                }else{

                    //decode orignal sequence and copy to corrected sequence
                    const int subjectLength = subjectColumnsEnd_excl - subjectColumnsBegin_incl;
                    const unsigned int* const subject = getSubjectPtr(subjectIndex);
                    for(int i = threadIdx.x; i < subjectLength; i += BLOCKSIZE){
                        my_corrected_subject[i] = to_nuc(get((const char*)subject, subjectLength, i));
                    }

                    bool foundAColumn = false;
                    int* globalUncorrectedPostitionsPtr = d_correctionResultPointers.uncorrected_positions_per_subject + subjectIndex * maximumSequenceLength;
                    int* const globalNumUncorrectedPositionsPtr = d_correctionResultPointers.num_uncorrected_positions_per_subject + subjectIndex;

                    //round up to next multiple of BLOCKSIZE;
                    const int loopIters = SDIV(subjectLength, BLOCKSIZE) * BLOCKSIZE;
                    for(int loopIter = 0; loopIter < loopIters; loopIter++){
                        if(threadIdx.x == 0){
                            numUncorrectedPositions = 0;
                        }
                        __syncthreads();

                        const int i = threadIdx.x + loopIter * BLOCKSIZE;

                        if(i < subjectLength){
                            const int globalIndex = subjectColumnsBegin_incl + i;

                            const int origCoverage = my_orig_coverage[globalIndex];
                            const char origBase = my_corrected_subject[i];
                            const char consensusBase = my_consensus[globalIndex];

                            float maxOverlapWeightOrigBase = 0.0f;
                            float maxOverlapWeightConsensusBase = 0.0f;
                            int origBaseCount = 1;
                            int consensusBaseCount = 0;

                            bool goodOrigOverlapExists = false;

                            const int globalOffset = d_candidates_per_subject_prefixsum[subjectIndex];

                            const int* myIndices = d_indices + globalOffset;

                            for(int candidatenr = 0; candidatenr < myNumIndices; candidatenr++){
                                const int arrayindex = myIndices[candidatenr] + globalOffset;

                                const unsigned int* candidateptr = getCandidatePtr(arrayindex);
                                const int candidateLength = getCandidateLength(arrayindex);
                                const int candidateShift = alignmentresultpointers.shifts[arrayindex];
                                const int candidateBasePosition = globalIndex - (subjectColumnsBegin_incl + candidateShift);
                                if(candidateBasePosition >= 0 && candidateBasePosition < candidateLength){
                                    char candidateBaseEnc = 0xFF;
                                    if(alignmentresultpointers.bestAlignmentFlags[arrayindex] == BestAlignment_t::ReverseComplement){
                                        candidateBaseEnc = get((const char*)candidateptr, candidateLength, candidateLength - candidateBasePosition-1);
                                        candidateBaseEnc = (~candidateBaseEnc) & 0x03;
                                    }else{
                                        candidateBaseEnc = get((const char*)candidateptr, candidateLength, candidateBasePosition);
                                    }
                                    const char candidateBase = to_nuc(candidateBaseEnc);

                                    const int nOps = alignmentresultpointers.nOps[arrayindex];
                                    const int overlapsize = alignmentresultpointers.overlaps[arrayindex];
                                    const float overlapweight = calculateOverlapWeight(subjectLength, nOps, overlapsize);
                                    assert(overlapweight <= 1.0f);
                                    assert(overlapweight >= 0.0f);

                                    constexpr float goodOverlapThreshold = 0.90f;

                                    if(origBase == candidateBase){
                                        maxOverlapWeightOrigBase = max(maxOverlapWeightOrigBase, overlapweight);
                                        origBaseCount++;

                                        if(fgeq(overlapweight, goodOverlapThreshold)){
                                            goodOrigOverlapExists = true;
                                        }
                                    }else{
                                        if(consensusBase == candidateBase){
                                            maxOverlapWeightConsensusBase = max(maxOverlapWeightConsensusBase, overlapweight);
                                            consensusBaseCount++;
                                        }
                                    }
                                }
                            }

                            if(my_support[globalIndex] > 0.5f){

                                constexpr float maxOverlapWeightLowerBound = 0.15f;

                                bool allowCorrectionToConsensus = false;

                                //if(maxOverlapWeightOrigBase < maxOverlapWeightConsensusBase){
                                    allowCorrectionToConsensus = true;
                                //}

                                // if(maxOverlapWeightOrigBase == 0 && maxOverlapWeightConsensusBase == 0){
                                //     //correct to orig;
                                //     allowCorrectionToConsensus = false;
                                // }else if(maxOverlapWeightConsensusBase < maxOverlapWeightLowerBound){
                                //     //correct to orig
                                //     allowCorrectionToConsensus = false;
                                // }else if(maxOverlapWeightOrigBase < maxOverlapWeightLowerBound){
                                //     //correct to consensus
                                //     allowCorrectionToConsensus = true;
                                //     if(origBaseCount < 4){
                                //         allowCorrectionToConsensus = true;
                                //     }
                                // }else if(maxOverlapWeightConsensusBase < maxOverlapWeightOrigBase - 0.2f){
                                //     //maybe correct to orig
                                //     allowCorrectionToConsensus = false;
                                // }else if(maxOverlapWeightConsensusBase  - 0.2f > maxOverlapWeightOrigBase){
                                //     //maybe correct to consensus
                                //     if(origBaseCount < 4){
                                //         allowCorrectionToConsensus = true;
                                //     }
                                // }

                                if(!goodOrigOverlapExists && allowCorrectionToConsensus){

                                    float avgsupportkregion = 0;
                                    int c = 0;
                                    bool kregioncoverageisgood = true;


                                    for(int j = i - k_region/2; j <= i + k_region/2 && kregioncoverageisgood; j++){
                                        if(j != i && j >= 0 && j < subjectLength){
                                            avgsupportkregion += my_support[subjectColumnsBegin_incl + j];
                                            kregioncoverageisgood &= fgeq(my_coverage[subjectColumnsBegin_incl + j], min_coverage_threshold);
                                            //kregioncoverageisgood &= (my_coverage[subjectColumnsBegin_incl + j] >= 1);
                                            c++;
                                        }
                                    }
                                    avgsupportkregion /= c;

                                    if(kregioncoverageisgood && fgeq(avgsupportkregion, 1.0f-4*estimatedErrorrate / 2.0f)){


                                        // constexpr float maxOverlapWeightLowerBound = 0.25f;
                                        //
                                        // bool correctToConsensus = false;//maxOverlapWeightOrigBase < maxOverlapWeightLowerBound;
                                        // // correctToConsensus |= maxOverlapWeightConsensusBase >= maxOverlapWeightOrigBase;
                                        // // correctToConsensus &= !goodOrigOverlapExists;
                                        // if(!goodOrigOverlapExists && (origBase != consensusBase && my_support[globalIndex] > 0.5f)){
                                        //     correctToConsensus = true;
                                        // }

                                        // if(maxOverlapWeightOrigBase == 0 && maxOverlapWeightConsensusBase == 0){
                                        //     //correct to orig;
                                        // }else if(maxOverlapWeightConsensusBase < maxOverlapWeightLowerBound){
                                        //     //correct to orig
                                        // }else if(maxOverlapWeightOrigBase < maxOverlapWeightLowerBound){
                                        //     //correct to consensus
                                        //     my_corrected_subject[i] = consensusBase;
                                        // }else if(maxOverlapWeightConsensusBase < maxOverlapWeightOrigBase){
                                        //     //maybe correct to orig
                                        // }else if(maxOverlapWeightConsensusBase >= maxOverlapWeightOrigBase){
                                        //     //maybe correct to consensus
                                        //     my_corrected_subject[i] = consensusBase;
                                        // }

                                        //if(correctToConsensus){
                                            my_corrected_subject[i] = consensusBase;
                                            foundAColumn = true;
                                        // }else{
                                        //     saveUncorrectedPositionInSmem(i);
                                        // }
                                    }else{
                                        saveUncorrectedPositionInSmem(i);
                                    }
                                }
                            }else{
                                saveUncorrectedPositionInSmem(i);
                            }
                        }

                        __syncthreads();

                        if(threadIdx.x == 0){
                            *globalNumUncorrectedPositionsPtr += numUncorrectedPositions;
                        }

                        for(int k = threadIdx.x; k < numUncorrectedPositions; k += BLOCKSIZE){
                            globalUncorrectedPostitionsPtr[k] = uncorrectedPositions[k];
                        }
                        globalUncorrectedPostitionsPtr += numUncorrectedPositions;

                        if(loopIter < loopIters - 1){
                            __syncthreads();
                        }
                    }

                    //perform block wide or-reduction on foundAColumn
                    foundAColumn = BlockReduceBool(temp_storage.boolreduce).Reduce(foundAColumn, [](bool a, bool b){return a || b;});
                    __syncthreads();

                    if(threadIdx.x == 0){
                        d_correctionResultPointers.subjectIsCorrected[subjectIndex] = true;//foundAColumn;
                    }
                }
            }
        }
    }





    template<int BLOCKSIZE>
    __global__
    void msa_correct_subject_implicit_kernel2(
                            MSAPointers msapointers,
                            AlignmentResultPointers alignmentresultpointers,
                            ReadSequencesPointers d_sequencePointers,
                            CorrectionResultPointers d_correctionResultPointers,
                            const int* __restrict__ d_indices_per_subject,
                            int n_subjects,
                            int encodedSequencePitchInInts,
                            size_t sequence_pitch,
                            size_t msa_pitch,
                            size_t msa_weights_pitch,
                            int maximumSequenceLength,
                            float estimatedErrorrate,
                            float desiredAlignmentMaxErrorRate,
                            float avg_support_threshold,
                            float min_support_threshold,
                            float min_coverage_threshold,
                            float max_coverage_threshold,
                            int k_region,
                            const read_number* readIds){

        using BlockReduceBool = hipcub::BlockReduce<bool, BLOCKSIZE>;
        using BlockReduceInt = hipcub::BlockReduce<int, BLOCKSIZE>;
        using BlockReduceFloat = hipcub::BlockReduce<float, BLOCKSIZE>;

        __shared__ union {
            typename BlockReduceBool::TempStorage boolreduce;
            typename BlockReduceInt::TempStorage intreduce;
            typename BlockReduceFloat::TempStorage floatreduce;
        } temp_storage;

        __shared__ int broadcastbuffer;

        __shared__ int numUncorrectedPositions;
        __shared__ int uncorrectedPositions[BLOCKSIZE];
        __shared__ float avgCountPerWeight[4];

        auto get = [] (const char* data, int length, int index){
            //return Sequence_t::get_as_nucleotide(data, length, index);
            return getEncodedNuc2Bit((const unsigned int*)data, length, index, [](auto i){return i;});
        };

        auto getSubjectPtr = [&] (int subjectIndex){
            const unsigned int* result = d_sequencePointers.subjectSequencesData + std::size_t(subjectIndex) * encodedSequencePitchInInts;
            return result;
        };

        auto getCandidatePtr = [&] (int candidateIndex){
            const unsigned int* result = d_sequencePointers.candidateSequencesData + std::size_t(candidateIndex) * encodedSequencePitchInInts;
            return result;
        };

        auto getCandidateLength = [&](int candidateIndex){
            return d_sequencePointers.candidateSequencesLength[candidateIndex];
        };

        auto isGoodAvgSupport = [&](float avgsupport){
            return fgeq(avgsupport, avg_support_threshold);
        };
        auto isGoodMinSupport = [&](float minsupport){
            return fgeq(minsupport, min_support_threshold);
        };
        auto isGoodMinCoverage = [&](float mincoverage){
            return fgeq(mincoverage, min_coverage_threshold);
        };

        constexpr char A_enc = 0x00;
        constexpr char C_enc = 0x01;
        constexpr char G_enc = 0x02;
        constexpr char T_enc = 0x03;

        auto to_nuc = [](char c){
            switch(c){
            case A_enc: return 'A';
            case C_enc: return 'C';
            case G_enc: return 'G';
            case T_enc: return 'T';
            default: return 'F';
            }
        };

        const size_t msa_weights_pitch_floats = msa_weights_pitch / sizeof(float);

        for(unsigned subjectIndex = blockIdx.x; subjectIndex < n_subjects; subjectIndex += gridDim.x){
            const int myNumIndices = d_indices_per_subject[subjectIndex];
            if(myNumIndices > 0){

                const float* const my_support = msapointers.support + msa_weights_pitch_floats * subjectIndex;
                const int* const my_coverage = msapointers.coverage + msa_weights_pitch_floats * subjectIndex;
                const int* const my_orig_coverage = msapointers.origCoverages + msa_weights_pitch_floats * subjectIndex;
                const char* const my_consensus = msapointers.consensus + msa_pitch  * subjectIndex;
                char* const my_corrected_subject = d_correctionResultPointers.correctedSubjects + subjectIndex * sequence_pitch;

                const int subjectColumnsBegin_incl = msapointers.msaColumnProperties[subjectIndex].subjectColumnsBegin_incl;
                const int subjectColumnsEnd_excl = msapointers.msaColumnProperties[subjectIndex].subjectColumnsEnd_excl;
                const int lastColumn_excl = msapointers.msaColumnProperties[subjectIndex].lastColumn_excl;

                float avg_support = 0;
                float min_support = 1.0f;
                //int max_coverage = 0;
                int min_coverage = std::numeric_limits<int>::max();

                for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                    assert(i < lastColumn_excl);

                    avg_support += my_support[i];
                    min_support = min(my_support[i], min_support);
                    //max_coverage = max(my_coverage[i], max_coverage);
                    min_coverage = min(my_coverage[i], min_coverage);
                }

                avg_support = BlockReduceFloat(temp_storage.floatreduce).Sum(avg_support);
                __syncthreads();

                min_support = BlockReduceFloat(temp_storage.floatreduce).Reduce(min_support, hipcub::Min());
                __syncthreads();

                //max_coverage = BlockReduceInt(temp_storage.intreduce).Reduce(max_coverage, hipcub::Max());

                min_coverage = BlockReduceInt(temp_storage.intreduce).Reduce(min_coverage, hipcub::Min());
                __syncthreads();

                avg_support /= (subjectColumnsEnd_excl - subjectColumnsBegin_incl);


                const float avg_support_threshold = 1.0f-1.0f*estimatedErrorrate;
        		const float min_support_threshold = 1.0f-3.0f*estimatedErrorrate;

                if(threadIdx.x == 0){
                    read_number readId = readIds[subjectIndex];
                    d_correctionResultPointers.subjectIsCorrected[subjectIndex] = true; //canBeCorrected;

                    const bool canBeCorrectedByConsensus = isGoodAvgSupport(avg_support) && isGoodMinSupport(min_support) && isGoodMinCoverage(min_coverage);
                    int flag = 0;

                    if(canBeCorrectedByConsensus){
                        int smallestErrorrateThatWouldMakeHQ = 100;

                        const int estimatedErrorratePercent = ceil(estimatedErrorrate * 100.0f);
                        for(int percent = estimatedErrorratePercent; percent >= 0; percent--){
                            const float factor = percent / 100.0f;
                            const float avg_threshold = 1.0f - 1.0f * factor;
                            const float min_threshold = 1.0f - 3.0f * factor;
                            if(fgeq(avg_support, avg_threshold) && fgeq(min_support, min_threshold)){
                                smallestErrorrateThatWouldMakeHQ = percent;
                            }
                        }

                        const bool isHQ = isGoodMinCoverage(min_coverage)
                                            && fleq(smallestErrorrateThatWouldMakeHQ, estimatedErrorratePercent * 0.5f);

                        //broadcastbuffer = isHQ;
                        d_correctionResultPointers.isHighQualitySubject[subjectIndex].hq(isHQ);

                        flag = isHQ ? 2 : 1;

                        // if(readId == 10307280){
                        //     printf("read 10307280 isHQ %d, min_coverage %d, avg_support %f, min_support %f, smallestErrorrateThatWouldMakeHQ %d, min_coverage_threshold %f\n", 
                        //         isHQ, min_coverage, avg_support, min_support, smallestErrorrateThatWouldMakeHQ, min_coverage_threshold);
                        // }
                    }

                    broadcastbuffer = flag;
                }
                __syncthreads();

                // for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                //     //assert(my_consensus[i] == 'A' || my_consensus[i] == 'C' || my_consensus[i] == 'G' || my_consensus[i] == 'T');
                //     if(my_support[i] > 0.90f && my_orig_coverage[i] <= 2){
                //         my_corrected_subject[i - subjectColumnsBegin_incl] = my_consensus[i];
                //     }else{
                //         const char* subject = getSubjectPtr(subjectIndex);
                //         const char encodedBase = get(subject, subjectColumnsEnd_excl- subjectColumnsBegin_incl, i - subjectColumnsBegin_incl);
                //         const char base = to_nuc(encodedBase);
                //         my_corrected_subject[i - subjectColumnsBegin_incl] = base;
                //     }
                // }

                const int flag = broadcastbuffer;

                if(flag > 0){
                    for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                        const char nuc = my_consensus[i];
                        assert(nuc == 'A' || nuc == 'C' || nuc == 'G' || nuc == 'T');

                        my_corrected_subject[i - subjectColumnsBegin_incl] = my_consensus[i];
                    }
                }else{
                    //correct only positions with high support.
                    for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                        //assert(my_consensus[i] == 'A' || my_consensus[i] == 'C' || my_consensus[i] == 'G' || my_consensus[i] == 'T');
                        if(my_support[i] > 0.90f && my_orig_coverage[i] <= 2){
                            my_corrected_subject[i - subjectColumnsBegin_incl] = my_consensus[i];
                        }else{
                            const unsigned int* subject = getSubjectPtr(subjectIndex);
                            const char encodedBase = get((const char*)subject, subjectColumnsEnd_excl- subjectColumnsBegin_incl, i - subjectColumnsBegin_incl);
                            const char base = to_nuc(encodedBase);
                            assert(base == 'A' || base == 'C' || base == 'G' || base == 'T');
                            my_corrected_subject[i - subjectColumnsBegin_incl] = base;
                        }
                    }
                }
            }else{
                if(threadIdx.x == 0){
                    d_correctionResultPointers.isHighQualitySubject[subjectIndex].hq(false);
                    d_correctionResultPointers.subjectIsCorrected[subjectIndex] = false;
                }
            }
        }
    }







    __device__ __forceinline__
    bool checkIfCandidateShouldBeCorrected(const MSAPointers& d_msapointers,
                        const AlignmentResultPointers& d_alignmentresultpointers,
                        const ReadSequencesPointers& d_sequencePointers,
                        const CorrectionResultPointers& d_correctionResultPointers,
                        const int* __restrict__ d_indices,
                        const int* __restrict__ d_candidates_per_subject_prefixsum,
                        size_t msa_weights_pitch_floats,
                        float min_support_threshold,
                        float min_coverage_threshold,
                        int new_columns_to_correct,
                        int subjectIndex,
                        int local_goodcandidate_index){

        const float* const my_support = d_msapointers.support + msa_weights_pitch_floats * subjectIndex;
        const int* const my_coverage = d_msapointers.coverage + msa_weights_pitch_floats * subjectIndex;

        const int globalOffset = d_candidates_per_subject_prefixsum[subjectIndex];
        const int* const my_indices = d_indices + globalOffset;

        const int subjectColumnsBegin_incl = d_msapointers.msaColumnProperties[subjectIndex].subjectColumnsBegin_incl;
        const int subjectColumnsEnd_excl = d_msapointers.msaColumnProperties[subjectIndex].subjectColumnsEnd_excl;
        const int lastColumn_excl = d_msapointers.msaColumnProperties[subjectIndex].lastColumn_excl;

        const int localCandidateIndex = my_indices[local_goodcandidate_index];
        const int global_candidate_index = localCandidateIndex + globalOffset;

        const int shift = d_alignmentresultpointers.shifts[global_candidate_index];
        const int candidate_length = d_sequencePointers.candidateSequencesLength[global_candidate_index];
        const int queryColumnsBegin_incl = subjectColumnsBegin_incl + shift;
        const int queryColumnsEnd_excl = subjectColumnsBegin_incl + shift + candidate_length;

        if(subjectColumnsBegin_incl - new_columns_to_correct <= queryColumnsBegin_incl
           && queryColumnsBegin_incl <= subjectColumnsBegin_incl + new_columns_to_correct
           && queryColumnsEnd_excl <= subjectColumnsEnd_excl + new_columns_to_correct) {

            float newColMinSupport = 1.0f;
            int newColMinCov = std::numeric_limits<int>::max();
            //check new columns left of subject
            for(int columnindex = subjectColumnsBegin_incl - new_columns_to_correct;
                columnindex < subjectColumnsBegin_incl;
                columnindex++) {

                assert(columnindex < lastColumn_excl);
                if(queryColumnsBegin_incl <= columnindex) {
                    newColMinSupport = my_support[columnindex] < newColMinSupport ? my_support[columnindex] : newColMinSupport;
                    newColMinCov = my_coverage[columnindex] < newColMinCov ? my_coverage[columnindex] : newColMinCov;
                }
            }
            //check new columns right of subject
            for(int columnindex = subjectColumnsEnd_excl;
                    columnindex < subjectColumnsEnd_excl + new_columns_to_correct
                        && columnindex < lastColumn_excl;
                    columnindex++) {

                newColMinSupport = my_support[columnindex] < newColMinSupport ? my_support[columnindex] : newColMinSupport;
                newColMinCov = my_coverage[columnindex] < newColMinCov ? my_coverage[columnindex] : newColMinCov;
            }

            bool result = fgeq(newColMinSupport, min_support_threshold)
                            && fgeq(newColMinCov, min_coverage_threshold);

            //return result;
            return true;
        }else{
            return false;
        }

    }

    __device__ __forceinline__
    bool checkIfCandidateShouldBeCorrectedGlobal(
            const float* __restrict__ support,
            const int* __restrict__ coverages,
            const MSAColumnProperties* __restrict__ msaColumnProperties,
            const int* __restrict__ alignmentShifts,
            const int* __restrict__ candidateSequencesLengths,
            size_t msa_weights_pitch_floats,
            float min_support_threshold,
            float min_coverage_threshold,
            int new_columns_to_correct,
            int subjectIndex,
            int global_candidate_index){

        const float* const my_support = support + msa_weights_pitch_floats * subjectIndex;
        const int* const my_coverage = coverages + msa_weights_pitch_floats * subjectIndex;

        const int subjectColumnsBegin_incl = msaColumnProperties[subjectIndex].subjectColumnsBegin_incl;
        const int subjectColumnsEnd_excl = msaColumnProperties[subjectIndex].subjectColumnsEnd_excl;
        const int lastColumn_excl = msaColumnProperties[subjectIndex].lastColumn_excl;

        const int shift = alignmentShifts[global_candidate_index];
        const int candidate_length = candidateSequencesLengths[global_candidate_index];
        const int queryColumnsBegin_incl = subjectColumnsBegin_incl + shift;
        const int queryColumnsEnd_excl = subjectColumnsBegin_incl + shift + candidate_length;

        if(subjectColumnsBegin_incl - new_columns_to_correct <= queryColumnsBegin_incl
           && queryColumnsBegin_incl <= subjectColumnsBegin_incl + new_columns_to_correct
           && queryColumnsEnd_excl <= subjectColumnsEnd_excl + new_columns_to_correct) {

            float newColMinSupport = 1.0f;
            int newColMinCov = std::numeric_limits<int>::max();
            //check new columns left of subject
            for(int columnindex = subjectColumnsBegin_incl - new_columns_to_correct;
                columnindex < subjectColumnsBegin_incl;
                columnindex++) {

                assert(columnindex < lastColumn_excl);
                if(queryColumnsBegin_incl <= columnindex) {
                    newColMinSupport = my_support[columnindex] < newColMinSupport ? my_support[columnindex] : newColMinSupport;
                    newColMinCov = my_coverage[columnindex] < newColMinCov ? my_coverage[columnindex] : newColMinCov;
                }
            }
            //check new columns right of subject
            for(int columnindex = subjectColumnsEnd_excl;
                    columnindex < subjectColumnsEnd_excl + new_columns_to_correct
                        && columnindex < lastColumn_excl;
                    columnindex++) {

                newColMinSupport = my_support[columnindex] < newColMinSupport ? my_support[columnindex] : newColMinSupport;
                newColMinCov = my_coverage[columnindex] < newColMinCov ? my_coverage[columnindex] : newColMinCov;
            }

            bool result = fgeq(newColMinSupport, min_support_threshold)
                            && fgeq(newColMinCov, min_coverage_threshold);

            //return result;
            return true;
        }else{
            return false;
        }

    }


    template<int blocksize, int tilesize>
    __global__
    void getNumCorrectedCandidatesPerAnchorKernel(
            int* __restrict__ d_numIndicesPerAnchor,
            const bool* __restrict__ d_isCorrectedCandidate,
            const int* __restrict__ d_numGoodIndicesPerSubject,
            const int* __restrict__ d_candidates_per_subject_prefixsum,
            const int* __restrict__ d_anchorIndicesOfCandidates,
            int numAnchors,
            int numCandidates){

        static_assert(blocksize % tilesize == 0);
        static_assert(tilesize == 32);

        constexpr int numTilesPerBlock = blocksize / tilesize;

        const int numTiles = (gridDim.x * blocksize) / tilesize;
        const int tileId = (threadIdx.x + blockIdx.x * blocksize) / tilesize;
        const int tileIdInBlock = threadIdx.x / tilesize;

        __shared__ int counts[numTilesPerBlock];


        auto tile = cg::tiled_partition<tilesize>(cg::this_thread_block());

        for(int anchorIndex = tileId; anchorIndex < numAnchors; anchorIndex += numTiles){

            const int offset = d_candidates_per_subject_prefixsum[anchorIndex];
            int* const numIndicesPtr = d_numIndicesPerAnchor + anchorIndex;

            const int numCandidatesForAnchor = d_numGoodIndicesPerSubject[anchorIndex];

            if(tile.thread_rank() == 0){
                counts[tileIdInBlock] = 0;
            }
            tile.sync();

            for(int localCandidateIndex = tile.thread_rank(); 
                    localCandidateIndex < numCandidatesForAnchor; 
                    localCandidateIndex += tile.size()){
                
                const int globalCandidateIndex = localCandidateIndex + offset;
                const bool isCorrected = d_isCorrectedCandidate[globalCandidateIndex];

                if(isCorrected){
                    cg::coalesced_group g = cg::coalesced_threads();
                    if (g.thread_rank() == 0) {
                        atomicAdd(&counts[tileIdInBlock], g.size());
                    }
                }
            }

            tile.sync();
            if(tile.thread_rank() == 0){
                atomicAdd(numIndicesPtr, counts[tileIdInBlock]);
            }

        }

    }

    template<int BLOCKSIZE>
    __global__
    void compactCandidateCorrectionResultsKernel(
            char* __restrict__ compactedCorrectedCandidates,
            TempCorrectedSequence::Edit* __restrict__ compactedEditsPerCorrectedCandidate,
            const int* __restrict__ numCorrectedCandidatesPerAnchor,
            const int* __restrict__ numCorrectedCandidatesPerAnchorPrefixsum, //exclusive
            const int* __restrict__ high_quality_subject_indices,
            const int* __restrict__ num_high_quality_subject_indices,
            const int* __restrict__ candidates_per_subject_prefixsum,
            const char* __restrict__ correctedCandidates,
            const int* __restrict__ correctedCandidateLengths,
            const TempCorrectedSequence::Edit* __restrict__ editsPerCorrectedCandidate,
            size_t decodedSequencePitch,
            int numEditsThreshold,
            int n_subjects){

        constexpr int groupsize = 32;
        static_assert(groupsize <= 32);
        static_assert(BLOCKSIZE % groupsize == 0);

        const int numHqSubjects = *num_high_quality_subject_indices;

        auto tgroup = cg::tiled_partition<groupsize>(cg::this_thread_block());

        const int numGroups = (gridDim.x * blockDim.x) / groupsize;
        const int groupId = (threadIdx.x + blockIdx.x * blockDim.x) / groupsize;
        const int groupIdInBlock = threadIdx.x / groupsize;

        for(int hqsubjectIndex = groupId;
                hqsubjectIndex < numHqSubjects;
                hqsubjectIndex += numGroups){

            const int subjectIndex = high_quality_subject_indices[hqsubjectIndex];
            const int myNumCorrectedCandidates = numCorrectedCandidatesPerAnchor[subjectIndex];

            if(myNumCorrectedCandidates > 0){
                const int inputbaseoffset = candidates_per_subject_prefixsum[subjectIndex];
                const int outputbaseoffset = numCorrectedCandidatesPerAnchorPrefixsum[subjectIndex];

                for(int cIndex = tgroup.thread_rank(); cIndex < myNumCorrectedCandidates; cIndex += tgroup.size()){

                    for(int i = 0; i < decodedSequencePitch; i++){
                        compactedCorrectedCandidates[(outputbaseoffset+cIndex) * decodedSequencePitch + i] 
                            = correctedCandidates[(inputbaseoffset+cIndex) * decodedSequencePitch + i];
                    }

                    for(int i = 0; i < numEditsThreshold; i++){
                        compactedEditsPerCorrectedCandidate[(outputbaseoffset+cIndex) * numEditsThreshold + i] 
                            = editsPerCorrectedCandidate[(inputbaseoffset+cIndex) * numEditsThreshold + i];
                    }
                }
            }
        }
    }


    __global__ 
    void flagCandidatesToBeCorrectedKernel(
            bool* __restrict__ candidateCanBeCorrected,
            int* __restrict__ numCorrectedCandidatesPerAnchor,
            const float* __restrict__ support,
            const int* __restrict__ coverages,
            const MSAColumnProperties* __restrict__ msaColumnProperties,
            const int* __restrict__ alignmentShifts,
            const int* __restrict__ candidateSequencesLengths,
            const int* __restrict__ anchorIndicesOfCandidates,
            const AnchorHighQualityFlag* __restrict__ hqflags,
            const int* __restrict__ numCandidatesPerSubjectPrefixsum,
            const int* __restrict__ localGoodCandidateIndices,
            const int* __restrict__ numLocalGoodCandidateIndicesPerSubject,
            size_t msa_weights_pitch_floats,
            float min_support_threshold,
            float min_coverage_threshold,
            int new_columns_to_correct,
            int n_subjects,
            int n_candidates){

        for(int anchorIndex = blockIdx.x; 
                anchorIndex < n_subjects; 
                anchorIndex += blockDim.x * gridDim.x){

            const bool isHighQualitySubject = hqflags[anchorIndex].hq();
            const int numGoodIndices = numLocalGoodCandidateIndicesPerSubject[anchorIndex];
            const int dataoffset = numCandidatesPerSubjectPrefixsum[anchorIndex];
            const int* myGoodIndices = localGoodCandidateIndices + dataoffset;

            if(isHighQualitySubject){

                for(int tid = threadIdx.x; tid < numGoodIndices; tid += blockDim.x){
                    const int localCandidateIndex = myGoodIndices[tid];
                    const int globalCandidateIndex = dataoffset + localCandidateIndex;

                    const bool canHandleCandidate = checkIfCandidateShouldBeCorrectedGlobal(
                        support,
                        coverages,
                        msaColumnProperties,
                        alignmentShifts,
                        candidateSequencesLengths,
                        msa_weights_pitch_floats,
                        min_support_threshold,
                        min_coverage_threshold,
                        new_columns_to_correct,
                        anchorIndex,
                        globalCandidateIndex
                    );

                    candidateCanBeCorrected[globalCandidateIndex] = canHandleCandidate;

                    if(canHandleCandidate){
                        atomicAdd(numCorrectedCandidatesPerAnchor + anchorIndex, 1);
                    }
                }
                
            }
        }

        // for(int candidateIndex = threadIdx.x + blockIdx.x * blockDim.x; 
        //         candidateIndex < n_candidates; 
        //         candidateIndex += blockDim.x * gridDim.x){
            
        //     const int anchorIndex = anchorIndicesOfCandidates[candidateIndex];
        //     const bool isHighQualitySubject = hqflags[anchorIndex].hq();

        //     if(isHighQualitySubject){

        //         const bool canHandleCandidate = checkIfCandidateShouldBeCorrectedGlobal(
        //             support,
        //             coverages,
        //             msaColumnProperties,
        //             alignmentShifts,
        //             candidateSequencesLengths,
        //             msa_weights_pitch_floats,
        //             min_support_threshold,
        //             min_coverage_threshold,
        //             new_columns_to_correct,
        //             anchorIndex,
        //             candidateIndex
        //         );

        //         candidateCanBeCorrected[candidateIndex] = canHandleCandidate;

        //         if(canHandleCandidate){
        //             atomicAdd(numCorrectedCandidatesPerAnchor + anchorIndex, 1);
        //         }
        //     }else{
        //         candidateCanBeCorrected[candidateIndex] = false;
        //     }
        // }
    }

    template<int BLOCKSIZE, int groupsize>
    __global__
    void msa_correct_candidates_with_group_kernel2(
            char* __restrict__ correctedCandidates,
            TempCorrectedSequence::Edit* __restrict__ d_editsPerCorrectedCandidate,
            int* __restrict__ d_numEditsPerCorrectedCandidate,
            const MSAColumnProperties* __restrict__ msaColumnProperties,
            const char* __restrict__ consensus,
            const float* __restrict__ support,
            const int* __restrict__ shifts,
            const BestAlignment_t* __restrict__ bestAlignmentFlags,
            const unsigned int* __restrict__ candidateSequencesData,
            const int* __restrict__ candidateSequencesLengths,
            const bool* __restrict__ d_candidateContainsN,
            const int* __restrict__ candidateIndicesOfCandidatesToBeCorrected,
            const int* __restrict__ numCandidatesToBeCorrected,
            const int* __restrict__ anchorIndicesOfCandidates,
            int doNotUseEditsValue,
            int numEditsThreshold,
            int n_subjects,
            int n_queries,
            int encodedSequencePitchInInts,
            size_t sequence_pitch,
            size_t msa_pitch,
            size_t msa_weights_pitch,
            size_t dynamicsmemPitchInInts){

        /*
            Use groupsize threads per candidate to perform correction
        */
        static_assert(BLOCKSIZE % groupsize == 0, "BLOCKSIZE % groupsize != 0");
        constexpr int groupsPerBlock = BLOCKSIZE / groupsize;


        auto make_unpacked_reverse_complement_inplace = [] (std::uint8_t* sequence, int sequencelength){
            return reverseComplementStringInplace((char*)sequence, sequencelength);
        };

        auto decodedReverseComplementInplaceGroup = [](auto group, char* sequence, int sequencelength){
            auto make_reverse_complement_nuc = [](char in){
                switch(in){
                    case 'A': return 'T';
                    case 'C': return 'G';
                    case 'G': return 'C';
                    case 'T': return 'A';
                    default :return 'F';
                }
            };

            for(int i = group.thread_rank(); i < sequencelength/2; i += group.size()){
                const std::uint8_t front = make_reverse_complement_nuc(sequence[i]);
                const std::uint8_t back = make_reverse_complement_nuc(sequence[sequencelength - 1 - i]);
                sequence[i] = back;
                sequence[sequencelength - 1 - i] = front;
            }

            if(sequencelength % 2 == 1 && group.thread_rank() == 0){
                const int middleindex = sequencelength/2;
                sequence[middleindex] = make_reverse_complement_nuc(sequence[middleindex]);
            }
        };

        auto getEncodedNucFromInt2Bit = [](unsigned int data, int pos){
            return ((data >> (30 - 2*pos)) & 0x00000003);
        };

        auto to_nuc = [](char c){
            constexpr char A_enc = 0x00;
            constexpr char C_enc = 0x01;
            constexpr char G_enc = 0x02;
            constexpr char T_enc = 0x03;

            switch(c){
            case A_enc: return 'A';
            case C_enc: return 'C';
            case G_enc: return 'G';
            case T_enc: return 'T';
            default: return 'F';
            }
        };

        __shared__ int shared_numEditsOfCandidate[groupsPerBlock];

        extern __shared__ int dynamicsmem[]; // for sequences



        auto tgroup = cg::tiled_partition<groupsize>(cg::this_thread_block());

        const int numGroups = (gridDim.x * blockDim.x) / groupsize;
        const int groupId = (threadIdx.x + blockIdx.x * blockDim.x) / groupsize;
        const int groupIdInBlock = threadIdx.x / groupsize;

        char* const shared_correctedCandidate = (char*)(dynamicsmem + dynamicsmemPitchInInts * groupIdInBlock);


        const size_t msa_weights_pitch_floats = msa_weights_pitch / sizeof(float);
        const int loopEnd = *numCandidatesToBeCorrected;

        for(int id = groupId;
                id < loopEnd;
                id += numGroups){

            const int candidateIndex = candidateIndicesOfCandidatesToBeCorrected[id];
            const int subjectIndex = anchorIndicesOfCandidates[candidateIndex];
            const int destinationIndex = id;

            char* const my_corrected_candidate = correctedCandidates + destinationIndex * sequence_pitch;
            const int candidate_length = candidateSequencesLengths[candidateIndex];

            const int shift = shifts[candidateIndex];
            const int subjectColumnsBegin_incl = msaColumnProperties[subjectIndex].subjectColumnsBegin_incl;
            const int queryColumnsBegin_incl = subjectColumnsBegin_incl + shift;
            const int queryColumnsEnd_excl = subjectColumnsBegin_incl + shift + candidate_length;

            const BestAlignment_t bestAlignmentFlag = bestAlignmentFlags[candidateIndex];

            const char* const my_consensus = consensus + msa_pitch * subjectIndex;

            if(tgroup.thread_rank() == 0){                        
                shared_numEditsOfCandidate[groupIdInBlock] = 0;
            }
            tgroup.sync();          

            
            

            const int copyposbegin = queryColumnsBegin_incl; //max(queryColumnsBegin_incl, subjectColumnsBegin_incl);
            const int copyposend = queryColumnsEnd_excl; //min(queryColumnsEnd_excl, subjectColumnsEnd_excl);
            assert(copyposend - copyposbegin == candidate_length);

            for(int i = copyposbegin + tgroup.thread_rank(); i < copyposend; i += tgroup.size()) {
                shared_correctedCandidate[i - queryColumnsBegin_incl] = my_consensus[i];
            }

            //const float* const my_support = support + msa_weights_pitch_floats * subjectIndex;
            

            // for(int i = copyposbegin; i < copyposend; i += 1) {
            //     //assert(my_consensus[i] == 'A' || my_consensus[i] == 'C' || my_consensus[i] == 'G' || my_consensus[i] == 'T');
            //     if(my_support[i] > 0.90f){
            //         my_corrected_candidates[destinationindex * sequence_pitch + (i - queryColumnsBegin_incl)] = my_consensus[i];
            //     }else{
            //         const char encodedBase = get(candidate, queryColumnsEnd_excl- queryColumnsBegin_incl, i - queryColumnsBegin_incl);
            //         const char base = to_nuc(encodedBase);
            //         my_corrected_candidates[destinationindex * sequence_pitch + (i - queryColumnsBegin_incl)] = base;
            //     }
            // }

            

            //the forward strand will be returned -> make reverse complement again
            if(bestAlignmentFlag == BestAlignment_t::ReverseComplement) {
                tgroup.sync(); // threads may access elements in shared memory which were written by another thread
                decodedReverseComplementInplaceGroup(tgroup, shared_correctedCandidate, candidate_length);
                tgroup.sync();
            }

            //copy from smem to global output
            for(int i = tgroup.thread_rank(); i < candidate_length; i += tgroup.size()) {
                my_corrected_candidate[i] = shared_correctedCandidate[i];
            }            

            //compare corrected candidate with uncorrected candidate, calculate edits   
            
            const unsigned int* const encUncorrectedCandidate = candidateSequencesData 
                        + std::size_t(candidateIndex) * encodedSequencePitchInInts;
            const bool thisSequenceContainsN = d_candidateContainsN[candidateIndex];            
            int* const myNumEdits = d_numEditsPerCorrectedCandidate + candidateIndex;
            TempCorrectedSequence::Edit* const myEdits = d_editsPerCorrectedCandidate + destinationIndex * numEditsThreshold;

            if(thisSequenceContainsN){
                if(tgroup.thread_rank() == 0){
                    *myNumEdits = doNotUseEditsValue;
                }
            }else{
                const int maxEdits = min(candidate_length / 7, numEditsThreshold);

                const int fullInts = candidate_length / 16;
                

                for(int i = 0; i < fullInts; i++){
                    const unsigned int encodedDataInt = encUncorrectedCandidate[i];

                    //compare with 16 bases of corrected sequence

                    for(int k = tgroup.thread_rank(); k < 16; k += tgroup.size()){
                        const int posInInt = k;
                        const int posInSequence = i * 16 + posInInt;
                        const char encodedUncorrectedNuc = getEncodedNucFromInt2Bit(encodedDataInt, posInInt);
                        const char correctedNuc = shared_correctedCandidate[posInSequence];

                        if(correctedNuc != to_nuc(encodedUncorrectedNuc)){
                            cg::coalesced_group g = cg::coalesced_threads();

                            int currentNumEdits = 0;
                            if(g.thread_rank() == 0){
                                currentNumEdits = atomicAdd(&shared_numEditsOfCandidate[groupIdInBlock], g.size());
                            }
                            currentNumEdits = g.shfl(currentNumEdits, 0);

                            if(currentNumEdits + g.size() <= maxEdits){
                                const int myEditOutputPos = g.thread_rank() + currentNumEdits;
                                myEdits[myEditOutputPos] = TempCorrectedSequence::Edit{posInSequence, correctedNuc};
                            }
                        }
                    }

                    tgroup.sync();

                    if(shared_numEditsOfCandidate[groupIdInBlock] > maxEdits){
                        break;
                    }
                }

                //process remaining positions
                if(shared_numEditsOfCandidate[groupIdInBlock] <= maxEdits){
                    const int remainingPositions = candidate_length - 16 * fullInts;
                    if(remainingPositions > 0){
                        const unsigned int encodedDataInt = encUncorrectedCandidate[fullInts];
                        for(int posInInt = tgroup.thread_rank(); posInInt < remainingPositions; posInInt += tgroup.size()){
                            const int posInSequence = fullInts * 16 + posInInt;
                            const char encodedUncorrectedNuc = getEncodedNucFromInt2Bit(encodedDataInt, posInInt);
                            const char correctedNuc = shared_correctedCandidate[posInSequence];

                            if(correctedNuc != to_nuc(encodedUncorrectedNuc)){
                                cg::coalesced_group g = cg::coalesced_threads();
                                
                                int currentNumEdits = 0;
                                if(g.thread_rank() == 0){
                                    currentNumEdits = atomicAdd(&shared_numEditsOfCandidate[groupIdInBlock], g.size());
                                }
                                currentNumEdits = g.shfl(currentNumEdits, 0);

                                if(currentNumEdits + g.size() <= maxEdits){
                                    const int myEditOutputPos = g.thread_rank() + currentNumEdits;
                                    if(myEditOutputPos < maxEdits){
                                        myEdits[myEditOutputPos] = TempCorrectedSequence::Edit{posInSequence, correctedNuc};
                                    }
                                }
                            }
                        }
                    }
                }

                tgroup.sync();

                if(tgroup.thread_rank() == 0){                            
                    if(shared_numEditsOfCandidate[groupIdInBlock] <= maxEdits){
                        *myNumEdits = shared_numEditsOfCandidate[groupIdInBlock];
                    }else{
                        *myNumEdits = doNotUseEditsValue;
                    }
                }
            }
            

            tgroup.sync(); //sync before handling next candidate
            
            //printf("subjectIndex %d global_candidate_index %d\n", subjectIndex, global_candidate_index);


            
        }
    }





    __global__
    void constructAnchorResultsKernel(
            TempCorrectedSequence::Edit* __restrict__ d_editsPerCorrectedSubject,
            int* __restrict__ d_numEditsPerCorrectedSubject,
            int doNotUseEditsValue,
            const int* __restrict__ d_indicesOfCorrectedSubjects,
            const int* __restrict__ d_numIndicesOfCorrectedSubjects,
            const bool* __restrict__ d_readContainsN,
            const unsigned int* __restrict__ d_uncorrectedSubjects,
            const int* __restrict__ d_subjectLengths,
            const char* __restrict__ d_correctedSubjects,
            int numEditsThreshold,
            size_t encodedSequencePitchInInts,
            size_t decodedSequencePitchInBytes){

        auto get = [] (const unsigned int* data, int length, int index, auto trafo){
            return getEncodedNuc2Bit(data, length, index, trafo);
        };
        
        auto getEncodedNucFromInt2Bit = [](unsigned int data, int pos){
            return ((data >> (30 - 2*pos)) & 0x00000003);
        };

        auto to_nuc = [](char c){
            constexpr char A_enc = 0x00;
            constexpr char C_enc = 0x01;
            constexpr char G_enc = 0x02;
            constexpr char T_enc = 0x03;
            
            switch(c){
            case A_enc: return 'A';
            case C_enc: return 'C';
            case G_enc: return 'G';
            case T_enc: return 'T';
            default: return 'F';
            }
        };

        const int numIndicesToProcess = *d_numIndicesOfCorrectedSubjects;

        for(int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < numIndicesToProcess; tid += blockDim.x * gridDim.x){
            const int indexOfCorrectedSubject = d_indicesOfCorrectedSubjects[tid];

            const bool thisSequenceContainsN = d_readContainsN[indexOfCorrectedSubject];            
            int* const myNumEdits = d_numEditsPerCorrectedSubject + tid;

            if(thisSequenceContainsN){
                *myNumEdits = doNotUseEditsValue;
            }else{
                const int length = d_subjectLengths[indexOfCorrectedSubject];

                //find correct pointers
                const unsigned int* const encodedUncorrectedSequence = d_uncorrectedSubjects + encodedSequencePitchInInts * indexOfCorrectedSubject;
                const char* const decodedCorrectedSequence = d_correctedSubjects + decodedSequencePitchInBytes * indexOfCorrectedSubject;
    
                TempCorrectedSequence::Edit* const myEdits = d_editsPerCorrectedSubject + numEditsThreshold * tid;

                const int maxEdits = min(length / 7, numEditsThreshold);
                int edits = 0;
                
                for(int i = 0; i < length && edits <= maxEdits; i++){
                    const char correctedNuc = decodedCorrectedSequence[i];
                    const char uncorrectedNuc = to_nuc(get(encodedUncorrectedSequence, length, i, [](auto i){return i;}));

                    if(correctedNuc != uncorrectedNuc){
                        if(edits < maxEdits){
                            myEdits[edits] = TempCorrectedSequence::Edit{i, correctedNuc};
                        }
                        edits++;
                    }
                }
                if(edits <= maxEdits){
                    *myNumEdits = edits;
                }else{
                    *myNumEdits = doNotUseEditsValue;
                }
            }
        }
    }



    template<int BLOCKSIZE>
    __global__
    void msaCorrectSubjectKernelWithOrigMismatchPositions(
                            MSAPointers msapointers,
                            AlignmentResultPointers alignmentresultpointers,
                            ReadSequencesPointers d_sequencePointers,
                            CorrectionResultPointers d_correctionResultPointers,
                            const int* __restrict__ d_indices,
                            const int* __restrict__ d_indices_per_subject,
                            int n_subjects,
                            int encodedSequencePitchInInts,
                            size_t sequence_pitch,
                            size_t msa_pitch,
                            size_t msa_weights_pitch,
                            int maximumSequenceLength,
                            float estimatedErrorrate,
                            float desiredAlignmentMaxErrorRate,
                            float avg_support_threshold,
                            float min_support_threshold,
                            float min_coverage_threshold,
                            float max_coverage_threshold,
                            int k_region){

        using BlockReduceBool = hipcub::BlockReduce<bool, BLOCKSIZE>;
        using BlockReduceInt = hipcub::BlockReduce<int, BLOCKSIZE>;
        using BlockReduceFloat = hipcub::BlockReduce<float, BLOCKSIZE>;

        __shared__ union {
            typename BlockReduceBool::TempStorage boolreduce;
            typename BlockReduceInt::TempStorage intreduce;
            typename BlockReduceFloat::TempStorage floatreduce;
        } temp_storage;

        __shared__ int broadcastbuffer;

        __shared__ int numUncorrectedPositions;
        __shared__ int uncorrectedPositions[BLOCKSIZE];
        __shared__ float avgCountPerWeight[4];

        auto get = [] (const char* data, int length, int index){
            //return Sequence_t::get_as_nucleotide(data, length, index);
            return getEncodedNuc2Bit((const unsigned int*)data, length, index, [](auto i){return i;});
        };

        auto getSubjectPtr = [&] (int subjectIndex){
            const unsigned int* result = d_sequencePointers.subjectSequencesData + std::size_t(subjectIndex) * encodedSequencePitchInInts;
            return result;
        };

        auto getCandidatePtr = [&] (int candidateIndex){
            const unsigned int* result = d_sequencePointers.candidateSequencesData + std::size_t(candidateIndex) * encodedSequencePitchInInts;
            return result;
        };

        auto getCandidateLength = [&](int candidateIndex){
            return d_sequencePointers.candidateSequencesLength[candidateIndex];
        };

        auto isGoodAvgSupport = [&](float avgsupport){
            return fgeq(avgsupport, avg_support_threshold);
        };
        auto isGoodMinSupport = [&](float minsupport){
            return fgeq(minsupport, min_support_threshold);
        };
        auto isGoodMinCoverage = [&](float mincoverage){
            return fgeq(mincoverage, min_coverage_threshold);
        };

        constexpr char A_enc = 0x00;
        constexpr char C_enc = 0x01;
        constexpr char G_enc = 0x02;
        constexpr char T_enc = 0x03;

        auto to_nuc = [](char c){
            switch(c){
            case A_enc: return 'A';
            case C_enc: return 'C';
            case G_enc: return 'G';
            case T_enc: return 'T';
            default: return 'F';
            }
        };

        const size_t msa_weights_pitch_floats = msa_weights_pitch / sizeof(float);

        for(unsigned subjectIndex = blockIdx.x; subjectIndex < n_subjects; subjectIndex += gridDim.x){
            const int myNumIndices = d_indices_per_subject[subjectIndex];
            if(myNumIndices > 0){

                const float* const my_support = msapointers.support + msa_weights_pitch_floats * subjectIndex;
                const int* const my_coverage = msapointers.coverage + msa_weights_pitch_floats * subjectIndex;
                const int* const my_orig_coverage = msapointers.origCoverages + msa_weights_pitch_floats * subjectIndex;
                const char* const my_consensus = msapointers.consensus + msa_pitch  * subjectIndex;
                char* const my_corrected_subject = d_correctionResultPointers.correctedSubjects + subjectIndex * sequence_pitch;

                const int subjectColumnsBegin_incl = msapointers.msaColumnProperties[subjectIndex].subjectColumnsBegin_incl;
                const int subjectColumnsEnd_excl = msapointers.msaColumnProperties[subjectIndex].subjectColumnsEnd_excl;
                const int lastColumn_excl = msapointers.msaColumnProperties[subjectIndex].lastColumn_excl;

                float avg_support = 0;
                float min_support = 1.0f;
                //int max_coverage = 0;
                int min_coverage = std::numeric_limits<int>::max();

                for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                    assert(i < lastColumn_excl);

                    avg_support += my_support[i];
                    min_support = min(my_support[i], min_support);
                    //max_coverage = max(my_coverage[i], max_coverage);
                    min_coverage = min(my_coverage[i], min_coverage);
                }

                avg_support = BlockReduceFloat(temp_storage.floatreduce).Sum(avg_support);
                __syncthreads();

                min_support = BlockReduceFloat(temp_storage.floatreduce).Reduce(min_support, hipcub::Min());
                __syncthreads();

                //max_coverage = BlockReduceInt(temp_storage.intreduce).Reduce(max_coverage, hipcub::Max());

                min_coverage = BlockReduceInt(temp_storage.intreduce).Reduce(min_coverage, hipcub::Min());
                __syncthreads();

                avg_support /= (subjectColumnsEnd_excl - subjectColumnsBegin_incl);


                const float avg_support_threshold = 1.0f-1.0f*estimatedErrorrate;
        		const float min_support_threshold = 1.0f-3.0f*estimatedErrorrate;

                if(threadIdx.x == 0){
                    d_correctionResultPointers.subjectIsCorrected[subjectIndex] = true; //canBeCorrected;

                    const bool canBeCorrectedByConsensus = isGoodAvgSupport(avg_support) && isGoodMinSupport(min_support) && isGoodMinCoverage(min_coverage);
                    int flag = 0;

                    if(canBeCorrectedByConsensus){
                        int smallestErrorrateThatWouldMakeHQ = 100;

                        const int estimatedErrorratePercent = ceil(estimatedErrorrate * 100.0f);
                        for(int percent = estimatedErrorratePercent; percent >= 0; percent--){
                            float factor = percent / 100.0f;
                            if(avg_support >= 1.0f - 1.0f * factor && min_support >= 1.0f - 3.0f * factor){
                                smallestErrorrateThatWouldMakeHQ = percent;
                            }
                        }

                        const bool isHQ = isGoodMinCoverage(min_coverage)
                                            && fleq(smallestErrorrateThatWouldMakeHQ, estimatedErrorratePercent * 0.5f);

                        //broadcastbuffer = isHQ;
                        d_correctionResultPointers.isHighQualitySubject[subjectIndex].hq(isHQ);

                        flag = isHQ ? 2 : 1;
                    }

                    broadcastbuffer = flag;
                }
                __syncthreads();

                // for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                //     //assert(my_consensus[i] == 'A' || my_consensus[i] == 'C' || my_consensus[i] == 'G' || my_consensus[i] == 'T');
                //     if(my_support[i] > 0.90f && my_orig_coverage[i] <= 2){
                //         my_corrected_subject[i - subjectColumnsBegin_incl] = my_consensus[i];
                //     }else{
                //         const char* subject = getSubjectPtr(subjectIndex);
                //         const char encodedBase = get(subject, subjectColumnsEnd_excl- subjectColumnsBegin_incl, i - subjectColumnsBegin_incl);
                //         const char base = to_nuc(encodedBase);
                //         my_corrected_subject[i - subjectColumnsBegin_incl] = base;
                //     }
                // }

                const int flag = broadcastbuffer;

                if(flag > 0){
                    for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                        my_corrected_subject[i - subjectColumnsBegin_incl] = my_consensus[i];
                    }
                }else{
                    //correct only positions with high support.
    #if 1                    
                    const int iterations = SDIV(subjectColumnsEnd_excl-subjectColumnsBegin_incl, BLOCKSIZE);

                    for(int iter = 0; iter < iterations; iter++){
                        const int begin = iter * BLOCKSIZE;
                        const int end = min(subjectColumnsEnd_excl-subjectColumnsBegin_incl, (iter+1) * BLOCKSIZE);

                        char editBase = 'F';
                        int editPos = -1;

                        if(threadIdx.x < end - begin){
                            const int i = subjectColumnsBegin_incl + begin + threadIdx.x;

                            const unsigned int* subject = getSubjectPtr(subjectIndex);
                            const char encodedBase = get((const char*)subject, subjectColumnsEnd_excl- subjectColumnsBegin_incl, i - subjectColumnsBegin_incl);
                            const char base = to_nuc(encodedBase);

                            if(my_support[i] > 0.90f && my_orig_coverage[i] <= 2){
                                editBase = my_consensus[i];
                                my_corrected_subject[i - subjectColumnsBegin_incl] = editBase;
                                if(editBase != base){
                                    editPos = i - subjectColumnsBegin_incl;
                                }
                            }else{                                
                                my_corrected_subject[i - subjectColumnsBegin_incl] = base;
                            }
                        }
                    }
    #else 

                    for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                        //assert(my_consensus[i] == 'A' || my_consensus[i] == 'C' || my_consensus[i] == 'G' || my_consensus[i] == 'T');
                        if(my_support[i] > 0.90f && my_orig_coverage[i] <= 2){
                            my_corrected_subject[i - subjectColumnsBegin_incl] = my_consensus[i];
                        }else{
                            const unsigned int* subject = getSubjectPtr(subjectIndex);
                            const char encodedBase = get((const char*)subject, subjectColumnsEnd_excl- subjectColumnsBegin_incl, i - subjectColumnsBegin_incl);
                            const char base = to_nuc(encodedBase);
                            my_corrected_subject[i - subjectColumnsBegin_incl] = base;
                        }
                    }


    #endif
                }
            }else{
                if(threadIdx.x == 0){
                    d_correctionResultPointers.isHighQualitySubject[subjectIndex].hq(false);
                    d_correctionResultPointers.subjectIsCorrected[subjectIndex] = false;
                }
            }
        }
    }










    //####################   KERNEL DISPATCH   ####################

    

    void callGetNumCorrectedCandidatesPerAnchorKernel(
            int* d_numIndicesPerAnchor,
            const bool* d_isCorrectedCandidate,
            const int* d_numGoodIndicesPerSubject,
            const int* d_candidates_per_subject_prefixsum,
            const int* d_anchorIndicesOfCandidates,
            int numAnchors,
            int numCandidates,
            hipStream_t stream,
            KernelLaunchHandle& handle){

        constexpr int blocksize = 128;
        constexpr int tilesize = 32;

        const std::size_t smem = 0;

        int max_blocks_per_device = 1;

        KernelLaunchConfig kernelLaunchConfig;
        kernelLaunchConfig.threads_per_block = blocksize;
        kernelLaunchConfig.smem = smem;

        auto iter = handle.kernelPropertiesMap.find(KernelId::GetNumCorrectedCandidatesPerAnchor);
        if(iter == handle.kernelPropertiesMap.end()){

            std::map<KernelLaunchConfig, KernelProperties> mymap;

            #define getProp(blocksize) { \
                KernelLaunchConfig kernelLaunchConfig; \
                kernelLaunchConfig.threads_per_block = (blocksize); \
                kernelLaunchConfig.smem = 0; \
                KernelProperties kernelProperties; \
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                    getNumCorrectedCandidatesPerAnchorKernel<(blocksize), tilesize>, \
                                                                kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
                mymap[kernelLaunchConfig] = kernelProperties; \
            }

            getProp(32);
            getProp(64);
            getProp(96);
            getProp(128);
            getProp(160);
            getProp(192);
            getProp(224);
            getProp(256);

            const auto& kernelProperties = mymap[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

            handle.kernelPropertiesMap[KernelId::GetNumCorrectedCandidatesPerAnchor] = std::move(mymap);

            #undef getProp
        }else{
            std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
            const KernelProperties& kernelProperties = map[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
        }

        hipMemsetAsync(d_numIndicesPerAnchor, 0, numAnchors * sizeof(int), stream); CUERR;

        dim3 block(blocksize, 1, 1);
        dim3 grid(std::min(SDIV(numCandidates, blocksize), max_blocks_per_device));

        getNumCorrectedCandidatesPerAnchorKernel<blocksize, tilesize><<<grid, block, 0, stream>>>(
            d_numIndicesPerAnchor,
            d_isCorrectedCandidate,
            d_numGoodIndicesPerSubject,
            d_candidates_per_subject_prefixsum,
            d_anchorIndicesOfCandidates,
            numAnchors,
            numCandidates
        );
    }




    void call_msa_correct_subject_implicit_kernel_async(
                            MSAPointers d_msapointers,
                            AlignmentResultPointers d_alignmentresultpointers,
                            ReadSequencesPointers d_sequencePointers,
                            CorrectionResultPointers d_correctionResultPointers,
                            const int* d_indices,
                            const int* d_indices_per_subject,
                            int n_subjects,
                            int encodedSequencePitchInInts,
                            size_t sequence_pitch,
                            size_t msa_pitch,
                            size_t msa_weights_pitch,
                            int maximumSequenceLength,
                            float estimatedErrorrate,
                            float desiredAlignmentMaxErrorRate,
                            float avg_support_threshold,
                            float min_support_threshold,
                            float min_coverage_threshold,
                            float max_coverage_threshold,
                            int k_region,
                            int maximum_sequence_length,
                            const read_number* readIds,
                            hipStream_t stream,
                            KernelLaunchHandle& handle){

        const int max_block_size = 256;
        const int blocksize = std::min(max_block_size, SDIV(maximum_sequence_length, 32) * 32);
        const std::size_t smem = 0;

        int max_blocks_per_device = 1;

        KernelLaunchConfig kernelLaunchConfig;
        kernelLaunchConfig.threads_per_block = blocksize;
        kernelLaunchConfig.smem = smem;

        auto iter = handle.kernelPropertiesMap.find(KernelId::MSACorrectSubjectImplicit);
        if(iter == handle.kernelPropertiesMap.end()){

            std::map<KernelLaunchConfig, KernelProperties> mymap;

            #define getProp(blocksize) { \
                KernelLaunchConfig kernelLaunchConfig; \
                kernelLaunchConfig.threads_per_block = (blocksize); \
                kernelLaunchConfig.smem = 0; \
                KernelProperties kernelProperties; \
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                    msa_correct_subject_implicit_kernel2<(blocksize)>, \
                                                                kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
                mymap[kernelLaunchConfig] = kernelProperties; \
            }

            getProp(32);
            getProp(64);
            getProp(96);
            getProp(128);
            getProp(160);
            getProp(192);
            getProp(224);
            getProp(256);

            const auto& kernelProperties = mymap[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

            handle.kernelPropertiesMap[KernelId::MSACorrectSubjectImplicit] = std::move(mymap);

            #undef getProp
        }else{
            std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
            const KernelProperties& kernelProperties = map[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
        }

        hipMemsetAsync(d_correctionResultPointers.isHighQualitySubject, 0, n_subjects * sizeof(AnchorHighQualityFlag), stream); CUERR;

        dim3 block(blocksize, 1, 1);
        dim3 grid(std::min(n_subjects, max_blocks_per_device));

        #define mycall(blocksize) msa_correct_subject_implicit_kernel2<(blocksize)> \
                                <<<grid, block, 0, stream>>>( \
                                    d_msapointers, \
                                    d_alignmentresultpointers, \
                                    d_sequencePointers, \
                                    d_correctionResultPointers, \
                                    d_indices_per_subject, \
                                    n_subjects, \
                                    encodedSequencePitchInInts, \
                                    sequence_pitch, \
                                    msa_pitch, \
                                    msa_weights_pitch, \
                                    maximumSequenceLength, \
                                    estimatedErrorrate, \
                                    desiredAlignmentMaxErrorRate, \
                                    avg_support_threshold, \
                                    min_support_threshold, \
                                    min_coverage_threshold, \
                                    max_coverage_threshold, \
                                    k_region, \
                                    readIds \
                                ); CUERR;

        assert(blocksize > 0 && blocksize <= max_block_size);

        switch(blocksize){
            case 32: mycall(32); break;
            case 64: mycall(64); break;
            case 96: mycall(96); break;
            case 128: mycall(128); break;
            case 160: mycall(160); break;
            case 192: mycall(192); break;
            case 224: mycall(224); break;
            case 256: mycall(256); break;
            default: mycall(256); break;
        }
        #undef mycall
    }



    void callFlagCandidatesToBeCorrectedKernel_async(
            bool* __restrict__ d_candidateCanBeCorrected,
            int* __restrict__ d_numCorrectedCandidatesPerAnchor,
            const float* __restrict__ d_support,
            const int* __restrict__ d_coverages,
            const MSAColumnProperties* __restrict__ d_msaColumnProperties,
            const int* __restrict__ d_alignmentShifts,
            const int* __restrict__ d_candidateSequencesLengths,
            const int* __restrict__ d_anchorIndicesOfCandidates,
            const AnchorHighQualityFlag* __restrict__ d_hqflags,
            const int* __restrict__ candidatesPerSubjectPrefixsum,
            const int* __restrict__ localGoodCandidateIndices,
            const int* __restrict__ numLocalGoodCandidateIndicesPerSubject,
            size_t msa_weights_pitch_floats,
            float min_support_threshold,
            float min_coverage_threshold,
            int new_columns_to_correct,
            int n_subjects,
            int n_candidates,
            hipStream_t stream,
            KernelLaunchHandle& handle){

        hipMemsetAsync(
            d_numCorrectedCandidatesPerAnchor, 
            0, 
            sizeof(int) * n_subjects, 
            stream
        ); CUERR;

        hipMemsetAsync(
            d_candidateCanBeCorrected, 
            0, 
            sizeof(bool) * n_candidates, 
            stream
        ); CUERR;

        constexpr int blocksize = 256;

        dim3 block(blocksize);
        dim3 grid(n_subjects);

        flagCandidatesToBeCorrectedKernel<<<grid, block, 0, stream>>>(
            d_candidateCanBeCorrected,
            d_numCorrectedCandidatesPerAnchor,
            d_support,
            d_coverages,
            d_msaColumnProperties,
            d_alignmentShifts,
            d_candidateSequencesLengths,
            d_anchorIndicesOfCandidates,
            d_hqflags,
            candidatesPerSubjectPrefixsum,
            localGoodCandidateIndices,
            numLocalGoodCandidateIndicesPerSubject,
            msa_weights_pitch_floats,
            min_support_threshold,
            min_coverage_threshold,
            new_columns_to_correct,
            n_subjects,
            n_candidates
        );

    }



    void callCorrectCandidatesWithGroupKernel2_async(
            char* __restrict__ correctedCandidates,
            TempCorrectedSequence::Edit* __restrict__ d_editsPerCorrectedCandidate,
            int* __restrict__ d_numEditsPerCorrectedCandidate,
            const MSAColumnProperties* __restrict__ msaColumnProperties,
            const char* __restrict__ consensus,
            const float* __restrict__ support,
            const int* __restrict__ shifts,
            const BestAlignment_t* __restrict__ bestAlignmentFlags,
            const unsigned int* __restrict__ candidateSequencesData,
            const int* __restrict__ candidateSequencesLengths,
            const bool* __restrict__ d_candidateContainsN,
            const int* __restrict__ candidateIndicesOfCandidatesToBeCorrected,
            const int* __restrict__ numCandidatesToBeCorrected,
            const int* __restrict__ anchorIndicesOfCandidates,
            int doNotUseEditsValue,
            int numEditsThreshold,
            int n_subjects,
            int n_candidates,
            int encodedSequencePitchInInts,
            size_t sequence_pitch,
            size_t msa_pitch,
            size_t msa_weights_pitch,
            int maximum_sequence_length,
            hipStream_t stream,
            KernelLaunchHandle& handle){


        constexpr int blocksize = 128;
        constexpr int groupsize = 32;
        constexpr int numGroupsPerBlock = blocksize / groupsize;

        const size_t dynamicsmemPitchInInts = SDIV(maximum_sequence_length, sizeof(int));
    	const std::size_t smem = numGroupsPerBlock * sizeof(int) * dynamicsmemPitchInInts;

    	int max_blocks_per_device = 1;

    	KernelLaunchConfig kernelLaunchConfig;
    	kernelLaunchConfig.threads_per_block = blocksize;
    	kernelLaunchConfig.smem = smem;

    	auto iter = handle.kernelPropertiesMap.find(KernelId::MSACorrectCandidates);
    	if(iter == handle.kernelPropertiesMap.end()) {

    		std::map<KernelLaunchConfig, KernelProperties> mymap;

    	    #define getProp(blocksize) { \
                KernelLaunchConfig kernelLaunchConfig; \
                kernelLaunchConfig.threads_per_block = (blocksize); \
                kernelLaunchConfig.smem = numGroupsPerBlock * sizeof(char) * (SDIV(maximum_sequence_length, 4) * 4); \
                KernelProperties kernelProperties; \
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                            msa_correct_candidates_with_group_kernel2<(blocksize), groupsize>, \
                            kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
                mymap[kernelLaunchConfig] = kernelProperties; \
            }

    		getProp(32);
    		getProp(64);
    		getProp(96);
    		getProp(128);
    		getProp(160);
    		getProp(192);
    		getProp(224);
    		getProp(256);

    		const auto& kernelProperties = mymap[kernelLaunchConfig];
    		max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

    		handle.kernelPropertiesMap[KernelId::MSACorrectCandidates] = std::move(mymap);

    	    #undef getProp
    	}else{
    		std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
    		const KernelProperties& kernelProperties = map[kernelLaunchConfig];
    		max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
    	}

    	dim3 block(blocksize, 1, 1);
        dim3 grid(std::min(max_blocks_per_device, n_candidates * numGroupsPerBlock));
        
        assert(smem % sizeof(int) == 0);

    	#define mycall(blocksize) msa_correct_candidates_with_group_kernel2<(blocksize), groupsize> \
    	        <<<grid, block, smem, stream>>>( \
                    correctedCandidates, \
                    d_editsPerCorrectedCandidate, \
                    d_numEditsPerCorrectedCandidate, \
                    msaColumnProperties, \
                    consensus, \
                    support, \
                    shifts, \
                    bestAlignmentFlags, \
                    candidateSequencesData, \
                    candidateSequencesLengths, \
                    d_candidateContainsN, \
                    candidateIndicesOfCandidatesToBeCorrected, \
                    numCandidatesToBeCorrected, \
                    anchorIndicesOfCandidates, \
                    doNotUseEditsValue, \
                    numEditsThreshold, \
                    n_subjects, \
                    n_candidates, \
                    encodedSequencePitchInInts, \
                    sequence_pitch, \
                    msa_pitch, \
                    msa_weights_pitch, \
                    dynamicsmemPitchInInts \
                ); CUERR;


    	switch(blocksize) {
    	case 32: mycall(32); break;
    	case 64: mycall(64); break;
    	case 96: mycall(96); break;
    	case 128: mycall(128); break;
    	case 160: mycall(160); break;
    	case 192: mycall(192); break;
    	case 224: mycall(224); break;
    	case 256: mycall(256); break;
    	default: mycall(256); break;
    	}

    		#undef mycall
    }





    void callCompactCandidateCorrectionResultsKernel_async(
            char* __restrict__ d_compactedCorrectedCandidates, 
            TempCorrectedSequence::Edit* __restrict__ d_compactedEditsPerCorrectedCandidate,
            const int* __restrict__ d_numCorrectedCandidatesPerAnchor,
            const int* __restrict__ d_numCorrectedCandidatesPerAnchorPrefixsum, //exclusive
            const int* __restrict__ d_high_quality_subject_indices,
            const int* __restrict__ d_num_high_quality_subject_indices,
            const int* __restrict__ d_candidates_per_subject_prefixsum,
            const char* __restrict__ d_correctedCandidates,
            const int* __restrict__ d_correctedCandidateLengths,
            const TempCorrectedSequence::Edit* __restrict__ d_editsPerCorrectedCandidate,
            size_t decodedSequencePitch,
            int numEditsThreshold,
            int n_subjects,
            hipStream_t stream,
            KernelLaunchHandle& /*handle*/){

        constexpr int blocksize = 256;

        dim3 block(blocksize);
        dim3 grid(SDIV(n_subjects, blocksize / 32));

        compactCandidateCorrectionResultsKernel<blocksize><<<grid, block, 0, stream>>>(
            d_compactedCorrectedCandidates,
            d_compactedEditsPerCorrectedCandidate,
            d_numCorrectedCandidatesPerAnchor,
            d_numCorrectedCandidatesPerAnchorPrefixsum,
            d_high_quality_subject_indices,
            d_num_high_quality_subject_indices,
            d_candidates_per_subject_prefixsum,
            d_correctedCandidates,
            d_correctedCandidateLengths,
            d_editsPerCorrectedCandidate,
            decodedSequencePitch,
            numEditsThreshold,
            n_subjects
        );
    }






    void callConstructAnchorResultsKernelAsync(
            TempCorrectedSequence::Edit* __restrict__ d_editsPerCorrectedSubject,
            int* __restrict__ d_numEditsPerCorrectedSubject,
            int doNotUseEditsValue,
            const int* __restrict__ d_indicesOfCorrectedSubjects,
            const int* __restrict__ d_numIndicesOfCorrectedSubjects,
            const bool* __restrict__ d_readContainsN,
            const unsigned int* __restrict__ d_uncorrectedSubjects,
            const int* __restrict__ d_subjectLengths,
            const char* __restrict__ d_correctedSubjects,
            int numEditsThreshold,
            size_t encodedSequencePitchInInts,
            size_t decodedSequencePitchInBytes,
            int numSubjects,
            hipStream_t stream,
            KernelLaunchHandle& handle){

        hipMemsetAsync(d_editsPerCorrectedSubject, 0, sizeof(TempCorrectedSequence::Edit) * numSubjects, stream);

        const int blocksize = 128;
        const std::size_t smem = 0;

        int max_blocks_per_device = 1;

        KernelLaunchConfig kernelLaunchConfig;
        kernelLaunchConfig.threads_per_block = blocksize;
        kernelLaunchConfig.smem = smem;

        auto iter = handle.kernelPropertiesMap.find(KernelId::ConstructAnchorResults);
        if(iter == handle.kernelPropertiesMap.end()){

            std::map<KernelLaunchConfig, KernelProperties> mymap;

            #define getProp(blocksize) { \
                KernelLaunchConfig kernelLaunchConfig; \
                kernelLaunchConfig.threads_per_block = (blocksize); \
                kernelLaunchConfig.smem = 0; \
                KernelProperties kernelProperties; \
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                    constructAnchorResultsKernel, \
                                                                kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
                mymap[kernelLaunchConfig] = kernelProperties; \
            }

            getProp(32);
            getProp(64);
            getProp(96);
            getProp(128);
            getProp(160);
            getProp(192);
            getProp(224);
            getProp(256);

            const auto& kernelProperties = mymap[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

            handle.kernelPropertiesMap[KernelId::ConstructAnchorResults] = std::move(mymap);

            #undef getProp
        }else{
            std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
            const KernelProperties& kernelProperties = map[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
        }

        dim3 block(blocksize, 1, 1);
        dim3 grid(std::min(SDIV(numSubjects, blocksize), max_blocks_per_device));

        #define mycall(blocksize) constructAnchorResultsKernel \
                                <<<grid, block, 0, stream>>>( \
                                        d_editsPerCorrectedSubject, \
                                        d_numEditsPerCorrectedSubject, \
                                        doNotUseEditsValue, \
                                        d_indicesOfCorrectedSubjects, \
                                        d_numIndicesOfCorrectedSubjects, \
                                        d_readContainsN, \
                                        d_uncorrectedSubjects, \
                                        d_subjectLengths, \
                                        d_correctedSubjects, \
                                        numEditsThreshold, \
                                        encodedSequencePitchInInts, \
                                        decodedSequencePitchInBytes); CUERR;

        mycall();

        // switch(blocksize){
        //     case 32: mycall(32); break;
        //     case 64: mycall(64); break;
        //     case 96: mycall(96); break;
        //     case 128: mycall(128); break;
        //     case 160: mycall(160); break;
        //     case 192: mycall(192); break;
        //     case 224: mycall(224); break;
        //     case 256: mycall(256); break;
        //     default: mycall(256); break;
        // }
         #undef mycall
    }











}
}
