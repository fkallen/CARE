#include "hip/hip_runtime.h"

#include <config.hpp>
#include <sequence.hpp>
#include <minhasher.hpp>
#include <readstorage.hpp>
#include <options.hpp>
#include <cpu_alignment.hpp>
#include <bestalignment.hpp>
#include <msa.hpp>

#include <hpc_helpers.cuh>

#include <algorithm>
#include <array>
#include <cstdint>
#include <vector>
#include <iostream>
#include <string>
#include <memory>
#include <mutex>
#include <numeric>

#include <gpu/readextender_gpu.hpp>
#include <extensionresultprocessing.hpp>
#include <rangegenerator.hpp>
#include <threadpool.hpp>
#include <memoryfile.hpp>
#include <util.hpp>
#include <filehelpers.hpp>

#include <omp.h>


namespace care{
namespace gpu{






MemoryFileFixedSize<ExtendedRead> 
//std::vector<ExtendedRead>
extend_gpu_pairedend(
    const GoodAlignmentProperties& goodAlignmentProperties,
    const CorrectionOptions& correctionOptions,
    const ExtensionOptions& extensionOptions,
    const RuntimeOptions& runtimeOptions,
    const FileOptions& fileOptions,
    const MemoryOptions& memoryOptions,
    const SequenceFileProperties& sequenceFileProperties,
    const GpuMinhasher& minhasher,
    const gpu::DistributedReadStorage& gpuReadStorage
){
    const auto rsMemInfo = gpuReadStorage.getMemoryInfo();
    const auto mhMemInfo = minhasher.getMemoryInfo();

    std::size_t memoryAvailableBytesHost = memoryOptions.memoryTotalLimit;
    if(memoryAvailableBytesHost > rsMemInfo.host){
        memoryAvailableBytesHost -= rsMemInfo.host;
    }else{
        memoryAvailableBytesHost = 0;
    }
    if(memoryAvailableBytesHost > mhMemInfo.host){
        memoryAvailableBytesHost -= mhMemInfo.host;
    }else{
        memoryAvailableBytesHost = 0;
    }

    std::unique_ptr<std::uint8_t[]> correctionStatusFlagsPerRead = std::make_unique<std::uint8_t[]>(sequenceFileProperties.nReads);

    #pragma omp parallel for
    for(read_number i = 0; i < sequenceFileProperties.nReads; i++){
        correctionStatusFlagsPerRead[i] = 0;
    }

    std::cerr << "correctionStatusFlagsPerRead bytes: " << sizeof(std::uint8_t) * sequenceFileProperties.nReads / 1024. / 1024. << " MB\n";

    if(memoryAvailableBytesHost > sizeof(std::uint8_t) * sequenceFileProperties.nReads){
        memoryAvailableBytesHost -= sizeof(std::uint8_t) * sequenceFileProperties.nReads;
    }else{
        memoryAvailableBytesHost = 0;
    }

    const std::size_t availableMemoryInBytes = memoryAvailableBytesHost; //getAvailableMemoryInKB() * 1024;
    std::size_t memoryForPartialResultsInBytes = 0;

    if(availableMemoryInBytes > 3*(std::size_t(1) << 30)){
        memoryForPartialResultsInBytes = availableMemoryInBytes - 3*(std::size_t(1) << 30);
    }

    const std::string tmpfilename{fileOptions.tempdirectory + "/" + "MemoryFileFixedSizetmp"};
    MemoryFileFixedSize<ExtendedRead> partialResults(memoryForPartialResultsInBytes, tmpfilename);

    std::vector<ExtendedRead> resultExtendedReads;

    cpu::RangeGenerator<read_number> readIdGenerator(sequenceFileProperties.nReads);
    //cpu::RangeGenerator<read_number> readIdGenerator(1000000);

    BackgroundThread outputThread(true);

    const std::uint64_t totalNumReadPairs = sequenceFileProperties.nReads / 2;

    auto showProgress = [&](auto totalCount, auto seconds){
        if(runtimeOptions.showProgress){

            printf("Processed %10u of %10lu read pairs (Runtime: %03d:%02d:%02d)\r",
                    totalCount, totalNumReadPairs,
                    int(seconds / 3600),
                    int(seconds / 60) % 60,
                    int(seconds) % 60);
            std::cout.flush();
        }

        if(totalCount == totalNumReadPairs){
            std::cerr << '\n';
        }
    };

    auto updateShowProgressInterval = [](auto duration){
        return duration;
    };

    ProgressThread<read_number> progressThread(totalNumReadPairs, showProgress, updateShowProgressInterval);

    
    const int insertSize = extensionOptions.insertSize;
    const int insertSizeStddev = extensionOptions.insertSizeStddev;
    const int maximumSequenceLength = sequenceFileProperties.maxSequenceLength;
    const std::size_t encodedSequencePitchInInts = getEncodedNumInts2Bit(maximumSequenceLength);

    std::mutex verboseMutex;
    std::mutex ompCriticalMutex;

    std::int64_t totalNumSuccess0 = 0;
    std::int64_t totalNumSuccess1 = 0;
    std::int64_t totalNumSuccess01 = 0;
    std::int64_t totalNumSuccessRead = 0;

    std::map<int, int> totalExtensionLengthsMap;

    std::map<int, int> totalMismatchesBetweenMateExtensions;

    //omp_set_num_threads(1);

    #pragma omp parallel
    {
        const int numDeviceIds = runtimeOptions.deviceIds.size();

        assert(numDeviceIds > 0);

        const int ompThreadId = omp_get_thread_num();
        const int deviceId = runtimeOptions.deviceIds.at(ompThreadId % numDeviceIds);
        hipSetDevice(deviceId); CUERR;

        GoodAlignmentProperties goodAlignmentProperties2 = goodAlignmentProperties;
        //goodAlignmentProperties2.maxErrorRate = 0.05;

        constexpr int maxextensionPerStep = 20;

        ReadExtenderGpu readExtenderGpu{
            insertSize,
            insertSizeStddev,
            maxextensionPerStep,
            maximumSequenceLength,
            gpuReadStorage, 
            minhasher,
            correctionOptions,
            goodAlignmentProperties2
        };

        std::int64_t numSuccess0 = 0;
        std::int64_t numSuccess1 = 0;
        std::int64_t numSuccess01 = 0;
        std::int64_t numSuccessRead = 0;

        std::map<int, int> extensionLengthsMap;
        std::map<int, int> mismatchesBetweenMateExtensions;

        auto gatherHandleSequences = gpuReadStorage.makeGatherHandleSequences();

        const int batchsizePairs = correctionOptions.batchsize;

        SimpleAllocationPinnedHost<read_number> currentIds(2 * batchsizePairs);
        SimpleAllocationPinnedHost<unsigned int> currentEncodedReads(2 * encodedSequencePitchInInts * batchsizePairs);
        SimpleAllocationPinnedHost<int> currentReadLengths(2 * batchsizePairs);

        hipStream_t stream;
        hipStreamCreate(&stream); CUERR;
        

        while(!(readIdGenerator.empty())){

            auto readIdsEnd = readIdGenerator.next_n_into_buffer(
                batchsizePairs * 2, 
                currentIds.get()
            );

            const int numReadsInBatch = std::distance(currentIds.get(), readIdsEnd);

            if(numReadsInBatch % 2 == 1){
                throw std::runtime_error("Input files not properly paired. Aborting read extension.");
            }
            
            if(numReadsInBatch == 0){
                continue; //this should only happen if all reads have been processed
            }

            gpuReadStorage.gatherSequenceDataToGpuBufferAsync(
                nullptr,
                gatherHandleSequences,
                currentEncodedReads.get(), //device acccessible
                encodedSequencePitchInInts,
                currentIds.get(),
                currentIds.get(), //device accessible
                numReadsInBatch,
                deviceId,
                stream
            ); CUERR;
    
            gpuReadStorage.gatherSequenceLengthsToGpuBufferAsync(
                currentReadLengths.get(), //device accessible
                deviceId,
                currentIds.get(), //device accessible
                numReadsInBatch,    
                stream
            ); CUERR;

            hipStreamSynchronize(stream);

            const int numReadPairsInBatch = numReadsInBatch / 2;

            std::vector<ReadExtenderGpu::ExtendInput> inputs(numReadPairsInBatch); 

            for(int i = 0; i < numReadPairsInBatch; i++){
                auto& input = inputs[i];

                input.readId1 = currentIds[2*i];
                input.readId2 = currentIds[2*i+1];
                input.encodedRead1 = currentEncodedReads.get() + (2*i) * encodedSequencePitchInInts;
                input.encodedRead2 = currentEncodedReads.get() + (2*i+1) * encodedSequencePitchInInts;
                input.readLength1 = currentReadLengths[2*i];
                input.readLength2 = currentReadLengths[2*i+1];
                input.numInts1 = getEncodedNumInts2Bit(currentReadLengths[2*i]);
                input.numInts2 = getEncodedNumInts2Bit(currentReadLengths[2*i+1]);
                input.verbose = false;
                input.verboseMutex = &verboseMutex;
            }

            auto extensionResultsBatch = readExtenderGpu.extendPairedReadBatch(inputs);

            //convert results of ReadExtender
            std::vector<ExtendedRead> resultvector(extensionResultsBatch.size());

            for(int i = 0; i < numReadPairsInBatch; i++){
                auto& extensionOutput = extensionResultsBatch[i];
                ExtendedRead& er = resultvector[i];

                er.readId = extensionOutput.readId1;
                er.extendedSequence = std::move(extensionOutput.extendedRead);

                if(extensionOutput.mateHasBeenFound){
                    er.status = ExtendedReadStatus::FoundMate;
                }else{
                    if(extensionOutput.aborted){
                        if(extensionOutput.abortReason == ReadExtender::AbortReason::NoPairedCandidates
                                || extensionOutput.abortReason == ReadExtender::AbortReason::NoPairedCandidatesAfterAlignment){

                            er.status = ExtendedReadStatus::CandidateAbort;
                        }else if(extensionOutput.abortReason == ReadExtender::AbortReason::MsaNotExtended){
                            er.status = ExtendedReadStatus::MSANoExtension;
                        }
                    }else{
                        er.status = ExtendedReadStatus::LengthAbort;
                    }
                }  
                
                if(extensionOutput.success){
                    numSuccessRead++;
                }                
            }

            auto outputfunc = [&, vec = std::move(resultvector)](){
                for(const auto& er : vec){
                    partialResults.storeElement(&er);
                }
            };

            outputThread.enqueue(
                std::move(outputfunc)
            );

            progressThread.addProgress(numReadPairsInBatch);            
        }


        hipStreamDestroy(stream); CUERR;

        //#pragma omp critical
        {
            std::lock_guard<std::mutex> lg(ompCriticalMutex);

            totalNumSuccess0 += numSuccess0;
            totalNumSuccess1 += numSuccess1;
            totalNumSuccess01 += numSuccess01;
            totalNumSuccessRead += numSuccessRead;

            for(const auto& pair : extensionLengthsMap){
                totalExtensionLengthsMap[pair.first] += pair.second;
            }

            for(const auto& pair : mismatchesBetweenMateExtensions){
                totalMismatchesBetweenMateExtensions[pair.first] += pair.second;
            }

            if(0 == ompThreadId){
                readExtenderGpu.printTimers();
            }      
        }

        
        
    } //end omp parallel

    progressThread.finished();

    outputThread.stopThread(BackgroundThread::StopType::FinishAndStop);

    //outputstream.flush();
    partialResults.flush();

    std::cout << "totalNumSuccess0: " << totalNumSuccess0 << std::endl;
    std::cout << "totalNumSuccess1: " << totalNumSuccess1 << std::endl;
    std::cout << "totalNumSuccess01: " << totalNumSuccess01 << std::endl;
    std::cout << "totalNumSuccessRead: " << totalNumSuccessRead << std::endl;

    // std::cout << "Extension lengths:\n";

    // for(const auto& pair : totalExtensionLengthsMap){
    //     std::cout << pair.first << ": " << pair.second << "\n";
    // }

    // std::cout << "mismatches between mate extensions:\n";

    // for(const auto& pair : totalMismatchesBetweenMateExtensions){
    //     std::cout << pair.first << ": " << pair.second << "\n";
    // }



    return partialResults;
    //return resultExtendedReads;
}




MemoryFileFixedSize<ExtendedRead> 
//std::vector<ExtendedRead>
extend_gpu_singleend(
    const GoodAlignmentProperties& goodAlignmentProperties,
    const CorrectionOptions& correctionOptions,
    const ExtensionOptions& extensionOptions,
    const RuntimeOptions& runtimeOptions,
    const FileOptions& fileOptions,
    const MemoryOptions& memoryOptions,
    const SequenceFileProperties& sequenceFileProperties,
    const GpuMinhasher& minhasher,
    const gpu::DistributedReadStorage& gpuReadStorage
){
    std::cerr << "extend_gpu_singleend\n";

    const auto rsMemInfo = gpuReadStorage.getMemoryInfo();
    const auto mhMemInfo = minhasher.getMemoryInfo();

    std::size_t memoryAvailableBytesHost = memoryOptions.memoryTotalLimit;
    if(memoryAvailableBytesHost > rsMemInfo.host){
        memoryAvailableBytesHost -= rsMemInfo.host;
    }else{
        memoryAvailableBytesHost = 0;
    }
    if(memoryAvailableBytesHost > mhMemInfo.host){
        memoryAvailableBytesHost -= mhMemInfo.host;
    }else{
        memoryAvailableBytesHost = 0;
    }

    std::unique_ptr<std::uint8_t[]> correctionStatusFlagsPerRead = std::make_unique<std::uint8_t[]>(sequenceFileProperties.nReads);

    #pragma omp parallel for
    for(read_number i = 0; i < sequenceFileProperties.nReads; i++){
        correctionStatusFlagsPerRead[i] = 0;
    }

    std::cerr << "correctionStatusFlagsPerRead bytes: " << sizeof(std::uint8_t) * sequenceFileProperties.nReads / 1024. / 1024. << " MB\n";

    if(memoryAvailableBytesHost > sizeof(std::uint8_t) * sequenceFileProperties.nReads){
        memoryAvailableBytesHost -= sizeof(std::uint8_t) * sequenceFileProperties.nReads;
    }else{
        memoryAvailableBytesHost = 0;
    }

    const std::size_t availableMemoryInBytes = memoryAvailableBytesHost; //getAvailableMemoryInKB() * 1024;
    std::size_t memoryForPartialResultsInBytes = 0;

    if(availableMemoryInBytes > 3*(std::size_t(1) << 30)){
        memoryForPartialResultsInBytes = availableMemoryInBytes - 3*(std::size_t(1) << 30);
    }

    const std::string tmpfilename{fileOptions.tempdirectory + "/" + "MemoryFileFixedSizetmp"};
    MemoryFileFixedSize<ExtendedRead> partialResults(memoryForPartialResultsInBytes, tmpfilename);

    std::vector<ExtendedRead> resultExtendedReads;

    cpu::RangeGenerator<read_number> readIdGenerator(sequenceFileProperties.nReads);
    //cpu::RangeGenerator<read_number> readIdGenerator(1000);

    BackgroundThread outputThread(true);

    auto showProgress = [&](auto totalCount, auto seconds){
        if(runtimeOptions.showProgress){

            printf("Processed %10u of %10lu read pairs (Runtime: %03d:%02d:%02d)\r",
                    totalCount, sequenceFileProperties.nReads,
                    int(seconds / 3600),
                    int(seconds / 60) % 60,
                    int(seconds) % 60);
            std::cout.flush();
        }

        if(totalCount == sequenceFileProperties.nReads){
            std::cerr << '\n';
        }
    };

    auto updateShowProgressInterval = [](auto duration){
        return duration;
    };

    ProgressThread<read_number> progressThread(sequenceFileProperties.nReads, showProgress, updateShowProgressInterval);

    
    const int insertSize = extensionOptions.insertSize;
    const int insertSizeStddev = extensionOptions.insertSizeStddev;
    const int maximumSequenceLength = sequenceFileProperties.maxSequenceLength;
    const std::size_t encodedSequencePitchInInts = getEncodedNumInts2Bit(maximumSequenceLength);

    std::mutex verboseMutex;
    std::mutex ompCriticalMutex;

    std::int64_t totalNumSuccess0 = 0;
    std::int64_t totalNumSuccess1 = 0;
    std::int64_t totalNumSuccess01 = 0;
    std::int64_t totalNumSuccessRead = 0;

    std::map<int, int> totalExtensionLengthsMap;

    std::map<int, int> totalMismatchesBetweenMateExtensions;

    //omp_set_num_threads(1);

    #pragma omp parallel
    {
        const int numDeviceIds = runtimeOptions.deviceIds.size();

        assert(numDeviceIds > 0);

        const int ompThreadId = omp_get_thread_num();
        const int deviceId = runtimeOptions.deviceIds.at(ompThreadId % numDeviceIds);
        hipSetDevice(deviceId); CUERR;

        GoodAlignmentProperties goodAlignmentProperties2 = goodAlignmentProperties;
        //goodAlignmentProperties2.maxErrorRate = 0.05;

        constexpr int maxextensionPerStep = 20;

        ReadExtenderGpu readExtenderGpu{
            insertSize,
            insertSizeStddev,
            maxextensionPerStep,
            maximumSequenceLength,
            gpuReadStorage, 
            minhasher,
            correctionOptions,
            goodAlignmentProperties2
        };

        std::int64_t numSuccess0 = 0;
        std::int64_t numSuccess1 = 0;
        std::int64_t numSuccess01 = 0;
        std::int64_t numSuccessRead = 0;

        std::map<int, int> extensionLengthsMap;
        std::map<int, int> mismatchesBetweenMateExtensions;

        auto gatherHandleSequences = gpuReadStorage.makeGatherHandleSequences();

        const int batchsize = correctionOptions.batchsize;

        SimpleAllocationPinnedHost<read_number> currentIds(batchsize);
        SimpleAllocationPinnedHost<unsigned int> currentEncodedReads(encodedSequencePitchInInts * batchsize);
        SimpleAllocationPinnedHost<int> currentReadLengths(batchsize);

        hipStream_t stream;
        hipStreamCreate(&stream); CUERR;
        

        while(!(readIdGenerator.empty())){

            auto readIdsEnd = readIdGenerator.next_n_into_buffer(
                batchsize, 
                currentIds.get()
            );

            const int numReadsInBatch = std::distance(currentIds.get(), readIdsEnd);
            
            if(numReadsInBatch == 0){
                continue; //this should only happen if all reads have been processed
            }

            gpuReadStorage.gatherSequenceDataToGpuBufferAsync(
                nullptr,
                gatherHandleSequences,
                currentEncodedReads.get(), //device acccessible
                encodedSequencePitchInInts,
                currentIds.get(),
                currentIds.get(), //device accessible
                numReadsInBatch,
                deviceId,
                stream
            ); CUERR;
    
            gpuReadStorage.gatherSequenceLengthsToGpuBufferAsync(
                currentReadLengths.get(), //device accessible
                deviceId,
                currentIds.get(), //device accessible
                numReadsInBatch,    
                stream
            ); CUERR;

            hipStreamSynchronize(stream);

            std::vector<ReadExtenderGpu::ExtendInput> inputs(numReadsInBatch); 

            for(int i = 0; i < numReadsInBatch; i++){
                auto& input = inputs[i];

                input.readId1 = currentIds[i];
                input.readId2 = std::numeric_limits<read_number>::max();
                input.encodedRead1 = currentEncodedReads.get() + i * encodedSequencePitchInInts;
                input.encodedRead2 = nullptr;
                input.readLength1 = currentReadLengths[i];
                input.readLength2 = 0;
                input.numInts1 = getEncodedNumInts2Bit(currentReadLengths[i]);
                input.numInts2 = 0;
                input.verbose = false;
                input.verboseMutex = &verboseMutex;
            }

            auto extensionResultsBatch = readExtenderGpu.extendSingleEndReadBatch(inputs);

            //convert results of ReadExtender
            std::vector<ExtendedRead> resultvector(extensionResultsBatch.size());

            for(int i = 0; i < numReadsInBatch; i++){
                auto& extensionOutput = extensionResultsBatch[i];
                ExtendedRead& er = resultvector[i];

                er.readId = extensionOutput.readId1;
                er.extendedSequence = std::move(extensionOutput.extendedRead);

                if(extensionOutput.mateHasBeenFound){
                    er.status = ExtendedReadStatus::FoundMate;
                }else{
                    if(extensionOutput.aborted){
                        if(extensionOutput.abortReason == ReadExtender::AbortReason::NoPairedCandidates
                                || extensionOutput.abortReason == ReadExtender::AbortReason::NoPairedCandidatesAfterAlignment){

                            er.status = ExtendedReadStatus::CandidateAbort;
                        }else if(extensionOutput.abortReason == ReadExtender::AbortReason::MsaNotExtended){
                            er.status = ExtendedReadStatus::MSANoExtension;
                        }
                    }else{
                        er.status = ExtendedReadStatus::LengthAbort;
                    }
                }  
                
                if(extensionOutput.success){
                    numSuccessRead++;
                }                
            }

            auto outputfunc = [&, vec = std::move(resultvector)](){
                for(const auto& er : vec){
                    partialResults.storeElement(&er);
                }
            };

            outputThread.enqueue(
                std::move(outputfunc)
            );

            progressThread.addProgress(numReadsInBatch);            
        }


        hipStreamDestroy(stream); CUERR;

        //#pragma omp critical
        {
            std::lock_guard<std::mutex> lg(ompCriticalMutex);

            totalNumSuccess0 += numSuccess0;
            totalNumSuccess1 += numSuccess1;
            totalNumSuccess01 += numSuccess01;
            totalNumSuccessRead += numSuccessRead;

            for(const auto& pair : extensionLengthsMap){
                totalExtensionLengthsMap[pair.first] += pair.second;
            }

            for(const auto& pair : mismatchesBetweenMateExtensions){
                totalMismatchesBetweenMateExtensions[pair.first] += pair.second;
            }

            if(0 == ompThreadId){
                readExtenderGpu.printTimers();
            }      
        }

        
        
    } //end omp parallel

    progressThread.finished();

    outputThread.stopThread(BackgroundThread::StopType::FinishAndStop);

    //outputstream.flush();
    partialResults.flush();

    std::cout << "totalNumSuccess0: " << totalNumSuccess0 << std::endl;
    std::cout << "totalNumSuccess1: " << totalNumSuccess1 << std::endl;
    std::cout << "totalNumSuccess01: " << totalNumSuccess01 << std::endl;
    std::cout << "totalNumSuccessRead: " << totalNumSuccessRead << std::endl;

    // std::cout << "Extension lengths:\n";

    // for(const auto& pair : totalExtensionLengthsMap){
    //     std::cout << pair.first << ": " << pair.second << "\n";
    // }

    // std::cout << "mismatches between mate extensions:\n";

    // for(const auto& pair : totalMismatchesBetweenMateExtensions){
    //     std::cout << pair.first << ": " << pair.second << "\n";
    // }



    return partialResults;
    //return resultExtendedReads;
}


MemoryFileFixedSize<ExtendedRead> 
//std::vector<ExtendedRead>
extend_gpu(
    const GoodAlignmentProperties& goodAlignmentProperties,
    const CorrectionOptions& correctionOptions,
    const ExtensionOptions& extensionOptions,
    const RuntimeOptions& runtimeOptions,
    const FileOptions& fileOptions,
    const MemoryOptions& memoryOptions,
    const SequenceFileProperties& sequenceFileProperties,
    const GpuMinhasher& gpumMinhasher,
    const gpu::DistributedReadStorage& gpuReadStorage
){
    if(fileOptions.pairType == SequencePairType::SingleEnd){
        return extend_gpu_singleend(
            goodAlignmentProperties,
            correctionOptions,
            extensionOptions,
            runtimeOptions,
            fileOptions,
            memoryOptions,
            sequenceFileProperties,
            gpumMinhasher,
            gpuReadStorage
        );
    }else{
        return extend_gpu_pairedend(
            goodAlignmentProperties,
            correctionOptions,
            extensionOptions,
            runtimeOptions,
            fileOptions,
            memoryOptions,
            sequenceFileProperties,
            gpumMinhasher,
            gpuReadStorage
        );
    }
}





} // namespace gpu

} // namespace care