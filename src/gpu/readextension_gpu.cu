#include "hip/hip_runtime.h"

#include <gpu/cudaerrorcheck.cuh>
#include <gpu/gpuminhasher.cuh>
#include <gpu/gpureadstorage.cuh>
#include <gpu/readextender_gpu.hpp>

#include <alignmentorientation.hpp>
#include <concurrencyhelpers.hpp>
#include <config.hpp>
#include <cpu_alignment.hpp>
#include <extendedread.hpp>
#include <filehelpers.hpp>
#include <hpc_helpers.cuh>
#include <msa.hpp>
#include <options.hpp>
#include <rangegenerator.hpp>
#include <readextender_common.hpp>
#include <sequencehelpers.hpp>
#include <serializedobjectstorage.hpp>
#include <threadpool.hpp>
#include <util.hpp>

#include <algorithm>
#include <array>
#include <cstdint>
#include <future>
#include <iostream>
#include <memory>
#include <mutex>
#include <numeric>
#include <string>
#include <vector>




#include <omp.h>
#include <hipcub/hipcub.hpp>
#include <thrust/iterator/transform_iterator.h>




namespace care{
namespace gpu{

template<class T>
struct IsGreaterThan{
    T value;

    __host__ __device__
    IsGreaterThan(T t) : value(t){}

    template<class V>
    __host__ __device__
    bool operator()(V item) const noexcept{
        return item > value;
    }
};

template<class IdGenerator>
void initializeExtenderInput(
    IdGenerator& readIdGenerator,
    int requestedSizeOfTasks,
    const GpuReadStorage& gpuReadStorage,
    ReadStorageHandle& readStorageHandle,
    read_number* currentIds, // pinned memory
    int* currentReadLengths, //device accessible
    unsigned int* currentEncodedReads, //device accessible
    bool useQualityScores,
    char* currentQualityScores, //device accessible
    std::size_t encodedSequencePitchInInts,
    std::size_t qualityPitchInBytes,
    GpuReadExtender::TaskData& tasks,
    hipStream_t stream,
    rmm::mr::device_memory_resource* mr
){
    nvtx::push_range("init", 2);

    const int maxNumPairs = (requestedSizeOfTasks - tasks.size()) / 4;

    int numNewReadsInBatch = 0;

    readIdGenerator.process_next_n(
        maxNumPairs * 2, 
        [&](auto begin, auto end){
            auto readIdsEnd = std::copy(begin, end, currentIds);
            numNewReadsInBatch = std::distance(currentIds, readIdsEnd);
        }
    );

    if(numNewReadsInBatch % 2 == 1){
        throw std::runtime_error("Input files not properly paired. Aborting read extension.");
    }
   
    if(numNewReadsInBatch > 0){
        
        gpuReadStorage.gatherSequences(
            readStorageHandle,
            currentEncodedReads,
            encodedSequencePitchInInts,
            makeAsyncConstBufferWrapper(currentIds),
            currentIds, //device accessible
            numNewReadsInBatch,
            stream,
            mr
        );

        gpuReadStorage.gatherSequenceLengths(
            readStorageHandle,
            currentReadLengths,
            currentIds,
            numNewReadsInBatch,
            stream
        );

        if(useQualityScores){
            gpuReadStorage.gatherQualities(
                readStorageHandle,
                currentQualityScores,
                qualityPitchInBytes,
                makeAsyncConstBufferWrapper(currentIds),
                currentIds, //device accessible
                numNewReadsInBatch,
                stream,
                mr
            );
        }
        
        const int numReadPairsInBatch = numNewReadsInBatch / 2; 

        //std::cerr << "thread " << std::this_thread::get_id() << "add tasks\n";
        tasks.addTasks(numReadPairsInBatch, currentIds, currentReadLengths, currentEncodedReads, currentQualityScores, stream);

        //gpuReadExtender->setState(GpuReadExtender::State::UpdateWorkingSet);

        //std::cerr << "Added " << (numReadPairsInBatch * 4) << " new tasks to batch\n";
    }

    nvtx::pop_range();
};


std::pair< std::vector<ExtendedRead>, std::vector<read_number> >
makeAndSplitExtensionOutput(
    GpuReadExtender::TaskData& finishedTasks, 
    GpuReadExtender::RawExtendResult& rawExtendResult, 
    const GpuReadExtender* gpuReadExtender, 
    bool isRepeatedIteration, 
    bool collectPairsToRepeat, 
    hipStream_t stream
){

    nvtx::push_range("constructRawResults", 4);
    gpuReadExtender->constructRawResults(finishedTasks, rawExtendResult, stream);
    nvtx::pop_range();

    CUDACHECK(cudaStreamSynchronizeWrapper(stream));

    std::vector<extension::ExtendResult> extensionResults = gpuReadExtender->convertRawExtendResults(rawExtendResult);

    const std::size_t maxNumExtendedReads = extensionResults.size();

    std::vector<read_number> pairsWhichShouldBeRepeatedTemp;

    std::vector<ExtendedRead> extendedReads;
    extendedReads.reserve(maxNumExtendedReads);

    int repeated = 0;

    nvtx::push_range("convert extension results", 7);

    for(std::size_t i = 0; i < maxNumExtendedReads; i++){
        auto& extensionOutput = extensionResults[i];
        const int extendedReadLength = extensionOutput.extendedRead.size();
        //if(extendedReadLength == extensionOutput.originalLength){
        //if(!extensionOutput.mateHasBeenFound){
        if(extendedReadLength > extensionOutput.originalLength && !extensionOutput.mateHasBeenFound && collectPairsToRepeat){
            //do not insert directly into pairsWhichShouldBeRepeated. it causes an infinite loop
            pairsWhichShouldBeRepeatedTemp.push_back(extensionOutput.readId1);
            pairsWhichShouldBeRepeatedTemp.push_back(extensionOutput.readId2);
            repeated++;
        }else{
            //assert(extensionOutput.extendedRead.size() > extensionOutput.originalLength);

            extension::ExtensionResultConversionOptions opts;
            opts.computedAfterRepetition = isRepeatedIteration;            
            
            extendedReads.emplace_back(extension::makeExtendedReadFromExtensionResult(extensionOutput, opts));

        }
                        
    }
    
    nvtx::pop_range();

    return std::make_pair(std::move(extendedReads), std::move(pairsWhichShouldBeRepeatedTemp));
}





SerializedObjectStorage extend_gpu_pairedend(
    const GoodAlignmentProperties& goodAlignmentProperties,
    const CorrectionOptions& correctionOptions,
    const ExtensionOptions& extensionOptions,
    const RuntimeOptions& runtimeOptions,
    const FileOptions& fileOptions,
    const MemoryOptions& memoryOptions,
    const GpuMinhasher& minhasher,
    const GpuReadStorage& gpuReadStorage
){
    const auto rsMemInfo = gpuReadStorage.getMemoryInfo();
    const auto mhMemInfo = minhasher.getMemoryInfo();

    std::size_t memoryAvailableBytesHost = memoryOptions.memoryTotalLimit;
    if(memoryAvailableBytesHost > rsMemInfo.host){
        memoryAvailableBytesHost -= rsMemInfo.host;
    }else{
        memoryAvailableBytesHost = 0;
    }
    if(memoryAvailableBytesHost > mhMemInfo.host){
        memoryAvailableBytesHost -= mhMemInfo.host;
    }else{
        memoryAvailableBytesHost = 0;
    }

    const std::size_t availableMemoryInBytes = memoryAvailableBytesHost; //getAvailableMemoryInKB() * 1024;
    std::size_t memoryForPartialResultsInBytes = 0;

    if(availableMemoryInBytes > 3*(std::size_t(1) << 30)){
        memoryForPartialResultsInBytes = availableMemoryInBytes - 3*(std::size_t(1) << 30);
    }

    std::cerr << "Partial results may occupy " << (memoryForPartialResultsInBytes /1024. / 1024. / 1024.) 
        << " GB in memory. Remaining partial results will be stored in temp directory. \n";

    const std::size_t memoryLimitData = memoryForPartialResultsInBytes * 0.75;
    const std::size_t memoryLimitOffsets = memoryForPartialResultsInBytes * 0.25;

    SerializedObjectStorage partialResults(memoryLimitData, memoryLimitOffsets, fileOptions.tempdirectory + "/");

    BackgroundThread outputThread(true);

    const std::uint64_t totalNumReadPairs = gpuReadStorage.getNumberOfReads() / 2;

    auto showProgress = [&](auto totalCount, auto seconds){
        if(runtimeOptions.showProgress){

            printf("Processed %10u of %10lu read pairs (Runtime: %03d:%02d:%02d)\r",
                    totalCount, totalNumReadPairs,
                    int(seconds / 3600),
                    int(seconds / 60) % 60,
                    int(seconds) % 60);
            std::cout.flush();
        }

        if(totalCount == totalNumReadPairs){
            std::cerr << '\n';
        }
    };

    auto updateShowProgressInterval = [](auto duration){
        return duration;
    };

    ProgressThread<read_number> progressThread(totalNumReadPairs, showProgress, updateShowProgressInterval);

    cpu::QualityScoreConversion qualityConversion{};

    
    const int insertSize = extensionOptions.insertSize;
    const int insertSizeStddev = extensionOptions.insertSizeStddev;
    const int maximumSequenceLength = gpuReadStorage.getSequenceLengthUpperBound();
    const std::size_t encodedSequencePitchInInts = SequenceHelpers::getEncodedNumInts2Bit(maximumSequenceLength);
    const std::size_t decodedSequencePitchInBytes = SDIV(maximumSequenceLength, 128) * 128;
    const std::size_t qualityPitchInBytes = SDIV(maximumSequenceLength, 128) * 128;

    const std::size_t min_overlap = std::max(
        1, 
        std::max(
            goodAlignmentProperties.min_overlap, 
            int(maximumSequenceLength * goodAlignmentProperties.min_overlap_ratio)
        )
    );
    const std::size_t msa_max_column_count = (3*gpuReadStorage.getSequenceLengthUpperBound() - 2*min_overlap);
    //round up to 32 elements
    const std::size_t msaColumnPitchInElements = SDIV(msa_max_column_count, 32) * 32;

    std::mutex ompCriticalMutex;

    std::int64_t totalNumSuccess0 = 0;
    std::int64_t totalNumSuccess1 = 0;
    std::int64_t totalNumSuccess01 = 0;
    std::int64_t totalNumSuccessRead = 0;

    std::map<int, int> totalExtensionLengthsMap;

    std::map<int, int> totalMismatchesBetweenMateExtensions;

    //omp_set_num_threads(1);

    CUDACHECK(hipSetDevice(runtimeOptions.deviceIds[0]));

    const int batchsizePairs = correctionOptions.batchsize;

    constexpr bool isPairedEnd = true;

    auto submitReadyResults = [&](std::vector<ExtendedRead> extendedReads, std::vector<EncodedExtendedRead> encodedExtendedReads){
        outputThread.enqueue(
            [&, vec = std::move(extendedReads), encvec = std::move(encodedExtendedReads)](){
                std::vector<std::uint8_t> tempbuffer(256);

                #if 0
                for(const auto& er : vec){
                    const std::size_t serializedSize = er.getSerializedNumBytes();
                    tempbuffer.resize(serializedSize);

                    auto end = er.copyToContiguousMemory(tempbuffer.data(), tempbuffer.data() + tempbuffer.size());
                    assert(end != nullptr);

                    partialResults.insert(tempbuffer.data(), end);
                }
                #else
                for(const auto& er : encvec){
                    const std::size_t serializedSize = er.getSerializedNumBytes();
                    tempbuffer.resize(serializedSize);

                    auto end = er.copyToContiguousMemory(tempbuffer.data(), tempbuffer.data() + tempbuffer.size());
                    assert(end != nullptr);

                    partialResults.insert(tempbuffer.data(), end);
                }
                #endif
            }
        );
    };
           

    assert(runtimeOptions.deviceIds.size() > 0);

    //will need at least one thread per gpu
    const int numDeviceIds = std::min(runtimeOptions.threads, int(runtimeOptions.deviceIds.size()));

    struct GpuData{
        int deviceId;
        std::unique_ptr<GpuReadExtender> gpuReadExtender;

        GpuData() = default;

        GpuData(const GpuData&) = delete;
        GpuData& operator=(const GpuData&) = delete;

        GpuData(GpuData&&) = default;
        GpuData& operator=(GpuData&&) = default;

        ~GpuData(){
            hipcub::SwitchDevice ds(deviceId);
            gpuReadExtender = nullptr;
        }
    };

    std::vector<GpuData> gpuDataVector;

    for(int d = 0; d < numDeviceIds; d++){
        const int deviceId = runtimeOptions.deviceIds[d];
        hipcub::SwitchDevice sd(deviceId);

        GpuData gpudata;
        gpudata.deviceId = deviceId;

        gpudata.gpuReadExtender = std::make_unique<GpuReadExtender>(
            encodedSequencePitchInInts,
            decodedSequencePitchInBytes,
            qualityPitchInBytes,
            msaColumnPitchInElements,
            isPairedEnd,
            gpuReadStorage, 
            correctionOptions,
            goodAlignmentProperties,
            qualityConversion,
            insertSize,
            insertSizeStddev,
            hipStreamPerThread,
            rmm::mr::get_current_device_resource()
        );

        CUDACHECK(hipStreamSynchronize(hipStreamPerThread));

        gpuDataVector.push_back(std::move(gpudata));
    }




    auto extenderThreadFunc = [&](int gpuIndex, int /*threadId*/, auto* readIdGenerator, bool isRepeatedIteration, bool isLastIteration, bool extraHashing, GpuReadExtender::IterationConfig iterationConfig){
        //std::cerr << "extenderThreadFunc( " << gpuIndex << ", " << threadId << ")\n";
        auto& gpudata = gpuDataVector[gpuIndex];

        hipStream_t stream = hipStreamPerThread;

        CUDACHECK(hipSetDevice(gpudata.deviceId));

        rmm::mr::device_memory_resource* rmmDeviceResource = rmm::mr::get_current_device_resource();

        std::int64_t numSuccess0 = 0;
        std::int64_t numSuccess1 = 0;
        std::int64_t numSuccess01 = 0;
        std::int64_t numSuccessRead = 0;

        std::map<int, int> extensionLengthsMap;
        std::map<int, int> mismatchesBetweenMateExtensions;

        ReadStorageHandle readStorageHandle = gpuReadStorage.makeHandle();


        helpers::SimpleAllocationPinnedHost<read_number> currentIds(2 * batchsizePairs);
        helpers::SimpleAllocationDevice<unsigned int> currentEncodedReads(2 * encodedSequencePitchInInts * batchsizePairs);
        helpers::SimpleAllocationDevice<int> currentReadLengths(2 * batchsizePairs);
        helpers::SimpleAllocationDevice<char> currentQualityScores(2 * qualityPitchInBytes * batchsizePairs);

        if(!correctionOptions.useQualityScores){
            helpers::call_fill_kernel_async(currentQualityScores.data(), currentQualityScores.size(), 'I', stream);
        }


        GpuReadExtender::Hasher anchorHasher(minhasher, rmmDeviceResource);

        GpuReadExtender::TaskData tasks(rmmDeviceResource, 0, encodedSequencePitchInInts, decodedSequencePitchInBytes, qualityPitchInBytes, stream);
        GpuReadExtender::TaskData finishedTasks(rmmDeviceResource, 0, encodedSequencePitchInInts, decodedSequencePitchInBytes, qualityPitchInBytes, stream);

        GpuReadExtender::AnchorData anchorData(rmmDeviceResource);
        GpuReadExtender::AnchorHashResult anchorHashResult(rmmDeviceResource);

        GpuReadExtender::RawExtendResult rawExtendResult{};

        std::vector<read_number> pairsWhichShouldBeRepeated;

        auto output = [&](){

            nvtx::push_range("output", 5);

            auto [extendedReads, pairsTmp] = makeAndSplitExtensionOutput(finishedTasks, rawExtendResult, gpudata.gpuReadExtender.get(), isRepeatedIteration, !isLastIteration, stream);

            pairsWhichShouldBeRepeated.insert(pairsWhichShouldBeRepeated.end(), pairsTmp.begin(), pairsTmp.end());

            const std::size_t numExtended = extendedReads.size();

            if(!extensionOptions.allowOutwardExtension){
                for(auto& er : extendedReads){
                    er.removeOutwardExtension();
                }
            }

            std::vector<EncodedExtendedRead> encvec(numExtended);
            for(std::size_t i = 0; i < numExtended; i++){
                extendedReads[i].encodeInto(encvec[i]);
            }

            submitReadyResults(std::move(extendedReads), std::move(encvec));

            progressThread.addProgress(numExtended);

            nvtx::pop_range();
        };

        while(!(readIdGenerator->empty() && tasks.size() == 0)){
            if(int(tasks.size()) < (batchsizePairs * 4) / 2){
                initializeExtenderInput(
                    *readIdGenerator,
                    batchsizePairs * 4,
                    gpuReadStorage,
                    readStorageHandle,
                    currentIds.data(), 
                    currentReadLengths.data(), 
                    currentEncodedReads.data(),
                    correctionOptions.useQualityScores,
                    currentQualityScores.data(), 
                    encodedSequencePitchInInts,
                    qualityPitchInBytes,
                    tasks,
                    stream,
                    rmmDeviceResource
                );
            }

            tasks.aggregateAnchorData(anchorData, stream);
            
            nvtx::push_range("getCandidateReadIds", 4);
            if(extraHashing){
            //if(false){
                anchorHasher.getCandidateReadIdsWithExtraExtensionHash(
                    anchorData, 
                    anchorHashResult,
                    iterationConfig, 
                    thrust::make_transform_iterator(
                        tasks.iteration.data(),
                        IsGreaterThan<int>{0}
                    ),
                    stream
                );
            }else{
                anchorHasher.getCandidateReadIds(anchorData, anchorHashResult, stream);
            }
            // #if 0
            // anchorHasher.getCandidateReadIds(anchorData, anchorHashResult, stream);
            // #else
            // anchorHasher.getCandidateReadIdsWithExtraExtensionHash(
            //     *gpudata.dataAllocator,
            //     anchorData, 
            //     anchorHashResult,
            //     iterationConfig, 
            //     thrust::make_transform_iterator(
            //         tasks.iteration.data(),
            //         IsGreaterThan<int>{0}
            //     ),
            //     stream
            // );
            // #endif
            nvtx::pop_range();

            gpudata.gpuReadExtender->processOneIteration(
                tasks,
                anchorData, 
                anchorHashResult, 
                finishedTasks, 
                iterationConfig,
                stream
            );

            CUDACHECK(cudaStreamSynchronizeWrapper(stream));
            
            if(finishedTasks.size() > std::size_t((batchsizePairs * 4) / 2)){
                output();
            }

            //std::cerr << "Remaining: tasks " << tasks.size() << ", finishedtasks " << gpuReadExtender->finishedTasks->size() << "\n";
        }

        output();
        assert(finishedTasks.size() == 0);

        {
            std::lock_guard<std::mutex> lg(ompCriticalMutex);

            totalNumSuccess0 += numSuccess0;
            totalNumSuccess1 += numSuccess1;
            totalNumSuccess01 += numSuccess01;
            totalNumSuccessRead += numSuccessRead;

            for(const auto& pair : extensionLengthsMap){
                totalExtensionLengthsMap[pair.first] += pair.second;
            }

            for(const auto& pair : mismatchesBetweenMateExtensions){
                totalMismatchesBetweenMateExtensions[pair.first] += pair.second;
            }   
        }

        
        gpuReadStorage.destroyHandle(readStorageHandle);

        return pairsWhichShouldBeRepeated;
    };

    bool isLastIteration = false;
    GpuReadExtender::IterationConfig iterationConfig;
    iterationConfig.maxextensionPerStep = extensionOptions.fixedStepsize == 0 ? 20 : extensionOptions.fixedStepsize;
    iterationConfig.minCoverageForExtension = 3;

    std::vector<read_number> pairsWhichShouldBeRepeated;
    std::vector<read_number> pairsWhichShouldBeRepeatedTmp;

    for(auto& x : gpuDataVector){
        x.gpuReadExtender->insertSizeStddev = extensionOptions.fixedStddev == 0 ? extensionOptions.insertSizeStddev : extensionOptions.fixedStddev;
    }


    {
        std::vector<std::future<std::vector<read_number>>> futures;

        const std::size_t numReadsToProcess = 100000;
        //const std::size_t numReadsToProcess = gpuReadStorage.getNumberOfReads();

        // std::vector<read_number> idsToExtend{
        //     0, 1, 22, 23, 44, 45, 68, 69, 78, 79, 86, 87, 98, 99,
        //     112, 113, 136,137,180,181,198,199,202,203,266,267,
        //     290,291,316,317,350,351,402,403,436,437,446,447,498,499,574,575,582,583,588,589,
        //     598,599,692,693,704,705,814,815,964,965,966,967,970,971,1026,1027
        // };

        // IteratorRangeTraversal<decltype(idsToExtend.begin())> readIdGenerator(
        //     idsToExtend.begin(),
        //     idsToExtend.end()
        // );

        IteratorRangeTraversal<thrust::counting_iterator<read_number>> readIdGenerator(
            thrust::make_counting_iterator<read_number>(0),
            thrust::make_counting_iterator<read_number>(0) + numReadsToProcess
        );

        // IteratorRangeTraversal<thrust::counting_iterator<read_number>> readIdGenerator(
        //     thrust::make_counting_iterator<read_number>(0) + 9779220,
        //     thrust::make_counting_iterator<read_number>(0) + 9779224
        // );

        const int maxNumThreads = runtimeOptions.threads;
        const bool extraHashing = false;

        std::cerr << "First iteration. insertsizedev: " << gpuDataVector[0].gpuReadExtender->insertSizeStddev 
        << ", maxextensionPerStep: " << iterationConfig.maxextensionPerStep
        << ", minCoverageForExtension: " << iterationConfig.minCoverageForExtension
        << ", isLastIteration: " << isLastIteration 
        << ", extraHashing: " << extraHashing << "\n";

        std::cerr << "use " << maxNumThreads << " threads\n";

        constexpr bool isRepeatedIteration = false;

        for(int t = 0; t < maxNumThreads; t++){
            futures.emplace_back(
                std::async(
                    std::launch::async,
                    extenderThreadFunc,
                    t % numDeviceIds,
                    t,
                    &readIdGenerator,
                    isRepeatedIteration,
                    isLastIteration,
                    extraHashing,
                    iterationConfig
                )
            );
        }

        for(auto& f : futures){
            auto vec = f.get();
            pairsWhichShouldBeRepeatedTmp.insert(pairsWhichShouldBeRepeatedTmp.end(), vec.begin(), vec.end());
        }

        std::swap(pairsWhichShouldBeRepeated, pairsWhichShouldBeRepeatedTmp);
        pairsWhichShouldBeRepeatedTmp.clear();
        std::sort(pairsWhichShouldBeRepeated.begin(), pairsWhichShouldBeRepeated.end());

        //iterationConfig.maxextensionPerStep -= 4;
    }

    if(!isLastIteration){

        for(auto& x : gpuDataVector){
            x.gpuReadExtender->insertSizeStddev = extensionOptions.fixedStddev == 0 ? 40 : extensionOptions.fixedStddev;
            //x.gpuReadExtender->insertSizeStddev = extensionOptions.fixedStddev == 0 ? 40 : 40;
        }

        const bool extraHashing = true;

        isLastIteration = true;
        constexpr bool isRepeatedIteration = true;
        //iterationConfig.maxextensionPerStep = 16;

        //while(pairsWhichShouldBeRepeated.size() > 0 && (iterationConfig.maxextensionPerStep > 0))
        {
            const int numPairsToRepeat = pairsWhichShouldBeRepeated.size() / 2;
            std::cerr << "Will repeat extension of " << numPairsToRepeat << " read pairs with fixedStepsize = " << iterationConfig.maxextensionPerStep << "\n";

            std::cerr << "Second iteration. insertsizedev: " << gpuDataVector[0].gpuReadExtender->insertSizeStddev 
            << ", maxextensionPerStep: " << iterationConfig.maxextensionPerStep
            << ", minCoverageForExtension: " << iterationConfig.minCoverageForExtension
            << ", isLastIteration: " << isLastIteration 
            << ", extraHashing: " << extraHashing << "\n";

            //isLastIteration = (iterationConfig.maxextensionPerStep <= 4);

            auto readIdGenerator = makeIteratorRangeTraversal(
                pairsWhichShouldBeRepeated.data(), 
                pairsWhichShouldBeRepeated.data() + pairsWhichShouldBeRepeated.size()
            );

            const int threadsForPairs = SDIV(numPairsToRepeat, batchsizePairs);
            const int maxNumThreads = std::min(threadsForPairs, runtimeOptions.threads);
            std::cerr << "use " << maxNumThreads << " threads\n";

            std::vector<std::future<std::vector<read_number>>> futures;

            for(int t = 0; t < maxNumThreads; t++){
                futures.emplace_back(
                    std::async(
                        std::launch::async,
                        extenderThreadFunc,
                        t % numDeviceIds,
                        t,
                        &readIdGenerator,
                        isRepeatedIteration,
                        isLastIteration,
                        extraHashing,
                        iterationConfig
                    )
                );
            }

            for(auto& f : futures){
                auto vec = f.get();
                pairsWhichShouldBeRepeatedTmp.insert(pairsWhichShouldBeRepeatedTmp.end(), vec.begin(), vec.end());
            }

            std::swap(pairsWhichShouldBeRepeated, pairsWhichShouldBeRepeatedTmp);
            pairsWhichShouldBeRepeatedTmp.clear();
            std::sort(pairsWhichShouldBeRepeated.begin(), pairsWhichShouldBeRepeated.end());

            iterationConfig.maxextensionPerStep -= 4;
        }
    }
   

    progressThread.finished();

    outputThread.stopThread(BackgroundThread::StopType::FinishAndStop);



    // std::cout << "totalNumSuccess0: " << totalNumSuccess0 << std::endl;
    // std::cout << "totalNumSuccess1: " << totalNumSuccess1 << std::endl;
    // std::cout << "totalNumSuccess01: " << totalNumSuccess01 << std::endl;
    // std::cout << "totalNumSuccessRead: " << totalNumSuccessRead << std::endl;

    // std::cout << "Extension lengths:\n";

    // for(const auto& pair : totalExtensionLengthsMap){
    //     std::cout << pair.first << ": " << pair.second << "\n";
    // }

    // std::cout << "mismatches between mate extensions:\n";

    // for(const auto& pair : totalMismatchesBetweenMateExtensions){
    //     std::cout << pair.first << ": " << pair.second << "\n";
    // }



    return partialResults;
}


#if 0

SerializedObjectStorage extend_gpu_singleend(
    const GoodAlignmentProperties& goodAlignmentProperties,
    const CorrectionOptions& correctionOptions,
    const ExtensionOptions& extensionOptions,
    const RuntimeOptions& runtimeOptions,
    const FileOptions& fileOptions,
    const MemoryOptions& memoryOptions,
    const GpuMinhasher& minhasher,
    const GpuReadStorage& gpuReadStorage
){
    std::cerr << "extend_gpu_singleend\n";
    throw std::runtime_error("extend_gpu_singleend NOT IMPLEMENTED");
    
}
#endif

SerializedObjectStorage extend_gpu(
    const GoodAlignmentProperties& goodAlignmentProperties,
    const CorrectionOptions& correctionOptions,
    const ExtensionOptions& extensionOptions,
    const RuntimeOptions& runtimeOptions,
    const FileOptions& fileOptions,
    const MemoryOptions& memoryOptions,
    const GpuMinhasher& gpumMinhasher,
    const GpuReadStorage& gpuReadStorage
){
    // if(fileOptions.pairType == SequencePairType::SingleEnd){
    //     return extend_gpu_singleend(
    //         goodAlignmentProperties,
    //         correctionOptions,
    //         extensionOptions,
    //         runtimeOptions,
    //         fileOptions,
    //         memoryOptions,
    //         gpumMinhasher,
    //         gpuReadStorage
    //     );
    // }else{
        return extend_gpu_pairedend(
            goodAlignmentProperties,
            correctionOptions,
            extensionOptions,
            runtimeOptions,
            fileOptions,
            memoryOptions,
            gpumMinhasher,
            gpuReadStorage
        );
    //}
}





} // namespace gpu

} // namespace care