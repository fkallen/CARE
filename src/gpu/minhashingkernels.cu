#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstddef>
#include <limits>

#include <config.hpp>
#include <hpc_helpers.cuh>
#include <sequencehelpers.hpp>


namespace care{
namespace gpu{


__global__
void minhashSignaturesKernel(
    std::uint64_t* __restrict__ signatures,
    std::size_t signaturesRowPitchElements,
    const unsigned int* __restrict__ sequences2Bit,
    std::size_t sequenceRowPitchElements,
    int numSequences,
    const int* __restrict__ sequenceLengths,
    int k,
    int numHashFuncs,
    int firstHashFunc
){
            
    //constexpr int blocksize = 128;
    constexpr int maximum_kmer_length = max_k<std::uint64_t>::value;

    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if(tid < numSequences * numHashFuncs){
        const int mySequenceIndex = tid / numHashFuncs;
        const int myNumHashFunc = tid % numHashFuncs;
        const int hashFuncId = myNumHashFunc + firstHashFunc;

        const unsigned int* const mySequence = sequences2Bit + mySequenceIndex * sequenceRowPitchElements;
        const int myLength = sequenceLengths[mySequenceIndex];

        std::uint64_t* const mySignature = signatures + mySequenceIndex * signaturesRowPitchElements;

        std::uint64_t minHashValue = std::numeric_limits<std::uint64_t>::max();

        auto handlekmer = [&](auto fwd, auto rc){
            using hasher = hashers::MurmurHash<std::uint64_t>;

            const auto smallest = min(fwd, rc);
            const auto hashvalue = hasher::hash(smallest + hashFuncId);
            minHashValue = min(minHashValue, hashvalue);
        };

        if(myLength >= k){
            //const int numKmers = myLength - k + 1;
            const std::uint64_t kmer_mask = std::numeric_limits<std::uint64_t>::max() >> ((maximum_kmer_length - k) * 2);
            const int rcshiftamount = (maximum_kmer_length - k) * 2;

            //Compute the first kmer
            std::uint64_t kmer_encoded = mySequence[0];
            if(k <= 16){
                kmer_encoded >>= (16 - k) * 2;
            }else{
                kmer_encoded = (kmer_encoded << 32) | mySequence[1];
                kmer_encoded >>= (32 - k) * 2;
            }

            kmer_encoded >>= 2; //k-1 bases, allows easier loop

            std::uint64_t rc_kmer_encoded = SequenceHelpers::reverseComplementInt2Bit(kmer_encoded);

            auto addBase = [&](std::uint64_t encBase){
                kmer_encoded <<= 2;
                rc_kmer_encoded >>= 2;

                const std::uint64_t revcBase = (~encBase) & 3;
                kmer_encoded |= encBase;
                rc_kmer_encoded |= revcBase << 62;
            };

            constexpr int basesPerInt = SequenceHelpers::basesPerInt2Bit();

            const int itersend1 = min(SDIV(k-1, basesPerInt) * basesPerInt, myLength);

            //process sequence positions one by one
            // until the next encoded sequence data element is reached
            for(int nextSequencePos = k - 1; nextSequencePos < itersend1; nextSequencePos++){
                const int nextIntIndex = nextSequencePos / basesPerInt;
                const int nextPositionInInt = nextSequencePos % basesPerInt;

                const std::uint64_t nextBase = mySequence[nextIntIndex] >> (30 - 2 * nextPositionInInt);

                addBase(nextBase);

                handlekmer(
                    kmer_encoded & kmer_mask, 
                    rc_kmer_encoded >> rcshiftamount
                );
            }

            const int fullIntIters = (myLength - itersend1) / basesPerInt;

            //process all fully occupied encoded sequence data elements
            // improves memory access
            for(int iter = 0; iter < fullIntIters; iter++){
                const int intIndex = (itersend1 + iter * basesPerInt) / basesPerInt;
                const unsigned int data = mySequence[intIndex];

                #pragma unroll
                for(int posInInt = 0; posInInt < basesPerInt; posInInt++){
                    const std::uint64_t nextBase = data >> (30 - 2 * posInInt);

                    addBase(nextBase);

                    handlekmer(
                        kmer_encoded & kmer_mask, 
                        rc_kmer_encoded >> rcshiftamount
                    );
                }
            }

            //process remaining positions one by one
            for(int nextSequencePos = fullIntIters * basesPerInt + itersend1; nextSequencePos < myLength; nextSequencePos++){
                const int nextIntIndex = nextSequencePos / basesPerInt;
                const int nextPositionInInt = nextSequencePos % basesPerInt;

                const std::uint64_t nextBase = mySequence[nextIntIndex] >> (30 - 2 * nextPositionInInt);

                addBase(nextBase);

                handlekmer(
                    kmer_encoded & kmer_mask, 
                    rc_kmer_encoded >> rcshiftamount
                );
            }

            mySignature[myNumHashFunc] = minHashValue & kmer_mask;

        }else{
            mySignature[myNumHashFunc] = std::numeric_limits<std::uint64_t>::max();
        }
    }
} 



void callMinhashSignaturesKernel(
    std::uint64_t* __restrict__ signatures,
    std::size_t signaturesRowPitchElements,
    const unsigned int* __restrict__ sequences2Bit,
    std::size_t sequenceRowPitchElements,
    int numSequences,
    const int* __restrict__ sequenceLengths,
    int k,
    int numHashFuncs,
    int firstHashFunc,
    hipStream_t stream
){
    dim3 block(128,1,1);
    dim3 grid(SDIV(numHashFuncs * numSequences, block.x),1,1);

    minhashSignaturesKernel<<<grid, block, 0, stream>>>(
        signatures,
        signaturesRowPitchElements,
        sequences2Bit,
        sequenceRowPitchElements,
        numSequences,
        sequenceLengths,
        k,
        numHashFuncs,
        firstHashFunc
    ); CUERR;
}




__global__
void minhashSignaturesKernel(
    std::uint64_t* __restrict__ signatures,
    std::size_t signaturesRowPitchElements,
    const unsigned int* __restrict__ sequences2Bit,
    std::size_t sequenceRowPitchElements,
    int numSequences,
    const int* __restrict__ sequenceLengths,
    int k,
    int numHashFuncs,
    const int* __restrict__ hashFunctionNumbers
){
            
    //constexpr int blocksize = 128;
    constexpr int maximum_kmer_length = max_k<std::uint64_t>::value;

    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if(tid < numSequences * numHashFuncs){
        const int mySequenceIndex = tid / numHashFuncs;
        const int myNumHashFunc = tid % numHashFuncs;
        const int hashFuncId = hashFunctionNumbers[myNumHashFunc];

        const unsigned int* const mySequence = sequences2Bit + mySequenceIndex * sequenceRowPitchElements;
        const int myLength = sequenceLengths[mySequenceIndex];

        std::uint64_t* const mySignature = signatures + mySequenceIndex * signaturesRowPitchElements;

        std::uint64_t minHashValue = std::numeric_limits<std::uint64_t>::max();

        auto handlekmer = [&](auto fwd, auto rc){
            using hasher = hashers::MurmurHash<std::uint64_t>;

            const auto smallest = min(fwd, rc);
            const auto hashvalue = hasher::hash(smallest + hashFuncId);
            minHashValue = min(minHashValue, hashvalue);
        };

        if(myLength >= k){
            //const int numKmers = myLength - k + 1;
            const std::uint64_t kmer_mask = std::numeric_limits<std::uint64_t>::max() >> ((maximum_kmer_length - k) * 2);
            const int rcshiftamount = (maximum_kmer_length - k) * 2;

            //Compute the first kmer
            std::uint64_t kmer_encoded = mySequence[0];
            if(k <= 16){
                kmer_encoded >>= (16 - k) * 2;
            }else{
                kmer_encoded = (kmer_encoded << 32) | mySequence[1];
                kmer_encoded >>= (32 - k) * 2;
            }

            kmer_encoded >>= 2; //k-1 bases, allows easier loop

            std::uint64_t rc_kmer_encoded = SequenceHelpers::reverseComplementInt2Bit(kmer_encoded);

            auto addBase = [&](std::uint64_t encBase){
                kmer_encoded <<= 2;
                rc_kmer_encoded >>= 2;

                const std::uint64_t revcBase = (~encBase) & 3;
                kmer_encoded |= encBase;
                rc_kmer_encoded |= revcBase << 62;
            };

            constexpr int basesPerInt = SequenceHelpers::basesPerInt2Bit();

            const int itersend1 = min(SDIV(k-1, basesPerInt) * basesPerInt, myLength);

            //process sequence positions one by one
            // until the next encoded sequence data element is reached
            for(int nextSequencePos = k - 1; nextSequencePos < itersend1; nextSequencePos++){
                const int nextIntIndex = nextSequencePos / basesPerInt;
                const int nextPositionInInt = nextSequencePos % basesPerInt;

                const std::uint64_t nextBase = mySequence[nextIntIndex] >> (30 - 2 * nextPositionInInt);

                addBase(nextBase);

                handlekmer(
                    kmer_encoded & kmer_mask, 
                    rc_kmer_encoded >> rcshiftamount
                );
            }

            const int fullIntIters = (myLength - itersend1) / basesPerInt;

            //process all fully occupied encoded sequence data elements
            // improves memory access
            for(int iter = 0; iter < fullIntIters; iter++){
                const int intIndex = (itersend1 + iter * basesPerInt) / basesPerInt;
                const unsigned int data = mySequence[intIndex];

                #pragma unroll
                for(int posInInt = 0; posInInt < basesPerInt; posInInt++){
                    const std::uint64_t nextBase = data >> (30 - 2 * posInInt);

                    addBase(nextBase);

                    handlekmer(
                        kmer_encoded & kmer_mask, 
                        rc_kmer_encoded >> rcshiftamount
                    );
                }
            }

            //process remaining positions one by one
            for(int nextSequencePos = fullIntIters * basesPerInt + itersend1; nextSequencePos < myLength; nextSequencePos++){
                const int nextIntIndex = nextSequencePos / basesPerInt;
                const int nextPositionInInt = nextSequencePos % basesPerInt;

                const std::uint64_t nextBase = mySequence[nextIntIndex] >> (30 - 2 * nextPositionInInt);

                addBase(nextBase);

                handlekmer(
                    kmer_encoded & kmer_mask, 
                    rc_kmer_encoded >> rcshiftamount
                );
            }

            mySignature[myNumHashFunc] = minHashValue & kmer_mask;

        }else{
            mySignature[myNumHashFunc] = std::numeric_limits<std::uint64_t>::max();
        }
    }
} 



void callMinhashSignaturesKernel(
    std::uint64_t* __restrict__ signatures,
    std::size_t signaturesRowPitchElements,
    const unsigned int* __restrict__ sequences2Bit,
    std::size_t sequenceRowPitchElements,
    int numSequences,
    const int* __restrict__ sequenceLengths,
    int k,
    int numHashFuncs,
    const int* __restrict__ hashFunctionNumbers,
    hipStream_t stream
){
    dim3 block(128,1,1);
    dim3 grid(SDIV(numHashFuncs * numSequences, block.x),1,1);

    minhashSignaturesKernel<<<grid, block, 0, stream>>>(
        signatures,
        signaturesRowPitchElements,
        sequences2Bit,
        sequenceRowPitchElements,
        numSequences,
        sequenceLengths,
        k,
        numHashFuncs,
        hashFunctionNumbers
    ); CUERR;
}


























} //namespace gpu
} //namespace care