#include "hip/hip_runtime.h"
#include <gpu/fakegpuminhasher.cuh>
#include <hpc_helpers.cuh>
#include <gpu/kernels.hpp>

namespace care{
    namespace gpu{


        


void FakeGpuMinhasher::queryPrecalculatedSignatures(
    const std::uint64_t* signatures, //getNumberOfMaps() elements per sequence
    FakeGpuMinhasher::Range_t* ranges, //getNumberOfMaps() elements per sequence
    int* totalNumResultsInRanges, 
    int numSequences) const{ 
    
    int numResults = 0;
    const std::uint64_t kmer_mask = getKmerMask();

    for(int i = 0; i < numSequences; i++){
        const std::uint64_t* const signature = &signatures[i * getNumberOfMaps()];
        FakeGpuMinhasher::Range_t* const range = &ranges[i * getNumberOfMaps()];            

        for(int map = 0; map < getNumberOfMaps(); ++map){
            kmer_type key = signature[map] & kmer_mask;
            auto entries_range = queryMap(map, key);
            numResults += std::distance(entries_range.first, entries_range.second);
            range[map] = entries_range;
        }
    }   

    *totalNumResultsInRanges = numResults;   
}



void FakeGpuMinhasher::writeToStream(std::ostream& os) const{

    os.write(reinterpret_cast<const char*>(&kmerSize), sizeof(int));
    os.write(reinterpret_cast<const char*>(&resultsPerMapThreshold), sizeof(int));

    const int numTables = getNumberOfMaps();
    os.write(reinterpret_cast<const char*>(&numTables), sizeof(int));

    for(const auto& tableptr : minhashTables){
        tableptr->writeToStream(os);
    }
}

int FakeGpuMinhasher::loadFromStream(std::ifstream& is, int numMapsUpperLimit){
    destroy();

    is.read(reinterpret_cast<char*>(&kmerSize), sizeof(int));
    is.read(reinterpret_cast<char*>(&resultsPerMapThreshold), sizeof(int));

    int numMaps = 0;

    is.read(reinterpret_cast<char*>(&numMaps), sizeof(int));

    const int mapsToLoad = std::min(numMapsUpperLimit, numMaps);

    for(int i = 0; i < mapsToLoad; i++){
        auto ptr = std::make_unique<HashTable>();
        ptr->loadFromStream(is);
        minhashTables.emplace_back(std::move(ptr));
    }

    return mapsToLoad;
}

void FakeGpuMinhasher::constructFromReadStorage(
    const FileOptions &fileOptions,
    const RuntimeOptions &runtimeOptions,
    const MemoryOptions& memoryOptions,
    std::uint64_t nReads,
    const CorrectionOptions& correctionOptions,
    const GpuReadStorage& gpuReadStorage
){
    
    auto& readStorage = gpuReadStorage;
    const auto& deviceIds = runtimeOptions.deviceIds;

    int deviceId = deviceIds[0];

    hipcub::SwitchDevice sd{deviceId};

    const int requestedNumberOfMaps = correctionOptions.numHashFunctions;

    const read_number numReads = readStorage.getNumberOfReads();
    const int maximumSequenceLength = readStorage.getSequenceLengthUpperBound();

    auto sequencehandle = gpuReadStorage.makeHandle();
    const std::size_t encodedSequencePitchInInts = SequenceHelpers::getEncodedNumInts2Bit(maximumSequenceLength);

    constexpr read_number parallelReads = 1000000;
    const int numIters = SDIV(numReads, parallelReads);

    const MemoryUsage memoryUsageOfReadStorage = readStorage.getMemoryInfo();
    std::size_t totalLimit = memoryOptions.memoryTotalLimit;
    if(totalLimit > memoryUsageOfReadStorage.host){
        totalLimit -= memoryUsageOfReadStorage.host;
    }else{
        totalLimit = 0;
    }
    if(totalLimit == 0){
        throw std::runtime_error("Not enough memory available for hash tables. Abort!");
    }
    std::size_t maxMemoryForTables = getAvailableMemoryInKB() * 1024;
    // std::cerr << "available: " << maxMemoryForTables 
    //         << ",memoryForHashtables: " << memoryOptions.memoryForHashtables
    //         << ", memoryTotalLimit: " << memoryOptions.memoryTotalLimit
    //         << ", rsHostUsage: " << memoryUsageOfReadStorage.host << "\n";

    maxMemoryForTables = std::min(maxMemoryForTables, 
                            std::min(memoryOptions.memoryForHashtables, totalLimit));

    std::cerr << "maxMemoryForTables = " << maxMemoryForTables << " bytes\n";

    const int hashFunctionOffset = 0;

    
    std::vector<int> usedHashFunctionNumbers;
    
    helpers::SimpleAllocationDevice<unsigned int, 1> d_sequenceData(encodedSequencePitchInInts * parallelReads);
    helpers::SimpleAllocationDevice<int, 0> d_lengths(parallelReads);
    
    helpers::SimpleAllocationPinnedHost<read_number, 0> h_indices(parallelReads);
    helpers::SimpleAllocationDevice<read_number, 0> d_indices(parallelReads);
    
    std::size_t d_insert_temp_size = 0;
    std::size_t h_insert_temp_size = 0;
    insert(
        nullptr,
        d_insert_temp_size,
        nullptr,
        h_insert_temp_size,
        (const unsigned int*)nullptr,
        int(parallelReads),
        (const int*)nullptr,
        encodedSequencePitchInInts,
        (const read_number*)nullptr,
        (const read_number*)nullptr,
        0,
        requestedNumberOfMaps,
        (const int*)nullptr,
        (hipStream_t)0
    );
    
    
    CudaStream stream{};
    ThreadPool tpForHashing(runtimeOptions.threads);
    ThreadPool tpForCompacting(std::min(2,runtimeOptions.threads));

    
    setMemoryLimitForConstruction(maxMemoryForTables);
    
    //std::size_t bytesOfCachedConstructedTables = 0;
    int remainingHashFunctions = requestedNumberOfMaps;
    bool keepGoing = true;

    while(remainingHashFunctions > 0 && keepGoing){

        setThreadPool(&tpForHashing);

        const int alreadyExistingHashFunctions = requestedNumberOfMaps - remainingHashFunctions;
        int addedHashFunctions = addHashfunctions(remainingHashFunctions);

        if(addedHashFunctions == 0){
            keepGoing = false;
            break;
        }

        helpers::SimpleAllocationDevice<char, 0> d_temp(d_insert_temp_size);
        helpers::SimpleAllocationPinnedHost<char> h_temp(h_insert_temp_size);

        std::cout << "Constructing maps: ";
        for(int i = 0; i < addedHashFunctions; i++){
            std::cout << (alreadyExistingHashFunctions + i) << "(" << (hashFunctionOffset + alreadyExistingHashFunctions + i) << ") ";
        }
        std::cout << '\n';

        std::vector<int> h_hashfunctionNumbers(addedHashFunctions);
        std::iota(
            h_hashfunctionNumbers.begin(),
            h_hashfunctionNumbers.end(),
            alreadyExistingHashFunctions + hashFunctionOffset
        );

        usedHashFunctionNumbers.insert(usedHashFunctionNumbers.end(), h_hashfunctionNumbers.begin(), h_hashfunctionNumbers.end());

        for (int iter = 0; iter < numIters; iter++){
            read_number readIdBegin = iter * parallelReads;
            read_number readIdEnd = std::min((iter + 1) * parallelReads, numReads);

            const std::size_t curBatchsize = readIdEnd - readIdBegin;

            std::iota(h_indices.get(), h_indices.get() + curBatchsize, readIdBegin);

            hipMemcpyAsync(d_indices, h_indices, sizeof(read_number) * curBatchsize, H2D, stream); CUERR;

            gpuReadStorage.gatherSequences(
                sequencehandle,
                d_sequenceData,
                encodedSequencePitchInInts,
                h_indices,
                d_indices,
                curBatchsize,
                stream
            );
        
            gpuReadStorage.gatherSequenceLengths(
                sequencehandle,
                d_lengths,
                d_indices,
                curBatchsize,
                stream
            );

            std::size_t s1 = d_insert_temp_size;
            std::size_t s2 = h_insert_temp_size;

            insert(
                d_temp.data(),
                s1,
                h_temp.data(),
                s2,
                d_sequenceData,
                curBatchsize,
                d_lengths,
                encodedSequencePitchInInts,
                d_indices,
                h_indices,
                alreadyExistingHashFunctions,
                addedHashFunctions,
                h_hashfunctionNumbers.data(),
                stream
            );

            hipStreamSynchronize(stream); CUERR;
        }

        d_temp.destroy();
        h_temp.destroy();

        std::cerr << "Compacting\n";
        if(tpForCompacting.getConcurrency() > 1){
            setThreadPool(&tpForCompacting);
        }else{
            setThreadPool(nullptr);
        }
        
        finalize();

        remainingHashFunctions -= addedHashFunctions;
    }

    setThreadPool(nullptr); 
    
    gpuReadStorage.destroyHandle(sequencehandle);
}


FakeGpuMinhasher::Range_t FakeGpuMinhasher::queryMap(int id, const Key_t& key) const{
    HashTable::QueryResult qr = minhashTables[id]->query(key);

    return std::make_pair(qr.valuesBegin, qr.valuesBegin + qr.numValues);
}





} //namespace gpu
} //namespace care
