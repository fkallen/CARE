#include "hip/hip_runtime.h"
#include <gpu/fakegpuminhasher.cuh>
#include <hpc_helpers.cuh>
#include <gpu/kernels.hpp>

namespace care{
    namespace gpu{


        


void FakeGpuMinhasher::queryPrecalculatedSignatures(
    const std::uint64_t* signatures, //getNumberOfMaps() elements per sequence
    FakeGpuMinhasher::Range_t* ranges, //getNumberOfMaps() elements per sequence
    int* totalNumResultsInRanges, 
    int numSequences) const{ 
    
    int numResults = 0;
    const std::uint64_t kmer_mask = getKmerMask();

    for(int i = 0; i < numSequences; i++){
        const std::uint64_t* const signature = &signatures[i * getNumberOfMaps()];
        FakeGpuMinhasher::Range_t* const range = &ranges[i * getNumberOfMaps()];            

        for(int map = 0; map < getNumberOfMaps(); ++map){
            kmer_type key = signature[map] & kmer_mask;
            auto entries_range = queryMap(map, key);
            numResults += std::distance(entries_range.first, entries_range.second);
            range[map] = entries_range;
        }
    }   

    *totalNumResultsInRanges = numResults;   
}



void FakeGpuMinhasher::writeToStream(std::ostream& os) const{

    os.write(reinterpret_cast<const char*>(&kmerSize), sizeof(int));
    os.write(reinterpret_cast<const char*>(&resultsPerMapThreshold), sizeof(int));

    const int numTables = getNumberOfMaps();
    os.write(reinterpret_cast<const char*>(&numTables), sizeof(int));

    for(const auto& tableptr : minhashTables){
        tableptr->writeToStream(os);
    }
}

int FakeGpuMinhasher::loadFromStream(std::ifstream& is, int numMapsUpperLimit){
    destroy();

    is.read(reinterpret_cast<char*>(&kmerSize), sizeof(int));
    is.read(reinterpret_cast<char*>(&resultsPerMapThreshold), sizeof(int));

    int numMaps = 0;

    is.read(reinterpret_cast<char*>(&numMaps), sizeof(int));

    const int mapsToLoad = std::min(numMapsUpperLimit, numMaps);

    for(int i = 0; i < mapsToLoad; i++){
        HashTable table;
        table.loadFromStream(is);
        addHashTable(std::move(table));
    }

    return mapsToLoad;
}

int FakeGpuMinhasher::calculateResultsPerMapThreshold(int coverage){
    int result = int(coverage * 2.5f);
    result = std::min(result, int(std::numeric_limits<BucketSize>::max()));
    result = std::max(10, result);
    return result;
}


void FakeGpuMinhasher::constructFromReadStorage(
    const FileOptions &fileOptions,
    const RuntimeOptions &runtimeOptions,
    const MemoryOptions& memoryOptions,
    std::uint64_t nReads,
    const CorrectionOptions& correctionOptions,
    const DistributedReadStorage& gpuReadStorage
){
    
    auto& readStorage = gpuReadStorage;
    const auto& deviceIds = runtimeOptions.deviceIds;

    const int requestedNumberOfMaps = correctionOptions.numHashFunctions;

    const read_number numReads = readStorage.getNumberOfReads();
    const int maximumSequenceLength = readStorage.getSequenceLengthUpperBound();

    auto sequencehandle = readStorage.makeGatherHandleSequences();
    std::size_t sequencepitch = SequenceHelpers::getEncodedNumInts2Bit(maximumSequenceLength) * sizeof(int);

    const std::string tmpmapsFilename = fileOptions.tempdirectory + "/tmpmaps";
    std::ofstream outstream(tmpmapsFilename, std::ios::binary);
    if(!outstream){
        throw std::runtime_error("Could not open temp file " + tmpmapsFilename + "!");
    }


    const MemoryUsage memoryUsageOfReadStorage = readStorage.getMemoryInfo();
    std::size_t totalLimit = memoryOptions.memoryTotalLimit;
    if(totalLimit > memoryUsageOfReadStorage.host){
        totalLimit -= memoryUsageOfReadStorage.host;
    }else{
        totalLimit = 0;
    }
    if(totalLimit == 0){
        throw std::runtime_error("Not enough memory available for hash tables. Abort!");
    }
    std::size_t maxMemoryForTables = getAvailableMemoryInKB() * 1024;
    // std::cerr << "available: " << maxMemoryForTables 
    //         << ",memoryForHashtables: " << memoryOptions.memoryForHashtables
    //         << ", memoryTotalLimit: " << memoryOptions.memoryTotalLimit
    //         << ", rsHostUsage: " << memoryUsageOfReadStorage.host << "\n";

    maxMemoryForTables = std::min(maxMemoryForTables, 
                            std::min(memoryOptions.memoryForHashtables, totalLimit));

    std::cerr << "maxMemoryForTables = " << maxMemoryForTables << " bytes\n";


    std::size_t writtenTableBytes = 0;
    int numSavedTables = 0;

    int numConstructedTables = 0;
    std::vector<HashTable> cachedConstructedTables;
    std::size_t bytesOfCachedConstructedTables = 0;
    bool allowCaching = false;

    // std::vector<std::ofstream> keysoutput;
    // for(int i = 0; i < requestedNumberOfMaps; i++){
    //     keysoutput.emplace_back("hashkeys" + std::to_string(i));
    //     std::size_t num = numReads;
    //     keysoutput.back().write((const char*)&num, sizeof(std::size_t));
    // }

    while(numConstructedTables < requestedNumberOfMaps && maxMemoryForTables > (writtenTableBytes + bytesOfCachedConstructedTables)){

        int maxNumTables = 0;

        auto updateMaxNumTables = [&](){
            // (1 kmer + readid) per read
            std::size_t requiredMemPerTable = (sizeof(kmer_type) + sizeof(read_number)) * numReads;
            maxNumTables = (maxMemoryForTables - bytesOfCachedConstructedTables) / requiredMemPerTable;
            maxNumTables -= 2; // keep free memory of 2 tables to perform transformation 
            std::cerr << "requiredMemPerTable = " << requiredMemPerTable << "\n";
            std::cerr << "maxNumTables = " << maxNumTables << "\n";
        };

        updateMaxNumTables();

        //if at least 75 percent of all tables can be constructed in first iteration, keep all constructed tables in memory
        //else save constructed tables to file if there are less than requestedNumberOfMaps
        if(numConstructedTables == 0 && float(maxNumTables) / requestedNumberOfMaps >= 0.75f){
            allowCaching = true;
        }

        bool savedTooManyTablesToFile = false;

        if(maxNumTables <= 0){
            if(cachedConstructedTables.empty() && !allowCaching){
                throw std::runtime_error("Not enough memory to construct 1 table");
            }else{
                //save cached constructed tables to file to make room for more tables

                std::cerr << "saving cached constructed tables to file to make room for more tables\n";
                for(int i = 0; i < int(cachedConstructedTables.size()); i++){                            
                    const auto& hashTable = cachedConstructedTables[i];

                    auto memoryUsage = hashTable.getMemoryInfo();
                    hashTable.writeToStream(outstream);
                    numSavedTables++;
                    writtenTableBytes = outstream.tellp();

                    std::cerr << "tablesize = " << memoryUsage.host << "\n";
                    std::cerr << "written total of " << writtenTableBytes << " / " << maxMemoryForTables << "\n";
                    std::cerr << "numSavedTables = " << numSavedTables << "\n";

                    if(maxMemoryForTables <= writtenTableBytes){
                        savedTooManyTablesToFile = true;
                        std::cerr << "savedTooManyTablesToFile\n";
                        break;
                    }
                }
                cachedConstructedTables.clear();
                bytesOfCachedConstructedTables = 0;

                updateMaxNumTables();

                if(maxNumTables <= 0){                        
                    throw std::runtime_error("Not enough memory to construct 1 table");
                }
            }
        }

        if(!savedTooManyTablesToFile){

            const int currentIterNumTables = std::min(requestedNumberOfMaps - numConstructedTables, maxNumTables);

            std::pair< std::vector<std::vector<kmer_type>>, std::vector<std::vector<read_number>> >              
            initialMinhashes = computeKeyValuePairsForHashtableUsingGpu(                      
                currentIterNumTables, 
                numConstructedTables,
                readStorage.getNumberOfReads(),
                readStorage.getSequenceLengthUpperBound(),
                runtimeOptions,
                readStorage
            );

            std::size_t availableMemoryToSaveGpuPartitions = totalLimit;
            // account for the currently calculated minhash signatures
            for(const auto& vec : initialMinhashes.first){
                const std::size_t sub = vec.capacity() * sizeof(kmer_type);
                if(availableMemoryToSaveGpuPartitions > sub){
                    availableMemoryToSaveGpuPartitions -= sub;
                }else{
                    availableMemoryToSaveGpuPartitions = 0;
                }
            }
            for(const auto& vec : initialMinhashes.second){
                const std::size_t sub = vec.capacity() * sizeof(read_number);
                if(availableMemoryToSaveGpuPartitions > sub){
                    availableMemoryToSaveGpuPartitions -= sub;
                }else{
                    availableMemoryToSaveGpuPartitions = 0;
                }
            }
            //account for constructed tables in previous iteration
            if(availableMemoryToSaveGpuPartitions > bytesOfCachedConstructedTables){
                availableMemoryToSaveGpuPartitions -= bytesOfCachedConstructedTables;
            }else{
                availableMemoryToSaveGpuPartitions = 0;
            }

            for(int i = 0; i < 2 + int(minhashTables.size()); i++){
                const std::size_t requiredMemPerTable = nReads * sizeof(Key_t) //keys
                                                        + nReads * sizeof(Value_t) // values
                                                        + nReads * sizeof(Value_t) // counts prefix sum
                                                        + 4 * 1024;
                if(availableMemoryToSaveGpuPartitions > requiredMemPerTable){
                    availableMemoryToSaveGpuPartitions -= requiredMemPerTable;
                }else{
                    availableMemoryToSaveGpuPartitions = 0;
                    break;
                }
            }         

            // std::cerr << "availableMemoryToSaveGpuPartitions: " << availableMemoryToSaveGpuPartitions << "\n";

            DistributedReadStorage::SavedGpuData savedReadstorageGpuData;
            const std::string rstempfile = fileOptions.tempdirectory+"/rstemp";
            const bool didSaveGpudata = true;

            std::ofstream rstempostream(rstempfile, std::ios::binary);
            savedReadstorageGpuData = std::move(readStorage.saveGpuDataAndFreeGpuMem(rstempostream, availableMemoryToSaveGpuPartitions));
            
            constexpr bool valuesOfSameKeyMustBeSorted = false;
            
            //if all tables could be constructed at once, no need to save them to temporary file
            if(requestedNumberOfMaps == currentIterNumTables){

                for(int i = 0; i < currentIterNumTables; i++){
                    int globalTableId = numConstructedTables;
                    
                    if(runtimeOptions.showProgress){
                        std::cout << "Constructing hash table " << globalTableId << "." << std::endl;
                    }                           
                    
                    auto& kmers = initialMinhashes.first[i];
                    auto& readIds = initialMinhashes.second[i];

                    // for(int i = 0; i < 10; i++){
                    //     std::cerr << kmers[i] << " " << readIds[i] << "\n";
                    // }
                    const int maxValuesPerKey = getNumResultsPerMapThreshold();

                    //keysoutput[globalTableId].write((const char*)(kmers.data()), kmers.size() * sizeof(kmer_type));

                    HashTable hashTable(
                        std::move(kmers),
                        std::move(readIds), 
                        maxValuesPerKey,
                        deviceIds,
                        valuesOfSameKeyMustBeSorted
                    );

                    addHashTable(std::move(hashTable));

                    numConstructedTables++;
                }

                if(didSaveGpudata){
                    std::ifstream rstempistream(rstempfile, std::ios::binary);
                    readStorage.allocGpuMemAndLoadGpuData(rstempistream, savedReadstorageGpuData);
                    savedReadstorageGpuData.clear();
                    filehelpers::removeFile(rstempfile);
                }
                
            }else{

                for(int i = 0; i < currentIterNumTables; i++){
                    const int globalTableId = numConstructedTables;
                    
                    if(runtimeOptions.showProgress){
                        std::cout << "Constructing hash table " << globalTableId << "." << std::endl;
                    }                           
                    
                    auto& kmers = initialMinhashes.first[i];
                    auto& readIds = initialMinhashes.second[i];
                    const int maxValuesPerKey = getNumResultsPerMapThreshold();

                    HashTable hashTable(
                        std::move(kmers), 
                        std::move(readIds), 
                        maxValuesPerKey,
                        deviceIds,
                        valuesOfSameKeyMustBeSorted
                    );

                    numConstructedTables++;     

                    auto memoryUsage = hashTable.getMemoryInfo();
                    if(allowCaching){
                        bytesOfCachedConstructedTables += memoryUsage.host;
                        cachedConstructedTables.emplace_back(std::move(hashTable));

                        std::cerr << "cached " << cachedConstructedTables.size() << " constructed tables in memory\n";

                        if(maxMemoryForTables <= bytesOfCachedConstructedTables){
                            break;
                        }
                    }else{
                        hashTable.writeToStream(outstream);
                        numSavedTables++;
                        writtenTableBytes = outstream.tellp();
    
                        std::cerr << "tablesize = " << memoryUsage.host << "\n";
                        std::cerr << "written total of " << writtenTableBytes << " / " << maxMemoryForTables << "\n";
                        std::cerr << "numSavedTables = " << numSavedTables << "\n";

                        if(maxMemoryForTables <= writtenTableBytes){
                            break;
                        }
                    }                            
                }

                initialMinhashes.first.clear();
                initialMinhashes.second.clear();

                if(didSaveGpudata){
                    std::ifstream rstempistream(rstempfile, std::ios::binary);
                    readStorage.allocGpuMemAndLoadGpuData(rstempistream, savedReadstorageGpuData);
                    savedReadstorageGpuData.clear();
                }

                if(int(cachedConstructedTables.size()) + numSavedTables >= requestedNumberOfMaps 
                            || maxMemoryForTables < writtenTableBytes){

                    outstream.flush();

                    //discard any cached table such that size of cached tables + size of tables in file < memory limit
                    std::size_t totalTableBytes = writtenTableBytes;
                    int end = 0;
                    for(int i = 0; i < int(cachedConstructedTables.size()); i++){
                        const auto& table = cachedConstructedTables[i];
                        auto memoryUsage = table.getMemoryInfo();

                        if(totalTableBytes + memoryUsage.host <= maxMemoryForTables){
                            totalTableBytes += memoryUsage.host;
                            end++;
                        }else{
                            break;
                        }
                    }
                    cachedConstructedTables.erase(cachedConstructedTables.begin() + end, cachedConstructedTables.end());
                    
                    int usableNumMaps = loadConstructedTablesFromFile(
                                                tmpmapsFilename, 
                                                numSavedTables, 
                                                maxMemoryForTables);

                    for(int i = 0; i < int(cachedConstructedTables.size()) && usableNumMaps < requestedNumberOfMaps; i++){
                        auto& table = cachedConstructedTables[i];
                        addHashTable(std::move(table));
                        
                        usableNumMaps++;
                    }

                    filehelpers::removeFile(tmpmapsFilename);
                    if(didSaveGpudata){
                        filehelpers::removeFile(rstempfile);
                    }

                    std::cout << "Can use " << usableNumMaps 
                        << " out of specified " << requestedNumberOfMaps
                        << " tables\n";
                } 
            } 
        }else{
            //all constructed tables have been saved to file, and no table is cached

            outstream.flush();

            int usableNumMaps = loadConstructedTablesFromFile(
                                            tmpmapsFilename, 
                                            numSavedTables, 
                                            maxMemoryForTables);

            std::cout << "Can use " << usableNumMaps 
                << " out of specified " << requestedNumberOfMaps
                << " tables\n";
        }
    }
}




FakeGpuMinhasher::Range_t FakeGpuMinhasher::queryMap(int id, const Key_t& key) const{
    HashTable::QueryResult qr = minhashTables[id]->query(key);

    return std::make_pair(qr.valuesBegin, qr.valuesBegin + qr.numValues);
}

void FakeGpuMinhasher::addHashTable(HashTable&& hm){
    minhashTables.emplace_back(std::make_unique<HashTable>(std::move(hm)));
}




std::pair< std::vector<std::vector<kmer_type>>, std::vector<std::vector<read_number>> > 
FakeGpuMinhasher::computeKeyValuePairsForHashtableUsingGpu(
    int numTables, 
    int firstTableId,
    std::int64_t numberOfReads,
    int upperBoundSequenceLength,
    const RuntimeOptions& runtimeOptions,
    const DistributedReadStorage& readStorage
){

    constexpr read_number parallelReads = 1000000;
    read_number numReads = numberOfReads;
    const int numIters = SDIV(numReads, parallelReads);
    const std::size_t encodedSequencePitchInInts = SequenceHelpers::getEncodedNumInts2Bit(upperBoundSequenceLength);

    const auto& deviceIds = runtimeOptions.deviceIds;
    const int numThreads = runtimeOptions.threads;

    const std::uint64_t kmer_mask = getKmerMask();

    assert(deviceIds.size() > 0);

    const int deviceId = deviceIds[0];

    hipSetDevice(deviceId); CUERR;

    const int numHashFuncs = numTables;
    const int firstHashFunc = firstTableId;
    const std::size_t signaturesRowPitchElements = numHashFuncs;

    ThreadPool::ParallelForHandle pforHandle;

    std::vector<std::vector<kmer_type>> kmersPerFunc(numTables);
    std::vector<std::vector<read_number>> readIdsPerFunc(numTables);

    for(auto& v : kmersPerFunc){
        v.resize(numberOfReads);
    }

    for(auto& v : readIdsPerFunc){
        v.resize(numberOfReads);
    }

    std::vector<int> tableIds(numTables);                
    std::vector<int> hashIds(numTables);
    
    std::iota(tableIds.begin(), tableIds.end(), 0);

    std::cout << "Constructing maps: ";
    for(int i = 0; i < numTables; i++){
        std::cout << (firstTableId + i) << ' ';
    }
    std::cout << '\n';

    auto showProgress = [&](auto totalCount, auto seconds){
        if(runtimeOptions.showProgress){
            std::cout << "Hashed " << totalCount << " / " << numReads << " reads. Elapsed time: " 
                    << seconds << " seconds.\n";
        }
    };

    auto updateShowProgressInterval = [](auto duration){
        return duration * 2;
    };

    ProgressThread<read_number> progressThread(numReads, showProgress, updateShowProgressInterval);

    ThreadPool threadPool(numThreads);

    helpers::SimpleAllocationDevice<unsigned int, 1> d_sequenceData(encodedSequencePitchInInts * parallelReads);
    helpers::SimpleAllocationDevice<int, 0> d_lengths(parallelReads);

    helpers::SimpleAllocationPinnedHost<read_number, 0> h_indices(parallelReads);
    helpers::SimpleAllocationDevice<read_number, 0> d_indices(parallelReads);

    helpers::SimpleAllocationPinnedHost<std::uint64_t, 0> h_signatures(signaturesRowPitchElements * parallelReads);
    helpers::SimpleAllocationDevice<std::uint64_t, 0> d_signatures(signaturesRowPitchElements * parallelReads);

    hipStream_t stream;
    hipStreamCreate(&stream); CUERR;

    auto sequencehandle = readStorage.makeGatherHandleSequences();


    for (int iter = 0; iter < numIters; iter++){
        read_number readIdBegin = iter * parallelReads;
        read_number readIdEnd = std::min((iter + 1) * parallelReads, numReads);

        const std::size_t curBatchsize = readIdEnd - readIdBegin;

        std::iota(h_indices.get(), h_indices.get() + curBatchsize, readIdBegin);

        hipMemcpyAsync(d_indices, h_indices, sizeof(read_number) * curBatchsize, H2D, stream); CUERR;

        readStorage.gatherSequenceDataToGpuBufferAsync(
            &threadPool,
            sequencehandle,
            d_sequenceData,
            encodedSequencePitchInInts,
            h_indices,
            d_indices,
            curBatchsize,
            deviceId,
            stream
        );
    
        readStorage.gatherSequenceLengthsToGpuBufferAsync(
            d_lengths,
            deviceId,
            d_indices,
            curBatchsize,
            stream
        );

        callMinhashSignaturesKernel(
            d_signatures,
            signaturesRowPitchElements,
            d_sequenceData,
            encodedSequencePitchInInts,
            curBatchsize,
            d_lengths,
            getKmerSize(),
            numHashFuncs,
            firstHashFunc,
            stream
        ); CUERR;

        hipMemcpyAsync(
            h_signatures, 
            d_signatures, 
            signaturesRowPitchElements * sizeof(std::uint64_t) * curBatchsize, 
            D2H, 
            stream
        ); CUERR;

        hipStreamSynchronize(stream); CUERR;


        auto lambda = [&, readIdBegin](auto begin, auto end, int threadId) {
            std::uint64_t countlimit = 10000;
            std::uint64_t count = 0;

            for (read_number readId = begin; readId < end; readId++){
                read_number localId = readId - readIdBegin;

                for(int i = 0; i < numHashFuncs; i++){
                    const kmer_type kmer = kmer_mask & h_signatures[signaturesRowPitchElements * localId + i];
                    kmersPerFunc[i][readId] = kmer;
                    readIdsPerFunc[i][readId] = readId;
                }
                
                count++;
                if(count == countlimit){
                    progressThread.addProgress(count);
                    count = 0;                                                         
                }
            }
            if(count > 0){
                progressThread.addProgress(count);
            }
        };

        threadPool.parallelFor(
            pforHandle,
            readIdBegin,
            readIdEnd,
            std::move(lambda));

        //TIMERSTOPCPU(insert);
    }

    progressThread.finished();

    hipStreamDestroy(stream); CUERR;

    return {std::move(kmersPerFunc), std::move(readIdsPerFunc)};
}





int FakeGpuMinhasher::loadConstructedTablesFromFile(
    const std::string& filename,
    int numTablesToLoad, 
    std::size_t availableMemory
){

    std::cerr << "available before loading maps: " << availableMemory << "\n";
    
    int assignedNumMaps = 0;

    //load as many transformed tables from file as possible and move them to minhasher
    std::ifstream instream(filename, std::ios::binary);
    for(int i = 0; i < numTablesToLoad; i++){
        try{
            std::cerr << "try loading table " << i << "\n";
            HashTable table;
            table.loadFromStream(instream);
            const auto memoryUsage = table.getMemoryInfo();
            const std::size_t tablesize = memoryUsage.host;

            if(availableMemory > tablesize){
                availableMemory -= tablesize;

                addHashTable(std::move(table));

                std::cerr << "available after loading table " << i << ": " << (getAvailableMemoryInKB() * 1024) << "\n";
                assignedNumMaps++;
                std::cerr << "usable num maps = " << assignedNumMaps << "\n";
            }else if(availableMemory == tablesize){
                availableMemory -= tablesize;

                addHashTable(std::move(table));
                
                std::cerr << "available after loading table " << i << ": " << (getAvailableMemoryInKB() * 1024) << "\n";
                assignedNumMaps++;
                std::cerr << "usable num maps = " << assignedNumMaps << "\n";
                break;
            }else{
                std::cerr << "Loading table " << i << " failed\n";
                break;
            }
        }catch(...){
            std::cerr << "Loading table " << i << " failed\n";
            break;
        }                        
    }

    return assignedNumMaps;
}




} //namespace gpu
} //namespace care
