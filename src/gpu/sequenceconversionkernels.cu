#include "hip/hip_runtime.h"
#include <gpu/kernels.hpp>
#include <gpu/utility_kernels.cuh>
#include <hpc_helpers.cuh>
#include <config.hpp>
#include <sequence.hpp>

#include <cassert>
#include <hip/hip_cooperative_groups.h>


//#define DO_CHECK_CONVERSIONS

namespace cg = cooperative_groups;

namespace care{
namespace gpu{

template<class First2Bit, class First2BitHilo, class Trafo2Bit, class Trafo2BitHilo>
__global__
void checkSequenceConversionKernel(const unsigned int* const __restrict__ normalData,
        size_t normalpitchInInts, // max num ints per input sequence
        const unsigned int*  const __restrict__ hiloData,
        size_t hilopitchInInts, // max num ints per output sequence
        const int* const __restrict__ sequenceLengths,
        int numSequences,
        First2Bit first2Bit,
        First2BitHilo first2BitHilo,
        Trafo2Bit trafo2Bit,
        Trafo2BitHilo trafo2BitHilo){

    constexpr char A_enc = 0x00;
    constexpr char C_enc = 0x01;
    constexpr char G_enc = 0x02;
    constexpr char T_enc = 0x03;

    auto to_nuc = [](char c){
        switch(c){
        case A_enc: return 'A';
        case C_enc: return 'C';
        case G_enc: return 'G';
        case T_enc: return 'T';
        default: assert(false); return 'F';
        }
    };

    //use one block per sequence
    for(int index = blockIdx.x; index < numSequences; index += gridDim.x){
        const int sequenceLength = sequenceLengths[index];
        const unsigned int* const normalSeq = normalData + first2Bit(index);
        const unsigned int* const hiloSeq = hiloData + first2BitHilo(index);    
        
        for(int p = threadIdx.x; p < sequenceLength; p += blockDim.x){
            char encnormal = getEncodedNuc2Bit(normalSeq, sequenceLength, p, trafo2Bit);
            char basenormal = to_nuc(encnormal);
            char enchilo = getEncodedNuc2BitHiLo(hiloSeq, sequenceLength, p, trafo2BitHilo);
            char basehilo = to_nuc(enchilo);
            if(basenormal != basehilo){
                printf("error seq %d position %d, normal %c hilo %c\n", index, p, basenormal, basehilo);
            }
            assert(basenormal == basehilo);
        }
    }
}   

void callCheckSequenceConversionKernelNN(const unsigned int* normalData,
        size_t normalpitchInInts,
        const unsigned int* hiloData,
        size_t hilopitchInInts,
        const int* sequenceLengths,
        int numSequences,
        hipStream_t stream){

    auto first2Bit = [=] __device__ (auto i){return i * normalpitchInInts;};
    auto first2BitHilo = [=] __device__ (auto i){return i * hilopitchInInts;};
    auto trafo2Bit = [=] __device__ (auto i){return i;};
    auto trafo2BitHilo = [=] __device__ (auto i){return i;};

    const int blocksize = 128;
    const int gridsize = std::min(numSequences, 65535);

    checkSequenceConversionKernel<<<gridsize,blocksize, 0, stream>>>(
        normalData,
        normalpitchInInts,
        hiloData,
        hilopitchInInts,
        sequenceLengths,
        numSequences,
        first2Bit,
        first2BitHilo,
        trafo2Bit,
        trafo2BitHilo
    ); CUERR;
}

void callCheckSequenceConversionKernelNT(const unsigned int* normalData,
        size_t normalpitchInInts,
        const unsigned int* hiloData,
        size_t hilopitchInInts,
        const int* sequenceLengths,
        int numSequences,
        hipStream_t stream){

    auto first2Bit = [=] __device__ (auto i){return i * normalpitchInInts;};
    auto first2BitHilo = [=] __device__ (auto i){return i;};
    auto trafo2Bit = [=] __device__ (auto i){return i;};
    auto trafo2BitHilo = [=] __device__ (auto i){return i * numSequences;};

    const int blocksize = 128;
    const int gridsize = std::min(numSequences, 65535);

    checkSequenceConversionKernel<<<gridsize,blocksize, 0, stream>>>(
        normalData,
        normalpitchInInts,
        hiloData,
        hilopitchInInts,
        sequenceLengths,
        numSequences,
        first2Bit,
        first2BitHilo,
        trafo2Bit,
        trafo2BitHilo
    ); CUERR;
}

void callCheckSequenceConversionKernelTT(const unsigned int* normalData,
        size_t normalpitchInInts,
        const unsigned int* hiloData,
        size_t hilopitchInInts,
        const int* sequenceLengths,
        int numSequences,
        hipStream_t stream){

    auto first2Bit = [=] __device__ (auto i){return i;};
    auto first2BitHilo = [=] __device__ (auto i){return i;};
    auto trafo2Bit = [=] __device__ (auto i){return i * numSequences;};
    auto trafo2BitHilo = [=] __device__ (auto i){return i * numSequences;};

    const int blocksize = 128;
    const int gridsize = std::min(numSequences, 65535);

    checkSequenceConversionKernel<<<gridsize, blocksize, 0, stream>>>(
        normalData,
        normalpitchInInts,
        hiloData,
        hilopitchInInts,
        sequenceLengths,
        numSequences,
        first2Bit,
        first2BitHilo,
        trafo2Bit,
        trafo2BitHilo
    ); CUERR;
}

 
template<int groupsize>
__global__
void convert2BitTo2BitHiloKernelNN(
        const unsigned int* const __restrict__ inputdata,
        size_t inputpitchInInts, // max num ints per input sequence
        unsigned int*  const __restrict__ outputdata,
        size_t outputpitchInInts, // max num ints per output sequence
        const int* const __restrict__ sequenceLengths,
        const int* __restrict__ numSequencesPtr){

    const int numSequences = *numSequencesPtr;

    auto inputStartIndex = [&](auto i){return i * inputpitchInInts;};
    auto outputStartIndex = [&](auto i){return i * outputpitchInInts;};
    auto inputTrafo = [&](auto i){return i;};
    auto outputTrafo = [&](auto i){return i;};

    auto extractEvenBits = [](unsigned int x){
        x = x & 0x55555555;
        x = (x | (x >> 1)) & 0x33333333;
        x = (x | (x >> 2)) & 0x0F0F0F0F;
        x = (x | (x >> 4)) & 0x00FF00FF;
        x = (x | (x >> 8)) & 0x0000FFFF;
        return x;
    };

    auto convert = [&](auto group,
                        unsigned int* out,
                        const unsigned int* in,
                        int length,
                        auto inindextrafo,
                        auto outindextrafo){

        const int inInts = getEncodedNumInts2Bit(length);
        const int outInts = getEncodedNumInts2BitHiLo(length);

        unsigned int* const outHi = out;
        unsigned int* const outLo = out + outindextrafo(outInts/2);

        for(int i = group.thread_rank(); i < outInts / 2; i += group.size()){
            const int outIndex = outindextrafo(i);
            const int inindex1 = inindextrafo(i*2);

            const unsigned int data1 = in[inindex1];
            const unsigned int even161 = extractEvenBits(data1);
            const unsigned int odd161 = extractEvenBits(data1 >> 1);

            unsigned int resultHi = odd161 << 16;
            unsigned int resultLo = even161 << 16;

            if((i < outInts / 2 - 1) || ((length-1) % 32) >= 16){
                const int inindex2 = inindextrafo(i*2 + 1);

                const unsigned int data2 = in[inindex2];
                const unsigned int even162 = extractEvenBits(data2);
                const unsigned int odd162 = extractEvenBits(data2 >> 1);

                resultHi = resultHi | odd162;
                resultLo = resultLo | even162;
            }

            outHi[outIndex] = resultHi;
            outLo[outIndex] = resultLo;
        }
    };

    auto group = cg::tiled_partition<groupsize>(cg::this_thread_block());
    const int numGroups = (blockDim.x * gridDim.x) / groupsize;
    const int groupId = (threadIdx.x + blockIdx.x * blockDim.x) / groupsize;

    for(int index = groupId; index < numSequences; index += numGroups){
        const int sequenceLength = sequenceLengths[index];
        const unsigned int* const in = inputdata + inputStartIndex(index);
        unsigned int* const out = outputdata + outputStartIndex(index);            

        convert(
            group,
            out,
            in,
            sequenceLength,
            inputTrafo,
            outputTrafo
        );
    } 
}

__global__
void convert2BitTo2BitHiloKernelNT(
        const unsigned int* const __restrict__ inputdata,
        size_t inputpitchInInts, // max num ints per input sequence
        unsigned int*  const __restrict__ outputdata,
        size_t outputpitchInInts, // max num ints per output sequence
        const int* const __restrict__ sequenceLengths,
        const int* __restrict__ numSequencesPtr){

    const int numSequences = *numSequencesPtr;

    auto inputStartIndex = [&](auto i){return i * inputpitchInInts;};
    auto outputStartIndex = [&](auto i){return i;};
    auto inputTrafo = [&](auto i){return i;};
    auto outputTrafo = [&](auto i){return i * numSequences;};

    auto extractEvenBits = [](unsigned int x){
        x = x & 0x55555555;
        x = (x | (x >> 1)) & 0x33333333;
        x = (x | (x >> 2)) & 0x0F0F0F0F;
        x = (x | (x >> 4)) & 0x00FF00FF;
        x = (x | (x >> 8)) & 0x0000FFFF;
        return x;
    };

    auto convert = [&](auto group,
                        unsigned int* out,
                        const unsigned int* in,
                        int length,
                        auto inindextrafo,
                        auto outindextrafo){

        const int inInts = getEncodedNumInts2Bit(length);
        const int outInts = getEncodedNumInts2BitHiLo(length);

        unsigned int* const outHi = out;
        unsigned int* const outLo = out + outindextrafo(outInts/2);

        for(int i = group.thread_rank(); i < outInts / 2; i += group.size()){
            const int outIndex = outindextrafo(i);
            const int inindex1 = inindextrafo(i*2);

            const unsigned int data1 = in[inindex1];
            const unsigned int even161 = extractEvenBits(data1);
            const unsigned int odd161 = extractEvenBits(data1 >> 1);

            unsigned int resultHi = odd161 << 16;
            unsigned int resultLo = even161 << 16;

            if((i < outInts / 2 - 1) || ((length-1) % 32) >= 16){
                const int inindex2 = inindextrafo(i*2 + 1);

                const unsigned int data2 = in[inindex2];
                const unsigned int even162 = extractEvenBits(data2);
                const unsigned int odd162 = extractEvenBits(data2 >> 1);

                resultHi = resultHi | odd162;
                resultLo = resultLo | even162;
            }

            outHi[outIndex] = resultHi;
            outLo[outIndex] = resultLo;
        }
    };

    auto group = cg::tiled_partition<1>(cg::this_thread_block());
    const int numGroups = (blockDim.x * gridDim.x) / group.size();
    const int groupId = (threadIdx.x + blockIdx.x * blockDim.x) / group.size();

    for(int index = groupId; index < numSequences; index += numGroups){
        const int sequenceLength = sequenceLengths[index];
        const unsigned int* const in = inputdata + inputStartIndex(index);
        unsigned int* const out = outputdata + outputStartIndex(index);            

        convert(
            group,
            out,
            in,
            sequenceLength,
            inputTrafo,
            outputTrafo
        );
    } 
}



__global__
void convert2BitTo2BitHiloKernelTT(
        const unsigned int* const __restrict__ inputdata,
        size_t inputpitchInInts, // max num ints per input sequence
        unsigned int*  const __restrict__ outputdata,
        size_t outputpitchInInts, // max num ints per output sequence
        const int* const __restrict__ sequenceLengths,
        const int* __restrict__ numSequencesPtr){

    const int numSequences = *numSequencesPtr;

    auto inputStartIndex = [&](auto i){return i;};
    auto outputStartIndex = [&](auto i){return i;};
    auto inputTrafo = [&](auto i){return i * numSequences;};
    auto outputTrafo = [&](auto i){return i * numSequences;};

    auto extractEvenBits = [](unsigned int x){
        x = x & 0x55555555;
        x = (x | (x >> 1)) & 0x33333333;
        x = (x | (x >> 2)) & 0x0F0F0F0F;
        x = (x | (x >> 4)) & 0x00FF00FF;
        x = (x | (x >> 8)) & 0x0000FFFF;
        return x;
    };

    auto convert = [&](auto group,
                        unsigned int* out,
                        const unsigned int* in,
                        int length,
                        auto inindextrafo,
                        auto outindextrafo){

        const int inInts = getEncodedNumInts2Bit(length);
        const int outInts = getEncodedNumInts2BitHiLo(length);

        unsigned int* const outHi = out;
        unsigned int* const outLo = out + outindextrafo(outInts/2);

        for(int i = group.thread_rank(); i < outInts / 2; i += group.size()){
            const int outIndex = outindextrafo(i);
            const int inindex1 = inindextrafo(i*2);

            const unsigned int data1 = in[inindex1];
            const unsigned int even161 = extractEvenBits(data1);
            const unsigned int odd161 = extractEvenBits(data1 >> 1);

            unsigned int resultHi = odd161 << 16;
            unsigned int resultLo = even161 << 16;

            if((i < outInts / 2 - 1) || ((length-1) % 32) >= 16){
                const int inindex2 = inindextrafo(i*2 + 1);

                const unsigned int data2 = in[inindex2];
                const unsigned int even162 = extractEvenBits(data2);
                const unsigned int odd162 = extractEvenBits(data2 >> 1);

                resultHi = resultHi | odd162;
                resultLo = resultLo | even162;
            }

            outHi[outIndex] = resultHi;
            outLo[outIndex] = resultLo;
        }
    };

    auto group = cg::tiled_partition<1>(cg::this_thread_block());
    const int numGroups = (blockDim.x * gridDim.x) / group.size();
    const int groupId = (threadIdx.x + blockIdx.x * blockDim.x) / group.size();

    for(int index = groupId; index < numSequences; index += numGroups){
        const int sequenceLength = sequenceLengths[index];
        const unsigned int* const in = inputdata + inputStartIndex(index);
        unsigned int* const out = outputdata + outputStartIndex(index);            

        convert(
            group,
            out,
            in,
            sequenceLength,
            inputTrafo,
            outputTrafo
        );
    } 
}





void callConversionKernel2BitTo2BitHiLoNN(
        const unsigned int* d_inputdata,
        size_t inputpitchInInts,
        unsigned int* d_outputdata,
        size_t outputpitchInInts,
        const int* d_sequenceLengths,
        const int* d_numSequences,
        int maxNumSequences,
        hipStream_t stream,
        KernelLaunchHandle& handle){

    
    constexpr int groupsize = 8;        
    constexpr int blocksize = 128;
    constexpr size_t smem = 0;
    
    int max_blocks_per_device = 1;

    KernelLaunchConfig kernelLaunchConfig;
    kernelLaunchConfig.threads_per_block = blocksize;
    kernelLaunchConfig.smem = smem;

    auto iter = handle.kernelPropertiesMap.find(KernelId::Conversion2BitTo2BitHiLoNN);
    if(iter == handle.kernelPropertiesMap.end()) {

        std::map<KernelLaunchConfig, KernelProperties> mymap;

        #define getProp(blocksize) { \
                KernelLaunchConfig kernelLaunchConfig; \
                kernelLaunchConfig.threads_per_block = (blocksize); \
                kernelLaunchConfig.smem = 0; \
                KernelProperties kernelProperties; \
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                    convert2BitTo2BitHiloKernelNN<groupsize>, \
                            kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
                mymap[kernelLaunchConfig] = kernelProperties; \
        }
        //getProp(1);
        getProp(32);
        getProp(64);
        getProp(96);
        getProp(128);
        getProp(160);
        getProp(192);
        getProp(224);
        getProp(256);

        const auto& kernelProperties = mymap[kernelLaunchConfig];
        max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

        handle.kernelPropertiesMap[KernelId::Conversion2BitTo2BitHiLoNN] = std::move(mymap);

        #undef getProp
    }else{
        std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
        const KernelProperties& kernelProperties = map[kernelLaunchConfig];
        max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
    }

    dim3 block(blocksize,1,1);
    //dim3 grid(std::min(max_blocks_per_device, SDIV(maxNumSequences * groupsize, blocksize)), 1, 1);
    dim3 grid(max_blocks_per_device, 1, 1);

    convert2BitTo2BitHiloKernelNN<groupsize><<<grid, block, 0, stream>>>(
        d_inputdata,
        inputpitchInInts,
        d_outputdata,
        outputpitchInInts,
        d_sequenceLengths,
        d_numSequences); CUERR;

#ifdef DO_CHECK_CONVERSIONS        

    callCheckSequenceConversionKernelNN(d_inputdata,
        inputpitchInInts,
        d_outputdata,
        outputpitchInInts,
        d_sequenceLengths,
        numSequences,
        stream);

#endif

}

void callConversionKernel2BitTo2BitHiLoNT(
        const unsigned int* d_inputdata,
        size_t inputpitchInInts,
        unsigned int* d_outputdata,
        size_t outputpitchInInts,
        const int* d_sequenceLengths,
        const int* d_numSequences,
        int maxNumSequences,
        hipStream_t stream,
        KernelLaunchHandle& handle){

    int max_blocks_per_device = 1;

    constexpr int blocksize = 128;
    constexpr size_t smem = 0;

    KernelLaunchConfig kernelLaunchConfig;
    kernelLaunchConfig.threads_per_block = blocksize;
    kernelLaunchConfig.smem = smem;

    auto iter = handle.kernelPropertiesMap.find(KernelId::Conversion2BitTo2BitHiLoNT);
    if(iter == handle.kernelPropertiesMap.end()) {

        std::map<KernelLaunchConfig, KernelProperties> mymap;

        #define getProp(blocksize) { \
                KernelLaunchConfig kernelLaunchConfig; \
                kernelLaunchConfig.threads_per_block = (blocksize); \
                kernelLaunchConfig.smem = 0; \
                KernelProperties kernelProperties; \
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                    convert2BitTo2BitHiloKernelNT, \
                            kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
                mymap[kernelLaunchConfig] = kernelProperties; \
        }
        getProp(1);
        getProp(32);
        getProp(64);
        getProp(96);
        getProp(128);
        getProp(160);
        getProp(192);
        getProp(224);
        getProp(256);

        const auto& kernelProperties = mymap[kernelLaunchConfig];
        max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

        handle.kernelPropertiesMap[KernelId::Conversion2BitTo2BitHiLoNT] = std::move(mymap);

        #undef getProp
    }else{
        std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
        const KernelProperties& kernelProperties = map[kernelLaunchConfig];
        max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
    }

    dim3 block(blocksize,1,1);
    //dim3 grid(std::min(max_blocks_per_device, SDIV(maxNumSequences, blocksize)), 1, 1);
    dim3 grid(max_blocks_per_device, 1, 1);

    convert2BitTo2BitHiloKernelNT<<<grid, block, 0, stream>>>(
        d_inputdata,
        inputpitchInInts,
        d_outputdata,
        outputpitchInInts,
        d_sequenceLengths,
        d_numSequences); CUERR;

#if 0    

    callCheckSequenceConversionKernelNT(d_inputdata,
        inputpitchInInts,
        d_outputdata,
        outputpitchInInts,
        d_sequenceLengths,
        numSequences,
        stream);

#endif

}

void callConversionKernel2BitTo2BitHiLoTT(
        const unsigned int* d_inputdata,
        size_t inputpitchInInts,
        unsigned int* d_outputdata,
        size_t outputpitchInInts,
        const int* d_sequenceLengths,
        const int* d_numSequences,
        int maxNumSequences,
        hipStream_t stream,
        KernelLaunchHandle& handle){

    int max_blocks_per_device = 1;

    constexpr int blocksize = 128;
    constexpr size_t smem = 0;

    KernelLaunchConfig kernelLaunchConfig;
    kernelLaunchConfig.threads_per_block = blocksize;
    kernelLaunchConfig.smem = smem;

    auto iter = handle.kernelPropertiesMap.find(KernelId::Conversion2BitTo2BitHiLoTT);
    if(iter == handle.kernelPropertiesMap.end()) {

        std::map<KernelLaunchConfig, KernelProperties> mymap;

        #define getProp(blocksize) { \
                KernelLaunchConfig kernelLaunchConfig; \
                kernelLaunchConfig.threads_per_block = (blocksize); \
                kernelLaunchConfig.smem = 0; \
                KernelProperties kernelProperties; \
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                    convert2BitTo2BitHiloKernelTT, \
                            kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
                mymap[kernelLaunchConfig] = kernelProperties; \
        }
        getProp(1);
        getProp(32);
        getProp(64);
        getProp(96);
        getProp(128);
        getProp(160);
        getProp(192);
        getProp(224);
        getProp(256);

        const auto& kernelProperties = mymap[kernelLaunchConfig];
        max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

        handle.kernelPropertiesMap[KernelId::Conversion2BitTo2BitHiLoTT] = std::move(mymap);

        #undef getProp
    }else{
        std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
        const KernelProperties& kernelProperties = map[kernelLaunchConfig];
        max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
    }

    dim3 block(blocksize,1,1);
    //dim3 grid(std::min(max_blocks_per_device, SDIV(maxNumSequences, blocksize)), 1, 1);
    dim3 grid(max_blocks_per_device, 1, 1);

    convert2BitTo2BitHiloKernelTT<<<grid, block, 0, stream>>>(
        d_inputdata,
        inputpitchInInts,
        d_outputdata,
        outputpitchInInts,
        d_sequenceLengths,
        d_numSequences); CUERR;

#if 0            

    callCheckSequenceConversionKernelTT(d_inputdata,
        inputpitchInInts,
        d_outputdata,
        outputpitchInInts,
        d_sequenceLengths,
        numSequences,
        stream);

#endif 
        
}



}
}




