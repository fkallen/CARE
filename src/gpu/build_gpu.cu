#include "hip/hip_runtime.h"
#include <build.hpp>

#include <config.hpp>

#include <options.hpp>

#include <minhasher.hpp>
#include <minhasher_transform.hpp>
#include "readstorage.hpp"
#include "sequencefileio.hpp"
#include "sequence.hpp"
#include "threadsafe_buffer.hpp"

#include <map>
#include <stdexcept>
#include <iostream>
#include <limits>
#include <thread>
#include <future>
#include <mutex>
#include <iterator>
#include <random>
#include <omp.h>


#ifdef __NVCC__

namespace care{
namespace gpu{



    BuiltDataStructure<GpuReadStorageWithFlags> buildGpuReadStorage(const FileOptions& fileOptions,
                                                const RuntimeOptions& runtimeOptions,
                                                bool useQualityScores,
                                                read_number expectedNumberOfReads,
                                                int expectedMaximumReadLength){



        if(fileOptions.load_binary_reads_from != ""){
            BuiltDataStructure<GpuReadStorageWithFlags> result;
            auto& readStorage = result.data.readStorage;

            readStorage.loadFromFile(fileOptions.load_binary_reads_from, runtimeOptions.deviceIds);
            result.builtType = BuiltType::Loaded;

            if(useQualityScores && !readStorage.canUseQualityScores())
                throw std::runtime_error("Quality scores are required but not present in compressed sequence file!");
            if(!useQualityScores && readStorage.canUseQualityScores())
                std::cerr << "Warning. The loaded compressed read file contains quality scores, but program does not use them!\n";

            std::cout << "Loaded binary reads from " << fileOptions.load_binary_reads_from << std::endl;

            return result;
        }else{
            //int nThreads = std::max(1, std::min(runtimeOptions.threads, 2));
            const int nThreads = std::max(1, runtimeOptions.threads);

            constexpr std::array<char, 4> bases = {'A', 'C', 'G', 'T'};
            int Ncount = 0;

            BuiltDataStructure<GpuReadStorageWithFlags> result;
            DistributedReadStorage& readstorage = result.data.readStorage;
            auto& validFlags = result.data.readIsValidFlags;

            readstorage = std::move(DistributedReadStorage{runtimeOptions.deviceIds, expectedNumberOfReads, useQualityScores, expectedMaximumReadLength});
            validFlags.resize(expectedNumberOfReads, false);
            result.builtType = BuiltType::Constructed;

            constexpr size_t maxbuffersize = 1000000;

            std::map<int,int> nmap{};

            auto flushBuffers = [&](std::vector<read_number>& indicesBuffer, std::vector<Read>& readsBuffer){
                if(indicesBuffer.size() > 0){
                    readstorage.setReads(indicesBuffer, readsBuffer, nThreads);
                    indicesBuffer.clear();
                    readsBuffer.clear();
                }
            };

            auto handle_read = [&](std::uint64_t readIndex, Read& read, std::vector<read_number>& indicesBuffer, std::vector<Read>& readsBuffer){
                const int readLength = int(read.sequence.size());

                if(readIndex >= expectedNumberOfReads){
                    throw std::runtime_error("Error! Expected " + std::to_string(expectedNumberOfReads)
                                            + " reads, but file contains at least "
                                            + std::to_string(readIndex) + " reads.");
                }

                if(readLength > expectedMaximumReadLength){
                    throw std::runtime_error("Error! Expected maximum read length = "
                                            + std::to_string(expectedMaximumReadLength)
                                            + ", but read " + std::to_string(readIndex)
                                            + "has length " + std::to_string(readLength));
                }

                const int undeterminedBasesInRead = std::count_if(read.sequence.begin(), read.sequence.end(), [](char c){
                    return c == 'N' || c == 'n';
                });

                nmap[undeterminedBasesInRead]++;

                //if(undeterminedBasesInRead > 10){
                //    validFlags[readIndex] = false;
                //}else{
                    for(auto& c : read.sequence){
                        if(c == 'a') c = 'A';
                        if(c == 'c') c = 'C';
                        if(c == 'g') c = 'G';
                        if(c == 't') c = 'T';
                        if(c == 'N' || c == 'n'){
                            c = bases[Ncount];
                            Ncount = (Ncount + 1) % 4;
                        }
                    }

                    indicesBuffer.emplace_back(readIndex);
                    readsBuffer.emplace_back(read);

                    validFlags[readIndex] = true;
                //}

                if(indicesBuffer.size() > maxbuffersize){
                    flushBuffers(indicesBuffer, readsBuffer);
                }
            };

            std::vector<read_number> indicesBuffer;
            std::vector<Read> readsBuffer;
            indicesBuffer.reserve(maxbuffersize);
            readsBuffer.reserve(maxbuffersize);

            forEachReadInFile(fileOptions.inputfile,
                            fileOptions.format,
                            [&](auto readnum, auto& read){
                                handle_read(readnum, read, indicesBuffer, readsBuffer);
                            }
            );

            if(indicesBuffer.size() > 0){
                flushBuffers(indicesBuffer, readsBuffer);
            }

            std::cerr << "occurences of n/N:\n";
            for(const auto& p : nmap){
                std::cerr << p.first << " " << p.second << '\n';
            }

            return result;
        }

    }


    BuiltDataStructure<Minhasher> build_minhasher(const FileOptions& fileOptions,
                                			   const RuntimeOptions& runtimeOptions,
                                			   std::uint64_t nReads,
                                               const MinhashOptions& minhashOptions,
                                			   const GpuReadStorageWithFlags& readStoragewFlags){

        BuiltDataStructure<Minhasher> result;
        auto& minhasher = result.data;

        auto identity = [](auto i){return i;};

        minhasher = std::move(Minhasher{minhashOptions});

        minhasher.init(nReads);

        if(fileOptions.load_hashtables_from != ""){
            minhasher.loadFromFile(fileOptions.load_hashtables_from);
            result.builtType = BuiltType::Loaded;

            std::cout << "Loaded hash tables from " << fileOptions.load_hashtables_from << std::endl;
        }else{
            result.builtType = BuiltType::Constructed;

            int oldnumthreads = 1;
            #pragma omp parallel
            {
                #pragma omp single
                oldnumthreads = omp_get_num_threads();
            }

            omp_set_num_threads(runtimeOptions.threads);
            //std::cerr << "setReads omp_set_num_threads end " << runtimeOptions.threads << "\n";

            const auto& readStorage = readStoragewFlags.readStorage;
            const auto& validFlags = readStoragewFlags.readIsValidFlags;

            constexpr int numMapsPerBatch = 16;
            constexpr read_number parallelReads = 10000000;

            const int numBatches = SDIV(minhashOptions.maps, numMapsPerBatch);

            for(int batch = 0; batch < numBatches; batch++){
                const int firstMap = batch * numMapsPerBatch;
                const int lastMap = std::min(minhashOptions.maps, (batch+1) * numMapsPerBatch);
                const int numMaps = lastMap - firstMap;
                std::vector<int> mapIds(numMaps);
                std::iota(mapIds.begin(), mapIds.end(), firstMap);

                for(auto mapId : mapIds){
                    minhasher.initMap(mapId);
                }

                read_number numReads = readStorage.getNumberOfReads();
                int numIters = SDIV(numReads, parallelReads);

                auto sequencehandle = readStorage.makeGatherHandleSequences();
                auto lengthhandle = readStorage.makeGatherHandleLengths();
                size_t sequencepitch = getEncodedNumInts2BitHiLo(readStorage.getSequenceLengthLimit()) * sizeof(int);

                for(int iter = 0; iter < numIters; iter++){
                    read_number readIdBegin = iter * parallelReads;
                    read_number readIdEnd = std::min((iter+1) * parallelReads, numReads);

                    std::vector<read_number> indices(readIdEnd - readIdBegin);
                    std::iota(indices.begin(), indices.end(), readIdBegin);

                    std::vector<char> sequenceData(indices.size() * sequencepitch);
                    std::vector<DistributedReadStorage::Length_t> lengths(indices.size());

                    auto future1 = readStorage.gatherSequenceDataToHostBufferAsync(
                                                sequencehandle,
                                                sequenceData.data(),
                                                sequencepitch,
                                                indices.data(),
                                                indices.size(),
                                                1);
                    auto future2 = readStorage.gatherSequenceLengthsToHostBufferAsync(
                                                lengthhandle,
                                                lengths.data(),
                                                indices.data(),
                                                indices.size(),
                                                1);

                    future1.wait();
                    future2.wait();

                    #pragma omp parallel for
                    for(read_number readId = readIdBegin; readId < readIdEnd; readId++){
                        //if(validFlags[readId]){
                            read_number localId = readId - readIdBegin;
            				const char* encodedsequence = (const char*)&sequenceData[localId * sequencepitch];
            				const int sequencelength = lengths[localId];
            				std::string sequencestring = get2BitHiLoString((const unsigned int*)encodedsequence, sequencelength);
                            minhasher.insertSequence(sequencestring, readId, mapIds);
                        //}else{
                        //    ; //invalid reads are discarded
                        //}
                    }
                }

                for(auto mapId : mapIds){
                    transform_minhasher_gpu(minhasher, mapId, runtimeOptions.deviceIds);
                }
            }
            omp_set_num_threads(oldnumthreads);

        }

        //TIMERSTARTCPU(finalize_hashtables);
        //minhasher.transform();
        //TIMERSTOPCPU(finalize_hashtables);

        return result;
    }



    BuiltGpuDataStructures buildGpuDataStructures(const MinhashOptions& minhashOptions,
                                			const CorrectionOptions& correctionOptions,
                                			const RuntimeOptions& runtimeOptions,
                                			const FileOptions& fileOptions){

        BuiltGpuDataStructures result;

        auto& sequenceFileProperties = result.sequenceFileProperties;

        if(fileOptions.load_binary_reads_from == "") {
            if(fileOptions.nReads == 0 || fileOptions.maximum_sequence_length == 0) {
                std::cout << "Scanning file to get number of reads and maximum sequence length." << std::endl;
                sequenceFileProperties = getSequenceFileProperties(fileOptions.inputfile, fileOptions.format);
            }else{
                sequenceFileProperties.maxSequenceLength = fileOptions.maximum_sequence_length;
                sequenceFileProperties.minSequenceLength = 0;
                sequenceFileProperties.nReads = fileOptions.nReads;
            }
        }

        TIMERSTARTCPU(build_readstorage);
        result.builtReadStorage = buildGpuReadStorage(fileOptions,
                                                  runtimeOptions,
                                                  correctionOptions.useQualityScores,
                                                  sequenceFileProperties.nReads,
                                                  sequenceFileProperties.maxSequenceLength);
        TIMERSTOPCPU(build_readstorage);

        const auto& readStorage = result.builtReadStorage.data.readStorage;

        //if(result.builtReadStorage.builtType == BuiltType::Loaded) {
            sequenceFileProperties.nReads = readStorage.getNumberOfReads();
            sequenceFileProperties.maxSequenceLength = readStorage.getStatistics().maximumSequenceLength;
            sequenceFileProperties.minSequenceLength = readStorage.getStatistics().minimumSequenceLength;
        //}

        TIMERSTARTCPU(build_minhasher);
        result.builtMinhasher = build_minhasher(fileOptions, runtimeOptions, sequenceFileProperties.nReads, minhashOptions, result.builtReadStorage.data);
        TIMERSTOPCPU(build_minhasher);

        return result;

    }

}
}


#endif
