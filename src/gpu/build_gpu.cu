#include "hip/hip_runtime.h"
#include <build.hpp>

#include <config.hpp>

#include <options.hpp>

#include <hpc_helpers.cuh>

#include <minhasher.hpp>
#include <minhasher_transform.hpp>
#include "readstorage.hpp"
#include "sequencefileio.hpp"
#include "sequence.hpp"
#include "threadsafe_buffer.hpp"

#include <threadpool.hpp>


#include <map>
#include <stdexcept>
#include <iostream>
#include <limits>
#include <thread>
#include <future>
#include <mutex>
#include <iterator>
#include <random>
#include <omp.h>
#include <mutex>
#include <condition_variable>

#ifdef __NVCC__

namespace care{
namespace gpu{



    BuiltDataStructure<GpuReadStorageWithFlags> buildGpuReadStorage(const FileOptions& fileOptions,
                                                const RuntimeOptions& runtimeOptions,
                                                bool useQualityScores,
                                                read_number expectedNumberOfReads,
                                                int expectedMaximumReadLength){



        if(fileOptions.load_binary_reads_from != ""){
            BuiltDataStructure<GpuReadStorageWithFlags> result;
            auto& readStorage = result.data.readStorage;

            readStorage.loadFromFile(fileOptions.load_binary_reads_from, runtimeOptions.deviceIds);
            result.builtType = BuiltType::Loaded;

            if(useQualityScores && !readStorage.canUseQualityScores())
                throw std::runtime_error("Quality scores are required but not present in compressed sequence file!");
            if(!useQualityScores && readStorage.canUseQualityScores())
                std::cerr << "Warning. The loaded compressed read file contains quality scores, but program does not use them!\n";

            std::cout << "Loaded binary reads from " << fileOptions.load_binary_reads_from << std::endl;

            return result;
        }else{
            //int nThreads = std::max(1, std::min(runtimeOptions.threads, 2));
            const int nThreads = std::max(1, runtimeOptions.threads);

            constexpr std::array<char, 4> bases = {'A', 'C', 'G', 'T'};
            int Ncount = 0;
            std::map<int,int> nmap{};

            BuiltDataStructure<GpuReadStorageWithFlags> result;
            DistributedReadStorage& readstorage = result.data.readStorage;
            //auto& validFlags = result.data.readIsValidFlags;

            readstorage = std::move(DistributedReadStorage{runtimeOptions.deviceIds, expectedNumberOfReads, useQualityScores, expectedMaximumReadLength});
            //validFlags.resize(expectedNumberOfReads, false);
            result.builtType = BuiltType::Constructed;

            auto flushBuffers = [&](std::vector<read_number>* indicesBuffer, std::vector<Read>* readsBuffer){
                if(indicesBuffer->size() > 0){
                    //TIMERSTARTCPU(setReads);
                    readstorage.setReads(*indicesBuffer, *readsBuffer);
                    //TIMERSTOPCPU(setReads);

                    //std::cerr << "clearing " << indicesBuffer->size() << " indices\n";
                    //TIMERSTARTCPU(fbclearindices);
                    indicesBuffer->clear();
                    //TIMERSTOPCPU(fbclearindices);

                    //std::cerr << "clearing " << readsBuffer->size() << " reads\n";
                    //TIMERSTARTCPU(fbclearreads);
                    readsBuffer->clear();
                    //TIMERSTOPCPU(fbclearreads);
                }
            };

            auto handle_read = [&](std::uint64_t readIndex, Read& read, std::vector<read_number>* indicesBuffer, std::vector<Read>* readsBuffer){
                const int readLength = int(read.sequence.size());

                if(readIndex >= expectedNumberOfReads){
                    throw std::runtime_error("Error! Expected " + std::to_string(expectedNumberOfReads)
                                            + " reads, but file contains at least "
                                            + std::to_string(readIndex) + " reads.");
                }

                if(readLength > expectedMaximumReadLength){
                    throw std::runtime_error("Error! Expected maximum read length = "
                                            + std::to_string(expectedMaximumReadLength)
                                            + ", but read " + std::to_string(readIndex)
                                            + "has length " + std::to_string(readLength));
                }

                const int undeterminedBasesInRead = std::count_if(read.sequence.begin(), read.sequence.end(), [](char c){
                    return c == 'N' || c == 'n';
                });

                nmap[undeterminedBasesInRead]++;

                if(undeterminedBasesInRead > 0){
                    readstorage.setReadContainsN(readIndex, true);
                }

                //if(undeterminedBasesInRead > 10){
                //    validFlags[readIndex] = false;
                //}else{
                    for(auto& c : read.sequence){
                        if(c == 'a') c = 'A';
                        if(c == 'c') c = 'C';
                        if(c == 'g') c = 'G';
                        if(c == 't') c = 'T';
                        if(c == 'N' || c == 'n'){
                            c = bases[Ncount];
                            Ncount = (Ncount + 1) % 4;
                        }
                    }

                    indicesBuffer->emplace_back(readIndex);
                    readsBuffer->emplace_back(read);

                    //validFlags[readIndex] = true;
                //}

            };

            constexpr size_t maxbuffersize = 1000000;
            constexpr int numBuffers = 2;

            std::array<std::vector<read_number>, numBuffers> indicesBuffers;
            std::array<std::vector<Read>, numBuffers> readsBuffers;
            std::array<bool, numBuffers> canBeUsed;
            std::array<std::mutex, numBuffers> mutex;
            std::array<std::condition_variable, numBuffers> cv;

            for(int i = 0; i < numBuffers; i++){
                indicesBuffers[i].reserve(maxbuffersize);
                readsBuffers[i].reserve(maxbuffersize);
                canBeUsed[i] = true;
            }

            int bufferindex = 0;

            forEachReadInFile(fileOptions.inputfile,
                            fileOptions.format,
                            [&](auto readnum, auto& read){

                    if(!canBeUsed[bufferindex]){
                        std::unique_lock<std::mutex> ul(mutex[bufferindex]);
                        if(!canBeUsed[bufferindex]){
                            //std::cerr << "waiting for other buffer\n";
                            cv[bufferindex].wait(ul, [&](){ return canBeUsed[bufferindex]; });
                        }
                    }

                    auto indicesBufferPtr = &indicesBuffers[bufferindex];
                    auto readsBufferPtr = &readsBuffers[bufferindex];

                    handle_read(readnum, read, indicesBufferPtr, readsBufferPtr);

                    if(indicesBufferPtr->size() > maxbuffersize){
                        //TIMERSTARTCPU(flushBuffers);
                        //flushBuffers(indicesBufferPtr, readsBufferPtr);
                        //TIMERSTOPCPU(flushBuffers);

                        canBeUsed[bufferindex] = false;

                        //std::cerr << "launch other thread\n";
                        threadpool.enqueue([&, indicesBufferPtr, readsBufferPtr, bufferindex](){
                            //TIMERSTARTCPU(flushBufferswithcv);

                            TIMERSTARTCPU(flushBuffers);

                            flushBuffers(indicesBufferPtr, readsBufferPtr);

                            TIMERSTOPCPU(flushBuffers);

                            std::lock_guard<std::mutex> l(mutex[bufferindex]);
                            canBeUsed[bufferindex] = true;
                            cv[bufferindex].notify_one();

                            //TIMERSTOPCPU(flushBufferswithcv);

                            //std::cerr << "other thread done\n";
                        });

                        bufferindex = (bufferindex + 1) % numBuffers; //swap buffers
                        //std::cerr << "bufferindex is now " << bufferindex << "\n";



                    }
                }
            );

            auto indicesBufferPtr = &indicesBuffers[bufferindex];
            auto readsBufferPtr = &readsBuffers[bufferindex];

            if(indicesBufferPtr->size() > 0){
                //TIMERSTARTCPU(flushBuffers);
                flushBuffers(indicesBufferPtr, readsBufferPtr);
                //TIMERSTOPCPU(flushBuffers);
            }

            // std::cerr << "occurences of n/N:\n";
            // for(const auto& p : nmap){
            //     std::cerr << p.first << " " << p.second << '\n';
            // }

            return result;
        }

    }


    BuiltDataStructure<Minhasher> build_minhasher(const FileOptions& fileOptions,
                                			   const RuntimeOptions& runtimeOptions,
                                			   std::uint64_t nReads,
                                               const MinhashOptions& minhashOptions,
                                			   const GpuReadStorageWithFlags& readStoragewFlags){

        BuiltDataStructure<Minhasher> result;
        auto& minhasher = result.data;

        auto identity = [](auto i){return i;};

        minhasher = std::move(Minhasher{minhashOptions});

        minhasher.init(nReads);

        if(fileOptions.load_hashtables_from != ""){
            minhasher.loadFromFile(fileOptions.load_hashtables_from);
            result.builtType = BuiltType::Loaded;

            std::cout << "Loaded hash tables from " << fileOptions.load_hashtables_from << std::endl;
        }else{
            result.builtType = BuiltType::Constructed;

            int oldnumthreads = 1;
            #pragma omp parallel
            {
                #pragma omp single
                oldnumthreads = omp_get_num_threads();
            }

            omp_set_num_threads(runtimeOptions.threads);
            //std::cerr << "setReads omp_set_num_threads end " << runtimeOptions.threads << "\n";

            const auto& readStorage = readStoragewFlags.readStorage;
            //const auto& validFlags = readStoragewFlags.readIsValidFlags;

            constexpr read_number parallelReads = 10000000;

            const int numBatches = SDIV(minhashOptions.maps, minhasherConstructionNumMaps);

            for(int batch = 0; batch < numBatches; batch++){
                const int firstMap = batch * minhasherConstructionNumMaps;
                const int lastMap = std::min(minhashOptions.maps, (batch+1) * minhasherConstructionNumMaps);
                const int numMaps = lastMap - firstMap;
                std::vector<int> mapIds(numMaps);
                std::iota(mapIds.begin(), mapIds.end(), firstMap);

                for(auto mapId : mapIds){
                    minhasher.initMap(mapId);
                }

                read_number numReads = readStorage.getNumberOfReads();
                int numIters = SDIV(numReads, parallelReads);

                auto sequencehandle = readStorage.makeGatherHandleSequences();
                auto lengthhandle = readStorage.makeGatherHandleLengths();
                size_t sequencepitch = getEncodedNumInts2BitHiLo(readStorage.getSequenceLengthLimit()) * sizeof(int);

                //TIMERSTARTCPU(iter);
                for(int iter = 0; iter < numIters; iter++){
                    read_number readIdBegin = iter * parallelReads;
                    read_number readIdEnd = std::min((iter+1) * parallelReads, numReads);

                    std::vector<read_number> indices(readIdEnd - readIdBegin);
                    std::iota(indices.begin(), indices.end(), readIdBegin);

                    std::vector<char> sequenceData(indices.size() * sequencepitch);
                    std::vector<DistributedReadStorage::Length_t> lengths(indices.size());

                    //TIMERSTARTCPU(gather);

                    auto future1 = readStorage.gatherSequenceDataToHostBufferAsync(
                                                sequencehandle,
                                                sequenceData.data(),
                                                sequencepitch,
                                                indices.data(),
                                                indices.size(),
                                                1);
                    auto future2 = readStorage.gatherSequenceLengthsToHostBufferAsync(
                                                lengthhandle,
                                                lengths.data(),
                                                indices.data(),
                                                indices.size(),
                                                1);

                    future1.wait();
                    future2.wait();

                    //TIMERSTOPCPU(gather);

                    //TIMERSTARTCPU(insert);

                    #pragma omp parallel for
                    for(read_number readId = readIdBegin; readId < readIdEnd; readId++){
                        //if(validFlags[readId]){
                            read_number localId = readId - readIdBegin;
            				const char* encodedsequence = (const char*)&sequenceData[localId * sequencepitch];
            				const int sequencelength = lengths[localId];
            				std::string sequencestring = get2BitHiLoString((const unsigned int*)encodedsequence, sequencelength);
                            minhasher.insertSequence(sequencestring, readId, mapIds);
                        //}else{
                        //    ; //invalid reads are discarded
                        //}
                    }

                    //TIMERSTOPCPU(insert);
                }
                //TIMERSTOPCPU(iter);

                for(auto mapId : mapIds){
                    transform_minhasher_gpu(minhasher, mapId, runtimeOptions.deviceIds);
                }
            }
            omp_set_num_threads(oldnumthreads);

        }

        //TIMERSTARTCPU(finalize_hashtables);
        //minhasher.transform();
        //TIMERSTOPCPU(finalize_hashtables);

        return result;
    }



    BuiltGpuDataStructures buildGpuDataStructures(const MinhashOptions& minhashOptions,
                                			const CorrectionOptions& correctionOptions,
                                			const RuntimeOptions& runtimeOptions,
                                			const FileOptions& fileOptions){

        BuiltGpuDataStructures result;

        auto& sequenceFileProperties = result.sequenceFileProperties;

        if(fileOptions.load_binary_reads_from == "") {
            if(fileOptions.nReads == 0 || fileOptions.maximum_sequence_length == 0) {
                std::cout << "Scanning file to get number of reads and maximum sequence length." << std::endl;
                sequenceFileProperties = getSequenceFileProperties(fileOptions.inputfile, fileOptions.format);
            }else{
                sequenceFileProperties.maxSequenceLength = fileOptions.maximum_sequence_length;
                sequenceFileProperties.minSequenceLength = 0;
                sequenceFileProperties.nReads = fileOptions.nReads;
            }
        }

        TIMERSTARTCPU(build_readstorage);
        result.builtReadStorage = buildGpuReadStorage(fileOptions,
                                                  runtimeOptions,
                                                  correctionOptions.useQualityScores,
                                                  sequenceFileProperties.nReads,
                                                  sequenceFileProperties.maxSequenceLength);
        TIMERSTOPCPU(build_readstorage);

        const auto& readStorage = result.builtReadStorage.data.readStorage;

        //if(result.builtReadStorage.builtType == BuiltType::Loaded) {
            sequenceFileProperties.nReads = readStorage.getNumberOfReads();
            sequenceFileProperties.maxSequenceLength = readStorage.getStatistics().maximumSequenceLength;
            sequenceFileProperties.minSequenceLength = readStorage.getStatistics().minimumSequenceLength;
        //}

        TIMERSTARTCPU(build_minhasher);
        result.builtMinhasher = build_minhasher(fileOptions, runtimeOptions, sequenceFileProperties.nReads, minhashOptions, result.builtReadStorage.data);
        TIMERSTOPCPU(build_minhasher);

        return result;
    }

    BuiltGpuDataStructures buildAndSaveGpuDataStructures(const MinhashOptions& minhashOptions,
                                                        const CorrectionOptions& correctionOptions,
                                                        const RuntimeOptions& runtimeOptions,
                                                        const FileOptions& fileOptions){

        BuiltGpuDataStructures result;

        auto& sequenceFileProperties = result.sequenceFileProperties;

        if(fileOptions.load_binary_reads_from == "") {
            if(fileOptions.nReads == 0 || fileOptions.maximum_sequence_length == 0) {
                std::cout << "Scanning file to get number of reads and maximum sequence length." << std::endl;
                sequenceFileProperties = getSequenceFileProperties(fileOptions.inputfile, fileOptions.format);
            }else{
                sequenceFileProperties.maxSequenceLength = fileOptions.maximum_sequence_length;
                sequenceFileProperties.minSequenceLength = 0;
                sequenceFileProperties.nReads = fileOptions.nReads;
            }
        }

        TIMERSTARTCPU(build_readstorage);
        result.builtReadStorage = buildGpuReadStorage(fileOptions,
                                                  runtimeOptions,
                                                  correctionOptions.useQualityScores,
                                                  sequenceFileProperties.nReads,
                                                  sequenceFileProperties.maxSequenceLength);
        TIMERSTOPCPU(build_readstorage);

        const auto& readStorage = result.builtReadStorage.data.readStorage;

        if(fileOptions.save_binary_reads_to != "") {
            std::cout << "Saving reads to file " << fileOptions.save_binary_reads_to << std::endl;
    		readStorage.saveToFile(fileOptions.save_binary_reads_to);
    		std::cout << "Saved reads" << std::endl;
    	}

        //if(result.builtReadStorage.builtType == BuiltType::Loaded) {
            sequenceFileProperties.nReads = readStorage.getNumberOfReads();
            sequenceFileProperties.maxSequenceLength = readStorage.getStatistics().maximumSequenceLength;
            sequenceFileProperties.minSequenceLength = readStorage.getStatistics().minimumSequenceLength;
        //}

        TIMERSTARTCPU(build_minhasher);
        result.builtMinhasher = build_minhasher(fileOptions, runtimeOptions, sequenceFileProperties.nReads, minhashOptions, result.builtReadStorage.data);
        TIMERSTOPCPU(build_minhasher);

        if(fileOptions.save_hashtables_to != "") {
            std::cout << "Saving minhasher to file " << fileOptions.save_hashtables_to << std::endl;
    		result.builtMinhasher.data.saveToFile(fileOptions.save_hashtables_to);
    		std::cout << "Saved minhasher" << std::endl;
    	}

        return result;
    }

}
}


#endif
