#include <cpugpuproxy.hpp>

#include <config.hpp>

#include <hpc_helpers.cuh>

#include <algorithm>
#include <iostream>
#include <vector>

namespace care{

    std::vector<int> getUsableDeviceIds(std::vector<int> deviceIds){
        int nDevices;

        hipGetDeviceCount(&nDevices); CUERR;

        std::vector<int> invalidIds;

        for(int id : deviceIds) {
            if(id >= nDevices) {
                invalidIds.emplace_back(id);
                std::cout << "Found invalid device Id: " << id << std::endl;
            }
        }

        if(invalidIds.size() > 0) {
            std::cout << "Available GPUs on your machine:" << std::endl;
            for(int j = 0; j < nDevices; j++) {
                hipDeviceProp_t prop;
                hipGetDeviceProperties(&prop, j); CUERR;
                std::cout << "Id " << j << " : " << prop.name << std::endl;
            }

            for(int invalidid : invalidIds) {
                deviceIds.erase(std::find(deviceIds.begin(), deviceIds.end(), invalidid));
            }
        }

        return deviceIds;
    }

}
