#include "hip/hip_runtime.h"
//#define NDEBUG

#include <gpu/kernels.hpp>
#include <gpu/devicefunctionsforkernels.cuh>

//#include <gpu/bestalignment.hpp>
#include <bestalignment.hpp>
#include <gpu/utility_kernels.cuh>
#include <gpu/cubcachingallocator.cuh>

#include <msa.hpp>
#include <sequence.hpp>
#include <correctionresultprocessing.hpp>

#include <shiftedhammingdistance_common.hpp>

#include <hpc_helpers.cuh>
#include <config.hpp>

#include <cassert>


#include <hipcub/hipcub.hpp>

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

#include <thrust/binary_search.h>


namespace care{
namespace gpu{

    KernelLaunchHandle make_kernel_launch_handle(int deviceId){
        KernelLaunchHandle handle;
        handle.deviceId = deviceId;
        hipGetDeviceProperties(&handle.deviceProperties, deviceId); CUERR;
        return handle;
    }


    template<int blocksize, int tilesize>
    __global__
    void selectIndicesOfGoodCandidatesKernel(
            int* __restrict__ d_indicesOfGoodCandidates,
            int* __restrict__ d_numIndicesPerAnchor,
            int* __restrict__ d_totalNumIndices,
            const BestAlignment_t* __restrict__ d_alignmentFlags,
            const int* __restrict__ d_candidates_per_subject,
            const int* __restrict__ d_candidates_per_subject_prefixsum,
            const int* __restrict__ d_anchorIndicesOfCandidates,
            int numAnchors,
            int numCandidates){

        static_assert(blocksize % tilesize == 0);
        static_assert(tilesize == 32);

        constexpr int numTilesPerBlock = blocksize / tilesize;

        const int numTiles = (gridDim.x * blocksize) / tilesize;
        const int tileId = (threadIdx.x + blockIdx.x * blocksize) / tilesize;
        const int tileIdInBlock = threadIdx.x / tilesize;

        __shared__ int totalIndices;
        __shared__ int counts[numTilesPerBlock];

        if(threadIdx.x == 0){
            totalIndices = 0;
        }
        __syncthreads();

        auto tile = cg::tiled_partition<tilesize>(cg::this_thread_block());

        for(int anchorIndex = tileId; anchorIndex < numAnchors; anchorIndex += numTiles){

            const int offset = d_candidates_per_subject_prefixsum[anchorIndex];
            int* const indicesPtr = d_indicesOfGoodCandidates + offset;
            int* const numIndicesPtr = d_numIndicesPerAnchor + anchorIndex;
            const BestAlignment_t* const myAlignmentFlagsPtr = d_alignmentFlags + offset;

            const int numCandidatesForAnchor = d_candidates_per_subject[anchorIndex];

            if(tile.thread_rank() == 0){
                counts[tileIdInBlock] = 0;
            }
            tile.sync();

            for(int localCandidateIndex = tile.thread_rank(); 
                    localCandidateIndex < numCandidatesForAnchor; 
                    localCandidateIndex += tile.size()){
                
                const BestAlignment_t alignmentflag = myAlignmentFlagsPtr[localCandidateIndex];

                if(alignmentflag != BestAlignment_t::None){
                    cg::coalesced_group g = cg::coalesced_threads();
                    int outputPos;
                    if (g.thread_rank() == 0) {
                        outputPos = atomicAdd(&counts[tileIdInBlock], g.size());
                        atomicAdd(&totalIndices, g.size());
                    }
                    outputPos = g.thread_rank() + g.shfl(outputPos, 0);
                    indicesPtr[outputPos] = localCandidateIndex;
                }
            }

            tile.sync();
            if(tile.thread_rank() == 0){
                atomicAdd(numIndicesPtr, counts[tileIdInBlock]);
            }

        }

        __syncthreads();

        if(threadIdx.x == 0){
            atomicAdd(d_totalNumIndices, totalIndices);
        }
    }





    /*

        For each candidate, compute the alignment of anchor|candidate and anchor|revc-candidate
        Compares both alignments and keeps the better one

        Sequences are stored in dynamic sized shared memory.
        To reduce shared memory usage, the candidates belonging to the same anchor
        are processed by a set of tiles. Each tile only computes alignments for a single anchor.
        This anchor is stored in shared memory and shared by all threads within a tile
    */

    template<int tilesize>
    __global__
    void
    popcount_shifted_hamming_distance_kernel(
                const unsigned int* subjectDataHiLo,
                const unsigned int* candidateDataHiLoTransposed,
                AlignmentResultPointers d_alignmentresultpointers,
                ReadSequencesPointers d_sequencePointers,
                const int* __restrict__ candidates_per_subject_prefixsum,
                const int* __restrict__ tiles_per_subject_prefixsum,
                int n_subjects,
                int n_candidates,
                int encodedSequencePitchInInts2BitHiLo,
                int min_overlap,
                float maxErrorRate,
                float min_overlap_ratio,
                float estimatedNucleotideErrorRate){

        auto make_reverse_complement_inplace = [&](unsigned int* sequence, int sequencelength, auto indextrafo){
            reverseComplementInplace2BitHiLo((unsigned int*)sequence, sequencelength, indextrafo);
        };

        auto no_bank_conflict_index_tile = [&](int logical_index) -> int {
            return logical_index * tilesize;
        };

        auto no_bank_conflict_index = [](int logical_index) -> int {
            return logical_index * blockDim.x;
        };

        auto identity = [](auto logical_index){
            return logical_index;
        };

        auto popcount = [](auto i){return __popc(i);};

        auto hammingDistanceWithShift = [&](bool doShift, int overlapsize, int max_errors,
                                    unsigned int* shiftptr_hi, unsigned int* shiftptr_lo, auto transfunc1,
                                    int shiftptr_size,
                                    const unsigned int* otherptr_hi, const unsigned int* otherptr_lo,
                                    auto transfunc2){

            if(doShift){
                shiftBitArrayLeftBy<1>(shiftptr_hi, shiftptr_size / 2, transfunc1);
                shiftBitArrayLeftBy<1>(shiftptr_lo, shiftptr_size / 2, transfunc1);
            }

            const int score = hammingdistanceHiLo(shiftptr_hi,
                                                shiftptr_lo,
                                                otherptr_hi,
                                                otherptr_lo,
                                                overlapsize,
                                                overlapsize,
                                                max_errors,
                                                transfunc1,
                                                transfunc2,
                                                popcount);

            return score;
        };


        auto alignmentComparator = [&] (int fwd_alignment_overlap,
            int revc_alignment_overlap,
            int fwd_alignment_nops,
            int revc_alignment_nops,
            bool fwd_alignment_isvalid,
            bool revc_alignment_isvalid,
            int subjectlength,
            int querylength)->BestAlignment_t{

            return choose_best_alignment(
                fwd_alignment_overlap,
                revc_alignment_overlap,
                fwd_alignment_nops,
                revc_alignment_nops,
                fwd_alignment_isvalid,
                revc_alignment_isvalid,
                subjectlength,
                querylength,
                min_overlap_ratio,
                min_overlap,
                estimatedNucleotideErrorRate * 4.0f
            );
        };

        // sizeof(char) * (max_sequence_bytes * num_tiles   // tiles share the subject
        //                    + max_sequence_bytes * num_threads // each thread works with its own candidate
        //                    + max_sequence_bytes * num_threads) // each thread needs memory to shift a sequence
        extern __shared__ unsigned int sharedmemory[];

        //set up shared memory pointers

        const int tiles = (blockDim.x * gridDim.x) / tilesize;
        const int globalTileId = (blockDim.x * blockIdx.x + threadIdx.x) / tilesize;
        const int localTileId = (threadIdx.x) / tilesize;
        const int tilesPerBlock = blockDim.x / tilesize;
        const int laneInTile = threadIdx.x % tilesize;
        const int requiredTiles = tiles_per_subject_prefixsum[n_subjects];

        unsigned int* const subjectBackupsBegin = sharedmemory; // per tile shared memory to store subject
        unsigned int* const queryBackupsBegin = subjectBackupsBegin + encodedSequencePitchInInts2BitHiLo * tilesPerBlock; // per thread shared memory to store query
        unsigned int* const mySequencesBegin = queryBackupsBegin + encodedSequencePitchInInts2BitHiLo * blockDim.x; // per thread shared memory to store shifted sequence

        unsigned int* const subjectBackup = subjectBackupsBegin + encodedSequencePitchInInts2BitHiLo * localTileId; // accesed via identity
        unsigned int* const queryBackup = queryBackupsBegin + threadIdx.x; // accesed via no_bank_conflict_index
        unsigned int* const mySequence = mySequencesBegin + threadIdx.x; // accesed via no_bank_conflict_index

        for(int logicalTileId = globalTileId; logicalTileId < requiredTiles ; logicalTileId += tiles){

            const int subjectIndex = thrust::distance(tiles_per_subject_prefixsum,
                                                    thrust::lower_bound(
                                                        thrust::seq,
                                                        tiles_per_subject_prefixsum,
                                                        tiles_per_subject_prefixsum + n_subjects + 1,
                                                        logicalTileId + 1))-1;

            const int candidatesBeforeThisSubject = candidates_per_subject_prefixsum[subjectIndex];
            const int maxCandidateIndex_excl = candidates_per_subject_prefixsum[subjectIndex+1];
            //const int tilesForThisSubject = tiles_per_subject_prefixsum[subjectIndex + 1] - tiles_per_subject_prefixsum[subjectIndex];
            const int tileForThisSubject = logicalTileId - tiles_per_subject_prefixsum[subjectIndex];
            const int candidateIndex = candidatesBeforeThisSubject + tileForThisSubject * tilesize + laneInTile;

            const int subjectbases = d_sequencePointers.subjectSequencesLength[subjectIndex];
            const int subjectints = getEncodedNumInts2BitHiLo(subjectbases);
            const unsigned int* subjectptr = subjectDataHiLo + std::size_t(subjectIndex) * encodedSequencePitchInInts2BitHiLo;

            //save subject in shared memory (in parallel, per tile)
            for(int lane = laneInTile; lane < encodedSequencePitchInInts2BitHiLo; lane += tilesize) {
                subjectBackup[identity(lane)] = subjectptr[lane];
                //transposed
                //subjectBackup[identity(lane)] = ((unsigned int*)(subjectptr))[lane * n_subjects];
            }

            cg::tiled_partition<tilesize>(cg::this_thread_block()).sync();


            if(candidateIndex < maxCandidateIndex_excl){

                const int querybases = d_sequencePointers.candidateSequencesLength[candidateIndex];
                const int queryints = getEncodedNumInts2BitHiLo(querybases);
                const int totalbases = subjectbases + querybases;
                const int minoverlap = max(min_overlap, int(float(subjectbases) * min_overlap_ratio));

                const unsigned int* candidateptr = candidateDataHiLoTransposed + std::size_t(candidateIndex);

                //save query in shared memory
                for(int i = 0; i < encodedSequencePitchInInts2BitHiLo; i += 1) {
                    //queryBackup[no_bank_conflict_index(i)] = ((unsigned int*)(candidateptr))[i];
                    //transposed
                    queryBackup[no_bank_conflict_index(i)] = candidateptr[i * n_candidates];
                }

                const unsigned int* const subjectBackup_hi = subjectBackup;
                const unsigned int* const subjectBackup_lo = subjectBackup + identity(subjectints/2);
                const unsigned int* const queryBackup_hi = queryBackup;
                const unsigned int* const queryBackup_lo = queryBackup + no_bank_conflict_index(queryints/2);

                int bestScore[2];
                int bestShift[2];
                int overlapsize[2];
                int opnr[2];

                #pragma unroll
                for(int orientation = 0; orientation < 2; orientation++){
                    const bool isReverseComplement = orientation == 1;

                    if(isReverseComplement) {
                        make_reverse_complement_inplace(queryBackup, querybases, no_bank_conflict_index);
                    }

                    //begin SHD algorithm

                    bestScore[orientation] = totalbases;     // score is number of mismatches
                    bestShift[orientation] = -querybases;    // shift of query relative to subject. shift < 0 if query begins before subject

                    auto handle_shift = [&](int shift, int overlapsize,
                                                unsigned int* shiftptr_hi, unsigned int* shiftptr_lo, auto transfunc1,
                                                int shiftptr_size,
                                                const unsigned int* otherptr_hi, const unsigned int* otherptr_lo,
                                                auto transfunc2){

                        //const int max_errors = int(float(overlapsize) * maxErrorRate);
                        const int max_errors_excl = min(int(float(overlapsize) * maxErrorRate),
                        bestScore[orientation] - totalbases + 2*overlapsize);

                        if(max_errors_excl > 0){

                            int score = hammingDistanceWithShift(shift != 0, overlapsize, max_errors_excl,
                                                shiftptr_hi,shiftptr_lo, transfunc1,
                                                shiftptr_size,
                                                otherptr_hi, otherptr_lo, transfunc2);

                            

                            // printf("%d, %d %d %d --- ", queryIndex, shift, overlapsize, score);

                            // printf("%d %d %d %d | %d %d %d %d --- ", 
                            //     shiftptr_hi[transfunc1(0)], shiftptr_hi[transfunc1(1)], shiftptr_hi[transfunc1(2)], shiftptr_hi[transfunc1(3)],
                            //     shiftptr_lo[transfunc1(0)], shiftptr_lo[transfunc1(1)], shiftptr_lo[transfunc1(2)], shiftptr_lo[transfunc1(3)]);

                            // printf("%d %d %d %d | %d %d %d %d\n", 
                            //     otherptr_hi[transfunc2(0)], otherptr_hi[transfunc2(1)], otherptr_hi[transfunc2(2)], otherptr_hi[transfunc2(3)],
                            //     otherptr_lo[transfunc2(0)], otherptr_lo[transfunc2(1)], otherptr_lo[transfunc2(2)], otherptr_lo[transfunc2(3)]);

                            score = (score < max_errors_excl ?
                                    score + totalbases - 2*overlapsize // non-overlapping regions count as mismatches
                                    : std::numeric_limits<int>::max()); // too many errors, discard

                            if(score < bestScore[orientation]){
                                bestScore[orientation] = score;
                                bestShift[orientation] = shift;
                            }

                            return true;
                        }else{
                            //printf("%d, %d %d %d max_errors_excl\n", queryIndex, shift, overlapsize, max_errors_excl);
                            return false;
                        }
                    };

                    //initialize threadlocal smem array with subject
                    for(int i = 0; i < encodedSequencePitchInInts2BitHiLo; i += 1) {
                        mySequence[no_bank_conflict_index(i)] = subjectBackup[identity(i)];
                    }

                    unsigned int* mySequence_hi = mySequence;
                    unsigned int* mySequence_lo = mySequence + no_bank_conflict_index(subjectints / 2);

                    for(int shift = 0; shift < subjectbases - minoverlap + 1; shift += 1) {
                        const int overlapsize = min(subjectbases - shift, querybases);

                        bool b = handle_shift(shift, overlapsize,
                                        mySequence_hi, mySequence_lo, no_bank_conflict_index,
                                        subjectints,
                                        queryBackup_hi, queryBackup_lo, no_bank_conflict_index);
                        if(!b){
                            break;
                        }
                    }

                    //initialize threadlocal smem array with query
                    for(int i = 0; i < encodedSequencePitchInInts2BitHiLo; i += 1) {
                        mySequence[no_bank_conflict_index(i)] = queryBackup[no_bank_conflict_index(i)];
                    }

                    mySequence_hi = mySequence;
                    mySequence_lo = mySequence + no_bank_conflict_index(queryints / 2);

                    for(int shift = -1; shift >= -querybases + minoverlap; shift -= 1) {
                        const int overlapsize = min(subjectbases, querybases + shift);

                        bool b = handle_shift(shift, overlapsize,
                                        mySequence_hi, mySequence_lo, no_bank_conflict_index,
                                        queryints,
                                        subjectBackup_hi, subjectBackup_lo, identity);
                        if(!b){
                            break;
                        }
                    }

                    const int queryoverlapbegin_incl = max(-bestShift[orientation], 0);
                    const int queryoverlapend_excl = min(querybases, subjectbases - bestShift[orientation]);
                    overlapsize[orientation] = queryoverlapend_excl - queryoverlapbegin_incl;
                    opnr[orientation] = bestScore[orientation] - totalbases + 2*overlapsize[orientation];
                }

                const BestAlignment_t flag = alignmentComparator(
                    overlapsize[0],
                    overlapsize[1],
                    opnr[0],
                    opnr[1],
                    bestShift[0] != -querybases,
                    bestShift[1] != -querybases,
                    subjectbases,
                    querybases
                );

                //int* const d_alignment_scores = d_alignmentresultpointers.scores;
                int* const d_alignment_overlaps = d_alignmentresultpointers.overlaps;
                int* const d_alignment_shifts = d_alignmentresultpointers.shifts;
                int* const d_alignment_nOps = d_alignmentresultpointers.nOps;
                bool* const d_alignment_isValid = d_alignmentresultpointers.isValid;
                BestAlignment_t* const d_alignment_best_alignment_flags = d_alignmentresultpointers.bestAlignmentFlags;

                d_alignment_best_alignment_flags[candidateIndex] = flag;
                //scores are unused in the program
                //d_alignment_scores[candidateIndex] = flag == BestAlignment_t::Forward ? bestScore[0] : bestScore[1];
                d_alignment_overlaps[candidateIndex] = flag == BestAlignment_t::Forward ? overlapsize[0] : overlapsize[1];
                d_alignment_shifts[candidateIndex] = flag == BestAlignment_t::Forward ? bestShift[0] : bestShift[1];
                d_alignment_nOps[candidateIndex] = flag == BestAlignment_t::Forward ? opnr[0] : opnr[1];
                d_alignment_isValid[candidateIndex] = flag == BestAlignment_t::Forward ? bestShift[0] != -querybases : bestShift[1] != -querybases;
            }
        }
    }



    /*
        Uses 1 thread per candidate to compute the alignment of anchor|candidate and anchor|revc-candidate
        Compares both alignments and keeps the better one

        Sequences are stored in registers
    */

    template<int blocksize, int maxValidIntsPerSequence>
    __global__
    void
    popcount_shifted_hamming_distance_ctpitch_kernel(
                const unsigned int* __restrict__ subjectDataHiLoTransposed,
                const unsigned int* __restrict__ candidateDataHiLoTransposed,
                const int* __restrict__ subjectSequencesLength,
                const int* __restrict__ candidateSequencesLength,
                BestAlignment_t* __restrict__ bestAlignmentFlags,
                int* __restrict__ alignment_scores,
                int* __restrict__ alignment_overlaps,
                int* __restrict__ alignment_shifts,
                int* __restrict__ alignment_nOps,
                bool* __restrict__ alignment_isValid,
                const int* __restrict__ d_anchorIndicesOfCandidates,
                int n_subjects,
                int n_candidates,
                size_t encodedSequencePitchInInts2BitHiLo,
                int min_overlap,
                float maxErrorRate,
                float min_overlap_ratio,
                float estimatedNucleotideErrorRate){

        static_assert(maxValidIntsPerSequence % 2 == 0, ""); //2bithilo has even number of ints

        auto identity = [](auto logical_index){
            return logical_index;
        };

        auto popcount = [](auto i){return __popc(i);};

        auto hammingdistanceHiLoReg = [&](
                            const auto& lhi,
                            const auto& llo,
                            const auto& rhi,
                            const auto& rlo,
                            int lhi_bitcount,
                            int rhi_bitcount,
                            int max_errors){

            constexpr int N = maxValidIntsPerSequence / 2;

            const int overlap_bitcount = std::min(lhi_bitcount, rhi_bitcount);

            if(overlap_bitcount == 0)
                return max_errors+1;

            const int partitions = SDIV(overlap_bitcount, (8 * sizeof(unsigned int)));
            const int remaining_bitcount = partitions * sizeof(unsigned int) * 8 - overlap_bitcount;

            int result = 0;

            #pragma unroll 
            for(int i = 0; i < N - 1; i++){
                if(i < partitions - 1 && result < max_errors){
                    const unsigned int hixor = lhi[i] ^ rhi[i];
                    const unsigned int loxor = llo[i] ^ rlo[i];
                    const unsigned int bits = hixor | loxor;
                    result += popcount(bits);
                }
            }

            if(result >= max_errors)
                return result;

            // i == partitions - 1

            #pragma unroll 
            for(int i = N-1; i >= 0; i--){
                if(partitions - 1 == i){
                    const unsigned int mask = remaining_bitcount == 0 ? 0xFFFFFFFF : 0xFFFFFFFF << (remaining_bitcount);
                    const unsigned int hixor = lhi[i] ^ rhi[i];
                    const unsigned int loxor = llo[i] ^ rlo[i];
                    const unsigned int bits = hixor | loxor;
                    result += popcount(bits & mask);
                }
            }

            return result;
        };

        auto shiftBitArrayLeftBy1 = [](auto& uintarray){
            constexpr int shift = 1;
            static_assert(shift < 32, "");

            constexpr int N = maxValidIntsPerSequence / 2;    
            #pragma unroll
            for(int i = 0; i < N - 1; i += 1) {
                const unsigned int a = uintarray[i];
                const unsigned int b = uintarray[i+1];
    
                uintarray[i] = (a << shift) | (b >> (8 * sizeof(unsigned int) - shift));
            }
    
            uintarray[N-1] <<= shift;
        };

        auto hammingDistanceWithShift = [&](bool doShift, int overlapsize, int max_errors,
                                    auto& shiftptr_hi, auto& shiftptr_lo,
                                    const auto& otherptr_hi, const auto& otherptr_lo
                                    ){

            if(doShift){
                shiftBitArrayLeftBy1(shiftptr_hi);
                shiftBitArrayLeftBy1(shiftptr_lo);
            }

            const int score = hammingdistanceHiLoReg(shiftptr_hi,
                                                shiftptr_lo,
                                                otherptr_hi,
                                                otherptr_lo,
                                                overlapsize,
                                                overlapsize,
                                                max_errors);

            return score;
        };

        auto alignmentComparator = [&] (int fwd_alignment_overlap,
            int revc_alignment_overlap,
            int fwd_alignment_nops,
            int revc_alignment_nops,
            bool fwd_alignment_isvalid,
            bool revc_alignment_isvalid,
            int subjectlength,
            int querylength)->BestAlignment_t{

            return choose_best_alignment(
                fwd_alignment_overlap,
                revc_alignment_overlap,
                fwd_alignment_nops,
                revc_alignment_nops,
                fwd_alignment_isvalid,
                revc_alignment_isvalid,
                subjectlength,
                querylength,
                min_overlap_ratio,
                min_overlap,
                estimatedNucleotideErrorRate * 4.0f
            );
        };


        unsigned int subjectBackupHi[maxValidIntsPerSequence / 2];
        unsigned int subjectBackupLo[maxValidIntsPerSequence / 2];
        unsigned int queryBackupHi[maxValidIntsPerSequence / 2];
        unsigned int queryBackupLo[maxValidIntsPerSequence / 2];
        unsigned int mySequenceHi[maxValidIntsPerSequence / 2];
        unsigned int mySequenceLo[maxValidIntsPerSequence / 2];

        auto reverseComplementQuery = [&](int querylength, int validInts){
            auto reverse_complement_int = [](auto n) {
                n = ((n >> 1) & 0x55555555) | ((n << 1) & 0xaaaaaaaa);
                n = ((n >> 2) & 0x33333333) | ((n << 2) & 0xcccccccc);
                n = ((n >> 4) & 0x0f0f0f0f) | ((n << 4) & 0xf0f0f0f0);
                n = ((n >> 8) & 0x00ff00ff) | ((n << 8) & 0xff00ff00);
                n = ((n >> 16) & 0x0000ffff) | ((n << 16) & 0xffff0000);
                return ~n;
            };

            constexpr int N = maxValidIntsPerSequence / 2;

            #pragma unroll
            for(int i = 0; i < N/2; ++i){
                const unsigned int hifront = reverse_complement_int(queryBackupHi[i]);
                const unsigned int hiback = reverse_complement_int(queryBackupHi[N - 1 - i]);
                queryBackupHi[i] = hiback;
                queryBackupHi[N - 1 - i] = hifront;
    
                const unsigned int lofront = reverse_complement_int(queryBackupLo[i]);
                const unsigned int loback = reverse_complement_int(queryBackupLo[N - 1 - i]);
                queryBackupLo[i] = loback;
                queryBackupLo[N - 1 - i] = lofront;
            }

            if(N % 2 == 1){
                constexpr int middleindex = N/2;
                queryBackupHi[middleindex] = reverse_complement_int(queryBackupHi[middleindex]);
                queryBackupLo[middleindex] = reverse_complement_int(queryBackupLo[middleindex]);
            }

            //fix unused data

            const int unusedInts = N - getEncodedNumInts2BitHiLo(querylength) / 2;
            if(unusedInts > 0){
                for(int iter = 0; iter < unusedInts; iter++){
                    #pragma unroll
                    for(int i = 0; i < N-1; ++i){
                        queryBackupHi[i] = queryBackupHi[i+1];
                        queryBackupLo[i] = queryBackupLo[i+1];
                    }
                }
            }

            const int unusedBitsInt = SDIV(querylength, 8 * sizeof(unsigned int)) * 8 * sizeof(unsigned int) - querylength;

            if(unusedBitsInt != 0){
                #pragma unroll
                for(int i = 0; i < N - 1; ++i){
                    queryBackupHi[i] = (queryBackupHi[i] << unusedBitsInt) | (queryBackupHi[i+1] >> (8 * sizeof(unsigned int) - unusedBitsInt));
                    queryBackupLo[i] = (queryBackupLo[i] << unusedBitsInt) | (queryBackupLo[i+1] >> (8 * sizeof(unsigned int) - unusedBitsInt));
                }
    
                queryBackupHi[N-1] <<= unusedBitsInt;
                queryBackupLo[N-1] <<= unusedBitsInt;
            }
        };

        for(int candidateIndex = threadIdx.x + blocksize * blockIdx.x; candidateIndex < n_candidates; candidateIndex += blocksize * gridDim.x){

            const int subjectIndex = d_anchorIndicesOfCandidates[candidateIndex];          
            const int subjectbases = subjectSequencesLength[subjectIndex];
            const int querybases = candidateSequencesLength[candidateIndex];

            const unsigned int* subjectptr = subjectDataHiLoTransposed + std::size_t(subjectIndex);

            #pragma unroll 
            for(int i = 0; i < maxValidIntsPerSequence / 2; i++){
                subjectBackupHi[i] = subjectptr[(i) * n_subjects];
                subjectBackupLo[i] = subjectptr[(i + maxValidIntsPerSequence / 2) * n_subjects];
            }

            const unsigned int* candidateptr = candidateDataHiLoTransposed + std::size_t(candidateIndex);

            //save query in reg

            #pragma unroll 
            for(int i = 0; i < maxValidIntsPerSequence / 2; i++){
                queryBackupHi[i] = candidateptr[i * n_candidates];
                queryBackupLo[i] = candidateptr[(i + maxValidIntsPerSequence / 2) * n_candidates];
            }

            //begin SHD algorithm

            const int subjectints = getEncodedNumInts2BitHiLo(subjectbases);
            const int queryints = getEncodedNumInts2BitHiLo(querybases);
            const int totalbases = subjectbases + querybases;
            const int minoverlap = max(min_overlap, int(float(subjectbases) * min_overlap_ratio));

            int bestScore[2];
            int bestShift[2];
            int overlapsize[2];
            int opnr[2];

            #pragma unroll
            for(int orientation = 0; orientation < 2; orientation++){
                const bool isReverseComplement = orientation == 1;

                if(isReverseComplement){
                    reverseComplementQuery(querybases, queryints);
                }

                bestScore[orientation] = totalbases;     // score is number of mismatches
                bestShift[orientation] = -querybases;    // shift of query relative to subject. shift < 0 if query begins before subject

                auto handle_shift = [&](int shift, int overlapsize,
                                        auto& shiftptr_hi, auto& shiftptr_lo,
                                        const auto& otherptr_hi, const auto& otherptr_lo){

                    //const int max_errors = int(float(overlapsize) * maxErrorRate);
                    const int max_errors_excl = min(int(float(overlapsize) * maxErrorRate),
                                                    bestScore[orientation] - totalbases + 2*overlapsize);

                    if(max_errors_excl > 0){

                        int score = hammingDistanceWithShift(shift != 0, overlapsize, max_errors_excl,
                                            shiftptr_hi, shiftptr_lo,
                                            otherptr_hi, otherptr_lo);

                        
                        // printf("%d, %d %d %d --- ", queryIndex, shift, overlapsize, score);

                        // printf("%d %d %d %d | %d %d %d %d --- ", 
                        //     shiftptr_hi[0], shiftptr_hi[1], shiftptr_hi[2], shiftptr_hi[3],
                        //     shiftptr_lo[0], shiftptr_lo[1], shiftptr_lo[2], shiftptr_lo[3]);

                        // printf("%d %d %d %d | %d %d %d %d\n", 
                        //     otherptr_hi[0], otherptr_hi[1], otherptr_hi[2], otherptr_hi[3],
                        //     otherptr_lo[0], otherptr_lo[1], otherptr_lo[2], otherptr_lo[3]);

                        score = (score < max_errors_excl ?
                                score + totalbases - 2*overlapsize // non-overlapping regions count as mismatches
                                : std::numeric_limits<int>::max()); // too many errors, discard

                        if(score < bestScore[orientation]){
                            bestScore[orientation] = score;
                            bestShift[orientation] = shift;
                        }

                        return true;
                    }else{
                        //printf("%d, %d %d %d max_errors_excl\n", queryIndex, shift, overlapsize, max_errors_excl);

                        return false;
                    }
                };

                #pragma unroll 
                for(int i = 0; i < maxValidIntsPerSequence / 2; i++){
                    mySequenceHi[i] = subjectBackupHi[i];
                    mySequenceLo[i] = subjectBackupLo[i];
                }

                for(int shift = 0; shift < subjectbases - minoverlap + 1; shift += 1) {
                    const int overlapsize = min(subjectbases - shift, querybases);

                    bool b = handle_shift(
                        shift, overlapsize,
                        mySequenceHi, mySequenceLo,
                        queryBackupHi, queryBackupLo
                    );
                    if(!b){
                        break;
                    }
                }

                //initialize threadlocal smem array with query
                #pragma unroll 
                for(int i = 0; i < maxValidIntsPerSequence / 2; i++){
                    mySequenceHi[i] = queryBackupHi[i];
                    mySequenceLo[i] = queryBackupLo[i];
                }

                for(int shift = -1; shift >= -querybases + minoverlap; shift -= 1) {
                    const int overlapsize = min(subjectbases, querybases + shift);

                    bool b = handle_shift(
                        shift, overlapsize,
                        mySequenceHi, mySequenceLo,
                        subjectBackupHi, subjectBackupLo
                    );
                    if(!b){
                        break;
                    }
                }

                const int queryoverlapbegin_incl = max(-bestShift[orientation], 0);
                const int queryoverlapend_excl = min(querybases, subjectbases - bestShift[orientation]);
                overlapsize[orientation] = queryoverlapend_excl - queryoverlapbegin_incl;
                opnr[orientation] = bestScore[orientation] - totalbases + 2*overlapsize[orientation];
            }

            const BestAlignment_t flag = alignmentComparator(
                overlapsize[0],
                overlapsize[1],
                opnr[0],
                opnr[1],
                bestShift[0] != -querybases,
                bestShift[1] != -querybases,
                subjectbases,
                querybases
            );

            bestAlignmentFlags[candidateIndex] = flag;
            //scores are unused in the program
            //d_alignment_scores[candidateIndex] = flag == BestAlignment_t::Forward ? bestScore[0] : bestScore[1];
            alignment_overlaps[candidateIndex] = flag == BestAlignment_t::Forward ? overlapsize[0] : overlapsize[1];
            alignment_shifts[candidateIndex] = flag == BestAlignment_t::Forward ? bestShift[0] : bestShift[1];
            alignment_nOps[candidateIndex] = flag == BestAlignment_t::Forward ? opnr[0] : opnr[1];
            alignment_isValid[candidateIndex] = flag == BestAlignment_t::Forward ? bestShift[0] != -querybases : bestShift[1] != -querybases;
        }
    }














    __global__
    void cuda_find_best_alignment_kernel_exp(
                AlignmentResultPointers d_alignmentresultpointers,
                ReadSequencesPointers d_sequencePointers,
                const int* __restrict__ d_candidates_per_subject_prefixsum,
                int n_subjects,
                int n_queries,
                float min_overlap_ratio,
                int min_overlap,
                float estimatedErrorrate,
                read_number debugsubjectreadid){

        auto getSubjectLength = [&] (int subjectIndex){
            const int length = d_sequencePointers.subjectSequencesLength[subjectIndex];
            return length;
        };

        auto getCandidateLength = [&] (int resultIndex){
            const int length = d_sequencePointers.candidateSequencesLength[resultIndex];
            return length;
        };

        auto comp = [&] (int fwd_alignment_overlap,
                        int revc_alignment_overlap,
                        int fwd_alignment_nops,
                        int revc_alignment_nops,
                        bool fwd_alignment_isvalid,
                        bool revc_alignment_isvalid,
                        int subjectlength,
                        int querylength)->BestAlignment_t{

            return choose_best_alignment(fwd_alignment_overlap,
                        revc_alignment_overlap,
                        fwd_alignment_nops,
                        revc_alignment_nops,
                        fwd_alignment_isvalid,
                        revc_alignment_isvalid,
                        subjectlength,
                        querylength,
                        min_overlap_ratio,
                        min_overlap,
                        estimatedErrorrate * 4.0f);
        };

        int* const d_alignment_scores = d_alignmentresultpointers.scores;
        int* const d_alignment_overlaps = d_alignmentresultpointers.overlaps;
        int* const d_alignment_shifts = d_alignmentresultpointers.shifts;
        int* const d_alignment_nOps = d_alignmentresultpointers.nOps;
        bool* const d_alignment_isValid = d_alignmentresultpointers.isValid;
        BestAlignment_t* const d_alignment_best_alignment_flags = d_alignmentresultpointers.bestAlignmentFlags;

        for(unsigned resultIndex = threadIdx.x + blockDim.x * blockIdx.x; resultIndex < n_queries; resultIndex += gridDim.x * blockDim.x) {
            const unsigned fwdIndex = resultIndex;
            const unsigned revcIndex = resultIndex + n_queries;

            const int fwd_alignment_score = d_alignment_scores[fwdIndex];
            const int fwd_alignment_overlap = d_alignment_overlaps[fwdIndex];
            const int fwd_alignment_shift = d_alignment_shifts[fwdIndex];
            const int fwd_alignment_nops = d_alignment_nOps[fwdIndex];
            const bool fwd_alignment_isvalid = d_alignment_isValid[fwdIndex];

            const int revc_alignment_score = d_alignment_scores[revcIndex];
            const int revc_alignment_overlap = d_alignment_overlaps[revcIndex];
            const int revc_alignment_shift = d_alignment_shifts[revcIndex];
            const int revc_alignment_nops = d_alignment_nOps[revcIndex];
            const bool revc_alignment_isvalid = d_alignment_isValid[revcIndex];

            //assert(fwd_alignment_isvalid || fwd_alignment_shift == -101);
            //assert(revc_alignment_isvalid || revc_alignment_shift == -101);

            //const int querylength = d_candidate_sequences_lengths[resultIndex];
            const int querylength = getCandidateLength(resultIndex);

            //find subjectindex
            /*int subjectIndex = 0;
            for(; subjectIndex < n_subjects; subjectIndex++) {
                if(resultIndex < d_candidates_per_subject_prefixsum[subjectIndex+1])
                    break;
            }*/

            const int subjectIndex = thrust::distance(d_candidates_per_subject_prefixsum,
                                                    thrust::lower_bound(
                                                        thrust::seq,
                                                        d_candidates_per_subject_prefixsum,
                                                        d_candidates_per_subject_prefixsum + n_subjects + 1,
                                                        resultIndex + 1))-1;

            //const int subjectlength = d_subject_sequences_lengths[subjectIndex];
            const int subjectlength = getSubjectLength(subjectIndex);

            const BestAlignment_t flag = comp(fwd_alignment_overlap,
                        revc_alignment_overlap,
                        fwd_alignment_nops,
                        revc_alignment_nops,
                        fwd_alignment_isvalid,
                        revc_alignment_isvalid,
                        subjectlength,
                        querylength);

            d_alignment_best_alignment_flags[resultIndex] = flag;

            d_alignment_scores[resultIndex] = flag == BestAlignment_t::Forward ? fwd_alignment_score : revc_alignment_score;
            d_alignment_overlaps[resultIndex] = flag == BestAlignment_t::Forward ? fwd_alignment_overlap : revc_alignment_overlap;
            d_alignment_shifts[resultIndex] = flag == BestAlignment_t::Forward ? fwd_alignment_shift : revc_alignment_shift;
            d_alignment_nOps[resultIndex] = flag == BestAlignment_t::Forward ? fwd_alignment_nops : revc_alignment_nops;
            d_alignment_isValid[resultIndex] = flag == BestAlignment_t::Forward ? fwd_alignment_isvalid : revc_alignment_isvalid;
        }
    }


    template<int BLOCKSIZE>
    __global__
    void cuda_filter_alignments_by_mismatchratio_kernel(
                AlignmentResultPointers d_alignmentresultpointers,
                const int* __restrict__ d_candidates_per_subject_prefixsum,
                int n_subjects,
                int n_candidates,
                float mismatchratioBaseFactor,
                float goodAlignmentsCountThreshold){

        using BlockReduceInt = hipcub::BlockReduce<int, BLOCKSIZE>;

        __shared__ union {
            typename BlockReduceInt::TempStorage intreduce;
            int broadcast[3];
        } temp_storage;


        for(int subjectindex = blockIdx.x; subjectindex < n_subjects; subjectindex += gridDim.x) {

            const int candidatesForSubject = d_candidates_per_subject_prefixsum[subjectindex+1]
                                            - d_candidates_per_subject_prefixsum[subjectindex];

            const int firstIndex = d_candidates_per_subject_prefixsum[subjectindex];

            //printf("subjectindex %d\n", subjectindex);

            int counts[3]{0,0,0};

            //if(threadIdx.x == 0){
            //    printf("my_n_indices %d\n", my_n_indices);
            //}

            for(int index = threadIdx.x; index < candidatesForSubject; index += blockDim.x) {

                const int candidate_index = firstIndex + index;
                if(d_alignmentresultpointers.bestAlignmentFlags[candidate_index] != BestAlignment_t::None) {

                    const int alignment_overlap = d_alignmentresultpointers.overlaps[candidate_index];
                    const int alignment_nops = d_alignmentresultpointers.nOps[candidate_index];

                    const float mismatchratio = float(alignment_nops) / alignment_overlap;

                    if(mismatchratio >= 4 * mismatchratioBaseFactor) {
                        d_alignmentresultpointers.bestAlignmentFlags[candidate_index] = BestAlignment_t::None;
                    }else{

                            #pragma unroll
                        for(int i = 2; i <= 4; i++) {
                            counts[i-2] += (mismatchratio < i * mismatchratioBaseFactor);
                        }
                    }

                }
            }

            //accumulate counts over block
                #pragma unroll
            for(int i = 0; i < 3; i++) {
                counts[i] = BlockReduceInt(temp_storage.intreduce).Sum(counts[i]);
                __syncthreads();
            }

            //broadcast accumulated counts to block
            if(threadIdx.x == 0) {
                #pragma unroll
                for(int i = 0; i < 3; i++) {
                    temp_storage.broadcast[i] = counts[i];
                    //printf("count[%d] = %d\n", i, counts[i]);
                }
                //printf("mismatchratioBaseFactor %f, goodAlignmentsCountThreshold %f\n", mismatchratioBaseFactor, goodAlignmentsCountThreshold);
            }

            __syncthreads();

            #pragma unroll
            for(int i = 0; i < 3; i++) {
                counts[i] = temp_storage.broadcast[i];
            }

            float mismatchratioThreshold = 0;
            if (counts[0] >= goodAlignmentsCountThreshold) {
                mismatchratioThreshold = 2 * mismatchratioBaseFactor;
            } else if (counts[1] >= goodAlignmentsCountThreshold) {
                mismatchratioThreshold = 3 * mismatchratioBaseFactor;
            } else if (counts[2] >= goodAlignmentsCountThreshold) {
                mismatchratioThreshold = 4 * mismatchratioBaseFactor;
            } else {
                mismatchratioThreshold = -1.0f;                         //this will invalidate all alignments for subject
                //mismatchratioThreshold = 4 * mismatchratioBaseFactor; //use alignments from every bin
                //mismatchratioThreshold = 1.1f;
            }

            // Invalidate all alignments for subject with mismatchratio >= mismatchratioThreshold
            for(int index = threadIdx.x; index < candidatesForSubject; index += blockDim.x) {
                const int candidate_index = firstIndex + index;
                if(d_alignmentresultpointers.bestAlignmentFlags[candidate_index] != BestAlignment_t::None) {

                    const int alignment_overlap = d_alignmentresultpointers.overlaps[candidate_index];
                    const int alignment_nops = d_alignmentresultpointers.nOps[candidate_index];

                    const float mismatchratio = float(alignment_nops) / alignment_overlap;

                    const bool doRemove = mismatchratio >= mismatchratioThreshold;
                    if(doRemove){
                        d_alignmentresultpointers.bestAlignmentFlags[candidate_index] = BestAlignment_t::None;
                    }
                }
            }
        }
    }

    










    template<int BLOCKSIZE>
    __global__
    void msa_correct_subject_implicit_kernel(
                            MSAPointers msapointers,
                            AlignmentResultPointers alignmentresultpointers,
                            ReadSequencesPointers d_sequencePointers,
                            CorrectionResultPointers d_correctionResultPointers,
                            const int* __restrict__ d_indices,
                            const int* __restrict__ d_indices_per_subject,
                            const int* __restrict__ d_candidates_per_subject_prefixsum,
                            int n_subjects,
                            int encodedSequencePitchInInts,
                            size_t sequence_pitch,
                            size_t msa_pitch,
                            size_t msa_weights_pitch,
                            int maximumSequenceLength,
                            float estimatedErrorrate,
                            float desiredAlignmentMaxErrorRate,
                            float avg_support_threshold,
                            float min_support_threshold,
                            float min_coverage_threshold,
                            float max_coverage_threshold,
                            int k_region){

        using BlockReduceBool = hipcub::BlockReduce<bool, BLOCKSIZE>;
        using BlockReduceInt = hipcub::BlockReduce<int, BLOCKSIZE>;
        using BlockReduceFloat = hipcub::BlockReduce<float, BLOCKSIZE>;

        __shared__ union {
            typename BlockReduceBool::TempStorage boolreduce;
            typename BlockReduceInt::TempStorage intreduce;
            typename BlockReduceFloat::TempStorage floatreduce;
        } temp_storage;

        __shared__ bool broadcastbuffer;

        __shared__ int numUncorrectedPositions;
        __shared__ int uncorrectedPositions[BLOCKSIZE];
        __shared__ float avgCountPerWeight[4];

        auto get = [] (const char* data, int length, int index){
            //return Sequence_t::get_as_nucleotide(data, length, index);
            return getEncodedNuc2Bit((const unsigned int*)data, length, index, [](auto i){return i;});
        };

        auto getSubjectPtr = [&] (int subjectIndex){
            const unsigned int* result = d_sequencePointers.subjectSequencesData + std::size_t(subjectIndex) * encodedSequencePitchInInts;
            return result;
        };

        auto getCandidatePtr = [&] (int candidateIndex){
            const unsigned int* result = d_sequencePointers.candidateSequencesData + std::size_t(candidateIndex) * encodedSequencePitchInInts;
            return result;
        };

        auto getCandidateLength = [&](int candidateIndex){
            return d_sequencePointers.candidateSequencesLength[candidateIndex];
        };

        auto isGoodAvgSupport = [&](float avgsupport){
            return avgsupport >= avg_support_threshold;
        };
        auto isGoodMinSupport = [&](float minsupport){
            return minsupport >= min_support_threshold;
        };
        auto isGoodMinCoverage = [&](float mincoverage){
            return mincoverage >= min_coverage_threshold;
        };

        constexpr char A_enc = 0x00;
        constexpr char C_enc = 0x01;
        constexpr char G_enc = 0x02;
        constexpr char T_enc = 0x03;

        auto to_nuc = [](char c){
            switch(c){
            case A_enc: return 'A';
            case C_enc: return 'C';
            case G_enc: return 'G';
            case T_enc: return 'T';
            default: return 'F';
            }
        };

        auto saveUncorrectedPositionInSmem = [&](int pos){
            const int smemindex = atomicAdd(&numUncorrectedPositions, 1);
            uncorrectedPositions[smemindex] = pos;
        };

        const size_t msa_weights_pitch_floats = msa_weights_pitch / sizeof(float);

        for(unsigned subjectIndex = blockIdx.x; subjectIndex < n_subjects; subjectIndex += gridDim.x){
            const int myNumIndices = d_indices_per_subject[subjectIndex];
            if(myNumIndices > 0){

                const float* const my_support = msapointers.support + msa_weights_pitch_floats * subjectIndex;
                const int* const my_coverage = msapointers.coverage + msa_weights_pitch_floats * subjectIndex;
                const int* const my_orig_coverage = msapointers.origCoverages + msa_weights_pitch_floats * subjectIndex;
                const char* const my_consensus = msapointers.consensus + msa_pitch  * subjectIndex;
                char* const my_corrected_subject = d_correctionResultPointers.correctedSubjects + subjectIndex * sequence_pitch;

                const int subjectColumnsBegin_incl = msapointers.msaColumnProperties[subjectIndex].subjectColumnsBegin_incl;
                const int subjectColumnsEnd_excl = msapointers.msaColumnProperties[subjectIndex].subjectColumnsEnd_excl;
                const int lastColumn_excl = msapointers.msaColumnProperties[subjectIndex].lastColumn_excl;

                float avg_support = 0;
                float min_support = 1.0f;
                //int max_coverage = 0;
                int min_coverage = std::numeric_limits<int>::max();

                for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                    assert(i < lastColumn_excl);

                    avg_support += my_support[i];
                    min_support = min(my_support[i], min_support);
                    //max_coverage = max(my_coverage[i], max_coverage);
                    min_coverage = min(my_coverage[i], min_coverage);
                }

                avg_support = BlockReduceFloat(temp_storage.floatreduce).Sum(avg_support);
                __syncthreads();

                min_support = BlockReduceFloat(temp_storage.floatreduce).Reduce(min_support, hipcub::Min());
                __syncthreads();

                //max_coverage = BlockReduceInt(temp_storage.intreduce).Reduce(max_coverage, hipcub::Max());

                min_coverage = BlockReduceInt(temp_storage.intreduce).Reduce(min_coverage, hipcub::Min());
                __syncthreads();

                avg_support /= (subjectColumnsEnd_excl - subjectColumnsBegin_incl);

                bool isHQ = isGoodAvgSupport(avg_support) && isGoodMinSupport(min_support) && isGoodMinCoverage(min_coverage);
                //bool isHQ = true;

                if(threadIdx.x == 0){
                    broadcastbuffer = isHQ;
                    d_correctionResultPointers.isHighQualitySubject[subjectIndex].hq(isHQ);
                    //printf("%f %f %d %d\n", avg_support, min_support, min_coverage, isHQ);
                }
                __syncthreads();

                isHQ = broadcastbuffer;

                if(isHQ){
                    for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                        //assert(my_consensus[i] == 'A' || my_consensus[i] == 'C' || my_consensus[i] == 'G' || my_consensus[i] == 'T');
                        my_corrected_subject[i - subjectColumnsBegin_incl] = my_consensus[i];
                    }
                    if(threadIdx.x == 0){
                        d_correctionResultPointers.subjectIsCorrected[subjectIndex] = true;
                    }
                }else{

                    //decode orignal sequence and copy to corrected sequence
                    const int subjectLength = subjectColumnsEnd_excl - subjectColumnsBegin_incl;
                    const unsigned int* const subject = getSubjectPtr(subjectIndex);
                    for(int i = threadIdx.x; i < subjectLength; i += BLOCKSIZE){
                        my_corrected_subject[i] = to_nuc(get((const char*)subject, subjectLength, i));
                    }

                    bool foundAColumn = false;
                    int* globalUncorrectedPostitionsPtr = d_correctionResultPointers.uncorrected_positions_per_subject + subjectIndex * maximumSequenceLength;
                    int* const globalNumUncorrectedPositionsPtr = d_correctionResultPointers.num_uncorrected_positions_per_subject + subjectIndex;

                    //round up to next multiple of BLOCKSIZE;
                    const int loopIters = SDIV(subjectLength, BLOCKSIZE) * BLOCKSIZE;
                    for(int loopIter = 0; loopIter < loopIters; loopIter++){
                        if(threadIdx.x == 0){
                            numUncorrectedPositions = 0;
                        }
                        __syncthreads();

                        const int i = threadIdx.x + loopIter * BLOCKSIZE;

                        if(i < subjectLength){
                            const int globalIndex = subjectColumnsBegin_incl + i;

                            const int origCoverage = my_orig_coverage[globalIndex];
                            const char origBase = my_corrected_subject[i];
                            const char consensusBase = my_consensus[globalIndex];

                            float maxOverlapWeightOrigBase = 0.0f;
                            float maxOverlapWeightConsensusBase = 0.0f;
                            int origBaseCount = 1;
                            int consensusBaseCount = 0;

                            bool goodOrigOverlapExists = false;

                            const int globalOffset = d_candidates_per_subject_prefixsum[subjectIndex];

                            const int* myIndices = d_indices + globalOffset;

                            for(int candidatenr = 0; candidatenr < myNumIndices; candidatenr++){
                                const int arrayindex = myIndices[candidatenr] + globalOffset;

                                const unsigned int* candidateptr = getCandidatePtr(arrayindex);
                                const int candidateLength = getCandidateLength(arrayindex);
                                const int candidateShift = alignmentresultpointers.shifts[arrayindex];
                                const int candidateBasePosition = globalIndex - (subjectColumnsBegin_incl + candidateShift);
                                if(candidateBasePosition >= 0 && candidateBasePosition < candidateLength){
                                    char candidateBaseEnc = 0xFF;
                                    if(alignmentresultpointers.bestAlignmentFlags[arrayindex] == BestAlignment_t::ReverseComplement){
                                        candidateBaseEnc = get((const char*)candidateptr, candidateLength, candidateLength - candidateBasePosition-1);
                                        candidateBaseEnc = (~candidateBaseEnc) & 0x03;
                                    }else{
                                        candidateBaseEnc = get((const char*)candidateptr, candidateLength, candidateBasePosition);
                                    }
                                    const char candidateBase = to_nuc(candidateBaseEnc);

                                    const int nOps = alignmentresultpointers.nOps[arrayindex];
                                    const int overlapsize = alignmentresultpointers.overlaps[arrayindex];
                                    const float overlapweight = calculateOverlapWeight(subjectLength, nOps, overlapsize);
                                    assert(overlapweight <= 1.0f);
                                    assert(overlapweight >= 0.0f);

                                    constexpr float goodOverlapThreshold = 0.90f;

                                    if(origBase == candidateBase){
                                        maxOverlapWeightOrigBase = max(maxOverlapWeightOrigBase, overlapweight);
                                        origBaseCount++;

                                        if(overlapweight >= goodOverlapThreshold){
                                            goodOrigOverlapExists = true;
                                        }
                                    }else{
                                        if(consensusBase == candidateBase){
                                            maxOverlapWeightConsensusBase = max(maxOverlapWeightConsensusBase, overlapweight);
                                            consensusBaseCount++;
                                        }
                                    }
                                }
                            }

                            if(my_support[globalIndex] > 0.5f){

                                constexpr float maxOverlapWeightLowerBound = 0.15f;

                                bool allowCorrectionToConsensus = false;

                                //if(maxOverlapWeightOrigBase < maxOverlapWeightConsensusBase){
                                    allowCorrectionToConsensus = true;
                                //}

                                // if(maxOverlapWeightOrigBase == 0 && maxOverlapWeightConsensusBase == 0){
                                //     //correct to orig;
                                //     allowCorrectionToConsensus = false;
                                // }else if(maxOverlapWeightConsensusBase < maxOverlapWeightLowerBound){
                                //     //correct to orig
                                //     allowCorrectionToConsensus = false;
                                // }else if(maxOverlapWeightOrigBase < maxOverlapWeightLowerBound){
                                //     //correct to consensus
                                //     allowCorrectionToConsensus = true;
                                //     if(origBaseCount < 4){
                                //         allowCorrectionToConsensus = true;
                                //     }
                                // }else if(maxOverlapWeightConsensusBase < maxOverlapWeightOrigBase - 0.2f){
                                //     //maybe correct to orig
                                //     allowCorrectionToConsensus = false;
                                // }else if(maxOverlapWeightConsensusBase  - 0.2f > maxOverlapWeightOrigBase){
                                //     //maybe correct to consensus
                                //     if(origBaseCount < 4){
                                //         allowCorrectionToConsensus = true;
                                //     }
                                // }

                                if(!goodOrigOverlapExists && allowCorrectionToConsensus){

                                    float avgsupportkregion = 0;
                                    int c = 0;
                                    bool kregioncoverageisgood = true;


                                    for(int j = i - k_region/2; j <= i + k_region/2 && kregioncoverageisgood; j++){
                                        if(j != i && j >= 0 && j < subjectLength){
                                            avgsupportkregion += my_support[subjectColumnsBegin_incl + j];
                                            kregioncoverageisgood &= (my_coverage[subjectColumnsBegin_incl + j] >= min_coverage_threshold);
                                            //kregioncoverageisgood &= (my_coverage[subjectColumnsBegin_incl + j] >= 1);
                                            c++;
                                        }
                                    }
                                    avgsupportkregion /= c;

                                    if(kregioncoverageisgood && avgsupportkregion >= 1.0f-4*estimatedErrorrate / 2.0f){


                                        // constexpr float maxOverlapWeightLowerBound = 0.25f;
                                        //
                                        // bool correctToConsensus = false;//maxOverlapWeightOrigBase < maxOverlapWeightLowerBound;
                                        // // correctToConsensus |= maxOverlapWeightConsensusBase >= maxOverlapWeightOrigBase;
                                        // // correctToConsensus &= !goodOrigOverlapExists;
                                        // if(!goodOrigOverlapExists && (origBase != consensusBase && my_support[globalIndex] > 0.5f)){
                                        //     correctToConsensus = true;
                                        // }

                                        // if(maxOverlapWeightOrigBase == 0 && maxOverlapWeightConsensusBase == 0){
                                        //     //correct to orig;
                                        // }else if(maxOverlapWeightConsensusBase < maxOverlapWeightLowerBound){
                                        //     //correct to orig
                                        // }else if(maxOverlapWeightOrigBase < maxOverlapWeightLowerBound){
                                        //     //correct to consensus
                                        //     my_corrected_subject[i] = consensusBase;
                                        // }else if(maxOverlapWeightConsensusBase < maxOverlapWeightOrigBase){
                                        //     //maybe correct to orig
                                        // }else if(maxOverlapWeightConsensusBase >= maxOverlapWeightOrigBase){
                                        //     //maybe correct to consensus
                                        //     my_corrected_subject[i] = consensusBase;
                                        // }

                                        //if(correctToConsensus){
                                            my_corrected_subject[i] = consensusBase;
                                            foundAColumn = true;
                                        // }else{
                                        //     saveUncorrectedPositionInSmem(i);
                                        // }
                                    }else{
                                        saveUncorrectedPositionInSmem(i);
                                    }
                                }
                            }else{
                                saveUncorrectedPositionInSmem(i);
                            }
                        }

                        __syncthreads();

                        if(threadIdx.x == 0){
                            *globalNumUncorrectedPositionsPtr += numUncorrectedPositions;
                        }

                        for(int k = threadIdx.x; k < numUncorrectedPositions; k += BLOCKSIZE){
                            globalUncorrectedPostitionsPtr[k] = uncorrectedPositions[k];
                        }
                        globalUncorrectedPostitionsPtr += numUncorrectedPositions;

                        if(loopIter < loopIters - 1){
                            __syncthreads();
                        }
                    }

                    //perform block wide or-reduction on foundAColumn
                    foundAColumn = BlockReduceBool(temp_storage.boolreduce).Reduce(foundAColumn, [](bool a, bool b){return a || b;});
                    __syncthreads();

                    if(threadIdx.x == 0){
                        d_correctionResultPointers.subjectIsCorrected[subjectIndex] = true;//foundAColumn;
                    }
                }
            }
        }
    }





    template<int BLOCKSIZE>
    __global__
    void msa_correct_subject_implicit_kernel2(
                            MSAPointers msapointers,
                            AlignmentResultPointers alignmentresultpointers,
                            ReadSequencesPointers d_sequencePointers,
                            CorrectionResultPointers d_correctionResultPointers,
                            const int* __restrict__ d_indices_per_subject,
                            int n_subjects,
                            int encodedSequencePitchInInts,
                            size_t sequence_pitch,
                            size_t msa_pitch,
                            size_t msa_weights_pitch,
                            int maximumSequenceLength,
                            float estimatedErrorrate,
                            float desiredAlignmentMaxErrorRate,
                            float avg_support_threshold,
                            float min_support_threshold,
                            float min_coverage_threshold,
                            float max_coverage_threshold,
                            int k_region){

        using BlockReduceBool = hipcub::BlockReduce<bool, BLOCKSIZE>;
        using BlockReduceInt = hipcub::BlockReduce<int, BLOCKSIZE>;
        using BlockReduceFloat = hipcub::BlockReduce<float, BLOCKSIZE>;

        __shared__ union {
            typename BlockReduceBool::TempStorage boolreduce;
            typename BlockReduceInt::TempStorage intreduce;
            typename BlockReduceFloat::TempStorage floatreduce;
        } temp_storage;

        __shared__ int broadcastbuffer;

        __shared__ int numUncorrectedPositions;
        __shared__ int uncorrectedPositions[BLOCKSIZE];
        __shared__ float avgCountPerWeight[4];

        auto get = [] (const char* data, int length, int index){
            //return Sequence_t::get_as_nucleotide(data, length, index);
            return getEncodedNuc2Bit((const unsigned int*)data, length, index, [](auto i){return i;});
        };

        auto getSubjectPtr = [&] (int subjectIndex){
            const unsigned int* result = d_sequencePointers.subjectSequencesData + std::size_t(subjectIndex) * encodedSequencePitchInInts;
            return result;
        };

        auto getCandidatePtr = [&] (int candidateIndex){
            const unsigned int* result = d_sequencePointers.candidateSequencesData + std::size_t(candidateIndex) * encodedSequencePitchInInts;
            return result;
        };

        auto getCandidateLength = [&](int candidateIndex){
            return d_sequencePointers.candidateSequencesLength[candidateIndex];
        };

        auto isGoodAvgSupport = [&](float avgsupport){
            return avgsupport >= avg_support_threshold;
        };
        auto isGoodMinSupport = [&](float minsupport){
            return minsupport >= min_support_threshold;
        };
        auto isGoodMinCoverage = [&](float mincoverage){
            return mincoverage >= min_coverage_threshold;
        };

        constexpr char A_enc = 0x00;
        constexpr char C_enc = 0x01;
        constexpr char G_enc = 0x02;
        constexpr char T_enc = 0x03;

        auto to_nuc = [](char c){
            switch(c){
            case A_enc: return 'A';
            case C_enc: return 'C';
            case G_enc: return 'G';
            case T_enc: return 'T';
            default: return 'F';
            }
        };

        const size_t msa_weights_pitch_floats = msa_weights_pitch / sizeof(float);

        for(unsigned subjectIndex = blockIdx.x; subjectIndex < n_subjects; subjectIndex += gridDim.x){
            const int myNumIndices = d_indices_per_subject[subjectIndex];
            if(myNumIndices > 0){

                const float* const my_support = msapointers.support + msa_weights_pitch_floats * subjectIndex;
                const int* const my_coverage = msapointers.coverage + msa_weights_pitch_floats * subjectIndex;
                const int* const my_orig_coverage = msapointers.origCoverages + msa_weights_pitch_floats * subjectIndex;
                const char* const my_consensus = msapointers.consensus + msa_pitch  * subjectIndex;
                char* const my_corrected_subject = d_correctionResultPointers.correctedSubjects + subjectIndex * sequence_pitch;

                const int subjectColumnsBegin_incl = msapointers.msaColumnProperties[subjectIndex].subjectColumnsBegin_incl;
                const int subjectColumnsEnd_excl = msapointers.msaColumnProperties[subjectIndex].subjectColumnsEnd_excl;
                const int lastColumn_excl = msapointers.msaColumnProperties[subjectIndex].lastColumn_excl;

                float avg_support = 0;
                float min_support = 1.0f;
                //int max_coverage = 0;
                int min_coverage = std::numeric_limits<int>::max();

                for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                    assert(i < lastColumn_excl);

                    avg_support += my_support[i];
                    min_support = min(my_support[i], min_support);
                    //max_coverage = max(my_coverage[i], max_coverage);
                    min_coverage = min(my_coverage[i], min_coverage);
                }

                avg_support = BlockReduceFloat(temp_storage.floatreduce).Sum(avg_support);
                __syncthreads();

                min_support = BlockReduceFloat(temp_storage.floatreduce).Reduce(min_support, hipcub::Min());
                __syncthreads();

                //max_coverage = BlockReduceInt(temp_storage.intreduce).Reduce(max_coverage, hipcub::Max());

                min_coverage = BlockReduceInt(temp_storage.intreduce).Reduce(min_coverage, hipcub::Min());
                __syncthreads();

                avg_support /= (subjectColumnsEnd_excl - subjectColumnsBegin_incl);


                const float avg_support_threshold = 1.0f-1.0f*estimatedErrorrate;
        		const float min_support_threshold = 1.0f-3.0f*estimatedErrorrate;

                if(threadIdx.x == 0){
                    d_correctionResultPointers.subjectIsCorrected[subjectIndex] = true; //canBeCorrected;

                    const bool canBeCorrectedByConsensus = isGoodAvgSupport(avg_support) && isGoodMinSupport(min_support) && isGoodMinCoverage(min_coverage);
                    int flag = 0;

                    if(canBeCorrectedByConsensus){
                        int smallestErrorrateThatWouldMakeHQ = 100;

                        const int estimatedErrorratePercent = ceil(estimatedErrorrate * 100.0f);
                        for(int percent = estimatedErrorratePercent; percent >= 0; percent--){
                            float factor = percent / 100.0f;
                            if(avg_support >= 1.0f - 1.0f * factor && min_support >= 1.0f - 3.0f * factor){
                                smallestErrorrateThatWouldMakeHQ = percent;
                            }
                        }

                        const bool isHQ = isGoodMinCoverage(min_coverage)
                                            && smallestErrorrateThatWouldMakeHQ <= estimatedErrorratePercent * 0.5f;

                        //broadcastbuffer = isHQ;
                        d_correctionResultPointers.isHighQualitySubject[subjectIndex].hq(isHQ);

                        flag = isHQ ? 2 : 1;
                    }

                    broadcastbuffer = flag;
                }
                __syncthreads();

                // for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                //     //assert(my_consensus[i] == 'A' || my_consensus[i] == 'C' || my_consensus[i] == 'G' || my_consensus[i] == 'T');
                //     if(my_support[i] > 0.90f && my_orig_coverage[i] <= 2){
                //         my_corrected_subject[i - subjectColumnsBegin_incl] = my_consensus[i];
                //     }else{
                //         const char* subject = getSubjectPtr(subjectIndex);
                //         const char encodedBase = get(subject, subjectColumnsEnd_excl- subjectColumnsBegin_incl, i - subjectColumnsBegin_incl);
                //         const char base = to_nuc(encodedBase);
                //         my_corrected_subject[i - subjectColumnsBegin_incl] = base;
                //     }
                // }

                const int flag = broadcastbuffer;

                if(flag > 0){
                    for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                        my_corrected_subject[i - subjectColumnsBegin_incl] = my_consensus[i];
                    }
                }else{
                    //correct only positions with high support.
                    for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                        //assert(my_consensus[i] == 'A' || my_consensus[i] == 'C' || my_consensus[i] == 'G' || my_consensus[i] == 'T');
                        if(my_support[i] > 0.90f && my_orig_coverage[i] <= 2){
                            my_corrected_subject[i - subjectColumnsBegin_incl] = my_consensus[i];
                        }else{
                            const unsigned int* subject = getSubjectPtr(subjectIndex);
                            const char encodedBase = get((const char*)subject, subjectColumnsEnd_excl- subjectColumnsBegin_incl, i - subjectColumnsBegin_incl);
                            const char base = to_nuc(encodedBase);
                            my_corrected_subject[i - subjectColumnsBegin_incl] = base;
                        }
                    }
                }
            }else{
                if(threadIdx.x == 0){
                    d_correctionResultPointers.isHighQualitySubject[subjectIndex].hq(false);
                    d_correctionResultPointers.subjectIsCorrected[subjectIndex] = false;
                }
            }
        }
    }







    __device__ __forceinline__
    bool checkIfCandidateShouldBeCorrected(const MSAPointers& d_msapointers,
                        const AlignmentResultPointers& d_alignmentresultpointers,
                        const ReadSequencesPointers& d_sequencePointers,
                        const CorrectionResultPointers& d_correctionResultPointers,
                        const int* __restrict__ d_indices,
                        const int* __restrict__ d_candidates_per_subject_prefixsum,
                        size_t msa_weights_pitch_floats,
                        float min_support_threshold,
                        float min_coverage_threshold,
                        int new_columns_to_correct,
                        int subjectIndex,
                        int local_goodcandidate_index){

        const float* const my_support = d_msapointers.support + msa_weights_pitch_floats * subjectIndex;
        const int* const my_coverage = d_msapointers.coverage + msa_weights_pitch_floats * subjectIndex;

        const int globalOffset = d_candidates_per_subject_prefixsum[subjectIndex];
        const int* const my_indices = d_indices + globalOffset;

        const int subjectColumnsBegin_incl = d_msapointers.msaColumnProperties[subjectIndex].subjectColumnsBegin_incl;
        const int subjectColumnsEnd_excl = d_msapointers.msaColumnProperties[subjectIndex].subjectColumnsEnd_excl;
        const int lastColumn_excl = d_msapointers.msaColumnProperties[subjectIndex].lastColumn_excl;

        const int localCandidateIndex = my_indices[local_goodcandidate_index];
        const int global_candidate_index = localCandidateIndex + globalOffset;

        const int shift = d_alignmentresultpointers.shifts[global_candidate_index];
        const int candidate_length = d_sequencePointers.candidateSequencesLength[global_candidate_index];
        const int queryColumnsBegin_incl = subjectColumnsBegin_incl + shift;
        const int queryColumnsEnd_excl = subjectColumnsBegin_incl + shift + candidate_length;

        if(subjectColumnsBegin_incl - new_columns_to_correct <= queryColumnsBegin_incl
           && queryColumnsBegin_incl <= subjectColumnsBegin_incl + new_columns_to_correct
           && queryColumnsEnd_excl <= subjectColumnsEnd_excl + new_columns_to_correct) {

            float newColMinSupport = 1.0f;
            int newColMinCov = std::numeric_limits<int>::max();
            //check new columns left of subject
            for(int columnindex = subjectColumnsBegin_incl - new_columns_to_correct;
                columnindex < subjectColumnsBegin_incl;
                columnindex++) {

                assert(columnindex < lastColumn_excl);
                if(queryColumnsBegin_incl <= columnindex) {
                    newColMinSupport = my_support[columnindex] < newColMinSupport ? my_support[columnindex] : newColMinSupport;
                    newColMinCov = my_coverage[columnindex] < newColMinCov ? my_coverage[columnindex] : newColMinCov;
                }
            }
            //check new columns right of subject
            for(int columnindex = subjectColumnsEnd_excl;
                columnindex < subjectColumnsEnd_excl + new_columns_to_correct
                && columnindex < lastColumn_excl;
                columnindex++) {

                newColMinSupport = my_support[columnindex] < newColMinSupport ? my_support[columnindex] : newColMinSupport;
                newColMinCov = my_coverage[columnindex] < newColMinCov ? my_coverage[columnindex] : newColMinCov;
            }

            bool result = newColMinSupport >= min_support_threshold
                            && newColMinCov >= min_coverage_threshold;

            //return result;
            return true;
        }else{
            return false;
        }

    }



    template<int BLOCKSIZE>
    __global__
    void msa_correct_candidates_kernel(
                MSAPointers d_msapointers,
                AlignmentResultPointers d_alignmentresultpointers,
                ReadSequencesPointers d_sequencePointers,
                CorrectionResultPointers d_correctionResultPointers,
                const int* __restrict__ d_indices,
                const int* __restrict__ d_indices_per_subject,
                const int* __restrict__ d_candidates_per_subject_prefixsum,
                const int* __restrict__ d_candidates_per_hq_subject_prefixsum, // inclusive, with leading zero
                //int* __restrict__ globalCommBuffer, // at least n_subjects elements, must be zero'd
                int n_subjects,
                int n_queries,
                const int* __restrict__ d_num_indices,
                int encodedSequencePitchInInts,
                size_t sequence_pitch,
                size_t msa_pitch,
                size_t msa_weights_pitch,
                float min_support_threshold,
                float min_coverage_threshold,
                int new_columns_to_correct){

        auto make_unpacked_reverse_complement_inplace = [] (std::uint8_t* sequence, int sequencelength){
            return reverseComplementStringInplace((char*)sequence, sequencelength);
        };

        auto get = [] (const char* data, int length, int index){
            //return Sequence_t::get_as_nucleotide(data, length, index);
            return getEncodedNuc2Bit((const unsigned int*)data, length, index, [](auto i){return i;});
        };

        constexpr char A_enc = 0x00;
        constexpr char C_enc = 0x01;
        constexpr char G_enc = 0x02;
        constexpr char T_enc = 0x03;

        auto to_nuc = [](char c){
            switch(c){
            case A_enc: return 'A';
            case C_enc: return 'C';
            case G_enc: return 'G';
            case T_enc: return 'T';
            default: return 'F';
            }
        };

        auto getCandidatePtr = [&] (int candidateIndex){
            const unsigned int* result = d_sequencePointers.candidateSequencesData + std::size_t(candidateIndex) * encodedSequencePitchInInts;
            return result;
        };

        using BlockReduceInt = hipcub::BlockReduce<int, BLOCKSIZE>;

        // __shared__ int numCandidatesForSubjectInThisBlockShared[BLOCKSIZE];
        // __shared__ int numCorrectedCandidatesForSubjectInThisBlockShared[BLOCKSIZE];
        // __shared__ int histogram[BLOCKSIZE];
        // __shared__ int hqsubjectIndices[BLOCKSIZE];
        // __shared__ int broadcastbuffer;
        // __shared__ union{
        //     typename BlockReduceInt::TempStorage intreduce;
        // } temp_storage;

        const size_t msa_weights_pitch_floats = msa_weights_pitch / sizeof(float);
        const int num_high_quality_subject_indices = *d_correctionResultPointers.numHighQualitySubjectIndices;
        const int num_candidates_of_hq_subjects = d_candidates_per_hq_subject_prefixsum[num_high_quality_subject_indices];

        //round up to next multiple of BLOCKSIZE;
        const int loopEnd = SDIV(num_candidates_of_hq_subjects, BLOCKSIZE) * BLOCKSIZE;

        for(int candidateHQid = threadIdx.x + blockIdx.x * blockDim.x;
                candidateHQid < loopEnd;
                candidateHQid += blockDim.x * gridDim.x){

            //__syncthreads();

            const int hqsubjectIndex = candidateHQid >= num_candidates_of_hq_subjects
                                        ?   std::numeric_limits<int>::max()
                                        :   thrust::distance(d_candidates_per_hq_subject_prefixsum,
                                                thrust::lower_bound(
                                                    thrust::seq,
                                                    d_candidates_per_hq_subject_prefixsum,
                                                    d_candidates_per_hq_subject_prefixsum + num_high_quality_subject_indices + 1,
                                                    candidateHQid + 1))-1;

            // if(candidateHQid < num_candidates_of_hq_subjects){
            //     hqsubjectIndices[threadIdx.x] = hqsubjectIndex;
            //     histogram[threadIdx.x] = 0;
            //     numCandidatesForSubjectInThisBlockShared[threadIdx.x] = 0;
            // }
            // __syncthreads();
            //
            // const int smallestHqsubjectIndexInBlock = hqsubjectIndices[0];
            //
            // //count histogram
            // if(candidateHQid < num_candidates_of_hq_subjects){
            //     atomicAdd(histogram + (hqsubjectIndex - smallestHqsubjectIndexInBlock), 1);
            // }
            //
            // int discontinuity = (threadIdx.x == 0);
            // if(threadIdx.x > 0 && candidateHQid < num_candidates_of_hq_subjects){
            //     discontinuity = (hqsubjectIndices[threadIdx.x] != hqsubjectIndices[threadIdx.x-1]);
            // }
            //
            // int numberOfUniquehqsubjectindices = BlockReduceInt(temp_storage.intreduce).Reduce(discontinuity, hipcub::Sum{});
            // if(threadIdx.x == 0){
            //     broadcastbuffer = numberOfUniquehqsubjectindices;
            // }
            // __syncthreads();
            // numberOfUniquehqsubjectindices = broadcastbuffer;
            //
            // if(threadIdx.x < numberOfUniquehqsubjectindices){
            //     const int localcount = histogram[threadIdx.x];
            //     const int hqsubindex = smallestHqsubjectIndexInBlock + threadIdx.x;
            //     const int subjectIndex = d_correctionResultPointers.highQualitySubjectIndices[hqsubindex];
            //
            //     assert(subjectIndex < n_subjects);
            //
            //     numCandidatesForSubjectInThisBlockShared[threadIdx.x] = atomicAdd(globalCommBuffer + subjectIndex, localcount);
            // }
            //
            // __syncthreads();

            if(candidateHQid < num_candidates_of_hq_subjects){

                const int subjectIndex = d_correctionResultPointers.highQualitySubjectIndices[hqsubjectIndex];
                const int local_goodcandidate_index = candidateHQid - d_candidates_per_hq_subject_prefixsum[hqsubjectIndex];

                const bool canHandleCandidate = checkIfCandidateShouldBeCorrected(
                                                        d_msapointers,
                                                        d_alignmentresultpointers,
                                                        d_sequencePointers,
                                                        d_correctionResultPointers,
                                                        d_indices,
                                                        d_candidates_per_subject_prefixsum,
                                                        msa_weights_pitch_floats,
                                                        min_support_threshold,
                                                        min_coverage_threshold,
                                                        new_columns_to_correct,
                                                        subjectIndex,
                                                        local_goodcandidate_index);

                if(canHandleCandidate) {

                    //assert((hqsubjectIndex - smallestHqsubjectIndexInBlock) < numberOfUniquehqsubjectindices);

                    //const int destinationindex = atomicAdd(numCandidatesForSubjectInThisBlockShared + (hqsubjectIndex - smallestHqsubjectIndexInBlock), 1);
                    //atomicAdd(numCorrectedCandidatesForSubjectInThisBlockShared + (hqsubjectIndex - smallestHqsubjectIndexInBlock), 1);

                    const int destinationindex = atomicAdd(d_correctionResultPointers.numCorrectedCandidates + subjectIndex, 1);
//TODO FIX INDICES
                    const int globalOffset = d_candidates_per_subject_prefixsum[subjectIndex];
                    const char* const my_consensus = d_msapointers.consensus + msa_pitch  * subjectIndex;
                    const int* const my_indices = d_indices + globalOffset;
                    char* const my_corrected_candidates = d_correctionResultPointers.correctedCandidates + globalOffset * sequence_pitch;
                    int* const my_indices_of_corrected_candidates = d_correctionResultPointers.indicesOfCorrectedCandidates + globalOffset;

                    const int localCandidateIndex = my_indices[local_goodcandidate_index];
                    const int global_candidate_index = localCandidateIndex;
                    const int candidate_length = d_sequencePointers.candidateSequencesLength[global_candidate_index];
                    const int shift = d_alignmentresultpointers.shifts[global_candidate_index];

                    const int subjectColumnsBegin_incl = d_msapointers.msaColumnProperties[subjectIndex].subjectColumnsBegin_incl;
                    //const int subjectColumnsEnd_excl = d_msapointers.msaColumnProperties[subjectIndex].subjectColumnsEnd_excl;
                    //const int lastColumn_excl = d_msapointers.msaColumnProperties[subjectIndex].lastColumn_excl;
                    const int queryColumnsBegin_incl = subjectColumnsBegin_incl + shift;
                    const int queryColumnsEnd_excl = subjectColumnsBegin_incl + shift + candidate_length;

                    const int copyposbegin = queryColumnsBegin_incl; //max(queryColumnsBegin_incl, subjectColumnsBegin_incl);
                    const int copyposend = queryColumnsEnd_excl; //min(queryColumnsEnd_excl, subjectColumnsEnd_excl);

                    for(int i = copyposbegin; i < copyposend; i += 1) {
                        my_corrected_candidates[destinationindex * sequence_pitch + (i - queryColumnsBegin_incl)] = my_consensus[i];
                    }

                    const float* const my_support = d_msapointers.support + msa_weights_pitch_floats * subjectIndex;
                    const unsigned int* candidate = d_sequencePointers.candidateSequencesData + std::size_t(global_candidate_index) * encodedSequencePitchInInts;

                    // for(int i = copyposbegin; i < copyposend; i += 1) {
                    //     //assert(my_consensus[i] == 'A' || my_consensus[i] == 'C' || my_consensus[i] == 'G' || my_consensus[i] == 'T');
                    //     if(my_support[i] > 0.90f){
                    //         my_corrected_candidates[destinationindex * sequence_pitch + (i - queryColumnsBegin_incl)] = my_consensus[i];
                    //     }else{
                    //         const char encodedBase = get(candidate, queryColumnsEnd_excl- queryColumnsBegin_incl, i - queryColumnsBegin_incl);
                    //         const char base = to_nuc(encodedBase);
                    //         my_corrected_candidates[destinationindex * sequence_pitch + (i - queryColumnsBegin_incl)] = base;
                    //     }
                    // }

                    const BestAlignment_t bestAlignmentFlag = d_alignmentresultpointers.bestAlignmentFlags[global_candidate_index];

                    //the forward strand will be returned -> make reverse complement again
                    if(bestAlignmentFlag == BestAlignment_t::ReverseComplement) {
                        make_unpacked_reverse_complement_inplace((std::uint8_t*)(my_corrected_candidates + destinationindex * sequence_pitch), candidate_length);
                    }

                    my_indices_of_corrected_candidates[destinationindex] = global_candidate_index;
                    //printf("subjectIndex %d global_candidate_index %d\n", subjectIndex, global_candidate_index);
                }

            }
        }
    }



    template<int BLOCKSIZE>
    __global__
    void compactCandidateCorrectionResultsKernel(
            char* __restrict__ compactedCorrectedCandidates,
            const int* __restrict__ numCorrectedCandidatesPerAnchor,
            const int* __restrict__ numCorrectedCandidatesPerAnchorPrefixsum, //exclusive
            const int* __restrict__ high_quality_subject_indices,
            const int* __restrict__ num_high_quality_subject_indices,
            const int* __restrict__ candidates_per_subject_prefixsum,
            const char* __restrict__ correctedCandidates,
            const int* __restrict__ correctedCandidateLengths,
            size_t decodedSequencePitch,
            int n_subjects){

        constexpr int groupsize = 32;
        static_assert(groupsize <= 32);
        static_assert(BLOCKSIZE % groupsize == 0);

        const int numHqSubjects = *num_high_quality_subject_indices;

        auto tgroup = cg::tiled_partition<groupsize>(cg::this_thread_block());

        const int numGroups = (gridDim.x * blockDim.x) / groupsize;
        const int groupId = (threadIdx.x + blockIdx.x * blockDim.x) / groupsize;
        const int groupIdInBlock = threadIdx.x / groupsize;

        for(int hqsubjectIndex = groupId;
                hqsubjectIndex < numHqSubjects;
                hqsubjectIndex += numGroups){

            const int subjectIndex = high_quality_subject_indices[hqsubjectIndex];
            const int myNumCorrectedCandidates = numCorrectedCandidatesPerAnchor[subjectIndex];

            if(myNumCorrectedCandidates > 0){
                const int inputbaseoffset = candidates_per_subject_prefixsum[subjectIndex];
                const int outputbaseoffset = numCorrectedCandidatesPerAnchorPrefixsum[subjectIndex];

                for(int cIndex = tgroup.thread_rank(); cIndex < myNumCorrectedCandidates; cIndex += tgroup.size()){

                    for(int i = 0; i < decodedSequencePitch; i++){
                        compactedCorrectedCandidates[(outputbaseoffset+cIndex) * decodedSequencePitch + i] 
                            = correctedCandidates[(inputbaseoffset+cIndex) * decodedSequencePitch + i];
                    }
                }
            }
        }
    }

    template<int BLOCKSIZE, int groupsize>
    __global__
    void msa_correct_candidates_with_group_kernel(
                MSAPointers d_msapointers,
                AlignmentResultPointers d_alignmentresultpointers,
                ReadSequencesPointers d_sequencePointers,
                CorrectionResultPointers d_correctionResultPointers,
                int* __restrict__ d_numTotalCorrectedCandidates,
                TempCorrectedSequence::Edit* __restrict__ d_editsPerCorrectedCandidate,
                int* __restrict__ d_numEditsPerCorrectedCandidate,
                const bool* __restrict__ d_candidateContainsN,
                int doNotUseEditsValue,
                int numEditsThreshold,
                const int* __restrict__ d_indices,
                const int* __restrict__ d_indices_per_subject,
                const int* __restrict__ d_candidates_per_subject_prefixsum,
                const int* __restrict__ d_candidates_per_hq_subject_prefixsum, // inclusive, with leading zero
                //int* __restrict__ globalCommBuffer, // at least n_subjects elements, must be zero'd
                int n_subjects,
                int n_queries,
                const int* __restrict__ d_num_indices,
                int encodedSequencePitchInInts,
                size_t sequence_pitch,
                size_t msa_pitch,
                size_t msa_weights_pitch,
                size_t dynamicsmemPitchInInts,
                float min_support_threshold,
                float min_coverage_threshold,
                int new_columns_to_correct){

        /*
            Use groupsize threads per candidate to perform correction
        */
        static_assert(BLOCKSIZE % groupsize == 0, "BLOCKSIZE % groupsize != 0");
        constexpr int groupsPerBlock = BLOCKSIZE / groupsize;


        auto make_unpacked_reverse_complement_inplace = [] (std::uint8_t* sequence, int sequencelength){
            return reverseComplementStringInplace((char*)sequence, sequencelength);
        };

        auto decodedReverseComplementInplaceGroup = [](auto group, char* sequence, int sequencelength){
            auto make_reverse_complement_nuc = [](char in){
                switch(in){
                    case 'A': return 'T';
                    case 'C': return 'G';
                    case 'G': return 'C';
                    case 'T': return 'A';
                    default :return 'F';
                }
            };
    
            for(int i = group.thread_rank(); i < sequencelength/2; i += group.size()){
                const std::uint8_t front = make_reverse_complement_nuc(sequence[i]);
                const std::uint8_t back = make_reverse_complement_nuc(sequence[sequencelength - 1 - i]);
                sequence[i] = back;
                sequence[sequencelength - 1 - i] = front;
            }
    
            if(sequencelength % 2 == 1 && group.thread_rank() == 0){
                const int middleindex = sequencelength/2;
                sequence[middleindex] = make_reverse_complement_nuc(sequence[middleindex]);
            }
        };

        auto getEncodedNucFromInt2Bit = [](unsigned int data, int pos){
            return ((data >> (30 - 2*pos)) & 0x00000003);
        };

        auto to_nuc = [](char c){
            constexpr char A_enc = 0x00;
            constexpr char C_enc = 0x01;
            constexpr char G_enc = 0x02;
            constexpr char T_enc = 0x03;

            switch(c){
            case A_enc: return 'A';
            case C_enc: return 'C';
            case G_enc: return 'G';
            case T_enc: return 'T';
            default: return 'F';
            }
        };

        using BlockReduceInt = hipcub::BlockReduce<int, BLOCKSIZE>;



        __shared__ int shared_destinationIndex[groupsPerBlock];
        __shared__ int shared_numEditsOfCandidate[groupsPerBlock];
        extern __shared__ int dynamicsmem[];



        auto tgroup = cg::tiled_partition<groupsize>(cg::this_thread_block());

        const int numGroups = (gridDim.x * blockDim.x) / groupsize;
        const int groupId = (threadIdx.x + blockIdx.x * blockDim.x) / groupsize;
        const int groupIdInBlock = threadIdx.x / groupsize;

        char* const shared_correctedCandidate = (char*)(dynamicsmem + dynamicsmemPitchInInts * groupIdInBlock);


        const size_t msa_weights_pitch_floats = msa_weights_pitch / sizeof(float);
        const int num_high_quality_subject_indices = *d_correctionResultPointers.numHighQualitySubjectIndices;
        const int num_candidates_of_hq_subjects = d_candidates_per_hq_subject_prefixsum[num_high_quality_subject_indices];

        for(int candidateHQid = groupId;
                candidateHQid < num_candidates_of_hq_subjects;
                candidateHQid +=  numGroups){

            if(candidateHQid < num_candidates_of_hq_subjects){

                const int hqsubjectIndex = thrust::distance(d_candidates_per_hq_subject_prefixsum,
                    thrust::lower_bound(
                        thrust::seq,
                        d_candidates_per_hq_subject_prefixsum,
                        d_candidates_per_hq_subject_prefixsum + num_high_quality_subject_indices + 1,
                        candidateHQid + 1))-1;

                const int subjectIndex = d_correctionResultPointers.highQualitySubjectIndices[hqsubjectIndex];
                const int local_goodcandidate_index = candidateHQid - d_candidates_per_hq_subject_prefixsum[hqsubjectIndex];

                const bool canHandleCandidate = checkIfCandidateShouldBeCorrected(
                                                        d_msapointers,
                                                        d_alignmentresultpointers,
                                                        d_sequencePointers,
                                                        d_correctionResultPointers,
                                                        d_indices,
                                                        d_candidates_per_subject_prefixsum,
                                                        msa_weights_pitch_floats,
                                                        min_support_threshold,
                                                        min_coverage_threshold,
                                                        new_columns_to_correct,
                                                        subjectIndex,
                                                        local_goodcandidate_index);

                if(canHandleCandidate) {
                    if(tgroup.thread_rank() == 0){                        
                        shared_destinationIndex[groupIdInBlock] = atomicAdd(d_correctionResultPointers.numCorrectedCandidates + subjectIndex, 1);
                        shared_numEditsOfCandidate[groupIdInBlock] = 0;
                        assert(shared_destinationIndex[groupIdInBlock] < d_indices_per_subject[subjectIndex]);
                    }
                    tgroup.sync();

                    const int globalOffset = d_candidates_per_subject_prefixsum[subjectIndex];
                    

                    const char* const my_consensus = d_msapointers.consensus + msa_pitch * subjectIndex;
                    const int* const my_indices = d_indices + globalOffset;
                    char* const my_corrected_candidates = d_correctionResultPointers.correctedCandidates + globalOffset * sequence_pitch;
                    int* const my_indices_of_corrected_candidates = d_correctionResultPointers.indicesOfCorrectedCandidates + globalOffset;

                    const int localCandidateIndex = my_indices[local_goodcandidate_index];
                    const int global_candidate_index = localCandidateIndex + globalOffset;
                    const int candidate_length = d_sequencePointers.candidateSequencesLength[global_candidate_index];
                    const int shift = d_alignmentresultpointers.shifts[global_candidate_index];

                    const int subjectColumnsBegin_incl = d_msapointers.msaColumnProperties[subjectIndex].subjectColumnsBegin_incl;
                    //const int subjectColumnsEnd_excl = d_msapointers.msaColumnProperties[subjectIndex].subjectColumnsEnd_excl;
                    //const int lastColumn_excl = d_msapointers.msaColumnProperties[subjectIndex].lastColumn_excl;
                    const int queryColumnsBegin_incl = subjectColumnsBegin_incl + shift;
                    const int queryColumnsEnd_excl = subjectColumnsBegin_incl + shift + candidate_length;

                    const int copyposbegin = queryColumnsBegin_incl; //max(queryColumnsBegin_incl, subjectColumnsBegin_incl);
                    const int copyposend = queryColumnsEnd_excl; //min(queryColumnsEnd_excl, subjectColumnsEnd_excl);
                    assert(copyposend - copyposbegin == candidate_length);

                    for(int i = copyposbegin + tgroup.thread_rank(); i < copyposend; i += tgroup.size()) {
                        shared_correctedCandidate[i - queryColumnsBegin_incl] = my_consensus[i];
                        //my_corrected_candidates[destinationindex * sequence_pitch + (i - queryColumnsBegin_incl)] = my_consensus[i];
                    }

                    const float* const my_support = d_msapointers.support + msa_weights_pitch_floats * subjectIndex;
                    const unsigned int* candidate = d_sequencePointers.candidateSequencesData + std::size_t(global_candidate_index) * encodedSequencePitchInInts;

                    // for(int i = copyposbegin; i < copyposend; i += 1) {
                    //     //assert(my_consensus[i] == 'A' || my_consensus[i] == 'C' || my_consensus[i] == 'G' || my_consensus[i] == 'T');
                    //     if(my_support[i] > 0.90f){
                    //         my_corrected_candidates[destinationindex * sequence_pitch + (i - queryColumnsBegin_incl)] = my_consensus[i];
                    //     }else{
                    //         const char encodedBase = get(candidate, queryColumnsEnd_excl- queryColumnsBegin_incl, i - queryColumnsBegin_incl);
                    //         const char base = to_nuc(encodedBase);
                    //         my_corrected_candidates[destinationindex * sequence_pitch + (i - queryColumnsBegin_incl)] = base;
                    //     }
                    // }

                    const BestAlignment_t bestAlignmentFlag = d_alignmentresultpointers.bestAlignmentFlags[global_candidate_index];

                    //the forward strand will be returned -> make reverse complement again
                    if(bestAlignmentFlag == BestAlignment_t::ReverseComplement) {
                        tgroup.sync(); // threads may access elements in shared memory which were written by another thread
                        decodedReverseComplementInplaceGroup(tgroup, shared_correctedCandidate, candidate_length);
                        tgroup.sync();
                    }

                    //copy from smem to global output
                    for(int i = tgroup.thread_rank(); i < candidate_length; i += tgroup.size()) {
                        const int destinationindex = shared_destinationIndex[groupIdInBlock];
                        my_corrected_candidates[destinationindex * sequence_pitch + i] = shared_correctedCandidate[i];
                    }

                    if(tgroup.thread_rank() == 0){
                        const int destinationindex = shared_destinationIndex[groupIdInBlock];
                        my_indices_of_corrected_candidates[destinationindex] = localCandidateIndex;
                    }
                    

                    //compare corrected candidate with uncorrected candidate, calculate edits
                    //const int numIntsOfSequence = getEncodedNumInts2Bit(candidate_length);

                    const bool thisSequenceContainsN = d_candidateContainsN[global_candidate_index];            
                    int* const myNumEdits = d_numEditsPerCorrectedCandidate + global_candidate_index;

                    if(thisSequenceContainsN){
                        if(tgroup.thread_rank() == 0){
                            *myNumEdits = doNotUseEditsValue;
                        }
                    }else{

                        TempCorrectedSequence::Edit* const myEdits = d_editsPerCorrectedCandidate + numEditsThreshold * global_candidate_index;

                        const int maxEdits = min(candidate_length / 7, numEditsThreshold);

                        const int fullInts = candidate_length / 16;
                        const unsigned int* const encUncorrectedCandidate = d_sequencePointers.candidateSequencesData 
                                + std::size_t(global_candidate_index) * encodedSequencePitchInInts;

                        for(int i = 0; i < fullInts; i++){
                            const unsigned int encodedDataInt = encUncorrectedCandidate[i];

                            //compare with 16 bases of corrected sequence

                            for(int k = tgroup.thread_rank(); k < 16; k += tgroup.size()){
                                const int posInInt = k;
                                const int posInSequence = i * 16 + posInInt;
                                const char encodedUncorrectedNuc = getEncodedNucFromInt2Bit(encodedDataInt, posInInt);
                                const char correctedNuc = shared_correctedCandidate[posInSequence];

                                if(correctedNuc != to_nuc(encodedUncorrectedNuc)){
                                    cg::coalesced_group g = cg::coalesced_threads();

                                    int currentNumEdits = 0;
                                    if(g.thread_rank() == 0){
                                        currentNumEdits = atomicAdd(&shared_numEditsOfCandidate[groupIdInBlock], g.size());
                                    }
                                    currentNumEdits = g.shfl(currentNumEdits, 0);

                                    if(currentNumEdits + g.size() <= maxEdits){
                                        const int myEditOutputPos = g.thread_rank() + currentNumEdits;
                                        myEdits[myEditOutputPos] = TempCorrectedSequence::Edit{posInSequence, correctedNuc};
                                    }
                                }
                            }

                            tgroup.sync();

                            if(shared_numEditsOfCandidate[groupIdInBlock] > maxEdits){
                                break;
                            }
                        }

                        //process remaining positions
                        if(shared_numEditsOfCandidate[groupIdInBlock] <= maxEdits){
                            const int remainingPositions = candidate_length - 16 * fullInts;
                            if(remainingPositions > 0){
                                const unsigned int encodedDataInt = encUncorrectedCandidate[fullInts];
                                for(int posInInt = tgroup.thread_rank(); posInInt < remainingPositions; posInInt += tgroup.size()){
                                    const int posInSequence = fullInts * 16 + posInInt;
                                    const char encodedUncorrectedNuc = getEncodedNucFromInt2Bit(encodedDataInt, posInInt);
                                    const char correctedNuc = shared_correctedCandidate[posInSequence];

                                    if(correctedNuc != to_nuc(encodedUncorrectedNuc)){
                                        cg::coalesced_group g = cg::coalesced_threads();
                                        
                                        int currentNumEdits = 0;
                                        if(g.thread_rank() == 0){
                                            currentNumEdits = atomicAdd(&shared_numEditsOfCandidate[groupIdInBlock], g.size());
                                        }
                                        currentNumEdits = g.shfl(currentNumEdits, 0);

                                        if(currentNumEdits + g.size() <= maxEdits){
                                            const int myEditOutputPos = g.thread_rank() + currentNumEdits;
                                            myEdits[myEditOutputPos] = TempCorrectedSequence::Edit{posInSequence, correctedNuc};
                                        }
                                    }
                                }
                            }
                        }

                        tgroup.sync();

                        if(tgroup.thread_rank() == 0){                            
                            if(shared_numEditsOfCandidate[groupIdInBlock] <= maxEdits){
                                *myNumEdits = shared_numEditsOfCandidate[groupIdInBlock];
                            }else{
                                *myNumEdits = doNotUseEditsValue;
                            }
                        }
                    }
                    

                    tgroup.sync(); //sync before handling next candidate
                    
                    //printf("subjectIndex %d global_candidate_index %d\n", subjectIndex, global_candidate_index);
                }

            }
        }
    }







    __global__
    void constructAnchorResultsKernel(
            TempCorrectedSequence::Edit* __restrict__ d_editsPerCorrectedSubject,
            int* __restrict__ d_numEditsPerCorrectedSubject,
            int doNotUseEditsValue,
            const int* __restrict__ d_indicesOfCorrectedSubjects,
            const int* __restrict__ d_numIndicesOfCorrectedSubjects,
            const bool* __restrict__ d_readContainsN,
            const unsigned int* __restrict__ d_uncorrectedSubjects,
            const int* __restrict__ d_subjectLengths,
            const char* __restrict__ d_correctedSubjects,
            int numEditsThreshold,
            size_t encodedSequencePitchInInts,
            size_t decodedSequencePitchInBytes){

        auto get = [] (const unsigned int* data, int length, int index, auto trafo){
            return getEncodedNuc2Bit(data, length, index, trafo);
        };
        
        auto getEncodedNucFromInt2Bit = [](unsigned int data, int pos){
            return ((data >> (30 - 2*pos)) & 0x00000003);
        };

        auto to_nuc = [](char c){
            constexpr char A_enc = 0x00;
            constexpr char C_enc = 0x01;
            constexpr char G_enc = 0x02;
            constexpr char T_enc = 0x03;
            
            switch(c){
            case A_enc: return 'A';
            case C_enc: return 'C';
            case G_enc: return 'G';
            case T_enc: return 'T';
            default: return 'F';
            }
        };

        const int numIndicesToProcess = *d_numIndicesOfCorrectedSubjects;

        for(int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < numIndicesToProcess; tid += blockDim.x * gridDim.x){
            const int indexOfCorrectedSubject = d_indicesOfCorrectedSubjects[tid];

            const bool thisSequenceContainsN = d_readContainsN[indexOfCorrectedSubject];            
            int* const myNumEdits = d_numEditsPerCorrectedSubject + tid;

            if(thisSequenceContainsN){
                *myNumEdits = doNotUseEditsValue;
            }else{
                const int length = d_subjectLengths[indexOfCorrectedSubject];

                //find correct pointers
                const unsigned int* const encodedUncorrectedSequence = d_uncorrectedSubjects + encodedSequencePitchInInts * indexOfCorrectedSubject;
                const char* const decodedCorrectedSequence = d_correctedSubjects + decodedSequencePitchInBytes * indexOfCorrectedSubject;
    
                TempCorrectedSequence::Edit* const myEdits = d_editsPerCorrectedSubject + numEditsThreshold * tid;

                const int maxEdits = min(length / 7, numEditsThreshold);
                int edits = 0;
                
                for(int i = 0; i < length && edits <= maxEdits; i++){
                    const char correctedNuc = decodedCorrectedSequence[i];
                    const char uncorrectedNuc = to_nuc(get(encodedUncorrectedSequence, length, i, [](auto i){return i;}));

                    if(correctedNuc != uncorrectedNuc){
                        myEdits[edits] = TempCorrectedSequence::Edit{i, correctedNuc};
                        edits++;
                    }
                }
                if(edits <= maxEdits){
                    *myNumEdits = edits;
                }else{
                    *myNumEdits = doNotUseEditsValue;
                }
            }
        }
    }



    template<int BLOCKSIZE>
    __global__
    void msaCorrectSubjectKernelWithOrigMismatchPositions(
                            MSAPointers msapointers,
                            AlignmentResultPointers alignmentresultpointers,
                            ReadSequencesPointers d_sequencePointers,
                            CorrectionResultPointers d_correctionResultPointers,
                            const int* __restrict__ d_indices,
                            const int* __restrict__ d_indices_per_subject,
                            int n_subjects,
                            int encodedSequencePitchInInts,
                            size_t sequence_pitch,
                            size_t msa_pitch,
                            size_t msa_weights_pitch,
                            int maximumSequenceLength,
                            float estimatedErrorrate,
                            float desiredAlignmentMaxErrorRate,
                            float avg_support_threshold,
                            float min_support_threshold,
                            float min_coverage_threshold,
                            float max_coverage_threshold,
                            int k_region){

        using BlockReduceBool = hipcub::BlockReduce<bool, BLOCKSIZE>;
        using BlockReduceInt = hipcub::BlockReduce<int, BLOCKSIZE>;
        using BlockReduceFloat = hipcub::BlockReduce<float, BLOCKSIZE>;

        __shared__ union {
            typename BlockReduceBool::TempStorage boolreduce;
            typename BlockReduceInt::TempStorage intreduce;
            typename BlockReduceFloat::TempStorage floatreduce;
        } temp_storage;

        __shared__ int broadcastbuffer;

        __shared__ int numUncorrectedPositions;
        __shared__ int uncorrectedPositions[BLOCKSIZE];
        __shared__ float avgCountPerWeight[4];

        auto get = [] (const char* data, int length, int index){
            //return Sequence_t::get_as_nucleotide(data, length, index);
            return getEncodedNuc2Bit((const unsigned int*)data, length, index, [](auto i){return i;});
        };

        auto getSubjectPtr = [&] (int subjectIndex){
            const unsigned int* result = d_sequencePointers.subjectSequencesData + std::size_t(subjectIndex) * encodedSequencePitchInInts;
            return result;
        };

        auto getCandidatePtr = [&] (int candidateIndex){
            const unsigned int* result = d_sequencePointers.candidateSequencesData + std::size_t(candidateIndex) * encodedSequencePitchInInts;
            return result;
        };

        auto getCandidateLength = [&](int candidateIndex){
            return d_sequencePointers.candidateSequencesLength[candidateIndex];
        };

        auto isGoodAvgSupport = [&](float avgsupport){
            return avgsupport >= avg_support_threshold;
        };
        auto isGoodMinSupport = [&](float minsupport){
            return minsupport >= min_support_threshold;
        };
        auto isGoodMinCoverage = [&](float mincoverage){
            return mincoverage >= min_coverage_threshold;
        };

        constexpr char A_enc = 0x00;
        constexpr char C_enc = 0x01;
        constexpr char G_enc = 0x02;
        constexpr char T_enc = 0x03;

        auto to_nuc = [](char c){
            switch(c){
            case A_enc: return 'A';
            case C_enc: return 'C';
            case G_enc: return 'G';
            case T_enc: return 'T';
            default: return 'F';
            }
        };

        const size_t msa_weights_pitch_floats = msa_weights_pitch / sizeof(float);

        for(unsigned subjectIndex = blockIdx.x; subjectIndex < n_subjects; subjectIndex += gridDim.x){
            const int myNumIndices = d_indices_per_subject[subjectIndex];
            if(myNumIndices > 0){

                const float* const my_support = msapointers.support + msa_weights_pitch_floats * subjectIndex;
                const int* const my_coverage = msapointers.coverage + msa_weights_pitch_floats * subjectIndex;
                const int* const my_orig_coverage = msapointers.origCoverages + msa_weights_pitch_floats * subjectIndex;
                const char* const my_consensus = msapointers.consensus + msa_pitch  * subjectIndex;
                char* const my_corrected_subject = d_correctionResultPointers.correctedSubjects + subjectIndex * sequence_pitch;

                const int subjectColumnsBegin_incl = msapointers.msaColumnProperties[subjectIndex].subjectColumnsBegin_incl;
                const int subjectColumnsEnd_excl = msapointers.msaColumnProperties[subjectIndex].subjectColumnsEnd_excl;
                const int lastColumn_excl = msapointers.msaColumnProperties[subjectIndex].lastColumn_excl;

                float avg_support = 0;
                float min_support = 1.0f;
                //int max_coverage = 0;
                int min_coverage = std::numeric_limits<int>::max();

                for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                    assert(i < lastColumn_excl);

                    avg_support += my_support[i];
                    min_support = min(my_support[i], min_support);
                    //max_coverage = max(my_coverage[i], max_coverage);
                    min_coverage = min(my_coverage[i], min_coverage);
                }

                avg_support = BlockReduceFloat(temp_storage.floatreduce).Sum(avg_support);
                __syncthreads();

                min_support = BlockReduceFloat(temp_storage.floatreduce).Reduce(min_support, hipcub::Min());
                __syncthreads();

                //max_coverage = BlockReduceInt(temp_storage.intreduce).Reduce(max_coverage, hipcub::Max());

                min_coverage = BlockReduceInt(temp_storage.intreduce).Reduce(min_coverage, hipcub::Min());
                __syncthreads();

                avg_support /= (subjectColumnsEnd_excl - subjectColumnsBegin_incl);


                const float avg_support_threshold = 1.0f-1.0f*estimatedErrorrate;
        		const float min_support_threshold = 1.0f-3.0f*estimatedErrorrate;

                if(threadIdx.x == 0){
                    d_correctionResultPointers.subjectIsCorrected[subjectIndex] = true; //canBeCorrected;

                    const bool canBeCorrectedByConsensus = isGoodAvgSupport(avg_support) && isGoodMinSupport(min_support) && isGoodMinCoverage(min_coverage);
                    int flag = 0;

                    if(canBeCorrectedByConsensus){
                        int smallestErrorrateThatWouldMakeHQ = 100;

                        const int estimatedErrorratePercent = ceil(estimatedErrorrate * 100.0f);
                        for(int percent = estimatedErrorratePercent; percent >= 0; percent--){
                            float factor = percent / 100.0f;
                            if(avg_support >= 1.0f - 1.0f * factor && min_support >= 1.0f - 3.0f * factor){
                                smallestErrorrateThatWouldMakeHQ = percent;
                            }
                        }

                        const bool isHQ = isGoodMinCoverage(min_coverage)
                                            && smallestErrorrateThatWouldMakeHQ <= estimatedErrorratePercent * 0.5f;

                        //broadcastbuffer = isHQ;
                        d_correctionResultPointers.isHighQualitySubject[subjectIndex].hq(isHQ);

                        flag = isHQ ? 2 : 1;
                    }

                    broadcastbuffer = flag;
                }
                __syncthreads();

                // for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                //     //assert(my_consensus[i] == 'A' || my_consensus[i] == 'C' || my_consensus[i] == 'G' || my_consensus[i] == 'T');
                //     if(my_support[i] > 0.90f && my_orig_coverage[i] <= 2){
                //         my_corrected_subject[i - subjectColumnsBegin_incl] = my_consensus[i];
                //     }else{
                //         const char* subject = getSubjectPtr(subjectIndex);
                //         const char encodedBase = get(subject, subjectColumnsEnd_excl- subjectColumnsBegin_incl, i - subjectColumnsBegin_incl);
                //         const char base = to_nuc(encodedBase);
                //         my_corrected_subject[i - subjectColumnsBegin_incl] = base;
                //     }
                // }

                const int flag = broadcastbuffer;

                if(flag > 0){
                    for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                        my_corrected_subject[i - subjectColumnsBegin_incl] = my_consensus[i];
                    }
                }else{
                    //correct only positions with high support.
#if 1                    
                    const int iterations = SDIV(subjectColumnsEnd_excl-subjectColumnsBegin_incl, BLOCKSIZE);

                    for(int iter = 0; iter < iterations; iter++){
                        const int begin = iter * BLOCKSIZE;
                        const int end = min(subjectColumnsEnd_excl-subjectColumnsBegin_incl, (iter+1) * BLOCKSIZE);

                        char editBase = 'F';
                        int editPos = -1;

                        if(threadIdx.x < end - begin){
                            const int i = subjectColumnsBegin_incl + begin + threadIdx.x;

                            const unsigned int* subject = getSubjectPtr(subjectIndex);
                            const char encodedBase = get((const char*)subject, subjectColumnsEnd_excl- subjectColumnsBegin_incl, i - subjectColumnsBegin_incl);
                            const char base = to_nuc(encodedBase);

                            if(my_support[i] > 0.90f && my_orig_coverage[i] <= 2){
                                editBase = my_consensus[i];
                                my_corrected_subject[i - subjectColumnsBegin_incl] = editBase;
                                if(editBase != base){
                                    editPos = i - subjectColumnsBegin_incl;
                                }
                            }else{                                
                                my_corrected_subject[i - subjectColumnsBegin_incl] = base;
                            }
                        }
                    }
#else 

                    for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                        //assert(my_consensus[i] == 'A' || my_consensus[i] == 'C' || my_consensus[i] == 'G' || my_consensus[i] == 'T');
                        if(my_support[i] > 0.90f && my_orig_coverage[i] <= 2){
                            my_corrected_subject[i - subjectColumnsBegin_incl] = my_consensus[i];
                        }else{
                            const unsigned int* subject = getSubjectPtr(subjectIndex);
                            const char encodedBase = get((const char*)subject, subjectColumnsEnd_excl- subjectColumnsBegin_incl, i - subjectColumnsBegin_incl);
                            const char base = to_nuc(encodedBase);
                            my_corrected_subject[i - subjectColumnsBegin_incl] = base;
                        }
                    }


#endif
                }
            }else{
                if(threadIdx.x == 0){
                    d_correctionResultPointers.isHighQualitySubject[subjectIndex].hq(false);
                    d_correctionResultPointers.subjectIsCorrected[subjectIndex] = false;
                }
            }
        }
    }










    //####################   KERNEL DISPATCH   ####################


    void call_popcount_shifted_hamming_distance_kernel_async(
    			AlignmentResultPointers d_alignmentresultpointers,
                ReadSequencesPointers d_sequencePointers,
    			const int* d_candidates_per_subject_prefixsum,
                const int* h_candidates_per_subject,
                const int* d_candidates_per_subject,
                const int* d_anchorIndicesOfCandidates,
    			int n_subjects,
    			int n_queries,
                int maximumSequenceLength,
                int encodedSequencePitchInInts2Bit,
    			int min_overlap,
    			float maxErrorRate,
                float min_overlap_ratio,
                float estimatedNucleotideErrorRate,
    			hipStream_t stream,
    			KernelLaunchHandle& handle){

            const int intsPerSequence2BitHiLo = getEncodedNumInts2BitHiLo(maximumSequenceLength);
            const int bytesPerSequence2BitHilo = intsPerSequence2BitHiLo * sizeof(unsigned int);

            unsigned int* d_subjectDataHiLoTransposed = nullptr;
            unsigned int* d_candidateDataHiLoTransposed = nullptr;

            cubCachingAllocator.DeviceAllocate(
                (void**)&d_candidateDataHiLoTransposed, 
                sizeof(unsigned int) * intsPerSequence2BitHiLo * n_queries, 
                stream
            ); CUERR;

            callConversionKernel2BitTo2BitHiLoNT(
                d_sequencePointers.candidateSequencesData,
                encodedSequencePitchInInts2Bit,
                d_candidateDataHiLoTransposed,
                intsPerSequence2BitHiLo,
                d_sequencePointers.candidateSequencesLength,
                n_queries,
                stream,
                handle
            );
            
            if(maximumSequenceLength){
                cubCachingAllocator.DeviceAllocate(
                    (void**)&d_subjectDataHiLoTransposed, 
                    sizeof(unsigned int) * intsPerSequence2BitHiLo * n_subjects, 
                    stream
                ); CUERR;

                callConversionKernel2BitTo2BitHiLoNT(
                    d_sequencePointers.subjectSequencesData,
                    encodedSequencePitchInInts2Bit,
                    d_subjectDataHiLoTransposed,
                    intsPerSequence2BitHiLo,
                    d_sequencePointers.subjectSequencesLength,
                    n_subjects,
                    stream,
                    handle
                );
            }
            

            if(maximumSequenceLength <= 128){
               
                constexpr int maxValidIntsPerSequence = 8;

                constexpr int blocksize = 128;
                int max_blocks_per_device = 1;

                KernelLaunchConfig kernelLaunchConfig;
                kernelLaunchConfig.threads_per_block = blocksize;
                kernelLaunchConfig.smem = 0;

                auto iter = handle.kernelPropertiesMap.find(KernelId::PopcountSHDTiledPitch8);
                if(iter == handle.kernelPropertiesMap.end()) {

                    std::map<KernelLaunchConfig, KernelProperties> mymap;

                    KernelProperties kernelProperties;
                    hipOccupancyMaxActiveBlocksPerMultiprocessor(
                        &kernelProperties.max_blocks_per_SM,
                        popcount_shifted_hamming_distance_ctpitch_kernel<blocksize, maxValidIntsPerSequence>,
                        kernelLaunchConfig.threads_per_block, 
                        kernelLaunchConfig.smem
                    ); CUERR;

                    mymap[kernelLaunchConfig] = kernelProperties;
                    max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

                    handle.kernelPropertiesMap[KernelId::PopcountSHDTiledPitch8] = std::move(mymap);
                }else{
                    std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
                    const KernelProperties& kernelProperties = map[kernelLaunchConfig];
                    max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
                }

                dim3 block(blocksize, 1, 1);
                const int numBlocks = SDIV(n_queries, blocksize);
                dim3 grid(std::min(numBlocks, max_blocks_per_device), 1, 1);

                popcount_shifted_hamming_distance_ctpitch_kernel<blocksize, maxValidIntsPerSequence>
                    <<<grid, block, 0, stream>>>(
                        d_subjectDataHiLoTransposed,
                        d_candidateDataHiLoTransposed,
                        d_sequencePointers.subjectSequencesLength,
                        d_sequencePointers.candidateSequencesLength,
                        d_alignmentresultpointers.bestAlignmentFlags,
                        d_alignmentresultpointers.scores,
                        d_alignmentresultpointers.overlaps,
                        d_alignmentresultpointers.shifts,
                        d_alignmentresultpointers.nOps,
                        d_alignmentresultpointers.isValid,
                        d_anchorIndicesOfCandidates,
                        n_subjects,
                        n_queries,
                        intsPerSequence2BitHiLo, 
                        min_overlap,
                        maxErrorRate,
                        min_overlap_ratio,
                        estimatedNucleotideErrorRate
                ); CUERR;

                cubCachingAllocator.DeviceFree(d_subjectDataHiLoTransposed);  CUERR;

            }else if(maximumSequenceLength <= 256){
               
                constexpr int maxValidIntsPerSequence = 16;

                constexpr int blocksize = 128;
                int max_blocks_per_device = 1;

                KernelLaunchConfig kernelLaunchConfig;
                kernelLaunchConfig.threads_per_block = blocksize;
                kernelLaunchConfig.smem = 0;

                auto iter = handle.kernelPropertiesMap.find(KernelId::PopcountSHDTiledPitch8);
                if(iter == handle.kernelPropertiesMap.end()) {

                    std::map<KernelLaunchConfig, KernelProperties> mymap;

                    KernelProperties kernelProperties;
                    hipOccupancyMaxActiveBlocksPerMultiprocessor(
                        &kernelProperties.max_blocks_per_SM,
                        popcount_shifted_hamming_distance_ctpitch_kernel<blocksize, maxValidIntsPerSequence>,
                        kernelLaunchConfig.threads_per_block, 
                        kernelLaunchConfig.smem
                    ); CUERR;

                    mymap[kernelLaunchConfig] = kernelProperties;
                    max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

                    handle.kernelPropertiesMap[KernelId::PopcountSHDTiledPitch8] = std::move(mymap);
                }else{
                    std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
                    const KernelProperties& kernelProperties = map[kernelLaunchConfig];
                    max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
                }

                dim3 block(blocksize, 1, 1);
                const int numBlocks = SDIV(n_queries, blocksize);
                dim3 grid(std::min(numBlocks, max_blocks_per_device), 1, 1);

                popcount_shifted_hamming_distance_ctpitch_kernel<blocksize, maxValidIntsPerSequence>
                    <<<grid, block, 0, stream>>>(
                        d_subjectDataHiLoTransposed,
                        d_candidateDataHiLoTransposed,
                        d_sequencePointers.subjectSequencesLength,
                        d_sequencePointers.candidateSequencesLength,
                        d_alignmentresultpointers.bestAlignmentFlags,
                        d_alignmentresultpointers.scores,
                        d_alignmentresultpointers.overlaps,
                        d_alignmentresultpointers.shifts,
                        d_alignmentresultpointers.nOps,
                        d_alignmentresultpointers.isValid,
                        d_anchorIndicesOfCandidates,
                        n_subjects,
                        n_queries,
                        intsPerSequence2BitHiLo, 
                        min_overlap,
                        maxErrorRate,
                        min_overlap_ratio,
                        estimatedNucleotideErrorRate
                ); CUERR;

                cubCachingAllocator.DeviceFree(d_subjectDataHiLoTransposed);  CUERR;

            }else{

                unsigned int* d_subjectDataHiLo = nullptr;

                cubCachingAllocator.DeviceAllocate(
                    (void**)&d_subjectDataHiLo, 
                    sizeof(unsigned int) * intsPerSequence2BitHiLo * n_subjects, 
                    stream
                ); CUERR;

                callConversionKernel2BitTo2BitHiLoNN(
                    d_sequencePointers.subjectSequencesData,
                    encodedSequencePitchInInts2Bit,
                    d_subjectDataHiLo,
                    intsPerSequence2BitHiLo,
                    d_sequencePointers.subjectSequencesLength,
                    n_subjects,
                    stream,
                    handle
                );

                constexpr int tilesize = 16;

                int* d_tiles_per_subject_prefixsum;
                cubCachingAllocator.DeviceAllocate((void**)&d_tiles_per_subject_prefixsum, sizeof(int) * (n_subjects+1), stream);  CUERR;

                // calculate blocks per subject prefixsum
                auto getTilesPerSubject = [=] __device__ (int candidates_for_subject){
                    return SDIV(candidates_for_subject, tilesize);
                };
                hipcub::TransformInputIterator<int,decltype(getTilesPerSubject), const int*>
                    d_tiles_per_subject(d_candidates_per_subject,
                                getTilesPerSubject);

                void* tempstorage = nullptr;
                size_t tempstoragesize = 0;

                hipcub::DeviceScan::InclusiveSum(nullptr,
                            tempstoragesize,
                            d_tiles_per_subject,
                            d_tiles_per_subject_prefixsum+1,
                            n_subjects,
                            stream); CUERR;

                cubCachingAllocator.DeviceAllocate((void**)&tempstorage, tempstoragesize, stream);  CUERR;

                hipcub::DeviceScan::InclusiveSum(tempstorage,
                            tempstoragesize,
                            d_tiles_per_subject,
                            d_tiles_per_subject_prefixsum+1,
                            n_subjects,
                            stream); CUERR;

                cubCachingAllocator.DeviceFree(tempstorage);  CUERR;

                call_set_kernel_async(d_tiles_per_subject_prefixsum,
                                        0,
                                        0,
                                        stream);




                constexpr int blocksize = 128;
                constexpr int tilesPerBlock = blocksize / tilesize;

                //const int requiredTiles = h_tiles_per_subject_prefixsum[n_subjects];

                int requiredTiles = 0;
                for(int i = 0; i < n_subjects;i++){
                    requiredTiles += SDIV(h_candidates_per_subject[i], tilesize);
                }

                const int requiredBlocks = SDIV(requiredTiles, tilesPerBlock);

                //printf("n_subjects %d, n_queries %d\n", n_subjects, n_queries);


                const std::size_t smem = sizeof(char) * (bytesPerSequence2BitHilo * tilesPerBlock + bytesPerSequence2BitHilo * blocksize * 2);

                int max_blocks_per_device = 1;

                KernelLaunchConfig kernelLaunchConfig;
                kernelLaunchConfig.threads_per_block = blocksize;
                kernelLaunchConfig.smem = smem;

                auto iter = handle.kernelPropertiesMap.find(KernelId::PopcountSHDTiled);
                if(iter == handle.kernelPropertiesMap.end()) {

                    std::map<KernelLaunchConfig, KernelProperties> mymap;

                    #define getProp(blocksize, tilesize) { \
                            KernelLaunchConfig kernelLaunchConfig; \
                            kernelLaunchConfig.threads_per_block = (blocksize); \
                            kernelLaunchConfig.smem = sizeof(char) * (bytesPerSequence2BitHilo * tilesPerBlock + bytesPerSequence2BitHilo * blocksize * 2); \
                            KernelProperties kernelProperties; \
                            hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                                popcount_shifted_hamming_distance_kernel<tilesize>, \
                                        kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
                            mymap[kernelLaunchConfig] = kernelProperties; \
                    }
                    getProp(1, tilesize);
                    getProp(32, tilesize);
                    getProp(64, tilesize);
                    getProp(96, tilesize);
                    getProp(128, tilesize);
                    getProp(160, tilesize);
                    getProp(192, tilesize);
                    getProp(224, tilesize);
                    getProp(256, tilesize);

                    const auto& kernelProperties = mymap[kernelLaunchConfig];
                    max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

                    handle.kernelPropertiesMap[KernelId::PopcountSHDTiled] = std::move(mymap);

                    #undef getProp
                }else{
                    std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
                    const KernelProperties& kernelProperties = map[kernelLaunchConfig];
                    max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
                }

                #define mycall popcount_shifted_hamming_distance_kernel<tilesize> \
                                                    <<<grid, block, smem, stream>>>( \
                                                    d_subjectDataHiLo, \
                                                    d_candidateDataHiLoTransposed, \
                                                    d_alignmentresultpointers, \
                                                    d_sequencePointers, \
                                                    d_candidates_per_subject_prefixsum, \
                                                    d_tiles_per_subject_prefixsum, \
                                                    n_subjects, \
                                                    n_queries, \
                                                    intsPerSequence2BitHiLo, \
                                                    min_overlap, \
                                                    maxErrorRate, \
                                                    min_overlap_ratio, \
                                                    estimatedNucleotideErrorRate); CUERR;

                dim3 block(blocksize, 1, 1);
                dim3 grid(std::min(requiredBlocks, max_blocks_per_device), 1, 1);

                mycall;

                #undef mycall

                cubCachingAllocator.DeviceFree(d_tiles_per_subject_prefixsum);  CUERR;

                cubCachingAllocator.DeviceFree(d_subjectDataHiLo);  CUERR;
            }

            
            
            cubCachingAllocator.DeviceFree(d_candidateDataHiLoTransposed);  CUERR;

            // hipDeviceSynchronize();
            // std::exit(0);
    }


    void call_cuda_find_best_alignment_kernel_async_exp(
                AlignmentResultPointers d_alignmentresultpointers,
                ReadSequencesPointers d_sequencePointers,
    			const int* d_candidates_per_subject_prefixsum,
    			int n_subjects,
    			int n_queries,
    			float min_overlap_ratio,
    			int min_overlap,
                float estimatedErrorrate,
    			hipStream_t stream,
                KernelLaunchHandle& handle,
                read_number debugsubjectreadid){

    	const int blocksize = 128;
    	const std::size_t smem = 0;

    	int max_blocks_per_device = 1;

    	KernelLaunchConfig kernelLaunchConfig;
    	kernelLaunchConfig.threads_per_block = blocksize;
    	kernelLaunchConfig.smem = smem;

    	auto iter = handle.kernelPropertiesMap.find(KernelId::FindBestAlignmentExp);
    	if(iter == handle.kernelPropertiesMap.end()) {

    		std::map<KernelLaunchConfig, KernelProperties> mymap;

    	    #define getProp(blocksize) { \
                KernelLaunchConfig kernelLaunchConfig; \
                kernelLaunchConfig.threads_per_block = (blocksize); \
                kernelLaunchConfig.smem = 0; \
                KernelProperties kernelProperties; \
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                            cuda_find_best_alignment_kernel_exp, \
                            kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
                mymap[kernelLaunchConfig] = kernelProperties; }

    		getProp(32);
    		getProp(64);
    		getProp(96);
    		getProp(128);
    		getProp(160);
    		getProp(192);
    		getProp(224);
    		getProp(256);

    		const auto& kernelProperties = mymap[kernelLaunchConfig];
    		max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

    		handle.kernelPropertiesMap[KernelId::FindBestAlignmentExp] = std::move(mymap);

    	    #undef getProp
    	}else{
    		std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
    		const KernelProperties& kernelProperties = map[kernelLaunchConfig];
    		max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
    	}

    	dim3 block(blocksize,1,1);
    	dim3 grid(std::min(max_blocks_per_device, SDIV(n_queries, blocksize)), 1, 1);

    	cuda_find_best_alignment_kernel_exp<<<grid, block, smem, stream>>>(
            		d_alignmentresultpointers,
                    d_sequencePointers,
            		d_candidates_per_subject_prefixsum,
            		n_subjects,
            		n_queries,
            		min_overlap_ratio,
            		min_overlap,
                    estimatedErrorrate,
                    debugsubjectreadid); CUERR;

    }


    void call_cuda_filter_alignments_by_mismatchratio_kernel_async(
    			AlignmentResultPointers d_alignmentresultpointers,
    			const int* d_candidates_per_subject_prefixsum,
    			int n_subjects,
    			int n_candidates,
    			float mismatchratioBaseFactor,
    			float goodAlignmentsCountThreshold,
    			hipStream_t stream,
    			KernelLaunchHandle& handle){

    	const int blocksize = 128;
    	const std::size_t smem = 0;

    	int max_blocks_per_device = 1;

    	KernelLaunchConfig kernelLaunchConfig;
    	kernelLaunchConfig.threads_per_block = blocksize;
    	kernelLaunchConfig.smem = smem;

    	auto iter = handle.kernelPropertiesMap.find(KernelId::FilterAlignmentsByMismatchRatio);
    	if(iter == handle.kernelPropertiesMap.end()) {

    		std::map<KernelLaunchConfig, KernelProperties> mymap;

    	    #define getProp(blocksize) { \
    		KernelLaunchConfig kernelLaunchConfig; \
    		kernelLaunchConfig.threads_per_block = (blocksize); \
    		kernelLaunchConfig.smem = 0; \
    		KernelProperties kernelProperties; \
    		hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
    					cuda_filter_alignments_by_mismatchratio_kernel<(blocksize)>, \
    					kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
    		mymap[kernelLaunchConfig] = kernelProperties; \
    }

    		getProp(32);
    		getProp(64);
    		getProp(96);
    		getProp(128);
    		getProp(160);
    		getProp(192);
    		getProp(224);
    		getProp(256);

    		const auto& kernelProperties = mymap[kernelLaunchConfig];
    		max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

    		handle.kernelPropertiesMap[KernelId::FilterAlignmentsByMismatchRatio] = std::move(mymap);

    	    #undef getProp
    	}else{
    		std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
    		const KernelProperties& kernelProperties = map[kernelLaunchConfig];
    		max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
    	}

    	dim3 block(blocksize, 1, 1);
    	dim3 grid(std::min(max_blocks_per_device, n_subjects));

    	#define mycall(blocksize) cuda_filter_alignments_by_mismatchratio_kernel<(blocksize)> \
    	        <<<grid, block, smem, stream>>>( \
    		d_alignmentresultpointers, \
    		d_candidates_per_subject_prefixsum, \
    		n_subjects, \
    		n_candidates, \
    		mismatchratioBaseFactor, \
    		goodAlignmentsCountThreshold); CUERR;

    	switch(blocksize) {
    	case 32: mycall(32); break;
    	case 64: mycall(64); break;
    	case 96: mycall(96); break;
    	case 128: mycall(128); break;
    	case 160: mycall(160); break;
    	case 192: mycall(192); break;
    	case 224: mycall(224); break;
    	case 256: mycall(256); break;
    	default: mycall(256); break;
    	}

    		#undef mycall
    }






    


    void callSelectIndicesOfGoodCandidatesKernelAsync(
            int* d_indicesOfGoodCandidates,
            int* d_numIndicesPerAnchor,
            int* d_totalNumIndices,
            const BestAlignment_t* d_alignmentFlags,
            const int* d_candidates_per_subject,
            const int* d_candidates_per_subject_prefixsum,
            const int* d_anchorIndicesOfCandidates,
            int numAnchors,
            int numCandidates,
            hipStream_t stream,
            KernelLaunchHandle& handle){

        constexpr int blocksize = 128;
        constexpr int tilesize = 32;

        const std::size_t smem = 0;

        int max_blocks_per_device = 1;

        KernelLaunchConfig kernelLaunchConfig;
        kernelLaunchConfig.threads_per_block = blocksize;
        kernelLaunchConfig.smem = smem;

        auto iter = handle.kernelPropertiesMap.find(KernelId::SelectIndicesOfGoodCandidates);
        if(iter == handle.kernelPropertiesMap.end()){

            std::map<KernelLaunchConfig, KernelProperties> mymap;

            #define getProp(blocksize) { \
                KernelLaunchConfig kernelLaunchConfig; \
                kernelLaunchConfig.threads_per_block = (blocksize); \
                kernelLaunchConfig.smem = 0; \
                KernelProperties kernelProperties; \
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                    selectIndicesOfGoodCandidatesKernel<(blocksize), tilesize>, \
                                                                kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
                mymap[kernelLaunchConfig] = kernelProperties; \
            }

            getProp(32);
            getProp(64);
            getProp(96);
            getProp(128);
            getProp(160);
            getProp(192);
            getProp(224);
            getProp(256);

            const auto& kernelProperties = mymap[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

            handle.kernelPropertiesMap[KernelId::SelectIndicesOfGoodCandidates] = std::move(mymap);

            #undef getProp
        }else{
            std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
            const KernelProperties& kernelProperties = map[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
        }

        hipMemsetAsync(d_numIndicesPerAnchor, 0, numAnchors * sizeof(int), stream); CUERR;
        hipMemsetAsync(d_totalNumIndices, 0, sizeof(int), stream); CUERR;

        dim3 block(blocksize, 1, 1);
        dim3 grid(std::min(SDIV(numCandidates, blocksize), max_blocks_per_device));

        selectIndicesOfGoodCandidatesKernel<blocksize, tilesize><<<grid, block, 0, stream>>>(
            d_indicesOfGoodCandidates,
            d_numIndicesPerAnchor,
            d_totalNumIndices,
            d_alignmentFlags,
            d_candidates_per_subject,
            d_candidates_per_subject_prefixsum,
            d_anchorIndicesOfCandidates,
            numAnchors,
            numCandidates
        );
    }











    void call_msa_correct_subject_implicit_kernel_async(
                            MSAPointers d_msapointers,
                            AlignmentResultPointers d_alignmentresultpointers,
                            ReadSequencesPointers d_sequencePointers,
                            CorrectionResultPointers d_correctionResultPointers,
                            const int* d_indices,
                            const int* d_indices_per_subject,
                            int n_subjects,
                            int encodedSequencePitchInInts,
                            size_t sequence_pitch,
                            size_t msa_pitch,
                            size_t msa_weights_pitch,
                            int maximumSequenceLength,
                            float estimatedErrorrate,
                            float desiredAlignmentMaxErrorRate,
                            float avg_support_threshold,
                            float min_support_threshold,
                            float min_coverage_threshold,
                            float max_coverage_threshold,
                            int k_region,
                            int maximum_sequence_length,
                            hipStream_t stream,
                            KernelLaunchHandle& handle){

        const int max_block_size = 256;
        const int blocksize = std::min(max_block_size, SDIV(maximum_sequence_length, 32) * 32);
        const std::size_t smem = 0;

        int max_blocks_per_device = 1;

        KernelLaunchConfig kernelLaunchConfig;
        kernelLaunchConfig.threads_per_block = blocksize;
        kernelLaunchConfig.smem = smem;

        auto iter = handle.kernelPropertiesMap.find(KernelId::MSACorrectSubjectImplicit);
        if(iter == handle.kernelPropertiesMap.end()){

            std::map<KernelLaunchConfig, KernelProperties> mymap;

            #define getProp(blocksize) { \
                KernelLaunchConfig kernelLaunchConfig; \
                kernelLaunchConfig.threads_per_block = (blocksize); \
                kernelLaunchConfig.smem = 0; \
                KernelProperties kernelProperties; \
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                    msa_correct_subject_implicit_kernel2<(blocksize)>, \
                                                                kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
                mymap[kernelLaunchConfig] = kernelProperties; \
            }

            getProp(32);
            getProp(64);
            getProp(96);
            getProp(128);
            getProp(160);
            getProp(192);
            getProp(224);
            getProp(256);

            const auto& kernelProperties = mymap[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

            handle.kernelPropertiesMap[KernelId::MSACorrectSubjectImplicit] = std::move(mymap);

            #undef getProp
        }else{
            std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
            const KernelProperties& kernelProperties = map[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
        }

        hipMemsetAsync(d_correctionResultPointers.isHighQualitySubject, 0, n_subjects * sizeof(AnchorHighQualityFlag), stream); CUERR;

        dim3 block(blocksize, 1, 1);
        dim3 grid(std::min(n_subjects, max_blocks_per_device));

        #define mycall(blocksize) msa_correct_subject_implicit_kernel2<(blocksize)> \
                                <<<grid, block, 0, stream>>>( \
                                    d_msapointers, \
                                    d_alignmentresultpointers, \
                                    d_sequencePointers, \
                                    d_correctionResultPointers, \
                                    d_indices_per_subject, \
                                    n_subjects, \
                                    encodedSequencePitchInInts, \
                                    sequence_pitch, \
                                    msa_pitch, \
                                    msa_weights_pitch, \
                                    maximumSequenceLength, \
                                    estimatedErrorrate, \
                                    desiredAlignmentMaxErrorRate, \
                                    avg_support_threshold, \
                                    min_support_threshold, \
                                    min_coverage_threshold, \
                                    max_coverage_threshold, \
                                    k_region); CUERR;

        assert(blocksize > 0 && blocksize <= max_block_size);

        switch(blocksize){
            case 32: mycall(32); break;
            case 64: mycall(64); break;
            case 96: mycall(96); break;
            case 128: mycall(128); break;
            case 160: mycall(160); break;
            case 192: mycall(192); break;
            case 224: mycall(224); break;
            case 256: mycall(256); break;
            default: mycall(256); break;
        }
        #undef mycall
    }




    void call_msa_correct_candidates_kernel_async(
                MSAPointers d_msapointers,
                AlignmentResultPointers d_alignmentresultpointers,
                ReadSequencesPointers d_sequencePointers,
                CorrectionResultPointers d_correctionResultPointers,
    			const int* d_indices,
    			const int* d_indices_per_subject,
    			const int* d_candidates_per_subject_prefixsum,
    			int n_subjects,
    			int n_queries,
    			const int* d_num_indices,
                int encodedSequencePitchInInts,
    			size_t sequence_pitch,
    			size_t msa_pitch,
    			size_t msa_weights_pitch,
    			float min_support_threshold,
    			float min_coverage_threshold,
    			int new_columns_to_correct,
    			int maximum_sequence_length,
    			hipStream_t stream,
    			KernelLaunchHandle& handle){

        //constexpr int tilesize = 32;
        const int max_block_size = 256;
        constexpr int blocksize = 64;// std::min(max_block_size, SDIV(maximum_sequence_length, 32) * 32);

        const int* d_highQualitySubjectIndices =  d_correctionResultPointers.highQualitySubjectIndices;

        auto getCandidatesPerHQAnchor = [=] __device__(int hqIndex){
            const int subjectIndex = d_highQualitySubjectIndices[hqIndex];
            return d_indices_per_subject[subjectIndex];
        };

        // auto getTilesPerHQAnchor = [=] __device__ (int hqIndex){
        //     const int numCandidatesOfAnchor = getCandidatesPerHQAnchor(hqIndex);
        //     return SDIV(numCandidatesOfAnchor, tilesize);
        // };

        using CperHQA_t = decltype(getCandidatesPerHQAnchor);
        //using TperHQA_t = decltype(getTilesPerHQAnchor);
        using CountIt = hipcub::CountingInputIterator<int>;

        void* tempstorage = nullptr;
        size_t tempstoragesize = 0;

        //const int numHQSubjects = *(h_correctionResultPointers.numHighQualitySubjectIndices);


        //make prefixsum of number of candidates per high quality subject
        int* d_candidatesPerHQAnchorPrefixSum = nullptr;
        cubCachingAllocator.DeviceAllocate((void**)&d_candidatesPerHQAnchorPrefixSum, sizeof(int) * (n_subjects+1), stream);  CUERR;

        hipcub::TransformInputIterator<int, CperHQA_t, CountIt> transformIter(CountIt{0}, getCandidatesPerHQAnchor);

        //calculate prefixsum of candidatesPerHQAnchor. 
        //only the first d_correctionResultPointersnumHighQualitySubjectIndices+1 entries will contain valid data.
        hipcub::DeviceScan::InclusiveSum(nullptr, tempstoragesize, transformIter, d_candidatesPerHQAnchorPrefixSum+1, n_subjects, stream);
        cubCachingAllocator.DeviceAllocate((void**)&tempstorage, tempstoragesize, stream);  CUERR;
        hipcub::DeviceScan::InclusiveSum(tempstorage, tempstoragesize, transformIter, d_candidatesPerHQAnchorPrefixSum+1, n_subjects, stream);
        cubCachingAllocator.DeviceFree(tempstorage);  CUERR;

        call_set_kernel_async(d_candidatesPerHQAnchorPrefixSum, 0, 0, stream);

        // set number of corrected candidates per subject to 0
        hipMemsetAsync(d_correctionResultPointers.numCorrectedCandidates, 0, sizeof(int) * n_subjects, stream); CUERR;

    	const std::size_t smem = 0;

    	int max_blocks_per_device = 1;

    	KernelLaunchConfig kernelLaunchConfig;
    	kernelLaunchConfig.threads_per_block = blocksize;
    	kernelLaunchConfig.smem = smem;

    	auto iter = handle.kernelPropertiesMap.find(KernelId::MSACorrectCandidates);
    	if(iter == handle.kernelPropertiesMap.end()) {

    		std::map<KernelLaunchConfig, KernelProperties> mymap;

    	    #define getProp(blocksize) { \
    		KernelLaunchConfig kernelLaunchConfig; \
    		kernelLaunchConfig.threads_per_block = (blocksize); \
    		kernelLaunchConfig.smem = 0; \
    		KernelProperties kernelProperties; \
    		hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                        msa_correct_candidates_kernel<(blocksize)>, \
    					kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
    		mymap[kernelLaunchConfig] = kernelProperties; \
    }

    		getProp(32);
    		getProp(64);
    		getProp(96);
    		getProp(128);
    		getProp(160);
    		getProp(192);
    		getProp(224);
    		getProp(256);

    		const auto& kernelProperties = mymap[kernelLaunchConfig];
    		max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

    		handle.kernelPropertiesMap[KernelId::MSACorrectCandidates] = std::move(mymap);

    	    #undef getProp
    	}else{
    		std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
    		const KernelProperties& kernelProperties = map[kernelLaunchConfig];
    		max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
    	}

    	dim3 block(blocksize, 1, 1);
    	dim3 grid(std::min(max_blocks_per_device, n_subjects));

    		#define mycall(blocksize) msa_correct_candidates_kernel<(blocksize)> \
    	        <<<grid, block, 0, stream>>>( \
            d_msapointers, \
            d_alignmentresultpointers, \
            d_sequencePointers, \
            d_correctionResultPointers, \
    		d_indices, \
    		d_indices_per_subject, \
    		d_candidates_per_subject_prefixsum, \
            d_candidatesPerHQAnchorPrefixSum, \
    		n_subjects, \
    		n_queries, \
    		d_num_indices, \
            encodedSequencePitchInInts, \
    		sequence_pitch, \
    		msa_pitch, \
            msa_weights_pitch, \
    		min_support_threshold, \
    		min_coverage_threshold, \
    		new_columns_to_correct); CUERR;

    	assert(blocksize > 0 && blocksize <= max_block_size);

    	switch(blocksize) {
    	case 32: mycall(32); break;
    	case 64: mycall(64); break;
    	case 96: mycall(96); break;
    	case 128: mycall(128); break;
    	case 160: mycall(160); break;
    	case 192: mycall(192); break;
    	case 224: mycall(224); break;
    	case 256: mycall(256); break;
    	default: mycall(256); break;
    	}

    		#undef mycall

        cubCachingAllocator.DeviceFree(d_candidatesPerHQAnchorPrefixSum);  CUERR;
    }




    void callCorrectCandidatesWithGroupKernel_async(
                MSAPointers d_msapointers,
                AlignmentResultPointers d_alignmentresultpointers,
                ReadSequencesPointers d_sequencePointers,
                CorrectionResultPointers d_correctionResultPointers,
                int* d_numTotalCorrectedCandidates,
                TempCorrectedSequence::Edit* __restrict__ d_editsPerCorrectedCandidate,
                int* __restrict__ d_numEditsPerCorrectedCandidate,
                const bool* __restrict__ d_candidateContainsN,
                int doNotUseEditsValue,
                int numEditsThreshold,
    			const int* d_indices,
    			const int* d_indices_per_subject,
    			const int* d_candidates_per_subject_prefixsum,
    			int n_subjects,
    			int n_queries,
    			const int* d_num_indices,
                int encodedSequencePitchInInts,
    			size_t sequence_pitch,
    			size_t msa_pitch,
    			size_t msa_weights_pitch,
    			float min_support_threshold,
    			float min_coverage_threshold,
    			int new_columns_to_correct,
    			int maximum_sequence_length,
    			hipStream_t stream,
    			KernelLaunchHandle& handle){

        const int* d_highQualitySubjectIndices =  d_correctionResultPointers.highQualitySubjectIndices;

        auto getNumGoodCandidatesPerHQAnchor = [=] __device__(int hqIndex){
            const int subjectIndex = d_highQualitySubjectIndices[hqIndex];
            return d_indices_per_subject[subjectIndex];
        };

        // auto getTilesPerHQAnchor = [=] __device__ (int hqIndex){
        //     const int numCandidatesOfAnchor = getNumGoodCandidatesPerHQAnchor(hqIndex);
        //     return SDIV(numCandidatesOfAnchor, tilesize);
        // };

        using CperHQA_t = decltype(getNumGoodCandidatesPerHQAnchor);
        //using TperHQA_t = decltype(getTilesPerHQAnchor);
        using CountIt = hipcub::CountingInputIterator<int>;

        void* tempstorage = nullptr;
        size_t tempstoragesize = 0;

        //const int numHQSubjects = *(h_correctionResultPointers.numHighQualitySubjectIndices);


        //make prefixsum of number of candidates per high quality subject
        int* d_candidatesPerHQAnchorPrefixSum = nullptr;
        cubCachingAllocator.DeviceAllocate((void**)&d_candidatesPerHQAnchorPrefixSum, sizeof(int) * (n_subjects+1), stream);  CUERR;

        hipcub::TransformInputIterator<int, CperHQA_t, CountIt> transformIter(CountIt{0}, getNumGoodCandidatesPerHQAnchor);

        //calculate prefixsum of candidatesPerHQAnchor. 
        //only the first d_correctionResultPointersnumHighQualitySubjectIndices+1 entries will contain valid data.
        hipcub::DeviceScan::InclusiveSum(nullptr, tempstoragesize, transformIter, d_candidatesPerHQAnchorPrefixSum+1, n_subjects, stream);
        cubCachingAllocator.DeviceAllocate((void**)&tempstorage, tempstoragesize, stream);  CUERR; //released at end of function
        hipcub::DeviceScan::InclusiveSum(tempstorage, tempstoragesize, transformIter, d_candidatesPerHQAnchorPrefixSum+1, n_subjects, stream);
        cubCachingAllocator.DeviceFree(tempstorage);  CUERR;

        call_set_kernel_async(d_candidatesPerHQAnchorPrefixSum, 0, 0, stream);

        // set number of corrected candidates per subject to 0
        hipMemsetAsync(d_correctionResultPointers.numCorrectedCandidates, 0, sizeof(int) * n_subjects, stream); CUERR;


        constexpr int blocksize = 128;
        constexpr int groupsize = 32;
        constexpr int numGroupsPerBlock = blocksize / groupsize;

        const size_t dynamicsmemPitchInInts = SDIV(maximum_sequence_length, sizeof(int));
    	const std::size_t smem = numGroupsPerBlock * sizeof(int) * dynamicsmemPitchInInts;

    	int max_blocks_per_device = 1;

    	KernelLaunchConfig kernelLaunchConfig;
    	kernelLaunchConfig.threads_per_block = blocksize;
    	kernelLaunchConfig.smem = smem;

    	auto iter = handle.kernelPropertiesMap.find(KernelId::MSACorrectCandidates);
    	if(iter == handle.kernelPropertiesMap.end()) {

    		std::map<KernelLaunchConfig, KernelProperties> mymap;

    	    #define getProp(blocksize) { \
                KernelLaunchConfig kernelLaunchConfig; \
                kernelLaunchConfig.threads_per_block = (blocksize); \
                kernelLaunchConfig.smem = numGroupsPerBlock * sizeof(char) * (SDIV(maximum_sequence_length, 4) * 4); \
                KernelProperties kernelProperties; \
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                            msa_correct_candidates_with_group_kernel<(blocksize), groupsize>, \
                            kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
                mymap[kernelLaunchConfig] = kernelProperties; \
            }

    		getProp(32);
    		getProp(64);
    		getProp(96);
    		getProp(128);
    		getProp(160);
    		getProp(192);
    		getProp(224);
    		getProp(256);

    		const auto& kernelProperties = mymap[kernelLaunchConfig];
    		max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

    		handle.kernelPropertiesMap[KernelId::MSACorrectCandidates] = std::move(mymap);

    	    #undef getProp
    	}else{
    		std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
    		const KernelProperties& kernelProperties = map[kernelLaunchConfig];
    		max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
    	}

    	dim3 block(blocksize, 1, 1);
        dim3 grid(std::min(max_blocks_per_device, n_subjects * numGroupsPerBlock));
        
        assert(smem % sizeof(int) == 0);

    	#define mycall(blocksize) msa_correct_candidates_with_group_kernel<(blocksize), groupsize> \
    	        <<<grid, block, smem, stream>>>( \
            d_msapointers, \
            d_alignmentresultpointers, \
            d_sequencePointers, \
            d_correctionResultPointers, \
            d_numTotalCorrectedCandidates, \
            d_editsPerCorrectedCandidate, \
            d_numEditsPerCorrectedCandidate, \
            d_candidateContainsN, \
            doNotUseEditsValue, \
            numEditsThreshold, \
    		d_indices, \
    		d_indices_per_subject, \
    		d_candidates_per_subject_prefixsum, \
            d_candidatesPerHQAnchorPrefixSum, \
    		n_subjects, \
    		n_queries, \
    		d_num_indices, \
            encodedSequencePitchInInts, \
    		sequence_pitch, \
    		msa_pitch, \
            msa_weights_pitch, \
            dynamicsmemPitchInInts, \
    		min_support_threshold, \
    		min_coverage_threshold, \
    		new_columns_to_correct); CUERR;


    	switch(blocksize) {
    	case 32: mycall(32); break;
    	case 64: mycall(64); break;
    	case 96: mycall(96); break;
    	case 128: mycall(128); break;
    	case 160: mycall(160); break;
    	case 192: mycall(192); break;
    	case 224: mycall(224); break;
    	case 256: mycall(256); break;
    	default: mycall(256); break;
    	}

    		#undef mycall

        cubCachingAllocator.DeviceFree(d_candidatesPerHQAnchorPrefixSum);  CUERR;
    }


    void callCompactCandidateCorrectionResultsKernel_async(
            char* __restrict__ d_compactedCorrectedCandidates,
            const int* __restrict__ d_numCorrectedCandidatesPerAnchor,
            const int* __restrict__ d_numCorrectedCandidatesPerAnchorPrefixsum, //exclusive
            const int* __restrict__ d_high_quality_subject_indices,
            const int* __restrict__ d_num_high_quality_subject_indices,
            const int* __restrict__ d_candidates_per_subject_prefixsum,
            const char* __restrict__ d_correctedCandidates,
            const int* __restrict__ d_correctedCandidateLengths,
            size_t decodedSequencePitch,
            int n_subjects,
            hipStream_t stream,
            KernelLaunchHandle& /*handle*/){

        constexpr int blocksize = 256;

        dim3 block(blocksize);
        dim3 grid(SDIV(n_subjects, blocksize / 32));

        compactCandidateCorrectionResultsKernel<blocksize><<<grid, block, 0, stream>>>(
            d_compactedCorrectedCandidates,
            d_numCorrectedCandidatesPerAnchor,
            d_numCorrectedCandidatesPerAnchorPrefixsum,
            d_high_quality_subject_indices,
            d_num_high_quality_subject_indices,
            d_candidates_per_subject_prefixsum,
            d_correctedCandidates,
            d_correctedCandidateLengths,
            decodedSequencePitch,
            n_subjects
        );
    }






    void callConstructAnchorResultsKernelAsync(
            TempCorrectedSequence::Edit* __restrict__ d_editsPerCorrectedSubject,
            int* __restrict__ d_numEditsPerCorrectedSubject,
            int doNotUseEditsValue,
            const int* __restrict__ d_indicesOfCorrectedSubjects,
            const int* __restrict__ d_numIndicesOfCorrectedSubjects,
            const bool* __restrict__ d_readContainsN,
            const unsigned int* __restrict__ d_uncorrectedSubjects,
            const int* __restrict__ d_subjectLengths,
            const char* __restrict__ d_correctedSubjects,
            int numEditsThreshold,
            size_t encodedSequencePitchInInts,
            size_t decodedSequencePitchInBytes,
            int numSubjects,
            hipStream_t stream,
            KernelLaunchHandle& handle){

        hipMemsetAsync(d_editsPerCorrectedSubject, 0, sizeof(TempCorrectedSequence::Edit) * numSubjects, stream);

        const int blocksize = 128;
        const std::size_t smem = 0;

        int max_blocks_per_device = 1;

        KernelLaunchConfig kernelLaunchConfig;
        kernelLaunchConfig.threads_per_block = blocksize;
        kernelLaunchConfig.smem = smem;

        auto iter = handle.kernelPropertiesMap.find(KernelId::ConstructAnchorResults);
        if(iter == handle.kernelPropertiesMap.end()){

            std::map<KernelLaunchConfig, KernelProperties> mymap;

            #define getProp(blocksize) { \
                KernelLaunchConfig kernelLaunchConfig; \
                kernelLaunchConfig.threads_per_block = (blocksize); \
                kernelLaunchConfig.smem = 0; \
                KernelProperties kernelProperties; \
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                    constructAnchorResultsKernel, \
                                                                kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
                mymap[kernelLaunchConfig] = kernelProperties; \
            }

            getProp(32);
            getProp(64);
            getProp(96);
            getProp(128);
            getProp(160);
            getProp(192);
            getProp(224);
            getProp(256);

            const auto& kernelProperties = mymap[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

            handle.kernelPropertiesMap[KernelId::ConstructAnchorResults] = std::move(mymap);

            #undef getProp
        }else{
            std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
            const KernelProperties& kernelProperties = map[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
        }

        dim3 block(blocksize, 1, 1);
        dim3 grid(std::min(SDIV(numSubjects, blocksize), max_blocks_per_device));

        #define mycall(blocksize) constructAnchorResultsKernel \
                                <<<grid, block, 0, stream>>>( \
                                        d_editsPerCorrectedSubject, \
                                        d_numEditsPerCorrectedSubject, \
                                        doNotUseEditsValue, \
                                        d_indicesOfCorrectedSubjects, \
                                        d_numIndicesOfCorrectedSubjects, \
                                        d_readContainsN, \
                                        d_uncorrectedSubjects, \
                                        d_subjectLengths, \
                                        d_correctedSubjects, \
                                        numEditsThreshold, \
                                        encodedSequencePitchInInts, \
                                        decodedSequencePitchInBytes); CUERR;

        mycall();

        // switch(blocksize){
        //     case 32: mycall(32); break;
        //     case 64: mycall(64); break;
        //     case 96: mycall(96); break;
        //     case 128: mycall(128); break;
        //     case 160: mycall(160); break;
        //     case 192: mycall(192); break;
        //     case 224: mycall(224); break;
        //     case 256: mycall(256); break;
        //     default: mycall(256); break;
        // }
         #undef mycall
    }











}
}
