#include "hip/hip_runtime.h"
#include <dispatch_care.hpp>
#include <gpu/gpuminhasher.cuh>

#include <config.hpp>
#include <options.hpp>
#include <readlibraryio.hpp>
#include <minhasher.hpp>
//#include <build.hpp>
#include <gpu/distributedreadstorage.hpp>
#include <gpu/correct_gpu.hpp>
#include <correctionresultprocessing.hpp>

#include <rangegenerator.hpp>

#include <algorithm>
#include <iostream>
#include <memory>
#include <vector>

#include <experimental/filesystem>

namespace filesys = std::experimental::filesystem;

namespace care{

    std::vector<int> getUsableDeviceIds(std::vector<int> deviceIds){
        int nDevices;

        hipGetDeviceCount(&nDevices); CUERR;

        std::vector<int> invalidIds;

        for(int id : deviceIds) {
            if(id >= nDevices) {
                invalidIds.emplace_back(id);
                std::cout << "Found invalid device Id: " << id << std::endl;
            }
        }

        if(invalidIds.size() > 0) {
            std::cout << "Available GPUs on your machine:" << std::endl;
            for(int j = 0; j < nDevices; j++) {
                hipDeviceProp_t prop;
                hipGetDeviceProperties(&prop, j); CUERR;
                std::cout << "Id " << j << " : " << prop.name << std::endl;
            }

            for(int invalidid : invalidIds) {
                deviceIds.erase(std::find(deviceIds.begin(), deviceIds.end(), invalidid));
            }
        }

        return deviceIds;
    }

    

    template<class T>
    void printDataStructureMemoryUsage(const T& datastructure, const std::string& name){
    	auto toGB = [](std::size_t bytes){
    			    double gb = bytes / 1024. / 1024. / 1024.0;
    			    return gb;
    		    };

        auto memInfo = datastructure.getMemoryInfo();
        
        std::cout << name << " memory usage: " << toGB(memInfo.host) << " GB on host\n";
        for(const auto& pair : memInfo.device){
            std::cout << name << " memory usage: " << toGB(pair.second) << " GB on device " << pair.first << '\n';
        }
    }

    void performCorrection(
                            CorrectionOptions correctionOptions,
                            RuntimeOptions runtimeOptions,
                            MemoryOptions memoryOptions,
                            FileOptions fileOptions,
                            GoodAlignmentProperties goodAlignmentProperties){

        std::cout << "Running CARE GPU" << std::endl;

        if(runtimeOptions.deviceIds.size() == 0){
            std::cout << "No device ids found. Abort!" << std::endl;
            return;
        }

        std::uint64_t maximumNumberOfReads = fileOptions.nReads;
        int maximumSequenceLength = fileOptions.maximum_sequence_length;
        int minimumSequenceLength = fileOptions.minimum_sequence_length;
        bool scanned = false;

        if(fileOptions.load_binary_reads_from == ""){

            if(maximumNumberOfReads == 0 || maximumSequenceLength == 0 || minimumSequenceLength == 0) {
                std::cout << "STEP 0: Determine input size" << std::endl;
                
                std::cout << "Scanning file(s) to get number of reads and min/max sequence length." << std::endl;

                maximumNumberOfReads = 0;
                maximumSequenceLength = 0;
                minimumSequenceLength = std::numeric_limits<int>::max();

                for(const auto& inputfile : fileOptions.inputfiles){
                    auto prop = getSequenceFileProperties(inputfile, runtimeOptions.showProgress);
                    maximumNumberOfReads += prop.nReads;
                    maximumSequenceLength = std::max(maximumSequenceLength, prop.maxSequenceLength);
                    minimumSequenceLength = std::min(minimumSequenceLength, prop.minSequenceLength);

                    std::cout << "----------------------------------------\n";
                    std::cout << "File: " << inputfile << "\n";
                    std::cout << "Reads: " << prop.nReads << "\n";
                    std::cout << "Minimum sequence length: " << prop.minSequenceLength << "\n";
                    std::cout << "Maximum sequence length: " << prop.maxSequenceLength << "\n";
                    std::cout << "----------------------------------------\n";

                    //result.inputFileProperties.emplace_back(prop);
                }

                scanned = true;
            }else{
                //std::cout << "Using the supplied max number of reads and min/max sequence length." << std::endl;
            }
        }

        /*
            Step 1: 
            - load all reads from all input files into (gpu-)memory
            - construct minhash signatures of all reads and store them in hash tables
        */

        TIMERSTARTCPU(STEP1);

        std::cout << "STEP 1: Database construction" << std::endl;

        TIMERSTARTCPU(build_readstorage);

        gpu::DistributedReadStorage readStorage(
            runtimeOptions.deviceIds, 
            maximumNumberOfReads, 
            correctionOptions.useQualityScores, 
            minimumSequenceLength, 
            maximumSequenceLength
        );

        if(fileOptions.load_binary_reads_from != ""){

            TIMERSTARTCPU(load_from_file);
            readStorage.loadFromFile(fileOptions.load_binary_reads_from, runtimeOptions.deviceIds);
            TIMERSTOPCPU(load_from_file);

            if(correctionOptions.useQualityScores && !readStorage.canUseQualityScores())
                throw std::runtime_error("Quality scores are required but not present in preprocessed reads file!");
            if(!correctionOptions.useQualityScores && readStorage.canUseQualityScores())
                std::cerr << "Warning. The loaded preprocessed reads file contains quality scores, but program does not use them!\n";

            std::cout << "Loaded preprocessed reads from " << fileOptions.load_binary_reads_from << std::endl;

            readStorage.constructionIsComplete();
        }else{
            readStorage.construct(
                fileOptions.inputfiles,
                correctionOptions.useQualityScores,
                maximumNumberOfReads,
                minimumSequenceLength,
                maximumSequenceLength,
                runtimeOptions.threads,
                runtimeOptions.showProgress
            );
        }

        if(fileOptions.save_binary_reads_to != "") {
            std::cout << "Saving reads to file " << fileOptions.save_binary_reads_to << std::endl;
            TIMERSTARTCPU(save_to_file);
            readStorage.saveToFile(fileOptions.save_binary_reads_to);
            TIMERSTOPCPU(save_to_file);
    		std::cout << "Saved reads" << std::endl;
        }

        TIMERSTOPCPU(build_readstorage);
        
        SequenceFileProperties totalInputFileProperties;

        totalInputFileProperties.nReads = readStorage.getNumberOfReads();
        totalInputFileProperties.maxSequenceLength = readStorage.getStatistics().maximumSequenceLength;
        totalInputFileProperties.minSequenceLength = readStorage.getStatistics().minimumSequenceLength;

        if(!scanned){
            std::cout << "Determined the following read properties:\n";
            std::cout << "----------------------------------------\n";
            std::cout << "Total number of reads: " << totalInputFileProperties.nReads << "\n";
            std::cout << "Minimum sequence length: " << totalInputFileProperties.minSequenceLength << "\n";
            std::cout << "Maximum sequence length: " << totalInputFileProperties.maxSequenceLength << "\n";
            std::cout << "----------------------------------------\n";
        }

        if(correctionOptions.autodetectKmerlength){
            const int maxlength = totalInputFileProperties.maxSequenceLength;

            auto getKmerSizeForHashing = [](int maximumReadLength){
                if(maximumReadLength < 160){
                    return 20;
                }else{
                    return 32;
                }
            };

            correctionOptions.kmerlength = getKmerSizeForHashing(maxlength);

            std::cout << "Will use k-mer length = " << correctionOptions.kmerlength << " for hashing.\n";
        }

        std::cout << "Reads with ambiguous bases: " << readStorage.getNumberOfReadsWithN() << std::endl;
        

        printDataStructureMemoryUsage(readStorage, "reads");


        TIMERSTARTCPU(build_newgpuminhasher);
        gpu::GpuMinhasher newGpuMinhasher(
            correctionOptions.kmerlength, 
            calculateResultsPerMapThreshold(correctionOptions.estimatedCoverage)
        );

        if(fileOptions.load_hashtables_from != ""){

            std::ifstream is(fileOptions.load_hashtables_from);
            assert((bool)is);

            newGpuMinhasher.loadFromStream(is);

            std::cout << "Loaded hash tables from " << fileOptions.load_hashtables_from << std::endl;
        }else{
            newGpuMinhasher.construct(
                fileOptions,
                runtimeOptions,
                memoryOptions,
                totalInputFileProperties.nReads, 
                correctionOptions,
                readStorage
            );

            if(correctionOptions.mustUseAllHashfunctions 
                && correctionOptions.numHashFunctions != newGpuMinhasher.getNumberOfMaps()){
                std::cout << "Cannot use specified number of hash functions (" 
                    << correctionOptions.numHashFunctions <<")\n";
                std::cout << "Abort!\n";
                return;
            }
        }

        if(fileOptions.save_hashtables_to != "") {
            std::cout << "Saving minhasher to file " << fileOptions.save_hashtables_to << std::endl;
            std::ofstream os(fileOptions.save_hashtables_to);
            assert((bool)os);

            newGpuMinhasher.writeToStream(os);

    		std::cout << "Saved minhasher" << std::endl;
        }

        printDataStructureMemoryUsage(newGpuMinhasher, "hash tables");



        TIMERSTOPCPU(build_newgpuminhasher);

        TIMERSTOPCPU(STEP1)



#if 0

        maximumNumberOfReads = totalInputFileProperties.nReads;
        maximumSequenceLength = totalInputFileProperties.maxSequenceLength;
        minimumSequenceLength = totalInputFileProperties.minSequenceLength;

        // timing of querying all reads. Executing on gpu 0

        const int deviceId = 0;
        hipSetDevice(deviceId); CUERR;

        hipStream_t stream;
        hipStreamCreate(&stream); CUERR;


        const int maxbatchsize = 1000; // number of reads to query simultaneously

        {

            using DeviceBufferRN = SimpleAllocationDevice<read_number, 0>;
            using PinnedBufferRN = SimpleAllocationPinnedHost<read_number, 0>;
            using DeviceBufferUint = SimpleAllocationDevice<unsigned int, 0>;
            using DeviceBufferInt = SimpleAllocationDevice<int, 0>;


            const int encodedSequencePitchInInts = getEncodedNumInts2Bit(maximumSequenceLength);
            const int numHashFunctions = newGpuMinhasher.getNumberOfMaps();
            const int resultsPerMap = calculateResultsPerMapThreshold(correctionOptions.estimatedCoverage);
            const int maxResultIds = resultsPerMap * numHashFunctions * maxbatchsize;


            DeviceBufferRN d_readIds(maxbatchsize);
            PinnedBufferRN h_readIds(maxbatchsize);
            DeviceBufferUint d_encodedSequences(encodedSequencePitchInInts * maxbatchsize);
            DeviceBufferInt d_sequenceLengths(maxbatchsize);
            
            DeviceBufferRN d_queryresults(maxResultIds);
            DeviceBufferInt d_numResultsPerReadId(maxbatchsize);
            DeviceBufferInt d_numResultsPerReadIdExclPrefixSum(maxbatchsize+1); // plus total count as last element


            SimpleAllocationDevice<std::uint64_t, 0> d_normalhashvalues(numHashFunctions * maxbatchsize);
            SimpleAllocationDevice<std::uint64_t, 0> d_uniquehashvalues(numHashFunctions * maxbatchsize);
            SimpleAllocationDevice<std::uint64_t, 0> d_temp(numHashFunctions * maxbatchsize);
            SimpleAllocationDevice<int, 0> d_hashfuncids(numHashFunctions * maxbatchsize);
            SimpleAllocationDevice<int, 0> d_signaturesizes(maxbatchsize);

            SimpleAllocationPinnedHost<int, 0> h_signaturesizes(maxbatchsize);

            const int threadPoolSize = std::max(1, runtimeOptions.threads - 3*int(runtimeOptions.deviceIds.size()));

            ThreadPool threadPool(threadPoolSize);
            auto sequenceGatherHandle = readStorage.makeGatherHandleSequences();
            auto minhasherQueryHandle = gpu::GpuMinhasher::makeQueryHandle();
            minhasherQueryHandle.resize(newGpuMinhasher, maxbatchsize, threadPoolSize);

            ThreadPool::ParallelForHandle pforHandle;

            cpu::RangeGenerator<read_number> readIdGenerator(maximumNumberOfReads);

            auto showProgress = [&](std::int64_t totalCount, int seconds){
                if(runtimeOptions.showProgress){
    
                    int hours = seconds / 3600;
                    seconds = seconds % 3600;
                    int minutes = seconds / 60;
                    seconds = seconds % 60;
                    
                    printf("Processed %10lu of %10lu reads (Runtime: %03d:%02d:%02d)\r",
                    totalCount, maximumNumberOfReads,
                    hours, minutes, seconds);
                }
            };
    
            auto updateShowProgressInterval = [](auto duration){
                return duration;
            };
    
            ProgressThread<std::int64_t> progressThread(maximumNumberOfReads, showProgress, updateShowProgressInterval);
    
            hipEvent_t eventbegin, eventend;
            hipEventCreate(&eventbegin); CUERR;
            hipEventCreate(&eventend); CUERR;

            float minhashtimeold = 0;
            float minhashtimenew = 0;

            std::map<int, int> numHashvaluesToFrequencyMap;
            
            while(!readIdGenerator.empty()){                

                auto hreadIdsEnd = readIdGenerator.next_n_into_buffer(
                    maxbatchsize, 
                    h_readIds.get()
                );

                const int currentbatchsize = std::distance(h_readIds.get(), hreadIdsEnd);

                hipMemcpyAsync(
                    d_readIds.get(),
                    h_readIds.get(),
                    sizeof(read_number) * currentbatchsize,
                    H2D,
                    stream
                ); CUERR;


                readStorage.gatherSequenceDataToGpuBufferAsync(
                    &threadPool,
                    sequenceGatherHandle,
                    d_encodedSequences.get(),
                    encodedSequencePitchInInts,
                    h_readIds.get(),
                    d_readIds.get(),
                    currentbatchsize,
                    deviceId,
                    stream
                );
        
                readStorage.gatherSequenceLengthsToGpuBufferAsync(
                    d_sequenceLengths.get(),
                    deviceId,
                    d_readIds.get(),
                    currentbatchsize,         
                    stream
                );

                ParallelForLoopExecutor parallelFor(&threadPool, &pforHandle);

                hipEventRecord(eventbegin, stream); CUERR;

                callMinhashSignaturesKernel_async(
                    d_normalhashvalues.get(),
                    numHashFunctions,
                    d_encodedSequences.get(),
                    encodedSequencePitchInInts,
                    currentbatchsize,
                    d_sequenceLengths.get(),
                    correctionOptions.kmerlength,
                    numHashFunctions,
                    stream
                );

                hipEventRecord(eventend, stream); CUERR;

                hipEventSynchronize(eventend); CUERR;

                float deltaold = 0;
                hipEventElapsedTime(&deltaold, eventbegin, eventend);

                minhashtimeold += deltaold;


                hipEventRecord(eventbegin, stream); CUERR;

                callUniqueMinhashSignaturesKernel_async(
                    d_temp.get(),
                    d_uniquehashvalues.get(),
                    numHashFunctions,
                    d_hashfuncids.get(),
                    numHashFunctions,
                    d_signaturesizes.get(),
                    d_encodedSequences.get(),
                    encodedSequencePitchInInts,
                    currentbatchsize,
                    d_sequenceLengths.get(),
                    correctionOptions.kmerlength,
                    numHashFunctions,
                    stream
                );

                hipEventRecord(eventend, stream); CUERR;

                hipEventSynchronize(eventend); CUERR;

                float deltanew = 0;
                hipEventElapsedTime(&deltanew, eventbegin, eventend);

                minhashtimenew += deltanew;

                //check 

                {
                    generic_kernel<<<currentbatchsize, 64,0, stream>>>(
                        [ = ,
                            d_normalhashvalues = d_normalhashvalues.get(),
                            d_uniquehashvalues = d_uniquehashvalues.get(),
                            d_hashfuncids = d_hashfuncids.get(),
                            d_signaturesizes = d_signaturesizes.get()
                        ] __device__ (){

                            for(int s = blockIdx.x; s < currentbatchsize; s += gridDim.x){
                                const int signaturesize = d_signaturesizes[s];

                                // printf("old: ");
                                // for(int k = 0; k < numHashFunctions; k += 1){
                                //     printf("%llu ", d_normalhashvalues[numHashFunctions * s + k]);
                                // }
                                // printf("\n");


                                // printf("new(%d): ", signaturesize);
                                // for(int k = 0; k < signaturesize; k += 1){
                                //     printf("(%llu %d) ", 
                                //         d_uniquehashvalues[numHashFunctions * s + k], 
                                //         d_hashfuncids[numHashFunctions * s + k]
                                //     );
                                // }
                                // printf("\n");

                                for(int k = threadIdx.x; k < signaturesize; k += blockDim.x){
                                    const auto newhashvalue = d_uniquehashvalues[numHashFunctions * s + k];
                                    const int hashfuncid = d_hashfuncids[numHashFunctions * s + k];

                                    const auto oldhashvalue = d_normalhashvalues[numHashFunctions * s + hashfuncid];

                                    if(newhashvalue != oldhashvalue){
                                        printf("error at s=%d, k=%d, hashfuncid=%d, old=%llu, new=%llu\n", 
                                            s,k, hashfuncid, oldhashvalue,newhashvalue);
                                    }
                                    assert(newhashvalue == oldhashvalue);
                                }
                            }
                        }
                    );
                    CUERR;
                }

                hipMemcpyAsync(
                    h_signaturesizes.get(),
                    d_signaturesizes.get(),
                    d_signaturesizes.sizeInBytes(),
                    D2H,
                    stream
                );

                hipStreamSynchronize(stream);

                progressThread.addProgress(currentbatchsize);

                for(int i = 0; i < currentbatchsize; i++){
                    numHashvaluesToFrequencyMap[h_signaturesizes[i]]++;
                }


            }

            progressThread.finished(); 
        
            std::cout << std::endl;

            std::cout << "Minhashing old took " << minhashtimeold << " ms\n";
            std::cout << "Minhashing new took " << minhashtimenew << " ms\n";

            for(auto pair : numHashvaluesToFrequencyMap){
                std::cout << pair.first << " " << pair.second << "\n";
            }

            hipEventDestroy(eventbegin); CUERR;
            hipEventDestroy(eventend); CUERR;

            gpu::GpuMinhasher::destroyQueryHandle(minhasherQueryHandle);

        }




        hipStreamDestroy(stream); CUERR;





#endif 






        std::cout << "STEP 2: Error correction" << std::endl;

        TIMERSTARTCPU(STEP2);

        auto partialResults = gpu::correct_gpu(
            goodAlignmentProperties, 
            correctionOptions,
            runtimeOptions, 
            fileOptions, 
            memoryOptions,
            totalInputFileProperties,
            //minhasher, 
            newGpuMinhasher,
            readStorage
        );

        TIMERSTOPCPU(STEP2);

        //minhasher.destroy();
        newGpuMinhasher.destroy();
        readStorage.destroy();

        //Merge corrected reads with input file to generate output file

        const std::size_t availableMemoryInBytes = getAvailableMemoryInKB() * 1024;
        std::size_t memoryForSorting = 0;

        if(availableMemoryInBytes > 1*(std::size_t(1) << 30)){
            memoryForSorting = availableMemoryInBytes - 1*(std::size_t(1) << 30);
        }               

        std::cout << "STEP 3: Constructing output file(s)" << std::endl;

        TIMERSTARTCPU(STEP3);

        std::vector<FileFormat> formats;
        for(const auto& inputfile : fileOptions.inputfiles){
            formats.emplace_back(getFileFormat(inputfile));
        }
        std::vector<std::string> outputfiles;
        for(const auto& outputfilename : fileOptions.outputfilenames){
            outputfiles.emplace_back(fileOptions.outputdirectory + "/" + outputfilename);
        }
        constructOutputFileFromCorrectionResults(
            fileOptions.tempdirectory,
            fileOptions.inputfiles, 
            partialResults, 
            memoryForSorting,
            formats[0],
            outputfiles, 
            false
        );

        TIMERSTOPCPU(STEP3);

        std::cout << "Construction of output file(s) finished." << std::endl;

    }

}
