#include "hip/hip_runtime.h"
#include <dispatch_care.hpp>
#include <gpu/gpuminhasher.cuh>

#include <config.hpp>
#include <options.hpp>
#include <readlibraryio.hpp>
#include <minhasher.hpp>
//#include <build.hpp>
#include <gpu/distributedreadstorage.hpp>
#include <gpu/correct_gpu.hpp>
#include <correctionresultprocessing.hpp>

#include <algorithm>
#include <iostream>
#include <memory>
#include <vector>

#include <experimental/filesystem>

namespace filesys = std::experimental::filesystem;

namespace care{

    std::vector<int> getUsableDeviceIds(std::vector<int> deviceIds){
        int nDevices;

        hipGetDeviceCount(&nDevices); CUERR;

        std::vector<int> invalidIds;

        for(int id : deviceIds) {
            if(id >= nDevices) {
                invalidIds.emplace_back(id);
                std::cout << "Found invalid device Id: " << id << std::endl;
            }
        }

        if(invalidIds.size() > 0) {
            std::cout << "Available GPUs on your machine:" << std::endl;
            for(int j = 0; j < nDevices; j++) {
                hipDeviceProp_t prop;
                hipGetDeviceProperties(&prop, j); CUERR;
                std::cout << "Id " << j << " : " << prop.name << std::endl;
            }

            for(int invalidid : invalidIds) {
                deviceIds.erase(std::find(deviceIds.begin(), deviceIds.end(), invalidid));
            }
        }

        return deviceIds;
    }

    

    template<class minhasher_t,
             class readStorage_t>
    void printDataStructureMemoryUsage(const minhasher_t& minhasher, const readStorage_t& readStorage){
    	auto toGB = [](std::size_t bytes){
    			    double gb = bytes / 1024. / 1024. / 1024.0;
    			    return gb;
    		    };

        auto rsMemInfo = readStorage.getMemoryInfo();

        std::cout << "Reads occupy " << toGB(rsMemInfo.host) << " GB on host\n";
        for(const auto& pair : rsMemInfo.device){
            std::cout << "Reads occupy " << toGB(pair.second) << " GB on device " << pair.first << '\n';
        }

    	//std::cout << "reads take up " << toGB(readStorage.size()) << " GB." << std::endl;
    	std::cout << "hash maps take up " << toGB(minhasher.numBytes()) << " GB on host." << std::endl;
    }

    void performCorrection(
                            CorrectionOptions correctionOptions,
                            RuntimeOptions runtimeOptions,
                            MemoryOptions memoryOptions,
                            FileOptions fileOptions,
                            GoodAlignmentProperties goodAlignmentProperties){

        std::cout << "Running CARE GPU" << std::endl;

        if(runtimeOptions.deviceIds.size() == 0){
            std::cout << "No device ids found. Abort!" << std::endl;
            return;
        }

        std::uint64_t maximumNumberOfReads = fileOptions.nReads;
        int maximumSequenceLength = fileOptions.maximum_sequence_length;
        int minimumSequenceLength = fileOptions.minimum_sequence_length;
        bool scanned = false;

        if(fileOptions.load_binary_reads_from == ""){

            if(maximumNumberOfReads == 0 || maximumSequenceLength == 0 || minimumSequenceLength == 0) {
                std::cout << "STEP 0: Determine input size" << std::endl;
                
                std::cout << "Scanning file(s) to get number of reads and min/max sequence length." << std::endl;

                maximumNumberOfReads = 0;
                maximumSequenceLength = 0;
                minimumSequenceLength = std::numeric_limits<int>::max();

                for(const auto& inputfile : fileOptions.inputfiles){
                    auto prop = getSequenceFileProperties(inputfile, runtimeOptions.showProgress);
                    maximumNumberOfReads += prop.nReads;
                    maximumSequenceLength = std::max(maximumSequenceLength, prop.maxSequenceLength);
                    minimumSequenceLength = std::min(minimumSequenceLength, prop.minSequenceLength);

                    std::cout << "----------------------------------------\n";
                    std::cout << "File: " << inputfile << "\n";
                    std::cout << "Reads: " << prop.nReads << "\n";
                    std::cout << "Minimum sequence length: " << prop.minSequenceLength << "\n";
                    std::cout << "Maximum sequence length: " << prop.maxSequenceLength << "\n";
                    std::cout << "----------------------------------------\n";

                    //result.inputFileProperties.emplace_back(prop);
                }

                scanned = true;
            }else{
                //std::cout << "Using the supplied max number of reads and min/max sequence length." << std::endl;
            }
        }

        /*
            Step 1: 
            - load all reads from all input files into (gpu-)memory
            - construct minhash signatures of all reads and store them in hash tables
        */

        TIMERSTARTCPU(STEP1);

        std::cout << "STEP 1: Database construction" << std::endl;

        TIMERSTARTCPU(build_readstorage);

        gpu::DistributedReadStorage readStorage(
            runtimeOptions.deviceIds, 
            maximumNumberOfReads, 
            correctionOptions.useQualityScores, 
            minimumSequenceLength, 
            maximumSequenceLength
        );

        if(fileOptions.load_binary_reads_from != ""){

            TIMERSTARTCPU(load_from_file);
            readStorage.loadFromFile(fileOptions.load_binary_reads_from, runtimeOptions.deviceIds);
            TIMERSTOPCPU(load_from_file);

            if(correctionOptions.useQualityScores && !readStorage.canUseQualityScores())
                throw std::runtime_error("Quality scores are required but not present in preprocessed reads file!");
            if(!correctionOptions.useQualityScores && readStorage.canUseQualityScores())
                std::cerr << "Warning. The loaded preprocessed reads file contains quality scores, but program does not use them!\n";

            std::cout << "Loaded preprocessed reads from " << fileOptions.load_binary_reads_from << std::endl;

            readStorage.constructionIsComplete();
        }else{
            readStorage.construct(
                fileOptions.inputfiles,
                correctionOptions.useQualityScores,
                maximumNumberOfReads,
                minimumSequenceLength,
                maximumSequenceLength,
                runtimeOptions.threads,
                runtimeOptions.showProgress
            );
        }

        if(fileOptions.save_binary_reads_to != "") {
            std::cout << "Saving reads to file " << fileOptions.save_binary_reads_to << std::endl;
            TIMERSTARTCPU(save_to_file);
            readStorage.saveToFile(fileOptions.save_binary_reads_to);
            TIMERSTOPCPU(save_to_file);
    		std::cout << "Saved reads" << std::endl;
        }

        TIMERSTOPCPU(build_readstorage);
        
        SequenceFileProperties totalInputFileProperties;

        totalInputFileProperties.nReads = readStorage.getNumberOfReads();
        totalInputFileProperties.maxSequenceLength = readStorage.getStatistics().maximumSequenceLength;
        totalInputFileProperties.minSequenceLength = readStorage.getStatistics().minimumSequenceLength;

        if(!scanned){
            std::cout << "Determined the following read properties:\n";
            std::cout << "----------------------------------------\n";
            std::cout << "Total number of reads: " << totalInputFileProperties.nReads << "\n";
            std::cout << "Minimum sequence length: " << totalInputFileProperties.minSequenceLength << "\n";
            std::cout << "Maximum sequence length: " << totalInputFileProperties.maxSequenceLength << "\n";
            std::cout << "----------------------------------------\n";
        }

        if(correctionOptions.autodetectKmerlength){
            const int maxlength = totalInputFileProperties.maxSequenceLength;

            auto getKmerSizeForHashing = [](int maximumReadLength){
                if(maximumReadLength < 160){
                    return 20;
                }else{
                    return 32;
                }
            };

            correctionOptions.kmerlength = getKmerSizeForHashing(maxlength);

            std::cout << "Will use k-mer length = " << correctionOptions.kmerlength << " for hashing.\n";
        }

        std::cout << "Reads with ambiguous bases: " << readStorage.getNumberOfReadsWithN() << std::endl;
        





        //printDataStructureMemoryUsage(minhasher, readStorage);

        //minhasher.destroy();

        TIMERSTARTCPU(build_newgpuminhasher);
        gpu::GpuMinhasher newGpuMinhasher(
            correctionOptions.kmerlength, 
            calculateResultsPerMapThreshold(correctionOptions.estimatedCoverage)
        );

        if(fileOptions.load_hashtables_from != ""){

            std::ifstream is(fileOptions.load_hashtables_from);
            assert((bool)is);

            newGpuMinhasher.loadFromStream(is);

            std::cout << "Loaded hash tables from " << fileOptions.load_hashtables_from << std::endl;
        }else{
            newGpuMinhasher.construct(
                fileOptions,
                runtimeOptions,
                memoryOptions,
                totalInputFileProperties.nReads, 
                correctionOptions,
                readStorage
            );

            if(correctionOptions.mustUseAllHashfunctions 
                && correctionOptions.numHashFunctions != newGpuMinhasher.getNumberOfMaps()){
                std::cout << "Cannot use specified number of hash functions (" 
                    << correctionOptions.numHashFunctions <<")\n";
                std::cout << "Abort!\n";
                return;
            }
        }

        if(fileOptions.save_hashtables_to != "") {
            std::cout << "Saving minhasher to file " << fileOptions.save_hashtables_to << std::endl;
            std::ofstream os(fileOptions.save_hashtables_to);
            assert((bool)os);

            newGpuMinhasher.writeToStream(os);

    		std::cout << "Saved minhasher" << std::endl;
        }



        TIMERSTOPCPU(build_newgpuminhasher);

        TIMERSTOPCPU(STEP1)




        std::cout << "STEP 2: Error correction" << std::endl;

        TIMERSTARTCPU(STEP2);

        auto partialResults = gpu::correct_gpu(
            goodAlignmentProperties, 
            correctionOptions,
            runtimeOptions, 
            fileOptions, 
            memoryOptions,
            totalInputFileProperties,
            //minhasher, 
            newGpuMinhasher,
            readStorage
        );

        TIMERSTOPCPU(STEP2);

        //minhasher.destroy();
        newGpuMinhasher.destroy();
        readStorage.destroy();

        //Merge corrected reads with input file to generate output file

        const std::size_t availableMemoryInBytes = getAvailableMemoryInKB() * 1024;
        std::size_t memoryForSorting = 0;

        if(availableMemoryInBytes > 1*(std::size_t(1) << 30)){
            memoryForSorting = availableMemoryInBytes - 1*(std::size_t(1) << 30);
        }

        std::cout << "STEP 3: Constructing output file(s)" << std::endl;

        TIMERSTARTCPU(STEP3);

        std::vector<FileFormat> formats;
        for(const auto& inputfile : fileOptions.inputfiles){
            formats.emplace_back(getFileFormat(inputfile));
        }
        std::vector<std::string> outputfiles;
        for(const auto& outputfilename : fileOptions.outputfilenames){
            outputfiles.emplace_back(fileOptions.outputdirectory + "/" + outputfilename);
        }
        constructOutputFileFromResults2(
            fileOptions.tempdirectory,
            fileOptions.inputfiles, 
            partialResults, 
            memoryForSorting,
            formats[0],
            outputfiles, 
            false
        );

        TIMERSTOPCPU(STEP3);

        std::cout << "Construction of output file(s) finished." << std::endl;

    }

}
