#include "hip/hip_runtime.h"
#include <gpu/readextender_gpu.hpp>
#include <readextenderbase.hpp>

#include <vector>
#include <algorithm>
#include <sequencehelpers.hpp>
#include <string>

#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/transform_iterator.h>

#include <gpu/segmented_set_operations.cuh>
#include <gpu/cachingallocator.cuh>
#include <hostdevicefunctions.cuh>

//#define checkdebugtasks

namespace care{

    struct SequenceFlagMultiplier{
        int pitch{};
        const bool* flags{};

        __host__ __device__
        SequenceFlagMultiplier(const bool* flags_, int pitch_)
            :pitch(pitch_), flags(flags_){

        }

        __host__ __device__
        bool operator()(int i) const{
            return flags[i / pitch];
        }
    };

    //flag candidates to remove because they are equal to anchor id or equal to mate id
    __global__
    void flagCandidateIdsWhichAreEqualToAnchorOrMateKernel(
        const read_number* __restrict__ candidateReadIds,
        const read_number* __restrict__ anchorReadIds,
        const read_number* __restrict__ mateReadIds,
        const int* __restrict__ numCandidatesPerAnchorPrefixSum,
        const int* __restrict__ numCandidatesPerAnchor,
        bool* __restrict__ keepflags, // size numCandidates
        bool* __restrict__ mateRemovedFlags, //size numTasks
        int* __restrict__ numCandidatesPerAnchorOut,
        int numTasks,
        bool isPairedEnd
    ){

        using BlockReduceInt = hipcub::BlockReduce<int, 128>;

        __shared__ typename BlockReduceInt::TempStorage intreduce1;
        __shared__ typename BlockReduceInt::TempStorage intreduce2;

        for(int a = blockIdx.x; a < numTasks; a += gridDim.x){
            const int size = numCandidatesPerAnchor[a];
            const int offset = numCandidatesPerAnchorPrefixSum[a];
            const read_number anchorId = anchorReadIds[a];
            read_number mateId = 0;
            if(isPairedEnd){
                mateId = mateReadIds[a];
            }

            int mateIsRemoved = 0;
            int numRemoved = 0;

            // if(threadIdx.x == 0){
            //     printf("looking for anchor %u, mate %u\n", anchorId, mateId);
            // }
            __syncthreads();

            for(int i = threadIdx.x; i < size; i+= blockDim.x){
                bool keep = true;

                const read_number candidateId = candidateReadIds[offset + i];
                //printf("tid %d, comp %u at position %d\n", threadIdx.x, candidateId, offset + i);

                if(candidateId == anchorId){
                    keep = false;
                    numRemoved++;
                }

                if(isPairedEnd && candidateId == mateId){
                    if(keep){
                        keep = false;
                        numRemoved++;
                    }
                    mateIsRemoved++;
                    //printf("mate removed. i = %d\n", i);
                }

                keepflags[offset + i] = keep;
            }
            //printf("tid = %d, mateIsRemoved = %d\n", threadIdx.x, mateIsRemoved);
            int numRemovedBlock = BlockReduceInt(intreduce1).Sum(numRemoved);
            int numMateRemovedBlock = BlockReduceInt(intreduce2).Sum(mateIsRemoved);
            if(threadIdx.x == 0){
                numCandidatesPerAnchorOut[a] = size - numRemovedBlock;
                //printf("numMateRemovedBlock %d\n", numMateRemovedBlock);
                if(numMateRemovedBlock > 0){
                    mateRemovedFlags[a] = true;
                }else{
                    mateRemovedFlags[a] = false;
                }
            }
        }
    }

    //output[map[i]] = input[i];
    template<class T, class U>
    __global__ 
    void setFirstSegmentIdsKernel(
        const T* __restrict__ segmentSizes,
        int* __restrict__ segmentIds,
        const U* __restrict__ segmentOffsets,
        int N
    ){
        const int tid = threadIdx.x + blockIdx.x * blockDim.x;
        const int stride = blockDim.x * gridDim.x;

        for(int i = tid; i < N; i += stride){
            if(segmentSizes[i] > 0){
                segmentIds[segmentOffsets[i]] = i;
            }
        }
    }













    std::vector<ReadExtenderBase::ExtendResult> ReadExtenderGpu::processPairedEndTasks(
        std::vector<ReadExtenderBase::Task>& tasks
    ) {
 
        std::vector<ExtendResult> extendResults;

        std::vector<int> indicesOfActiveTasks(tasks.size());
        std::vector<int> indicesOfActiveTasksTmp(tasks.size());
        std::iota(indicesOfActiveTasks.begin(), indicesOfActiveTasks.end(), 0);

        std::map<read_number, int> splitTracker; //counts number of tasks per read id, which can change by splitting a task
        for(const auto& t : tasks){
            splitTracker[t.myReadId] = 1;
        }

        //set input string as current anchor
        for(auto& task : tasks){
            std::string decodedAnchor(task.currentAnchorLength, '\0');

            SequenceHelpers::decode2BitSequence(
                &decodedAnchor[0],
                task.currentAnchor.data(),
                task.currentAnchorLength
            );

            task.totalDecodedAnchors.emplace_back(std::move(decodedAnchor));
            task.totalAnchorBeginInExtendedRead.emplace_back(0);
        }


#if 1
        //undo: replace vecAccess\(([a-zA-z]+), ([a-zA-z]+)\) by $1[$2]
        auto vecAccess = [](auto& vec, auto index) -> decltype(vec[index]){
            return vec[index];
        };
#else 
        auto vecAccess = [](auto& vec, auto index) -> decltype(vec.at(index)){
            return vec.at(index);
        };
#endif 

        int deviceId = 0;
        hipGetDevice(&deviceId); CUERR;

        

        hipStream_t firstStream = streams[0];
        hipStream_t secondStream = streams[1];

        ThrustCachingAllocator<char> thrustCachingAllocator1(deviceId, cubAllocator, firstStream);
        auto thrustPolicy1 = thrust::cuda::par(thrustCachingAllocator1).on(firstStream);

        const int numTasks = tasks.size();

        batchData.h_numAnchors.resize(1);
        batchData.h_numCandidates.resize(1);
        batchData.d_numAnchors.resize(1);
        batchData.d_numCandidates.resize(1);

        auto processTasksOldStyle = [this, vecAccess, &indicesOfActiveTasks](auto& tasks){

            getCandidateReadIds(tasks, indicesOfActiveTasks);

            for(int indexOfActiveTask : indicesOfActiveTasks){
                auto& task = vecAccess(tasks, indexOfActiveTask);

                // remove self from candidate list
                auto readIdPos = std::lower_bound(
                    task.candidateReadIds.begin(),                                            
                    task.candidateReadIds.end(),
                    task.myReadId
                );

                if(readIdPos != task.candidateReadIds.end() && *readIdPos == task.myReadId){
                    task.candidateReadIds.erase(readIdPos);
                }

                if(task.pairedEnd){

                    //remove mate of input from candidate list
                    auto mateReadIdPos = std::lower_bound(
                        task.candidateReadIds.begin(),                                            
                        task.candidateReadIds.end(),
                        task.mateReadId
                    );

                    if(mateReadIdPos != task.candidateReadIds.end() && *mateReadIdPos == task.mateReadId){
                        task.candidateReadIds.erase(mateReadIdPos);
                        task.mateRemovedFromCandidates = true;
                    }
                }
            }

            for(int indexOfActiveTask : indicesOfActiveTasks){
                auto& task = vecAccess(tasks, indexOfActiveTask);

                std::vector<read_number> tmp(task.candidateReadIds.size());

                auto end = std::set_difference(
                    task.candidateReadIds.begin(),
                    task.candidateReadIds.end(),
                    task.allUsedCandidateReadIdPairs.begin(),
                    task.allUsedCandidateReadIdPairs.end(),
                    tmp.begin()
                );

                tmp.erase(end, tmp.end());

                std::swap(task.candidateReadIds, tmp);
            }

            loadCandidateSequenceData(tasks, indicesOfActiveTasks);

            eraseDataOfRemovedMates(tasks, indicesOfActiveTasks);

            calculateAlignments(tasks, indicesOfActiveTasks);

            #if 1

            for(int indexOfActiveTask : indicesOfActiveTasks){
                auto& task = vecAccess(tasks, indexOfActiveTask);

                /*
                    Remove bad alignments
                */        

                const int size = task.alignments.size();

                std::vector<int> positionsOfCandidatesToKeep(size);
                std::vector<int> tmpPositionsOfCandidatesToKeep(size);

                task.numRemainingCandidates = 0;

                //select candidates with good alignment and positive shift
                for(int c = 0; c < size; c++){
                    const BestAlignment_t alignmentFlag0 = vecAccess(task.alignmentFlags, c);
                    
                    if(alignmentFlag0 != BestAlignment_t::None && vecAccess(task.alignments, c).shift >= 0){
                        vecAccess(positionsOfCandidatesToKeep, task.numRemainingCandidates) = c;
                        task.numRemainingCandidates++;
                    }else{
                        ; // remove alignment
                    }
                }

                positionsOfCandidatesToKeep.erase(
                    positionsOfCandidatesToKeep.begin() + task.numRemainingCandidates, 
                    positionsOfCandidatesToKeep.end()
                );

                if(task.numRemainingCandidates == 0){
                    task.abort = true;
                    task.abortReason = AbortReason::NoPairedCandidatesAfterAlignment;

                    task.candidateReadIds.erase(task.candidateReadIds.begin(), task.candidateReadIds.end());
                    task.candidateSequenceLengths.erase(task.candidateSequenceLengths.begin(), task.candidateSequenceLengths.end());
                    task.candidateSequenceData.erase(task.candidateSequenceData.begin(), task.candidateSequenceData.end());
                    task.alignments.erase(task.alignments.begin(), task.alignments.end());
                    task.alignmentFlags.erase(task.alignmentFlags.begin(), task.alignmentFlags.end());

                    continue; //stop processing task
                }

                float relativeOverlapThreshold = 0.9f;
                bool goodAlignmentExists = false;

                while(!goodAlignmentExists && fgeq(relativeOverlapThreshold, goodAlignmentProperties.min_overlap_ratio)){                    

                    goodAlignmentExists = std::any_of(
                        positionsOfCandidatesToKeep.begin(), 
                        positionsOfCandidatesToKeep.end(),
                        [&](const auto& position){
                            const auto& alignment = vecAccess(task.alignments, position);
                            const float relativeOverlap = float(alignment.overlap) / float(task.currentAnchorLength);
                            return fgeq(relativeOverlap, relativeOverlapThreshold) && relativeOverlap < 1.0f;
                        }
                    );

                    if(!goodAlignmentExists){
                        relativeOverlapThreshold -= 0.1f;
                    }
                }

                // std::cerr << "task " << indexOfActiveTask
                //      << ", goodAlignmentExists " << goodAlignmentExists
                //      <<", relativeOverlapThreshold " << relativeOverlapThreshold;
                

                if(goodAlignmentExists){
                    positionsOfCandidatesToKeep.erase(
                        std::remove_if(
                            positionsOfCandidatesToKeep.begin(), 
                            positionsOfCandidatesToKeep.end(),
                            [&](const auto& position){
                                const auto& alignment = vecAccess(task.alignments, position);
                                const float relativeOverlap = float(alignment.overlap) / float(task.currentAnchorLength);
                                return !fgeq(relativeOverlap, relativeOverlapThreshold);
                            }
                        ),
                        positionsOfCandidatesToKeep.end()
                    );
                    task.numRemainingCandidates = positionsOfCandidatesToKeep.size();
                }

                // std::cerr << ", numRemainingCandidates = " << task.numRemainingCandidates << "\n";

                // std::cerr << "positionsOfCandidatesToKeep: ";

                // for(int x : positionsOfCandidatesToKeep){
                //     std::cerr << x << " ";
                // }
                // std::cerr << "\n";

                //std::cerr << ", remaining candidates " << task.numRemainingCandidates << "\n";


                //compact selected candidates inplace

                

                {
                    task.candidateSequenceData.resize(task.numRemainingCandidates * encodedSequencePitchInInts);

                    for(int c = 0; c < task.numRemainingCandidates; c++){
                        const int index = vecAccess(positionsOfCandidatesToKeep, c);

                        vecAccess(task.alignments, c) = vecAccess(task.alignments, index);
                        vecAccess(task.alignmentFlags, c) = vecAccess(task.alignmentFlags, index);
                        vecAccess(task.candidateReadIds, c) = vecAccess(task.candidateReadIds, index);
                        vecAccess(task.candidateSequenceLengths, c) = vecAccess(task.candidateSequenceLengths, index);
                        
                        assert(vecAccess(task.alignmentFlags, index) != BestAlignment_t::None);

                        // std::cerr << "cand " << index << " dir " 
                        //     << ((vecAccess(task.alignmentFlags, index) == BestAlignment_t::Forward) ? 'f' : 'r') << "\n";

                        if(vecAccess(task.alignmentFlags, index) == BestAlignment_t::Forward){
                            std::copy_n(
                                task.candidateSequencesFwdData.data() + index * encodedSequencePitchInInts,
                                encodedSequencePitchInInts,
                                task.candidateSequenceData.data() + c * encodedSequencePitchInInts
                            );
                        }else{
                            //BestAlignment_t::ReverseComplement

                            std::copy_n(
                                task.candidateSequencesRevcData.data() + index * encodedSequencePitchInInts,
                                encodedSequencePitchInInts,
                                task.candidateSequenceData.data() + c * encodedSequencePitchInInts
                            );
                        }

                        // //not sure if these 2 arrays will be required further on
                        // std::copy_n(
                        //     candidateSequencesFwdData.data() + index * encodedSequencePitchInInts,
                        //     encodedSequencePitchInInts,
                        //     candidateSequencesFwdData.data() + c * encodedSequencePitchInInts
                        // );

                        // std::copy_n(
                        //     candidateSequencesRevcData.data() + index * encodedSequencePitchInInts,
                        //     encodedSequencePitchInInts,
                        //     candidateSequencesRevcData.data() + c * encodedSequencePitchInInts
                        // );
                        
                    }

                    //erase past-end elements
                    task.alignments.erase(
                        task.alignments.begin() + task.numRemainingCandidates, 
                        task.alignments.end()
                    );
                    task.alignmentFlags.erase(
                        task.alignmentFlags.begin() + task.numRemainingCandidates, 
                        task.alignmentFlags.end()
                    );
                    task.candidateReadIds.erase(
                        task.candidateReadIds.begin() + task.numRemainingCandidates, 
                        task.candidateReadIds.end()
                    );
                    task.candidateSequenceLengths.erase(
                        task.candidateSequenceLengths.begin() + task.numRemainingCandidates, 
                        task.candidateSequenceLengths.end()
                    );
                    // //not sure if these 2 arrays will be required further on
                    // candidateSequencesFwdData.erase(
                    //     candidateSequencesFwdData.begin() + task.numRemainingCandidates * encodedSequencePitchInInts, 
                    //     candidateSequencesFwdData.end()
                    // );
                    // candidateSequencesRevcData.erase(
                    //     candidateSequencesRevcData.begin() + task.numRemainingCandidates * encodedSequencePitchInInts, 
                    //     candidateSequencesRevcData.end()
                    // );
                    
                }

            }
            #endif
        };
        

        while(indicesOfActiveTasks.size() > 0){

            #ifdef checkdebugtasks
            auto debugtasks = tasks;
            processTasksOldStyle(debugtasks);
            #endif

            //perform one extension iteration for active tasks

            //setup batchdata for active tasks
            const int numActiveTasks = indicesOfActiveTasks.size();
            batchData.numTasks = numActiveTasks;

            batchData.h_numAnchors.resize(1);
            batchData.d_numAnchors.resize(1);
            batchData.h_numCandidates.resize(1);
            batchData.d_numCandidates.resize(1);
            batchData.d_numCandidates2.resize(1);

            batchData.h_anchorReadIds.resize(numActiveTasks);
            batchData.d_anchorReadIds.resize(numActiveTasks);
            batchData.h_mateReadIds.resize(numActiveTasks);
            batchData.d_mateReadIds.resize(numActiveTasks);
            
            batchData.h_subjectSequencesData.resize(encodedSequencePitchInInts * numActiveTasks);
            batchData.d_subjectSequencesData.resize(encodedSequencePitchInInts * numActiveTasks);
            batchData.h_anchorSequencesLength.resize(numActiveTasks);
            batchData.d_anchorSequencesLength.resize(numActiveTasks);

            batchData.h_anchormatedata.resize(numActiveTasks * encodedSequencePitchInInts);
            batchData.d_anchormatedata.resize(numActiveTasks * encodedSequencePitchInInts);

            batchData.h_inputanchormatedata.resize(numActiveTasks * encodedSequencePitchInInts);
            batchData.d_inputanchormatedata.resize(numActiveTasks * encodedSequencePitchInInts);

            batchData.h_numCandidatesPerAnchor.resize(numActiveTasks);
            batchData.d_numCandidatesPerAnchor.resize(numActiveTasks);
            batchData.h_numCandidatesPerAnchor2.resize(numActiveTasks);
            batchData.d_numCandidatesPerAnchor2.resize(numActiveTasks);
            batchData.h_numCandidatesPerAnchor3.resize(numActiveTasks);
            batchData.d_numCandidatesPerAnchor3.resize(numActiveTasks);
            batchData.h_numCandidatesPerAnchorPrefixSum.resize(numActiveTasks+1);
            batchData.h_numCandidatesPerAnchorPrefixSum2.resize(numActiveTasks+1);
            batchData.h_numCandidatesPerAnchorPrefixSum3.resize(numActiveTasks+1);
            batchData.d_numCandidatesPerAnchorPrefixSum.resize(numActiveTasks+1);
            batchData.d_numCandidatesPerAnchorPrefixSum2.resize(numActiveTasks+1);
            batchData.d_numCandidatesPerAnchorPrefixSum3.resize(numActiveTasks+1);

            batchData.h_numCandidatesPerAnchorPrefixSum[0] = 0;
            batchData.h_numCandidatesPerAnchorPrefixSum2[0] = 0;
            batchData.h_numCandidatesPerAnchorPrefixSum3[0] = 0;

            batchData.h_indexlist1.resize(numActiveTasks);
            batchData.d_indexlist1.resize(numActiveTasks);

            batchData.h_indexlist2.resize(numActiveTasks);
            batchData.d_indexlist2.resize(numActiveTasks);

            batchData.totalNumberOfUsedIds = 0;

            for(int t = 0; t < numActiveTasks; t++){
                auto& task = tasks[indicesOfActiveTasks[t]];
                task.dataIsAvailable = false;

                batchData.h_anchorReadIds[t] = task.myReadId;
                batchData.h_mateReadIds[t] = task.mateReadId;
                batchData.totalNumberOfUsedIds += task.allUsedCandidateReadIdPairs.size();

                std::copy(
                    task.encodedMate.begin(),
                    task.encodedMate.end(),
                    batchData.h_inputanchormatedata.begin() + t * encodedSequencePitchInInts
                );
    
                if(task.iteration >= 0){
    
                    batchData.h_anchorSequencesLength[t] = task.currentAnchorLength;
    
                    std::copy(
                        task.currentAnchor.begin(),
                        task.currentAnchor.end(),
                        batchData.h_subjectSequencesData.begin() + t * encodedSequencePitchInInts
                    );
                }else{
                    //only hash kmers which include extended positions
    
                    const int extendedPositionsPreviousIteration 
                        = task.totalAnchorBeginInExtendedRead.at(task.iteration) - task.totalAnchorBeginInExtendedRead.at(task.iteration-1);
    
                    const int lengthToHash = std::min(task.currentAnchorLength, kmerLength + extendedPositionsPreviousIteration - 1);
                    batchData.h_anchorSequencesLength[t] = lengthToHash;
    
                    //std::cerr << "lengthToHash = " << lengthToHash << "\n";
    
                    std::vector<char> buf(task.currentAnchorLength);
                    SequenceHelpers::decode2BitSequence(buf.data(), task.currentAnchor.data(), task.currentAnchorLength);
                    SequenceHelpers::encodeSequence2Bit(
                        batchData.h_subjectSequencesData.get() + t * encodedSequencePitchInInts, 
                        buf.data() + task.currentAnchorLength - lengthToHash, 
                        lengthToHash
                    );    

                    if(task.myReadId == 33){
                        std::cerr << "Anchor for iteration " << task.iteration << "\n";
                        assert(lengthToHash == task.currentAnchorLength);
                        for(auto c : buf){
                            std::cerr << c;
                        }
                        std::cerr << "\n";
                    }
                }
            }

            batchData.h_usedReadIds.resize(batchData.totalNumberOfUsedIds);
            batchData.h_numUsedReadIdsPerAnchor.resize(batchData.numTasks);
            batchData.h_numUsedReadIdsPerAnchorPrefixSum.resize(batchData.numTasks);
            batchData.d_segmentIds2.resize(batchData.totalNumberOfUsedIds);

            hipMemcpyAsync(
                batchData.d_inputanchormatedata.get(),
                batchData.h_inputanchormatedata.get(),
                sizeof(unsigned int) * numActiveTasks * encodedSequencePitchInInts,
                H2D,
                firstStream
            ); CUERR;            

            hipMemcpyAsync(
                batchData.d_subjectSequencesData.get(),
                batchData.h_subjectSequencesData.get(),
                sizeof(unsigned int) * numActiveTasks * encodedSequencePitchInInts,
                H2D,
                firstStream
            ); CUERR;
    
            hipMemcpyAsync(
                batchData.d_anchorSequencesLength.get(),
                batchData.h_anchorSequencesLength.get(),
                sizeof(int) * numActiveTasks,
                H2D,
                firstStream
            ); CUERR;

            hipMemcpyAsync(
                batchData.d_anchorReadIds.get(),
                batchData.h_anchorReadIds.get(),
                sizeof(read_number) * batchData.numTasks,
                H2D,
                secondStream
            ); CUERR;

            hipMemcpyAsync(
                batchData.d_mateReadIds.get(),
                batchData.h_mateReadIds.get(),
                sizeof(read_number) * batchData.numTasks,
                H2D,
                secondStream
            ); CUERR;

            batchData.h_numUsedReadIdsPerAnchorPrefixSum[0] = 0;
            {
                auto h_usedReadIdsIter = batchData.h_usedReadIds.begin();
                for(int i = 0; i < batchData.numTasks; i++){
                    auto& task = vecAccess(tasks, indicesOfActiveTasks[i]);
                    h_usedReadIdsIter = std::copy(
                        task.allUsedCandidateReadIdPairs.begin(),
                        task.allUsedCandidateReadIdPairs.end(),
                        h_usedReadIdsIter
                    );
                    batchData.h_numUsedReadIdsPerAnchor[i] = task.allUsedCandidateReadIdPairs.size();

                    if(i < batchData.numTasks - 1){
                        batchData.h_numUsedReadIdsPerAnchorPrefixSum[i+1] 
                            = batchData.h_numUsedReadIdsPerAnchorPrefixSum[i] + batchData.h_numUsedReadIdsPerAnchor[i];
                    }
                }
            }

            hashTimer.start();

            getCandidateReadIds(batchData, firstStream);

            hipMemcpyAsync(
                &batchData.totalNumCandidates,
                batchData.d_numCandidatesPerAnchorPrefixSum.get() + batchData.numTasks,
                sizeof(int),
                D2H,
                firstStream
            ); CUERR;
    
            hipStreamSynchronize(firstStream); CUERR;
            hipEventRecord(events[0], secondStream); CUERR;
            hipStreamWaitEvent(firstStream, events[0], 0); CUERR;

            batchData.h_candidateReadIds.resize(batchData.totalNumCandidates);

            batchData.d_anchorIndicesOfCandidates.resize(batchData.totalNumCandidates);
            batchData.d_anchorIndicesOfCandidates2.resize(batchData.totalNumCandidates);
            batchData.h_flagscandidates.resize(batchData.totalNumCandidates);
            batchData.d_flagscandidates.resize(batchData.totalNumCandidates);
            batchData.h_flagsanchors.resize(batchData.numTasks);
            batchData.d_flagsanchors.resize(batchData.numTasks);
            batchData.h_candidateReadIds2.resize(batchData.totalNumCandidates);
            batchData.d_candidateReadIds2.resize(batchData.totalNumCandidates);
            batchData.h_candidateReadIds3.resize(batchData.totalNumCandidates);

            //determine required temp bytes for following cub calls, and allocate temp storage

            std::size_t cubBytes = 0;
            std::size_t cubBytes2 = 0;

            hipcub::DeviceSelect::Flagged(
                nullptr,
                cubBytes2,
                batchData.d_candidateReadIds.data(),
                batchData.d_flagscandidates.data(),
                batchData.d_candidateReadIds2.data(),
                batchData.h_numCandidates.data(),
                batchData.totalNumCandidates,
                firstStream
            );
            
            cubBytes = std::max(cubBytes, cubBytes2);
            
            hipcub::DeviceScan::InclusiveSum(
                nullptr, 
                cubBytes2, 
                batchData.d_numCandidatesPerAnchor2.data(), 
                batchData.d_numCandidatesPerAnchorPrefixSum2.data() + 1, 
                batchData.numTasks,
                firstStream
            );
            
            cubBytes = std::max(cubBytes, cubBytes2);
            
            hipcub::DeviceScan::InclusiveScan(
                nullptr, 
                cubBytes2, 
                batchData.d_anchorIndicesOfCandidates.data(), 
                batchData.d_anchorIndicesOfCandidates.data(), 
                hipcub::Max{},
                batchData.totalNumCandidates,
                firstStream
            );
            
            cubBytes = std::max(cubBytes, cubBytes2);
            
            hipcub::DeviceScan::InclusiveScan(
                nullptr, 
                cubBytes2, 
                batchData.d_segmentIds2.data(), 
                batchData.d_segmentIds2.data(), 
                hipcub::Max{},
                batchData.totalNumberOfUsedIds,
                firstStream
            );
            
            cubBytes = std::max(cubBytes, cubBytes2);
            
            hipcub::DeviceScan::InclusiveSum(
                nullptr, 
                cubBytes2, 
                batchData.d_numCandidatesPerAnchor.data(), 
                batchData.d_numCandidatesPerAnchorPrefixSum.data() + 1, 
                batchData.numTasks,
                firstStream
            );
            
            cubBytes = std::max(cubBytes, cubBytes2);
            
            hipcub::DeviceSelect::Flagged(
                nullptr,
                cubBytes2,
                thrust::make_counting_iterator(0),
                batchData.d_flagsanchors.data(),
                batchData.d_indexlist1.data(),
                batchData.h_numAnchors.data(),
                batchData.numTasks,
                firstStream
            );
            
            cubBytes = std::max(cubBytes, cubBytes2);
            
            hipcub::DeviceSelect::Flagged(
                nullptr,
                cubBytes2,
                batchData.d_inputanchormatedata.data(),
                thrust::make_transform_iterator(
                    thrust::make_counting_iterator(0),
                    SequenceFlagMultiplier{batchData.d_flagsanchors.data(), int(encodedSequencePitchInInts)}
                ),
                batchData.d_anchormatedata.data(),
                thrust::make_discard_iterator(),
                batchData.numTasks * encodedSequencePitchInInts,
                firstStream
            );
            
            cubBytes = std::max(cubBytes, cubBytes2);

            //auto cubtempstorage = thrustCachingAllocator1.allocate(cubBytes);

            void* cubtempstorage; cubAllocator->DeviceAllocate((void**)&cubtempstorage, cubBytes, firstStream);

            //cub temp allocated          
            
            helpers::call_fill_kernel_async(batchData.d_flagscandidates.data(), batchData.d_flagscandidates.size(), false, firstStream);
            
            //flag candidates to remove because they are equal to anchor id or equal to mate id
            flagCandidateIdsWhichAreEqualToAnchorOrMateKernel<<<4096, 128, 0, firstStream>>>(
                batchData.d_candidateReadIds.data(),
                batchData.d_anchorReadIds.data(),
                batchData.d_mateReadIds.data(),
                batchData.d_numCandidatesPerAnchorPrefixSum.data(),
                batchData.d_numCandidatesPerAnchor.data(),
                batchData.d_flagscandidates.data(),
                batchData.d_flagsanchors.data(),
                batchData.d_numCandidatesPerAnchor2.data(),
                batchData.numTasks,
                tasks[0].pairedEnd
            );
            CUERR;

            //copy selected candidate ids

            hipcub::DeviceSelect::Flagged(
                cubtempstorage,
                cubBytes,
                batchData.d_candidateReadIds.data(),
                batchData.d_flagscandidates.data(),
                batchData.d_candidateReadIds2.data(),
                batchData.h_numCandidates.data(),
                batchData.totalNumCandidates,
                firstStream
            );

            hipStreamSynchronize(firstStream); CUERR;

            batchData.totalNumCandidates = *batchData.h_numCandidates;

            //compute prefix sum of number of candidates per anchor
            helpers::call_set_kernel_async(batchData.d_numCandidatesPerAnchorPrefixSum2.data(), 0, 0, firstStream);

            hipcub::DeviceScan::InclusiveSum(
                cubtempstorage, 
                cubBytes, 
                batchData.d_numCandidatesPerAnchor2.data(), 
                batchData.d_numCandidatesPerAnchorPrefixSum2.data() + 1, 
                batchData.numTasks,
                firstStream
            );

            //compute segment ids for candidate read ids
            helpers::call_fill_kernel_async(batchData.d_anchorIndicesOfCandidates.data(), batchData.totalNumCandidates, 0, firstStream);

            setFirstSegmentIdsKernel<<<SDIV(batchData.numTasks, 256), 256, 0, firstStream>>>(
                batchData.d_numCandidatesPerAnchor2.data(),
                batchData.d_anchorIndicesOfCandidates.data(),
                batchData.d_numCandidatesPerAnchorPrefixSum2.data(),
                batchData.numTasks
            );

            hipcub::DeviceScan::InclusiveScan(
                cubtempstorage, 
                cubBytes, 
                batchData.d_anchorIndicesOfCandidates.data(), 
                batchData.d_anchorIndicesOfCandidates.data(), 
                hipcub::Max{},
                batchData.totalNumCandidates,
                firstStream
            );

            //compute segment ids for used candidate read ids
            helpers::call_fill_kernel_async(batchData.d_segmentIds2.data(), batchData.totalNumberOfUsedIds, 0, firstStream);

            setFirstSegmentIdsKernel<<<SDIV(batchData.numTasks, 256), 256, 0, firstStream>>>(
                batchData.h_numUsedReadIdsPerAnchor.data(),
                batchData.d_segmentIds2.data(),
                batchData.h_numUsedReadIdsPerAnchorPrefixSum.data(),
                batchData.numTasks
            );

            hipcub::DeviceScan::InclusiveScan(
                cubtempstorage, 
                cubBytes, 
                batchData.d_segmentIds2.data(), 
                batchData.d_segmentIds2.data(), 
                hipcub::Max{},
                batchData.totalNumberOfUsedIds,
                firstStream
            );
            
            //compute segmented set difference between candidate read ids and used candidate read ids
            auto d_candidateReadIds_end = GpuSegmentedSetOperation{}.difference(
                thrustCachingAllocator1,
                batchData.d_candidateReadIds2.data(),
                batchData.d_numCandidatesPerAnchor2.data(),
                batchData.d_numCandidatesPerAnchorPrefixSum2.data(),
                batchData.d_anchorIndicesOfCandidates.data(),
                batchData.totalNumCandidates,
                batchData.h_usedReadIds.data(),
                batchData.h_numUsedReadIdsPerAnchor.data(),
                batchData.h_numUsedReadIdsPerAnchorPrefixSum.data(),
                batchData.d_segmentIds2.data(),
                batchData.totalNumberOfUsedIds,
                batchData.numTasks,        
                batchData.d_candidateReadIds.data(),
                batchData.d_numCandidatesPerAnchor.data(),
                batchData.d_anchorIndicesOfCandidates2.data(),
                firstStream
            );

            std::swap(batchData.d_anchorIndicesOfCandidates, batchData.d_anchorIndicesOfCandidates2);

            batchData.totalNumCandidates = std::distance(batchData.d_candidateReadIds.data(), d_candidateReadIds_end);

            //compute prefix sum of new segment sizes
            hipcub::DeviceScan::InclusiveSum(
                cubtempstorage, 
                cubBytes, 
                batchData.d_numCandidatesPerAnchor.data(), 
                batchData.d_numCandidatesPerAnchorPrefixSum.data() + 1, 
                batchData.numTasks,
                firstStream
            );

            //determine task ids with removed mates

            hipcub::DeviceSelect::Flagged(
                cubtempstorage,
                cubBytes,
                thrust::make_counting_iterator(0),
                batchData.d_flagsanchors.data(),
                batchData.d_indexlist1.data(),
                batchData.h_numAnchors.data(),
                batchData.numTasks,
                firstStream
            );

            hipStreamSynchronize(firstStream); CUERR;

            batchData.numTasksWithMateRemoved = *batchData.h_numAnchors;

            if(batchData.numTasksWithMateRemoved > 0){

                //copy mate sequence data of removed mates
                    
                hipcub::DeviceSelect::Flagged(
                    cubtempstorage,
                    cubBytes,
                    batchData.d_inputanchormatedata.data(),
                    thrust::make_transform_iterator(
                        thrust::make_counting_iterator(0),
                        SequenceFlagMultiplier{batchData.d_flagsanchors.data(), int(encodedSequencePitchInInts)}
                    ),
                    batchData.d_anchormatedata.data(),
                    thrust::make_discard_iterator(),
                    batchData.numTasks * encodedSequencePitchInInts,
                    firstStream
                );
            }

            cubAllocator->DeviceFree(cubtempstorage);

            hashTimer.stop();
   
            //allocate data for candidate sequences 
            batchData.h_candidateSequencesLength.resize(batchData.totalNumCandidates);
            batchData.h_candidateSequencesData.resize(encodedSequencePitchInInts * batchData.totalNumCandidates);
            batchData.h_candidateSequencesRevcData.resize(encodedSequencePitchInInts * batchData.totalNumCandidates);

            batchData.d_candidateSequencesLength.resize(batchData.totalNumCandidates);
            batchData.d_candidateSequencesData.resize(encodedSequencePitchInInts * batchData.totalNumCandidates);
            batchData.d_candidateSequencesRevcData.resize(encodedSequencePitchInInts * batchData.totalNumCandidates);

            batchData.d_candidateSequencesLength2.resize(batchData.totalNumCandidates);
            batchData.d_candidateSequencesData2.resize(encodedSequencePitchInInts * batchData.totalNumCandidates);
            batchData.d_candidateSequencesRevcData2.resize(encodedSequencePitchInInts * batchData.totalNumCandidates);
            batchData.d_candidateReadIds2.resize(batchData.totalNumCandidates);

            batchData.h_intbuffercandidates.resize(batchData.totalNumCandidates);
            batchData.d_intbuffercandidates.resize(batchData.totalNumCandidates);
            batchData.h_flagscandidates.resize(batchData.totalNumCandidates);
            batchData.d_flagscandidates.resize(batchData.totalNumCandidates);

            batchData.h_alignment_overlaps.resize(batchData.totalNumCandidates);
            batchData.h_alignment_shifts.resize(batchData.totalNumCandidates);
            batchData.h_alignment_nOps.resize(batchData.totalNumCandidates);
            batchData.h_alignment_isValid.resize(batchData.totalNumCandidates);
            batchData.h_alignment_best_alignment_flags.resize(batchData.totalNumCandidates);

            batchData.d_alignment_overlaps.resize(batchData.totalNumCandidates);
            batchData.d_alignment_shifts.resize(batchData.totalNumCandidates);
            batchData.d_alignment_nOps.resize(batchData.totalNumCandidates);
            batchData.d_alignment_isValid.resize(batchData.totalNumCandidates);
            batchData.d_alignment_best_alignment_flags.resize(batchData.totalNumCandidates);

            batchData.d_alignment_overlaps2.resize(batchData.totalNumCandidates);
            batchData.d_alignment_shifts2.resize(batchData.totalNumCandidates);
            batchData.d_alignment_nOps2.resize(batchData.totalNumCandidates);
            batchData.d_alignment_isValid2.resize(batchData.totalNumCandidates);
            batchData.d_alignment_best_alignment_flags2.resize(batchData.totalNumCandidates);

            collectTimer.start();

            loadCandidateSequenceData(batchData, firstStream);      

            eraseDataOfRemovedMates(batchData, firstStream);

            thrust::copy_n(
                thrustPolicy1,
                thrust::make_zip_iterator(thrust::make_tuple(
                    batchData.d_numCandidatesPerAnchorPrefixSum.data() + 1,
                    batchData.d_numCandidatesPerAnchor.data()
                )),
                batchData.numTasks,
                thrust::make_zip_iterator(thrust::make_tuple(
                    batchData.h_numCandidatesPerAnchorPrefixSum.data() + 1,
                    batchData.h_numCandidatesPerAnchor.data()
                ))
            );

            // hipMemcpyAsync(
            //     batchData.h_numCandidatesPerAnchorPrefixSum.get(),
            //     batchData.d_numCandidatesPerAnchorPrefixSum.get(),
            //     sizeof(int) * (batchData.numTasks+1),
            //     D2H,
            //     firstStream
            // ); CUERR;

            // hipMemcpyAsync(
            //     batchData.h_numCandidatesPerAnchor.get(),
            //     batchData.d_numCandidatesPerAnchor.get(),
            //     sizeof(int) * batchData.numTasks,
            //     D2H,
            //     firstStream
            // ); CUERR;
    
            hipStreamSynchronize(firstStream); CUERR;

            batchData.totalNumCandidates = batchData.h_numCandidatesPerAnchorPrefixSum[batchData.numTasks];
    
            #ifdef checkdebugtasks
            hipMemcpyAsync(
                batchData.h_candidateSequencesData.get(),
                batchData.d_candidateSequencesData.get(),
                sizeof(unsigned int) * batchData.totalNumCandidates * encodedSequencePitchInInts,
                H2D,
                secondStream
            ); CUERR;
    
            hipMemcpyAsync(
                batchData.h_candidateSequencesRevcData.get(),
                batchData.d_candidateSequencesRevcData.get(),
                sizeof(unsigned int) * batchData.totalNumCandidates * encodedSequencePitchInInts,
                H2D,
                secondStream
            ); CUERR;
            #endif
            


            collectTimer.stop();

            /*
                Compute alignments
            */

            alignmentTimer.start();

            calculateAlignments(batchData, firstStream);

            alignmentTimer.stop();

            

            #ifdef checkdebugtasks
            hipStreamSynchronize(secondStream); CUERR;
            // unpack batchData sequences into tasks
            for(int i = 0; i < numActiveTasks; i++){
                auto& task = vecAccess(tasks, indicesOfActiveTasks[i]);

                const int numCandidates = batchData.h_numCandidatesPerAnchor[i];
                const int offset = batchData.h_numCandidatesPerAnchorPrefixSum[i];

                task.candidateSequencesFwdData.resize(numCandidates * encodedSequencePitchInInts);
                std::copy_n(batchData.h_candidateSequencesData.data() + offset * encodedSequencePitchInInts, numCandidates * encodedSequencePitchInInts, task.candidateSequencesFwdData.begin());

                task.candidateSequencesRevcData.resize(numCandidates * encodedSequencePitchInInts);
                std::copy_n(batchData.h_candidateSequencesRevcData.data() + offset * encodedSequencePitchInInts, numCandidates * encodedSequencePitchInInts, task.candidateSequencesRevcData.begin());
            }
            #endif
           
       
            filterAlignments(batchData, firstStream);

            thrust::copy_n(
                thrustPolicy1,
                thrust::make_zip_iterator(thrust::make_tuple(
                    batchData.d_numCandidatesPerAnchorPrefixSum.data() + 1,
                    batchData.d_numCandidatesPerAnchor.data()
                )),
                batchData.numTasks,
                thrust::make_zip_iterator(thrust::make_tuple(
                    batchData.h_numCandidatesPerAnchorPrefixSum.data() + 1,
                    batchData.h_numCandidatesPerAnchor.data()
                ))
            );
    
            hipStreamSynchronize(firstStream); CUERR;

            batchData.totalNumCandidates = batchData.h_numCandidatesPerAnchorPrefixSum[batchData.numTasks];

            hipMemcpyAsync(
                batchData.h_candidateSequencesData.get(),
                batchData.d_candidateSequencesData.get(),
                sizeof(unsigned int) * batchData.totalNumCandidates * encodedSequencePitchInInts,
                H2D,
                firstStream
            ); CUERR;

            auto d_zipped_begin = thrust::make_zip_iterator(
                thrust::make_tuple(
                    batchData.d_candidateReadIds.data(),
                    batchData.d_candidateSequencesLength.data(),
                    batchData.d_alignment_overlaps.data(),
                    batchData.d_alignment_isValid.data(),
                    batchData.d_alignment_shifts.data(),
                    batchData.d_alignment_nOps.data(),
                    batchData.d_alignment_best_alignment_flags.data()
                )
            );

            auto h_zipped_begin = thrust::make_zip_iterator(
                thrust::make_tuple(
                    batchData.h_candidateReadIds.data(),
                    batchData.h_candidateSequencesLength.data(),
                    batchData.h_alignment_overlaps.data(),
                    batchData.h_alignment_isValid.data(),
                    batchData.h_alignment_shifts.data(),
                    batchData.h_alignment_nOps.data(),
                    batchData.h_alignment_best_alignment_flags.data()
                )
            );

            thrust::copy_n(
                thrustPolicy1,
                d_zipped_begin,
                batchData.totalNumCandidates,
                h_zipped_begin
            );

            hipStreamSynchronize(firstStream); CUERR;

            // unpack batchData into tasks

            auto copyBatchDataIntoTask = [&](auto& task, int taskindex){
                const int numCandidates = batchData.h_numCandidatesPerAnchor[taskindex];
                const int offset = batchData.h_numCandidatesPerAnchorPrefixSum[taskindex];

                task.candidateReadIds.resize(numCandidates);
                std::copy_n(batchData.h_candidateReadIds.data() + offset, numCandidates, task.candidateReadIds.begin());

                task.candidateSequenceLengths.resize(numCandidates);
                std::copy_n(batchData.h_candidateSequencesLength.data() + offset, numCandidates, task.candidateSequenceLengths.begin());

                task.candidateSequenceData.resize(numCandidates * encodedSequencePitchInInts);
                std::copy_n(
                    batchData.h_candidateSequencesData.data() + offset * encodedSequencePitchInInts, 
                    numCandidates * encodedSequencePitchInInts, 
                    task.candidateSequenceData.begin()
                );

                task.alignmentFlags.resize(numCandidates);
                task.alignments.resize(numCandidates);

                for(int c = 0; c < numCandidates; c++){
                    task.alignments[c].shift = batchData.h_alignment_shifts[offset + c];
                    task.alignments[c].overlap = batchData.h_alignment_overlaps[offset + c];
                    task.alignments[c].nOps = batchData.h_alignment_nOps[offset + c];
                    #ifdef checkdebugtasks
                    task.alignments[c].isValid = batchData.h_alignment_isValid[offset + c];
                    #endif
                    task.alignmentFlags[c] = batchData.h_alignment_best_alignment_flags[offset + c];
                }

                task.mateRemovedFromCandidates = false; //debug. not required
                task.numRemainingCandidates = numCandidates;

                if(task.numRemainingCandidates == 0){
                    task.abort = true;
                    task.abortReason = AbortReason::NoPairedCandidatesAfterAlignment;
                }
            };


            for(int i = 0; i < numActiveTasks; i++){
                auto& task = vecAccess(tasks, indicesOfActiveTasks[i]);

                #ifdef checkdebugtasks
                copyBatchDataIntoTask(task, i);
                task.dataIsAvailable = true;
                #else
                task.mateRemovedFromCandidates = false; //debug. not required
                task.numRemainingCandidates = batchData.h_numCandidatesPerAnchor[i];

                if(task.numRemainingCandidates == 0){
                    task.abort = true;
                    task.abortReason = AbortReason::NoPairedCandidatesAfterAlignment;
                }
                #endif
            }

            #ifdef checkdebugtasks
            for(int i = 0; i < numActiveTasks; i++){
                auto& newtask = tasks[indicesOfActiveTasks[i]];
                auto& oldtask = debugtasks[indicesOfActiveTasks[i]];

                if(newtask != oldtask){
                    std::cerr << "old task and new task differ. i=" 
                        << i << ", indicesOfActiveTasks[i] " << indicesOfActiveTasks[i] << "\n";
                    assert(false);
                }
            }
            #endif

            if(true){
                //construct gpu msa
                
                batchData.h_consensus.resize(numActiveTasks * msaColumnPitchInElements);
                batchData.h_msa_column_properties.resize(numActiveTasks);

                batchData.d_consensus.resize(numActiveTasks * msaColumnPitchInElements);
                batchData.d_support.resize(numActiveTasks * msaColumnPitchInElements);
                batchData.d_coverage.resize(numActiveTasks * msaColumnPitchInElements);
                batchData.d_origWeights.resize(numActiveTasks * msaColumnPitchInElements);
                batchData.d_origCoverages.resize(numActiveTasks * msaColumnPitchInElements);
                batchData.d_msa_column_properties.resize(numActiveTasks);
                batchData.d_counts.resize(numActiveTasks * 4 * msaColumnPitchInElements);
                batchData.d_weights.resize(numActiveTasks * 4 * msaColumnPitchInElements);

                batchData.d_intbuffercandidates.resize(batchData.totalNumCandidates);
                batchData.d_indexlist1.resize(batchData.totalNumCandidates);
                batchData.d_numCandidatesPerAnchor2.resize(batchData.numTasks);
                batchData.d_flagscandidates.resize(batchData.totalNumCandidates);

                helpers::call_fill_kernel_async(batchData.d_msa_column_properties.data(), numActiveTasks, gpu::MSAColumnProperties{-1,-1,-1,-1}, firstStream); CUERR;                

                int* const indices1 = batchData.d_intbuffercandidates.data();
                int* const indices2 = batchData.d_indexlist1.data();

                helpers::lambda_kernel<<<batchData.numTasks, 128, 0, firstStream>>>(
                    [
                        indices1,
                        d_numCandidatesPerAnchor = batchData.d_numCandidatesPerAnchor.get(),
                        d_numCandidatesPerAnchorPrefixSum = batchData.d_numCandidatesPerAnchorPrefixSum.get()
                    ] __device__ (){
                        const int num = d_numCandidatesPerAnchor[blockIdx.x];
                        const int offset = d_numCandidatesPerAnchorPrefixSum[blockIdx.x];
                        
                        for(int i = threadIdx.x; i < num; i += blockDim.x){
                            indices1[offset + i] = i;
                        }
                    }
                );

                gpu::GPUMultiMSA multiMSA;

                *batchData.h_numAnchors = numActiveTasks;

                multiMSA.numMSAs = numActiveTasks;
                multiMSA.columnPitchInElements = msaColumnPitchInElements;
                multiMSA.counts = batchData.d_counts.get();
                multiMSA.weights = batchData.d_weights.get();
                multiMSA.coverages = batchData.d_coverage.get();
                multiMSA.consensus = batchData.d_consensus.get();
                multiMSA.support = batchData.d_support.get();
                multiMSA.origWeights = batchData.d_origWeights.get();
                multiMSA.origCoverages = batchData.d_origCoverages.get();
                multiMSA.columnProperties = batchData.d_msa_column_properties.get();

                //sequences which have alignment flag BestAlignment_t::ReverseComplement are already present as reverse complement in d_candidateSequencesData.
                //To not reverse complement them again during msa construction, treat all stored sequences are forward
                helpers::call_fill_kernel_async(batchData.d_alignment_best_alignment_flags2.data(), batchData.totalNumCandidates, BestAlignment_t::Forward, firstStream); CUERR;

                callConstructMultipleSequenceAlignmentsKernel_async(
                    multiMSA,
                    batchData.d_alignment_overlaps.get(),
                    batchData.d_alignment_shifts.get(),
                    batchData.d_alignment_nOps.get(),
                    //batchData.d_alignment_best_alignment_flags.get(),
                    batchData.d_alignment_best_alignment_flags2.get(),
                    batchData.d_anchorSequencesLength.get(),
                    batchData.d_candidateSequencesLength.get(),
                    indices1, //d_indices,
                    batchData.d_numCandidatesPerAnchor.get(),
                    batchData.d_numCandidatesPerAnchorPrefixSum.get(),
                    batchData.d_subjectSequencesData.get(),
                    batchData.d_candidateSequencesData.get(),
                    nullptr, //d_anchor_qualities.get(),
                    nullptr, //d_candidate_qualities.get(),
                    batchData.h_numAnchors.get(), //d_numAnchors
                    goodAlignmentProperties.maxErrorRate,
                    batchData.numTasks,
                    batchData.totalNumCandidates,
                    false, //correctionOptions->useQualityScores,
                    encodedSequencePitchInInts,
                    qualityPitchInBytes,
                    firstStream,
                    kernelLaunchHandle
                );

                //refine msa
                bool* d_shouldBeKept = (bool*)batchData.d_flagscandidates.get();
                helpers::call_fill_kernel_async(indices2, batchData.totalNumCandidates, -42, firstStream); CUERR;
                helpers::call_fill_kernel_async(batchData.d_numCandidatesPerAnchor2.data(), batchData.numTasks, -13, firstStream); CUERR;
                helpers::call_fill_kernel_async(batchData.d_numCandidates2.data(), 1, -9, firstStream); CUERR;
                callMsaCandidateRefinementKernel_multiiter_async(
                    indices2,
                    batchData.d_numCandidatesPerAnchor2.data(),
                    batchData.d_numCandidates2.get(),
                    multiMSA,
                    //batchData.d_alignment_best_alignment_flags.get(),
                    batchData.d_alignment_best_alignment_flags2.get(),
                    batchData.d_alignment_shifts.get(),
                    batchData.d_alignment_nOps.get(),
                    batchData.d_alignment_overlaps.get(),
                    batchData.d_subjectSequencesData.get(),
                    batchData.d_candidateSequencesData.get(),
                    batchData.d_anchorSequencesLength.get(),
                    batchData.d_candidateSequencesLength.get(),
                    nullptr, //d_anchor_qualities.get(),
                    nullptr, //d_candidate_qualities.get(),
                    d_shouldBeKept,
                    batchData.d_numCandidatesPerAnchorPrefixSum.get(),
                    batchData.h_numAnchors.get(),
                    goodAlignmentProperties.maxErrorRate,
                    batchData.numTasks,
                    batchData.totalNumCandidates,
                    false, //correctionOptions->useQualityScores,
                    encodedSequencePitchInInts,
                    qualityPitchInBytes,
                    indices1, //d_indices,
                    batchData.d_numCandidatesPerAnchor.get(),
                    correctionOptions.estimatedCoverage,
                    getNumRefinementIterations(),
                    firstStream,
                    kernelLaunchHandle
                );

                if(tasks[1].myReadId == 3 && tasks[1].iteration == 0){
                    helpers::lambda_kernel<<<1, 1, 0, firstStream>>>(
                        [
                            shifts = batchData.d_alignment_shifts.data(),
                            d_numCandidatesPerAnchor = batchData.d_numCandidatesPerAnchor.data(),
                            d_counts = batchData.d_counts.data(),
                            d_coverage = batchData.d_coverage.data(),
                            msaColumnPitchInElements = this->msaColumnPitchInElements,
                            d_msa_column_properties = batchData.d_msa_column_properties.data(),
                            numTasks = batchData.numTasks,
                            splitcolumnsPitchElements = 32,
                            d_possibleSplitColumns = batchData.d_possibleSplitColumns.data(),
                            d_numPossibleSplitColumnsPerTask = batchData.d_numPossibleSplitColumnsPerAnchor.data(),
                            d_candidateSequencesLength = batchData.d_candidateSequencesLength.data(),
                            d_candidateSequencesData = batchData.d_candidateSequencesData.data(),
                            encodedSequencePitchInInts = this->encodedSequencePitchInInts,
                            d_subjectSequencesData = batchData.d_subjectSequencesData.data(),
                            d_numCandidatesPerAnchorPrefixSum = batchData.d_numCandidatesPerAnchorPrefixSum.data()
                        ] __device__ (){
        
                            const int task = 1;

                            const int offset = d_numCandidatesPerAnchorPrefixSum[task];
    
                                const int firstColumn = d_msa_column_properties[task].firstColumn_incl;
                                const int lastColumnExcl = d_msa_column_properties[task].lastColumn_excl;                                

                                int* myCountsPtr[4];
                                myCountsPtr[0] = d_counts + 4 * msaColumnPitchInElements * task + 0 * msaColumnPitchInElements;
                                myCountsPtr[1] = d_counts + 4 * msaColumnPitchInElements * task + 1 * msaColumnPitchInElements;
                                myCountsPtr[2] = d_counts + 4 * msaColumnPitchInElements * task + 2 * msaColumnPitchInElements;
                                myCountsPtr[3] = d_counts + 4 * msaColumnPitchInElements * task + 3 * msaColumnPitchInElements;

                                printf("anchor\n");
                                for (int k = 0; k < 100; k++){
                                    const std::uint8_t base = SequenceHelpers::getEncodedNuc2Bit(d_subjectSequencesData + task * encodedSequencePitchInInts, 100, k);
                                    const char aaaa = SequenceHelpers::decodeBase(base);
                                    printf("%c", aaaa);
                                }
                                printf("\n");

                                printf("candidates:\n");
                                for(int i = 0; i < d_numCandidatesPerAnchor[task]; i++){
                                    printf("len %d\n", d_candidateSequencesLength[i]);
                                    for (int k = 0; k < d_candidateSequencesLength[i]; k++){
                                        const std::uint8_t base = SequenceHelpers::getEncodedNuc2Bit(
                                            d_candidateSequencesData + (offset + i) * encodedSequencePitchInInts, 
                                            d_candidateSequencesLength[i], 
                                            k
                                        );
                                        const char aaaa = SequenceHelpers::decodeBase(base);
                                        printf("%c", aaaa);
                                    }
                                    printf("\n");
                                }
                                printf("\n");

                                printf("shifts:\n");
                                for(int i = 0; i < d_numCandidatesPerAnchor[task]; i++){
                                    printf("%d ", shifts[offset + i]);
                                }
                                printf("\n");

                                printf("A:\n");
                                for(int i = firstColumn; i < lastColumnExcl; i++){
                                    printf("%d ", myCountsPtr[0][i]);
                                }
                                printf("\n");

                                printf("C:\n");
                                for(int i = firstColumn; i < lastColumnExcl; i++){
                                    printf("%d ", myCountsPtr[1][i]);
                                }
                                printf("\n");

                                printf("G:\n");
                                for(int i = firstColumn; i < lastColumnExcl; i++){
                                    printf("%d ", myCountsPtr[2][i]);
                                }
                                printf("\n");

                                printf("T:\n");
                                for(int i = firstColumn; i < lastColumnExcl; i++){
                                    printf("%d ", myCountsPtr[3][i]);
                                }
                                printf("\n");


                                printf("cov:\n");
                                for(int i = firstColumn; i < lastColumnExcl; i++){
                                    printf("%d ", d_coverage[i]);
                                }
                                printf("\n");
                        }
                    ); CUERR;

                    hipDeviceSynchronize(); CUERR; //DEBUG
                }



                hipEventRecord(events[0], firstStream); CUERR;
                hipStreamWaitEvent(secondStream, events[0], 0); CUERR;

                hipMemcpyAsync(
                    batchData.h_numCandidates.data(),
                    batchData.d_numCandidates.data(),
                    sizeof(int),
                    D2H,
                    secondStream
                ); CUERR;

                hipEventRecord(events[0], secondStream); CUERR;

                hipMemcpyAsync(
                    batchData.h_msa_column_properties.data(),
                    batchData.d_msa_column_properties.data(),
                    sizeof(gpu::MSAColumnProperties) * batchData.numTasks,
                    D2H,
                    secondStream
                ); CUERR;

                //convert encoded consensus to characters and copy to host
                helpers::lambda_kernel<<<batchData.numTasks, 128, 0, secondStream>>>(
                    [
                        msaColumnPitchInElements = this->msaColumnPitchInElements,
                        d_consensus = batchData.d_consensus.data(),
                        h_consensus = batchData.h_consensus.data()
                    ] __device__ (){
                        
                        for(int i = threadIdx.x; i < msaColumnPitchInElements; i += blockDim.x){
                            std::uint8_t encoded = d_consensus[blockIdx.x * msaColumnPitchInElements + i];

                            char decoded = 'F';
                            if(encoded == std::uint8_t{0}){
                                decoded = 'A';
                            }else if(encoded == std::uint8_t{1}){
                                decoded = 'C';
                            }else if(encoded == std::uint8_t{2}){
                                decoded = 'G';
                            }else if(encoded == std::uint8_t{3}){
                                decoded = 'T';
                            }

                            h_consensus[blockIdx.x * msaColumnPitchInElements + i] = decoded;
                        }
                    }
                );



                bool cubdebugsync = false;

                //allocate cub storage
                std::size_t cubBytes = 0;
                std::size_t cubBytes2 = 0;
                hipError_t cubstatus = hipcub::DeviceScan::InclusiveSum(
                    nullptr,
                    cubBytes,
                    batchData.d_numCandidatesPerAnchor2.get(), 
                    batchData.d_numCandidatesPerAnchorPrefixSum2.get() + 1, 
                    batchData.numTasks, 
                    firstStream,
                    cubdebugsync
                );
                assert(cubstatus == hipSuccess);

                auto in_zipped_begin = thrust::make_zip_iterator(
                    thrust::make_tuple(
                        batchData.d_candidateReadIds.data(),
                        batchData.d_candidateSequencesLength.data(),
                        batchData.d_alignment_overlaps.data(),
                        batchData.d_alignment_isValid.data(),
                        batchData.d_alignment_shifts.data(),
                        batchData.d_alignment_nOps.data(),
                        batchData.d_alignment_best_alignment_flags.data()
                    )
                );
    
                auto out_zipped_begin = thrust::make_zip_iterator(
                    thrust::make_tuple(
                        batchData.d_candidateReadIds2.data(),
                        batchData.d_candidateSequencesLength2.data(),
                        batchData.d_alignment_overlaps2.data(),
                        batchData.d_alignment_isValid2.data(),
                        batchData.d_alignment_shifts2.data(),
                        batchData.d_alignment_nOps2.data(),
                        batchData.d_alignment_best_alignment_flags2.data()
                    )
                );

                cubstatus = hipcub::DeviceSelect::Flagged(
                    nullptr,
                    cubBytes2,
                    in_zipped_begin,
                    batchData.d_flagscandidates.data(),
                    out_zipped_begin,
                    thrust::make_discard_iterator(),
                    batchData.totalNumCandidates,
                    firstStream,
                    cubdebugsync
                );

                assert(cubstatus == hipSuccess);
                cubBytes = std::max(cubBytes, cubBytes2);

                cubstatus = hipcub::DeviceSelect::Flagged(
                    nullptr,
                    cubBytes2,
                    batchData.d_candidateSequencesData.data(),
                    thrust::make_transform_iterator(
                        thrust::make_counting_iterator(0),
                        SequenceFlagMultiplier{batchData.d_flagscandidates.data(), int(encodedSequencePitchInInts)}
                    ),
                    batchData.d_candidateSequencesData2.data(),
                    thrust::make_discard_iterator(),
                    batchData.totalNumCandidates * encodedSequencePitchInInts,
                    firstStream,
                    cubdebugsync
                );

                assert(cubstatus == hipSuccess);
                cubBytes = std::max(cubBytes, cubBytes2);

                void* cubtemp; cubAllocator->DeviceAllocate((void**)&cubtemp, cubBytes, firstStream);

                cubstatus = hipcub::DeviceScan::InclusiveSum(
                    cubtemp,
                    cubBytes,
                    batchData.d_numCandidatesPerAnchor2.get(), 
                    batchData.d_numCandidatesPerAnchorPrefixSum2.get() + 1, 
                    batchData.numTasks, 
                    firstStream,
                    cubdebugsync
                );
                assert(cubstatus == hipSuccess);

                helpers::call_fill_kernel_async(batchData.d_flagscandidates.data(), batchData.totalNumCandidates, false, firstStream); CUERR;

                //convert output indices from task-local indices to global indices
                helpers::lambda_kernel<<<batchData.numTasks, 128, 0, firstStream>>>(
                    [
                        d_flagscandidates = batchData.d_flagscandidates.data(),
                        indices2,
                        d_numCandidatesPerAnchor2 = batchData.d_numCandidatesPerAnchor2.get(),
                        d_numCandidatesPerAnchorPrefixSum = batchData.d_numCandidatesPerAnchorPrefixSum.get()
                    ] __device__ (){
                        /*
                            Input:
                            indices2: 0,1,2,0,0,0,0,3,5,0
                            d_numCandidatesPerAnchorPrefixSum: 0,6,10

                            Output:
                            d_flagscandidates: 1,1,1,0,0,0,1,0,0,1,0,1
                        */
                        const int num = d_numCandidatesPerAnchor2[blockIdx.x];
                        const int offset = d_numCandidatesPerAnchorPrefixSum[blockIdx.x];
                        
                        for(int i = threadIdx.x; i < num; i += blockDim.x){
                            const int globalIndex = indices2[offset + i] + offset;
                            d_flagscandidates[globalIndex] = true;
                        }
                    }
                ); CUERR;

                //compact candidate sequences according to flags                

                cubstatus = hipcub::DeviceSelect::Flagged(
                    cubtemp,
                    cubBytes,
                    batchData.d_candidateSequencesData.data(),
                    thrust::make_transform_iterator(
                        thrust::make_counting_iterator(0),
                        SequenceFlagMultiplier{batchData.d_flagscandidates.data(), int(encodedSequencePitchInInts)}
                    ),
                    batchData.d_candidateSequencesData2.data(),
                    thrust::make_discard_iterator(),
                    batchData.totalNumCandidates * encodedSequencePitchInInts,
                    firstStream,
                    cubdebugsync
                );

                assert(cubstatus == hipSuccess);

                //compact other candidate buffers according to flags

                cubstatus = hipcub::DeviceSelect::Flagged(
                    cubtemp,
                    cubBytes,
                    in_zipped_begin,
                    batchData.d_flagscandidates.data(),
                    out_zipped_begin,
                    thrust::make_discard_iterator(),
                    batchData.totalNumCandidates,
                    firstStream,
                    cubdebugsync
                );

                assert(cubstatus == hipSuccess);

                std::swap(batchData.d_numCandidatesPerAnchor, batchData.d_numCandidatesPerAnchor2);
                std::swap(batchData.d_numCandidatesPerAnchorPrefixSum, batchData.d_numCandidatesPerAnchorPrefixSum2);                
                std::swap(batchData.d_candidateSequencesData, batchData.d_candidateSequencesData2);
                std::swap(batchData.d_candidateReadIds, batchData.d_candidateReadIds2);
                std::swap(batchData.d_candidateSequencesLength, batchData.d_candidateSequencesLength2);
                std::swap(batchData.d_alignment_overlaps, batchData.d_alignment_overlaps2);
                std::swap(batchData.d_alignment_isValid, batchData.d_alignment_isValid2);
                std::swap(batchData.d_alignment_shifts, batchData.d_alignment_shifts2);
                std::swap(batchData.d_alignment_nOps, batchData.d_alignment_nOps2);
                std::swap(batchData.d_alignment_best_alignment_flags, batchData.d_alignment_best_alignment_flags2);


                //compute possible msa splits

                batchData.h_possibleSplitColumns.resize(32 * batchData.numTasks);
                batchData.h_numPossibleSplitColumnsPerAnchor.resize(batchData.numTasks);

                batchData.d_possibleSplitColumns.resize(32 * batchData.numTasks);
                batchData.d_numPossibleSplitColumnsPerAnchor.resize(batchData.numTasks);

                helpers::lambda_kernel<<<batchData.numTasks, 128, 0, firstStream>>>(
                    [
                        d_numCandidatesPerAnchor = batchData.d_numCandidatesPerAnchor.data(),
                        d_counts = batchData.d_counts.data(),
                        d_coverage = batchData.d_coverage.data(),
                        msaColumnPitchInElements = this->msaColumnPitchInElements,
                        d_msa_column_properties = batchData.d_msa_column_properties.data(),
                        numTasks = batchData.numTasks,
                        splitcolumnsPitchElements = 32,
                        d_possibleSplitColumns = batchData.d_possibleSplitColumns.data(),
                        d_numPossibleSplitColumnsPerTask = batchData.d_numPossibleSplitColumnsPerAnchor.data(),
                        size1 = batchData.d_msa_column_properties.size(),
                        size2 = batchData.d_counts.size(),
                        size3 = batchData.d_coverage.size()
                    ] __device__ (){

                        using PSC = MultipleSequenceAlignment::PossibleSplitColumn;
                        constexpr int maxColumnsPerTask = 32;

                        __shared__ PSC sharedPSC[maxColumnsPerTask];
                        __shared__ int broadcastint;

                        using BlockReduce = hipcub::BlockReduce<int, 128>;
                        using BlockScan = hipcub::BlockScan<int, 128>;
                        __shared__ typename BlockReduce::TempStorage blockreducetemp;
                        __shared__ typename BlockScan::TempStorage blockscantemp;

                        for(int task = blockIdx.x; task < numTasks; task += gridDim.x){

                            assert(task < size1);

                            int* const numSplitColumnsPtr = d_numPossibleSplitColumnsPerTask + task;
                            PSC* const splitColumnsPtr = d_possibleSplitColumns + splitcolumnsPitchElements * task;

                            if(d_numCandidatesPerAnchor[task] > 0){

                                //only check columns to the right of anchor
                                const int firstColumn = d_msa_column_properties[task].subjectColumnsEnd_excl;
                                const int lastColumnExcl = d_msa_column_properties[task].lastColumn_excl;                                

                                int* myCountsPtr[4];
                                myCountsPtr[0] = d_counts + 4 * msaColumnPitchInElements * task + 0 * msaColumnPitchInElements;
                                myCountsPtr[1] = d_counts + 4 * msaColumnPitchInElements * task + 1 * msaColumnPitchInElements;
                                myCountsPtr[2] = d_counts + 4 * msaColumnPitchInElements * task + 2 * msaColumnPitchInElements;
                                myCountsPtr[3] = d_counts + 4 * msaColumnPitchInElements * task + 3 * msaColumnPitchInElements;   
                                
                                int* myCoveragesPtr = d_coverage + msaColumnPitchInElements * task;

                                int totalNumResults = 0;

                                const int numIterations = SDIV(lastColumnExcl - firstColumn, blockDim.x);

                                for(int iteration = 0; iteration < numIterations; iteration++){
                                    const int col = firstColumn + iteration * blockDim.x + threadIdx.x;

                                    PSC myresults[3];
                                    int myNumResults = 0;

                                    if(col < lastColumnExcl){                                   
        
                                        auto checkNuc = [&](const auto& counts, const char nuc){
                                            if(myNumResults < 3){


                                                const float ratio = float(counts[col]) / float(myCoveragesPtr[col]);
                                                //if((counts[col] == 2 && fgeq(ratio, 0.4f) && fleq(ratio, 0.6f)) || counts[col] > 2){
                                                if(counts[col] >= 2 && fgeq(ratio, 0.4f) && fleq(ratio, 0.6f)){
        
                                                    #pragma unroll
                                                    for(int k = 0; k < 3; k++){
                                                        if(myNumResults == k){
                                                            myresults[k] = {nuc, col, ratio};
                                                            myNumResults++;
                                                            break;
                                                        }
                                                    }
                                                    
                                                }
                                            }
                                        };
        
                                        checkNuc(myCountsPtr[0], 'A');
                                        checkNuc(myCountsPtr[1], 'C');
                                        checkNuc(myCountsPtr[2], 'G');
                                        checkNuc(myCountsPtr[3], 'T');
        
                                        if(myNumResults != 2){
                                            myNumResults = 0;
                                        }    
                                    }

                                    int totalNumResultsIteration = BlockReduce(blockreducetemp).Sum(myNumResults);
                                    if(threadIdx.x == 0){
                                        broadcastint = totalNumResultsIteration;
                                    }
                                    __syncthreads();
                                    totalNumResultsIteration = broadcastint;

                                    if(totalNumResultsIteration + totalNumResults > maxColumnsPerTask){
                                        totalNumResults = 0;
                                        break;
                                    }else{
                                        int outputoffset = 0;

                                        BlockScan(blockscantemp).ExclusiveSum(myNumResults, outputoffset);

                                        if(myNumResults == 2){
                                            sharedPSC[totalNumResults + outputoffset + 0] = myresults[0];
                                            sharedPSC[totalNumResults + outputoffset + 1] = myresults[1];
                                        }

                                        totalNumResults += totalNumResultsIteration;
                                    }

                                    __syncthreads();
                                }

                                for(int i = threadIdx.x; i < totalNumResults; i += blockDim.x){
                                    splitColumnsPtr[i] = sharedPSC[i];
                                }

                                if(threadIdx.x == 0){
                                    *numSplitColumnsPtr = totalNumResults;
                                }
                            }else{
                                if(threadIdx.x == 0){
                                    *numSplitColumnsPtr = 0;
                                }
                            }
                        }
                    }
                ); CUERR;

                hipMemcpyAsync(
                    batchData.h_possibleSplitColumns.get(),
                    batchData.d_possibleSplitColumns.get(),
                    sizeof(MultipleSequenceAlignment::PossibleSplitColumn) * 32 * batchData.numTasks,
                    D2H,
                    firstStream
                ); CUERR;

                hipMemcpyAsync(
                    batchData.h_numPossibleSplitColumnsPerAnchor.get(),
                    batchData.d_numPossibleSplitColumnsPerAnchor.get(),
                    sizeof(int) * batchData.numTasks,
                    D2H,
                    firstStream
                ); CUERR;

                hipEventSynchronize(events[0]); CUERR; //wait for h_numCandidates

                #if 0

                batchData.totalNumCandidates = *batchData.h_numCandidates;

                thrust::copy_n(
                    thrustPolicy1,
                    thrust::make_zip_iterator(thrust::make_tuple(
                        batchData.d_numCandidatesPerAnchorPrefixSum.data() + 1,
                        batchData.d_numCandidatesPerAnchor.data()
                    )),
                    batchData.numTasks,
                    thrust::make_zip_iterator(thrust::make_tuple(
                        batchData.h_numCandidatesPerAnchorPrefixSum.data() + 1,
                        batchData.h_numCandidatesPerAnchor.data()
                    ))
                );

                //copy data to host
                hipMemcpyAsync(
                    batchData.h_candidateSequencesData.get(),
                    batchData.d_candidateSequencesData.get(),
                    sizeof(unsigned int) * batchData.totalNumCandidates * encodedSequencePitchInInts,
                    D2H,
                    firstStream
                ); CUERR;
    
                auto d_zipped_begin = thrust::make_zip_iterator(
                    thrust::make_tuple(
                        batchData.d_candidateReadIds.data(),
                        batchData.d_candidateSequencesLength.data(),
                        batchData.d_alignment_overlaps.data(),
                        batchData.d_alignment_isValid.data(),
                        batchData.d_alignment_shifts.data(),
                        batchData.d_alignment_nOps.data(),
                        batchData.d_alignment_best_alignment_flags.data()
                    )
                );
    
                auto h_zipped_begin = thrust::make_zip_iterator(
                    thrust::make_tuple(
                        batchData.h_candidateReadIds.data(),
                        batchData.h_candidateSequencesLength.data(),
                        batchData.h_alignment_overlaps.data(),
                        batchData.h_alignment_isValid.data(),
                        batchData.h_alignment_shifts.data(),
                        batchData.h_alignment_nOps.data(),
                        batchData.h_alignment_best_alignment_flags.data()
                    )
                );
    
                thrust::copy_n(
                    thrustPolicy1,
                    d_zipped_begin,
                    batchData.totalNumCandidates,
                    h_zipped_begin
                );
                
    
                hipStreamSynchronize(firstStream); CUERR;

                for(int i = 0; i < numActiveTasks; i++){
                    auto& task = vecAccess(tasks, indicesOfActiveTasks[i]);
    
                    #ifdef checkdebugtasks
                    copyBatchDataIntoTask(task, i);
                    task.dataIsAvailable = true;
                    #else
                    task.mateRemovedFromCandidates = false; //debug. not required
                    task.numRemainingCandidates = batchData.h_numCandidatesPerAnchor[i];
    
                    if(task.numRemainingCandidates == 0){
                        task.abort = true;
                        task.abortReason = AbortReason::NoPairedCandidatesAfterAlignment;
                    }
                    #endif
                }

                #endif

                hipStreamSynchronize(firstStream); CUERR;
                hipStreamSynchronize(secondStream); CUERR;

                cubAllocator->DeviceFree(cubtemp);

            }
    

            std::vector<Task> newTasksFromSplit;
            std::vector<int> newTaskIndices;


            auto constructMsaWithDataFromTask = [&](auto& task, int taskindex){
                const std::string& decodedAnchor = task.totalDecodedAnchors.back();

                MultipleSequenceAlignment msa;

                auto build = [&](){

                    task.candidateShifts.resize(task.numRemainingCandidates);
                    task.candidateOverlapWeights.resize(task.numRemainingCandidates);

                    //gather data required for msa
                    for(int c = 0; c < task.numRemainingCandidates; c++){
                        vecAccess(task.candidateShifts, c) = vecAccess(task.alignments, c).shift;

                        vecAccess(task.candidateOverlapWeights, c) = calculateOverlapWeight(
                            task.currentAnchorLength, 
                            vecAccess(task.alignments, c).nOps,
                            vecAccess(task.alignments, c).overlap,
                            goodAlignmentProperties.maxErrorRate
                        );
                    }

                    task.candidateStrings.resize(decodedSequencePitchInBytes * task.numRemainingCandidates, '\0');

                    //decode the candidates for msa
                    for(int c = 0; c < task.numRemainingCandidates; c++){
                        SequenceHelpers::decode2BitSequence(
                            task.candidateStrings.data() + c * decodedSequencePitchInBytes,
                            task.candidateSequenceData.data() + c * encodedSequencePitchInInts,
                            vecAccess(task.candidateSequenceLengths, c)
                        );
                    }

                    MultipleSequenceAlignment::InputData msaInput;
                    msaInput.useQualityScores = false;
                    msaInput.subjectLength = task.currentAnchorLength;
                    msaInput.nCandidates = task.numRemainingCandidates;
                    msaInput.candidatesPitch = decodedSequencePitchInBytes;
                    msaInput.candidateQualitiesPitch = 0;
                    msaInput.subject = decodedAnchor.c_str();
                    msaInput.candidates = task.candidateStrings.data();
                    msaInput.subjectQualities = nullptr;
                    msaInput.candidateQualities = nullptr;
                    msaInput.candidateLengths = task.candidateSequenceLengths.data();
                    msaInput.candidateShifts = task.candidateShifts.data();
                    msaInput.candidateDefaultWeightFactors = task.candidateOverlapWeights.data();                    

                    msa.build(msaInput);
                };

                build();

                #if 1

                constexpr int max_num_minimizations = 5;

                auto removeCandidatesOfDifferentRegion = [&](const auto& minimizationResult){
                    const int numCandidates = task.candidateReadIds.size();

                    int insertpos = 0;
                    for(int i = 0; i < numCandidates; i++){
                        if(!minimizationResult.differentRegionCandidate[i]){               
                            //keep candidate

                            task.candidateReadIds[insertpos] = task.candidateReadIds[i];

                            std::copy_n(
                                task.candidateSequenceData.data() + i * size_t(encodedSequencePitchInInts),
                                encodedSequencePitchInInts,
                                task.candidateSequenceData.data() + insertpos * size_t(encodedSequencePitchInInts)
                            );

                            task.candidateSequenceLengths[insertpos] = task.candidateSequenceLengths[i];
                            task.alignmentFlags[insertpos] = task.alignmentFlags[i];
                            task.alignments[insertpos] = task.alignments[i];
                            task.candidateOverlapWeights[insertpos] = task.candidateOverlapWeights[i];
                            task.candidateShifts[insertpos] = task.candidateShifts[i];

                            std::copy_n(
                                task.candidateStrings.data() + i * size_t(decodedSequencePitchInBytes),
                                decodedSequencePitchInBytes,
                                task.candidateStrings.data() + insertpos * size_t(decodedSequencePitchInBytes)
                            );

                            insertpos++;
                        }
                    }

                    task.numRemainingCandidates = insertpos;

                    task.candidateReadIds.erase(
                        task.candidateReadIds.begin() + insertpos, 
                        task.candidateReadIds.end()
                    );
                    task.candidateSequenceData.erase(
                        task.candidateSequenceData.begin() + encodedSequencePitchInInts * insertpos, 
                        task.candidateSequenceData.end()
                    );
                    task.candidateSequenceLengths.erase(
                        task.candidateSequenceLengths.begin() + insertpos, 
                        task.candidateSequenceLengths.end()
                    );
                    task.alignmentFlags.erase(
                        task.alignmentFlags.begin() + insertpos, 
                        task.alignmentFlags.end()
                    );
                    task.alignments.erase(
                        task.alignments.begin() + insertpos, 
                        task.alignments.end()
                    );

                    task.candidateStrings.erase(
                        task.candidateStrings.begin() + decodedSequencePitchInBytes * insertpos, 
                        task.candidateStrings.end()
                    );
                    task.candidateOverlapWeights.erase(
                        task.candidateOverlapWeights.begin() + insertpos, 
                        task.candidateOverlapWeights.end()
                    );
                    task.candidateShifts.erase(
                        task.candidateShifts.begin() + insertpos, 
                        task.candidateShifts.end()
                    );
                    
                };

                if(max_num_minimizations > 0){                

                    for(int numIterations = 0; numIterations < max_num_minimizations; numIterations++){
                        const auto minimizationResult = msa.findCandidatesOfDifferentRegion(
                            correctionOptions.estimatedCoverage
                        );

                        if(minimizationResult.performedMinimization){
                            removeCandidatesOfDifferentRegion(minimizationResult);

                            //build minimized multiple sequence alignment
                            build();
                        }else{
                            break;
                        }               
                        
                    }
                }   

                #endif

                return msa;
            };

            auto constructMsaWithDataFromBatchData = [&](auto& task, int taskindex){
                const std::string& decodedAnchor = task.totalDecodedAnchors.back();

                const int offset = batchData.h_numCandidatesPerAnchorPrefixSum[taskindex];

                MultipleSequenceAlignment msa;

                int* const myShifts = &batchData.h_alignment_shifts[offset];
                int* const myNops = &batchData.h_alignment_nOps[offset];
                int* const myOverlaps = &batchData.h_alignment_overlaps[offset];
                auto* const myAlignmentFlags = &batchData.h_alignment_best_alignment_flags[offset];
                unsigned int* const myCandidateSequencesData = &batchData.h_candidateSequencesData[offset * encodedSequencePitchInInts];
                int* const myCandidateSequencesLengths = &batchData.h_candidateSequencesLength[offset];
                read_number* const myCandidateReadIds = &batchData.h_candidateReadIds[offset];


                auto build = [&](){

                    task.candidateOverlapWeights.resize(task.numRemainingCandidates);

                    //gather data required for msa
                    for(int c = 0; c < task.numRemainingCandidates; c++){

                        vecAccess(task.candidateOverlapWeights, c) = calculateOverlapWeight(
                            task.currentAnchorLength, 
                            myNops[c],
                            myOverlaps[c],
                            goodAlignmentProperties.maxErrorRate
                        );
                    }

                    task.candidateStrings.resize(decodedSequencePitchInBytes * task.numRemainingCandidates, '\0');

                    //decode the candidates for msa
                    for(int c = 0; c < task.numRemainingCandidates; c++){
                        SequenceHelpers::decode2BitSequence(
                            task.candidateStrings.data() + c * decodedSequencePitchInBytes,
                            myCandidateSequencesData + c * encodedSequencePitchInInts,
                            myCandidateSequencesLengths[c]
                        );
                    }

                    MultipleSequenceAlignment::InputData msaInput;
                    msaInput.useQualityScores = false;
                    msaInput.subjectLength = task.currentAnchorLength;
                    msaInput.nCandidates = task.numRemainingCandidates;
                    msaInput.candidatesPitch = decodedSequencePitchInBytes;
                    msaInput.candidateQualitiesPitch = 0;
                    msaInput.subject = decodedAnchor.c_str();
                    msaInput.candidates = task.candidateStrings.data();
                    msaInput.subjectQualities = nullptr;
                    msaInput.candidateQualities = nullptr;
                    msaInput.candidateLengths = myCandidateSequencesLengths;
                    msaInput.candidateShifts = myShifts;
                    msaInput.candidateDefaultWeightFactors = task.candidateOverlapWeights.data();                    

                    msa.build(msaInput);
                };

                build();

                #if 1

                constexpr int max_num_minimizations = 5;

                auto removeCandidatesOfDifferentRegion = [&](const auto& minimizationResult){
                    const int numCandidates = batchData.h_numCandidatesPerAnchor[taskindex];

                    int insertpos = 0;
                    for(int i = 0; i < numCandidates; i++){
                        if(!minimizationResult.differentRegionCandidate[i]){               
                            //keep candidate

                            myCandidateReadIds[insertpos] = myCandidateReadIds[i];

                            std::copy_n(
                                myCandidateSequencesData + i * size_t(encodedSequencePitchInInts),
                                encodedSequencePitchInInts,
                                myCandidateSequencesData + insertpos * size_t(encodedSequencePitchInInts)
                            );

                            myCandidateSequencesLengths[insertpos] = myCandidateSequencesLengths[i];
                            myAlignmentFlags[insertpos] = myAlignmentFlags[i];
                            myShifts[insertpos] = myShifts[i];
                            myNops[insertpos] = myNops[i];
                            myOverlaps[insertpos] = myOverlaps[i];
                            task.candidateOverlapWeights[insertpos] = task.candidateOverlapWeights[i];
                            myShifts[insertpos] = myShifts[i];

                            std::copy_n(
                                task.candidateStrings.data() + i * size_t(decodedSequencePitchInBytes),
                                decodedSequencePitchInBytes,
                                task.candidateStrings.data() + insertpos * size_t(decodedSequencePitchInBytes)
                            );

                            insertpos++;
                        }
                    }

                    batchData.h_numCandidatesPerAnchor[taskindex] = insertpos;
                    task.numRemainingCandidates = insertpos;

                    task.candidateStrings.erase(
                        task.candidateStrings.begin() + decodedSequencePitchInBytes * insertpos, 
                        task.candidateStrings.end()
                    );
                    task.candidateOverlapWeights.erase(
                        task.candidateOverlapWeights.begin() + insertpos, 
                        task.candidateOverlapWeights.end()
                    );                    
                };

                if(max_num_minimizations > 0){                

                    for(int numIterations = 0; numIterations < max_num_minimizations; numIterations++){
                        const auto minimizationResult = msa.findCandidatesOfDifferentRegion(
                            correctionOptions.estimatedCoverage
                        );

                        if(minimizationResult.performedMinimization){
                            removeCandidatesOfDifferentRegion(minimizationResult);

                            //build minimized multiple sequence alignment
                            build();
                        }else{
                            break;
                        }               
                        
                    }
                }   

                #endif

                return msa;
            };

            auto constructMsa = [&](auto& task, int taskindex){
                if(task.dataIsAvailable){
                    return constructMsaWithDataFromTask(task, taskindex);
                }else{
                    return constructMsaWithDataFromBatchData(task, taskindex);
                }
            };

            auto extendWithMsa = [&](auto& task, const auto& msa, int taskIndex){

                int consensusLength = msa.consensus.size();
                //can extend by at most maxextensionPerStep bps
                int extendBy = std::min(
                    consensusLength - task.currentAnchorLength, 
                    maxextensionPerStep
                );
                //cannot extend over fragment 
                extendBy = std::min(extendBy, (insertSize + insertSizeStddev - task.mateLength) - task.accumExtensionLengths);

                auto makeAnchorForNextIteration = [&](){
                    if(extendBy == 0){
                        task.abort = true;
                        task.abortReason = AbortReason::MsaNotExtended;
                    }else{
                        task.accumExtensionLengths += extendBy;

                        //update data for next iteration of outer while loop                           

                        std::string decodedAnchor(msa.consensus.data() + extendBy, task.currentAnchorLength);

                        const int numInts = SequenceHelpers::getEncodedNumInts2Bit(task.currentAnchorLength);

                        task.currentAnchor.resize(numInts);

                        SequenceHelpers::encodeSequence2Bit(
                            task.currentAnchor.data(), 
                            decodedAnchor.data(), 
                            task.currentAnchorLength
                        );

                        task.totalDecodedAnchors.emplace_back(std::move(decodedAnchor));
                        task.totalAnchorBeginInExtendedRead.emplace_back(task.accumExtensionLengths);

                        // task.resultsequence.insert(
                        //     task.resultsequence.end(), 
                        //     msa.consensus.data() + task.currentAnchorLength, 
                        //     msa.consensus.data() + task.currentAnchorLength + extendBy
                        // );


                        // std::string tmp(task.currentAnchorLength, '\0');

                        // decode2BitSequence(
                        //     &tmp[0],
                        //     task.currentAnchor.data(),
                        //     task.currentAnchorLength
                        // );

                        // auto sub = task.resultsequence.substr(task.resultsequence.length() - task.currentAnchorLength);

                        // assert(sub == tmp);
                    }
                };

                constexpr int requiredOverlapMate = 70; //TODO relative overlap 
                constexpr int numMismatchesUpperBound = 2;

                if(task.pairedEnd && task.accumExtensionLengths + consensusLength - requiredOverlapMate + task.mateLength >= insertSize - insertSizeStddev){
                    //check if mate can be overlapped with consensus 

                    //hamMap[i] stores possible starting positions of overlaps which would have hamming distance i
                    std::map<int, std::vector<int>> hamMap;

                    //longmatchMap[i] stores possible starting positions of overlaps which would have a longest match of length i between mate and msa consensus
                    //std::map<int, std::vector<int>> longmatchMap; //map length of longest match to list start positions

                    //for each possibility to overlap the mate and consensus such that the merged sequence would end in the desired range [insertSize - insertSizeStddev, insertSize + insertSizeStddev]

                    const int firstStartpos = std::max(0, insertSize - insertSizeStddev - task.accumExtensionLengths - task.mateLength);
                    const int lastStartposExcl = std::min(
                        std::max(0, insertSize + insertSizeStddev - task.accumExtensionLengths - task.mateLength) + 1,
                        consensusLength - requiredOverlapMate
                    );

                    for(int startpos = firstStartpos; startpos < lastStartposExcl; startpos++){
                        //compute metrics of overlap
                            
                        const int ham = cpu::hammingDistanceOverlap(
                            msa.consensus.begin() + startpos, msa.consensus.end(), 
                            task.decodedMateRevC.begin(), task.decodedMateRevC.end()
                        );

                        hamMap[ham].emplace_back(startpos);

                        // const int longest = cpu::longestMatch(
                        //     msa.consensus.begin() + startpos, msa.consensus.end(), 
                        //     task.decodedMateRevC.begin(), task.decodedMateRevC.end()
                        // );

                        // longmatchMap[longest].emplace_back(startpos);
                    }
                    
                    std::vector<std::pair<int, std::vector<int>>> flatMap(hamMap.begin(), hamMap.end());
                    //sort by hamming distance, ascending
                    std::sort(flatMap.begin(), flatMap.end(), [](const auto& p1, const auto& p2){return p1.first < p2.first;});

                    //std::vector<std::pair<int, std::vector<int>>> flatMap2(longmatchMap.begin(), longmatchMap.end());
                    //sort by length of longest match, descending
                    //std::sort(flatMap2.begin(), flatMap2.end(), [](const auto& p1, const auto& p2){return p2.first < p1.first;});

                    //if there exists an overlap between msa consensus and mate which would end the merge, use the best one
                    if(flatMap.size() > 0 && flatMap[0].first <= numMismatchesUpperBound){
                    //if(flatMap2.size() > 0 && flatMap2[0].first >= 40){
                        const int mateStartposInConsensus = flatMap[0].second.front();
                        const int missingPositionsBetweenAnchorEndAndMateBegin = std::max(0, mateStartposInConsensus - task.currentAnchorLength);

                        if(missingPositionsBetweenAnchorEndAndMateBegin > 0){
                            //bridge the gap between current anchor and mate
                            task.totalDecodedAnchors.emplace_back(
                                msa.consensus.data() + missingPositionsBetweenAnchorEndAndMateBegin,
                                msa.consensus.data() + missingPositionsBetweenAnchorEndAndMateBegin + mateStartposInConsensus
                            );
                            task.totalAnchorBeginInExtendedRead.emplace_back(task.accumExtensionLengths + missingPositionsBetweenAnchorEndAndMateBegin);
                        }


                        task.mateHasBeenFound = true;

                        //const int currentAccumExtensionLengths = task.accumExtensionLengths;
                        
                        task.accumExtensionLengths += mateStartposInConsensus;
                        std::string decodedAnchor(task.decodedMateRevC);

                        task.totalDecodedAnchors.emplace_back(std::move(decodedAnchor));
                        task.totalAnchorBeginInExtendedRead.emplace_back(task.accumExtensionLengths);

                        // const int startpos = mateStartposInConsensus;
                        // task.resultsequence.resize(currentAccumExtensionLengths + startpos + task.decodedMateRevC.length());
                        // const auto replaceBegin = task.resultsequence.begin() + currentAccumExtensionLengths + startpos;
                        // task.resultsequence.replace(
                        //     replaceBegin, 
                        //     replaceBegin + task.decodedMateRevC.length(), 
                        //     task.decodedMateRevC.begin(), 
                        //     task.decodedMateRevC.end()
                        // );

                    }else{
                        makeAnchorForNextIteration();
                    }
                }else{
                    makeAnchorForNextIteration();
                }
            };

            auto keepSelectedCandidates = [&](auto& task, const auto& selectedCandidateIndices, int taskIndex){
                assert(task.dataIsAvailable);

                const int numCandidateIndices = selectedCandidateIndices.size();
                assert(numCandidateIndices <= task.numRemainingCandidates);

                for(int i = 0; i < numCandidateIndices; i++){
                    const int c = vecAccess(selectedCandidateIndices, i);
                    // if(!(0 <= c && c < task.candidateReadIds.size())){
                    //     std::cerr << "c = " << c << ", candidateReadIds.size() = " << task.candidateReadIds.size() << "\n";
                    // }

                    // assert(0 <= c && c < task.candidateReadIds.size());
                    // assert(0 <= c && c < task.candidateSequenceLengths.size());
                    // assert(0 <= c && c < task.alignments.size());
                    // assert(0 <= c && c < task.alignmentFlags.size());

                    // assert(0 <= c && c*encodedSequencePitchInInts < task.candidateSequencesFwdData.size());
                    // assert(0 <= c && c*encodedSequencePitchInInts < task.candidateSequencesRevcData.size());
                    // assert(0 <= c && c*encodedSequencePitchInInts < task.candidateSequenceData.size());

                    vecAccess(task.candidateReadIds, i) = vecAccess(task.candidateReadIds, c);
                    vecAccess(task.candidateSequenceLengths , i) = vecAccess(task.candidateSequenceLengths, c);
                    vecAccess(task.alignments, i) = vecAccess(task.alignments, c);
                    vecAccess(task.alignmentFlags, i) = vecAccess(task.alignmentFlags, c);
                    vecAccess(task.candidateShifts, i) = vecAccess(task.candidateShifts, c);
                    vecAccess(task.candidateOverlapWeights, i) = vecAccess(task.candidateOverlapWeights, c);

                    // std::copy_n(
                    //     task.candidateSequencesFwdData.begin() + c * encodedSequencePitchInInts,
                    //     encodedSequencePitchInInts,
                    //     task.candidateSequencesFwdData.begin() + i * encodedSequencePitchInInts
                    // );

                    // std::copy_n(
                    //     task.candidateSequencesRevcData.begin() + c * encodedSequencePitchInInts,
                    //     encodedSequencePitchInInts,
                    //     task.candidateSequencesRevcData.begin() + i * encodedSequencePitchInInts
                    // );

                    std::copy_n(
                        task.candidateSequenceData.begin() + c * encodedSequencePitchInInts,
                        encodedSequencePitchInInts,
                        task.candidateSequenceData.begin() + i * encodedSequencePitchInInts
                    );

                    std::copy_n(
                        task.candidateStrings.begin() + c * decodedSequencePitchInBytes,
                        decodedSequencePitchInBytes,
                        task.candidateStrings.begin() + i * decodedSequencePitchInBytes
                    );
                }

                task.candidateReadIds.erase(
                    task.candidateReadIds.begin() + numCandidateIndices,
                    task.candidateReadIds.end()
                );
                task.candidateSequenceLengths.erase(
                    task.candidateSequenceLengths.begin() + numCandidateIndices,
                    task.candidateSequenceLengths.end()
                );
                task.alignments.erase(
                    task.alignments.begin() + numCandidateIndices,
                    task.alignments.end()
                );
                task.alignmentFlags.erase(
                    task.alignmentFlags.begin() + numCandidateIndices,
                    task.alignmentFlags.end()
                );
                // task.candidateSequencesFwdData.erase(
                //     task.candidateSequencesFwdData.begin() + numCandidateIndices * encodedSequencePitchInInts,
                //     task.candidateSequencesFwdData.end()
                // );
                // task.candidateSequencesRevcData.erase(
                //     task.candidateSequencesRevcData.begin() + numCandidateIndices * encodedSequencePitchInInts,
                //     task.candidateSequencesRevcData.end()
                // );
                task.candidateSequenceData.erase(
                    task.candidateSequenceData.begin() + numCandidateIndices * encodedSequencePitchInInts,
                    task.candidateSequenceData.end()
                );
                if(task.pairedEnd){
                    task.mateIdLocationIter = std::lower_bound(
                        task.candidateReadIds.begin(),
                        task.candidateReadIds.end(),
                        task.mateReadId
                    );

                    task.mateHasBeenFound = (task.mateIdLocationIter != task.candidateReadIds.end() 
                        && *task.mateIdLocationIter == task.mateReadId);
                }
                task.numRemainingCandidates = numCandidateIndices;
            };

            nvtx::push_range("MSA", 6);
            msaTimer.start();

            for(int i = 0; i < numActiveTasks; i++){
                const int indexOfActiveTask = indicesOfActiveTasks[i];
                auto& task = vecAccess(tasks, indexOfActiveTask);

                if(task.numRemainingCandidates == 0){
                    continue;
                }
                assert(task.numRemainingCandidates > 0);

                //nvtx::push_range("constructMsa", 7);
                const MultipleSequenceAlignment msa = constructMsa(task, i);
                //nvtx::pop_range();

                // std::cerr << "original msa\n";
                // msa.print(std::cerr);
                // std::cerr << "\n";

                
#if 1
                //if(task.splitDepth == 0){
                if(splitTracker[task.myReadId] <= 4){
                    std::vector<MultipleSequenceAlignment::PossibleSplitColumn> possibleSplitColumns = computePossibleSplitColumns(
                        task.currentAnchorLength, 
                        msa.nColumns,
                        msa.countsA.data(),
                        msa.countsC.data(),
                        msa.countsG.data(),
                        msa.countsT.data(),
                        msa.coverage.data()
                    );

                    if(batchData.h_numPossibleSplitColumnsPerAnchor[i] == 0){
                        if(!(possibleSplitColumns.size() == 0 || possibleSplitColumns.size() > 32)){
                            std::cerr << "possibleSplitColumns.size() = " << possibleSplitColumns.size() << "\n";
                        }
                        assert(possibleSplitColumns.size() == 0 || possibleSplitColumns.size() > 32);
                    }else{
                        if(!(possibleSplitColumns.size() == batchData.h_numPossibleSplitColumnsPerAnchor[i])){
                            std::cerr << "possibleSplitColumns.size() = " << possibleSplitColumns.size() 
                                << "i = " << i << ", batchData.h_numPossibleSplitColumnsPerAnchor[i] = " << batchData.h_numPossibleSplitColumnsPerAnchor[i] << "\n";
                        }
                        assert(possibleSplitColumns.size() == batchData.h_numPossibleSplitColumnsPerAnchor[i]);

                        auto* ptr = batchData.h_possibleSplitColumns.data() + 32 * i;
                        for(int k = 0; k < batchData.h_numPossibleSplitColumnsPerAnchor[i]; k++){
                            assert(possibleSplitColumns[k] == ptr[k]);
                        }
                    }

                    assert(batchData.h_msa_column_properties[i].lastColumn_excl - batchData.h_msa_column_properties[i].firstColumn_incl == msa.consensus.size());

                    // for(int k = 0; k < int(msa.consensus.size()); k++){
                    //     assert(batchData.h_consensus[i * msaColumnPitchInElements + k] == msa.consensus[k]);
                    // }

                    auto possibleSplitsNew = inspectColumnsRegionSplit(
                        possibleSplitColumns.data(),
                        possibleSplitColumns.size(),
                        task.currentAnchorLength, 
                        msa.nColumns,
                        msa.subjectColumnsBegin_incl,
                        msa.nCandidates,
                        task.candidateStrings.data(),
                        decodedSequencePitchInBytes,
                        msa.inputData.candidateShifts,
                        msa.inputData.candidateLengths
                    );

                    //auto possibleSplitsOld = msa.inspectColumnsRegionSplit(task.currentAnchorLength);
                    //assert(possibleSplitsOld == possibleSplitsNew);

                    auto& possibleSplits = possibleSplitsNew;

                    if(possibleSplits.splits.size() > 1){
                        //nvtx::push_range("split msa", 8);
                        //auto& task = tasks[indexOfActiveTask];
                        #if 1
                        std::sort(
                            possibleSplits.splits.begin(), 
                            possibleSplits.splits.end(),
                            [](const auto& split1, const auto& split2){
                                //sort by size, descending
                                return split2.listOfCandidates.size() < split1.listOfCandidates.size();
                            }
                        );
                        #else
                        std::nth_element(
                            possibleSplits.splits.begin(), 
                            possibleSplits.splits.begin() + 1, 
                            possibleSplits.splits.end(),
                            [](const auto& split1, const auto& split2){
                                //sort by size, descending
                                return split2.listOfCandidates.size() < split1.listOfCandidates.size();
                            }
                        );
                        #endif

                        // std::cerr << "split[0] = ";
                        // for(auto x : possibleSplits.splits[0].listOfCandidates) std::cerr << x << " ";
                        // std::cerr << "\nsplit[1] = ";
                        // for(auto x : possibleSplits.splits[1].listOfCandidates) std::cerr << x << " ";
                        // std::cerr << "\n";

                        // auto printColumnInfo = [](const auto& x){
                        //     std::cerr << "(" << x.column << ", " << x.letter << ", " << x.ratio << ") ";
                        // };

                        // std::cerr << "columns[0] = ";
                        // for(auto x : possibleSplits.splits[0].columnInfo) printColumnInfo(x);
                        // std::cerr << "\ncolumns[1] = ";
                        // for(auto x : possibleSplits.splits[1].columnInfo) printColumnInfo(x);
                        // std::cerr << "\n";


                        //copy task's data from batchData into task

                        copyBatchDataIntoTask(task, i);
                        //create the separate shifts array
                        task.candidateShifts.resize(task.alignments.size());
                        for(int c = 0; c < task.numRemainingCandidates; c++){
                            task.candidateShifts[c] = task.alignments[c].shift;
                        }
                        task.dataIsAvailable = true;
                        //create a copy of task, and only keep candidates of first split
                        
                        Task taskCopy = task;
                        taskCopy.splitDepth++;

                        // std::cerr << "split\n";
                        // msa.print(std::cerr); 
                        // std::cerr << "\n into \n";

                        keepSelectedCandidates(taskCopy, possibleSplits.splits[0].listOfCandidates, -1);
                        const MultipleSequenceAlignment msaOfCopy = constructMsa(taskCopy, -1);

                        // msaOfCopy.print(std::cerr); 
                        // std::cerr << "\n and \n";

                        extendWithMsa(taskCopy, msaOfCopy, -1);

                        //only keep canddiates of second split
                        keepSelectedCandidates(task, possibleSplits.splits[1].listOfCandidates, -1);
                        const MultipleSequenceAlignment newMsa = constructMsa(task, -1);

                        // newMsa.print(std::cerr); 
                        // std::cerr << "\n";

                        extendWithMsa(task, newMsa, -1);

                        //if extension was not possible in task, replace task by task copy
                        if(task.abort && task.abortReason == AbortReason::MsaNotExtended){
                            //replace task by taskCopy
                            task = std::move(taskCopy);
                        }else if(!taskCopy.abort){
                            //if extension was possible in both task and taskCopy, taskCopy will be added to tasks and list of active tasks
                            newTaskIndices.emplace_back(tasks.size() + newTasksFromSplit.size());
                            newTasksFromSplit.emplace_back(std::move(taskCopy));

                            splitTracker[task.myReadId]++;


                        }
                        //nvtx::pop_range();                     
                    }else{
                        extendWithMsa(task, msa, indexOfActiveTask);
                    }
                }else{
                    extendWithMsa(task, msa, indexOfActiveTask);
                }
#else 
                extendWithMsa(task, msa, indexOfActiveTask);
#endif

            }

            msaTimer.stop();

            nvtx::pop_range();

            if(newTasksFromSplit.size() > 0){
                //std::cerr << "Added " << newTasksFromSplit.size() << " tasks\n";
                tasks.insert(tasks.end(), std::make_move_iterator(newTasksFromSplit.begin()), std::make_move_iterator(newTasksFromSplit.end()));
                indicesOfActiveTasks.insert(indicesOfActiveTasks.end(), newTaskIndices.begin(), newTaskIndices.end());

                indicesOfActiveTasksTmp.resize(indicesOfActiveTasks.size());
            }           

            /*
                update book-keeping of used candidates
            */  

            for(int i = 0; i < numActiveTasks; i++){
                auto& task = vecAccess(tasks, indicesOfActiveTasks[i]);

                                      
                {
                    if(task.dataIsAvailable){
                        std::vector<read_number> tmp(task.allUsedCandidateReadIdPairs.size() + task.candidateReadIds.size());
                        auto tmp_end = std::merge(
                            task.allUsedCandidateReadIdPairs.begin(),
                            task.allUsedCandidateReadIdPairs.end(),
                            task.candidateReadIds.begin(),
                            task.candidateReadIds.end(),
                            tmp.begin()
                        );

                        tmp.erase(tmp_end, tmp.end());

                        std::swap(task.allUsedCandidateReadIdPairs, tmp);
                    }else{
                        const int numCandidates = batchData.h_numCandidatesPerAnchor[i];
                        const int offset = batchData.h_numCandidatesPerAnchorPrefixSum[i];
                        const read_number* ids = &batchData.h_candidateReadIds[offset];

                        std::vector<read_number> tmp(task.allUsedCandidateReadIdPairs.size() + numCandidates);
                        auto tmp_end = std::merge(
                            task.allUsedCandidateReadIdPairs.begin(),
                            task.allUsedCandidateReadIdPairs.end(),
                            ids,
                            ids + numCandidates,
                            tmp.begin()
                        );

                        tmp.erase(tmp_end, tmp.end());

                        std::swap(task.allUsedCandidateReadIdPairs, tmp);
                    }
                }

                // task.usedCandidateReadIdsPerIteration.emplace_back(std::move(task.candidateReadIds));
                // task.usedAlignmentsPerIteration.emplace_back(std::move(task.alignments));
                // task.usedAlignmentFlagsPerIteration.emplace_back(std::move(task.alignmentFlags));

                task.iteration++;
            }
            
            //update list of active task indices

            indicesOfActiveTasks.erase(
                std::remove_if(
                    indicesOfActiveTasks.begin(), 
                    indicesOfActiveTasks.end(),
                    [&](int index){
                        return !tasks[index].isActive(insertSize, insertSizeStddev);
                    }
                ),
                indicesOfActiveTasks.end()
            );
        }

        //construct results

        for(const auto& task : tasks){

            ExtendResult extendResult;
            extendResult.direction = task.direction;
            extendResult.numIterations = task.iteration;
            extendResult.aborted = task.abort;
            extendResult.abortReason = task.abortReason;
            extendResult.readId1 = task.myReadId;
            extendResult.readId2 = task.mateReadId;
            extendResult.originalLength = task.myLength;

#if 0
            //extendResult.extendedRead = std::move(task.resultsequence);
            extendResult.success = true;
            extendResult.mateHasBeenFound = task.mateHasBeenFound;

#else
            // if(abort){
            //     ; //no read extension possible
            // }else
            {
                //if(mateHasBeenFound){
                {
                    //construct extended read
                    //build msa of all saved totalDecodedAnchors[0]

                    const int numsteps = task.totalDecodedAnchors.size();

                    // if(task.myReadId == 90 || task.mateReadId == 90){
                    //     std::cerr << "task.totalDecodedAnchors\n";
                    // }

                    int maxlen = 0;
                    for(const auto& s: task.totalDecodedAnchors){
                        const int len = s.length();
                        if(len > maxlen){
                            maxlen = len;
                        }

                        // if(task.myReadId == 90 || task.mateReadId == 90){
                        //     std::cerr << s << "\n";
                        // }
                    }

                    // if(task.myReadId == 90 || task.mateReadId == 90){
                    //     std::cerr << "\n";
                    // }

                    const std::string& decodedAnchor = vecAccess(task.totalDecodedAnchors, 0);

                    const std::vector<int> shifts(task.totalAnchorBeginInExtendedRead.begin() + 1, task.totalAnchorBeginInExtendedRead.end());
                    std::vector<float> initialWeights(numsteps-1, 1.0f);


                    std::vector<char> stepstrings(maxlen * (numsteps-1), '\0');
                    std::vector<int> stepstringlengths(numsteps-1);
                    for(int c = 1; c < numsteps; c++){
                        std::copy(
                            vecAccess(task.totalDecodedAnchors, c).begin(),
                            vecAccess(task.totalDecodedAnchors, c).end(),
                            stepstrings.begin() + (c-1) * maxlen
                        );
                        vecAccess(stepstringlengths, c-1) = vecAccess(task.totalDecodedAnchors, c).size();
                    }

                    MultipleSequenceAlignment::InputData msaInput;
                    msaInput.useQualityScores = false;
                    msaInput.subjectLength = decodedAnchor.length();
                    msaInput.nCandidates = numsteps-1;
                    msaInput.candidatesPitch = maxlen;
                    msaInput.candidateQualitiesPitch = 0;
                    msaInput.subject = decodedAnchor.c_str();
                    msaInput.candidates = stepstrings.data();
                    msaInput.subjectQualities = nullptr;
                    msaInput.candidateQualities = nullptr;
                    msaInput.candidateLengths = stepstringlengths.data();
                    msaInput.candidateShifts = shifts.data();
                    msaInput.candidateDefaultWeightFactors = initialWeights.data();

                    MultipleSequenceAlignment msa;

                    msa.build(msaInput);

                    // if(task.myReadId == 90 || task.mateReadId == 90){
                    //     std::cerr << "Id " << task.myReadId << ", Final\n";
                    //     msa.print(std::cerr);
                    //     std::cerr << "\n";
                    // }

                    extendResult.success = true;

                    std::string extendedRead(msa.consensus.begin(), msa.consensus.end());
                    //std::cerr << "before: " << extendedRead << "\n";
                    std::copy(decodedAnchor.begin(), decodedAnchor.end(), extendedRead.begin());
                    if(task.mateHasBeenFound){
                        std::copy(
                            task.decodedMateRevC.begin(),
                            task.decodedMateRevC.end(),
                            extendedRead.begin() + extendedRead.length() - task.decodedMateRevC.length()
                        );
                    }
                    // extendedRead.replace(extendedRead.begin(), extendedRead.begin() + decodedAnchor, decodedAnchor.begin(), decodedAnchor.end());
                    // std::cerr << "after : " << extendedRead << "\n";
                    
                    // msa.print(std::cerr);
                    // std::cerr << "msa cons:\n";
                    // std::cerr << extendedRead << "\n";
                    // std::cerr << "new cons:\n";
                    // std::cerr << task.resultsequence << "\n";


                    extendResult.extendedRead = std::move(extendedRead);

                    extendResult.mateHasBeenFound = task.mateHasBeenFound;
                }
                // else{
                //     ; //no read extension possible
                // }
            }

            // if(extendResult.extendedRead.length() != task.resultsequence.length()){
            //     std::cerr << task.myReadId << "\n";
            //     std::cerr << extendResult.extendedRead << "\n";
            //     std::cerr << task.resultsequence << "\n";
            //     std::exit(0);
            // }
#endif
            extendResults.emplace_back(std::move(extendResult));

        }

        return extendResults;
    }


    std::vector<ReadExtenderBase::ExtendResult> ReadExtenderGpu::processSingleEndTasks(
        std::vector<ReadExtenderBase::Task>& tasks
    ){
        return processPairedEndTasks(tasks);
    }


    void ReadExtenderGpu::getCandidateReadIds(BatchData& batchData, hipStream_t stream) const{
        nvtx::push_range("gpu_hashing", 2);

        int totalNumValues = 0;

        gpuMinhasher->determineNumValues(
            minhashHandle,
            batchData.d_subjectSequencesData.get(),
            encodedSequencePitchInInts,
            batchData.d_anchorSequencesLength.get(),
            batchData.numTasks,
            batchData.d_numCandidatesPerAnchor.get(),
            totalNumValues,
            stream
        );

        hipStreamSynchronize(stream); CUERR;

        batchData.d_candidateReadIds.resize(totalNumValues);        

        if(totalNumValues == 0){
            hipMemsetAsync(batchData.d_numCandidatesPerAnchor.get(), 0, sizeof(int) * batchData.numTasks , stream); CUERR;
            hipMemsetAsync(batchData.d_numCandidatesPerAnchorPrefixSum.get(), 0, sizeof(int) * (1 + batchData.numTasks), stream); CUERR;
            return;
        }

        gpuMinhasher->retrieveValues(
            minhashHandle,
            nullptr,
            batchData.numTasks,              
            totalNumValues,
            batchData.d_candidateReadIds.get(),
            batchData.d_numCandidatesPerAnchor.get(),
            batchData.d_numCandidatesPerAnchorPrefixSum.get(),
            stream
        );

        nvtx::pop_range();
    }


    void ReadExtenderGpu::loadCandidateSequenceData(BatchData& batchData, hipStream_t stream) const{

        nvtx::push_range("gpu_loadCandidates", 2);

        const int totalNumCandidates = batchData.totalNumCandidates;

        gpuReadStorage->gatherSequences(
            readStorageHandle,
            batchData.d_candidateSequencesData.get(),
            encodedSequencePitchInInts,
            batchData.h_candidateReadIds.get(),
            batchData.d_candidateReadIds.get(), //device accessible
            totalNumCandidates,
            stream
        );

        gpuReadStorage->gatherSequenceLengths(
            readStorageHandle,
            batchData.d_candidateSequencesLength.get(),
            batchData.d_candidateReadIds.get(),
            totalNumCandidates,
            stream
        );

        readextendergpukernels::reverseComplement2bitKernel<128><<<320,128,0,stream>>>(
            batchData.d_candidateSequencesLength.get(),
            batchData.d_candidateSequencesData.get(),
            batchData.d_candidateSequencesRevcData.get(),
            totalNumCandidates,
            encodedSequencePitchInInts
        ); CUERR;

        nvtx::pop_range();
    }


    void ReadExtenderGpu::eraseDataOfRemovedMates(BatchData& batchData, hipStream_t stream) const{
        nvtx::push_range("gpu_eraseDataOfRemovedMates", 3);

        auto vecAccess = [](auto& vec, auto index) -> decltype(vec.at(index)){
            return vec.at(index);
        };

        if(batchData.numTasksWithMateRemoved > 0){
            const int totalNumCandidates = batchData.totalNumCandidates;

            constexpr int groupsize = 32;
            constexpr int blocksize = 128;
            constexpr int groupsperblock = blocksize / groupsize;
            dim3 block(blocksize,1,1);
            dim3 grid(SDIV(batchData.numTasksWithMateRemoved * groupsize, blocksize), 1, 1);
            const std::size_t smembytes = sizeof(unsigned int) * groupsperblock * encodedSequencePitchInInts;

            helpers::call_fill_kernel_async(batchData.d_flagscandidates.data(), batchData.d_flagscandidates.size(), false, stream);

            readextendergpukernels::filtermatekernel<blocksize,groupsize><<<grid, block, smembytes, stream>>>(
                batchData.d_anchormatedata.data(),
                batchData.d_candidateSequencesData.data(),
                encodedSequencePitchInInts,
                batchData.d_numCandidatesPerAnchor.data(),
                batchData.d_numCandidatesPerAnchorPrefixSum.data(),
                batchData.d_indexlist1.data(),
                batchData.numTasksWithMateRemoved,
                batchData.d_flagscandidates.data()
            ); CUERR;

            auto negate = [] __device__ (bool b){
                return !b;
            };

            hipcub::TransformInputIterator<bool, decltype(negate), bool*> d_keepflags(batchData.d_flagscandidates.data(), negate);

            std::size_t requiredCubSize = 0;
            std::size_t requiredCubSize1 = 0;
            std::size_t requiredCubSize2 = 0;
            hipError_t cubstatus = hipcub::DeviceScan::ExclusiveSum(
                nullptr,
                requiredCubSize1,
                d_keepflags, 
                batchData.d_intbuffercandidates.data(), 
                totalNumCandidates, 
                stream
            );
            assert(hipSuccess == cubstatus);

            cubstatus = hipcub::DeviceScan::InclusiveSum(
                nullptr,
                requiredCubSize2,
                batchData.d_numCandidatesPerAnchor.data(), 
                batchData.d_numCandidatesPerAnchorPrefixSum.data() + 1, 
                batchData.numTasks, 
                stream
            );
            assert(hipSuccess == cubstatus);

            requiredCubSize = std::max(requiredCubSize1, requiredCubSize2);

            void* cubtemp; cubAllocator->DeviceAllocate((void**)&cubtemp, requiredCubSize, stream);

            cubstatus = hipcub::DeviceScan::ExclusiveSum(
                cubtemp,
                requiredCubSize,
                d_keepflags, 
                batchData.d_intbuffercandidates.data(), 
                totalNumCandidates, 
                stream
            );
            assert(hipSuccess == cubstatus);

            helpers::lambda_kernel<<<4096, 128, 0, stream>>>(
                [
                    numTasks = batchData.numTasks,
                    encodedSequencePitchInInts = encodedSequencePitchInInts,
                    d_numCandidatesPerAnchor = batchData.d_numCandidatesPerAnchor.data(),
                    d_numCandidatesPerAnchorPrefixSum = batchData.d_numCandidatesPerAnchorPrefixSum.data(),
                    d_removalflags = batchData.d_flagscandidates.data(),
                    d_outputpositions = batchData.d_intbuffercandidates.data(),
                    d_candidateReadIds = batchData.d_candidateReadIds.data(),
                    d_candidateSequencesLength = batchData.d_candidateSequencesLength.data(),
                    d_candidateSequencesData = batchData.d_candidateSequencesData.data(),
                    d_candidateSequencesRevcData = batchData.d_candidateSequencesRevcData.data(),
                    d_anchorIndicesOfCandidates = batchData.d_anchorIndicesOfCandidates.data(),
                    d_candidateReadIdsOut = batchData.d_candidateReadIds2.data(),
                    d_candidateSequencesLengthOut = batchData.d_candidateSequencesLength2.data(),
                    d_candidateSequencesDataOut = batchData.d_candidateSequencesData2.data(),
                    d_candidateSequencesRevcDataOut = batchData.d_candidateSequencesRevcData2.data(),
                    d_anchorIndicesOfCandidatesOut = batchData.d_anchorIndicesOfCandidates2.data()
                ] __device__ (){

                    constexpr int elementsPerIteration = 128;
                    __shared__ bool smem_removalflags[elementsPerIteration];
                    __shared__ int smem_outputpositions[elementsPerIteration];

                    using BlockReduce = hipcub::BlockReduce<int, elementsPerIteration>;
                    __shared__ typename BlockReduce::TempStorage temp_storage;

                    auto group = cg::tiled_partition<8>(cg::this_thread_block());
                    const int numGroupsInBlock = blockDim.x / 8;
                    const int groupInBlock = threadIdx.x / 8;

                    for(int t = blockIdx.x; t < numTasks; t += gridDim.x){
                        const int numCandidates = d_numCandidatesPerAnchor[t];
                        const int inputOffset = d_numCandidatesPerAnchorPrefixSum[t];

                        int numSelected = 0;

                        for(int i = threadIdx.x; i < numCandidates; i += blockDim.x){
                            if(!d_removalflags[inputOffset + i]){
                                const int outputLocation = d_outputpositions[inputOffset + i];

                                d_candidateReadIdsOut[outputLocation] = d_candidateReadIds[inputOffset + i];
                                d_candidateSequencesLengthOut[outputLocation] = d_candidateSequencesLength[inputOffset + i];
                                d_anchorIndicesOfCandidatesOut[outputLocation] = d_anchorIndicesOfCandidates[inputOffset + i];

                                numSelected++;
                            }
                        }

                        for(int i = threadIdx.x; i < numCandidates * encodedSequencePitchInInts; i += blockDim.x){
                            const int which = i / encodedSequencePitchInInts;
                            const int what = i % encodedSequencePitchInInts;

                            if(!d_removalflags[inputOffset + which]){
                                d_candidateSequencesDataOut[d_outputpositions[inputOffset + which] * encodedSequencePitchInInts + what] = d_candidateSequencesData[(inputOffset + which) * encodedSequencePitchInInts + what];
                                d_candidateSequencesRevcDataOut[d_outputpositions[inputOffset + which] * encodedSequencePitchInInts + what] = d_candidateSequencesRevcData[(inputOffset + which) * encodedSequencePitchInInts + what];
                            }
                        }

                        numSelected = BlockReduce(temp_storage).Sum(numSelected);
                        __syncthreads();
                        
                        if(threadIdx.x == 0){
                            if(numSelected != numCandidates){
                                assert(numSelected < numCandidates);
                                d_numCandidatesPerAnchor[t] = numSelected;
                                //printf("task %d, removed %d\n", t, numCandidates - numSelected);
                            }
                        }

                    }
                }
            ); CUERR;

            //update prefix sum

            cubstatus = hipcub::DeviceScan::InclusiveSum(
                cubtemp,
                requiredCubSize,
                batchData.d_numCandidatesPerAnchor.data(), 
                batchData.d_numCandidatesPerAnchorPrefixSum.data() + 1, 
                batchData.numTasks, 
                stream
            );
            assert(hipSuccess == cubstatus);

            cubAllocator->DeviceFree(cubtemp);

            std::swap(batchData.d_candidateReadIds2, batchData.d_candidateReadIds);
            std::swap(batchData.d_candidateSequencesLength2, batchData.d_candidateSequencesLength);
            std::swap(batchData.d_candidateSequencesData2, batchData.d_candidateSequencesData);
            std::swap(batchData.d_candidateSequencesRevcData2, batchData.d_candidateSequencesRevcData);
            std::swap(batchData.d_anchorIndicesOfCandidates2, batchData.d_anchorIndicesOfCandidates);

        }

        nvtx::pop_range();
       
    }


    void ReadExtenderGpu::calculateAlignments(BatchData& batchData, hipStream_t stream) const{
        nvtx::push_range("gpu_alignment", 4);

        
        batchData.h_numAnchors[0] = batchData.numTasks;

        const bool* const d_anchorContainsN = nullptr;
        const bool* const d_candidateContainsN = nullptr;
        const bool removeAmbiguousAnchors = false;
        const bool removeAmbiguousCandidates = false;
        const int maxNumAnchors = batchData.numTasks;
        const int maxNumCandidates = batchData.totalNumCandidates;
        const int maximumSequenceLength = 100; //encodedSequencePitchInInts * 16;
        const int encodedSequencePitchInInts2Bit = encodedSequencePitchInInts;
        const int min_overlap = goodAlignmentProperties.min_overlap;
        const float maxErrorRate = goodAlignmentProperties.maxErrorRate;
        const float min_overlap_ratio = goodAlignmentProperties.min_overlap_ratio;
        const float estimatedNucleotideErrorRate = correctionOptions.estimatedErrorrate;

        auto callAlignmentKernel = [&](void* d_tempstorage, size_t& tempstoragebytes){

            call_popcount_rightshifted_hamming_distance_kernel_async(
                d_tempstorage,
                tempstoragebytes,
                batchData.d_alignment_overlaps.get(),
                batchData.d_alignment_shifts.get(),
                batchData.d_alignment_nOps.get(),
                batchData.d_alignment_isValid.get(),
                batchData.d_alignment_best_alignment_flags.get(),
                batchData.d_subjectSequencesData.get(),
                batchData.d_candidateSequencesData.get(),
                batchData.d_anchorSequencesLength.get(),
                batchData.d_candidateSequencesLength.get(),
                batchData.d_numCandidatesPerAnchorPrefixSum.get(),
                batchData.d_numCandidatesPerAnchor.get(),
                batchData.d_anchorIndicesOfCandidates.get(),
                //batchData.d_intbuffercandidates.get(),
                batchData.h_numAnchors.get(),
                &batchData.d_numCandidatesPerAnchorPrefixSum[batchData.numTasks],
                d_anchorContainsN,
                removeAmbiguousAnchors,
                d_candidateContainsN,
                removeAmbiguousCandidates,
                maxNumAnchors,
                maxNumCandidates,
                maximumSequenceLength,
                encodedSequencePitchInInts2Bit,
                min_overlap,
                maxErrorRate,
                min_overlap_ratio,
                estimatedNucleotideErrorRate,
                stream,
                kernelLaunchHandle
            );
        };

        const int num = batchData.totalNumCandidates;

        // assert(batchData.d_intbuffercandidates.size() >= num);
        // assert(batchData.h_numAnchors[0] == batchData.numTasks);
        // assert(batchData.d_numCandidatesPerAnchor.size() >= batchData.numTasks);
        // assert(batchData.d_numCandidatesPerAnchorPrefixSum.size() >= batchData.numTasks+1);

        // readextendergpukernels::setAnchorIndicesOfCandidateskernel<<<1024, 128, 0, stream>>>(
        //     batchData.d_intbuffercandidates.data(),
        //     batchData.h_numAnchors.data(),
        //     batchData.d_numCandidatesPerAnchor.get(),
        //     batchData.d_numCandidatesPerAnchorPrefixSum.get()
        // );

        size_t tempstoragebytes = 0;
        callAlignmentKernel(nullptr, tempstoragebytes);

        batchData.d_tempstorage.resize(tempstoragebytes);

        callAlignmentKernel(batchData.d_tempstorage.get(), tempstoragebytes);

        nvtx::pop_range();
    }



    void ReadExtenderGpu::filterAlignments(BatchData& batchData, hipStream_t stream) const{
        nvtx::push_range("gpu_filterAlignments", 5);

        const int totalNumCandidates = batchData.totalNumCandidates;
        const int numAnchors = batchData.numTasks;

        helpers::call_fill_kernel_async(batchData.d_flagscandidates.data(), batchData.d_flagscandidates.size(), true, stream);

        bool* const d_keepflags = batchData.d_flagscandidates.data();

        dim3 block(128,1,1);
        dim3 grid(numAnchors, 1, 1);

        //filter alignments of candidates. d_keepflags[i] will be set to false if candidate[i] should be removed
        //batchData.d_numCandidatesPerAnchor2[i] contains new number of candidates for anchor i
        helpers::lambda_kernel<<<grid, block, 0, stream>>>(
            [
                d_alignment_best_alignment_flags = batchData.d_alignment_best_alignment_flags.data(),
                d_alignment_shifts = batchData.d_alignment_shifts.data(),
                d_alignment_overlaps = batchData.d_alignment_overlaps.data(),
                d_anchorSequencesLength = batchData.d_anchorSequencesLength.data(),
                d_numCandidatesPerAnchor = batchData.d_numCandidatesPerAnchor.data(),
                d_numCandidatesPerAnchor2 = batchData.d_numCandidatesPerAnchor2.data(),
                d_numCandidatesPerAnchorPrefixSum = batchData.d_numCandidatesPerAnchorPrefixSum.data(),
                d_keepflags,
                min_overlap_ratio = goodAlignmentProperties.min_overlap_ratio,
                numAnchors
            ] __device__ (){

                using BlockReduceFloat = hipcub::BlockReduce<float, 128>;
                using BlockReduceInt = hipcub::BlockReduce<int, 128>;

                __shared__ union {
                    typename BlockReduceFloat::TempStorage floatreduce;
                    typename BlockReduceInt::TempStorage intreduce;
                } cubtemp;

                __shared__ int intbroadcast;
                __shared__ float floatbroadcast;

                for(int a = blockIdx.x; a < numAnchors; a += gridDim.x){
                    const int num = d_numCandidatesPerAnchor[a];
                    const int offset = d_numCandidatesPerAnchorPrefixSum[a];
                    const float anchorLength = d_anchorSequencesLength[a];
                    int removed = 0;

                    int threadReducedGoodAlignmentExists = 0;
                    float threadReducedRelativeOverlapThreshold = 0.0f;

                    //loop over candidates to compute relative overlap threshold

                    for(int c = threadIdx.x; c < num; c += blockDim.x){
                        const auto alignmentflag = d_alignment_best_alignment_flags[offset + c];
                        const int shift = d_alignment_shifts[offset + c];

                        if(alignmentflag != BestAlignment_t::None && shift >= 0){
                            bool goodAlignmentExists = false;
                            const float overlap = d_alignment_overlaps[offset + c];                            
                            const float relativeOverlap = overlap / anchorLength;
                            
                            if(relativeOverlap < 1.0f && fgeq(relativeOverlap, min_overlap_ratio)){
                                threadReducedGoodAlignmentExists = 1;
                                const float tmp = floorf(relativeOverlap * 10.0f) / 10.0f;
                                threadReducedRelativeOverlapThreshold = fmaxf(threadReducedRelativeOverlapThreshold, tmp);
                            }

                            // while(!goodAlignmentExists && fgeq(relativeOverlapThreshold, min_overlap_ratio)){

                            //     goodAlignmentExists = fgeq(relativeOverlap, relativeOverlapThreshold) && relativeOverlap < 1.0f;

                            //     if(!goodAlignmentExists){
                            //         relativeOverlapThreshold -= 0.1f;
                            //     }
                            // }

                            // if(goodAlignmentExists){
                            //     threadReducedGoodAlignmentExists = 1;
                            //     threadReducedRelativeOverlapThreshold = max(threadReducedRelativeOverlapThreshold, relativeOverlapThreshold);
                            // }

                            // if(a == 1){
                            //     printf("a %d c %d relativeOverlap %f, thread good %d, thresh %f\n", 
                            //         a, c, relativeOverlap, goodAlignmentExists, threadReducedRelativeOverlapThreshold);
                            // }
                        }else{
                            //remove alignment with negative shift
                            d_keepflags[offset + c] = false;
                            removed++;
                        }

                        
                    }
                    // __syncthreads(); //debug
                    // if(threadIdx.x < num){
                    //     printf("a %d thread good %d, thresh %f\n", a, threadReducedGoodAlignmentExists, threadReducedRelativeOverlapThreshold);
                    // }
                    // __syncthreads(); //debug

                    int blockreducedGoodAlignmentExists = BlockReduceInt(cubtemp.intreduce)
                        .Sum(threadReducedGoodAlignmentExists);
                    if(threadIdx.x == 0){
                        intbroadcast = blockreducedGoodAlignmentExists;
                        //printf("task %d good: %d\n", a, blockreducedGoodAlignmentExists);
                    }
                    __syncthreads();

                    blockreducedGoodAlignmentExists = intbroadcast;

                    if(blockreducedGoodAlignmentExists > 0){
                        float blockreducedRelativeOverlapThreshold = BlockReduceFloat(cubtemp.floatreduce)
                            .Reduce(threadReducedRelativeOverlapThreshold, hipcub::Max());
                        if(threadIdx.x == 0){
                            floatbroadcast = blockreducedRelativeOverlapThreshold;
                            //printf("task %d thresh: %f\n", a, blockreducedRelativeOverlapThreshold);
                        }
                        __syncthreads();

                        blockreducedRelativeOverlapThreshold = floatbroadcast;

                        // loop over candidates and remove those with an alignment overlap threshold smaller than the computed threshold
                        for(int c = threadIdx.x; c < num; c += blockDim.x){
    
                            if(d_keepflags[offset + c]){
                                const float overlap = d_alignment_overlaps[offset + c];                            
                                const float relativeOverlap = overlap / anchorLength;                 
    
                                if(!fgeq(relativeOverlap, blockreducedRelativeOverlapThreshold)){
                                    d_keepflags[offset + c] = false;
                                    removed++;
                                }
                            }
                        }
                    }else{
                        //NOOP.
                        //if no good alignment exists, no candidate is removed. we will try to work with the not-so-good alignments
                    }

                    removed = BlockReduceInt(cubtemp.intreduce).Sum(removed);

                    if(threadIdx.x == 0){
                        d_numCandidatesPerAnchor2[a] = num - removed;
                        //printf("task %d remaining: %d - %d = %d\n", a, num, removed, num - removed);
                    }
                    __syncthreads();
                }
            }
        ); CUERR;

        //setup cub 
        auto d_zip_input = thrust::make_zip_iterator(
            thrust::make_tuple(
                batchData.d_alignment_nOps.data(),
                batchData.d_alignment_overlaps.data(),
                batchData.d_alignment_shifts.data(),
                batchData.d_alignment_isValid.data(),
                batchData.d_alignment_best_alignment_flags.data(),
                batchData.d_candidateReadIds.data(),
                batchData.d_candidateSequencesLength.data()
            )
        );

        auto d_zip_output = thrust::make_zip_iterator(
            thrust::make_tuple(
                batchData.d_alignment_nOps2.data(),
                batchData.d_alignment_overlaps2.data(),
                batchData.d_alignment_shifts2.data(),
                batchData.d_alignment_isValid2.data(),
                batchData.d_alignment_best_alignment_flags2.data(),
                batchData.d_candidateReadIds2.data(),
                batchData.d_candidateSequencesLength2.data()
            )
        );

        std::size_t requiredCubSize1 = 0;
        hipError_t cubstatus = hipcub::DeviceSelect::Flagged(
            nullptr, 
            requiredCubSize1, 
            d_zip_input, 
            d_keepflags, 
            d_zip_output, 
            batchData.d_numCandidates.data(), 
            totalNumCandidates, 
            stream
        );
        assert(cubstatus == hipSuccess);

        std::size_t requiredCubSize2 = 0;
        cubstatus = hipcub::DeviceScan::ExclusiveSum(
            nullptr,
            requiredCubSize2,
            d_keepflags, 
            batchData.d_intbuffercandidates.data(), 
            totalNumCandidates, 
            stream
        );
        assert(cubstatus == hipSuccess);

        std::size_t requiredCubSize3 = 0;
        cubstatus = hipcub::DeviceScan::InclusiveSum(
            nullptr,
            requiredCubSize3,
            batchData.d_numCandidatesPerAnchor2.data(), 
            batchData.d_numCandidatesPerAnchorPrefixSum.data() + 1, 
            batchData.numTasks, 
            stream
        );
        assert(hipSuccess == cubstatus);

        std::size_t requiredCubSize = std::max(std::max(requiredCubSize1, requiredCubSize2), requiredCubSize3);
        void* cubtemp; cubAllocator->DeviceAllocate((void**)&cubtemp, requiredCubSize, stream);
        //batchData.d_tempstorage.resize(requiredCubSize);

        //compute output positions for selected candidates

        cubstatus = hipcub::DeviceScan::ExclusiveSum(
            cubtemp, 
            requiredCubSize,
            d_keepflags, 
            batchData.d_intbuffercandidates.data(), 
            totalNumCandidates, 
            stream
        );
        assert(cubstatus == hipSuccess);

        //compact zip data
        cubstatus = hipcub::DeviceSelect::Flagged(
            cubtemp, 
            requiredCubSize, 
            d_zip_input, 
            d_keepflags, 
            d_zip_output, 
            batchData.d_numCandidates.data(), 
            totalNumCandidates, 
            stream
        );
        assert(cubstatus == hipSuccess);

        //compact sequence data. if alignmentflag is forward, forward sequence data will be copied, 
        //else reverse complement will be copied
        helpers::lambda_kernel<<<4096, 128, 0, stream>>>(
            [
                encodedSequencePitchInInts = encodedSequencePitchInInts,
                d_keepflags,
                totalNumCandidates,
                d_outputpositions = batchData.d_intbuffercandidates.data(),
                d_alignment_best_alignment_flags = batchData.d_alignment_best_alignment_flags.data(),
                d_candidateSequencesData = batchData.d_candidateSequencesData.data(),
                d_candidateSequencesRevcData = batchData.d_candidateSequencesRevcData.data(),
                d_candidateSequencesDataOut = batchData.d_candidateSequencesData2.data()
            ] __device__ (){

                const int tid = threadIdx.x + blockIdx.x * blockDim.x;
                const int stride = blockDim.x * gridDim.x;
                const int elements = totalNumCandidates * encodedSequencePitchInInts;

                for(int i = tid; i < elements; i += stride){
                    const int which = i / encodedSequencePitchInInts;
                    const int what = i % encodedSequencePitchInInts;

                    if(d_keepflags[which]){

                        const int outputindex = d_outputpositions[which] * encodedSequencePitchInInts + what;
                        const int inputindex = which * encodedSequencePitchInInts + what;

                        const auto alignmentflag = d_alignment_best_alignment_flags[which];
                        
                        if(alignmentflag == BestAlignment_t::Forward){                             
                            d_candidateSequencesDataOut[outputindex] = d_candidateSequencesData[inputindex];
                        }else{
                            d_candidateSequencesDataOut[outputindex] = d_candidateSequencesRevcData[inputindex];
                        }
                    }
                }
            }
        ); CUERR;


        //update prefix sum
        cubstatus = hipcub::DeviceScan::InclusiveSum(
            cubtemp, 
            requiredCubSize, 
            batchData.d_numCandidatesPerAnchor2.data(), 
            batchData.d_numCandidatesPerAnchorPrefixSum.data() + 1, 
            batchData.numTasks, 
            stream
        );
        assert(hipSuccess == cubstatus);

        cubAllocator->DeviceFree(cubtemp);

        std::swap(batchData.d_alignment_nOps2, batchData.d_alignment_nOps);
        std::swap(batchData.d_alignment_overlaps2, batchData.d_alignment_overlaps);
        std::swap(batchData.d_alignment_shifts2, batchData.d_alignment_shifts);
        std::swap(batchData.d_alignment_isValid2, batchData.d_alignment_isValid);
        std::swap(batchData.d_alignment_best_alignment_flags2, batchData.d_alignment_best_alignment_flags);
        std::swap(batchData.d_candidateReadIds2, batchData.d_candidateReadIds);
        std::swap(batchData.d_candidateSequencesLength2, batchData.d_candidateSequencesLength);
        std::swap(batchData.d_numCandidatesPerAnchor2, batchData.d_numCandidatesPerAnchor);
        std::swap(batchData.d_candidateSequencesData2, batchData.d_candidateSequencesData);

        nvtx::pop_range();
    }




}