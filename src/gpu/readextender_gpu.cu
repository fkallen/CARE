#include "hip/hip_runtime.h"
#include <gpu/readextender_gpu.hpp>
#include <readextenderbase.hpp>

#include <vector>
#include <algorithm>
#include <sequencehelpers.hpp>
#include <string>

#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/transform_iterator.h>

#include <gpu/segmented_set_operations.cuh>

namespace care{


    //flag candidates to remove because they are equal to anchor id or equal to mate id
    __global__
    void flagCandidateIdsWhichAreEqualToAnchorOrMateKernel(
        const read_number* candidateReadIds,
        const read_number* anchorReadIds,
        const read_number* mateReadIds,
        const int* numCandidatesPerAnchorPrefixSum,
        const int* numCandidatesPerAnchor,
        bool* keepflags, // size numCandidates
        bool* mateRemovedFlags, //size numTasks
        int* numCandidatesPerAnchorOut,
        int numTasks,
        bool isPairedEnd
    ){

        using BlockReduceInt = hipcub::BlockReduce<int, 128>;

        __shared__ typename BlockReduceInt::TempStorage intreduce1;
        __shared__ typename BlockReduceInt::TempStorage intreduce2;

        for(int a = blockIdx.x; a < numTasks; a += gridDim.x){
            const int size = numCandidatesPerAnchor[a];
            const int offset = numCandidatesPerAnchorPrefixSum[a];
            const read_number anchorId = anchorReadIds[a];
            read_number mateId = 0;
            if(isPairedEnd){
                mateId = mateReadIds[a];
            }

            int mateIsRemoved = 0;
            int numRemoved = 0;

            // if(threadIdx.x == 0){
            //     printf("looking for anchor %u, mate %u\n", anchorId, mateId);
            // }
            __syncthreads();

            for(int i = threadIdx.x; i < size; i+= blockDim.x){
                bool keep = true;

                const read_number candidateId = candidateReadIds[offset + i];
                //printf("tid %d, comp %u at position %d\n", threadIdx.x, candidateId, offset + i);

                if(candidateId == anchorId){
                    keep = false;
                    numRemoved++;
                }

                if(isPairedEnd && candidateId == mateId){
                    if(keep){
                        keep = false;
                        numRemoved++;
                    }
                    mateIsRemoved++;
                    //printf("mate removed. i = %d\n", i);
                }

                keepflags[offset + i] = keep;
            }
            //printf("tid = %d, mateIsRemoved = %d\n", threadIdx.x, mateIsRemoved);
            int numRemovedBlock = BlockReduceInt(intreduce1).Sum(numRemoved);
            int numMateRemovedBlock = BlockReduceInt(intreduce2).Sum(mateIsRemoved);
            if(threadIdx.x == 0){
                numCandidatesPerAnchorOut[a] = size - numRemovedBlock;
                //printf("numMateRemovedBlock %d\n", numMateRemovedBlock);
                if(numMateRemovedBlock > 0){
                    mateRemovedFlags[a] = true;
                }else{
                    mateRemovedFlags[a] = false;
                }
            }
        }
    }















    std::vector<ReadExtenderBase::ExtendResult> ReadExtenderGpu::processPairedEndTasks(
        std::vector<ReadExtenderBase::Task>& tasks
    ) {
 
        std::vector<ExtendResult> extendResults;

        std::vector<int> indicesOfActiveTasks(tasks.size());
        std::vector<int> indicesOfActiveTasksTmp(tasks.size());
        std::iota(indicesOfActiveTasks.begin(), indicesOfActiveTasks.end(), 0);

        std::map<read_number, int> splitTracker; //counts number of tasks per read id, which can change by splitting a task
        for(const auto& t : tasks){
            splitTracker[t.myReadId] = 1;
        }

        //set input string as current anchor
        for(auto& task : tasks){
            std::string decodedAnchor(task.currentAnchorLength, '\0');

            SequenceHelpers::decode2BitSequence(
                &decodedAnchor[0],
                task.currentAnchor.data(),
                task.currentAnchorLength
            );

            task.totalDecodedAnchors.emplace_back(std::move(decodedAnchor));
            task.totalAnchorBeginInExtendedRead.emplace_back(0);
        }


#if 1
        //undo: replace vecAccess\(([a-zA-z]+), ([a-zA-z]+)\) by $1[$2]
        auto vecAccess = [](auto& vec, auto index) -> decltype(vec[index]){
            return vec[index];
        };
#else 
        auto vecAccess = [](auto& vec, auto index) -> decltype(vec.at(index)){
            return vec.at(index);
        };
#endif 

        hipStream_t firstStream = streams[0];
        hipStream_t secondStream = streams[1];

        const int numTasks = tasks.size();

        batchData.h_numAnchors.resize(1);
        batchData.h_numCandidates.resize(1);
        batchData.d_numAnchors.resize(1);
        batchData.d_numCandidates.resize(1);

        auto processTasksOldStyle = [this, vecAccess, &indicesOfActiveTasks](auto& tasks){

            getCandidateReadIds(tasks, indicesOfActiveTasks);

            for(int indexOfActiveTask : indicesOfActiveTasks){
                auto& task = vecAccess(tasks, indexOfActiveTask);

                // remove self from candidate list
                auto readIdPos = std::lower_bound(
                    task.candidateReadIds.begin(),                                            
                    task.candidateReadIds.end(),
                    task.myReadId
                );

                if(readIdPos != task.candidateReadIds.end() && *readIdPos == task.myReadId){
                    task.candidateReadIds.erase(readIdPos);
                }

                if(task.pairedEnd){

                    //remove mate of input from candidate list
                    auto mateReadIdPos = std::lower_bound(
                        task.candidateReadIds.begin(),                                            
                        task.candidateReadIds.end(),
                        task.mateReadId
                    );

                    if(mateReadIdPos != task.candidateReadIds.end() && *mateReadIdPos == task.mateReadId){
                        task.candidateReadIds.erase(mateReadIdPos);
                        task.mateRemovedFromCandidates = true;
                    }
                }
            }

            for(int indexOfActiveTask : indicesOfActiveTasks){
                auto& task = vecAccess(tasks, indexOfActiveTask);

                std::vector<read_number> tmp(task.candidateReadIds.size());

                auto end = std::set_difference(
                    task.candidateReadIds.begin(),
                    task.candidateReadIds.end(),
                    task.allUsedCandidateReadIdPairs.begin(),
                    task.allUsedCandidateReadIdPairs.end(),
                    tmp.begin()
                );

                tmp.erase(end, tmp.end());

                std::swap(task.candidateReadIds, tmp);
            }

            loadCandidateSequenceData(tasks, indicesOfActiveTasks);

            eraseDataOfRemovedMates(tasks, indicesOfActiveTasks);

            calculateAlignments(tasks, indicesOfActiveTasks);

            #if 1

            for(int indexOfActiveTask : indicesOfActiveTasks){
                auto& task = vecAccess(tasks, indexOfActiveTask);

                /*
                    Remove bad alignments
                */        

                const int size = task.alignments.size();

                std::vector<int> positionsOfCandidatesToKeep(size);
                std::vector<int> tmpPositionsOfCandidatesToKeep(size);

                task.numRemainingCandidates = 0;

                //select candidates with good alignment and positive shift
                for(int c = 0; c < size; c++){
                    const BestAlignment_t alignmentFlag0 = vecAccess(task.alignmentFlags, c);
                    
                    if(alignmentFlag0 != BestAlignment_t::None && vecAccess(task.alignments, c).shift >= 0){
                        vecAccess(positionsOfCandidatesToKeep, task.numRemainingCandidates) = c;
                        task.numRemainingCandidates++;
                    }else{
                        ; // remove alignment
                    }
                }

                positionsOfCandidatesToKeep.erase(
                    positionsOfCandidatesToKeep.begin() + task.numRemainingCandidates, 
                    positionsOfCandidatesToKeep.end()
                );

                if(task.numRemainingCandidates == 0){
                    task.abort = true;
                    task.abortReason = AbortReason::NoPairedCandidatesAfterAlignment;

                    task.candidateReadIds.erase(task.candidateReadIds.begin(), task.candidateReadIds.end());
                    task.candidateSequenceLengths.erase(task.candidateSequenceLengths.begin(), task.candidateSequenceLengths.end());
                    task.candidateSequenceData.erase(task.candidateSequenceData.begin(), task.candidateSequenceData.end());
                    task.alignments.erase(task.alignments.begin(), task.alignments.end());
                    task.alignmentFlags.erase(task.alignmentFlags.begin(), task.alignmentFlags.end());

                    continue; //stop processing task
                }

                float relativeOverlapThreshold = 0.9f;
                bool goodAlignmentExists = false;

                while(!goodAlignmentExists && fgeq(relativeOverlapThreshold, goodAlignmentProperties.min_overlap_ratio)){                    

                    goodAlignmentExists = std::any_of(
                        positionsOfCandidatesToKeep.begin(), 
                        positionsOfCandidatesToKeep.end(),
                        [&](const auto& position){
                            const auto& alignment = vecAccess(task.alignments, position);
                            const float relativeOverlap = float(alignment.overlap) / float(task.currentAnchorLength);
                            return fgeq(relativeOverlap, relativeOverlapThreshold) && relativeOverlap < 1.0f;
                        }
                    );

                    if(!goodAlignmentExists){
                        relativeOverlapThreshold -= 0.1f;
                    }
                }

                // std::cerr << "task " << indexOfActiveTask
                //      << ", goodAlignmentExists " << goodAlignmentExists
                //      <<", relativeOverlapThreshold " << relativeOverlapThreshold;
                

                if(goodAlignmentExists){
                    positionsOfCandidatesToKeep.erase(
                        std::remove_if(
                            positionsOfCandidatesToKeep.begin(), 
                            positionsOfCandidatesToKeep.end(),
                            [&](const auto& position){
                                const auto& alignment = vecAccess(task.alignments, position);
                                const float relativeOverlap = float(alignment.overlap) / float(task.currentAnchorLength);
                                return !fgeq(relativeOverlap, relativeOverlapThreshold);
                            }
                        ),
                        positionsOfCandidatesToKeep.end()
                    );
                    task.numRemainingCandidates = positionsOfCandidatesToKeep.size();
                }

                // std::cerr << ", numRemainingCandidates = " << task.numRemainingCandidates << "\n";

                // std::cerr << "positionsOfCandidatesToKeep: ";

                // for(int x : positionsOfCandidatesToKeep){
                //     std::cerr << x << " ";
                // }
                // std::cerr << "\n";

                //std::cerr << ", remaining candidates " << task.numRemainingCandidates << "\n";


                //compact selected candidates inplace

                

                {
                    task.candidateSequenceData.resize(task.numRemainingCandidates * encodedSequencePitchInInts);

                    for(int c = 0; c < task.numRemainingCandidates; c++){
                        const int index = vecAccess(positionsOfCandidatesToKeep, c);

                        vecAccess(task.alignments, c) = vecAccess(task.alignments, index);
                        vecAccess(task.alignmentFlags, c) = vecAccess(task.alignmentFlags, index);
                        vecAccess(task.candidateReadIds, c) = vecAccess(task.candidateReadIds, index);
                        vecAccess(task.candidateSequenceLengths, c) = vecAccess(task.candidateSequenceLengths, index);
                        
                        assert(vecAccess(task.alignmentFlags, index) != BestAlignment_t::None);

                        // std::cerr << "cand " << index << " dir " 
                        //     << ((vecAccess(task.alignmentFlags, index) == BestAlignment_t::Forward) ? 'f' : 'r') << "\n";

                        if(vecAccess(task.alignmentFlags, index) == BestAlignment_t::Forward){
                            std::copy_n(
                                task.candidateSequencesFwdData.data() + index * encodedSequencePitchInInts,
                                encodedSequencePitchInInts,
                                task.candidateSequenceData.data() + c * encodedSequencePitchInInts
                            );
                        }else{
                            //BestAlignment_t::ReverseComplement

                            std::copy_n(
                                task.candidateSequencesRevcData.data() + index * encodedSequencePitchInInts,
                                encodedSequencePitchInInts,
                                task.candidateSequenceData.data() + c * encodedSequencePitchInInts
                            );
                        }

                        // //not sure if these 2 arrays will be required further on
                        // std::copy_n(
                        //     candidateSequencesFwdData.data() + index * encodedSequencePitchInInts,
                        //     encodedSequencePitchInInts,
                        //     candidateSequencesFwdData.data() + c * encodedSequencePitchInInts
                        // );

                        // std::copy_n(
                        //     candidateSequencesRevcData.data() + index * encodedSequencePitchInInts,
                        //     encodedSequencePitchInInts,
                        //     candidateSequencesRevcData.data() + c * encodedSequencePitchInInts
                        // );
                        
                    }

                    //erase past-end elements
                    task.alignments.erase(
                        task.alignments.begin() + task.numRemainingCandidates, 
                        task.alignments.end()
                    );
                    task.alignmentFlags.erase(
                        task.alignmentFlags.begin() + task.numRemainingCandidates, 
                        task.alignmentFlags.end()
                    );
                    task.candidateReadIds.erase(
                        task.candidateReadIds.begin() + task.numRemainingCandidates, 
                        task.candidateReadIds.end()
                    );
                    task.candidateSequenceLengths.erase(
                        task.candidateSequenceLengths.begin() + task.numRemainingCandidates, 
                        task.candidateSequenceLengths.end()
                    );
                    // //not sure if these 2 arrays will be required further on
                    // candidateSequencesFwdData.erase(
                    //     candidateSequencesFwdData.begin() + task.numRemainingCandidates * encodedSequencePitchInInts, 
                    //     candidateSequencesFwdData.end()
                    // );
                    // candidateSequencesRevcData.erase(
                    //     candidateSequencesRevcData.begin() + task.numRemainingCandidates * encodedSequencePitchInInts, 
                    //     candidateSequencesRevcData.end()
                    // );
                    
                }

            }
            #endif
        };
        

        while(indicesOfActiveTasks.size() > 0){

            // auto debugtasks = tasks;
            // processTasksOldStyle(debugtasks);

            //perform one extension iteration for active tasks

            //setup batchdata for active tasks
            const int numActiveTasks = indicesOfActiveTasks.size();
            batchData.numTasks = numActiveTasks;

            batchData.h_numAnchors.resize(1);
            batchData.d_numAnchors.resize(1);
            batchData.h_numCandidates.resize(1);
            batchData.d_numCandidates.resize(1);

            batchData.h_anchorReadIds.resize(numActiveTasks);
            batchData.d_anchorReadIds.resize(numActiveTasks);
            batchData.h_mateReadIds.resize(numActiveTasks);
            batchData.d_mateReadIds.resize(numActiveTasks);
            
            batchData.h_subjectSequencesData.resize(encodedSequencePitchInInts * numActiveTasks);
            batchData.d_subjectSequencesData.resize(encodedSequencePitchInInts * numActiveTasks);
            batchData.h_anchorSequencesLength.resize(numActiveTasks);
            batchData.d_anchorSequencesLength.resize(numActiveTasks);

            batchData.h_anchormatedata.resize(numActiveTasks * encodedSequencePitchInInts);
            batchData.d_anchormatedata.resize(numActiveTasks * encodedSequencePitchInInts);

            batchData.h_numCandidatesPerAnchor.resize(numActiveTasks);
            batchData.d_numCandidatesPerAnchor.resize(numActiveTasks);
            batchData.h_numCandidatesPerAnchor2.resize(numActiveTasks);
            batchData.d_numCandidatesPerAnchor2.resize(numActiveTasks);
            batchData.h_numCandidatesPerAnchor3.resize(numActiveTasks);
            batchData.d_numCandidatesPerAnchor3.resize(numActiveTasks);
            batchData.h_numCandidatesPerAnchorPrefixSum.resize(numActiveTasks+1);
            batchData.h_numCandidatesPerAnchorPrefixSum2.resize(numActiveTasks+1);
            batchData.h_numCandidatesPerAnchorPrefixSum3.resize(numActiveTasks+1);
            batchData.d_numCandidatesPerAnchorPrefixSum.resize(numActiveTasks+1);
            batchData.d_numCandidatesPerAnchorPrefixSum2.resize(numActiveTasks+1);
            batchData.d_numCandidatesPerAnchorPrefixSum3.resize(numActiveTasks+1);

            batchData.h_indexlist1.resize(numActiveTasks);
            batchData.d_indexlist1.resize(numActiveTasks);

            batchData.h_indexlist2.resize(numActiveTasks);
            batchData.d_indexlist2.resize(numActiveTasks);


            for(int t = 0; t < numActiveTasks; t++){
                const auto& task = tasks[indicesOfActiveTasks[t]];

                batchData.h_anchorReadIds[t] = task.myReadId;
                batchData.h_mateReadIds[t] = task.mateReadId;
    
                if(task.iteration >= 0){
    
                    batchData.h_anchorSequencesLength[t] = task.currentAnchorLength;
    
                    std::copy(
                        task.currentAnchor.begin(),
                        task.currentAnchor.end(),
                        batchData.h_subjectSequencesData.get() + t * encodedSequencePitchInInts
                    );
                }else{
                    //only hash kmers which include extended positions
    
                    const int extendedPositionsPreviousIteration 
                        = task.totalAnchorBeginInExtendedRead.at(task.iteration) - task.totalAnchorBeginInExtendedRead.at(task.iteration-1);
    
                    const int lengthToHash = std::min(task.currentAnchorLength, kmerLength + extendedPositionsPreviousIteration - 1);
                    batchData.h_anchorSequencesLength[t] = lengthToHash;
    
                    //std::cerr << "lengthToHash = " << lengthToHash << "\n";
    
                    std::vector<char> buf(task.currentAnchorLength);
                    SequenceHelpers::decode2BitSequence(buf.data(), task.currentAnchor.data(), task.currentAnchorLength);
                    SequenceHelpers::encodeSequence2Bit(
                        batchData.h_subjectSequencesData.get() + t * encodedSequencePitchInInts, 
                        buf.data() + task.currentAnchorLength - lengthToHash, 
                        lengthToHash
                    );    
                }
            }

            hipMemcpyAsync(
                batchData.d_subjectSequencesData.get(),
                batchData.h_subjectSequencesData.get(),
                sizeof(unsigned int) * numActiveTasks * encodedSequencePitchInInts,
                H2D,
                firstStream
            ); CUERR;
    
            hipMemcpyAsync(
                batchData.d_anchorSequencesLength.get(),
                batchData.h_anchorSequencesLength.get(),
                sizeof(int) * numActiveTasks,
                H2D,
                firstStream
            ); CUERR;

            hashTimer.start();

            getCandidateReadIds(batchData, firstStream);

            hipMemcpyAsync(
                batchData.h_numCandidatesPerAnchorPrefixSum.get(),
                batchData.d_numCandidatesPerAnchorPrefixSum.get(),
                sizeof(int) * (batchData.numTasks+1),
                D2H,
                firstStream
            ); CUERR;
    
            hipStreamSynchronize(firstStream); CUERR;

            int totalNumCandidates = batchData.h_numCandidatesPerAnchorPrefixSum[batchData.numTasks];
    
            hipMemcpyAsync(
                batchData.h_candidateReadIds.get(),
                batchData.d_candidateReadIds.get(),
                sizeof(read_number) * totalNumCandidates,
                D2H,
                firstStream
            ); CUERR;

            hipMemcpyAsync(
                batchData.h_numCandidatesPerAnchor.get(),
                batchData.d_numCandidatesPerAnchor.get(),
                sizeof(int) * batchData.numTasks,
                D2H,
                firstStream
            ); CUERR;
    
            hipStreamSynchronize(firstStream); CUERR;

            batchData.h_segmentIds1.resize(totalNumCandidates);
            batchData.h_segmentIds3.resize(totalNumCandidates);
            batchData.d_segmentIds1.resize(totalNumCandidates);
            batchData.d_segmentIds3.resize(totalNumCandidates);
            batchData.h_flagscandidates.resize(totalNumCandidates);
            batchData.d_flagscandidates.resize(totalNumCandidates);
            batchData.h_flagsanchors.resize(batchData.numTasks);
            batchData.d_flagsanchors.resize(batchData.numTasks);
            batchData.h_candidateReadIds2.resize(totalNumCandidates);
            batchData.d_candidateReadIds2.resize(totalNumCandidates);
            batchData.h_candidateReadIds3.resize(totalNumCandidates);

            //compare (segmentid, value) tuples
            // auto comp = [] __device__ (const auto& t1, const auto& t2){
            //     const int idl = thrust::get<0>(t1);
            //     const int idr = thrust::get<0>(t2);
    
            //     if(idl < idr) return true;
            //     if(idl > idr) return false;
    
            //     return thrust::get<1>(t1) < thrust::get<1>(t2);
            // };

            helpers::call_fill_kernel_async(batchData.d_flagscandidates.data(), batchData.d_flagscandidates.size(), false, firstStream);

            //flag candidates to remove because they are equal to anchor id or equal to mate id
            flagCandidateIdsWhichAreEqualToAnchorOrMateKernel<<<4096, 128, 0, firstStream>>>(
                batchData.h_candidateReadIds.data(),
                batchData.h_anchorReadIds.data(),
                batchData.h_mateReadIds.data(),
                batchData.h_numCandidatesPerAnchorPrefixSum.data(),
                batchData.h_numCandidatesPerAnchor.data(),
                batchData.d_flagscandidates.data(),
                batchData.d_flagsanchors.data(),
                batchData.d_numCandidatesPerAnchor2.data(),
                batchData.numTasks,
                tasks[0].pairedEnd
            );
            CUERR;

            int newNumCandidates = thrust::distance(
                batchData.d_candidateReadIds2.data(),
                thrust::copy_if(
                    thrust::cuda::par(thrustallocator).on(firstStream),
                    batchData.d_candidateReadIds.data(),
                    batchData.d_candidateReadIds.data() + totalNumCandidates,
                    batchData.d_flagscandidates.data(),
                    batchData.d_candidateReadIds2.data(),
                    thrust::identity<int>()
                )
            );

            helpers::call_set_kernel_async(batchData.d_numCandidatesPerAnchorPrefixSum2.data(), 0, 0, firstStream);

            thrust::inclusive_scan(
                thrust::cuda::par(thrustallocator).on(firstStream),
                batchData.d_numCandidatesPerAnchor2.begin(),
                batchData.d_numCandidatesPerAnchor2.end(),
                batchData.d_numCandidatesPerAnchorPrefixSum2.begin() + 1
            );

            helpers::call_fill_kernel_async(batchData.d_segmentIds1.data(), newNumCandidates, 0, firstStream);

            thrust::scatter_if(
                thrust::cuda::par(thrustallocator).on(firstStream),
                thrust::counting_iterator<int>(0),
                thrust::counting_iterator<int>(batchData.numTasks),
                batchData.d_numCandidatesPerAnchorPrefixSum2.begin(),
                batchData.d_numCandidatesPerAnchor2.begin(),
                batchData.d_segmentIds1.begin()
            );

            thrust::inclusive_scan(
                thrust::cuda::par(thrustallocator).on(firstStream),
                batchData.d_segmentIds1.begin(),
                batchData.d_segmentIds1.end(),
                batchData.d_segmentIds1.begin(),
                thrust::maximum<int>()
            );

            std::size_t totalNumberOfUsedIds = 0;
            for(int i = 0; i < batchData.numTasks; i++){
                auto& task = vecAccess(tasks, indicesOfActiveTasks[i]);
                totalNumberOfUsedIds += task.allUsedCandidateReadIdPairs.size();
            }
            batchData.h_usedReadIds.resize(totalNumberOfUsedIds);
            batchData.h_numUsedReadIdsPerAnchor.resize(batchData.numTasks);
            batchData.h_numUsedReadIdsPerAnchorPrefixSum.resize(batchData.numTasks);

            batchData.h_numUsedReadIdsPerAnchorPrefixSum[0] = 0;
            auto h_usedReadIdsIter = batchData.h_usedReadIds.begin();
            for(int i = 0; i < batchData.numTasks; i++){
                auto& task = vecAccess(tasks, indicesOfActiveTasks[i]);
                h_usedReadIdsIter = std::copy(
                    task.allUsedCandidateReadIdPairs.begin(),
                    task.allUsedCandidateReadIdPairs.end(),
                    h_usedReadIdsIter
                );
                batchData.h_numUsedReadIdsPerAnchor[i] = task.allUsedCandidateReadIdPairs.size();

                if(i < batchData.numTasks - 1){
                    batchData.h_numUsedReadIdsPerAnchorPrefixSum[i+1] 
                        = batchData.h_numUsedReadIdsPerAnchorPrefixSum[i] + batchData.h_numUsedReadIdsPerAnchor[i];
                }
            }

            batchData.d_segmentIds2.resize(totalNumberOfUsedIds);

            helpers::call_fill_kernel_async(batchData.d_segmentIds2.data(), totalNumberOfUsedIds, 0, firstStream);

            thrust::scatter_if(
                thrust::cuda::par(thrustallocator).on(firstStream),
                thrust::counting_iterator<int>(0),
                thrust::counting_iterator<int>(batchData.numTasks),
                batchData.h_numUsedReadIdsPerAnchorPrefixSum.begin(),
                batchData.h_numUsedReadIdsPerAnchor.begin(),
                batchData.d_segmentIds2.begin()
            );

            thrust::inclusive_scan(
                thrust::cuda::par(thrustallocator).on(firstStream),
                batchData.d_segmentIds2.begin(),
                batchData.d_segmentIds2.end(),
                batchData.d_segmentIds2.begin(),
                thrust::maximum<int>()
            );

            // std::cerr
            // << "\n" << batchData.d_candidateReadIds2.data() 
            // << "\n " << batchData.d_numCandidatesPerAnchor2.data()
            // << "\n " << batchData.d_numCandidatesPerAnchorPrefixSum2.data()
            // << "\n " << batchData.d_segmentIds1.data()
            // << "\n " << newNumCandidates
            // << "\n " << batchData.h_usedReadIds.data()
            // << "\n " << batchData.h_numUsedReadIdsPerAnchor.data()
            // << "\n " << batchData.h_numUsedReadIdsPerAnchorPrefixSum.data()
            // << "\n " << batchData.d_segmentIds2.data()
            // << "\n " << totalNumberOfUsedIds
            // << "\n " << batchData.numTasks
            // << "\n " << batchData.h_candidateReadIds3.data()
            // << "\n " << batchData.h_numCandidatesPerAnchor3.data()
            // << "\n " << batchData.h_segmentIds3.data() << "\n";

            auto h_candidateReadIds3_end = GpuSegmentedSetOperation{}.difference(
                thrustallocator,
                batchData.d_candidateReadIds2.data(),
                batchData.d_numCandidatesPerAnchor2.data(),
                batchData.d_numCandidatesPerAnchorPrefixSum2.data(),
                batchData.d_segmentIds1.data(),
                newNumCandidates,
                batchData.h_usedReadIds.data(),
                batchData.h_numUsedReadIdsPerAnchor.data(),
                batchData.h_numUsedReadIdsPerAnchorPrefixSum.data(),
                batchData.d_segmentIds2.data(),
                totalNumberOfUsedIds,
                batchData.numTasks,        
                batchData.h_candidateReadIds3.data(),
                batchData.h_numCandidatesPerAnchor3.data(),
                batchData.d_segmentIds3.data(),
                firstStream
            );

            newNumCandidates = std::distance(batchData.h_candidateReadIds3.data(), h_candidateReadIds3_end);

            //determine task ids with removed mates
            int numTasksWithMateRemovedaaaaa = thrust::distance(
                batchData.h_indexlist2.data(),
                thrust::copy_if(
                    thrust::cuda::par(thrustallocator).on(firstStream),
                    thrust::make_counting_iterator(0),
                    thrust::make_counting_iterator(batchData.numTasks),
                    batchData.d_flagsanchors.data(),
                    batchData.h_indexlist2.data(),
                    thrust::identity<int>()
                )
            );

            hipStreamSynchronize(firstStream); CUERR;

            std::vector<unsigned int> anchormatedataaaa(encodedSequencePitchInInts * numTasksWithMateRemovedaaaaa);

            for(int i = 0; i < numTasksWithMateRemovedaaaaa; i++){
                const int indexForActiveTasks = batchData.h_indexlist2[i];
                const int indexForTasks = indicesOfActiveTasks[indexForActiveTasks];
                const auto& task = vecAccess(tasks, indexForTasks);

                std::copy(
                    task.encodedMate.begin(), 
                    task.encodedMate.end(), 
                    anchormatedataaaa.begin() + i * encodedSequencePitchInInts
                );
            }

            // hipDeviceSynchronize(); CUERR;
            // for(int i = 0; i < batchData.h_numCandidatesPerAnchor[0]; i++){
            //     std::cerr << batchData.h_candidateReadIds[i] << " " << int(batchData.h_flagscandidates[i]) << "\n";
            // }
            // std::cerr << "\n";
            

            batchData.numTasksWithMateRemoved = 0;

            read_number* outputiter = batchData.h_candidateReadIds.data();

            for(int i = 0; i < batchData.numTasks; i++){
                auto& task = vecAccess(tasks, indicesOfActiveTasks[i]);

                int& numCandidates = batchData.h_numCandidatesPerAnchor[i];
                const int candidatesOffset = batchData.h_numCandidatesPerAnchorPrefixSum[i];
                read_number* candidates = batchData.h_candidateReadIds + candidatesOffset;
                read_number* candidatesEnd = candidates + numCandidates;

                auto readIdPos = std::lower_bound(
                    candidates,
                    candidatesEnd,
                    task.myReadId
                );

                if(readIdPos != candidatesEnd && *readIdPos == task.myReadId){
                    std::copy(readIdPos+1, candidatesEnd, readIdPos);
                    --numCandidates;
                    --candidatesEnd;
                }

                if(task.pairedEnd){

                    //remove mate of input from candidate list
                    auto mateReadIdPos = std::lower_bound(
                        candidates,
                        candidates + numCandidates,
                        task.mateReadId
                    );

                    if(mateReadIdPos != candidatesEnd && *mateReadIdPos == task.mateReadId){
                        std::copy(mateReadIdPos+1, candidatesEnd, mateReadIdPos);
                        --numCandidates;
                        --candidatesEnd;

                        std::copy(
                            task.encodedMate.begin(), 
                            task.encodedMate.end(), 
                            batchData.h_anchormatedata.begin() + batchData.numTasksWithMateRemoved * encodedSequencePitchInInts
                        );

                        batchData.h_indexlist1[batchData.numTasksWithMateRemoved] = i;

                        batchData.numTasksWithMateRemoved++;

                        task.mateRemovedFromCandidates = true; //debug. not required
                    }
                }

                /*
                    Remove candidate pairs which have already been used for extension
                */

                std::vector<read_number> tmp(numCandidates);

                auto end = std::set_difference(
                    candidates,
                    candidatesEnd,
                    task.allUsedCandidateReadIdPairs.begin(),
                    task.allUsedCandidateReadIdPairs.end(),
                    tmp.begin()
                );

                numCandidates = std::distance(tmp.begin(), end);

                //std::copy(tmp.begin(), end, candidates);
                outputiter = std::copy(tmp.begin(), end, outputiter);
            }

            hashTimer.stop();

            hipStreamSynchronize(firstStream); CUERR;
            if(numTasksWithMateRemovedaaaaa != batchData.numTasksWithMateRemoved){
                std::cerr << "numTasksWithMateRemovedaaaaa = " << numTasksWithMateRemovedaaaaa << ", batchData.numTasksWithMateRemoved = " << batchData.numTasksWithMateRemoved << "\n";
            }else{
                for(int i = 0; i < batchData.numTasksWithMateRemoved * encodedSequencePitchInInts; i++){
                    const int l = batchData.h_anchormatedata[i];
                    const int r = anchormatedataaaa[i];
    
                    if(l != r){
                        std::cerr << "anchormatedataaaa " << i << " " << l << " " << r << "\n";
                    }
                }
                
            }

            for(int i = 0; i < batchData.numTasks; i++){
                const int l = batchData.h_numCandidatesPerAnchor[i];
                const int r = batchData.h_numCandidatesPerAnchor3[i];

                if(l != r){
                    std::cerr << "numPerAnchor " << i << " " << l << " " << r << "\n";
                }
            }

            for(int i = 0; i < newNumCandidates; i++){
                const read_number l = batchData.h_candidateReadIds[i];
                const read_number r = batchData.h_candidateReadIds3[i];

                if(l != r){
                    std::cerr << "candidateids " << i << " " << l << " " << r << "\n";
                }
            }

            if(batchData.numTasksWithMateRemoved > 0){
                hipMemcpyAsync(
                    batchData.d_anchormatedata.data(),
                    batchData.h_anchormatedata.data(),
                    sizeof(unsigned int) * batchData.numTasksWithMateRemoved * encodedSequencePitchInInts,
                    H2D,
                    firstStream
                ); CUERR;
    
                hipMemcpyAsync(
                    batchData.d_indexlist1.data(),
                    batchData.h_indexlist1.data(),
                    sizeof(int) * batchData.numTasksWithMateRemoved,
                    H2D,
                    firstStream
                ); CUERR;
            }

            //compact candidate ids and update offsets accordingly

            // {
            //     read_number* outputposition = batchData.h_candidateReadIds.data();

                 for(int i = 0; i < batchData.numTasks; i++){
    
                     const int numCandidates = batchData.h_numCandidatesPerAnchor[i];
            //         const int candidatesOffset = batchData.h_numCandidatesPerAnchorPrefixSum[i];
            //         const read_number* candidates = batchData.h_candidateReadIds + candidatesOffset;
                    
            //         outputposition = std::copy_n(candidates, numCandidates, outputposition);
                     batchData.h_numCandidatesPerAnchorPrefixSum[i+1] = batchData.h_numCandidatesPerAnchorPrefixSum[i] + numCandidates;                 
                 }
            // }

            totalNumCandidates = batchData.h_numCandidatesPerAnchorPrefixSum[batchData.numTasks];

            hipMemcpyAsync(
                batchData.d_numCandidatesPerAnchor.data(),
                batchData.h_numCandidatesPerAnchor.data(),
                sizeof(int) * batchData.numTasks,
                H2D,
                firstStream
            ); CUERR;

            hipMemcpyAsync(
                batchData.d_numCandidatesPerAnchorPrefixSum.data(),
                batchData.h_numCandidatesPerAnchorPrefixSum.data(),
                sizeof(int) * (batchData.numTasks + 1),
                H2D,
                firstStream
            ); CUERR;

            hipMemcpyAsync(
                batchData.d_candidateReadIds.data(),
                batchData.h_candidateReadIds.data(),
                sizeof(read_number) * totalNumCandidates,
                H2D,
                firstStream
            ); CUERR;

   
            //allocate data for candidate sequences 
            batchData.h_candidateSequencesLength.resize(totalNumCandidates);
            batchData.h_candidateSequencesData.resize(encodedSequencePitchInInts * totalNumCandidates);
            batchData.h_candidateSequencesRevcData.resize(encodedSequencePitchInInts * totalNumCandidates);

            batchData.d_candidateSequencesLength.resize(totalNumCandidates);
            batchData.d_candidateSequencesData.resize(encodedSequencePitchInInts * totalNumCandidates);
            batchData.d_candidateSequencesRevcData.resize(encodedSequencePitchInInts * totalNumCandidates);

            batchData.d_candidateSequencesLength2.resize(totalNumCandidates);
            batchData.d_candidateSequencesData2.resize(encodedSequencePitchInInts * totalNumCandidates);
            batchData.d_candidateSequencesRevcData2.resize(encodedSequencePitchInInts * totalNumCandidates);
            batchData.d_candidateReadIds2.resize(totalNumCandidates);

            batchData.h_intbuffercandidates.resize(totalNumCandidates);
            batchData.d_intbuffercandidates.resize(totalNumCandidates);
            batchData.h_flagscandidates.resize(totalNumCandidates);
            batchData.d_flagscandidates.resize(totalNumCandidates);

            batchData.h_alignment_overlaps.resize(totalNumCandidates);
            batchData.h_alignment_shifts.resize(totalNumCandidates);
            batchData.h_alignment_nOps.resize(totalNumCandidates);
            batchData.h_alignment_isValid.resize(totalNumCandidates);
            batchData.h_alignment_best_alignment_flags.resize(totalNumCandidates);

            batchData.d_alignment_overlaps.resize(totalNumCandidates);
            batchData.d_alignment_shifts.resize(totalNumCandidates);
            batchData.d_alignment_nOps.resize(totalNumCandidates);
            batchData.d_alignment_isValid.resize(totalNumCandidates);
            batchData.d_alignment_best_alignment_flags.resize(totalNumCandidates);

            batchData.d_alignment_overlaps2.resize(totalNumCandidates);
            batchData.d_alignment_shifts2.resize(totalNumCandidates);
            batchData.d_alignment_nOps2.resize(totalNumCandidates);
            batchData.d_alignment_isValid2.resize(totalNumCandidates);
            batchData.d_alignment_best_alignment_flags2.resize(totalNumCandidates);

            collectTimer.start();

            loadCandidateSequenceData(batchData, firstStream);      

            eraseDataOfRemovedMates(batchData, firstStream);

            hipMemcpyAsync(
                batchData.h_numCandidatesPerAnchorPrefixSum.get(),
                batchData.d_numCandidatesPerAnchorPrefixSum.get(),
                sizeof(int) * (batchData.numTasks+1),
                D2H,
                firstStream
            ); CUERR;

            hipMemcpyAsync(
                batchData.h_numCandidatesPerAnchor.get(),
                batchData.d_numCandidatesPerAnchor.get(),
                sizeof(int) * batchData.numTasks,
                D2H,
                firstStream
            ); CUERR;
    
            hipStreamSynchronize(firstStream); CUERR;

            totalNumCandidates = batchData.h_numCandidatesPerAnchorPrefixSum[batchData.numTasks];

            // hipMemcpyAsync(
            //     batchData.h_candidateReadIds.data(),
            //     batchData.d_candidateReadIds.data(),
            //     sizeof(read_number) * totalNumCandidates,
            //     H2D,
            //     secondStream
            // ); CUERR;

            // hipMemcpyAsync(
            //     batchData.h_candidateSequencesLength.get(),
            //     batchData.d_candidateSequencesLength.get(),
            //     sizeof(int) * totalNumCandidates,
            //     H2D,
            //     secondStream
            // ); CUERR;
    
            hipMemcpyAsync(
                batchData.h_candidateSequencesData.get(),
                batchData.d_candidateSequencesData.get(),
                sizeof(unsigned int) * totalNumCandidates * encodedSequencePitchInInts,
                H2D,
                secondStream
            ); CUERR;
    
            hipMemcpyAsync(
                batchData.h_candidateSequencesRevcData.get(),
                batchData.d_candidateSequencesRevcData.get(),
                sizeof(unsigned int) * totalNumCandidates * encodedSequencePitchInInts,
                H2D,
                secondStream
            ); CUERR;    
            


            collectTimer.stop();

            /*
                Compute alignments
            */

            alignmentTimer.start();

            calculateAlignments(batchData, firstStream);

            hipStreamSynchronize(firstStream);

            alignmentTimer.stop();


            // hipMemcpyAsync(
            //     batchData.h_alignment_overlaps.get(),
            //     batchData.d_alignment_overlaps.get(),
            //     sizeof(int) * totalNumCandidates,
            //     D2H,
            //     firstStream
            // ); CUERR;

            // hipMemcpyAsync(
            //     batchData.h_alignment_isValid.get(),
            //     batchData.d_alignment_isValid.get(),
            //     sizeof(bool) * totalNumCandidates,
            //     D2H,
            //     firstStream
            // ); CUERR;

            // hipMemcpyAsync(
            //     batchData.h_alignment_shifts.get(),
            //     batchData.d_alignment_shifts.get(),
            //     sizeof(int) * totalNumCandidates,
            //     D2H,
            //     firstStream
            // ); CUERR;

            // hipMemcpyAsync(
            //     batchData.h_alignment_nOps.get(),
            //     batchData.d_alignment_nOps.get(),
            //     sizeof(int) * totalNumCandidates,
            //     D2H,
            //     firstStream
            // ); CUERR;

            // hipMemcpyAsync(
            //     batchData.h_alignment_best_alignment_flags.get(),
            //     batchData.d_alignment_best_alignment_flags.get(),
            //     sizeof(BestAlignment_t) * totalNumCandidates,
            //     D2H,
            //     firstStream
            // ); CUERR;

            // hipStreamSynchronize(firstStream); CUERR;
            hipStreamSynchronize(secondStream); CUERR;

            // unpack batchData into tasks


            for(int i = 0; i < numActiveTasks; i++){
                auto& task = vecAccess(tasks, indicesOfActiveTasks[i]);

                const int numCandidates = batchData.h_numCandidatesPerAnchor[i];
                const int offset = batchData.h_numCandidatesPerAnchorPrefixSum[i];

                // task.candidateReadIds.resize(numCandidates);
                // std::copy_n(batchData.h_candidateReadIds.data() + offset, numCandidates, task.candidateReadIds.begin());

                // task.candidateSequenceLengths.resize(numCandidates);
                // std::copy_n(batchData.h_candidateSequencesLength.data() + offset, numCandidates, task.candidateSequenceLengths.begin());

                task.candidateSequencesFwdData.resize(numCandidates * encodedSequencePitchInInts);
                std::copy_n(batchData.h_candidateSequencesData.data() + offset * encodedSequencePitchInInts, numCandidates * encodedSequencePitchInInts, task.candidateSequencesFwdData.begin());

                task.candidateSequencesRevcData.resize(numCandidates * encodedSequencePitchInInts);
                std::copy_n(batchData.h_candidateSequencesRevcData.data() + offset * encodedSequencePitchInInts, numCandidates * encodedSequencePitchInInts, task.candidateSequencesRevcData.begin());

                // task.alignmentFlags.resize(numCandidates);
                // task.alignments.resize(numCandidates);

                // for(int c = 0; c < numCandidates; c++){
                //     task.alignments[c].shift = batchData.h_alignment_shifts[offset + c];
                //     task.alignments[c].overlap = batchData.h_alignment_overlaps[offset + c];
                //     task.alignments[c].nOps = batchData.h_alignment_nOps[offset + c];
                //     task.alignments[c].isValid = batchData.h_alignment_isValid[offset + c];
                //     task.alignmentFlags[c] = batchData.h_alignment_best_alignment_flags[offset + c];
                // }

                task.mateRemovedFromCandidates = false; //debug. not required
            }


            // for(int i = 0; i < numActiveTasks; i++){
            //     auto& newtask = tasks[indicesOfActiveTasks[i]];
            //     auto& oldtask = debugtasks[indicesOfActiveTasks[i]];

            //     if(newtask != oldtask){
            //         std::cerr << "old task and new task differ. i=" 
            //             << i << ", indicesOfActiveTasks[i] " << indicesOfActiveTasks[i] << "\n";
            //     }
            // }

            

#if 0
            alignmentFilterTimer.start();

            for(int indexOfActiveTask : indicesOfActiveTasks){
                auto& task = vecAccess(tasks, indexOfActiveTask);

                /*
                    Remove bad alignments
                */        

                const int size = task.alignments.size();

                std::vector<int> positionsOfCandidatesToKeep(size);
                std::vector<int> tmpPositionsOfCandidatesToKeep(size);

                task.numRemainingCandidates = 0;

                //select candidates with good alignment and positive shift
                for(int c = 0; c < size; c++){
                    const BestAlignment_t alignmentFlag0 = vecAccess(task.alignmentFlags, c);
                    
                    if(alignmentFlag0 != BestAlignment_t::None && vecAccess(task.alignments, c).shift >= 0){
                        vecAccess(positionsOfCandidatesToKeep, task.numRemainingCandidates) = c;
                        task.numRemainingCandidates++;
                    }else{
                        ; // remove alignment
                    }
                }

                positionsOfCandidatesToKeep.erase(
                    positionsOfCandidatesToKeep.begin() + task.numRemainingCandidates, 
                    positionsOfCandidatesToKeep.end()
                );

                if(task.numRemainingCandidates == 0){
                    task.abort = true;
                    task.abortReason = AbortReason::NoPairedCandidatesAfterAlignment;

                    continue; //stop processing task
                }

                float relativeOverlapThreshold = 0.9f;
                bool goodAlignmentExists = false;

                #if 0

                while(!goodAlignmentExists && fgeq(relativeOverlapThreshold, goodAlignmentProperties.min_overlap_ratio)){                    

                    goodAlignmentExists = std::any_of(
                        positionsOfCandidatesToKeep.begin(), 
                        positionsOfCandidatesToKeep.end(),
                        [&](const auto& position){
                            const auto& alignment = vecAccess(task.alignments, position);
                            const float relativeOverlap = float(alignment.overlap) / float(task.currentAnchorLength);
                            // if(fgeq(relativeOverlap, relativeOverlapThreshold) && relativeOverlap < 1.0f){
                            //     std::cerr << position << " " << relativeOverlap << " " << relativeOverlapThreshold << "\n";
                            // }
                            return fgeq(relativeOverlap, relativeOverlapThreshold) && relativeOverlap < 1.0f;
                        }
                    );

                    if(!goodAlignmentExists){
                        relativeOverlapThreshold -= 0.1f;
                    }
                }

                #else 
                {

                    bool hasmax = false;
                    float maxRel = 0.0f;

                    for(auto p : positionsOfCandidatesToKeep){
                        const auto& alignment = vecAccess(task.alignments, p);
                        const float relativeOverlap = float(alignment.overlap) / float(task.currentAnchorLength);
                        if(relativeOverlap < 1.0f && fgeq(relativeOverlap, goodAlignmentProperties.min_overlap_ratio)){
                            hasmax = true;
                            const float tmp = std::floor(relativeOverlap * 10.0f) / 10.0f;
                            maxRel = std::max(maxRel, tmp);
                        }
                    }
                    // assert(hasmax == goodAlignmentExists);
                    // if(hasmax){
                    //     assert(feq(maxRel, relativeOverlapThreshold));
                    // }

                    relativeOverlapThreshold = maxRel;
                    goodAlignmentExists = hasmax;

                }

                #endif

                


                

                if(goodAlignmentExists){
                    positionsOfCandidatesToKeep.erase(
                        std::remove_if(
                            positionsOfCandidatesToKeep.begin(), 
                            positionsOfCandidatesToKeep.end(),
                            [&](const auto& position){
                                const auto& alignment = vecAccess(task.alignments, position);
                                const float relativeOverlap = float(alignment.overlap) / float(task.currentAnchorLength);
                                return !fgeq(relativeOverlap, relativeOverlapThreshold);
                            }
                        ),
                        positionsOfCandidatesToKeep.end()
                    );
                    task.numRemainingCandidates = positionsOfCandidatesToKeep.size();
                }

                //std::cerr << ", remaining candidates " << task.numRemainingCandidates << "\n";


                //compact selected candidates inplace

                

                {
                    task.candidateSequenceData.resize(task.numRemainingCandidates * encodedSequencePitchInInts);

                    for(int c = 0; c < task.numRemainingCandidates; c++){
                        const int index = vecAccess(positionsOfCandidatesToKeep, c);

                        vecAccess(task.alignments, c) = vecAccess(task.alignments, index);
                        vecAccess(task.alignmentFlags, c) = vecAccess(task.alignmentFlags, index);
                        vecAccess(task.candidateReadIds, c) = vecAccess(task.candidateReadIds, index);
                        vecAccess(task.candidateSequenceLengths, c) = vecAccess(task.candidateSequenceLengths, index);
                        
                        assert(vecAccess(task.alignmentFlags, index) != BestAlignment_t::None);

                        if(vecAccess(task.alignmentFlags, index) == BestAlignment_t::Forward){
                            std::copy_n(
                                task.candidateSequencesFwdData.data() + index * encodedSequencePitchInInts,
                                encodedSequencePitchInInts,
                                task.candidateSequenceData.data() + c * encodedSequencePitchInInts
                            );
                        }else{
                            //BestAlignment_t::ReverseComplement

                            std::copy_n(
                                task.candidateSequencesRevcData.data() + index * encodedSequencePitchInInts,
                                encodedSequencePitchInInts,
                                task.candidateSequenceData.data() + c * encodedSequencePitchInInts
                            );
                        }

                    }

                    //erase past-end elements
                    task.alignments.erase(
                        task.alignments.begin() + task.numRemainingCandidates, 
                        task.alignments.end()
                    );
                    task.alignmentFlags.erase(
                        task.alignmentFlags.begin() + task.numRemainingCandidates, 
                        task.alignmentFlags.end()
                    );
                    task.candidateReadIds.erase(
                        task.candidateReadIds.begin() + task.numRemainingCandidates, 
                        task.candidateReadIds.end()
                    );
                    task.candidateSequenceLengths.erase(
                        task.candidateSequenceLengths.begin() + task.numRemainingCandidates, 
                        task.candidateSequenceLengths.end()
                    );                    
                }

            }

            alignmentFilterTimer.stop();
#else       
            filterAlignments(batchData, firstStream);

            hipMemcpyAsync(
                batchData.h_numCandidatesPerAnchorPrefixSum.get(),
                batchData.d_numCandidatesPerAnchorPrefixSum.get(),
                sizeof(int) * (batchData.numTasks+1),
                D2H,
                firstStream
            ); CUERR;

            hipMemcpyAsync(
                batchData.h_numCandidatesPerAnchor.get(),
                batchData.d_numCandidatesPerAnchor.get(),
                sizeof(int) * batchData.numTasks,
                D2H,
                firstStream
            ); CUERR;
    
            hipStreamSynchronize(firstStream); CUERR;

            //auto old = totalNumCandidates;

            totalNumCandidates = batchData.h_numCandidatesPerAnchorPrefixSum[batchData.numTasks];

            //std::cerr << old << " -> " << totalNumCandidates << "\n";

            hipMemcpyAsync(
                batchData.h_candidateReadIds.data(),
                batchData.d_candidateReadIds.data(),
                sizeof(read_number) * totalNumCandidates,
                H2D,
                firstStream
            ); CUERR;

            hipMemcpyAsync(
                batchData.h_candidateSequencesLength.get(),
                batchData.d_candidateSequencesLength.get(),
                sizeof(int) * totalNumCandidates,
                H2D,
                firstStream
            ); CUERR;
    
            hipMemcpyAsync(
                batchData.h_candidateSequencesData.get(),
                batchData.d_candidateSequencesData.get(),
                sizeof(unsigned int) * totalNumCandidates * encodedSequencePitchInInts,
                H2D,
                firstStream
            ); CUERR;

            hipMemcpyAsync(
                batchData.h_alignment_overlaps.get(),
                batchData.d_alignment_overlaps.get(),
                sizeof(int) * totalNumCandidates,
                D2H,
                firstStream
            ); CUERR;

            hipMemcpyAsync(
                batchData.h_alignment_isValid.get(),
                batchData.d_alignment_isValid.get(),
                sizeof(bool) * totalNumCandidates,
                D2H,
                firstStream
            ); CUERR;

            hipMemcpyAsync(
                batchData.h_alignment_shifts.get(),
                batchData.d_alignment_shifts.get(),
                sizeof(int) * totalNumCandidates,
                D2H,
                firstStream
            ); CUERR;

            hipMemcpyAsync(
                batchData.h_alignment_nOps.get(),
                batchData.d_alignment_nOps.get(),
                sizeof(int) * totalNumCandidates,
                D2H,
                firstStream
            ); CUERR;

            hipMemcpyAsync(
                batchData.h_alignment_best_alignment_flags.get(),
                batchData.d_alignment_best_alignment_flags.get(),
                sizeof(BestAlignment_t) * totalNumCandidates,
                D2H,
                firstStream
            ); CUERR;

            hipStreamSynchronize(firstStream); CUERR;

            // unpack batchData into tasks


            for(int i = 0; i < numActiveTasks; i++){
                auto& task = vecAccess(tasks, indicesOfActiveTasks[i]);

                const int numCandidates = batchData.h_numCandidatesPerAnchor[i];
                const int offset = batchData.h_numCandidatesPerAnchorPrefixSum[i];

                task.candidateReadIds.resize(numCandidates);
                std::copy_n(batchData.h_candidateReadIds.data() + offset, numCandidates, task.candidateReadIds.begin());

                task.candidateSequenceLengths.resize(numCandidates);
                std::copy_n(batchData.h_candidateSequencesLength.data() + offset, numCandidates, task.candidateSequenceLengths.begin());

                task.candidateSequenceData.resize(numCandidates * encodedSequencePitchInInts);
                std::copy_n(
                    batchData.h_candidateSequencesData.data() + offset * encodedSequencePitchInInts, 
                    numCandidates * encodedSequencePitchInInts, 
                    task.candidateSequenceData.begin()
                );

                task.alignmentFlags.resize(numCandidates);
                task.alignments.resize(numCandidates);

                for(int c = 0; c < numCandidates; c++){
                    task.alignments[c].shift = batchData.h_alignment_shifts[offset + c];
                    task.alignments[c].overlap = batchData.h_alignment_overlaps[offset + c];
                    task.alignments[c].nOps = batchData.h_alignment_nOps[offset + c];
                    task.alignments[c].isValid = batchData.h_alignment_isValid[offset + c];
                    task.alignmentFlags[c] = batchData.h_alignment_best_alignment_flags[offset + c];
                }

                task.mateRemovedFromCandidates = false; //debug. not required
                task.numRemainingCandidates = numCandidates;

                if(task.numRemainingCandidates == 0){
                    task.abort = true;
                    task.abortReason = AbortReason::NoPairedCandidatesAfterAlignment;
                }
            }

            // for(int i = 0; i < numActiveTasks; i++){
            //     auto& newtask = tasks[indicesOfActiveTasks[i]];
            //     auto& oldtask = debugtasks[indicesOfActiveTasks[i]];

            //     if(newtask != oldtask){
            //         std::cerr << "old task and new task differ. i=" 
            //             << i << ", indicesOfActiveTasks[i] " << indicesOfActiveTasks[i] << "\n";
            //     }
            // }
    
#endif        
            std::vector<Task> newTasksFromSplit;
            std::vector<int> newTaskIndices;


            auto constructMsa = [&](auto& task, int taskIndex){
                const std::string& decodedAnchor = task.totalDecodedAnchors.back();

                auto calculateOverlapWeight = [](int anchorlength, int nOps, int overlapsize){
                    constexpr float maxErrorPercentInOverlap = 0.2f;

                    return 1.0f - sqrtf(nOps / (overlapsize * maxErrorPercentInOverlap));
                };

                MultipleSequenceAlignment msa;

                auto build = [&](){

                    task.candidateShifts.resize(task.numRemainingCandidates);
                    task.candidateOverlapWeights.resize(task.numRemainingCandidates);

                    //gather data required for msa
                    for(int c = 0; c < task.numRemainingCandidates; c++){
                        vecAccess(task.candidateShifts, c) = vecAccess(task.alignments, c).shift;

                        vecAccess(task.candidateOverlapWeights, c) = calculateOverlapWeight(
                            task.currentAnchorLength, 
                            vecAccess(task.alignments, c).nOps,
                            vecAccess(task.alignments, c).overlap
                        );
                    }

                    task.candidateStrings.resize(decodedSequencePitchInBytes * task.numRemainingCandidates, '\0');

                    //decode the candidates for msa
                    for(int c = 0; c < task.numRemainingCandidates; c++){
                        SequenceHelpers::decode2BitSequence(
                            task.candidateStrings.data() + c * decodedSequencePitchInBytes,
                            task.candidateSequenceData.data() + c * encodedSequencePitchInInts,
                            vecAccess(task.candidateSequenceLengths, c)
                        );
                    }

                    MultipleSequenceAlignment::InputData msaInput;
                    msaInput.useQualityScores = false;
                    msaInput.subjectLength = task.currentAnchorLength;
                    msaInput.nCandidates = task.numRemainingCandidates;
                    msaInput.candidatesPitch = decodedSequencePitchInBytes;
                    msaInput.candidateQualitiesPitch = 0;
                    msaInput.subject = decodedAnchor.c_str();
                    msaInput.candidates = task.candidateStrings.data();
                    msaInput.subjectQualities = nullptr;
                    msaInput.candidateQualities = nullptr;
                    msaInput.candidateLengths = task.candidateSequenceLengths.data();
                    msaInput.candidateShifts = task.candidateShifts.data();
                    msaInput.candidateDefaultWeightFactors = task.candidateOverlapWeights.data();                    

                    msa.build(msaInput);
                };

                build();

                #if 1

                constexpr int max_num_minimizations = 5;

                auto removeCandidatesOfDifferentRegion = [&](const auto& minimizationResult){
                    const int numCandidates = task.candidateReadIds.size();

                    int insertpos = 0;
                    for(int i = 0; i < numCandidates; i++){
                        if(!minimizationResult.differentRegionCandidate[i]){               
                            //keep candidate

                            task.candidateReadIds[insertpos] = task.candidateReadIds[i];

                            std::copy_n(
                                task.candidateSequenceData.data() + i * size_t(encodedSequencePitchInInts),
                                encodedSequencePitchInInts,
                                task.candidateSequenceData.data() + insertpos * size_t(encodedSequencePitchInInts)
                            );

                            task.candidateSequenceLengths[insertpos] = task.candidateSequenceLengths[i];
                            task.alignmentFlags[insertpos] = task.alignmentFlags[i];
                            task.alignments[insertpos] = task.alignments[i];
                            task.candidateOverlapWeights[insertpos] = task.candidateOverlapWeights[i];
                            task.candidateShifts[insertpos] = task.candidateShifts[i];

                            std::copy_n(
                                task.candidateStrings.data() + i * size_t(decodedSequencePitchInBytes),
                                decodedSequencePitchInBytes,
                                task.candidateStrings.data() + insertpos * size_t(decodedSequencePitchInBytes)
                            );

                            insertpos++;
                        }
                    }

                    task.numRemainingCandidates = insertpos;

                    task.candidateReadIds.erase(
                        task.candidateReadIds.begin() + insertpos, 
                        task.candidateReadIds.end()
                    );
                    task.candidateSequenceData.erase(
                        task.candidateSequenceData.begin() + encodedSequencePitchInInts * insertpos, 
                        task.candidateSequenceData.end()
                    );
                    task.candidateSequenceLengths.erase(
                        task.candidateSequenceLengths.begin() + insertpos, 
                        task.candidateSequenceLengths.end()
                    );
                    task.alignmentFlags.erase(
                        task.alignmentFlags.begin() + insertpos, 
                        task.alignmentFlags.end()
                    );
                    task.alignments.erase(
                        task.alignments.begin() + insertpos, 
                        task.alignments.end()
                    );

                    task.candidateStrings.erase(
                        task.candidateStrings.begin() + decodedSequencePitchInBytes * insertpos, 
                        task.candidateStrings.end()
                    );
                    task.candidateOverlapWeights.erase(
                        task.candidateOverlapWeights.begin() + insertpos, 
                        task.candidateOverlapWeights.end()
                    );
                    task.candidateShifts.erase(
                        task.candidateShifts.begin() + insertpos, 
                        task.candidateShifts.end()
                    );
                    
                };

                if(max_num_minimizations > 0){                

                    for(int numIterations = 0; numIterations < max_num_minimizations; numIterations++){
                        const auto minimizationResult = msa.findCandidatesOfDifferentRegion(
                            correctionOptions.estimatedCoverage
                        );

                        if(minimizationResult.performedMinimization){
                            removeCandidatesOfDifferentRegion(minimizationResult);

                            //build minimized multiple sequence alignment
                            build();
                        }else{
                            break;
                        }               
                        
                    }
                }   

                #endif

                return msa;
            };

            auto extendWithMsa = [&](auto& task, const auto& msa, int taskIndex){

                int consensusLength = msa.consensus.size();
                //can extend by at most maxextensionPerStep bps
                int extendBy = std::min(
                    consensusLength - task.currentAnchorLength, 
                    maxextensionPerStep
                );
                //cannot extend over fragment 
                extendBy = std::min(extendBy, (insertSize + insertSizeStddev - task.mateLength) - task.accumExtensionLengths);

                auto makeAnchorForNextIteration = [&](){
                    if(extendBy == 0){
                        task.abort = true;
                        task.abortReason = AbortReason::MsaNotExtended;
                    }else{
                        task.accumExtensionLengths += extendBy;

                        //update data for next iteration of outer while loop                           

                        std::string decodedAnchor(msa.consensus.data() + extendBy, task.currentAnchorLength);

                        const int numInts = SequenceHelpers::getEncodedNumInts2Bit(task.currentAnchorLength);

                        task.currentAnchor.resize(numInts);

                        SequenceHelpers::encodeSequence2Bit(
                            task.currentAnchor.data(), 
                            decodedAnchor.data(), 
                            task.currentAnchorLength
                        );

                        task.totalDecodedAnchors.emplace_back(std::move(decodedAnchor));
                        task.totalAnchorBeginInExtendedRead.emplace_back(task.accumExtensionLengths);

                        // task.resultsequence.insert(
                        //     task.resultsequence.end(), 
                        //     msa.consensus.data() + task.currentAnchorLength, 
                        //     msa.consensus.data() + task.currentAnchorLength + extendBy
                        // );


                        // std::string tmp(task.currentAnchorLength, '\0');

                        // decode2BitSequence(
                        //     &tmp[0],
                        //     task.currentAnchor.data(),
                        //     task.currentAnchorLength
                        // );

                        // auto sub = task.resultsequence.substr(task.resultsequence.length() - task.currentAnchorLength);

                        // assert(sub == tmp);
                    }
                };

                constexpr int requiredOverlapMate = 70; //TODO relative overlap 
                constexpr int numMismatchesUpperBound = 2;

                if(task.pairedEnd && task.accumExtensionLengths + consensusLength - requiredOverlapMate + task.mateLength >= insertSize - insertSizeStddev){
                    //check if mate can be overlapped with consensus 

                    //hamMap[i] stores possible starting positions of overlaps which would have hamming distance i
                    std::map<int, std::vector<int>> hamMap;

                    //longmatchMap[i] stores possible starting positions of overlaps which would have a longest match of length i between mate and msa consensus
                    //std::map<int, std::vector<int>> longmatchMap; //map length of longest match to list start positions

                    //for each possibility to overlap the mate and consensus such that the merged sequence would end in the desired range [insertSize - insertSizeStddev, insertSize + insertSizeStddev]

                    const int firstStartpos = std::max(0, insertSize - insertSizeStddev - task.accumExtensionLengths - task.mateLength);
                    const int lastStartposExcl = std::min(
                        std::max(0, insertSize + insertSizeStddev - task.accumExtensionLengths - task.mateLength) + 1,
                        consensusLength - requiredOverlapMate
                    );

                    for(int startpos = firstStartpos; startpos < lastStartposExcl; startpos++){
                        //compute metrics of overlap
                            
                        const int ham = cpu::hammingDistanceOverlap(
                            msa.consensus.begin() + startpos, msa.consensus.end(), 
                            task.decodedMateRevC.begin(), task.decodedMateRevC.end()
                        );

                        hamMap[ham].emplace_back(startpos);

                        // const int longest = cpu::longestMatch(
                        //     msa.consensus.begin() + startpos, msa.consensus.end(), 
                        //     task.decodedMateRevC.begin(), task.decodedMateRevC.end()
                        // );

                        // longmatchMap[longest].emplace_back(startpos);
                    }
                    
                    std::vector<std::pair<int, std::vector<int>>> flatMap(hamMap.begin(), hamMap.end());
                    //sort by hamming distance, ascending
                    std::sort(flatMap.begin(), flatMap.end(), [](const auto& p1, const auto& p2){return p1.first < p2.first;});

                    //std::vector<std::pair<int, std::vector<int>>> flatMap2(longmatchMap.begin(), longmatchMap.end());
                    //sort by length of longest match, descending
                    //std::sort(flatMap2.begin(), flatMap2.end(), [](const auto& p1, const auto& p2){return p2.first < p1.first;});

                    //if there exists an overlap between msa consensus and mate which would end the merge, use the best one
                    if(flatMap.size() > 0 && flatMap[0].first <= numMismatchesUpperBound){
                    //if(flatMap2.size() > 0 && flatMap2[0].first >= 40){
                        const int mateStartposInConsensus = flatMap[0].second.front();
                        const int missingPositionsBetweenAnchorEndAndMateBegin = std::max(0, mateStartposInConsensus - task.currentAnchorLength);

                        if(missingPositionsBetweenAnchorEndAndMateBegin > 0){
                            //bridge the gap between current anchor and mate
                            task.totalDecodedAnchors.emplace_back(
                                msa.consensus.data() + missingPositionsBetweenAnchorEndAndMateBegin,
                                msa.consensus.data() + missingPositionsBetweenAnchorEndAndMateBegin + mateStartposInConsensus
                            );
                            task.totalAnchorBeginInExtendedRead.emplace_back(task.accumExtensionLengths + missingPositionsBetweenAnchorEndAndMateBegin);
                        }


                        task.mateHasBeenFound = true;

                        //const int currentAccumExtensionLengths = task.accumExtensionLengths;
                        
                        task.accumExtensionLengths += mateStartposInConsensus;
                        std::string decodedAnchor(task.decodedMateRevC);

                        task.totalDecodedAnchors.emplace_back(std::move(decodedAnchor));
                        task.totalAnchorBeginInExtendedRead.emplace_back(task.accumExtensionLengths);

                        // const int startpos = mateStartposInConsensus;
                        // task.resultsequence.resize(currentAccumExtensionLengths + startpos + task.decodedMateRevC.length());
                        // const auto replaceBegin = task.resultsequence.begin() + currentAccumExtensionLengths + startpos;
                        // task.resultsequence.replace(
                        //     replaceBegin, 
                        //     replaceBegin + task.decodedMateRevC.length(), 
                        //     task.decodedMateRevC.begin(), 
                        //     task.decodedMateRevC.end()
                        // );

                    }else{
                        makeAnchorForNextIteration();
                    }
                }else{
                    makeAnchorForNextIteration();
                }
            };

            auto keepSelectedCandidates = [&](auto& task, const auto& selectedCandidateIndices, int taskIndex){
                const int numCandidateIndices = selectedCandidateIndices.size();
                assert(numCandidateIndices <= task.numRemainingCandidates);

                for(int i = 0; i < numCandidateIndices; i++){
                    const int c = vecAccess(selectedCandidateIndices, i);
                    // if(!(0 <= c && c < task.candidateReadIds.size())){
                    //     std::cerr << "c = " << c << ", candidateReadIds.size() = " << task.candidateReadIds.size() << "\n";
                    // }

                    // assert(0 <= c && c < task.candidateReadIds.size());
                    // assert(0 <= c && c < task.candidateSequenceLengths.size());
                    // assert(0 <= c && c < task.alignments.size());
                    // assert(0 <= c && c < task.alignmentFlags.size());

                    // assert(0 <= c && c*encodedSequencePitchInInts < task.candidateSequencesFwdData.size());
                    // assert(0 <= c && c*encodedSequencePitchInInts < task.candidateSequencesRevcData.size());
                    // assert(0 <= c && c*encodedSequencePitchInInts < task.candidateSequenceData.size());

                    vecAccess(task.candidateReadIds, i) = vecAccess(task.candidateReadIds, c);
                    vecAccess(task.candidateSequenceLengths , i) = vecAccess(task.candidateSequenceLengths, c);
                    vecAccess(task.alignments, i) = vecAccess(task.alignments, c);
                    vecAccess(task.alignmentFlags, i) = vecAccess(task.alignmentFlags, c);
                    vecAccess(task.candidateShifts, i) = vecAccess(task.candidateShifts, c);
                    vecAccess(task.candidateOverlapWeights, i) = vecAccess(task.candidateOverlapWeights, c);

                    std::copy_n(
                        task.candidateSequencesFwdData.begin() + c * encodedSequencePitchInInts,
                        encodedSequencePitchInInts,
                        task.candidateSequencesFwdData.begin() + i * encodedSequencePitchInInts
                    );

                    std::copy_n(
                        task.candidateSequencesRevcData.begin() + c * encodedSequencePitchInInts,
                        encodedSequencePitchInInts,
                        task.candidateSequencesRevcData.begin() + i * encodedSequencePitchInInts
                    );

                    std::copy_n(
                        task.candidateSequenceData.begin() + c * encodedSequencePitchInInts,
                        encodedSequencePitchInInts,
                        task.candidateSequenceData.begin() + i * encodedSequencePitchInInts
                    );

                    std::copy_n(
                        task.candidateStrings.begin() + c * decodedSequencePitchInBytes,
                        decodedSequencePitchInBytes,
                        task.candidateStrings.begin() + i * decodedSequencePitchInBytes
                    );
                }

                task.candidateReadIds.erase(
                    task.candidateReadIds.begin() + numCandidateIndices,
                    task.candidateReadIds.end()
                );
                task.candidateSequenceLengths.erase(
                    task.candidateSequenceLengths.begin() + numCandidateIndices,
                    task.candidateSequenceLengths.end()
                );
                task.alignments.erase(
                    task.alignments.begin() + numCandidateIndices,
                    task.alignments.end()
                );
                task.alignmentFlags.erase(
                    task.alignmentFlags.begin() + numCandidateIndices,
                    task.alignmentFlags.end()
                );
                task.candidateSequencesFwdData.erase(
                    task.candidateSequencesFwdData.begin() + numCandidateIndices * encodedSequencePitchInInts,
                    task.candidateSequencesFwdData.end()
                );
                task.candidateSequencesRevcData.erase(
                    task.candidateSequencesRevcData.begin() + numCandidateIndices * encodedSequencePitchInInts,
                    task.candidateSequencesRevcData.end()
                );
                task.candidateSequenceData.erase(
                    task.candidateSequenceData.begin() + numCandidateIndices * encodedSequencePitchInInts,
                    task.candidateSequenceData.end()
                );
                if(task.pairedEnd){
                    task.mateIdLocationIter = std::lower_bound(
                        task.candidateReadIds.begin(),
                        task.candidateReadIds.end(),
                        task.mateReadId
                    );

                    task.mateHasBeenFound = (task.mateIdLocationIter != task.candidateReadIds.end() 
                        && *task.mateIdLocationIter == task.mateReadId);
                }
                task.numRemainingCandidates = numCandidateIndices;
            };

            nvtx::push_range("MSA", 6);
            msaTimer.start();

            for(int indexOfActiveTask : indicesOfActiveTasks){
                auto& task = vecAccess(tasks, indexOfActiveTask);

                const MultipleSequenceAlignment msa = constructMsa(task, indexOfActiveTask);

                // std::cerr << "original msa\n";
                // msa.print(std::cerr);
                // std::cerr << "\n";

                
#if 1
                //if(task.splitDepth == 0){
                if(splitTracker[task.myReadId] <= 4){
                    auto possibleSplits = msa.inspectColumnsRegionSplit(task.currentAnchorLength);

                    if(possibleSplits.splits.size() > 1){
                        //auto& task = tasks[indexOfActiveTask];
                        
                        std::sort(
                            possibleSplits.splits.begin(), 
                            possibleSplits.splits.end(),
                            [](const auto& split1, const auto& split2){
                                //sort by size, descending
                                return split2.listOfCandidates.size() < split1.listOfCandidates.size();
                            }
                        );

                        // std::cerr << "split[0] = ";
                        // for(auto x : possibleSplits.splits[0].listOfCandidates) std::cerr << x << " ";
                        // std::cerr << "\nsplit[1] = ";
                        // for(auto x : possibleSplits.splits[1].listOfCandidates) std::cerr << x << " ";
                        // std::cerr << "\n";

                        // auto printColumnInfo = [](const auto& x){
                        //     std::cerr << "(" << x.column << ", " << x.letter << ", " << x.ratio << ") ";
                        // };

                        // std::cerr << "columns[0] = ";
                        // for(auto x : possibleSplits.splits[0].columnInfo) printColumnInfo(x);
                        // std::cerr << "\ncolumns[1] = ";
                        // for(auto x : possibleSplits.splits[1].columnInfo) printColumnInfo(x);
                        // std::cerr << "\n";


                        //create a copy of task, and only keep candidates of first split
                        Task taskCopy = task;
                        taskCopy.splitDepth++;

                        // std::cerr << "split\n";
                        // msa.print(std::cerr); 
                        // std::cerr << "\n into \n";

                        keepSelectedCandidates(taskCopy, possibleSplits.splits[0].listOfCandidates, indexOfActiveTask);
                        const MultipleSequenceAlignment msaOfCopy = constructMsa(taskCopy, indexOfActiveTask);

                        // msaOfCopy.print(std::cerr); 
                        // std::cerr << "\n and \n";

                        extendWithMsa(taskCopy, msaOfCopy, indexOfActiveTask);

                        //only keep canddiates of second split
                        keepSelectedCandidates(task, possibleSplits.splits[1].listOfCandidates, indexOfActiveTask);
                        const MultipleSequenceAlignment newMsa = constructMsa(task, indexOfActiveTask);

                        // newMsa.print(std::cerr); 
                        // std::cerr << "\n";

                        extendWithMsa(task, newMsa, indexOfActiveTask);

                        //if extension was not possible in task, replace task by task copy
                        if(task.abort && task.abortReason == AbortReason::MsaNotExtended){
                            //replace task by taskCopy
                            task = std::move(taskCopy);
                        }else if(!taskCopy.abort){
                            //if extension was possible in both task and taskCopy, taskCopy will be added to tasks and list of active tasks
                            newTaskIndices.emplace_back(tasks.size() + newTasksFromSplit.size());
                            newTasksFromSplit.emplace_back(std::move(taskCopy));

                            splitTracker[task.myReadId]++;


                        }                        
                    }else{
                        extendWithMsa(task, msa, indexOfActiveTask);
                    }
                }else{
                    extendWithMsa(task, msa, indexOfActiveTask);
                }
#else 
                extendWithMsa(task, msa, indexOfActiveTask);
#endif

            }

            msaTimer.stop();

            nvtx::pop_range();

            if(newTasksFromSplit.size() > 0){
                //std::cerr << "Added " << newTasksFromSplit.size() << " tasks\n";
                tasks.insert(tasks.end(), std::make_move_iterator(newTasksFromSplit.begin()), std::make_move_iterator(newTasksFromSplit.end()));
                indicesOfActiveTasks.insert(indicesOfActiveTasks.end(), newTaskIndices.begin(), newTaskIndices.end());

                indicesOfActiveTasksTmp.resize(indicesOfActiveTasks.size());
            }           

            /*
                update book-keeping of used candidates
            */  

            for(int indexOfActiveTask : indicesOfActiveTasks){
                auto& task = tasks[indexOfActiveTask];

                                      
                {
                    std::vector<read_number> tmp(task.allUsedCandidateReadIdPairs.size() + task.candidateReadIds.size());
                    auto tmp_end = std::merge(
                        task.allUsedCandidateReadIdPairs.begin(),
                        task.allUsedCandidateReadIdPairs.end(),
                        task.candidateReadIds.begin(),
                        task.candidateReadIds.end(),
                        tmp.begin()
                    );

                    tmp.erase(tmp_end, tmp.end());

                    std::swap(task.allUsedCandidateReadIdPairs, tmp);
                }

                // task.usedCandidateReadIdsPerIteration.emplace_back(std::move(task.candidateReadIds));
                // task.usedAlignmentsPerIteration.emplace_back(std::move(task.alignments));
                // task.usedAlignmentFlagsPerIteration.emplace_back(std::move(task.alignmentFlags));

                task.iteration++;
            }
            
            //update list of active task indices

            indicesOfActiveTasks.erase(
                std::remove_if(
                    indicesOfActiveTasks.begin(), 
                    indicesOfActiveTasks.end(),
                    [&](int index){
                        return !tasks[index].isActive(insertSize, insertSizeStddev);
                    }
                ),
                indicesOfActiveTasks.end()
            );
        }

        //construct results

        for(const auto& task : tasks){

            ExtendResult extendResult;
            extendResult.direction = task.direction;
            extendResult.numIterations = task.iteration;
            extendResult.aborted = task.abort;
            extendResult.abortReason = task.abortReason;
            extendResult.readId1 = task.myReadId;
            extendResult.readId2 = task.mateReadId;
            extendResult.originalLength = task.myLength;

#if 0
            //extendResult.extendedRead = std::move(task.resultsequence);
            extendResult.success = true;
            extendResult.mateHasBeenFound = task.mateHasBeenFound;

#else
            // if(abort){
            //     ; //no read extension possible
            // }else
            {
                //if(mateHasBeenFound){
                {
                    //construct extended read
                    //build msa of all saved totalDecodedAnchors[0]

                    const int numsteps = task.totalDecodedAnchors.size();

                    // if(task.myReadId == 90 || task.mateReadId == 90){
                    //     std::cerr << "task.totalDecodedAnchors\n";
                    // }

                    int maxlen = 0;
                    for(const auto& s: task.totalDecodedAnchors){
                        const int len = s.length();
                        if(len > maxlen){
                            maxlen = len;
                        }

                        // if(task.myReadId == 90 || task.mateReadId == 90){
                        //     std::cerr << s << "\n";
                        // }
                    }

                    // if(task.myReadId == 90 || task.mateReadId == 90){
                    //     std::cerr << "\n";
                    // }

                    const std::string& decodedAnchor = vecAccess(task.totalDecodedAnchors, 0);

                    const std::vector<int> shifts(task.totalAnchorBeginInExtendedRead.begin() + 1, task.totalAnchorBeginInExtendedRead.end());
                    std::vector<float> initialWeights(numsteps-1, 1.0f);


                    std::vector<char> stepstrings(maxlen * (numsteps-1), '\0');
                    std::vector<int> stepstringlengths(numsteps-1);
                    for(int c = 1; c < numsteps; c++){
                        std::copy(
                            vecAccess(task.totalDecodedAnchors, c).begin(),
                            vecAccess(task.totalDecodedAnchors, c).end(),
                            stepstrings.begin() + (c-1) * maxlen
                        );
                        vecAccess(stepstringlengths, c-1) = vecAccess(task.totalDecodedAnchors, c).size();
                    }

                    MultipleSequenceAlignment::InputData msaInput;
                    msaInput.useQualityScores = false;
                    msaInput.subjectLength = decodedAnchor.length();
                    msaInput.nCandidates = numsteps-1;
                    msaInput.candidatesPitch = maxlen;
                    msaInput.candidateQualitiesPitch = 0;
                    msaInput.subject = decodedAnchor.c_str();
                    msaInput.candidates = stepstrings.data();
                    msaInput.subjectQualities = nullptr;
                    msaInput.candidateQualities = nullptr;
                    msaInput.candidateLengths = stepstringlengths.data();
                    msaInput.candidateShifts = shifts.data();
                    msaInput.candidateDefaultWeightFactors = initialWeights.data();

                    MultipleSequenceAlignment msa;

                    msa.build(msaInput);

                    // if(task.myReadId == 90 || task.mateReadId == 90){
                    //     std::cerr << "Id " << task.myReadId << ", Final\n";
                    //     msa.print(std::cerr);
                    //     std::cerr << "\n";
                    // }

                    extendResult.success = true;

                    std::string extendedRead(msa.consensus.begin(), msa.consensus.end());
                    //std::cerr << "before: " << extendedRead << "\n";
                    std::copy(decodedAnchor.begin(), decodedAnchor.end(), extendedRead.begin());
                    if(task.mateHasBeenFound){
                        std::copy(
                            task.decodedMateRevC.begin(),
                            task.decodedMateRevC.end(),
                            extendedRead.begin() + extendedRead.length() - task.decodedMateRevC.length()
                        );
                    }
                    // extendedRead.replace(extendedRead.begin(), extendedRead.begin() + decodedAnchor, decodedAnchor.begin(), decodedAnchor.end());
                    // std::cerr << "after : " << extendedRead << "\n";
                    
                    // msa.print(std::cerr);
                    // std::cerr << "msa cons:\n";
                    // std::cerr << extendedRead << "\n";
                    // std::cerr << "new cons:\n";
                    // std::cerr << task.resultsequence << "\n";


                    extendResult.extendedRead = std::move(extendedRead);

                    extendResult.mateHasBeenFound = task.mateHasBeenFound;
                }
                // else{
                //     ; //no read extension possible
                // }
            }

            // if(extendResult.extendedRead.length() != task.resultsequence.length()){
            //     std::cerr << task.myReadId << "\n";
            //     std::cerr << extendResult.extendedRead << "\n";
            //     std::cerr << task.resultsequence << "\n";
            //     std::exit(0);
            // }
#endif
            extendResults.emplace_back(std::move(extendResult));

        }

        return extendResults;
    }


    std::vector<ReadExtenderBase::ExtendResult> ReadExtenderGpu::processSingleEndTasks(
        std::vector<ReadExtenderBase::Task>& tasks
    ){
        return processPairedEndTasks(tasks);
    }


    void ReadExtenderGpu::getCandidateReadIds(BatchData& batchData, hipStream_t stream) const{
        nvtx::push_range("gpu_hashing", 2);

        int totalNumValues = 0;

        gpuMinhasher->determineNumValues(
            minhashHandle,
            batchData.d_subjectSequencesData.get(),
            encodedSequencePitchInInts,
            batchData.d_anchorSequencesLength.get(),
            batchData.numTasks,
            batchData.d_numCandidatesPerAnchor.get(),
            totalNumValues,
            stream
        );

        hipStreamSynchronize(stream); CUERR;

        batchData.d_candidateReadIds.resize(totalNumValues);
        batchData.h_candidateReadIds.resize(totalNumValues);

        if(totalNumValues == 0){
            hipMemsetAsync(batchData.d_numCandidatesPerAnchor.get(), 0, sizeof(int) * batchData.numTasks , stream); CUERR;
            hipMemsetAsync(batchData.d_numCandidatesPerAnchorPrefixSum.get(), 0, sizeof(int) * (1 + batchData.numTasks), stream); CUERR;
            return;
        }

        gpuMinhasher->retrieveValues(
            minhashHandle,
            nullptr,
            batchData.numTasks,              
            totalNumValues,
            batchData.d_candidateReadIds.get(),
            batchData.d_numCandidatesPerAnchor.get(),
            batchData.d_numCandidatesPerAnchorPrefixSum.get(),
            stream
        );

        nvtx::pop_range();
    }


    void ReadExtenderGpu::loadCandidateSequenceData(BatchData& batchData, hipStream_t stream) const{

        nvtx::push_range("gpu_loadCandidates", 2);

        const int totalNumCandidates = batchData.h_numCandidatesPerAnchorPrefixSum[batchData.numTasks];

        gpuReadStorage->gatherSequences(
            readStorageHandle,
            batchData.d_candidateSequencesData.get(),
            encodedSequencePitchInInts,
            batchData.h_candidateReadIds.get(),
            batchData.d_candidateReadIds.get(), //device accessible
            totalNumCandidates,
            stream
        );

        gpuReadStorage->gatherSequenceLengths(
            readStorageHandle,
            batchData.d_candidateSequencesLength.get(),
            batchData.d_candidateReadIds.get(),
            totalNumCandidates,
            stream
        );

        readextendergpukernels::reverseComplement2bitKernel<128><<<320,128,0,stream>>>(
            batchData.d_candidateSequencesLength.get(),
            batchData.d_candidateSequencesData.get(),
            batchData.d_candidateSequencesRevcData.get(),
            totalNumCandidates,
            encodedSequencePitchInInts
        ); CUERR;

        nvtx::pop_range();
    }


    void ReadExtenderGpu::eraseDataOfRemovedMates(BatchData& batchData, hipStream_t stream) const{
        nvtx::push_range("gpu_eraseDataOfRemovedMates", 3);

        auto vecAccess = [](auto& vec, auto index) -> decltype(vec.at(index)){
            return vec.at(index);
        };

        if(batchData.numTasksWithMateRemoved > 0){
            const int totalNumCandidates = batchData.h_numCandidatesPerAnchorPrefixSum[batchData.numTasks];

            constexpr int groupsize = 32;
            constexpr int blocksize = 128;
            constexpr int groupsperblock = blocksize / groupsize;
            dim3 block(blocksize,1,1);
            dim3 grid(SDIV(batchData.numTasksWithMateRemoved * groupsize, blocksize), 1, 1);
            const std::size_t smembytes = sizeof(unsigned int) * groupsperblock * encodedSequencePitchInInts;

            helpers::call_fill_kernel_async(batchData.d_flagscandidates.data(), batchData.d_flagscandidates.size(), false, stream);

            readextendergpukernels::filtermatekernel<blocksize,groupsize><<<grid, block, smembytes, stream>>>(
                batchData.d_anchormatedata.data(),
                batchData.d_candidateSequencesData.data(),
                encodedSequencePitchInInts,
                batchData.d_numCandidatesPerAnchor.data(),
                batchData.d_numCandidatesPerAnchorPrefixSum.data(),
                batchData.d_indexlist1.data(),
                batchData.numTasksWithMateRemoved,
                batchData.d_flagscandidates.data()
            ); CUERR;

            auto negate = [] __device__ (bool b){
                return !b;
            };

            hipcub::TransformInputIterator<bool, decltype(negate), bool*> d_keepflags(batchData.d_flagscandidates.data(), negate);

            std::size_t requiredCubSize = 0;
            std::size_t requiredCubSize1 = 0;
            std::size_t requiredCubSize2 = 0;
            hipError_t cubstatus = hipcub::DeviceScan::ExclusiveSum(
                nullptr,
                requiredCubSize1,
                d_keepflags, 
                batchData.d_intbuffercandidates.data(), 
                totalNumCandidates, 
                stream
            );
            assert(hipSuccess == cubstatus);

            cubstatus = hipcub::DeviceScan::InclusiveSum(
                nullptr,
                requiredCubSize2,
                batchData.d_numCandidatesPerAnchor.data(), 
                batchData.d_numCandidatesPerAnchorPrefixSum.data() + 1, 
                batchData.numTasks, 
                stream
            );
            assert(hipSuccess == cubstatus);

            requiredCubSize = std::max(requiredCubSize1, requiredCubSize2);

            batchData.d_tempstorage.resize(requiredCubSize);

            cubstatus = hipcub::DeviceScan::ExclusiveSum(
                batchData.d_tempstorage.data(),
                requiredCubSize,
                d_keepflags, 
                batchData.d_intbuffercandidates.data(), 
                totalNumCandidates, 
                stream
            );
            assert(hipSuccess == cubstatus);

#if 0
            helpers::lambda_kernel<<<4096, 128, 0, stream>>>(
                [
                    numTasks = batchData.numTasks,
                    encodedSequencePitchInInts = encodedSequencePitchInInts,
                    d_numCandidatesPerAnchor = batchData.d_numCandidatesPerAnchor.data(),
                    d_numCandidatesPerAnchorPrefixSum = batchData.d_numCandidatesPerAnchorPrefixSum.data(),
                    d_removalflags = batchData.d_flagscandidates.data(),
                    d_outputpositions = batchData.d_intbuffercandidates.data(),
                    d_candidateReadIds = batchData.d_candidateReadIds.data(),
                    d_candidateSequencesLength = batchData.d_candidateSequencesLength.data(),
                    d_candidateSequencesData = batchData.d_candidateSequencesData.data(),
                    d_candidateSequencesRevcData = batchData.d_candidateSequencesRevcData.data(),
                    d_candidateReadIdsOut = batchData.d_candidateReadIds2.data(),
                    d_candidateSequencesLengthOut = batchData.d_candidateSequencesLength2.data(),
                    d_candidateSequencesDataOut = batchData.d_candidateSequencesData2.data(),
                    d_candidateSequencesRevcDataOut = batchData.d_candidateSequencesRevcData2.data(),
                    d_candidateReadIdsOutSize = batchData.d_candidateReadIds2.size(),
                    d_candidateSequencesLengthOutSize = batchData.d_candidateSequencesLength2.size(),
                    d_candidateSequencesDataOutSize = batchData.d_candidateSequencesData2.size(),
                    d_candidateSequencesRevcDataOutSize = batchData.d_candidateSequencesRevcData2.size()
                ] __device__ (){

                    constexpr int elementsPerIteration = 128;
                    __shared__ bool smem_removalflags[elementsPerIteration];
                    __shared__ int smem_outputpositions[elementsPerIteration];

                    using BlockReduce = hipcub::BlockReduce<int, elementsPerIteration>;
                    __shared__ typename BlockReduce::TempStorage temp_storage;

                    auto group = cg::tiled_partition<8>(cg::this_thread_block());
                    const int numGroupsInBlock = blockDim.x / 8;
                    const int groupInBlock = threadIdx.x / 8;

                    for(int t = blockIdx.x; t < numTasks; t += gridDim.x){
                        const int numCandidates = d_numCandidatesPerAnchor[t];
                        const int inputOffset = d_numCandidatesPerAnchorPrefixSum[t];

                        int numSelected = 0;

                        const int numSmemIterations = SDIV(numCandidates, elementsPerIteration);
                        for(int smemiter = 0; smemiter < numSmemIterations; smemiter++){
                            const int first = smemiter * elementsPerIteration;
                            const int last = min((smemiter+1) * elementsPerIteration, numCandidates);
                            const int num = last - first;

                            for(int i = threadIdx.x; i < num; i += blockDim.x){
                                smem_removalflags[i] = d_removalflags[inputOffset + first + i];
                                smem_outputpositions[i] = d_outputpositions[inputOffset + first + i];
                            }
                            __syncthreads();

                            for(int i = threadIdx.x; i < num; i += blockDim.x){
                                if(!smem_removalflags[i]){
                                    assert(d_candidateReadIdsOutSize > smem_outputpositions[i]);
                                    assert(d_candidateSequencesLengthOutSize > smem_outputpositions[i]);
                                    d_candidateReadIdsOut[smem_outputpositions[i]] = d_candidateReadIds[inputOffset + first + i];
                                    d_candidateSequencesLengthOut[smem_outputpositions[i]] = d_candidateSequencesLength[inputOffset + first + i];
                                }
                            }

                            for(int i = threadIdx.x; i < num * encodedSequencePitchInInts; i += blockDim.x){
                                const int which = i / encodedSequencePitchInInts;
                                const int what = i % encodedSequencePitchInInts;

                                if(!smem_removalflags[which]){
                                    assert(d_candidateSequencesDataOutSize > smem_outputpositions[i]);
                                    assert(d_candidateSequencesRevcDataOutSize > smem_outputpositions[i]);
                                    d_candidateSequencesDataOut[smem_outputpositions[which] * encodedSequencePitchInInts + what] = d_candidateSequencesData[(inputOffset + first) * encodedSequencePitchInInts + what];
                                    d_candidateSequencesRevcDataOut[smem_outputpositions[which] * encodedSequencePitchInInts + what] = d_candidateSequencesRevcData[(inputOffset + first) * encodedSequencePitchInInts + what];
                                }
                            }

                            int flag = 0;
                            if(threadIdx.x < num){
                                flag = !smem_removalflags[threadIdx.x];
                            }
                            numSelected += BlockReduce(temp_storage).Sum(flag);

                            __syncthreads();
                        }

                        if(threadIdx.x == 0){
                            d_numCandidatesPerAnchor[t] = numSelected;
                        }
                    }
                }
            ); CUERR;
#else 
            // helpers::lambda_kernel<<<1,1, 0, stream>>>(
            //     [
            //         numTasks = batchData.numTasks,
            //         d_numCandidatesPerAnchor = batchData.d_numCandidatesPerAnchor.data(),
            //         d_numCandidatesPerAnchorPrefixSum = batchData.d_numCandidatesPerAnchorPrefixSum.data(),
            //         d_removalflags = batchData.d_flagscandidates.data(),
            //         d_outputpositions = batchData.d_intbuffercandidates.data()
            //     ] __device__ (){
            //         for(int t = blockIdx.x; t < numTasks; t += gridDim.x){
            //             const int numCandidates = d_numCandidatesPerAnchor[t];
            //             const int inputOffset = d_numCandidatesPerAnchorPrefixSum[t];
            //             assert(d_numCandidatesPerAnchorPrefixSum[t+1] == numCandidates + inputOffset);

            //             for(int i = 0; i < numCandidates; i++){
            //                 if(d_removalflags[inputOffset + i]){
            //                     for(int k = 0; k < numCandidates; k++){
            //                         printf("%d %d %d\n", inputOffset + k, int(d_removalflags[inputOffset + k]), d_outputpositions[inputOffset + k]);
            //                     }
            //                     break;
            //                 }
            //             }
            //         }
            //     }
            // ); CUERR;
            // hipDeviceSynchronize(); CUERR;

            helpers::lambda_kernel<<<4096, 128, 0, stream>>>(
                [
                    numTasks = batchData.numTasks,
                    encodedSequencePitchInInts = encodedSequencePitchInInts,
                    d_numCandidatesPerAnchor = batchData.d_numCandidatesPerAnchor.data(),
                    d_numCandidatesPerAnchorPrefixSum = batchData.d_numCandidatesPerAnchorPrefixSum.data(),
                    d_removalflags = batchData.d_flagscandidates.data(),
                    d_outputpositions = batchData.d_intbuffercandidates.data(),
                    d_candidateReadIds = batchData.d_candidateReadIds.data(),
                    d_candidateSequencesLength = batchData.d_candidateSequencesLength.data(),
                    d_candidateSequencesData = batchData.d_candidateSequencesData.data(),
                    d_candidateSequencesRevcData = batchData.d_candidateSequencesRevcData.data(),
                    d_candidateReadIdsOut = batchData.d_candidateReadIds2.data(),
                    d_candidateSequencesLengthOut = batchData.d_candidateSequencesLength2.data(),
                    d_candidateSequencesDataOut = batchData.d_candidateSequencesData2.data(),
                    d_candidateSequencesRevcDataOut = batchData.d_candidateSequencesRevcData2.data(),
                    d_candidateReadIdsOutSize = batchData.d_candidateReadIds2.size(),
                    d_candidateSequencesLengthOutSize = batchData.d_candidateSequencesLength2.size(),
                    d_candidateSequencesDataOutSize = batchData.d_candidateSequencesData2.size(),
                    d_candidateSequencesRevcDataOutSize = batchData.d_candidateSequencesRevcData2.size()
                ] __device__ (){

                    constexpr int elementsPerIteration = 128;
                    __shared__ bool smem_removalflags[elementsPerIteration];
                    __shared__ int smem_outputpositions[elementsPerIteration];

                    using BlockReduce = hipcub::BlockReduce<int, elementsPerIteration>;
                    __shared__ typename BlockReduce::TempStorage temp_storage;

                    auto group = cg::tiled_partition<8>(cg::this_thread_block());
                    const int numGroupsInBlock = blockDim.x / 8;
                    const int groupInBlock = threadIdx.x / 8;

                    for(int t = blockIdx.x; t < numTasks; t += gridDim.x){
                        const int numCandidates = d_numCandidatesPerAnchor[t];
                        const int inputOffset = d_numCandidatesPerAnchorPrefixSum[t];

                        int numSelected = 0;

                        for(int i = threadIdx.x; i < numCandidates; i += blockDim.x){
                            if(!d_removalflags[inputOffset + i]){
                                d_candidateReadIdsOut[d_outputpositions[inputOffset + i]] = d_candidateReadIds[inputOffset + i];
                                d_candidateSequencesLengthOut[d_outputpositions[inputOffset + i]] = d_candidateSequencesLength[inputOffset + i];

                                numSelected++;
                            }
                        }

                        for(int i = threadIdx.x; i < numCandidates * encodedSequencePitchInInts; i += blockDim.x){
                            const int which = i / encodedSequencePitchInInts;
                            const int what = i % encodedSequencePitchInInts;

                            if(!d_removalflags[inputOffset + which]){
                                d_candidateSequencesDataOut[d_outputpositions[inputOffset + which] * encodedSequencePitchInInts + what] = d_candidateSequencesData[(inputOffset + which) * encodedSequencePitchInInts + what];
                                d_candidateSequencesRevcDataOut[d_outputpositions[inputOffset + which] * encodedSequencePitchInInts + what] = d_candidateSequencesRevcData[(inputOffset + which) * encodedSequencePitchInInts + what];
                            }
                        }

                        numSelected = BlockReduce(temp_storage).Sum(numSelected);
                        __syncthreads();
                        
                        if(threadIdx.x == 0){
                            if(numSelected != numCandidates){
                                assert(numSelected < numCandidates);
                                d_numCandidatesPerAnchor[t] = numSelected;
                                //printf("task %d, removed %d\n", t, numCandidates - numSelected);
                            }
                        }

                    }
                }
            ); CUERR;

#endif

            //update prefix sum

            cubstatus = hipcub::DeviceScan::InclusiveSum(
                batchData.d_tempstorage.data(),
                requiredCubSize,
                batchData.d_numCandidatesPerAnchor.data(), 
                batchData.d_numCandidatesPerAnchorPrefixSum.data() + 1, 
                batchData.numTasks, 
                stream
            );
            if(cubstatus != hipSuccess){
                CUERR;
                assert(batchData.d_tempstorage.data() != nullptr);

                std::cerr << "cub error: " << hipGetErrorString(cubstatus) << ", batchData.numTasks: " << batchData.numTasks << ", requiredCubSize: " << requiredCubSize << "\n";
                //std::cerr << batchData.h_readIds[0] << "\n";
                std::size_t foo = 0;

                hipcub::DeviceScan::InclusiveSum(
                    nullptr,
                    foo,
                    batchData.d_numCandidatesPerAnchor.data(), 
                    batchData.d_numCandidatesPerAnchorPrefixSum.data() + 1, 
                    batchData.numTasks, 
                    stream
                );

                std::cerr << "required cub size for inclusive sum: " << foo << "\n";
            }
            assert(hipSuccess == cubstatus);

            std::swap(batchData.d_candidateReadIds2, batchData.d_candidateReadIds);
            std::swap(batchData.d_candidateSequencesLength2, batchData.d_candidateSequencesLength);
            std::swap(batchData.d_candidateSequencesData2, batchData.d_candidateSequencesData);
            std::swap(batchData.d_candidateSequencesRevcData2, batchData.d_candidateSequencesRevcData);

        }

        nvtx::pop_range();
       
    }


    void ReadExtenderGpu::calculateAlignments(BatchData& batchData, hipStream_t stream) const{
        nvtx::push_range("gpu_alignment", 4);

        
        batchData.h_numAnchors[0] = batchData.numTasks;

        const bool* const d_anchorContainsN = nullptr;
        const bool* const d_candidateContainsN = nullptr;
        const bool removeAmbiguousAnchors = false;
        const bool removeAmbiguousCandidates = false;
        const int maxNumAnchors = batchData.numTasks;
        const int maxNumCandidates = batchData.h_numCandidatesPerAnchorPrefixSum[batchData.numTasks];
        const int maximumSequenceLength = 100; //encodedSequencePitchInInts * 16;
        const int encodedSequencePitchInInts2Bit = encodedSequencePitchInInts;
        const int min_overlap = goodAlignmentProperties.min_overlap;
        const float maxErrorRate = goodAlignmentProperties.maxErrorRate;
        const float min_overlap_ratio = goodAlignmentProperties.min_overlap_ratio;
        const float estimatedNucleotideErrorRate = correctionOptions.estimatedErrorrate;

        auto callAlignmentKernel = [&](void* d_tempstorage, size_t& tempstoragebytes){

            call_popcount_rightshifted_hamming_distance_kernel_async(
                d_tempstorage,
                tempstoragebytes,
                batchData.d_alignment_overlaps.get(),
                batchData.d_alignment_shifts.get(),
                batchData.d_alignment_nOps.get(),
                batchData.d_alignment_isValid.get(),
                batchData.d_alignment_best_alignment_flags.get(),
                batchData.d_subjectSequencesData.get(),
                batchData.d_candidateSequencesData.get(),
                batchData.d_anchorSequencesLength.get(),
                batchData.d_candidateSequencesLength.get(),
                batchData.d_numCandidatesPerAnchorPrefixSum.get(),
                batchData.d_numCandidatesPerAnchor.get(),
                batchData.d_intbuffercandidates.get(),
                batchData.h_numAnchors.get(),
                &batchData.d_numCandidatesPerAnchorPrefixSum[batchData.numTasks],
                d_anchorContainsN,
                removeAmbiguousAnchors,
                d_candidateContainsN,
                removeAmbiguousCandidates,
                maxNumAnchors,
                maxNumCandidates,
                maximumSequenceLength,
                encodedSequencePitchInInts2Bit,
                min_overlap,
                maxErrorRate,
                min_overlap_ratio,
                estimatedNucleotideErrorRate,
                stream,
                kernelLaunchHandle
            );
        };

        const int num = batchData.h_numCandidatesPerAnchorPrefixSum[batchData.numTasks];

        assert(batchData.d_intbuffercandidates.size() >= num);
        assert(batchData.h_numAnchors[0] == batchData.numTasks);
        assert(batchData.d_numCandidatesPerAnchor.size() >= batchData.numTasks);
        assert(batchData.d_numCandidatesPerAnchorPrefixSum.size() >= batchData.numTasks+1);

        readextendergpukernels::setAnchorIndicesOfCandidateskernel<<<1024, 128, 0, stream>>>(
            batchData.d_intbuffercandidates.data(),
            batchData.h_numAnchors.data(),
            batchData.d_numCandidatesPerAnchor.get(),
            batchData.d_numCandidatesPerAnchorPrefixSum.get()
        );

        size_t tempstoragebytes = 0;
        callAlignmentKernel(nullptr, tempstoragebytes);

        batchData.d_tempstorage.resize(tempstoragebytes);

        callAlignmentKernel(batchData.d_tempstorage.get(), tempstoragebytes);

        nvtx::pop_range();
    }



    void ReadExtenderGpu::filterAlignments(BatchData& batchData, hipStream_t stream) const{
        nvtx::push_range("gpu_filterAlignments", 5);

        const int totalNumCandidates = batchData.h_numCandidatesPerAnchorPrefixSum[batchData.numTasks];
        const int numAnchors = batchData.numTasks;

        helpers::call_fill_kernel_async(batchData.d_flagscandidates.data(), batchData.d_flagscandidates.size(), true, stream);

        bool* const d_keepflags = batchData.d_flagscandidates.data();

        dim3 block(128,1,1);
        dim3 grid(numAnchors, 1, 1);

        //filter alignments of candidates. d_keepflags[i] will be set to false if candidate[i] should be removed
        //batchData.d_numCandidatesPerAnchor2[i] contains new number of candidates for anchor i
        helpers::lambda_kernel<<<grid, block, 0, stream>>>(
            [
                d_alignment_best_alignment_flags = batchData.d_alignment_best_alignment_flags.data(),
                d_alignment_shifts = batchData.d_alignment_shifts.data(),
                d_alignment_overlaps = batchData.d_alignment_overlaps.data(),
                d_anchorSequencesLength = batchData.d_anchorSequencesLength.data(),
                d_numCandidatesPerAnchor = batchData.d_numCandidatesPerAnchor.data(),
                d_numCandidatesPerAnchor2 = batchData.d_numCandidatesPerAnchor2.data(),
                d_numCandidatesPerAnchorPrefixSum = batchData.d_numCandidatesPerAnchorPrefixSum.data(),
                d_keepflags,
                min_overlap_ratio = goodAlignmentProperties.min_overlap_ratio,
                numAnchors
            ] __device__ (){

                using BlockReduceFloat = hipcub::BlockReduce<float, 128>;
                using BlockReduceInt = hipcub::BlockReduce<int, 128>;

                __shared__ union {
                    typename BlockReduceFloat::TempStorage floatreduce;
                    typename BlockReduceInt::TempStorage intreduce;
                } cubtemp;

                __shared__ int intbroadcast;
                __shared__ float floatbroadcast;

                for(int a = blockIdx.x; a < numAnchors; a += gridDim.x){
                    const int num = d_numCandidatesPerAnchor[a];
                    const int offset = d_numCandidatesPerAnchorPrefixSum[a];
                    const float anchorLength = d_anchorSequencesLength[a];
                    int removed = 0;

                    int threadReducedGoodAlignmentExists = 0;
                    float threadReducedRelativeOverlapThreshold = 0.0f;

                    //loop over candidates to compute relative overlap threshold

                    for(int c = threadIdx.x; c < num; c += blockDim.x){
                        const auto alignmentflag = d_alignment_best_alignment_flags[offset + c];
                        const int shift = d_alignment_shifts[offset + c];

                        if(alignmentflag != BestAlignment_t::None && shift >= 0){
                            bool goodAlignmentExists = false;
                            const float overlap = d_alignment_overlaps[offset + c];                            
                            const float relativeOverlap = overlap / anchorLength;
                            
                            if(relativeOverlap < 1.0f && fgeq(relativeOverlap, min_overlap_ratio)){
                                threadReducedGoodAlignmentExists = 1;
                                const float tmp = floorf(relativeOverlap * 10.0f) / 10.0f;
                                threadReducedRelativeOverlapThreshold = fmaxf(threadReducedRelativeOverlapThreshold, tmp);
                            }

                            // while(!goodAlignmentExists && fgeq(relativeOverlapThreshold, min_overlap_ratio)){

                            //     goodAlignmentExists = fgeq(relativeOverlap, relativeOverlapThreshold) && relativeOverlap < 1.0f;

                            //     if(!goodAlignmentExists){
                            //         relativeOverlapThreshold -= 0.1f;
                            //     }
                            // }

                            // if(goodAlignmentExists){
                            //     threadReducedGoodAlignmentExists = 1;
                            //     threadReducedRelativeOverlapThreshold = max(threadReducedRelativeOverlapThreshold, relativeOverlapThreshold);
                            // }

                            // if(a == 1){
                            //     printf("a %d c %d relativeOverlap %f, thread good %d, thresh %f\n", 
                            //         a, c, relativeOverlap, goodAlignmentExists, threadReducedRelativeOverlapThreshold);
                            // }
                        }else{
                            //remove alignment with negative shift
                            d_keepflags[offset + c] = false;
                            removed++;
                        }

                        
                    }
                    // __syncthreads(); //debug
                    // if(threadIdx.x < num){
                    //     printf("a %d thread good %d, thresh %f\n", a, threadReducedGoodAlignmentExists, threadReducedRelativeOverlapThreshold);
                    // }
                    // __syncthreads(); //debug

                    int blockreducedGoodAlignmentExists = BlockReduceInt(cubtemp.intreduce)
                        .Sum(threadReducedGoodAlignmentExists);
                    if(threadIdx.x == 0){
                        intbroadcast = blockreducedGoodAlignmentExists;
                        //printf("task %d good: %d\n", a, blockreducedGoodAlignmentExists);
                    }
                    __syncthreads();

                    blockreducedGoodAlignmentExists = intbroadcast;

                    if(blockreducedGoodAlignmentExists > 0){
                        float blockreducedRelativeOverlapThreshold = BlockReduceFloat(cubtemp.floatreduce)
                            .Reduce(threadReducedRelativeOverlapThreshold, hipcub::Max());
                        if(threadIdx.x == 0){
                            floatbroadcast = blockreducedRelativeOverlapThreshold;
                            //printf("task %d thresh: %f\n", a, blockreducedRelativeOverlapThreshold);
                        }
                        __syncthreads();

                        blockreducedRelativeOverlapThreshold = floatbroadcast;

                        // loop over candidates and remove those with an alignment overlap threshold smaller than the computed threshold
                        for(int c = threadIdx.x; c < num; c += blockDim.x){
    
                            if(d_keepflags[offset + c]){
                                const float overlap = d_alignment_overlaps[offset + c];                            
                                const float relativeOverlap = overlap / anchorLength;                 
    
                                if(!fgeq(relativeOverlap, blockreducedRelativeOverlapThreshold)){
                                    d_keepflags[offset + c] = false;
                                    removed++;
                                }
                            }
                        }
                    }else{
                        //NOOP.
                        //if no good alignment exists, no candidate is removed. we will try to work with the not-so-good alignments
                    }

                    removed = BlockReduceInt(cubtemp.intreduce).Sum(removed);

                    if(threadIdx.x == 0){
                        d_numCandidatesPerAnchor2[a] = num - removed;
                        //printf("task %d remaining: %d - %d = %d\n", a, num, removed, num - removed);
                    }
                    __syncthreads();
                }
            }
        ); CUERR;

        //setup cub 
        auto d_zip_input = thrust::make_zip_iterator(
            thrust::make_tuple(
                batchData.d_alignment_nOps.data(),
                batchData.d_alignment_overlaps.data(),
                batchData.d_alignment_shifts.data(),
                batchData.d_alignment_isValid.data(),
                batchData.d_alignment_best_alignment_flags.data(),
                batchData.d_candidateReadIds.data(),
                batchData.d_candidateSequencesLength.data()
            )
        );

        auto d_zip_output = thrust::make_zip_iterator(
            thrust::make_tuple(
                batchData.d_alignment_nOps2.data(),
                batchData.d_alignment_overlaps2.data(),
                batchData.d_alignment_shifts2.data(),
                batchData.d_alignment_isValid2.data(),
                batchData.d_alignment_best_alignment_flags2.data(),
                batchData.d_candidateReadIds2.data(),
                batchData.d_candidateSequencesLength2.data()
            )
        );

        std::size_t requiredCubSize1 = 0;
        hipError_t cubstatus = hipcub::DeviceSelect::Flagged(
            nullptr, 
            requiredCubSize1, 
            d_zip_input, 
            d_keepflags, 
            d_zip_output, 
            batchData.d_numCandidates.data(), 
            totalNumCandidates, 
            stream
        );
        assert(cubstatus == hipSuccess);

        std::size_t requiredCubSize2 = 0;
        cubstatus = hipcub::DeviceScan::ExclusiveSum(
            nullptr,
            requiredCubSize2,
            d_keepflags, 
            batchData.d_intbuffercandidates.data(), 
            totalNumCandidates, 
            stream
        );
        assert(cubstatus == hipSuccess);

        std::size_t requiredCubSize3 = 0;
        cubstatus = hipcub::DeviceScan::InclusiveSum(
            nullptr,
            requiredCubSize3,
            batchData.d_numCandidatesPerAnchor2.data(), 
            batchData.d_numCandidatesPerAnchorPrefixSum.data() + 1, 
            batchData.numTasks, 
            stream
        );
        assert(hipSuccess == cubstatus);

        std::size_t requiredCubSize = std::max(std::max(requiredCubSize1, requiredCubSize2), requiredCubSize3);
        batchData.d_tempstorage.resize(requiredCubSize);

        //compute output positions for selected candidates

        cubstatus = hipcub::DeviceScan::ExclusiveSum(
            batchData.d_tempstorage.data(), 
            requiredCubSize,
            d_keepflags, 
            batchData.d_intbuffercandidates.data(), 
            totalNumCandidates, 
            stream
        );
        assert(cubstatus == hipSuccess);

        //compact zip data
        cubstatus = hipcub::DeviceSelect::Flagged(
            batchData.d_tempstorage.data(), 
            requiredCubSize, 
            d_zip_input, 
            d_keepflags, 
            d_zip_output, 
            batchData.d_numCandidates.data(), 
            totalNumCandidates, 
            stream
        );
        assert(cubstatus == hipSuccess);

        //compact sequence data. if alignmentflag is forward, forward sequence data will be copied, 
        //else reverse complement will be copied
        helpers::lambda_kernel<<<4096, 128, 0, stream>>>(
            [
                encodedSequencePitchInInts = encodedSequencePitchInInts,
                d_keepflags,
                totalNumCandidates,
                d_outputpositions = batchData.d_intbuffercandidates.data(),
                d_alignment_best_alignment_flags = batchData.d_alignment_best_alignment_flags.data(),
                d_candidateSequencesData = batchData.d_candidateSequencesData.data(),
                d_candidateSequencesRevcData = batchData.d_candidateSequencesRevcData.data(),
                d_candidateSequencesDataOut = batchData.d_candidateSequencesData2.data()
            ] __device__ (){

                const int tid = threadIdx.x + blockIdx.x * blockDim.x;
                const int stride = blockDim.x * gridDim.x;
                const int elements = totalNumCandidates * encodedSequencePitchInInts;

                for(int i = tid; i < elements; i += stride){
                    const int which = i / encodedSequencePitchInInts;
                    const int what = i % encodedSequencePitchInInts;

                    if(d_keepflags[which]){

                        const int outputindex = d_outputpositions[which] * encodedSequencePitchInInts + what;
                        const int inputindex = which * encodedSequencePitchInInts + what;

                        const auto alignmentflag = d_alignment_best_alignment_flags[which];
                        
                        if(alignmentflag == BestAlignment_t::Forward){                             
                            d_candidateSequencesDataOut[outputindex] = d_candidateSequencesData[inputindex];
                        }else{
                            d_candidateSequencesDataOut[outputindex] = d_candidateSequencesRevcData[inputindex];
                        }
                    }
                }
            }
        ); CUERR;

        //hipDeviceSynchronize(); CUERR;

        //update prefix sum
        cubstatus = hipcub::DeviceScan::InclusiveSum(
            batchData.d_tempstorage.data(), 
            requiredCubSize, 
            batchData.d_numCandidatesPerAnchor2.data(), 
            batchData.d_numCandidatesPerAnchorPrefixSum.data() + 1, 
            batchData.numTasks, 
            stream
        );
        assert(hipSuccess == cubstatus);

        std::swap(batchData.d_alignment_nOps2, batchData.d_alignment_nOps);
        std::swap(batchData.d_alignment_overlaps2, batchData.d_alignment_overlaps);
        std::swap(batchData.d_alignment_shifts2, batchData.d_alignment_shifts);
        std::swap(batchData.d_alignment_isValid2, batchData.d_alignment_isValid);
        std::swap(batchData.d_alignment_best_alignment_flags2, batchData.d_alignment_best_alignment_flags);
        std::swap(batchData.d_candidateReadIds2, batchData.d_candidateReadIds);
        std::swap(batchData.d_candidateSequencesLength2, batchData.d_candidateSequencesLength);
        std::swap(batchData.d_numCandidatesPerAnchor2, batchData.d_numCandidatesPerAnchor);
        std::swap(batchData.d_candidateSequencesData2, batchData.d_candidateSequencesData);

        nvtx::pop_range();
    }




}