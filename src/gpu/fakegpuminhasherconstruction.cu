#include "hip/hip_runtime.h"

#include <gpu/fakegpuminhasherconstruction.cuh>

#include <gpu/fakegpuminhasher.cuh>
#include <gpu/fakegpusinglehashminhasher.cuh>

#include <minhasherlimit.hpp>

#include <options.hpp>

#include <memory>
#include <utility>


namespace care{
namespace gpu{

        void constructFromReadStorage(
            const ProgramOptions& programOptions,
            const GpuReadStorage& gpuReadStorage,
            GpuMinhasher* gpuMinhasher
        ){
            
            auto& readStorage = gpuReadStorage;
            const auto& deviceIds = programOptions.deviceIds;

            int deviceId = deviceIds[0];

            hipcub::SwitchDevice sd{deviceId};

            const int requestedNumberOfMaps = programOptions.numHashFunctions;

            const read_number numReads = readStorage.getNumberOfReads();
            const int maximumSequenceLength = readStorage.getSequenceLengthUpperBound();

            auto sequencehandle = gpuReadStorage.makeHandle();
            const std::size_t encodedSequencePitchInInts = SequenceHelpers::getEncodedNumInts2Bit(maximumSequenceLength);

            rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource();

            constexpr read_number parallelReads = 1000000;
            const int numIters = SDIV(numReads, parallelReads);

            const MemoryUsage memoryUsageOfReadStorage = readStorage.getMemoryInfo();
            std::size_t totalLimit = programOptions.memoryTotalLimit;
            if(totalLimit > memoryUsageOfReadStorage.host){
                totalLimit -= memoryUsageOfReadStorage.host;
            }else{
                totalLimit = 0;
            }
            if(totalLimit == 0){
                throw std::runtime_error("Not enough memory available for hash tables. Abort!");
            }
            std::size_t maxMemoryForTables = getAvailableMemoryInKB() * 1024;
            // std::cerr << "available: " << maxMemoryForTables 
            //         << ",memoryForHashtables: " << programOptions.memoryForHashtables
            //         << ", memoryTotalLimit: " << programOptions.memoryTotalLimit
            //         << ", rsHostUsage: " << memoryUsageOfReadStorage.host << "\n";

            maxMemoryForTables = std::min(maxMemoryForTables, 
                                    std::min(programOptions.memoryForHashtables, totalLimit));

            std::cerr << "maxMemoryForTables = " << maxMemoryForTables << " bytes\n";

            const int hashFunctionOffset = 0;

            
            std::vector<int> usedHashFunctionNumbers;

            hipStream_t stream = hipStreamPerThread;
            
            rmm::device_uvector<unsigned int> d_sequenceData(encodedSequencePitchInInts * parallelReads, stream, mr);
            rmm::device_uvector<int> d_lengths(parallelReads, stream, mr);
            rmm::device_uvector<read_number> d_indices(parallelReads, stream, mr);
            
            helpers::SimpleAllocationPinnedHost<read_number, 0> h_indices(parallelReads);

            
            ThreadPool tpForHashing(programOptions.threads);
            ThreadPool tpForCompacting(std::min(2,programOptions.threads));

            
            gpuMinhasher->setHostMemoryLimitForConstruction(maxMemoryForTables);
            gpuMinhasher->setDeviceMemoryLimitsForConstruction({1*1024*1024*1024});
            
            //std::size_t bytesOfCachedConstructedTables = 0;
            int remainingHashFunctions = requestedNumberOfMaps;
            bool keepGoing = true;

            while(remainingHashFunctions > 0 && keepGoing){

                gpuMinhasher->setThreadPool(&tpForHashing);

                const int alreadyExistingHashFunctions = requestedNumberOfMaps - remainingHashFunctions;
                std::vector<int> h_hashfunctionNumbers(remainingHashFunctions);
                std::iota(
                    h_hashfunctionNumbers.begin(),
                    h_hashfunctionNumbers.end(),
                    alreadyExistingHashFunctions + hashFunctionOffset
                );
    
                int addedHashFunctions = gpuMinhasher->addHashTables(remainingHashFunctions,h_hashfunctionNumbers.data(), stream);
    
                if(addedHashFunctions == 0){
                    keepGoing = false;
                    break;
                }
    
                std::cout << "Constructing maps: ";
                for(int i = 0; i < addedHashFunctions; i++){
                    std::cout << (alreadyExistingHashFunctions + i) << "(" << (hashFunctionOffset + alreadyExistingHashFunctions + i) << ") ";
                }
                std::cout << '\n';
    
                usedHashFunctionNumbers.insert(usedHashFunctionNumbers.end(), h_hashfunctionNumbers.begin(), h_hashfunctionNumbers.begin() + addedHashFunctions);

                for (int iter = 0; iter < numIters; iter++){
                    read_number readIdBegin = iter * parallelReads;
                    read_number readIdEnd = std::min((iter + 1) * parallelReads, numReads);

                    const std::size_t curBatchsize = readIdEnd - readIdBegin;

                    std::iota(h_indices.get(), h_indices.get() + curBatchsize, readIdBegin);

                    CUDACHECK(hipMemcpyAsync(d_indices.data(), h_indices, sizeof(read_number) * curBatchsize, H2D, stream));

                    gpuReadStorage.gatherSequences(
                        sequencehandle,
                        d_sequenceData.data(),
                        encodedSequencePitchInInts,
                        makeAsyncConstBufferWrapper(h_indices.data()),
                        d_indices.data(),
                        curBatchsize,
                        stream,
                        mr
                    );
                
                    gpuReadStorage.gatherSequenceLengths(
                        sequencehandle,
                        d_lengths.data(),
                        d_indices.data(),
                        curBatchsize,
                        stream
                    );

                    gpuMinhasher->insert(
                        d_sequenceData.data(),
                        curBatchsize,
                        d_lengths.data(),
                        encodedSequencePitchInInts,
                        d_indices.data(),
                        h_indices,
                        alreadyExistingHashFunctions,
                        addedHashFunctions,
                        h_hashfunctionNumbers.data(),
                        stream,
                        mr
                    );

                    CUDACHECK(hipStreamSynchronize(stream));
                }

                CUDACHECK(hipStreamSynchronize(stream));

                std::cerr << "Compacting\n";
                if(tpForCompacting.getConcurrency() > 1){
                    gpuMinhasher->setThreadPool(&tpForCompacting);
                }else{
                    gpuMinhasher->setThreadPool(nullptr);
                }
                
                gpuMinhasher->compact(stream);
                CUDACHECK(hipStreamSynchronize(stream));

                remainingHashFunctions -= addedHashFunctions;
            }

            gpuMinhasher->setThreadPool(nullptr); 
            
            gpuReadStorage.destroyHandle(sequencehandle);

            gpuMinhasher->constructionIsFinished(stream);
            CUDACHECK(hipStreamSynchronize(stream));
        }

        #if 1
        std::unique_ptr<FakeGpuMinhasher>
        constructFakeGpuMinhasherFromGpuReadStorage(
            const ProgramOptions& programOptions,
            const GpuReadStorage& gpuReadStorage
        ){
            std::cerr << "USE NEW FAKE MIN CONSTRUCTION\n";
            float loadfactor = programOptions.hashtableLoadfactor;
            
            auto gpuMinhasher = std::make_unique<FakeGpuMinhasher>(
                gpuReadStorage.getNumberOfReads(),
                calculateResultsPerMapThreshold(programOptions.estimatedCoverage),
                programOptions.kmerlength,
                loadfactor
            );

            if(programOptions.load_hashtables_from != ""){

                std::ifstream is(programOptions.load_hashtables_from);
                assert((bool)is);

                const int loadedMaps = gpuMinhasher->loadFromStream(is, programOptions.numHashFunctions);

                std::cout << "Loaded " << loadedMaps << " hash tables from " << programOptions.load_hashtables_from << std::endl;
            }else{
                constructFromReadStorage(
                    programOptions,
                    gpuReadStorage,
                    gpuMinhasher.get()
                );
            }

            return gpuMinhasher;
        }

        #else

        std::unique_ptr<FakeGpuMinhasher>
        constructFakeGpuMinhasherFromGpuReadStorage(
            const ProgramOptions& programOptions,
            const GpuReadStorage& gpuReadStorage
        ){
            float loadfactor = programOptions.hashtableLoadfactor;
            
            auto gpuMinhasher = std::make_unique<FakeGpuMinhasher>(
                gpuReadStorage.getNumberOfReads(),
                calculateResultsPerMapThreshold(programOptions.estimatedCoverage),
                programOptions.kmerlength,
                loadfactor
            );

            if(programOptions.load_hashtables_from != ""){

                std::ifstream is(programOptions.load_hashtables_from);
                assert((bool)is);
    
                const int loadedMaps = gpuMinhasher->loadFromStream(is, programOptions.numHashFunctions);
    
                std::cout << "Loaded " << loadedMaps << " hash tables from " << programOptions.load_hashtables_from << std::endl;
            }else{
                gpuMinhasher->constructFromReadStorage(
                    programOptions,
                    gpuReadStorage.getNumberOfReads(), 
                    gpuReadStorage
                );
            }

            return gpuMinhasher;
        }

        #endif

        #if 0
        std::unique_ptr<FakeGpuSingleHashMinhasher>
        constructFakeGpuSingleHashMinhasherFromGpuReadStorage(
            const ProgramOptions& programOptions,
            const GpuReadStorage& gpuReadStorage
        ){
            float loadfactor = programOptions.hashtableLoadfactor;
            
            auto gpuMinhasher = std::make_unique<FakeGpuSingleHashMinhasher>(
                gpuReadStorage.getNumberOfReads(),
                255,
                programOptions.kmerlength,
                loadfactor
            );

            if(programOptions.load_hashtables_from != ""){

                std::ifstream is(programOptions.load_hashtables_from);
                assert((bool)is);

                const int loadedMaps = gpuMinhasher->loadFromStream(is, programOptions.numHashFunctions);

                std::cout << "Loaded " << loadedMaps << " hash tables from " << programOptions.load_hashtables_from << std::endl;
            }else{
                gpuMinhasher->constructFromReadStorage(
                    programOptions,
                    gpuReadStorage.getNumberOfReads(), 
                    gpuReadStorage
                );
            }

            return gpuMinhasher;
        }
        #endif
}
}