#include "hip/hip_runtime.h"
#include <gpu/minhashkernels.hpp>
#include <gpu/nvtxtimelinemarkers.hpp>

#include <hpc_helpers.cuh>
#include <config.hpp>


#include <nvToolsExt.h>

#include <hipcub/hipcub.hpp>

#include <algorithm>
#include <cassert>
#include <vector>


namespace care{










template<int blocksize>
__global__
void compactDataOfUniqueRanges(
        read_number* __restrict__ output,
        const read_number* __restrict__ input,
        const int* __restrict__ sizesOfUniqueRangesPrefixsum,
        const int* __restrict__ rangesPerSequenceBegins,
        int numSequences){

    for(int sequenceIndex = blockIdx.x; sequenceIndex < numSequences; sequenceIndex += gridDim.x){

        const read_number* blockinput = input + rangesPerSequenceBegins[sequenceIndex];
        read_number* blockoutput = output + sizesOfUniqueRangesPrefixsum[sequenceIndex];
        const int numElements = sizesOfUniqueRangesPrefixsum[sequenceIndex + 1] - sizesOfUniqueRangesPrefixsum[sequenceIndex];

        for(int index = threadIdx.x; index < numElements; index += blocksize){
            blockoutput[index] = blockinput[index];
        }
    }

}


template<int blocksize, int numtempregs>
__global__
void makeUniqueRangesKernel(
        read_number* __restrict__ inoutData, 
        int* __restrict__ sizesOfUniqueRanges, 
        int numSequences,
        const read_number* __restrict__ anchorIds,
        const int* __restrict__ rangesPerSequenceBegins,
        int globalOffset){

    using BlockRadixSort = hipcub::BlockRadixSort<read_number, blocksize, numtempregs>;
    using BlockLoad = hipcub::BlockLoad<read_number, blocksize, numtempregs, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
    using BlockDiscontinuity = hipcub::BlockDiscontinuity<read_number, blocksize>;
    using BlockScan = hipcub::BlockScan<int, blocksize>; 

    __shared__ union{
        typename BlockRadixSort::TempStorage sort;
        typename BlockLoad::TempStorage load;
        typename BlockDiscontinuity::TempStorage discontinuity;
        typename BlockScan::TempStorage scan;
    } temp_storage;

    __shared__ read_number anchorId;

    for(int sequenceIndex = blockIdx.x; sequenceIndex < numSequences; sequenceIndex += gridDim.x){

        read_number tempregs[numtempregs];   

        #pragma unroll
        for(int i = 0; i <numtempregs; i++){
            tempregs[i] = std::numeric_limits<read_number>::max();
        }

        const int sizeOfRange = rangesPerSequenceBegins[sequenceIndex + 1] - rangesPerSequenceBegins[sequenceIndex];
        if(sizeOfRange == 0){
            if(threadIdx.x == 0){
                sizesOfUniqueRanges[sequenceIndex] = 0;
            }
        }else{
        
            read_number* const myRange = inoutData + rangesPerSequenceBegins[sequenceIndex] - globalOffset;

            assert(sizeOfRange <= numtempregs * blocksize);            

            BlockLoad(temp_storage.load).Load(
                myRange, 
                tempregs, 
                sizeOfRange
            );

            if(threadIdx.x == 0){
                anchorId = anchorIds[sequenceIndex];
            }

            __syncthreads();

            BlockRadixSort(temp_storage.sort).Sort(tempregs);

            __syncthreads();

            int head_flags[numtempregs];

            BlockDiscontinuity(temp_storage.discontinuity).FlagHeads(
                head_flags, 
                tempregs, 
                hipcub::Inequality()
            );

            __syncthreads();            

            #pragma unroll
            for(int i = 0; i < numtempregs; i++){
                if(threadIdx.x * numtempregs + i >= sizeOfRange){
                    head_flags[i] = 0;
                }else{
                    if(tempregs[i] == anchorId){
                        head_flags[i] = 0;
                    }
                }
            }

            int prefixsum[numtempregs];
            int numberOfSetHeadFlags = 0;

            BlockScan(temp_storage.scan).ExclusiveSum(head_flags, prefixsum, numberOfSetHeadFlags);

            __syncthreads();

            #pragma unroll
            for(int i = 0; i < numtempregs; i++){
                if(threadIdx.x * numtempregs + i < sizeOfRange && head_flags[i] == 1){
                    myRange[prefixsum[i]] = tempregs[i];
                }
            }

            if(threadIdx.x == 0){
                sizesOfUniqueRanges[sequenceIndex] = numberOfSetHeadFlags;
            }
        }
    }

}



template<int blocksize, int numtempregs>
__device__
void makeUniqueRangeSingleWarp(
        read_number* __restrict__ myRange, 
        int sizeOfRange, 
        int* __restrict__ sizeOfUniqueRange,
        typename hipcub::BlockRadixSort<read_number, blocksize, numtempregs>::TempStorage& sorttemp,
        typename hipcub::BlockLoad<read_number, blocksize, numtempregs, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>::TempStorage& loadtemp){

    using BlockRadixSort = hipcub::BlockRadixSort<read_number, blocksize, numtempregs>;
    using BlockLoad = hipcub::BlockLoad<read_number, blocksize, numtempregs, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;


    read_number tempregs[numtempregs];   

    #pragma unroll
    for(int i = 0; i <numtempregs; i++){
        tempregs[i] = std::numeric_limits<read_number>::max();
    }

    BlockLoad(loadtemp).Load(
        myRange, 
        tempregs, 
        sizeOfRange
    );

    __syncthreads();

    BlockRadixSort(sorttemp).SortBlockedToStriped(tempregs);

    int numUniqueElements = 0;
 
    #pragma unroll
    for(int i = 0; i < numtempregs; i++){
        const read_number curElement = tempregs[i];

        read_number nextElement = threadIdx.x == 0 ? tempregs[(i+1) % numtempregs] : tempregs[i];
        nextElement = __shfl_sync(0xFFFFFFFF, nextElement, threadIdx.x+1);               

        //find elements which are not equal to their right neighbor and not out of range
        const bool predicate = (curElement != nextElement) && (i * blocksize + threadIdx.x < sizeOfRange);

        const uint32_t mask = __ballot_sync(0xFFFFFFFF, predicate);
        const uint32_t count = __popc(mask);

        //get position
        const uint32_t numPre = __popc(mask & ((1 << threadIdx.x) -1));
        const int position = numUniqueElements + numPre;

        if(predicate){
            myRange[position] = curElement;
        }
        numUniqueElements += count;
    }
    
    if(threadIdx.x == 0){
        *sizeOfUniqueRange = numUniqueElements;
    }
}


template<int blocksize, int numtempregs>
__global__
void makeUniqueRangesKernelWithIntrinsicsSingleWarp(
        read_number* __restrict__ inoutData, 
        int* __restrict__ sizesOfUniqueRanges, 
        int numSequences,
        const int* __restrict__ rangesPerSequenceBegins,
        int globalOffset){

    static_assert(blocksize == 32, "blocksize must be 32 for SingleWarp kernel");

    using BlockRadixSort = hipcub::BlockRadixSort<read_number, blocksize, numtempregs>;
    using BlockLoad = hipcub::BlockLoad<read_number, blocksize, numtempregs, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
    
    __shared__ union{
        typename BlockRadixSort::TempStorage sort;
        typename BlockLoad::TempStorage load;
    } temp_storage;

    for(int sequenceIndex = blockIdx.x; sequenceIndex < numSequences; sequenceIndex += gridDim.x){

        const int sizeOfRange = rangesPerSequenceBegins[sequenceIndex + 1] - rangesPerSequenceBegins[sequenceIndex];
        assert(sizeOfRange <= numtempregs * blocksize);

        if(sizeOfRange == 0){
            if(threadIdx.x == 0){
                sizesOfUniqueRanges[sequenceIndex] = 0;
            }
        }else{
        
            read_number* const myRange = inoutData + rangesPerSequenceBegins[sequenceIndex] - globalOffset;

            makeUniqueRangeSingleWarp<blocksize, numtempregs>(
                myRange, 
                sizeOfRange, 
                &sizesOfUniqueRanges[sequenceIndex],
                temp_storage.sort,
                temp_storage.load
            );
        }
    }

}





template<int blocksize, int numtempregs>
__global__
void makeUniqueRangesKernelWithIntrinsicsSingleWarpChunked(
        read_number* __restrict__ inoutData, 
        int* __restrict__ sizesOfUniqueRanges, 
        int numSequences,
        const int* __restrict__ rangesPerSequenceBegins,
        int globalOffset){

    static_assert(blocksize == 32, "blocksize must be 32 for SingleWarp kernel");

    using BlockRadixSortFull = hipcub::BlockRadixSort<read_number, blocksize, numtempregs>;
    using BlockLoadFull = hipcub::BlockLoad<read_number, blocksize, numtempregs, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;

    using BlockRadixSort1 = hipcub::BlockRadixSort<read_number, blocksize, 1>;
    using BlockLoad1 = hipcub::BlockLoad<read_number, blocksize, 1, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;

    using BlockRadixSort2 = hipcub::BlockRadixSort<read_number, blocksize, 2>;
    using BlockLoad2 = hipcub::BlockLoad<read_number, blocksize, 2, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;

    using BlockRadixSort4 = hipcub::BlockRadixSort<read_number, blocksize, 4>;
    using BlockLoad4 = hipcub::BlockLoad<read_number, blocksize, 4, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;

    using BlockRadixSort8 = hipcub::BlockRadixSort<read_number, blocksize, 8>;
    using BlockLoad8 = hipcub::BlockLoad<read_number, blocksize, 8, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;

    using BlockRadixSort16 = hipcub::BlockRadixSort<read_number, blocksize, 16>;
    using BlockLoad16 = hipcub::BlockLoad<read_number, blocksize, 16, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;

    using BlockRadixSort32 = hipcub::BlockRadixSort<read_number, blocksize, 32>;
    using BlockLoad32 = hipcub::BlockLoad<read_number, blocksize, 32, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;

    using BlockRadixSort64 = hipcub::BlockRadixSort<read_number, blocksize, 64>;
    using BlockLoad64 = hipcub::BlockLoad<read_number, blocksize, 64, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
    
    __shared__ union{
        typename BlockRadixSortFull::TempStorage sortFull;
        typename BlockLoadFull::TempStorage loadFull;

        typename BlockRadixSort1::TempStorage sort1;
        typename BlockLoad1::TempStorage load1;

        typename BlockRadixSort2::TempStorage sort2;
        typename BlockLoad2::TempStorage load2;

        typename BlockRadixSort4::TempStorage sort4;
        typename BlockLoad4::TempStorage load4;

        typename BlockRadixSort8::TempStorage sort8;
        typename BlockLoad8::TempStorage load8;

        typename BlockRadixSort16::TempStorage sort16;
        typename BlockLoad16::TempStorage load16;

        typename BlockRadixSort32::TempStorage sort32;
        typename BlockLoad32::TempStorage load32;

        typename BlockRadixSort64::TempStorage sort64;
        typename BlockLoad64::TempStorage load64;
    } temp_storage;

    for(int sequenceIndex = blockIdx.x; sequenceIndex < numSequences; sequenceIndex += gridDim.x){

        const int sizeOfRange = rangesPerSequenceBegins[sequenceIndex + 1] - rangesPerSequenceBegins[sequenceIndex];
        assert(sizeOfRange <= numtempregs * blocksize);

        if(sizeOfRange == 0){
            if(threadIdx.x == 0){
                sizesOfUniqueRanges[sequenceIndex] = 0;
            }
        }else{
            read_number* const myRange = inoutData + rangesPerSequenceBegins[sequenceIndex] - globalOffset;
            int* const sizeOfUniqueRange = sizesOfUniqueRanges + sequenceIndex;

            if(sizeOfRange <= blocksize * 1){
                makeUniqueRangeSingleWarp<blocksize, 1>(
                    myRange, 
                    sizeOfRange, 
                    sizeOfUniqueRange,
                    temp_storage.sort1,
                    temp_storage.load1
                );
            }else if(sizeOfRange <= blocksize * 2){
                makeUniqueRangeSingleWarp<blocksize, 2>(
                    myRange, 
                    sizeOfRange, 
                    sizeOfUniqueRange,
                    temp_storage.sort2,
                    temp_storage.load2
                );
            }else if(sizeOfRange <= blocksize * 4){
                makeUniqueRangeSingleWarp<blocksize, 4>(
                    myRange, 
                    sizeOfRange, 
                    sizeOfUniqueRange,
                    temp_storage.sort4,
                    temp_storage.load4
                );
            }else if(sizeOfRange <= blocksize * 8){
                makeUniqueRangeSingleWarp<blocksize, 8>(
                    myRange, 
                    sizeOfRange, 
                    sizeOfUniqueRange,
                    temp_storage.sort8,
                    temp_storage.load8
                );
            }else if(sizeOfRange <= blocksize * 16){
                makeUniqueRangeSingleWarp<blocksize, 16>(
                    myRange, 
                    sizeOfRange, 
                    sizeOfUniqueRange,
                    temp_storage.sort16,
                    temp_storage.load16
                );
            }else if(sizeOfRange <= blocksize * 32){
                makeUniqueRangeSingleWarp<blocksize, 32>(
                    myRange, 
                    sizeOfRange, 
                    sizeOfUniqueRange,
                    temp_storage.sort32,
                    temp_storage.load32
                );
            }else if(sizeOfRange <= blocksize * 64){
                makeUniqueRangeSingleWarp<blocksize, 64>(
                    myRange, 
                    sizeOfRange, 
                    sizeOfUniqueRange,
                    temp_storage.sort64,
                    temp_storage.load64
                );
            }else{
                makeUniqueRangeSingleWarp<blocksize, numtempregs>(
                    myRange, 
                    sizeOfRange, 
                    sizeOfUniqueRange,
                    temp_storage.sortFull,
                    temp_storage.loadFull
                );
            }

            
        }
    }

}




template<int blocksize, int numtempregs>
__global__
void makeUniqueRangesKernelWithIntrinsicsMultiWarp(
        read_number* __restrict__ inoutData, 
        int* __restrict__ sizesOfUniqueRanges, 
        int numSequences,
        const int* __restrict__ rangesPerSequenceBegins,
        int globalOffset){

    using BlockRadixSort = hipcub::BlockRadixSort<read_number, blocksize, numtempregs>;
    using BlockLoad = hipcub::BlockLoad<read_number, blocksize, numtempregs, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
    using WarpScan = hipcub::WarpScan<int>;
    
    constexpr int warpsize = 32;
    constexpr int numWarpsPerBlock = blocksize / warpsize;
    static_assert(blocksize % warpsize == 0, "blocksize not multiple of warpsize");

    __shared__ union{
        typename BlockRadixSort::TempStorage sort;
        typename BlockLoad::TempStorage load;
        typename WarpScan::TempStorage warpscan[numWarpsPerBlock];  
    } temp_storage;

    __shared__ read_number rightNeighborPerWarp[numWarpsPerBlock];
    __shared__ int countsPerWarp[numWarpsPerBlock];
    __shared__ int countsPrefixsum[numWarpsPerBlock+1];

    const int warpId = threadIdx.x / warpsize;
    const int laneId = threadIdx.x % warpsize;

    for(int sequenceIndex = blockIdx.x; sequenceIndex < numSequences; sequenceIndex += gridDim.x){

        read_number tempregs[numtempregs];   

        #pragma unroll
        for(int i = 0; i <numtempregs; i++){
            tempregs[i] = std::numeric_limits<read_number>::max();
        }

        const int sizeOfRange = rangesPerSequenceBegins[sequenceIndex + 1] - rangesPerSequenceBegins[sequenceIndex];
        if(sizeOfRange == 0){
            if(threadIdx.x == 0){
                sizesOfUniqueRanges[sequenceIndex] = 0;
            }
        }else{
        
            read_number* const myRange = inoutData + rangesPerSequenceBegins[sequenceIndex] - globalOffset;

            assert(sizeOfRange <= numtempregs * blocksize);

            BlockLoad(temp_storage.load).Load(
                myRange, 
                tempregs, 
                sizeOfRange
            );

            __syncthreads();

            BlockRadixSort(temp_storage.sort).SortBlockedToStriped(tempregs);

            int numUniqueElements = 0;
            #pragma unroll
            for(int i = 0; i < numtempregs; i++){                

                //save input for previous warp in shared memory
                if(laneId == 0){
                    if(warpId == 0){
                        rightNeighborPerWarp[numWarpsPerBlock-1] = tempregs[(i+1) % numtempregs];
                    }else{
                        rightNeighborPerWarp[warpId-1] = tempregs[i];
                    }
                    countsPerWarp[warpId] = 0;
                }
                __syncthreads();

                const read_number curElement = tempregs[i];

                read_number nextElement = laneId == 0 ? rightNeighborPerWarp[warpId] : tempregs[i];
                nextElement = __shfl_sync(0xFFFFFFFF, nextElement, laneId+1);               

                //find elements which are not equal to their right neighbor and not out of range
                const bool predicate = (curElement != nextElement) && (i * blocksize + threadIdx.x < sizeOfRange);

                const uint32_t mask = __ballot_sync(0xFFFFFFFF, predicate);
                const int count = __popc(mask);

                if(laneId == 0){
                    countsPerWarp[warpId] = count;
                }
                __syncthreads();

                if(warpId == 0){
                    int c = laneId < numWarpsPerBlock ? countsPerWarp[laneId] : 0;
                    int warp_aggregate = 0;
                    WarpScan(temp_storage.warpscan[warpId]).ExclusiveSum(c, c, warp_aggregate);
                    //__syncwarp();

                    if(laneId < numWarpsPerBlock){
                        countsPrefixsum[laneId] = c;
                    }
                    if(laneId == 0){
                        countsPrefixsum[numWarpsPerBlock] = warp_aggregate;
                    }
                }

                const int numPre = __popc(mask & ((1 << laneId) -1));

                __syncthreads();

                //get position                
                const int position = numUniqueElements + countsPrefixsum[warpId] + numPre;

                if(predicate){
                    myRange[position] = curElement;
                }
                numUniqueElements += countsPrefixsum[numWarpsPerBlock];
            }

            if(threadIdx.x == 0){
                sizesOfUniqueRanges[sequenceIndex] = numUniqueElements;
            }
        }
    }

}









void makeCompactUniqueRangesGmem(
        MergeRangesGpuHandle<read_number>& handle, 
        const std::pair<const read_number*, const read_number*>* ranges,
        int numRanges, 
        const read_number* d_anchorIds, 
        int rangesPerSequence, 
        int totalNumElements, 
        bool onlyAlloc,
        hipStream_t stream){

    assert(false && "cannot use gmem merge currently");

    const int numSequences = numRanges / rangesPerSequence;
    if(numSequences == 0){
        return;
    }
    
    
    {
        size_t temp_storage_bytes = 0;
        size_t temp_storage_bytes2 = 0;

        hipcub::DeviceSegmentedRadixSort::SortKeys(
            nullptr, 
            temp_storage_bytes, 
            handle.d_data.get(), 
            handle.d_results.get(),
            totalNumElements, 
            numSequences, 
            handle.d_rangesBeginPerSequence.get(), 
            handle.d_rangesBeginPerSequence.get() + 1,
            0,
            sizeof(read_number) * 8,
            stream
        );

        using CountIter = hipcub::CountingInputIterator<int>;
        read_number* d_data = handle.d_data.get();
        auto toHeadFlag = [=] __device__ (int index){
            if(index == 0 || d_data[index-1] != d_data[index]){
                return 1;
            }else{
                return 0;
            }
        };

        using TransformOp = decltype(toHeadFlag);
        using HeadFlagsIter = hipcub::TransformInputIterator<int, TransformOp, CountIter>;

        HeadFlagsIter head_flags(CountIter{0}, toHeadFlag);

        hipcub::DeviceSegmentedReduce::Sum(
            nullptr, 
            temp_storage_bytes2, 
            head_flags, 
            handle.d_uniqueRangeLengths.get(),
            numSequences, 
            handle.d_rangesBeginPerSequence.get(), 
            handle.d_rangesBeginPerSequence.get() + 1,
            stream
        );

        temp_storage_bytes = std::max(temp_storage_bytes, temp_storage_bytes2);
    
        hipcub::DeviceScan::InclusiveSum(
            nullptr, 
            temp_storage_bytes2, 
            handle.d_uniqueRangeLengths.get(), 
            handle.d_uniqueRangeLengthsPrefixsum.get() + 1, 
            numSequences,
            stream
        );

        temp_storage_bytes = std::max(temp_storage_bytes, temp_storage_bytes2);
    
        hipcub::DeviceSelect::Flagged(
            nullptr, 
            temp_storage_bytes2,
            handle.d_data.get(), 
            head_flags, 
            handle.d_results.get(), 
            hipcub::DiscardOutputIterator<int>{}, //d_num_selected_out
            totalNumElements,
            stream
        );

        temp_storage_bytes = std::max(temp_storage_bytes, temp_storage_bytes2);
        handle.cubTempStorage.resize(temp_storage_bytes);
    }

    if(onlyAlloc){
        return;
    }

    hipMemcpyAsync(
        handle.d_rangesBeginPerSequence.get(), 
        handle.h_rangesBeginPerSequence.get(),
        sizeof(int) * (numSequences+1),
        H2D,
        stream
    ); CUERR;

    //copy data of ranges into contiguous pinned memory, then to device async
    {
        auto begin = handle.h_data.get();
        auto end = handle.h_data.get();
        for(int rangeIndex = 0; rangeIndex < numRanges; rangeIndex++){
            end = std::copy(
                ranges[rangeIndex].first, 
                ranges[rangeIndex].second,
                begin
            );
            
            begin = end;
        }
    }

    hipMemcpyAsync(
        handle.d_data.get(), 
        handle.h_data.get(),
        sizeof(read_number) * totalNumElements,
        H2D,
        stream
    ); CUERR;

    size_t cubTempStorageSize = handle.cubTempStorage.sizeInBytes();

    hipcub::DeviceSegmentedRadixSort::SortKeys(
        handle.cubTempStorage.get(), 
        cubTempStorageSize, 
        handle.d_data.get(), 
        handle.d_results.get(),
        totalNumElements, 
        numSequences, 
        handle.d_rangesBeginPerSequence.get(), 
        handle.d_rangesBeginPerSequence.get() + 1,
        0,
        sizeof(read_number) * 8,
        stream
    );

    std::swap(handle.d_data, handle.d_results);

    using CountIter = hipcub::CountingInputIterator<int>;
    read_number* d_data = handle.d_data.get();
    auto toHeadFlag = [=] __device__ (int index){
        if(index == 0 || d_data[index-1] != d_data[index]){
            return 1;
        }else{
            return 0;
        }
    };

    using TransformOp = decltype(toHeadFlag);
    using HeadFlagsIter = hipcub::TransformInputIterator<int, TransformOp, CountIter>;


    HeadFlagsIter head_flags(CountIter{0}, toHeadFlag);

    hipcub::DeviceSegmentedReduce::Sum(
        handle.cubTempStorage.get(), 
        cubTempStorageSize, 
        head_flags, 
        handle.d_uniqueRangeLengths.get(),
        numSequences, 
        handle.d_rangesBeginPerSequence.get(), 
        handle.d_rangesBeginPerSequence.get() + 1,
        stream
    );

    hipcub::DeviceScan::InclusiveSum(
        handle.cubTempStorage.get(), 
        cubTempStorageSize, 
        handle.d_uniqueRangeLengths.get(), 
        handle.d_uniqueRangeLengthsPrefixsum.get() + 1, 
        numSequences,
        stream
    );

    hipcub::DeviceSelect::Flagged(
        handle.cubTempStorage.get(), 
        cubTempStorageSize,
        handle.d_data.get(), 
        head_flags, 
        handle.d_results.get(), 
        hipcub::DiscardOutputIterator<int>{}, //d_num_selected_out == uniqueRangelengthsPrefixSum[numSequences]
        totalNumElements,
        stream
    );

       
}



void makeCompactUniqueRangesSmem(
        MergeRangesGpuHandle<read_number>& handle, 
        read_number* d_compactUniqueCandidateIds,
        int* d_candidatesPerAnchor,
        int* d_candidatesPerAnchorPrefixSum,
        read_number* d_candidateIds,
        const std::pair<const read_number*, const read_number*>* ranges,
        int numRanges, 
        const read_number* d_anchorIds, 
        int rangesPerSequence, 
        int totalNumElements, 
        bool onlyAlloc,
        MergeRangesKernelType kernelType,
        hipStream_t stream){

    const int numSequences = numRanges / rangesPerSequence;
    if(numSequences == 0){
        return;
    }

    {
        size_t temp_storage_bytes = 0;

        hipcub::DeviceScan::InclusiveSum(
            nullptr, 
            temp_storage_bytes,
            d_candidatesPerAnchor,
            d_candidatesPerAnchorPrefixSum + 1,
            numSequences,
            stream
        ); CUERR;
    
        handle.cubTempStorage.resize(temp_storage_bytes);
    }

    if(onlyAlloc){
        return;
    }

    hipMemcpyAsync(
        handle.d_rangesBeginPerSequence.get(), 
        handle.h_rangesBeginPerSequence.get(),
        sizeof(int) * (numSequences+1),
        H2D,
        stream
    ); CUERR;

    hipEventRecord(handle.events.back(), stream); CUERR;

    for(auto& pipelinestream : handle.streams){
        hipStreamWaitEvent(pipelinestream, handle.events.back(), 0); CUERR;
    }

    {
        const int numstreams = handle.streams.size();

        int elementOffset = 0;
        int sequenceOffset = 0;

        for(int i = 0; i < numstreams; i++){
            const int sequenceBegin = i * numSequences / numstreams;
            const int sequenceEnd = std::min((i+1) * numSequences / numstreams, numSequences);

            const int mynumSequences = sequenceEnd - sequenceBegin;

            int mynumElements = 0;
            int largestNumElementsPerSequence = 0;
            for(int sequenceIndex = sequenceBegin; sequenceIndex < sequenceEnd; sequenceIndex++){
                
                int numElementsForSequence = 0;

                for(int k = 0; k < rangesPerSequence; k++){
                    const int rangeIndex = sequenceIndex * rangesPerSequence + k;    
                    const int sizeOfRange = std::distance(ranges[rangeIndex].first, ranges[rangeIndex].second);
                    numElementsForSequence += sizeOfRange;
                }

                largestNumElementsPerSequence = std::max(largestNumElementsPerSequence, numElementsForSequence);                
            } 

            #define processData(blocksize, numtempregs) \
            { \
                switch(kernelType){ \
                case MergeRangesKernelType::allcub: \
                    makeUniqueRangesKernel<(blocksize), (numtempregs)><<<mynumSequences, (blocksize), 0, handle.streams[i]>>>( \
                        d_candidateIds + elementOffset,  \
                        d_candidatesPerAnchor + sequenceOffset,  \
                        mynumSequences, \
                        d_anchorIds, \
                        handle.d_rangesBeginPerSequence.get() + sequenceOffset, \
                        elementOffset \
                    ); CUERR; \
                    break; \
                case MergeRangesKernelType::popcmultiwarp: \
                    makeUniqueRangesKernelWithIntrinsicsMultiWarp<(blocksize), (numtempregs)><<<mynumSequences, (blocksize), 0, handle.streams[i]>>>( \
                        d_candidateIds + elementOffset,  \
                        d_candidatesPerAnchor + sequenceOffset,  \
                        mynumSequences, \
                        handle.d_rangesBeginPerSequence.get() + sequenceOffset, \
                        elementOffset \
                    ); CUERR; \
                    break; \
                case MergeRangesKernelType::popcsinglewarp: \
                    makeUniqueRangesKernelWithIntrinsicsSingleWarp<32, 64><<<mynumSequences, 32, 0, handle.streams[i]>>>( \
                        d_candidateIds + elementOffset,  \
                        d_candidatesPerAnchor + sequenceOffset,  \
                        mynumSequences, \
                        handle.d_rangesBeginPerSequence.get() + sequenceOffset, \
                        elementOffset \
                    ); CUERR; \
                    break; \
                case MergeRangesKernelType::popcsinglewarpchunked: \
                    makeUniqueRangesKernelWithIntrinsicsSingleWarpChunked<32, 64><<<mynumSequences, 32, 0, handle.streams[i]>>>( \
                        d_candidateIds + elementOffset,  \
                        d_candidatesPerAnchor + sequenceOffset,  \
                        mynumSequences, \
                        handle.d_rangesBeginPerSequence.get() + sequenceOffset, \
                        elementOffset \
                    ); CUERR; \
                    break; \
                default: std::cerr << "unknown kernel type\n"; \
                } \
            }

            if(largestNumElementsPerSequence <= 32){
                constexpr int blocksize = 32;
                constexpr int numtempregs = 1;
                assert(largestNumElementsPerSequence <= blocksize * numtempregs);

                processData(blocksize, numtempregs);
            }else if(largestNumElementsPerSequence <= 64){
                constexpr int blocksize = 64;
                constexpr int numtempregs = 1;
                assert(largestNumElementsPerSequence <= blocksize * numtempregs);

                processData(blocksize, numtempregs);
            }else if(largestNumElementsPerSequence <= 96){
                constexpr int blocksize = 96;
                constexpr int numtempregs = 1;
                assert(largestNumElementsPerSequence <= blocksize * numtempregs);

                processData(blocksize, numtempregs);
            }else if(largestNumElementsPerSequence <= 128){
                constexpr int blocksize = 128;
                constexpr int numtempregs = 1;
                assert(largestNumElementsPerSequence <= blocksize * numtempregs);

                processData(blocksize, numtempregs);
            }else if(largestNumElementsPerSequence <= 256){
                constexpr int blocksize = 128;
                constexpr int numtempregs = 2;
                assert(largestNumElementsPerSequence <= blocksize * numtempregs);

                processData(blocksize, numtempregs);
            }else if(largestNumElementsPerSequence <= 512){
                constexpr int blocksize = 128;
                constexpr int numtempregs = 4;
                assert(largestNumElementsPerSequence <= blocksize * numtempregs);

                processData(blocksize, numtempregs);
            }else if(largestNumElementsPerSequence <= 1024){
                constexpr int blocksize = 128;
                constexpr int numtempregs = 8;
                assert(largestNumElementsPerSequence <= blocksize * numtempregs);

                processData(blocksize, numtempregs);
            }else if(largestNumElementsPerSequence <= 2048){
                constexpr int blocksize = 128;
                constexpr int numtempregs = 16;
                assert(largestNumElementsPerSequence <= blocksize * numtempregs);

                processData(blocksize, numtempregs);
            }else if(largestNumElementsPerSequence <= 4096){
                constexpr int blocksize = 128;
                constexpr int numtempregs = 32;
                assert(largestNumElementsPerSequence <= blocksize * numtempregs);

                processData(blocksize, numtempregs);
            }else{
                constexpr int blocksize = 128;
                constexpr int numtempregs = 64;
                assert(largestNumElementsPerSequence <= blocksize * numtempregs);
                
                processData(blocksize, numtempregs);
            }


            hipEventRecord(handle.events[i], handle.streams[i]); CUERR;
            hipStreamWaitEvent(stream, handle.events[i], 0); CUERR;

            elementOffset += mynumElements;
            sequenceOffset += mynumSequences;
        }
        
    }    

    size_t tmpsize = handle.cubTempStorage.sizeInBytes();

    hipcub::DeviceScan::InclusiveSum(
        handle.cubTempStorage.get(), 
        tmpsize,
        d_candidatesPerAnchor,
        d_candidatesPerAnchorPrefixSum + 1,
        numSequences,
        stream
    ); CUERR;

    compactDataOfUniqueRanges<256><<<numSequences, 256, 0, stream>>>(
        d_compactUniqueCandidateIds,
        d_candidateIds,
        d_candidatesPerAnchorPrefixSum,
        handle.d_rangesBeginPerSequence.get(),
        numSequences
    ); CUERR;

    //hipEventRecord(handle.events.back(), stream); CUERR;   

}



void mergeRangesGpuAsync(
        MergeRangesGpuHandle<read_number>& handle, 
        read_number* d_compactUniqueCandidateIds,
        int* d_candidatesPerAnchor,
        int* d_candidatesPerAnchorPrefixSum,
        read_number* d_candidateIds,
        const std::pair<const read_number*, const read_number*>* h_ranges, 
        int numRanges, 
        const read_number* d_anchorIds, 
        int rangesPerSequence, 
        hipStream_t stream,
        MergeRangesKernelType kernelType){
    
    const int numSequences = numRanges / rangesPerSequence;
    if(numSequences == 0){
        return;
    }

    handle.d_rangesBeginPerSequence.resize(numSequences+1);
    handle.h_rangesBeginPerSequence.resize(numSequences+1);

    handle.h_rangesBeginPerSequence[0] = 0;

    //int longestRange = 0;

    //nvtx::push_range("longestrange", 4);
    int maxNumResults = 0;
    for(int i = 0; i < numSequences; i++){   
        int rangeOfSequence = 0;     
        for(int k = 0; k < rangesPerSequence; k++){
            const int rangeIndex = i * rangesPerSequence + k;
            maxNumResults += std::distance(h_ranges[rangeIndex].first, h_ranges[rangeIndex].second);
            //rangeOfSequence += std::distance(h_ranges[rangeIndex].first, h_ranges[rangeIndex].second);
        }
        //longestRange = std::max(longestRange, rangeOfSequence);
        handle.h_rangesBeginPerSequence[i+1] = maxNumResults;
    }
    //nvtx::pop_range();

    //std::cerr << "longestRange = " << longestRange << "\n";
    handle.d_data.resize(maxNumResults);
    handle.h_data.resize(maxNumResults);

    handle.d_results.resize(maxNumResults);
    handle.h_results.resize(maxNumResults);

    handle.h_numresults.resize(1);

    handle.d_uniqueRangeLengths.resize(numSequences);
    handle.h_uniqueRangeLengths.resize(numSequences);

    handle.d_uniqueRangeLengthsPrefixsum.resize(numSequences + 1);
    hipMemsetAsync(handle.d_uniqueRangeLengthsPrefixsum, 0, sizeof(int), stream); CUERR;


    //TIMERSTARTCPU(makeCompactUniqueRanges);

    if(kernelType == MergeRangesKernelType::devicewide){
        makeCompactUniqueRangesGmem(
            handle, 
            h_ranges,
            numRanges, 
            d_anchorIds,
            rangesPerSequence, 
            maxNumResults, 
            false,
            stream
        );
        
    }else{
        makeCompactUniqueRangesSmem(
            handle, 
            d_compactUniqueCandidateIds,
            d_candidatesPerAnchor,
            d_candidatesPerAnchorPrefixSum,
            d_candidateIds,
            h_ranges,
            numRanges, 
            d_anchorIds,
            rangesPerSequence, 
            maxNumResults, 
            false,
            kernelType,
            stream
        );
    }

}



OperationResult mergeRangesGpu(
        MergeRangesGpuHandle<read_number>& handle, 
        const std::pair<const read_number*, const read_number*>* h_ranges, 
        int numRanges, 
        const read_number* d_anchorIds, 
        int rangesPerSequence, 
        hipStream_t stream,
        MergeRangesKernelType kernelType){
    
    const int numSequences = numRanges / rangesPerSequence;
    if(numSequences == 0){
        return OperationResult{};
    }

    // mergeRangesGpuAsync(
    //     handle, 
    //     h_ranges, 
    //     numRanges, 
    //     d_anchorIds, 
    //     rangesPerSequence, 
    //     stream,
    //     kernelType
    // );

    hipMemcpyAsync(
        handle.h_uniqueRangeLengths.get(), 
        handle.d_uniqueRangeLengths.get(), 
        sizeof(int) * numSequences, 
        D2H, 
        stream
    ); CUERR;
    hipMemcpyAsync(
        handle.h_numresults.get(), 
        handle.d_uniqueRangeLengthsPrefixsum.get() + numSequences, 
        sizeof(int), 
        D2H, 
        stream
    ); CUERR;
    hipMemcpyAsync(
        handle.h_results.get(), 
        handle.d_results.get(), 
        sizeof(read_number) * handle.d_results.size(), 
        D2H, 
        stream
    ); CUERR; 


    hipStreamSynchronize(stream); CUERR;

    //TIMERSTOPCPU(makeCompactUniqueRanges);

    OperationResult result;

    result.candidateIds.clear();
    result.candidateIds.resize(*handle.h_numresults.get());

    result.candidateIdsPerSequence.clear();
    result.candidateIdsPerSequence.resize(numSequences, 0);

    std::copy_n(
        handle.h_results.get(), 
        *handle.h_numresults.get(), 
        result.candidateIds.begin()
    );

    std::copy_n(
        handle.h_uniqueRangeLengths.get(), 
        numSequences, 
        result.candidateIdsPerSequence.begin()
    );

    return result;

}



} //namespace care