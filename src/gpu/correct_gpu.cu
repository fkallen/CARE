#include "hip/hip_runtime.h"




#include <gpu/gpucorrector.cuh>
#include <gpu/gpureadstorage.cuh>

#include <gpu/gpuminhasher.cuh>

#include <options.hpp>
#include <readlibraryio.hpp>
#include <memorymanagement.hpp>
#include <memoryfile.hpp>
#include <threadpool.hpp>
#include <rangegenerator.hpp>
#include <concurrencyhelpers.hpp>
#include <corrector.hpp>
#include <corrector_common.hpp>

#include <classification.hpp>
#include <forest_gpu.cuh>


#include <cassert>
#include <cstdint>
#include <iostream>
#include <string>
#include <vector>
#include <future>

namespace care{
namespace gpu{


#if 0

//TODO classifier?
class SimpleCpuCorrectionPipeline{
    template<class T>
    using HostContainer = helpers::SimpleAllocationPinnedHost<T, 0>;

public:
    template<class ResultProcessor, class BatchCompletion>
    void runToCompletion(
        cpu::RangeGenerator<read_number>& readIdGenerator,
        const CorrectionOptions& correctionOptions,
        const GoodAlignmentProperties& goodAlignmentProperties,
        ReadCorrectionFlags& correctionFlags,
        ResultProcessor processResults,
        BatchCompletion batchCompleted
    ) const {
        assert(false);
#if 0                
        //const int threadId = omp_get_thread_num();

        const std::size_t encodedSequencePitchInInts2Bit = SequenceHelpers::getEncodedNumInts2Bit(gpuReadStorage->getSequenceLengthUpperBound());
        const std::size_t decodedSequencePitchInBytes = gpuReadStorage->getSequenceLengthUpperBound();
        const std::size_t qualityPitchInBytes = gpuReadStorage->getSequenceLengthUpperBound();

        CpuErrorCorrector errorCorrector(
            encodedSequencePitchInInts2Bit,
            decodedSequencePitchInBytes,
            qualityPitchInBytes,
            correctionOptions,
            goodAlignmentProperties,
            *candidateIdsProvider,
            *readProvider,
            correctionFlags
        );

        HostContainer<read_number> batchReadIds(correctionOptions.batchsize);
        HostContainer<unsigned int> batchEncodedData(correctionOptions.batchsize * encodedSequencePitchInInts2Bit);
        HostContainer<char> batchQualities(correctionOptions.batchsize * qualityPitchInBytes);
        HostContainer<int> batchReadLengths(correctionOptions.batchsize);

        std::vector<read_number> tmpids(correctionOptions.batchsize);

        while(!(readIdGenerator.empty())){
            tmpids.resize(correctionOptions.batchsize);            

            auto readIdsEnd = readIdGenerator.next_n_into_buffer(
                correctionOptions.batchsize, 
                tmpids.begin()
            );
            
            tmpids.erase(readIdsEnd, tmpids.end());

            if(tmpids.empty()){
                continue;
            }

            const int numAnchors = tmpids.size();

            batchReadIds.resize(numAnchors);
            std::copy(tmpids.begin(), tmpids.end(), batchReadIds.begin());

            //collect input data of all reads in batch
            readProvider->setReadIds(batchReadIds.data(), batchReadIds.size());

            readProvider->gatherSequenceLengths(
                batchReadLengths.data()
            );

            readProvider->gatherSequenceData(
                batchEncodedData.data(),
                encodedSequencePitchInInts2Bit
            );

            if(correctionOptions.useQualityScores){
                readProvider->gatherSequenceQualities(
                    batchQualities.data(),
                    qualityPitchInBytes
                );
            }

            CpuErrorCorrector::MultiCorrectionInput input;
            input.anchorLengths.insert(input.anchorLengths.end(), batchReadLengths.begin(), batchReadLengths.end());
            input.anchorReadIds.insert(input.anchorReadIds.end(), batchReadIds.begin(), batchReadIds.end());

            input.encodedAnchors.resize(numAnchors);            
            for(int i = 0; i < numAnchors; i++){
                input.encodedAnchors[i] = batchEncodedData.data() + encodedSequencePitchInInts2Bit * i;
            }

            if(correctionOptions.useQualityScores){
                input.anchorQualityscores.resize(numAnchors);
                for(int i = 0; i < numAnchors; i++){
                    input.anchorQualityscores[i] = batchQualities.data() + qualityPitchInBytes * i;
                }
            }
            
            auto errorCorrectorOutputVector = errorCorrector.processMulti(input);
            
            CorrectionOutput correctionOutput;

            for(auto& output : errorCorrectorOutputVector){
                if(output.hasAnchorCorrection){
                    correctionOutput.encodedAnchorCorrections.emplace_back(output.anchorCorrection.encode());
                    correctionOutput.anchorCorrections.emplace_back(std::move(output.anchorCorrection));
                }

                for(auto& tmp : output.candidateCorrections){
                    correctionOutput.encodedCandidateCorrections.emplace_back(tmp.encode());
                    correctionOutput.candidateCorrections.emplace_back(std::move(tmp));
                }
            }

            processResults(std::move(correctionOutput));

            batchCompleted(batchReadIds.size()); 
            
        } //while unprocessed reads exist loop end   
#endif
    }
};
#endif

template<class Minhasher>
class SimpleGpuCorrectionPipeline{    
    /*
        SimpleGpuCorrectionPipeline uses
        thread which is responsible for everything.
        Threadpool may be used for internal parallelization.
    */

    using AnchorHasher = GpuAnchorHasher;
public:
    struct RunStatistics{
        double hasherTimeAverage{};
        double correctorTimeAverage{};
        double outputconstructorTimeAverage{};
        MemoryUsage memoryInputData{};
        MemoryUsage memoryRawOutputData{};
        MemoryUsage memoryHasher{};
        MemoryUsage memoryCorrector{};
        MemoryUsage memoryOutputConstructor{};
    };

    SimpleGpuCorrectionPipeline(
        const GpuReadStorage& readStorage_,
        const Minhasher& minhasher_,
        ThreadPool* threadPool_,
        ClfAgent* clfAgent_
    ) :
        readStorage(&readStorage_),
        minhasher(&minhasher_),
        threadPool(threadPool_),
        clfAgent(clfAgent_)
    {

    }

    template<class ResultProcessor, class BatchCompletion>
    RunStatistics runToCompletion(
        int deviceId,
        cpu::RangeGenerator<read_number>& readIdGenerator,
        const CorrectionOptions& correctionOptions,
        const GoodAlignmentProperties& goodAlignmentProperties,
        ReadCorrectionFlags& correctionFlags,
        ResultProcessor processResults,
        BatchCompletion batchCompleted
    ) const {

        auto continueCondition = [&](){ return !readIdGenerator.empty(); };

        return run_impl(
            deviceId,
            readIdGenerator,
            correctionOptions,
            goodAlignmentProperties,
            correctionFlags,
            processResults,
            batchCompleted,
            continueCondition
        );
    }

    template<class ResultProcessor, class BatchCompletion>
    RunStatistics runSomeBatches(
        int deviceId,
        cpu::RangeGenerator<read_number>& readIdGenerator,
        const CorrectionOptions& correctionOptions,
        const GoodAlignmentProperties& goodAlignmentProperties,
        ReadCorrectionFlags& correctionFlags,
        ResultProcessor processResults,
        BatchCompletion batchCompleted,
        int numBatches
    ) const {

        auto continueCondition = [&](){ bool success = !readIdGenerator.empty() && numBatches > 0; numBatches--; return success;};

        return run_impl(
            deviceId,
            readIdGenerator,
            correctionOptions,
            goodAlignmentProperties,
            correctionFlags,
            processResults,
            batchCompleted,
            continueCondition
        );
    }

    template<class ResultProcessor, class BatchCompletion>
    RunStatistics runToCompletionDoubleBuffered(
        int deviceId,
        cpu::RangeGenerator<read_number>& readIdGenerator,
        const CorrectionOptions& correctionOptions,
        const GoodAlignmentProperties& goodAlignmentProperties,
        ReadCorrectionFlags& correctionFlags,
        ResultProcessor processResults,
        BatchCompletion batchCompleted
    ) const {

        auto continueCondition = [&](){ return !readIdGenerator.empty(); };

        return runDoubleBuffered_impl(
            deviceId,
            readIdGenerator,
            correctionOptions,
            goodAlignmentProperties,
            correctionFlags,
            processResults,
            batchCompleted,
            continueCondition
        );
    }

    template<class ResultProcessor, class BatchCompletion>
    RunStatistics runSomeBatchesDoubleBuffered(
        int deviceId,
        cpu::RangeGenerator<read_number>& readIdGenerator,
        const CorrectionOptions& correctionOptions,
        const GoodAlignmentProperties& goodAlignmentProperties,
        ReadCorrectionFlags& correctionFlags,
        ResultProcessor processResults,
        BatchCompletion batchCompleted,
        int numBatches
    ) const {

        auto continueCondition = [&](){ bool success = !readIdGenerator.empty() && numBatches > 0; numBatches--; return success;};

        return runDoubleBuffered_impl(
            deviceId,
            readIdGenerator,
            correctionOptions,
            goodAlignmentProperties,
            correctionFlags,
            processResults,
            batchCompleted,
            continueCondition
        );
    }

    template<class ResultProcessor, class BatchCompletion, class ContinueCondition>
    RunStatistics runDoubleBuffered_impl(
        int deviceId,
        cpu::RangeGenerator<read_number>& readIdGenerator,
        const CorrectionOptions& correctionOptions,
        const GoodAlignmentProperties& goodAlignmentProperties,
        ReadCorrectionFlags& correctionFlags,
        ResultProcessor processResults,
        BatchCompletion batchCompleted,
        ContinueCondition continueCondition
    ) const {
        int cur = 0;
        hipGetDevice(&cur); CUERR;
        hipSetDevice(deviceId);

        constexpr int numextra = 1;

        CudaStream stream;
        GpuErrorCorrectorInput input;
        std::array<GpuErrorCorrectorRawOutput, 1 + numextra> rawOutputArray;
        std::queue<GpuErrorCorrectorRawOutput*> freeRawOutputQueue;
        std::queue<GpuErrorCorrectorRawOutput*> unprocessedRawOutputQueue;
        for(auto& a : rawOutputArray){
            freeRawOutputQueue.push(&a);
        }
        //GpuErrorCorrectorRawOutput rawOutput;

        hipError_t querystatus = input.event.query();
        if(querystatus != hipSuccess){
            std::cout << "CUDA error: " << hipGetErrorString(querystatus) << " : "
                << __FILE__ << ", line " << __LINE__ << std::endl;
        }
        assert(hipSuccess == querystatus);

        //ThreadPool::ParallelForHandle pforHandle;
        //ForLoopExecutor forLoopExecutor(threadPool, &pforHandle);
        SequentialForLoopExecutor forLoopExecutor;

        AnchorHasher gpuAnchorHasher(
            *readStorage,
            *minhasher,
            threadPool
        );

        GpuErrorCorrector gpuErrorCorrector{
            *readStorage,
            correctionFlags,
            correctionOptions,
            goodAlignmentProperties,
            correctionOptions.batchsize,
            threadPool,
            clfAgent
        };

        OutputConstructor outputConstructor(            
            correctionFlags,
            correctionOptions
        );

        RunStatistics runStatistics;

        std::vector<read_number> anchorIds(correctionOptions.batchsize);

        int iterations = 0;
        std::vector<double> elapsedHashingTimes;
        std::vector<double> elapsedCorrectionTimes;
        std::vector<double> elapsedOutputTimes;

        double elapsedHashingTime = 0.0;
        double elapsedCorrectionTime = 0.0;
        double elapsedOutputTime = 0.0;

        int globalcounter = 0;

        for(int i = 0; i < numextra; i++){
            if(continueCondition()){
                helpers::CpuTimer hashingTimer;
            
                anchorIds.resize(correctionOptions.batchsize);
                auto readIdsEnd = readIdGenerator.next_n_into_buffer(correctionOptions.batchsize, anchorIds.begin());
                anchorIds.erase(readIdsEnd, anchorIds.end());

                if(anchorIds.size() == 0){
                    continue;
                }

                //std::cerr << "globalcounter " << globalcounter << "\n";
        
                nvtx::push_range("makeErrorCorrectorInput", 0);
                gpuAnchorHasher.makeErrorCorrectorInput(
                    anchorIds.data(),
                    anchorIds.size(),
                    input,
                    stream
                );
                nvtx::pop_range();

                input.event.synchronize();

                globalcounter++;

                GpuErrorCorrectorRawOutput* rawOutputPtr = freeRawOutputQueue.front();
                freeRawOutputQueue.pop();

                hashingTimer.stop();
                //elapsedHashingTimes.emplace_back(hashingTimer.elapsed());
                elapsedHashingTime += hashingTimer.elapsed();

                nvtx::push_range("correct", 1);
                gpuErrorCorrector.correct(input, *rawOutputPtr, stream);
                nvtx::pop_range();

                unprocessedRawOutputQueue.push(rawOutputPtr);
            }
        }


        while(continueCondition()){

            
            
            anchorIds.resize(correctionOptions.batchsize);
            auto readIdsEnd = readIdGenerator.next_n_into_buffer(correctionOptions.batchsize, anchorIds.begin());
            anchorIds.erase(readIdsEnd, anchorIds.end());

            helpers::CpuTimer correctionTimer;

            if(anchorIds.size() > 0){
                helpers::CpuTimer hashingTimer;

                input.event.synchronize();

                //std::cerr << "globalcounter " << globalcounter << "\n";
        
                nvtx::push_range("makeErrorCorrectorInput", 0);
                gpuAnchorHasher.makeErrorCorrectorInput(
                    anchorIds.data(),
                    anchorIds.size(),
                    input,
                    stream
                );
                nvtx::pop_range();

                input.event.synchronize();

                //globalcounter++;

                hashingTimer.stop();
                //elapsedHashingTimes.emplace_back(hashingTimer.elapsed());
                elapsedHashingTime += hashingTimer.elapsed();

                GpuErrorCorrectorRawOutput* rawOutputPtr = freeRawOutputQueue.front();
                freeRawOutputQueue.pop();

                //helpers::CpuTimer correctionTimer;
                correctionTimer.reset();
                correctionTimer.start();

                nvtx::push_range("correct", 1);
                gpuErrorCorrector.correct(input, *rawOutputPtr, stream);
                nvtx::pop_range();

                unprocessedRawOutputQueue.push(rawOutputPtr);
            }

            if(unprocessedRawOutputQueue.size() > 0){

                GpuErrorCorrectorRawOutput* rawOutputPtr = unprocessedRawOutputQueue.front();
                unprocessedRawOutputQueue.pop();

                rawOutputPtr->event.synchronize();

                if(anchorIds.size() > 0){
                    correctionTimer.stop();
                    //elapsedCorrectionTimes.emplace_back(correctionTimer.elapsed());
                    elapsedCorrectionTime += correctionTimer.elapsed();
                }


                helpers::CpuTimer outputTimer;

                nvtx::push_range("constructResults", 2);
                auto correctionOutput = outputConstructor.constructResults(*rawOutputPtr, forLoopExecutor);
                nvtx::pop_range();

                freeRawOutputQueue.push(rawOutputPtr);


                nvtx::push_range("encodeResults", 3);

                correctionOutput.encode();

                nvtx::pop_range();

                outputTimer.stop();
                //elapsedOutputTimes.emplace_back(outputTimer.elapsed());
                elapsedOutputTime += outputTimer.elapsed();

                processResults(
                    std::move(correctionOutput)
                );

            }

            batchCompleted(anchorIds.size());

            iterations++;
        }

        //process remaining cached results
        while(unprocessedRawOutputQueue.size() > 0){
            GpuErrorCorrectorRawOutput* rawOutputPtr = unprocessedRawOutputQueue.front();
            unprocessedRawOutputQueue.pop();

            rawOutputPtr->event.synchronize();

            //correctionTimer.stop();
            //elapsedCorrectionTimes.emplace_back(correctionTimer.elapsed());
            //elapsedCorrectionTime += correctionTimer.elapsed();


            helpers::CpuTimer outputTimer;

            nvtx::push_range("constructResults", 2);
            auto correctionOutput = outputConstructor.constructResults(*rawOutputPtr, forLoopExecutor);
            nvtx::pop_range();

            freeRawOutputQueue.push(rawOutputPtr);

            nvtx::push_range("encodeResults", 3);

            correctionOutput.encode();

            nvtx::pop_range();

            outputTimer.stop();
            //elapsedOutputTimes.emplace_back(outputTimer.elapsed());
            elapsedOutputTime += outputTimer.elapsed();

            processResults(
                std::move(correctionOutput)
            );

            batchCompleted(*input.h_numAnchors.get());

            iterations++;
        }

        hipSetDevice(cur); CUERR;

        runStatistics.hasherTimeAverage = elapsedHashingTime / iterations;
        runStatistics.correctorTimeAverage = elapsedCorrectionTime / iterations;
        runStatistics.outputconstructorTimeAverage = elapsedOutputTime / iterations;
        runStatistics.memoryHasher = gpuAnchorHasher.getMemoryInfo();
        runStatistics.memoryCorrector = gpuErrorCorrector.getMemoryInfo();
        runStatistics.memoryOutputConstructor = outputConstructor.getMemoryInfo();
        runStatistics.memoryInputData = input.getMemoryInfo();
        //runStatistics.memoryRawOutputData = rawOutput.getMemoryInfo();

        return runStatistics;

        // std::cerr << "hashing times: ";
        // for(auto d : elapsedHashingTimes) std::cerr << d << ", ";
        // std::cerr << "\n";
        // //std::cerr << "Average: " << std::accumulate(elapsedHashingTimes.begin(), elapsedHashingTimes.end(), 0.0) / iterations << "\n";
        // std::cerr << "Average: " << elapsedHashingTime / iterations << "\n";

        // std::cerr << "correction times: ";
        // for(auto d : elapsedCorrectionTimes) std::cerr << d << ", ";
        // std::cerr << "\n";
        // //std::cerr << "Average: " << std::accumulate(elapsedCorrectionTimes.begin(), elapsedCorrectionTimes.end(), 0.0) / iterations << "\n";
        // std::cerr << "Average: " << elapsedCorrectionTime / iterations << "\n";

        // std::cerr << "output times: ";
        // for(auto d : elapsedOutputTimes) std::cerr << d << ", ";
        // std::cerr << "\n";
        // //std::cerr << "Average: " << std::accumulate(elapsedOutputTimes.begin(), elapsedOutputTimes.end(), 0.0) / iterations << "\n";
        // std::cerr << "Average: " << elapsedOutputTime / iterations << "\n";
    }

    template<class ResultProcessor, class BatchCompletion, class ContinueCondition>
    RunStatistics run_impl(
        int deviceId,
        cpu::RangeGenerator<read_number>& readIdGenerator,
        const CorrectionOptions& correctionOptions,
        const GoodAlignmentProperties& goodAlignmentProperties,
        ReadCorrectionFlags& correctionFlags,
        ResultProcessor processResults,
        BatchCompletion batchCompleted,
        ContinueCondition continueCondition
    ) const {
        int cur = 0;
        hipGetDevice(&cur); CUERR;
        hipSetDevice(deviceId);

        //constexpr int numextra = 1;

        CudaStream stream;
        GpuErrorCorrectorInput input;

        GpuErrorCorrectorRawOutput rawOutput;

        //ThreadPool::ParallelForHandle pforHandle;
        //ForLoopExecutor forLoopExecutor(threadPool, &pforHandle);
        SequentialForLoopExecutor forLoopExecutor;

        AnchorHasher gpuAnchorHasher(
            *readStorage,
            *minhasher,
            threadPool
        );

        GpuErrorCorrector gpuErrorCorrector{
            *readStorage,
            correctionFlags,
            correctionOptions,
            goodAlignmentProperties,
            correctionOptions.batchsize,
            threadPool,
            clfAgent
        };

        OutputConstructor outputConstructor(            
            correctionFlags,
            correctionOptions
        );

        RunStatistics runStatistics;

        std::vector<read_number> anchorIds(correctionOptions.batchsize);

        int iterations = 0;
        std::vector<double> elapsedHashingTimes;
        std::vector<double> elapsedCorrectionTimes;
        std::vector<double> elapsedOutputTimes;

        double elapsedHashingTime = 0.0;
        double elapsedCorrectionTime = 0.0;
        double elapsedOutputTime = 0.0;

        //int globalcounter = 0;

        while(continueCondition()){

            helpers::CpuTimer hashingTimer;
            
            anchorIds.resize(correctionOptions.batchsize);
            auto readIdsEnd = readIdGenerator.next_n_into_buffer(correctionOptions.batchsize, anchorIds.begin());
            anchorIds.erase(readIdsEnd, anchorIds.end());

            if(anchorIds.size() > 0){

                input.event.synchronize();

                //std::cerr << "globalcounter " << globalcounter << "\n";
        
                nvtx::push_range("makeErrorCorrectorInput", 0);
                gpuAnchorHasher.makeErrorCorrectorInput(
                    anchorIds.data(),
                    anchorIds.size(),
                    input,
                    stream
                );
                nvtx::pop_range();

                input.event.synchronize();

                //globalcounter++;

                hashingTimer.stop();
                //elapsedHashingTimes.emplace_back(hashingTimer.elapsed());
                if(iterations >= 10){
                    elapsedHashingTime += hashingTimer.elapsed();
                }

                helpers::CpuTimer correctionTimer;

                nvtx::push_range("correct", 1);
                gpuErrorCorrector.correct(input, rawOutput, stream);
                nvtx::pop_range();

                rawOutput.event.synchronize();

                correctionTimer.stop();
                //elapsedCorrectionTimes.emplace_back(correctionTimer.elapsed());
                if(iterations >= 10){
                    elapsedCorrectionTime += correctionTimer.elapsed();
                }

                helpers::CpuTimer outputTimer;

                nvtx::push_range("constructResults", 2);
                auto correctionOutput = outputConstructor.constructResults(rawOutput, forLoopExecutor);
                nvtx::pop_range();

                nvtx::push_range("encodeResults", 3);

                correctionOutput.encode();

                nvtx::pop_range();

                outputTimer.stop();
                //elapsedOutputTimes.emplace_back(outputTimer.elapsed());
                if(iterations >= 10){
                    elapsedOutputTime += outputTimer.elapsed();
                }

                processResults(
                    std::move(correctionOutput)
                );
            }

            batchCompleted(anchorIds.size());

            iterations++;
        }

        hipSetDevice(cur); CUERR;

        const int timediterations = std::max(1, iterations - 10);

        runStatistics.hasherTimeAverage = elapsedHashingTime / timediterations;
        runStatistics.correctorTimeAverage = elapsedCorrectionTime / timediterations;
        runStatistics.outputconstructorTimeAverage = elapsedOutputTime / timediterations;
        runStatistics.memoryHasher = gpuAnchorHasher.getMemoryInfo();
        runStatistics.memoryCorrector = gpuErrorCorrector.getMemoryInfo();
        runStatistics.memoryOutputConstructor = outputConstructor.getMemoryInfo();
        runStatistics.memoryInputData = input.getMemoryInfo();
        //runStatistics.memoryRawOutputData = rawOutput.getMemoryInfo();

        return runStatistics;

        // std::cerr << "hashing times: ";
        // for(auto d : elapsedHashingTimes) std::cerr << d << ", ";
        // std::cerr << "\n";
        // //std::cerr << "Average: " << std::accumulate(elapsedHashingTimes.begin(), elapsedHashingTimes.end(), 0.0) / iterations << "\n";
        // std::cerr << "Average: " << elapsedHashingTime / iterations << "\n";

        // std::cerr << "correction times: ";
        // for(auto d : elapsedCorrectionTimes) std::cerr << d << ", ";
        // std::cerr << "\n";
        // //std::cerr << "Average: " << std::accumulate(elapsedCorrectionTimes.begin(), elapsedCorrectionTimes.end(), 0.0) / iterations << "\n";
        // std::cerr << "Average: " << elapsedCorrectionTime / iterations << "\n";

        // std::cerr << "output times: ";
        // for(auto d : elapsedOutputTimes) std::cerr << d << ", ";
        // std::cerr << "\n";
        // //std::cerr << "Average: " << std::accumulate(elapsedOutputTimes.begin(), elapsedOutputTimes.end(), 0.0) / iterations << "\n";
        // std::cerr << "Average: " << elapsedOutputTime / iterations << "\n";
    }

private:
    const GpuReadStorage* readStorage;
    const Minhasher* minhasher;
    ThreadPool* threadPool;
    ClfAgent* clfAgent;
};


template<class Minhasher>
class ComplexGpuCorrectionPipeline{
    using AnchorHasher = GpuAnchorHasher;
public:
    struct Config{
        int numHashers;
        int numCorrectors;
        int numOutputConstructors;
    };

    ComplexGpuCorrectionPipeline(
        const GpuReadStorage& readStorage_,
        const Minhasher& minhasher_,
        ThreadPool* threadPool_,
        ClfAgent* clfAgent_ 
    ) :
        readStorage(&readStorage_),
        minhasher(&minhasher_),
        threadPool(threadPool_),
        clfAgent(clfAgent_)
    {

    }

    template<class ResultProcessor, class BatchCompletion>
    void run(
        int deviceId,
        const Config& config,
        cpu::RangeGenerator<read_number>& readIdGenerator,
        const CorrectionOptions& correctionOptions,
        const GoodAlignmentProperties& goodAlignmentProperties,
        ReadCorrectionFlags& correctionFlags,
        ResultProcessor processResults,
        BatchCompletion batchCompleted
    ){
        int curDevice = 0;
        hipGetDevice(&curDevice); CUERR;
        hipSetDevice(deviceId); CUERR;

        noMoreInputs = false;
        activeHasherThreads = config.numHashers;
        noMoreRawOutputs = false;
        activeCorrectorThreads = config.numCorrectors;
        currentConfig = config;

        bool combinedCorrectionAndOutputconstruction = config.numOutputConstructors == 0;

        if(combinedCorrectionAndOutputconstruction){

            int numBatches = config.numHashers + config.numCorrectors; // such that all hashers and all correctors could be busy simultaneously
            numBatches += config.numCorrectors; //double buffer in correctors

            int numInputBatches = config.numHashers + config.numCorrectors;
            numInputBatches += config.numCorrectors * getNumExtraBuffers();

            std::vector<GpuErrorCorrectorInput> inputs(numInputBatches);
            for(auto& i : inputs){
                freeInputs.push(&i);
            }

            //int numOutputBatches = config.numHashers + config.numCorrectors;
            //numOutputBatches += config.numCorrectors;

            // std::vector<GpuErrorCorrectorRawOutput> rawOutputs(numOutputBatches);
            // for(auto& i : rawOutputs){
            //     freeRawOutputs.push(&i);
            // }

            std::vector<std::future<void>> futures;

            for(int i = 0; i < config.numHashers; i++){
                futures.emplace_back(
                    std::async(
                        std::launch::async,
                        [&](){ 
                            hasherThreadFunction(deviceId, readIdGenerator, 
                                correctionOptions); 
                        }
                    )
                );
            }

            for(int i = 0; i < config.numCorrectors; i++){
                futures.emplace_back(
                    std::async(
                        std::launch::async,
                        [&](){ 
                            correctorThreadFunctionMultiBufferWithOutput(deviceId, correctionOptions, 
                                goodAlignmentProperties, 
                                correctionFlags,
                                processResults, batchCompleted);                          
                        }
                    )
                );
            }            

            for(auto& future : futures){
                future.wait();
            }

        }else{
            int numInputBatches = config.numHashers + config.numCorrectors; // such that all hashers and all correctors could be busy simultaneously

            std::vector<GpuErrorCorrectorInput> inputs(numInputBatches);
            for(auto& i : inputs){
                freeInputs.push(&i);
            }

            int numOutputBatches = config.numHashers + config.numCorrectors;

            std::vector<GpuErrorCorrectorRawOutput> rawOutputs(numOutputBatches);
            for(auto& i : rawOutputs){
                freeRawOutputs.push(&i);
            }

            std::vector<std::future<void>> futures;

            for(int i = 0; i < config.numHashers; i++){
                futures.emplace_back(
                    std::async(
                        std::launch::async,
                        [&](){ 
                            hasherThreadFunction(deviceId, readIdGenerator, 
                                correctionOptions); 
                        }
                    )
                );
            }

            for(int i = 0; i < config.numCorrectors; i++){
                futures.emplace_back(
                    std::async(
                        std::launch::async,
                        [&](){ 
                            correctorThreadFunction(
                                deviceId, 
                                correctionOptions, 
                                goodAlignmentProperties,
                                correctionFlags
                            );                          
                        }
                    )
                );
            }

            for(int i = 0; i < config.numOutputConstructors; i++){
                futures.emplace_back(
                    std::async(
                        std::launch::async,
                        [&](){ 
                            outputConstructorThreadFunction(correctionOptions, correctionFlags,
                                processResults, batchCompleted); 
                        }
                    )
                );
            }

            for(auto& future : futures){
                future.wait();
            }

        }

        // std::cerr << "input data sizes\n";
        // for(const auto& i : inputs){
        //     auto meminfo = i.getMemoryInfo();
        //     std::cerr << "host: " << meminfo.host << ", ";
        //     for(auto d : meminfo.device){
        //         std::cerr << "device " << d.first << ": " << d.second << " ";
        //     }
        //     std::cerr << "\n";
        // }

        // std::cerr << "output data sizes\n";
        // for(const auto& o : rawOutputs){
        //     auto meminfo = o.getMemoryInfo();
        //     std::cerr << "host: " << meminfo.host << ", ";
        //     for(auto d : meminfo.device){
        //         std::cerr << "device " << d.first << ": " << d.second << " ";
        //     }
        //     std::cerr << "\n";
        // }

        hipSetDevice(curDevice); CUERR;
    }
    

    void hasherThreadFunction(
        int deviceId,
        cpu::RangeGenerator<read_number>& readIdGenerator,
        const CorrectionOptions& correctionOptions
    ){
        hipSetDevice(deviceId);

        AnchorHasher gpuAnchorHasher(
            *readStorage,
            *minhasher,
            nullptr//threadPool
        );

        CudaStream hasherStream;
        ThreadPool::ParallelForHandle pforHandle;


        while(!readIdGenerator.empty()){
            hipStreamSynchronize(hasherStream);

            std::vector<read_number> anchorIds(correctionOptions.batchsize);

            auto readIdsEnd = readIdGenerator.next_n_into_buffer(correctionOptions.batchsize, anchorIds.begin());
            anchorIds.erase(readIdsEnd, anchorIds.end());

            nvtx::push_range("getFreeInput",1);
            GpuErrorCorrectorInput* const inputPtr = freeInputs.pop();
            nvtx::pop_range();

            assert(hipSuccess == inputPtr->event.query());

            nvtx::push_range("makeErrorCorrectorInput", 0);
            gpuAnchorHasher.makeErrorCorrectorInput(
                anchorIds.data(),
                anchorIds.size(),
                *inputPtr,
                hasherStream
            );
            nvtx::pop_range();

            inputPtr->event.synchronize();

            unprocessedInputs.push(inputPtr);
            
        }

        activeHasherThreads--;

        if(activeHasherThreads == 0){
            noMoreInputs = true;
        }

        hipStreamSynchronize(hasherStream);

        // std::cerr << "Hasher memory usage\n";
        // {
        //     auto meminfo = gpuAnchorHasher.getMemoryInfo();
        //     std::cerr << "host: " << meminfo.host << ", ";
        //     for(auto d : meminfo.device){
        //         std::cerr << "device " << d.first << ": " << d.second << " ";
        //     }
        //     std::cerr << "\n";
        // }
    };

    void correctorThreadFunction(
        int deviceId,
        const CorrectionOptions& correctionOptions,
        const GoodAlignmentProperties& goodAlignmentProperties,
        const ReadCorrectionFlags& correctionFlags
    ){
        hipSetDevice(deviceId);

        GpuErrorCorrector gpuErrorCorrector{
            *readStorage,
            correctionFlags,
            correctionOptions,
            goodAlignmentProperties,
            correctionOptions.batchsize,
            threadPool,
            clfAgent
        };

        CudaStream stream;

        GpuErrorCorrectorInput* inputPtr = unprocessedInputs.popOrDefault(
            [&](){
                return !noMoreInputs;  //if noMoreInputs, return nullptr
            },
            nullptr
        ); 

        while(inputPtr != nullptr){
            nvtx::push_range("getFreeRawOutput",1);
            GpuErrorCorrectorRawOutput* rawOutputPtr = freeRawOutputs.pop();
            nvtx::pop_range();

            hipError_t cstatus = hipSuccess;
            cstatus = inputPtr->event.query();
            if(cstatus != hipSuccess){
                std::cerr << hipGetErrorString(cstatus) << "\n";
                assert(false);
            }
            cstatus = rawOutputPtr->event.query();
            if(cstatus != hipSuccess){
                std::cerr << hipGetErrorString(cstatus) << "\n";
                assert(false);
            }

            hipStreamSynchronize(stream); CUERR;

            // assert(hipSuccess == inputPtr->event.query());
            // assert(hipSuccess == rawOutputPtr->event.query());

            nvtx::push_range("correct", 0);
            gpuErrorCorrector.correct(*inputPtr, *rawOutputPtr, stream);
            nvtx::pop_range();

            inputPtr->event.synchronize();
            freeInputs.push(inputPtr);

            //hipStreamSynchronize(stream);
            
            rawOutputPtr->event.synchronize();
            //std::cerr << "Synchronized output " << rawOutputPtr << "\n";
            unprocessedRawOutputs.push(rawOutputPtr);
        
            nvtx::push_range("getUnprocessedInput",2);
            inputPtr = unprocessedInputs.popOrDefault(
                [&](){
                    return !noMoreInputs;  //if noMoreInputs, return nullptr
                },
                nullptr
            ); 
            nvtx::pop_range();

        };

        activeCorrectorThreads--;

        if(activeCorrectorThreads == 0){
            noMoreRawOutputs = true;
        }

        hipStreamSynchronize(stream); CUERR;
    };

    static constexpr int getNumExtraBuffers() noexcept{
        return 1;
    }

    template<class ResultProcessor, class BatchCompletion>
    void correctorThreadFunctionMultiBufferWithOutput(
        int deviceId,
        const CorrectionOptions& correctionOptions,
        const GoodAlignmentProperties& goodAlignmentProperties,
        ReadCorrectionFlags& correctionFlags,
        ResultProcessor processResults,
        BatchCompletion batchCompleted
    ){
        hipSetDevice(deviceId);

        GpuErrorCorrector gpuErrorCorrector{
            *readStorage,
            correctionFlags,
            correctionOptions,
            goodAlignmentProperties,
            correctionOptions.batchsize,
            threadPool,
            clfAgent
        };

        OutputConstructor outputConstructor(            
            correctionFlags,
            correctionOptions
        );

        ThreadPool::ParallelForHandle pforHandle;
        //ForLoopExecutor forLoopExecutor(&threadPool, &pforHandle);
        SequentialForLoopExecutor forLoopExecutor;

        std::array<GpuErrorCorrectorRawOutput, 1 + getNumExtraBuffers()> rawOutputs{};
        std::queue<GpuErrorCorrectorRawOutput*> myFreeOutputsQueue;

        for(auto& i : rawOutputs){
            myFreeOutputsQueue.push(&i);
        }

        auto constructOutput = [&](GpuErrorCorrectorRawOutput* rawOutputPtr){
            nvtx::push_range("constructResults", 0);
            auto correctionOutput = outputConstructor.constructResults(*rawOutputPtr, forLoopExecutor);
            nvtx::pop_range();

            nvtx::push_range("encodeResults", 1);
            correctionOutput.encode();

            nvtx::pop_range();

            processResults(
                std::move(correctionOutput)
            );

            batchCompleted(rawOutputPtr->numAnchors); 

            myFreeOutputsQueue.push(rawOutputPtr);
        };

        CudaStream stream;

        std::queue<std::pair<GpuErrorCorrectorInput*,
            GpuErrorCorrectorRawOutput*>> dataInFlight;

        GpuErrorCorrectorInput* inputPtr = unprocessedInputs.popOrDefault(
            [&](){
                return !noMoreInputs;  //if noMoreInputs, return nullptr
            },
            nullptr
        ); 

        for(int preIters = 0; preIters < getNumExtraBuffers(); preIters++){

            if(inputPtr != nullptr){
                nvtx::push_range("getFreeRawOutput",1);
                //GpuErrorCorrectorRawOutput* rawOutputPtr = freeRawOutputs.pop();
                GpuErrorCorrectorRawOutput* rawOutputPtr = myFreeOutputsQueue.front();
                myFreeOutputsQueue.pop();
                nvtx::pop_range();

                nvtx::push_range("correct", 0);
                gpuErrorCorrector.correct(*inputPtr, *rawOutputPtr, stream);
                nvtx::pop_range();

                dataInFlight.emplace(inputPtr, rawOutputPtr);

                inputPtr = unprocessedInputs.popOrDefault(
                    [&](){
                        return !noMoreInputs;  //if noMoreInputs, return nullptr
                    },
                    nullptr
                ); 
            }
        }

        while(inputPtr != nullptr){
            nvtx::push_range("getFreeRawOutput",1);
            //GpuErrorCorrectorRawOutput* rawOutputPtr = freeRawOutputs.pop();
            GpuErrorCorrectorRawOutput* rawOutputPtr = myFreeOutputsQueue.front();
            myFreeOutputsQueue.pop();
            nvtx::pop_range();

            hipError_t cstatus = hipSuccess;
            cstatus = inputPtr->event.query();
            if(cstatus != hipSuccess){
                std::cerr << hipGetErrorString(cstatus) << "\n";
                assert(false);
            }
            cstatus = rawOutputPtr->event.query();
            if(cstatus != hipSuccess){
                std::cerr << hipGetErrorString(cstatus) << "\n";
                assert(false);
            }

            hipStreamSynchronize(stream); CUERR;

            // assert(hipSuccess == inputPtr->event.query());
            // assert(hipSuccess == rawOutputPtr->event.query());

            nvtx::push_range("correct", 0);
            gpuErrorCorrector.correct(*inputPtr, *rawOutputPtr, stream);
            nvtx::pop_range();

            dataInFlight.emplace(inputPtr, rawOutputPtr);

            if(!dataInFlight.empty()){
                auto pointers = dataInFlight.front();
                dataInFlight.pop();

                pointers.first->event.synchronize();
                freeInputs.push(pointers.first);

                pointers.second->event.synchronize();
                //std::cerr << "Synchronized output " << pointers.second << "\n";
                constructOutput(pointers.second);
            }            

            nvtx::push_range("getUnprocessedInput",2);
            inputPtr = unprocessedInputs.popOrDefault(
                [&](){
                    return !noMoreInputs;  //if noMoreInputs, return nullptr
                },
                nullptr
            ); 
            nvtx::pop_range();

        };

        //process outstanding buffered work
        while(!dataInFlight.empty()){
            auto pointers = dataInFlight.front();
            dataInFlight.pop();

            pointers.first->event.synchronize();
            freeInputs.push(pointers.first);

            pointers.second->event.synchronize();
            constructOutput(pointers.second);
        }

        activeCorrectorThreads--;

        if(activeCorrectorThreads == 0){
            noMoreRawOutputs = true;
        }

        hipStreamSynchronize(stream); CUERR;
    };


    template<class ResultProcessor, class BatchCompletion>
    void outputConstructorThreadFunction(
        const CorrectionOptions& correctionOptions,
        ReadCorrectionFlags& correctionFlags,
        ResultProcessor processResults,
        BatchCompletion batchCompleted
    ){

        OutputConstructor outputConstructor(            
            correctionFlags,
            correctionOptions
        );

        ThreadPool::ParallelForHandle pforHandle;
        //ForLoopExecutor forLoopExecutor(&threadPool, &pforHandle);
        SequentialForLoopExecutor forLoopExecutor;

        GpuErrorCorrectorRawOutput* rawOutputPtr = unprocessedRawOutputs.popOrDefault(
            [&](){
                return !noMoreRawOutputs;  //if noMoreRawOutputs, return nullptr
            },
            nullptr
        );

        while(rawOutputPtr != nullptr){
            nvtx::push_range("constructResults", 0);
            auto correctionOutput = outputConstructor.constructResults(*rawOutputPtr, forLoopExecutor);
            nvtx::pop_range();

            nvtx::push_range("encodeResults", 1);
            correctionOutput.encode();

            // std::vector<EncodedTempCorrectedSequence> encodedAnchorCorrections;
            // std::vector<EncodedTempCorrectedSequence> encodedCandidateCorrections;

            // if(correctionOutput.anchorCorrections.size() > 0){
            //     encodedAnchorCorrections.resize(correctionOutput.anchorCorrections.size());

            //     forLoopExecutor(std::size_t(0), correctionOutput.anchorCorrections.size(), 
            //         [&](auto begin, auto end, auto /*threadId*/){
            //             for(auto i = begin; i < end; i++){
            //                 correctionOutput.anchorCorrections[i].encodeInto(encodedAnchorCorrections[i]);
            //             }
            //         }
            //     );
            // }

            // if(correctionOutput.candidateCorrections.size() > 0){
            //     encodedCandidateCorrections.resize(correctionOutput.candidateCorrections.size());

            //     forLoopExecutor(std::size_t(0), correctionOutput.candidateCorrections.size(), 
            //         [&](auto begin, auto end, auto /*threadId*/){
            //             for(auto i = begin; i < end; i++){
            //                 correctionOutput.candidateCorrections[i].encodeInto(encodedCandidateCorrections[i]);
            //             }
            //         }
            //     );
            // }

            nvtx::pop_range();

            processResults(
                std::move(correctionOutput)
                // std::move(correctionOutput.anchorCorrections),
                // std::move(correctionOutput.candidateCorrections),
                // std::move(encodedAnchorCorrections),
                // std::move(encodedCandidateCorrections)
            );

            batchCompleted(rawOutputPtr->numAnchors); 


            freeRawOutputs.push(rawOutputPtr);

            nvtx::push_range("getUnprocessedRawOutput", 2);
            rawOutputPtr = unprocessedRawOutputs.popOrDefault(
                [&](){
                    return !noMoreRawOutputs;  //if noMoreRawOutputs, return nullptr
                },
                nullptr
            );  

            nvtx::pop_range();
        }
    };

private:
    const GpuReadStorage* readStorage;
    const Minhasher* minhasher;
    ThreadPool* threadPool;
    ClfAgent* clfAgent;

    SimpleSingleProducerSingleConsumerQueue<GpuErrorCorrectorInput*> freeInputs;
    SimpleSingleProducerSingleConsumerQueue<GpuErrorCorrectorInput*> unprocessedInputs;
    SimpleSingleProducerSingleConsumerQueue<GpuErrorCorrectorRawOutput*> freeRawOutputs;
    SimpleSingleProducerSingleConsumerQueue<GpuErrorCorrectorRawOutput*> unprocessedRawOutputs;

    std::atomic<bool> noMoreInputs{false};
    std::atomic<int> activeHasherThreads{0};
    std::atomic<bool> noMoreRawOutputs{false};
    std::atomic<int> activeCorrectorThreads{0};

    Config currentConfig;
};


template<class Minhasher>
MemoryFileFixedSize<EncodedTempCorrectedSequence> 
correct_gpu_impl(
        const GoodAlignmentProperties& goodAlignmentProperties,
        const CorrectionOptions& correctionOptions,
        const RuntimeOptions& runtimeOptions,
        const FileOptions& fileOptions,
        const MemoryOptions& memoryOptions,
        Minhasher& minhasher,
        GpuReadStorage& readStorage){

    assert(runtimeOptions.canUseGpu);
    //assert(runtimeOptions.max_candidates > 0);
    assert(runtimeOptions.deviceIds.size() > 0);

    const auto& deviceIds = runtimeOptions.deviceIds;

    const auto rsMemInfo = readStorage.getMemoryInfo();
    const auto mhMemInfo = minhasher.getMemoryInfo();

    std::size_t memoryAvailableBytesHost = memoryOptions.memoryTotalLimit;

    if(memoryAvailableBytesHost > rsMemInfo.host){
        memoryAvailableBytesHost -= rsMemInfo.host;
    }else{
        memoryAvailableBytesHost = 0;
    }

    if(memoryAvailableBytesHost > mhMemInfo.host){
        memoryAvailableBytesHost -= mhMemInfo.host;
    }else{
        memoryAvailableBytesHost = 0;
    }

    ReadCorrectionFlags correctionFlags(readStorage.getNumberOfReads());

    std::cerr << "Status flags per reads require " << correctionFlags.sizeInBytes() / 1024. / 1024. << " MB\n";

    if(memoryAvailableBytesHost > correctionFlags.sizeInBytes()){
        memoryAvailableBytesHost -= correctionFlags.sizeInBytes();
    }else{
        memoryAvailableBytesHost = 0;
    }

    const std::size_t availableMemoryInBytes = memoryAvailableBytesHost; //getAvailableMemoryInKB() * 1024;
    std::size_t memoryForPartialResultsInBytes = 0;

    if(availableMemoryInBytes > 2*(std::size_t(1) << 30)){
        memoryForPartialResultsInBytes = availableMemoryInBytes - 2*(std::size_t(1) << 30);
    }

    std::cerr << "Partial results may occupy " << (memoryForPartialResultsInBytes /1024. / 1024. / 1024.) 
        << " GB in memory. Remaining partial results will be stored in temp directory. \n";

    const std::string tmpfilename{fileOptions.tempdirectory + "/" + "MemoryFileFixedSizetmp"};
    MemoryFileFixedSize<EncodedTempCorrectedSequence> partialResults(memoryForPartialResultsInBytes, tmpfilename);

    //std::mutex outputstreamlock;

    BackgroundThread outputThread;

    auto saveCorrectedSequence = [&](const TempCorrectedSequence* tmp, const EncodedTempCorrectedSequence* encoded){
        //std::cerr << *tmp << "\n";
        //useEditsCountMap[tmp.useEdits]++;
        //std::unique_lock<std::mutex> l(outputstreammutex);
        if(!(tmp->hq && tmp->useEdits && tmp->edits.empty())){
            //outputstream << tmp << '\n';
            partialResults.storeElement(encoded);
            //useEditsSavedCountMap[tmp.useEdits]++;
            //numEditsHistogram[tmp.edits.size()]++;

            // std::cerr << tmp.edits.size() << " " << encoded.data.capacity() << "\n";
        }
    };

    // auto processResults = [&](
    //     std::vector<TempCorrectedSequence>&& anchorCorrections,
    //     std::vector<TempCorrectedSequence>&& candidateCorrections,
    //     std::vector<EncodedTempCorrectedSequence>&& encodedAnchorCorrections,
    //     std::vector<EncodedTempCorrectedSequence>&& encodedCandidateCorrections
    // ){
    //     assert(anchorCorrections.size() == encodedAnchorCorrections.size());
    //     assert(candidateCorrections.size() == encodedCandidateCorrections.size());

    //     const int numA = encodedAnchorCorrections.size();
    //     const int numC = encodedCandidateCorrections.size();

    //     auto outputFunction = [
    //         &,
    //         anchorCorrections = std::move(anchorCorrections),
    //         candidateCorrections = std::move(candidateCorrections),
    //         encodedAnchorCorrections = std::move(encodedAnchorCorrections),
    //         encodedCandidateCorrections = std::move(encodedCandidateCorrections)
    //     ](){

    //         const int numA = encodedAnchorCorrections.size();
    //         const int numC = encodedCandidateCorrections.size();

    //         for(int i = 0; i < numA; i++){
    //             saveCorrectedSequence(
    //                 &anchorCorrections[i], 
    //                 &encodedAnchorCorrections[i]
    //             );
    //         }

    //         for(int i = 0; i < numC; i++){
    //             saveCorrectedSequence(
    //                 &candidateCorrections[i], 
    //                 &encodedCandidateCorrections[i]
    //             );
    //         }
    //     };

    //     if(numA > 0 || numC > 0){
    //         outputThread.enqueue(std::move(outputFunction));
    //         //outputFunction();
    //     }
    // };

    auto processResults = [&](
        CorrectionOutput&& correctionOutput
    ){
        assert(correctionOutput.anchorCorrections.size() == correctionOutput.encodedAnchorCorrections.size());
        assert(correctionOutput.candidateCorrections.size() == correctionOutput.encodedCandidateCorrections.size());

        const int numA = correctionOutput.encodedAnchorCorrections.size();
        const int numC = correctionOutput.encodedCandidateCorrections.size();

        auto outputFunction = [
            &,
            correctionOutput = std::move(correctionOutput)
        ](){

            const int numA = correctionOutput.encodedAnchorCorrections.size();
            const int numC = correctionOutput.encodedCandidateCorrections.size();

            for(int i = 0; i < numA; i++){
                saveCorrectedSequence(
                    &correctionOutput.anchorCorrections[i], 
                    &correctionOutput.encodedAnchorCorrections[i]
                );
            }

            for(int i = 0; i < numC; i++){
                saveCorrectedSequence(
                    &correctionOutput.candidateCorrections[i], 
                    &correctionOutput.encodedCandidateCorrections[i]
                );
            }
        };

        if(numA > 0 || numC > 0){
            outputThread.enqueue(std::move(outputFunction));
            //outputFunction();
        }
    };

    outputThread.setMaximumQueueSize(runtimeOptions.threads);

    outputThread.start();

    //const int threadPoolSize = std::max(1, runtimeOptions.threads - 2*int(deviceIds.size()));
    //std::cerr << "threadpool size for correction = " << threadPoolSize << "\n";
    //ThreadPool threadPool(threadPoolSize);

    auto showProgress = [&](std::int64_t totalCount, int seconds){
        if(runtimeOptions.showProgress){

            int hours = seconds / 3600;
            seconds = seconds % 3600;
            int minutes = seconds / 60;
            seconds = seconds % 60;

            std::size_t numreads = readStorage.getNumberOfReads();
            
            printf("Processed %10lu of %10lu reads (Runtime: %03d:%02d:%02d)\r",
            totalCount, numreads,
            hours, minutes, seconds);

            std::fflush(stdout);
        }
    };

    auto updateShowProgressInterval = [](auto duration){
        return duration;
    };

    ProgressThread<std::int64_t> progressThread(readStorage.getNumberOfReads(), showProgress, updateShowProgressInterval);

    auto batchCompleted = [&](int size){
        //std::cerr << "Add progress " << size << "\n";
        progressThread.addProgress(size);
    };

    ClfAgent clfAgent_(correctionOptions, fileOptions);



    cpu::RangeGenerator<read_number> readIdGenerator(readStorage.getNumberOfReads());
    //cpu::RangeGenerator<read_number> readIdGenerator(std::min(1500000u, readStorage.getNumberOfReads()));

    if(false /* && runtimeOptions.threads <= 6*/){
        //execute a single thread pipeline with each available thread

        auto runPipeline = [&](int deviceId){    
            SimpleGpuCorrectionPipeline<Minhasher> pipeline(
                readStorage,
                minhasher,
                nullptr, //&threadPool
                &clfAgent_
            );
    
            pipeline.runToCompletion(
                deviceId,
                readIdGenerator,
                correctionOptions,
                goodAlignmentProperties,
                correctionFlags,
                processResults,
                batchCompleted
            );
        };
    
        std::vector<std::future<void>> futures;
    
        for(int i = 0; i < runtimeOptions.threads; i++){
            const int deviceId = deviceIds[i % deviceIds.size()];

            futures.emplace_back(std::async(
                std::launch::async,
                runPipeline,
                deviceId
            ));
        }
    
        for(auto& f : futures){
            f.wait();
        }
    }else{

     

        //Process a few batches on the first gpu to estimate runtime per step
        //These estimates will be used to spawn an appropriate number of threads for each gpu (assuming all gpus are similar)


        typename SimpleGpuCorrectionPipeline<Minhasher>::RunStatistics runStatistics;

        {
            SimpleGpuCorrectionPipeline<Minhasher> pipeline(
                readStorage,
                minhasher,
                nullptr, //&threadPool
                &clfAgent_
            );

            constexpr int numBatches = 50;

            runStatistics = pipeline.runSomeBatches(
                deviceIds[0],
                readIdGenerator,
                correctionOptions,
                goodAlignmentProperties,
                correctionFlags,
                processResults,
                batchCompleted,
                numBatches
            );   
                
        }

        const int numHashersPerCorrectorByTime = std::ceil(runStatistics.hasherTimeAverage / runStatistics.correctorTimeAverage);
        //std::cerr << runStatistics.hasherTimeAverage << " " << runStatistics.correctorTimeAverage << "\n";

        // auto runSimpleCpuPipeline = [&](int deviceId){
        //     // hipSetDevice(deviceId); CUERR;

        //     // SimpleCpuCorrectionPipeline pipeline;

        //     // std::unique_ptr<ReadProvider> readProvider = std::make_unique<GpuReadStorageReadProvider>(readStorage);
        //     // std::unique_ptr<CandidateIdsProvider> candidateIdsProvider = std::make_unique<GpuMinhasherCandidateIdsProvider>(minhasher);

        //     // pipeline.runToCompletion(
        //     //     readIdGenerator,
        //     //     correctionOptions,
        //     //     goodAlignmentProperties,
        //     //     correctionFlags,
        //     //     readProvider.get(),
        //     //     candidateIdsProvider.get(),
        //     //     processResults,
        //     //     batchCompleted
        //     // ); 
        // };

        auto runSimpleGpuPipeline = [&](int deviceId){
            SimpleGpuCorrectionPipeline<Minhasher> pipeline(
                readStorage,
                minhasher,
                nullptr, //&threadPool        
                &clfAgent_
            );

            pipeline.runToCompletionDoubleBuffered(
                deviceIds[0],
                readIdGenerator,
                correctionOptions,
                goodAlignmentProperties,
                correctionFlags,
                processResults,
                batchCompleted
            );  
        };

        auto runComplexGpuPipeline = [&](int deviceId, typename ComplexGpuCorrectionPipeline<Minhasher>::Config config){
            
            ComplexGpuCorrectionPipeline<Minhasher> pipeline(readStorage, minhasher, nullptr, &clfAgent_);

            pipeline.run(
                deviceId,
                config,
                readIdGenerator,
                correctionOptions,
                goodAlignmentProperties,
                correctionFlags,
                processResults,
                batchCompleted
            );
        };

        std::vector<std::future<void>> futures;

        const int numDevices = deviceIds.size();
        const int requiredNumThreadsForComplex = numHashersPerCorrectorByTime + (2 + 1 + 1);
        int availableThreads = runtimeOptions.threads;

        //std::cerr << "numDevice " << numDevices << ", requiredNumThreadsForComplex " << requiredNumThreadsForComplex << ", availableThreads " << availableThreads << "\n";

        // auto launchSimplePipelines = [&](int firstIdIndex, int lastIdIndex){
        //     constexpr int maxNumThreadsPerDevice = 3;
        //     assert(lastIdIndex <= numDevices);

        //     std::vector<int> numThreadsPerDevice(numDevices, 0);

        //     for(int i = 0; i < maxNumThreadsPerDevice; i++){
        //         for(int d = firstIdIndex; d < lastIdIndex; d++){
        //             if(availableThreads > 0){
        //                 futures.emplace_back(std::async(
        //                     std::launch::async,
        //                     runSimpleGpuPipeline,
        //                     deviceIds[d]
        //                 ));

        //                 availableThreads--;

        //                 numThreadsPerDevice[d]++;
        //             }
        //         }
        //     }

        //     for(int d = firstIdIndex; d < lastIdIndex; d++){
        //         if(numThreadsPerDevice[d] > 0){
        //             std::cerr << "Use " << numThreadsPerDevice[d] << " simple threads on device " << deviceIds[d] << "\n";
        //         }else{
        //             std::cerr << "Device " << deviceIds[d] << " will be unused. (Not enough threads available.)\n";
        //         }
        //     }
        // };

        for(int i = 0; i < numDevices; i++){ 
            const int deviceId = deviceIds[i];

            int threadsForDevice = std::max(1,std::min(availableThreads, requiredNumThreadsForComplex));

            if(threadsForDevice > 3){

                typename ComplexGpuCorrectionPipeline<Minhasher>::Config pipelineConfig;
                #if 1
                pipelineConfig.numOutputConstructors = 0; //always 0

                pipelineConfig.numCorrectors = 1;
                threadsForDevice -= pipelineConfig.numCorrectors;
                
                pipelineConfig.numHashers = std::max(1, std::min(threadsForDevice, numHashersPerCorrectorByTime));
                threadsForDevice -= pipelineConfig.numHashers;

                if(threadsForDevice > 0){
                    pipelineConfig.numCorrectors++;
                    threadsForDevice--;
                }

                pipelineConfig.numHashers += threadsForDevice;
                #else
                pipelineConfig.numOutputConstructors = 0; //always 0
                pipelineConfig.numCorrectors = 13;
                pipelineConfig.numHashers = 3;
                #endif

                std::cerr << "\nWill use " << pipelineConfig.numHashers << " hasher(s), "
                << pipelineConfig.numCorrectors << " corrector(s) "
                << "on device " << deviceId << "\n";                

                futures.emplace_back(
                    std::async(
                        std::launch::async,
                        runComplexGpuPipeline,
                        deviceId, pipelineConfig
                    )
                );

                availableThreads -= pipelineConfig.numOutputConstructors;
                availableThreads -= pipelineConfig.numCorrectors;
                availableThreads -= pipelineConfig.numHashers;
            }else{
                std::cerr << "\nWill use " << threadsForDevice << " simple pipelines on device " << deviceId << "\n";

                while(threadsForDevice > 0){
                    futures.emplace_back(std::async(
                        std::launch::async,
                        runSimpleGpuPipeline,
                        deviceId
                    ));

                    threadsForDevice--;
                }
            }
        }

        //if there are not enough threads to run one complex pipeline on any device, only use simple pipelines
        // if(requiredNumThreadsForComplex > availableThreads){
        //     launchSimplePipelines(0, numDevices);
        // }else{

            // std::vector<bool> useComplexPipeline(numDevices, false);
            // int numSimple = 0;
            // int firstSimpleDevice = numDevices;

            // for(int i = 0; i < numDevices; i++){            

            //     if(availableThreads >= requiredNumThreadsForComplex){  
            //         availableThreads -= requiredNumThreadsForComplex;
            //     }else{
            //         numSimple++;

            //         if(firstSimpleDevice == numDevices){
            //             firstSimpleDevice = i;
            //         }
            //     }
            // }

            // for(int i = 0; i < firstSimpleDevice; i++){
            //     const int deviceId = deviceIds[i];

            //     typename ComplexGpuCorrectionPipeline<Minhasher>::Config pipelineConfig;
            //     pipelineConfig.numHashers = numHashersPerCorrectorByTime + 2;
            //     pipelineConfig.numCorrectors = 1 + 1;
            //     pipelineConfig.numOutputConstructors = 0;

            //     // std::cerr << "\nWill use " << pipelineConfig.numHashers << " hasher(s), " 
            //     // << pipelineConfig.numCorrectors << " corrector(s), " 
            //     // << pipelineConfig.numOutputConstructors << " output constructor(s) "
            //     // << "on device " << deviceId << "\n";                

            //     futures.emplace_back(
            //         std::async(
            //             std::launch::async,
            //             runComplexGpuPipeline,
            //             deviceId, pipelineConfig
            //         )
            //     );
            // }

            // launchSimplePipelines(firstSimpleDevice, numDevices);
        //}

        //std::cerr << "Remaing threads after launching gpu pipelines: " << availableThreads << "\n";

        //use remaining threads to correct on the host
        // for(int i = 0; i < availableThreads; i++){
        //     futures.emplace_back(
        //         std::async(
        //             std::launch::async,
        //             runSimpleCpuPipeline,
        //             deviceIds[i % deviceIds.size()]
        //         )
        //     );                
        // }

        for(auto& f : futures){
            f.wait();
        }


        
    }

#if 0

auto runPipeline = [&](int deviceId){
    auto printRunStats = [](const auto& runStatistics){
        std::cerr << "hashing time average: " << runStatistics.hasherTimeAverage << "\n";
        std::cerr << "corrector time average: " << runStatistics.correctorTimeAverage << "\n";
        std::cerr << "output constructor time average: " << runStatistics.outputconstructorTimeAverage << "\n";

        std::cerr << "input size: ";
        std::cerr << "host: " << runStatistics.memoryInputData.host << ", ";
        for(const auto& d : runStatistics.memoryInputData.device){
            std::cerr << "device " << d.first << ": " << d.second << " ";
        }
        std::cerr << "\n";

        std::cerr << "raw output size ";
        std::cerr << "host: " << runStatistics.memoryRawOutputData.host << ", ";
        for(const auto& d : runStatistics.memoryRawOutputData.device){
            std::cerr << "device " << d.first << ": " << d.second << " ";
        }
        std::cerr << "\n";

        std::cerr << "hasher size ";
        std::cerr << "host: " << runStatistics.memoryHasher.host << ", ";
        for(const auto& d : runStatistics.memoryHasher.device){
            std::cerr << "device " << d.first << ": " << d.second << " ";
        }
        std::cerr << "\n";

        std::cerr << "corrector size ";
        std::cerr << "host: " << runStatistics.memoryCorrector.host << ", ";
        for(const auto& d : runStatistics.memoryCorrector.device){
            std::cerr << "device " << d.first << ": " << d.second << " ";
        }
        std::cerr << "\n";

        std::cerr << "output constructor size ";
        std::cerr << "host: " << runStatistics.memoryOutputConstructor.host << ", ";
        for(const auto& d : runStatistics.memoryOutputConstructor.device){
            std::cerr << "device " << d.first << ": " << d.second << " ";
        }
        std::cerr << "\n";
    };
#endif   

    progressThread.finished(); 
        
    std::cout << std::endl;

    //threadPool.wait();
    outputThread.stopThread(BackgroundThread::StopType::FinishAndStop);

    //assert(threadPool.empty());

    partialResults.flush();

    // std::ofstream flagsstream(fileOptions.outputfilenames[0] + "_flags");

    // for(std::uint64_t i = 0; i < gpuReadStorage->getNumberOfReads(); i++){
    //     flagsstream << correctionFlags.isCorrectedAsHQAnchor(i) << " " 
    //         << correctionFlags.isNotCorrectedAsAnchor(i) << "\n";
    // }

    return partialResults;
}


MemoryFileFixedSize<EncodedTempCorrectedSequence> 
correct_gpu(
        const GoodAlignmentProperties& goodAlignmentProperties,
        const CorrectionOptions& correctionOptions,
        const RuntimeOptions& runtimeOptions,
        const FileOptions& fileOptions,
        const MemoryOptions& memoryOptions,
        GpuMinhasher& minhasher,
        GpuReadStorage& readStorage){

    return correct_gpu_impl(
        goodAlignmentProperties,
        correctionOptions,
        runtimeOptions,
        fileOptions,
        memoryOptions,
        minhasher,
        readStorage
    );
}


}
}

