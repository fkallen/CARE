#include "hip/hip_runtime.h"
//#define NDEBUG

#include <gpu/kernels.hpp>
#include <gpu/kernellaunch.hpp>

#include <bestalignment.hpp>

#include <sequencehelpers.hpp>

#include <hostdevicefunctions.cuh>

#include <hpc_helpers.cuh>
#include <config.hpp>

#include <cassert>


#include <hipcub/hipcub.hpp>
#include <hip/hip_cooperative_groups.h>
#include <thrust/binary_search.h>

namespace cg = cooperative_groups;



namespace care{
namespace gpu{


    template<int blocksize, int tilesize>
    __global__
    void selectIndicesOfGoodCandidatesKernel(
            int* __restrict__ d_indicesOfGoodCandidates,
            int* __restrict__ d_numIndicesPerAnchor,
            int* __restrict__ d_totalNumIndices,
            const BestAlignment_t* __restrict__ d_alignmentFlags,
            const int* __restrict__ d_candidates_per_subject,
            const int* __restrict__ d_candidates_per_subject_prefixsum,
            const int* __restrict__ d_anchorIndicesOfCandidates,
            const int* __restrict__ d_numAnchors,
            const int* __restrict__ d_numCandidates
            ){

        static_assert(blocksize % tilesize == 0);
        static_assert(tilesize == 32);

        constexpr int numTilesPerBlock = blocksize / tilesize;

        const int numAnchors = *d_numAnchors;
        //const int numCandidates = *d_numCandidates;

        const int numTiles = (gridDim.x * blocksize) / tilesize;
        const int tileId = (threadIdx.x + blockIdx.x * blocksize) / tilesize;
        const int tileIdInBlock = threadIdx.x / tilesize;

        __shared__ int totalIndices;
        __shared__ int counts[numTilesPerBlock];

        if(threadIdx.x == 0){
            totalIndices = 0;
        }
        __syncthreads();

        auto tile = cg::tiled_partition<tilesize>(cg::this_thread_block());

        for(int anchorIndex = tileId; anchorIndex < numAnchors; anchorIndex += numTiles){

            const int offset = d_candidates_per_subject_prefixsum[anchorIndex];
            int* const indicesPtr = d_indicesOfGoodCandidates + offset;
            int* const numIndicesPtr = d_numIndicesPerAnchor + anchorIndex;
            const BestAlignment_t* const myAlignmentFlagsPtr = d_alignmentFlags + offset;

            const int numCandidatesForAnchor = d_candidates_per_subject[anchorIndex];

            if(tile.thread_rank() == 0){
                counts[tileIdInBlock] = 0;
            }
            tile.sync();

            for(int localCandidateIndex = tile.thread_rank(); 
                    localCandidateIndex < numCandidatesForAnchor; 
                    localCandidateIndex += tile.size()){
                
                const BestAlignment_t alignmentflag = myAlignmentFlagsPtr[localCandidateIndex];

                if(alignmentflag != BestAlignment_t::None){
                    cg::coalesced_group g = cg::coalesced_threads();
                    int outputPos;
                    if (g.thread_rank() == 0) {
                        outputPos = atomicAdd(&counts[tileIdInBlock], g.size());
                        atomicAdd(&totalIndices, g.size());
                    }
                    outputPos = g.thread_rank() + g.shfl(outputPos, 0);
                    indicesPtr[outputPos] = localCandidateIndex;
                }
            }

            tile.sync();
            if(tile.thread_rank() == 0){
                atomicAdd(numIndicesPtr, counts[tileIdInBlock]);
            }

        }

        __syncthreads();

        if(threadIdx.x == 0){
            atomicAdd(d_totalNumIndices, totalIndices);
        }
    }




    /*

        For each candidate, compute the alignment of anchor|candidate and anchor|revc-candidate
        Compares both alignments and keeps the better one

        Sequences are stored in dynamic sized shared memory.
        To reduce shared memory usage, the candidates belonging to the same anchor
        are processed by a set of tiles. Each tile only computes alignments for a single anchor.
        This anchor is stored in shared memory and shared by all threads within a tile
    */

    template<int tilesize>
    __global__
    void
    popcount_shifted_hamming_distance_smem_kernel(
                const unsigned int* __restrict__ subjectDataHiLo,
                const unsigned int* __restrict__ candidateDataHiLoTransposed,
                int* __restrict__ d_alignment_overlaps,
                int* __restrict__ d_alignment_shifts,
                int* __restrict__ d_alignment_nOps,
                bool* __restrict__ d_alignment_isValid,
                BestAlignment_t* __restrict__ d_alignment_best_alignment_flags,
                const int* __restrict__ subjectSequencesLength,
                const int* __restrict__ candidateSequencesLength,
                const int* __restrict__ candidates_per_subject_prefixsum,
                const int* __restrict__ tiles_per_subject_prefixsum,
                const int* __restrict__ numAnchorsPtr,
                const int* __restrict__ numCandidatesPtr,
                const bool* __restrict__ anchorContainsN,
                bool removeAmbiguousAnchors,
                const bool* __restrict__ candidateContainsN,
                bool removeAmbiguousCandidates,
                int encodedSequencePitchInInts2BitHiLo,
                int min_overlap,
                float maxErrorRate,
                float min_overlap_ratio,
                float estimatedNucleotideErrorRate){

        const int n_subjects = *numAnchorsPtr;
        const int n_candidates = *numCandidatesPtr;

        auto no_bank_conflict_index = [](int logical_index) -> int {
            return logical_index * blockDim.x;
        };

        auto identity = [](auto logical_index){
            return logical_index;
        };

        auto popcount = [](auto i){return __popc(i);};

        auto hammingDistanceWithShift = [&](bool doShift, int overlapsize, int max_errors,
                                    unsigned int* shiftptr_hi, unsigned int* shiftptr_lo, auto transfunc1,
                                    int shiftptr_size,
                                    const unsigned int* otherptr_hi, const unsigned int* otherptr_lo,
                                    auto transfunc2){

            if(doShift){
                shiftBitArrayLeftBy<1>(shiftptr_hi, shiftptr_size / 2, transfunc1);
                shiftBitArrayLeftBy<1>(shiftptr_lo, shiftptr_size / 2, transfunc1);
            }

            const int score = hammingdistanceHiLo(shiftptr_hi,
                                                shiftptr_lo,
                                                otherptr_hi,
                                                otherptr_lo,
                                                overlapsize,
                                                overlapsize,
                                                max_errors,
                                                transfunc1,
                                                transfunc2,
                                                popcount);

            return score;
        };


        auto alignmentComparator = [&] (int fwd_alignment_overlap,
            int revc_alignment_overlap,
            int fwd_alignment_nops,
            int revc_alignment_nops,
            bool fwd_alignment_isvalid,
            bool revc_alignment_isvalid,
            int subjectlength,
            int querylength)->BestAlignment_t{

            return choose_best_alignment(
                fwd_alignment_overlap,
                revc_alignment_overlap,
                fwd_alignment_nops,
                revc_alignment_nops,
                fwd_alignment_isvalid,
                revc_alignment_isvalid,
                subjectlength,
                querylength,
                min_overlap_ratio,
                min_overlap,
                estimatedNucleotideErrorRate * 4.0f
            );
        };

        // sizeof(char) * (max_sequence_bytes * num_tiles   // tiles share the subject
        //                    + max_sequence_bytes * num_threads // each thread works with its own candidate
        //                    + max_sequence_bytes * num_threads) // each thread needs memory to shift a sequence
        extern __shared__ unsigned int sharedmemory[];

        //set up shared memory pointers

        const int tiles = (blockDim.x * gridDim.x) / tilesize;
        const int globalTileId = (blockDim.x * blockIdx.x + threadIdx.x) / tilesize;
        const int localTileId = (threadIdx.x) / tilesize;
        const int tilesPerBlock = blockDim.x / tilesize;
        const int laneInTile = threadIdx.x % tilesize;
        const int requiredTiles = tiles_per_subject_prefixsum[n_subjects];

        unsigned int* const subjectBackupsBegin = sharedmemory; // per tile shared memory to store subject
        unsigned int* const queryBackupsBegin = subjectBackupsBegin + encodedSequencePitchInInts2BitHiLo * tilesPerBlock; // per thread shared memory to store query
        unsigned int* const mySequencesBegin = queryBackupsBegin + encodedSequencePitchInInts2BitHiLo * blockDim.x; // per thread shared memory to store shifted sequence

        unsigned int* const subjectBackup = subjectBackupsBegin + encodedSequencePitchInInts2BitHiLo * localTileId; // accesed via identity
        unsigned int* const queryBackup = queryBackupsBegin + threadIdx.x; // accesed via no_bank_conflict_index
        unsigned int* const mySequence = mySequencesBegin + threadIdx.x; // accesed via no_bank_conflict_index

        for(int logicalTileId = globalTileId; logicalTileId < requiredTiles ; logicalTileId += tiles){

            const int subjectIndex = thrust::distance(tiles_per_subject_prefixsum,
                                                    thrust::lower_bound(
                                                        thrust::seq,
                                                        tiles_per_subject_prefixsum,
                                                        tiles_per_subject_prefixsum + n_subjects + 1,
                                                        logicalTileId + 1))-1;

            const int candidatesBeforeThisSubject = candidates_per_subject_prefixsum[subjectIndex];
            const int maxCandidateIndex_excl = candidates_per_subject_prefixsum[subjectIndex+1];
            //const int tilesForThisSubject = tiles_per_subject_prefixsum[subjectIndex + 1] - tiles_per_subject_prefixsum[subjectIndex];
            const int tileForThisSubject = logicalTileId - tiles_per_subject_prefixsum[subjectIndex];
            const int candidateIndex = candidatesBeforeThisSubject + tileForThisSubject * tilesize + laneInTile;

            const int subjectbases = subjectSequencesLength[subjectIndex];
            const int subjectints = SequenceHelpers::getEncodedNumInts2BitHiLo(subjectbases);
            const unsigned int* subjectptr = subjectDataHiLo + std::size_t(subjectIndex) * encodedSequencePitchInInts2BitHiLo;

            //save subject in shared memory (in parallel, per tile)
            for(int lane = laneInTile; lane < encodedSequencePitchInInts2BitHiLo; lane += tilesize) {
                subjectBackup[identity(lane)] = subjectptr[lane];
                //transposed
                //subjectBackup[identity(lane)] = ((unsigned int*)(subjectptr))[lane * n_subjects];
            }

            cg::tiled_partition<tilesize>(cg::this_thread_block()).sync();


            if(candidateIndex < maxCandidateIndex_excl){
                if(!(removeAmbiguousAnchors && anchorContainsN[subjectIndex]) && !(removeAmbiguousCandidates && candidateContainsN[candidateIndex])){
                    const int querybases = candidateSequencesLength[candidateIndex];
                    const int queryints = SequenceHelpers::getEncodedNumInts2BitHiLo(querybases);
                    const int totalbases = subjectbases + querybases;
                    const int minoverlap = max(min_overlap, int(float(subjectbases) * min_overlap_ratio));

                    const unsigned int* candidateptr = candidateDataHiLoTransposed + std::size_t(candidateIndex);

                    //save query in shared memory
                    for(int i = 0; i < encodedSequencePitchInInts2BitHiLo; i += 1) {
                        //queryBackup[no_bank_conflict_index(i)] = ((unsigned int*)(candidateptr))[i];
                        //transposed
                        queryBackup[no_bank_conflict_index(i)] = candidateptr[i * n_candidates];
                    }

                    const unsigned int* const subjectBackup_hi = subjectBackup;
                    const unsigned int* const subjectBackup_lo = subjectBackup + identity(subjectints/2);
                    const unsigned int* const queryBackup_hi = queryBackup;
                    const unsigned int* const queryBackup_lo = queryBackup + no_bank_conflict_index(queryints/2);

                    int bestScore[2];
                    int bestShift[2];
                    int overlapsize[2];
                    int opnr[2];

                    #pragma unroll
                    for(int orientation = 0; orientation < 2; orientation++){
                        const bool isReverseComplement = orientation == 1;

                        if(isReverseComplement) {
                            SequenceHelpers::reverseComplementSequenceInplace2BitHiLo(queryBackup, querybases, no_bank_conflict_index);
                        }

                        //begin SHD algorithm

                        bestScore[orientation] = totalbases;     // score is number of mismatches
                        bestShift[orientation] = -querybases;    // shift of query relative to subject. shift < 0 if query begins before subject

                        auto handle_shift = [&](int shift, int overlapsize,
                                                    unsigned int* shiftptr_hi, unsigned int* shiftptr_lo, auto transfunc1,
                                                    int shiftptr_size,
                                                    const unsigned int* otherptr_hi, const unsigned int* otherptr_lo,
                                                    auto transfunc2){

                            //const int max_errors = int(float(overlapsize) * maxErrorRate);
                            const int max_errors_excl = min(int(float(overlapsize) * maxErrorRate),
                            bestScore[orientation] - totalbases + 2*overlapsize);

                            if(max_errors_excl > 0){

                                int score = hammingDistanceWithShift(shift != 0, overlapsize, max_errors_excl,
                                                    shiftptr_hi,shiftptr_lo, transfunc1,
                                                    shiftptr_size,
                                                    otherptr_hi, otherptr_lo, transfunc2);

                                

                                // printf("%d, %d %d %d --- ", queryIndex, shift, overlapsize, score);

                                // printf("%d %d %d %d | %d %d %d %d --- ", 
                                //     shiftptr_hi[transfunc1(0)], shiftptr_hi[transfunc1(1)], shiftptr_hi[transfunc1(2)], shiftptr_hi[transfunc1(3)],
                                //     shiftptr_lo[transfunc1(0)], shiftptr_lo[transfunc1(1)], shiftptr_lo[transfunc1(2)], shiftptr_lo[transfunc1(3)]);

                                // printf("%d %d %d %d | %d %d %d %d\n", 
                                //     otherptr_hi[transfunc2(0)], otherptr_hi[transfunc2(1)], otherptr_hi[transfunc2(2)], otherptr_hi[transfunc2(3)],
                                //     otherptr_lo[transfunc2(0)], otherptr_lo[transfunc2(1)], otherptr_lo[transfunc2(2)], otherptr_lo[transfunc2(3)]);

                                score = (score < max_errors_excl ?
                                        score + totalbases - 2*overlapsize // non-overlapping regions count as mismatches
                                        : std::numeric_limits<int>::max()); // too many errors, discard

                                if(score < bestScore[orientation]){
                                    bestScore[orientation] = score;
                                    bestShift[orientation] = shift;
                                }

                                return true;
                            }else{
                                //printf("%d, %d %d %d max_errors_excl\n", queryIndex, shift, overlapsize, max_errors_excl);
                                return false;
                            }
                        };

                        //initialize threadlocal smem array with subject
                        for(int i = 0; i < encodedSequencePitchInInts2BitHiLo; i += 1) {
                            mySequence[no_bank_conflict_index(i)] = subjectBackup[identity(i)];
                        }

                        unsigned int* mySequence_hi = mySequence;
                        unsigned int* mySequence_lo = mySequence + no_bank_conflict_index(subjectints / 2);

                        for(int shift = 0; shift < subjectbases - minoverlap + 1; shift += 1) {
                            const int overlapsize = min(subjectbases - shift, querybases);

                            bool b = handle_shift(shift, overlapsize,
                                            mySequence_hi, mySequence_lo, no_bank_conflict_index,
                                            subjectints,
                                            queryBackup_hi, queryBackup_lo, no_bank_conflict_index);
                            if(!b){
                                break;
                            }
                        }

                        //initialize threadlocal smem array with query
                        for(int i = 0; i < encodedSequencePitchInInts2BitHiLo; i += 1) {
                            mySequence[no_bank_conflict_index(i)] = queryBackup[no_bank_conflict_index(i)];
                        }

                        mySequence_hi = mySequence;
                        mySequence_lo = mySequence + no_bank_conflict_index(queryints / 2);

                        for(int shift = -1; shift >= -querybases + minoverlap; shift -= 1) {
                            const int overlapsize = min(subjectbases, querybases + shift);

                            bool b = handle_shift(shift, overlapsize,
                                            mySequence_hi, mySequence_lo, no_bank_conflict_index,
                                            queryints,
                                            subjectBackup_hi, subjectBackup_lo, identity);
                            if(!b){
                                break;
                            }
                        }

                        const int queryoverlapbegin_incl = max(-bestShift[orientation], 0);
                        const int queryoverlapend_excl = min(querybases, subjectbases - bestShift[orientation]);
                        overlapsize[orientation] = queryoverlapend_excl - queryoverlapbegin_incl;
                        opnr[orientation] = bestScore[orientation] - totalbases + 2*overlapsize[orientation];
                    }

                    const BestAlignment_t flag = alignmentComparator(
                        overlapsize[0],
                        overlapsize[1],
                        opnr[0],
                        opnr[1],
                        bestShift[0] != -querybases,
                        bestShift[1] != -querybases,
                        subjectbases,
                        querybases
                    );

                    d_alignment_best_alignment_flags[candidateIndex] = flag;
                    d_alignment_overlaps[candidateIndex] = flag == BestAlignment_t::Forward ? overlapsize[0] : overlapsize[1];
                    d_alignment_shifts[candidateIndex] = flag == BestAlignment_t::Forward ? bestShift[0] : bestShift[1];
                    d_alignment_nOps[candidateIndex] = flag == BestAlignment_t::Forward ? opnr[0] : opnr[1];
                    d_alignment_isValid[candidateIndex] = flag == BestAlignment_t::Forward ? bestShift[0] != -querybases : bestShift[1] != -querybases;
                }else{
                    d_alignment_best_alignment_flags[candidateIndex] = BestAlignment_t::None;
                    d_alignment_isValid[candidateIndex] = false;
                }
            }
        }
    }

    template<int tilesize>
    __global__
    void
    popcount_rightshifted_hamming_distance_smem_kernel(
                const unsigned int* __restrict__ subjectDataHiLo,
                const unsigned int* __restrict__ candidateDataHiLoTransposed,
                int* __restrict__ d_alignment_overlaps,
                int* __restrict__ d_alignment_shifts,
                int* __restrict__ d_alignment_nOps,
                bool* __restrict__ d_alignment_isValid,
                BestAlignment_t* __restrict__ d_alignment_best_alignment_flags,
                const int* __restrict__ subjectSequencesLength,
                const int* __restrict__ candidateSequencesLength,
                const int* __restrict__ candidates_per_subject_prefixsum,
                const int* __restrict__ tiles_per_subject_prefixsum,
                const int* __restrict__ numAnchorsPtr,
                const int* __restrict__ numCandidatesPtr,
                const bool* __restrict__ anchorContainsN,
                bool removeAmbiguousAnchors,
                const bool* __restrict__ candidateContainsN,
                bool removeAmbiguousCandidates,
                int encodedSequencePitchInInts2BitHiLo,
                int min_overlap,
                float maxErrorRate,
                float min_overlap_ratio,
                float estimatedNucleotideErrorRate){

        const int n_subjects = *numAnchorsPtr;
        const int n_candidates = *numCandidatesPtr;

        auto make_reverse_complement_inplace = [&](unsigned int* sequence, int sequencelength, auto indextrafo){
            reverseComplementInplace2BitHiLo((unsigned int*)sequence, sequencelength, indextrafo);
        };

        auto no_bank_conflict_index = [](int logical_index) -> int {
            return logical_index * blockDim.x;
        };

        auto identity = [](auto logical_index){
            return logical_index;
        };

        auto popcount = [](auto i){return __popc(i);};

        auto hammingDistanceWithShift = [&](bool doShift, int overlapsize, int max_errors,
                                    unsigned int* shiftptr_hi, unsigned int* shiftptr_lo, auto transfunc1,
                                    int shiftptr_size,
                                    const unsigned int* otherptr_hi, const unsigned int* otherptr_lo,
                                    auto transfunc2){

            if(doShift){
                shiftBitArrayLeftBy<1>(shiftptr_hi, shiftptr_size / 2, transfunc1);
                shiftBitArrayLeftBy<1>(shiftptr_lo, shiftptr_size / 2, transfunc1);
            }

            const int score = hammingdistanceHiLo(shiftptr_hi,
                                                shiftptr_lo,
                                                otherptr_hi,
                                                otherptr_lo,
                                                overlapsize,
                                                overlapsize,
                                                max_errors,
                                                transfunc1,
                                                transfunc2,
                                                popcount);

            return score;
        };


        auto alignmentComparator = [&] (int fwd_alignment_overlap,
            int revc_alignment_overlap,
            int fwd_alignment_nops,
            int revc_alignment_nops,
            bool fwd_alignment_isvalid,
            bool revc_alignment_isvalid,
            int subjectlength,
            int querylength)->BestAlignment_t{

            return choose_best_alignment(
                fwd_alignment_overlap,
                revc_alignment_overlap,
                fwd_alignment_nops,
                revc_alignment_nops,
                fwd_alignment_isvalid,
                revc_alignment_isvalid,
                subjectlength,
                querylength,
                min_overlap_ratio,
                min_overlap,
                estimatedNucleotideErrorRate * 4.0f
            );
        };

        // sizeof(char) * (max_sequence_bytes * num_tiles   // tiles share the subject
        //                    + max_sequence_bytes * num_threads // each thread works with its own candidate
        //                    + max_sequence_bytes * num_threads) // each thread needs memory to shift a sequence
        extern __shared__ unsigned int sharedmemory[];

        //set up shared memory pointers

        const int tiles = (blockDim.x * gridDim.x) / tilesize;
        const int globalTileId = (blockDim.x * blockIdx.x + threadIdx.x) / tilesize;
        const int localTileId = (threadIdx.x) / tilesize;
        const int tilesPerBlock = blockDim.x / tilesize;
        const int laneInTile = threadIdx.x % tilesize;
        const int requiredTiles = tiles_per_subject_prefixsum[n_subjects];

        unsigned int* const subjectBackupsBegin = sharedmemory; // per tile shared memory to store subject
        unsigned int* const queryBackupsBegin = subjectBackupsBegin + encodedSequencePitchInInts2BitHiLo * tilesPerBlock; // per thread shared memory to store query
        unsigned int* const mySequencesBegin = queryBackupsBegin + encodedSequencePitchInInts2BitHiLo * blockDim.x; // per thread shared memory to store shifted sequence

        unsigned int* const subjectBackup = subjectBackupsBegin + encodedSequencePitchInInts2BitHiLo * localTileId; // accesed via identity
        unsigned int* const queryBackup = queryBackupsBegin + threadIdx.x; // accesed via no_bank_conflict_index
        unsigned int* const mySequence = mySequencesBegin + threadIdx.x; // accesed via no_bank_conflict_index

        for(int logicalTileId = globalTileId; logicalTileId < requiredTiles ; logicalTileId += tiles){

            const int subjectIndex = thrust::distance(tiles_per_subject_prefixsum,
                                                    thrust::lower_bound(
                                                        thrust::seq,
                                                        tiles_per_subject_prefixsum,
                                                        tiles_per_subject_prefixsum + n_subjects + 1,
                                                        logicalTileId + 1))-1;

            const int candidatesBeforeThisSubject = candidates_per_subject_prefixsum[subjectIndex];
            const int maxCandidateIndex_excl = candidates_per_subject_prefixsum[subjectIndex+1];
            //const int tilesForThisSubject = tiles_per_subject_prefixsum[subjectIndex + 1] - tiles_per_subject_prefixsum[subjectIndex];
            const int tileForThisSubject = logicalTileId - tiles_per_subject_prefixsum[subjectIndex];
            const int candidateIndex = candidatesBeforeThisSubject + tileForThisSubject * tilesize + laneInTile;

            const int subjectbases = subjectSequencesLength[subjectIndex];
            const int subjectints = getEncodedNumInts2BitHiLo(subjectbases);
            const unsigned int* subjectptr = subjectDataHiLo + std::size_t(subjectIndex) * encodedSequencePitchInInts2BitHiLo;

            //save subject in shared memory (in parallel, per tile)
            for(int lane = laneInTile; lane < encodedSequencePitchInInts2BitHiLo; lane += tilesize) {
                subjectBackup[identity(lane)] = subjectptr[lane];
                //transposed
                //subjectBackup[identity(lane)] = ((unsigned int*)(subjectptr))[lane * n_subjects];
            }

            cg::tiled_partition<tilesize>(cg::this_thread_block()).sync();


            if(candidateIndex < maxCandidateIndex_excl){
                if(!(removeAmbiguousAnchors && anchorContainsN[subjectIndex]) && !(removeAmbiguousCandidates && candidateContainsN[candidateIndex])){
                    const int querybases = candidateSequencesLength[candidateIndex];
                    const int queryints = getEncodedNumInts2BitHiLo(querybases);
                    const int totalbases = subjectbases + querybases;
                    const int minoverlap = max(min_overlap, int(float(subjectbases) * min_overlap_ratio));

                    const unsigned int* candidateptr = candidateDataHiLoTransposed + std::size_t(candidateIndex);

                    //save query in shared memory
                    for(int i = 0; i < encodedSequencePitchInInts2BitHiLo; i += 1) {
                        //queryBackup[no_bank_conflict_index(i)] = ((unsigned int*)(candidateptr))[i];
                        //transposed
                        queryBackup[no_bank_conflict_index(i)] = candidateptr[i * n_candidates];
                    }

                    const unsigned int* const queryBackup_hi = queryBackup;
                    const unsigned int* const queryBackup_lo = queryBackup + no_bank_conflict_index(queryints/2);

                    int bestScore[2];
                    int bestShift[2];
                    int overlapsize[2];
                    int opnr[2];

                    #pragma unroll
                    for(int orientation = 0; orientation < 2; orientation++){
                        const bool isReverseComplement = orientation == 1;

                        if(isReverseComplement) {
                            make_reverse_complement_inplace(queryBackup, querybases, no_bank_conflict_index);
                        }

                        //begin SHD algorithm

                        bestScore[orientation] = totalbases;     // score is number of mismatches
                        bestShift[orientation] = -querybases;    // shift of query relative to subject. shift < 0 if query begins before subject

                        auto handle_shift = [&](int shift, int overlapsize,
                                                    unsigned int* shiftptr_hi, unsigned int* shiftptr_lo, auto transfunc1,
                                                    int shiftptr_size,
                                                    const unsigned int* otherptr_hi, const unsigned int* otherptr_lo,
                                                    auto transfunc2){

                            //const int max_errors = int(float(overlapsize) * maxErrorRate);
                            const int max_errors_excl = min(int(float(overlapsize) * maxErrorRate),
                            bestScore[orientation] - totalbases + 2*overlapsize);

                            if(max_errors_excl > 0){

                                int score = hammingDistanceWithShift(shift != 0, overlapsize, max_errors_excl,
                                                    shiftptr_hi,shiftptr_lo, transfunc1,
                                                    shiftptr_size,
                                                    otherptr_hi, otherptr_lo, transfunc2);

                                

                                // printf("%d, %d %d %d --- ", queryIndex, shift, overlapsize, score);

                                // printf("%d %d %d %d | %d %d %d %d --- ", 
                                //     shiftptr_hi[transfunc1(0)], shiftptr_hi[transfunc1(1)], shiftptr_hi[transfunc1(2)], shiftptr_hi[transfunc1(3)],
                                //     shiftptr_lo[transfunc1(0)], shiftptr_lo[transfunc1(1)], shiftptr_lo[transfunc1(2)], shiftptr_lo[transfunc1(3)]);

                                // printf("%d %d %d %d | %d %d %d %d\n", 
                                //     otherptr_hi[transfunc2(0)], otherptr_hi[transfunc2(1)], otherptr_hi[transfunc2(2)], otherptr_hi[transfunc2(3)],
                                //     otherptr_lo[transfunc2(0)], otherptr_lo[transfunc2(1)], otherptr_lo[transfunc2(2)], otherptr_lo[transfunc2(3)]);

                                score = (score < max_errors_excl ?
                                        score + totalbases - 2*overlapsize // non-overlapping regions count as mismatches
                                        : std::numeric_limits<int>::max()); // too many errors, discard

                                if(score < bestScore[orientation]){
                                    bestScore[orientation] = score;
                                    bestShift[orientation] = shift;
                                }

                                return true;
                            }else{
                                //printf("%d, %d %d %d max_errors_excl\n", queryIndex, shift, overlapsize, max_errors_excl);
                                return false;
                            }
                        };

                        //initialize threadlocal smem array with subject
                        for(int i = 0; i < encodedSequencePitchInInts2BitHiLo; i += 1) {
                            mySequence[no_bank_conflict_index(i)] = subjectBackup[identity(i)];
                        }

                        unsigned int* mySequence_hi = mySequence;
                        unsigned int* mySequence_lo = mySequence + no_bank_conflict_index(subjectints / 2);

                        for(int shift = 0; shift < subjectbases - minoverlap + 1; shift += 1) {
                            const int overlapsize = min(subjectbases - shift, querybases);

                            bool b = handle_shift(shift, overlapsize,
                                            mySequence_hi, mySequence_lo, no_bank_conflict_index,
                                            subjectints,
                                            queryBackup_hi, queryBackup_lo, no_bank_conflict_index);
                            if(!b){
                                break;
                            }
                        }

                        const int queryoverlapbegin_incl = max(-bestShift[orientation], 0);
                        const int queryoverlapend_excl = min(querybases, subjectbases - bestShift[orientation]);
                        overlapsize[orientation] = queryoverlapend_excl - queryoverlapbegin_incl;
                        opnr[orientation] = bestScore[orientation] - totalbases + 2*overlapsize[orientation];
                    }

                    const BestAlignment_t flag = alignmentComparator(
                        overlapsize[0],
                        overlapsize[1],
                        opnr[0],
                        opnr[1],
                        bestShift[0] != -querybases,
                        bestShift[1] != -querybases,
                        subjectbases,
                        querybases
                    );

                    d_alignment_best_alignment_flags[candidateIndex] = flag;
                    d_alignment_overlaps[candidateIndex] = flag == BestAlignment_t::Forward ? overlapsize[0] : overlapsize[1];
                    d_alignment_shifts[candidateIndex] = flag == BestAlignment_t::Forward ? bestShift[0] : bestShift[1];
                    d_alignment_nOps[candidateIndex] = flag == BestAlignment_t::Forward ? opnr[0] : opnr[1];
                    d_alignment_isValid[candidateIndex] = flag == BestAlignment_t::Forward ? bestShift[0] != -querybases : bestShift[1] != -querybases;
                }else{
                    d_alignment_best_alignment_flags[candidateIndex] = BestAlignment_t::None;
                    d_alignment_isValid[candidateIndex] = false;
                }
            }
        }
    }



    /*
        Uses 1 thread per candidate to compute the alignment of anchor|candidate and anchor|revc-candidate
        Compares both alignments and keeps the better one

        Sequences are stored in registers
    */

    template<int blocksize, int maxValidIntsPerSequence>
    __global__
    void
    popcount_shifted_hamming_distance_reg_kernel(
                const unsigned int* __restrict__ subjectDataHiLoTransposed,
                const unsigned int* __restrict__ candidateDataHiLoTransposed,
                const int* __restrict__ subjectSequencesLength,
                const int* __restrict__ candidateSequencesLength,
                BestAlignment_t* __restrict__ bestAlignmentFlags,
                int* __restrict__ alignment_overlaps,
                int* __restrict__ alignment_shifts,
                int* __restrict__ alignment_nOps,
                bool* __restrict__ alignment_isValid,
                const int* __restrict__ d_anchorIndicesOfCandidates,
                const int* __restrict__ numAnchorsPtr,
                const int* __restrict__ numCandidatesPtr,
                const bool* __restrict__ anchorContainsN,
                bool removeAmbiguousAnchors,
                const bool* __restrict__ candidateContainsN,
                bool removeAmbiguousCandidates,
                size_t encodedSequencePitchInInts2BitHiLo,
                int min_overlap,
                float maxErrorRate,
                float min_overlap_ratio,
                float estimatedNucleotideErrorRate){

        static_assert(maxValidIntsPerSequence % 2 == 0, ""); //2bithilo has even number of ints


        const int n_subjects = *numAnchorsPtr;
        const int n_candidates = *numCandidatesPtr;

        auto popcount = [](auto i){return __popc(i);};

        auto hammingdistanceHiLoReg = [&](
                            const auto& lhi,
                            const auto& llo,
                            const auto& rhi,
                            const auto& rlo,
                            int lhi_bitcount,
                            int rhi_bitcount,
                            int max_errors){

            constexpr int N = maxValidIntsPerSequence / 2;

            const int overlap_bitcount = std::min(lhi_bitcount, rhi_bitcount);

            if(overlap_bitcount == 0)
                return max_errors+1;

            const int partitions = SDIV(overlap_bitcount, (8 * sizeof(unsigned int)));
            const int remaining_bitcount = partitions * sizeof(unsigned int) * 8 - overlap_bitcount;

            int result = 0;

            #pragma unroll 
            for(int i = 0; i < N - 1; i++){
                if(i < partitions - 1 && result < max_errors){
                    const unsigned int hixor = lhi[i] ^ rhi[i];
                    const unsigned int loxor = llo[i] ^ rlo[i];
                    const unsigned int bits = hixor | loxor;
                    result += popcount(bits);
                }
            }

            if(result >= max_errors)
                return result;

            // i == partitions - 1

            #pragma unroll 
            for(int i = N-1; i >= 0; i--){
                if(partitions - 1 == i){
                    const unsigned int mask = remaining_bitcount == 0 ? 0xFFFFFFFF : 0xFFFFFFFF << (remaining_bitcount);
                    const unsigned int hixor = lhi[i] ^ rhi[i];
                    const unsigned int loxor = llo[i] ^ rlo[i];
                    const unsigned int bits = hixor | loxor;
                    result += popcount(bits & mask);
                }
            }

            return result;
        };

        auto maskBitArray = [](auto& uintarrayHi, auto& uintarrayLo, int keeplength){
            //only keep the first keeplength bits, set remaining bits to 0
            constexpr int N = maxValidIntsPerSequence / 2;

            const int unusedInts = N - SDIV(keeplength, 32);
            if(unusedInts > 0){
                #pragma unroll
                for(int i = 0; i < N; ++i){
                    if(i >= N-unusedInts){
                        uintarrayHi[i] = 0;
                        uintarrayLo[i] = 0;
                    }
                }
            }

            const int unusedBitsInt = SDIV(keeplength, 32) * 32 - keeplength;

            if(unusedBitsInt != 0){
                #pragma unroll
                for(int i = 0; i < N - 1; ++i){
                    if(i == N-unusedInts-1){
                        unsigned int mask = ~((1u << unusedBitsInt)-1);
                        uintarrayHi[i] &= mask;
                        uintarrayLo[i] &= mask;
                        break;
                    }
                }
            }
        };

        auto shiftBitArrayLeftBy1 = [](auto& uintarray){
            constexpr int shift = 1;
            static_assert(shift < 32, "");

            constexpr int N = maxValidIntsPerSequence / 2;    
            #pragma unroll
            for(int i = 0; i < N - 1; i += 1) {
                const unsigned int a = uintarray[i];
                const unsigned int b = uintarray[i+1];
    
                uintarray[i] = (a << shift) | (b >> (8 * sizeof(unsigned int) - shift));
            }
    
            uintarray[N-1] <<= shift;
        };

        auto hammingDistanceWithShift = [&](bool doShift, int overlapsize, int max_errors,
                                    auto& shiftptr_hi, auto& shiftptr_lo,
                                    const auto& otherptr_hi, const auto& otherptr_lo
                                    ){

            if(doShift){
                shiftBitArrayLeftBy1(shiftptr_hi);
                shiftBitArrayLeftBy1(shiftptr_lo);
            }

            const int score = hammingdistanceHiLoReg(shiftptr_hi,
                                                shiftptr_lo,
                                                otherptr_hi,
                                                otherptr_lo,
                                                overlapsize,
                                                overlapsize,
                                                max_errors);

            return score;
        };

        auto alignmentComparator = [&] (int fwd_alignment_overlap,
            int revc_alignment_overlap,
            int fwd_alignment_nops,
            int revc_alignment_nops,
            bool fwd_alignment_isvalid,
            bool revc_alignment_isvalid,
            int subjectlength,
            int querylength)->BestAlignment_t{

            return choose_best_alignment(
                fwd_alignment_overlap,
                revc_alignment_overlap,
                fwd_alignment_nops,
                revc_alignment_nops,
                fwd_alignment_isvalid,
                revc_alignment_isvalid,
                subjectlength,
                querylength,
                min_overlap_ratio,
                min_overlap,
                estimatedNucleotideErrorRate * 4.0f
            );
        };


        unsigned int subjectBackupHi[maxValidIntsPerSequence / 2];
        unsigned int subjectBackupLo[maxValidIntsPerSequence / 2];
        unsigned int queryBackupHi[maxValidIntsPerSequence / 2];
        unsigned int queryBackupLo[maxValidIntsPerSequence / 2];
        unsigned int mySequenceHi[maxValidIntsPerSequence / 2];
        unsigned int mySequenceLo[maxValidIntsPerSequence / 2];

        auto reverseComplementQuery = [&](int querylength, int validInts){
            auto reverse_complement_int = [](auto n) {
                n = ((n >> 1) & 0x55555555) | ((n << 1) & 0xaaaaaaaa);
                n = ((n >> 2) & 0x33333333) | ((n << 2) & 0xcccccccc);
                n = ((n >> 4) & 0x0f0f0f0f) | ((n << 4) & 0xf0f0f0f0);
                n = ((n >> 8) & 0x00ff00ff) | ((n << 8) & 0xff00ff00);
                n = ((n >> 16) & 0x0000ffff) | ((n << 16) & 0xffff0000);
                return ~n;
            };

            constexpr int N = maxValidIntsPerSequence / 2;

            #pragma unroll
            for(int i = 0; i < N/2; ++i){
                const unsigned int hifront = reverse_complement_int(queryBackupHi[i]);
                const unsigned int hiback = reverse_complement_int(queryBackupHi[N - 1 - i]);
                queryBackupHi[i] = hiback;
                queryBackupHi[N - 1 - i] = hifront;
    
                const unsigned int lofront = reverse_complement_int(queryBackupLo[i]);
                const unsigned int loback = reverse_complement_int(queryBackupLo[N - 1 - i]);
                queryBackupLo[i] = loback;
                queryBackupLo[N - 1 - i] = lofront;
            }

            if(N % 2 == 1){
                constexpr int middleindex = N/2;
                queryBackupHi[middleindex] = reverse_complement_int(queryBackupHi[middleindex]);
                queryBackupLo[middleindex] = reverse_complement_int(queryBackupLo[middleindex]);
            }

            //fix unused data

            const int unusedInts = N - getEncodedNumInts2BitHiLo(querylength) / 2;
            if(unusedInts > 0){
                for(int iter = 0; iter < unusedInts; iter++){
                    #pragma unroll
                    for(int i = 0; i < N-1; ++i){
                        queryBackupHi[i] = queryBackupHi[i+1];
                        queryBackupLo[i] = queryBackupLo[i+1];
                    }
                }
            }

            const int unusedBitsInt = SDIV(querylength, 8 * sizeof(unsigned int)) * 8 * sizeof(unsigned int) - querylength;

            if(unusedBitsInt != 0){
                #pragma unroll
                for(int i = 0; i < N - 1; ++i){
                    queryBackupHi[i] = (queryBackupHi[i] << unusedBitsInt) | (queryBackupHi[i+1] >> (8 * sizeof(unsigned int) - unusedBitsInt));
                    queryBackupLo[i] = (queryBackupLo[i] << unusedBitsInt) | (queryBackupLo[i+1] >> (8 * sizeof(unsigned int) - unusedBitsInt));
                }
    
                queryBackupHi[N-1] <<= unusedBitsInt;
                queryBackupLo[N-1] <<= unusedBitsInt;
            }
        };

        for(int candidateIndex = threadIdx.x + blocksize * blockIdx.x; candidateIndex < n_candidates; candidateIndex += blocksize * gridDim.x){

            if(!(removeAmbiguousCandidates && candidateContainsN[candidateIndex])){

                const int subjectIndex = d_anchorIndicesOfCandidates[candidateIndex];  

                if(!(removeAmbiguousAnchors && anchorContainsN[subjectIndex])){

                    const int subjectbases = subjectSequencesLength[subjectIndex];
                    const int querybases = candidateSequencesLength[candidateIndex];

                    const unsigned int* subjectptr = subjectDataHiLoTransposed + std::size_t(subjectIndex);

                    #pragma unroll 
                    for(int i = 0; i < maxValidIntsPerSequence / 2; i++){
                        subjectBackupHi[i] = subjectptr[(i) * n_subjects];
                        subjectBackupLo[i] = subjectptr[(i + maxValidIntsPerSequence / 2) * n_subjects];
                    }

                    maskBitArray(subjectBackupHi, subjectBackupLo, subjectbases);

                    const unsigned int* candidateptr = candidateDataHiLoTransposed + std::size_t(candidateIndex);

                    //save query in reg

                    #pragma unroll 
                    for(int i = 0; i < maxValidIntsPerSequence / 2; i++){
                        queryBackupHi[i] = candidateptr[i * n_candidates];
                        queryBackupLo[i] = candidateptr[(i + maxValidIntsPerSequence / 2) * n_candidates];
                    }

                    maskBitArray(queryBackupHi, queryBackupLo, querybases);

                    //begin SHD algorithm

                    const int subjectints = getEncodedNumInts2BitHiLo(subjectbases);
                    const int queryints = getEncodedNumInts2BitHiLo(querybases);
                    const int totalbases = subjectbases + querybases;
                    const int minoverlap = max(min_overlap, int(float(subjectbases) * min_overlap_ratio));

                    int bestScore[2];
                    int bestShift[2];
                    int overlapsize[2];
                    int opnr[2];

                    #pragma unroll
                    for(int orientation = 0; orientation < 2; orientation++){
                        const bool isReverseComplement = orientation == 1;

                        if(isReverseComplement){
                            reverseComplementQuery(querybases, queryints);
                        }

                        bestScore[orientation] = totalbases;     // score is number of mismatches
                        bestShift[orientation] = -querybases;    // shift of query relative to subject. shift < 0 if query begins before subject

                        auto handle_shift = [&](int shift, int overlapsize,
                                                auto& shiftptr_hi, auto& shiftptr_lo,
                                                const auto& otherptr_hi, const auto& otherptr_lo){

                            //const int max_errors = int(float(overlapsize) * maxErrorRate);
                            const int max_errors_excl = min(int(float(overlapsize) * maxErrorRate),
                                                            bestScore[orientation] - totalbases + 2*overlapsize);

                            if(max_errors_excl > 0){

                                int score = hammingDistanceWithShift(shift != 0, overlapsize, max_errors_excl,
                                                    shiftptr_hi, shiftptr_lo,
                                                    otherptr_hi, otherptr_lo);

                                
                                // printf("%d, %d %d %d --- ", queryIndex, shift, overlapsize, score);

                                // printf("%d %d %d %d | %d %d %d %d --- ", 
                                //     shiftptr_hi[0], shiftptr_hi[1], shiftptr_hi[2], shiftptr_hi[3],
                                //     shiftptr_lo[0], shiftptr_lo[1], shiftptr_lo[2], shiftptr_lo[3]);

                                // printf("%d %d %d %d | %d %d %d %d\n", 
                                //     otherptr_hi[0], otherptr_hi[1], otherptr_hi[2], otherptr_hi[3],
                                //     otherptr_lo[0], otherptr_lo[1], otherptr_lo[2], otherptr_lo[3]);

                                score = (score < max_errors_excl ?
                                        score + totalbases - 2*overlapsize // non-overlapping regions count as mismatches
                                        : std::numeric_limits<int>::max()); // too many errors, discard

                                if(score < bestScore[orientation]){
                                    bestScore[orientation] = score;
                                    bestShift[orientation] = shift;
                                }

                                return true;
                            }else{
                                //printf("%d, %d %d %d max_errors_excl\n", queryIndex, shift, overlapsize, max_errors_excl);

                                return false;
                            }
                        };

                        #pragma unroll 
                        for(int i = 0; i < maxValidIntsPerSequence / 2; i++){
                            mySequenceHi[i] = subjectBackupHi[i];
                            mySequenceLo[i] = subjectBackupLo[i];
                        }

                        for(int shift = 0; shift < subjectbases - minoverlap + 1; shift += 1) {
                            const int overlapsize = min(subjectbases - shift, querybases);

                            bool b = handle_shift(
                                shift, overlapsize,
                                mySequenceHi, mySequenceLo,
                                queryBackupHi, queryBackupLo
                            );
                            if(!b){
                                break;
                            }
                        }

                        //initialize threadlocal smem array with query
                        #pragma unroll 
                        for(int i = 0; i < maxValidIntsPerSequence / 2; i++){
                            mySequenceHi[i] = queryBackupHi[i];
                            mySequenceLo[i] = queryBackupLo[i];
                        }

                        for(int shift = -1; shift >= -querybases + minoverlap; shift -= 1) {
                            const int overlapsize = min(subjectbases, querybases + shift);

                            bool b = handle_shift(
                                shift, overlapsize,
                                mySequenceHi, mySequenceLo,
                                subjectBackupHi, subjectBackupLo
                            );
                            if(!b){
                                break;
                            }
                        }

                        const int queryoverlapbegin_incl = max(-bestShift[orientation], 0);
                        const int queryoverlapend_excl = min(querybases, subjectbases - bestShift[orientation]);
                        overlapsize[orientation] = queryoverlapend_excl - queryoverlapbegin_incl;
                        opnr[orientation] = bestScore[orientation] - totalbases + 2*overlapsize[orientation];
                    }

                    const BestAlignment_t flag = alignmentComparator(
                        overlapsize[0],
                        overlapsize[1],
                        opnr[0],
                        opnr[1],
                        bestShift[0] != -querybases,
                        bestShift[1] != -querybases,
                        subjectbases,
                        querybases
                    );

                    bestAlignmentFlags[candidateIndex] = flag;
                    alignment_overlaps[candidateIndex] = flag == BestAlignment_t::Forward ? overlapsize[0] : overlapsize[1];
                    alignment_shifts[candidateIndex] = flag == BestAlignment_t::Forward ? bestShift[0] : bestShift[1];
                    alignment_nOps[candidateIndex] = flag == BestAlignment_t::Forward ? opnr[0] : opnr[1];
                    alignment_isValid[candidateIndex] = flag == BestAlignment_t::Forward ? bestShift[0] != -querybases : bestShift[1] != -querybases;
                }else{
                    bestAlignmentFlags[candidateIndex] = BestAlignment_t::None;
                    alignment_isValid[candidateIndex] = false;
                }
            }else{
                bestAlignmentFlags[candidateIndex] = BestAlignment_t::None;
                alignment_isValid[candidateIndex] = false;
            }
        }
    }










    template<int blocksize, int maxValidIntsPerSequence>
    __global__
    void
    popcount_rightshifted_hamming_distance_reg_kernel(
                const unsigned int* __restrict__ subjectDataHiLoTransposed,
                const unsigned int* __restrict__ candidateDataHiLoTransposed,
                int* __restrict__ d_alignment_overlaps,
                int* __restrict__ d_alignment_shifts,
                int* __restrict__ d_alignment_nOps,
                bool* __restrict__ d_alignment_isValid,
                BestAlignment_t* __restrict__ d_alignment_best_alignment_flags,
                const int* __restrict__ subjectSequencesLength,
                const int* __restrict__ candidateSequencesLength,
                const int* __restrict__ candidates_per_subject_prefixsum,
                const int* __restrict__ tiles_per_subject_prefixsum,
                const int* __restrict__ numAnchorsPtr,
                const int* __restrict__ numCandidatesPtr,
                const bool* __restrict__ anchorContainsN,
                bool removeAmbiguousAnchors,
                const bool* __restrict__ candidateContainsN,
                bool removeAmbiguousCandidates,
                int encodedSequencePitchInInts2BitHiLo,
                int min_overlap,
                float maxErrorRate,
                float min_overlap_ratio,
                float estimatedNucleotideErrorRate){

        const int n_subjects = *numAnchorsPtr;
        const int n_candidates = *numCandidatesPtr;


        auto no_bank_conflict_index = [](int logical_index) -> int {
            return logical_index * blockDim.x;
        };

        auto identity = [](auto logical_index){
            return logical_index;
        };

        auto popcount = [](auto i){return __popc(i);};

        auto hammingDistanceWithShift = [&](bool doShift, int overlapsize, int max_errors,
                                    unsigned int* shiftptr_hi, unsigned int* shiftptr_lo, auto transfunc1,
                                    int shiftptr_size,
                                    const unsigned int* otherptr_hi, const unsigned int* otherptr_lo,
                                    auto transfunc2){

            if(doShift){
                shiftBitArrayLeftBy<1>(shiftptr_hi, shiftptr_size / 2, transfunc1);
                shiftBitArrayLeftBy<1>(shiftptr_lo, shiftptr_size / 2, transfunc1);
            }

            const int score = hammingdistanceHiLo(shiftptr_hi,
                                                shiftptr_lo,
                                                otherptr_hi,
                                                otherptr_lo,
                                                overlapsize,
                                                overlapsize,
                                                max_errors,
                                                transfunc1,
                                                transfunc2,
                                                popcount);

            return score;
        };


        auto alignmentComparator = [&] (int fwd_alignment_overlap,
            int revc_alignment_overlap,
            int fwd_alignment_nops,
            int revc_alignment_nops,
            bool fwd_alignment_isvalid,
            bool revc_alignment_isvalid,
            int subjectlength,
            int querylength)->BestAlignment_t{

            return choose_best_alignment(
                fwd_alignment_overlap,
                revc_alignment_overlap,
                fwd_alignment_nops,
                revc_alignment_nops,
                fwd_alignment_isvalid,
                revc_alignment_isvalid,
                subjectlength,
                querylength,
                min_overlap_ratio,
                min_overlap,
                estimatedNucleotideErrorRate * 4.0f
            );
        };

        // sizeof(char) * (max_sequence_bytes * num_tiles   // tiles share the subject
        //                    + max_sequence_bytes * num_threads // each thread works with its own candidate
        //                    + max_sequence_bytes * num_threads) // each thread needs memory to shift a sequence
        extern __shared__ unsigned int sharedmemory[];

        //set up shared memory pointers

        const int tiles = (blockDim.x * gridDim.x) / tilesize;
        const int globalTileId = (blockDim.x * blockIdx.x + threadIdx.x) / tilesize;
        const int localTileId = (threadIdx.x) / tilesize;
        const int tilesPerBlock = blockDim.x / tilesize;
        const int laneInTile = threadIdx.x % tilesize;
        const int requiredTiles = tiles_per_subject_prefixsum[n_subjects];

        unsigned int* const subjectBackupsBegin = sharedmemory; // per tile shared memory to store subject
        unsigned int* const queryBackupsBegin = subjectBackupsBegin + encodedSequencePitchInInts2BitHiLo * tilesPerBlock; // per thread shared memory to store query
        unsigned int* const mySequencesBegin = queryBackupsBegin + encodedSequencePitchInInts2BitHiLo * blockDim.x; // per thread shared memory to store shifted sequence

        unsigned int* const subjectBackup = subjectBackupsBegin + encodedSequencePitchInInts2BitHiLo * localTileId; // accesed via identity
        unsigned int* const queryBackup = queryBackupsBegin + threadIdx.x; // accesed via no_bank_conflict_index
        unsigned int* const mySequence = mySequencesBegin + threadIdx.x; // accesed via no_bank_conflict_index

        for(int logicalTileId = globalTileId; logicalTileId < requiredTiles ; logicalTileId += tiles){

            const int subjectIndex = thrust::distance(tiles_per_subject_prefixsum,
                                                    thrust::lower_bound(
                                                        thrust::seq,
                                                        tiles_per_subject_prefixsum,
                                                        tiles_per_subject_prefixsum + n_subjects + 1,
                                                        logicalTileId + 1))-1;

            const int candidatesBeforeThisSubject = candidates_per_subject_prefixsum[subjectIndex];
            const int maxCandidateIndex_excl = candidates_per_subject_prefixsum[subjectIndex+1];
            //const int tilesForThisSubject = tiles_per_subject_prefixsum[subjectIndex + 1] - tiles_per_subject_prefixsum[subjectIndex];
            const int tileForThisSubject = logicalTileId - tiles_per_subject_prefixsum[subjectIndex];
            const int candidateIndex = candidatesBeforeThisSubject + tileForThisSubject * tilesize + laneInTile;

            const int subjectbases = subjectSequencesLength[subjectIndex];
            const int subjectints = SequenceHelpers::getEncodedNumInts2BitHiLo(subjectbases);
            const unsigned int* subjectptr = subjectDataHiLo + std::size_t(subjectIndex) * encodedSequencePitchInInts2BitHiLo;

            //save subject in shared memory (in parallel, per tile)
            for(int lane = laneInTile; lane < encodedSequencePitchInInts2BitHiLo; lane += tilesize) {
                subjectBackup[identity(lane)] = subjectptr[lane];
                //transposed
                //subjectBackup[identity(lane)] = ((unsigned int*)(subjectptr))[lane * n_subjects];
            }

            cg::tiled_partition<tilesize>(cg::this_thread_block()).sync();


            if(candidateIndex < maxCandidateIndex_excl){
                if(!(removeAmbiguousAnchors && anchorContainsN[subjectIndex]) && !(removeAmbiguousCandidates && candidateContainsN[candidateIndex])){
                    const int querybases = candidateSequencesLength[candidateIndex];
                    const int queryints = SequenceHelpers::getEncodedNumInts2BitHiLo(querybases);
                    const int totalbases = subjectbases + querybases;
                    const int minoverlap = max(min_overlap, int(float(subjectbases) * min_overlap_ratio));

                    const unsigned int* candidateptr = candidateDataHiLoTransposed + std::size_t(candidateIndex);

                    //save query in shared memory
                    for(int i = 0; i < encodedSequencePitchInInts2BitHiLo; i += 1) {
                        //queryBackup[no_bank_conflict_index(i)] = ((unsigned int*)(candidateptr))[i];
                        //transposed
                        queryBackup[no_bank_conflict_index(i)] = candidateptr[i * n_candidates];
                    }

                    const unsigned int* const queryBackup_hi = queryBackup;
                    const unsigned int* const queryBackup_lo = queryBackup + no_bank_conflict_index(queryints/2);

                    int bestScore[2];
                    int bestShift[2];
                    int overlapsize[2];
                    int opnr[2];

                    #pragma unroll
                    for(int orientation = 0; orientation < 2; orientation++){
                        const bool isReverseComplement = orientation == 1;

                        if(isReverseComplement) {
                            SequenceHelpers::reverseComplementSequenceInplace2BitHiLo(queryBackup, querybases, no_bank_conflict_index);
                        }

                        //begin SHD algorithm

                        bestScore[orientation] = totalbases;     // score is number of mismatches
                        bestShift[orientation] = -querybases;    // shift of query relative to subject. shift < 0 if query begins before subject

                        auto handle_shift = [&](int shift, int overlapsize,
                                                    unsigned int* shiftptr_hi, unsigned int* shiftptr_lo, auto transfunc1,
                                                    int shiftptr_size,
                                                    const unsigned int* otherptr_hi, const unsigned int* otherptr_lo,
                                                    auto transfunc2){

                            //const int max_errors = int(float(overlapsize) * maxErrorRate);
                            const int max_errors_excl = min(int(float(overlapsize) * maxErrorRate),
                            bestScore[orientation] - totalbases + 2*overlapsize);

                            if(max_errors_excl > 0){

                                int score = hammingDistanceWithShift(shift != 0, overlapsize, max_errors_excl,
                                                    shiftptr_hi,shiftptr_lo, transfunc1,
                                                    shiftptr_size,
                                                    otherptr_hi, otherptr_lo, transfunc2);

                                

                                // printf("%d, %d %d %d --- ", queryIndex, shift, overlapsize, score);

                                // printf("%d %d %d %d | %d %d %d %d --- ", 
                                //     shiftptr_hi[transfunc1(0)], shiftptr_hi[transfunc1(1)], shiftptr_hi[transfunc1(2)], shiftptr_hi[transfunc1(3)],
                                //     shiftptr_lo[transfunc1(0)], shiftptr_lo[transfunc1(1)], shiftptr_lo[transfunc1(2)], shiftptr_lo[transfunc1(3)]);

                                // printf("%d %d %d %d | %d %d %d %d\n", 
                                //     otherptr_hi[transfunc2(0)], otherptr_hi[transfunc2(1)], otherptr_hi[transfunc2(2)], otherptr_hi[transfunc2(3)],
                                //     otherptr_lo[transfunc2(0)], otherptr_lo[transfunc2(1)], otherptr_lo[transfunc2(2)], otherptr_lo[transfunc2(3)]);

                                score = (score < max_errors_excl ?
                                        score + totalbases - 2*overlapsize // non-overlapping regions count as mismatches
                                        : std::numeric_limits<int>::max()); // too many errors, discard

                                if(score < bestScore[orientation]){
                                    bestScore[orientation] = score;
                                    bestShift[orientation] = shift;
                                }

                                return true;
                            }else{
                                //printf("%d, %d %d %d max_errors_excl\n", queryIndex, shift, overlapsize, max_errors_excl);
                                return false;
                            }
                        };

                        //initialize threadlocal smem array with subject
                        for(int i = 0; i < encodedSequencePitchInInts2BitHiLo; i += 1) {
                            mySequence[no_bank_conflict_index(i)] = subjectBackup[identity(i)];
                        }

                        unsigned int* mySequence_hi = mySequence;
                        unsigned int* mySequence_lo = mySequence + no_bank_conflict_index(subjectints / 2);

                        for(int shift = 0; shift < subjectbases - minoverlap + 1; shift += 1) {
                            const int overlapsize = min(subjectbases - shift, querybases);

                            bool b = handle_shift(shift, overlapsize,
                                            mySequence_hi, mySequence_lo, no_bank_conflict_index,
                                            subjectints,
                                            queryBackup_hi, queryBackup_lo, no_bank_conflict_index);
                            if(!b){
                                break;
                            }
                        }

                        const int queryoverlapbegin_incl = max(-bestShift[orientation], 0);
                        const int queryoverlapend_excl = min(querybases, subjectbases - bestShift[orientation]);
                        overlapsize[orientation] = queryoverlapend_excl - queryoverlapbegin_incl;
                        opnr[orientation] = bestScore[orientation] - totalbases + 2*overlapsize[orientation];
                    }

                    // if(candidateIndex == 8){
                    //     printf("(%d, %d, %d, %d) (%d, %d, %d, %d)", 
                    //         overlapsize[0], bestShift[0], opnr[0], bestShift[0] != -querybases,
                    //         overlapsize[1], bestShift[1], opnr[1], bestShift[1] != -querybases);
                    // }

                    const BestAlignment_t flag = alignmentComparator(
                        overlapsize[0],
                        overlapsize[1],
                        opnr[0],
                        opnr[1],
                        bestShift[0] != -querybases,
                        bestShift[1] != -querybases,
                        subjectbases,
                        querybases
                    );

                    d_alignment_best_alignment_flags[candidateIndex] = flag;
                    d_alignment_overlaps[candidateIndex] = flag == BestAlignment_t::Forward ? overlapsize[0] : overlapsize[1];
                    d_alignment_shifts[candidateIndex] = flag == BestAlignment_t::Forward ? bestShift[0] : bestShift[1];
                    d_alignment_nOps[candidateIndex] = flag == BestAlignment_t::Forward ? opnr[0] : opnr[1];
                    d_alignment_isValid[candidateIndex] = flag == BestAlignment_t::Forward ? bestShift[0] != -querybases : bestShift[1] != -querybases;
                }else{
                    d_alignment_best_alignment_flags[candidateIndex] = BestAlignment_t::None;
                    d_alignment_isValid[candidateIndex] = false;
                }
            }
        }
    }



    /*
        Uses 1 thread per candidate to compute the alignment of anchor|candidate and anchor|revc-candidate
        Compares both alignments and keeps the better one

        Sequences are stored in registers
    */

    template<int blocksize, int maxValidIntsPerSequence>
    __global__
    void
    popcount_shifted_hamming_distance_reg_kernel(
                const unsigned int* __restrict__ subjectDataHiLoTransposed,
                const unsigned int* __restrict__ candidateDataHiLoTransposed,
                const int* __restrict__ subjectSequencesLength,
                const int* __restrict__ candidateSequencesLength,
                BestAlignment_t* __restrict__ bestAlignmentFlags,
                int* __restrict__ alignment_overlaps,
                int* __restrict__ alignment_shifts,
                int* __restrict__ alignment_nOps,
                bool* __restrict__ alignment_isValid,
                const int* __restrict__ d_anchorIndicesOfCandidates,
                const int* __restrict__ numAnchorsPtr,
                const int* __restrict__ numCandidatesPtr,
                const bool* __restrict__ anchorContainsN,
                bool removeAmbiguousAnchors,
                const bool* __restrict__ candidateContainsN,
                bool removeAmbiguousCandidates,
                size_t encodedSequencePitchInInts2BitHiLo,
                int min_overlap,
                float maxErrorRate,
                float min_overlap_ratio,
                float estimatedNucleotideErrorRate){

        static_assert(maxValidIntsPerSequence % 2 == 0, ""); //2bithilo has even number of ints


        const int n_subjects = *numAnchorsPtr;
        const int n_candidates = *numCandidatesPtr;

        auto popcount = [](auto i){return __popc(i);};

        auto hammingdistanceHiLoReg = [&](
                            const auto& lhi,
                            const auto& llo,
                            const auto& rhi,
                            const auto& rlo,
                            int lhi_bitcount,
                            int rhi_bitcount,
                            int max_errors){

            constexpr int N = maxValidIntsPerSequence / 2;

            const int overlap_bitcount = std::min(lhi_bitcount, rhi_bitcount);

            if(overlap_bitcount == 0)
                return max_errors+1;

            const int partitions = SDIV(overlap_bitcount, (8 * sizeof(unsigned int)));
            const int remaining_bitcount = partitions * sizeof(unsigned int) * 8 - overlap_bitcount;

            int result = 0;

            #pragma unroll 
            for(int i = 0; i < N - 1; i++){
                if(i < partitions - 1 && result < max_errors){
                    const unsigned int hixor = lhi[i] ^ rhi[i];
                    const unsigned int loxor = llo[i] ^ rlo[i];
                    const unsigned int bits = hixor | loxor;
                    result += popcount(bits);
                }
            }

            if(result >= max_errors)
                return result;

            // i == partitions - 1

            #pragma unroll 
            for(int i = N-1; i >= 0; i--){
                if(partitions - 1 == i){
                    const unsigned int mask = remaining_bitcount == 0 ? 0xFFFFFFFF : 0xFFFFFFFF << (remaining_bitcount);
                    const unsigned int hixor = lhi[i] ^ rhi[i];
                    const unsigned int loxor = llo[i] ^ rlo[i];
                    const unsigned int bits = hixor | loxor;
                    result += popcount(bits & mask);
                }
            }

            return result;
        };

        auto maskBitArray = [](auto& uintarrayHi, auto& uintarrayLo, int keeplength){
            //only keep the first keeplength bits, set remaining bits to 0
            constexpr int N = maxValidIntsPerSequence / 2;

            const int unusedInts = N - SDIV(keeplength, 32);
            if(unusedInts > 0){
                #pragma unroll
                for(int i = 0; i < N; ++i){
                    if(i >= N-unusedInts){
                        uintarrayHi[i] = 0;
                        uintarrayLo[i] = 0;
                    }
                }
            }

            const int unusedBitsInt = SDIV(keeplength, 32) * 32 - keeplength;

            if(unusedBitsInt != 0){
                #pragma unroll
                for(int i = 0; i < N - 1; ++i){
                    if(i == N-unusedInts-1){
                        unsigned int mask = ~((1u << unusedBitsInt)-1);
                        uintarrayHi[i] &= mask;
                        uintarrayLo[i] &= mask;
                        break;
                    }
                }
            }
        };

        auto shiftBitArrayLeftBy1 = [](auto& uintarray){
            constexpr int shift = 1;
            static_assert(shift < 32, "");

            constexpr int N = maxValidIntsPerSequence / 2;    
            #pragma unroll
            for(int i = 0; i < N - 1; i += 1) {
                const unsigned int a = uintarray[i];
                const unsigned int b = uintarray[i+1];
    
                uintarray[i] = (a << shift) | (b >> (8 * sizeof(unsigned int) - shift));
            }
    
            uintarray[N-1] <<= shift;
        };

        auto hammingDistanceWithShift = [&](bool doShift, int overlapsize, int max_errors,
                                    auto& shiftptr_hi, auto& shiftptr_lo,
                                    const auto& otherptr_hi, const auto& otherptr_lo
                                    ){

            if(doShift){
                shiftBitArrayLeftBy1(shiftptr_hi);
                shiftBitArrayLeftBy1(shiftptr_lo);
            }

            const int score = hammingdistanceHiLoReg(shiftptr_hi,
                                                shiftptr_lo,
                                                otherptr_hi,
                                                otherptr_lo,
                                                overlapsize,
                                                overlapsize,
                                                max_errors);

            return score;
        };

        auto alignmentComparator = [&] (int fwd_alignment_overlap,
            int revc_alignment_overlap,
            int fwd_alignment_nops,
            int revc_alignment_nops,
            bool fwd_alignment_isvalid,
            bool revc_alignment_isvalid,
            int subjectlength,
            int querylength)->BestAlignment_t{

            return choose_best_alignment(
                fwd_alignment_overlap,
                revc_alignment_overlap,
                fwd_alignment_nops,
                revc_alignment_nops,
                fwd_alignment_isvalid,
                revc_alignment_isvalid,
                subjectlength,
                querylength,
                min_overlap_ratio,
                min_overlap,
                estimatedNucleotideErrorRate * 4.0f
            );
        };


        unsigned int subjectBackupHi[maxValidIntsPerSequence / 2];
        unsigned int subjectBackupLo[maxValidIntsPerSequence / 2];
        unsigned int queryBackupHi[maxValidIntsPerSequence / 2];
        unsigned int queryBackupLo[maxValidIntsPerSequence / 2];
        unsigned int mySequenceHi[maxValidIntsPerSequence / 2];
        unsigned int mySequenceLo[maxValidIntsPerSequence / 2];

        auto reverseComplementQuery = [&](int querylength, int validInts){

            constexpr int N = maxValidIntsPerSequence / 2;

            #pragma unroll
            for(int i = 0; i < N/2; ++i){
                const unsigned int hifront = SequenceHelpers::reverseComplementInt2BitHiLoHalf(queryBackupHi[i]);
                const unsigned int hiback = SequenceHelpers::reverseComplementInt2BitHiLoHalf(queryBackupHi[N - 1 - i]);
                queryBackupHi[i] = hiback;
                queryBackupHi[N - 1 - i] = hifront;
    
                const unsigned int lofront = SequenceHelpers::reverseComplementInt2BitHiLoHalf(queryBackupLo[i]);
                const unsigned int loback = SequenceHelpers::reverseComplementInt2BitHiLoHalf(queryBackupLo[N - 1 - i]);
                queryBackupLo[i] = loback;
                queryBackupLo[N - 1 - i] = lofront;
            }

            if(N % 2 == 1){
                constexpr int middleindex = N/2;
                queryBackupHi[middleindex] = SequenceHelpers::reverseComplementInt2BitHiLoHalf(queryBackupHi[middleindex]);
                queryBackupLo[middleindex] = SequenceHelpers::reverseComplementInt2BitHiLoHalf(queryBackupLo[middleindex]);
            }

            //fix unused data

            const int unusedInts = N - SequenceHelpers::getEncodedNumInts2BitHiLo(querylength) / 2;
            if(unusedInts > 0){
                for(int iter = 0; iter < unusedInts; iter++){
                    #pragma unroll
                    for(int i = 0; i < N-1; ++i){
                        queryBackupHi[i] = queryBackupHi[i+1];
                        queryBackupLo[i] = queryBackupLo[i+1];
                    }
                }
            }

            const int unusedBitsInt = SDIV(querylength, 8 * sizeof(unsigned int)) * 8 * sizeof(unsigned int) - querylength;

            if(unusedBitsInt != 0){
                #pragma unroll
                for(int i = 0; i < N - 1; ++i){
                    queryBackupHi[i] = (queryBackupHi[i] << unusedBitsInt) | (queryBackupHi[i+1] >> (8 * sizeof(unsigned int) - unusedBitsInt));
                    queryBackupLo[i] = (queryBackupLo[i] << unusedBitsInt) | (queryBackupLo[i+1] >> (8 * sizeof(unsigned int) - unusedBitsInt));
                }
    
                queryBackupHi[N-1] <<= unusedBitsInt;
                queryBackupLo[N-1] <<= unusedBitsInt;
            }
        };

        for(int candidateIndex = threadIdx.x + blocksize * blockIdx.x; candidateIndex < n_candidates; candidateIndex += blocksize * gridDim.x){

            if(!(removeAmbiguousCandidates && candidateContainsN[candidateIndex])){

                const int subjectIndex = d_anchorIndicesOfCandidates[candidateIndex];  

                if(!(removeAmbiguousAnchors && anchorContainsN[subjectIndex])){

                    const int subjectbases = subjectSequencesLength[subjectIndex];
                    const int querybases = candidateSequencesLength[candidateIndex];

                    const unsigned int* subjectptr = subjectDataHiLoTransposed + std::size_t(subjectIndex);

                    #pragma unroll 
                    for(int i = 0; i < maxValidIntsPerSequence / 2; i++){
                        subjectBackupHi[i] = subjectptr[(i) * n_subjects];
                        subjectBackupLo[i] = subjectptr[(i + maxValidIntsPerSequence / 2) * n_subjects];
                    }

                    maskBitArray(subjectBackupHi, subjectBackupLo, subjectbases);

                    const unsigned int* candidateptr = candidateDataHiLoTransposed + std::size_t(candidateIndex);

                    //save query in reg

                    #pragma unroll 
                    for(int i = 0; i < maxValidIntsPerSequence / 2; i++){
                        queryBackupHi[i] = candidateptr[i * n_candidates];
                        queryBackupLo[i] = candidateptr[(i + maxValidIntsPerSequence / 2) * n_candidates];
                    }

                    maskBitArray(queryBackupHi, queryBackupLo, querybases);

                    //begin SHD algorithm

                    const int subjectints = SequenceHelpers::getEncodedNumInts2BitHiLo(subjectbases);
                    const int queryints = SequenceHelpers::getEncodedNumInts2BitHiLo(querybases);
                    const int totalbases = subjectbases + querybases;
                    const int minoverlap = max(min_overlap, int(float(subjectbases) * min_overlap_ratio));

                    int bestScore[2];
                    int bestShift[2];
                    int overlapsize[2];
                    int opnr[2];

                    #pragma unroll
                    for(int orientation = 0; orientation < 2; orientation++){
                        const bool isReverseComplement = orientation == 1;

                        if(isReverseComplement){
                            reverseComplementQuery(querybases, queryints);
                        }

                        bestScore[orientation] = totalbases;     // score is number of mismatches
                        bestShift[orientation] = -querybases;    // shift of query relative to subject. shift < 0 if query begins before subject

                        auto handle_shift = [&](int shift, int overlapsize,
                                                auto& shiftptr_hi, auto& shiftptr_lo,
                                                const auto& otherptr_hi, const auto& otherptr_lo){

                            //const int max_errors = int(float(overlapsize) * maxErrorRate);
                            const int max_errors_excl = min(int(float(overlapsize) * maxErrorRate),
                                                            bestScore[orientation] - totalbases + 2*overlapsize);

                            if(max_errors_excl > 0){

                                int score = hammingDistanceWithShift(shift != 0, overlapsize, max_errors_excl,
                                                    shiftptr_hi, shiftptr_lo,
                                                    otherptr_hi, otherptr_lo);

                                
                                // printf("%d, %d %d %d --- ", queryIndex, shift, overlapsize, score);

                                // printf("%d %d %d %d | %d %d %d %d --- ", 
                                //     shiftptr_hi[0], shiftptr_hi[1], shiftptr_hi[2], shiftptr_hi[3],
                                //     shiftptr_lo[0], shiftptr_lo[1], shiftptr_lo[2], shiftptr_lo[3]);

                                // printf("%d %d %d %d | %d %d %d %d\n", 
                                //     otherptr_hi[0], otherptr_hi[1], otherptr_hi[2], otherptr_hi[3],
                                //     otherptr_lo[0], otherptr_lo[1], otherptr_lo[2], otherptr_lo[3]);

                                score = (score < max_errors_excl ?
                                        score + totalbases - 2*overlapsize // non-overlapping regions count as mismatches
                                        : std::numeric_limits<int>::max()); // too many errors, discard

                                if(score < bestScore[orientation]){
                                    bestScore[orientation] = score;
                                    bestShift[orientation] = shift;
                                }

                                return true;
                            }else{
                                //printf("%d, %d %d %d max_errors_excl\n", queryIndex, shift, overlapsize, max_errors_excl);

                                return false;
                            }
                        };

                        #pragma unroll 
                        for(int i = 0; i < maxValidIntsPerSequence / 2; i++){
                            mySequenceHi[i] = subjectBackupHi[i];
                            mySequenceLo[i] = subjectBackupLo[i];
                        }

                        for(int shift = 0; shift < subjectbases - minoverlap + 1; shift += 1) {
                            const int overlapsize = min(subjectbases - shift, querybases);

                            bool b = handle_shift(
                                shift, overlapsize,
                                mySequenceHi, mySequenceLo,
                                queryBackupHi, queryBackupLo
                            );
                            if(!b){
                                break;
                            }
                        }

                        //initialize threadlocal smem array with query
                        #pragma unroll 
                        for(int i = 0; i < maxValidIntsPerSequence / 2; i++){
                            mySequenceHi[i] = queryBackupHi[i];
                            mySequenceLo[i] = queryBackupLo[i];
                        }

                        for(int shift = -1; shift >= -querybases + minoverlap; shift -= 1) {
                            const int overlapsize = min(subjectbases, querybases + shift);

                            bool b = handle_shift(
                                shift, overlapsize,
                                mySequenceHi, mySequenceLo,
                                subjectBackupHi, subjectBackupLo
                            );
                            if(!b){
                                break;
                            }
                        }

                        const int queryoverlapbegin_incl = max(-bestShift[orientation], 0);
                        const int queryoverlapend_excl = min(querybases, subjectbases - bestShift[orientation]);
                        overlapsize[orientation] = queryoverlapend_excl - queryoverlapbegin_incl;
                        opnr[orientation] = bestScore[orientation] - totalbases + 2*overlapsize[orientation];
                    }

                    const BestAlignment_t flag = alignmentComparator(
                        overlapsize[0],
                        overlapsize[1],
                        opnr[0],
                        opnr[1],
                        bestShift[0] != -querybases,
                        bestShift[1] != -querybases,
                        subjectbases,
                        querybases
                    );

                    bestAlignmentFlags[candidateIndex] = flag;
                    alignment_overlaps[candidateIndex] = flag == BestAlignment_t::Forward ? overlapsize[0] : overlapsize[1];
                    alignment_shifts[candidateIndex] = flag == BestAlignment_t::Forward ? bestShift[0] : bestShift[1];
                    alignment_nOps[candidateIndex] = flag == BestAlignment_t::Forward ? opnr[0] : opnr[1];
                    alignment_isValid[candidateIndex] = flag == BestAlignment_t::Forward ? bestShift[0] != -querybases : bestShift[1] != -querybases;
                }else{
                    bestAlignmentFlags[candidateIndex] = BestAlignment_t::None;
                    alignment_isValid[candidateIndex] = false;
                }
            }else{
                bestAlignmentFlags[candidateIndex] = BestAlignment_t::None;
                alignment_isValid[candidateIndex] = false;
            }
        }
    }










    template<int blocksize, int maxValidIntsPerSequence>
    __global__
    void
    popcount_rightshifted_hamming_distance_reg_kernel(
                const unsigned int* __restrict__ subjectDataHiLoTransposed,
                const unsigned int* __restrict__ candidateDataHiLoTransposed,
                const int* __restrict__ subjectSequencesLength,
                const int* __restrict__ candidateSequencesLength,
                BestAlignment_t* __restrict__ bestAlignmentFlags,
                int* __restrict__ alignment_overlaps,
                int* __restrict__ alignment_shifts,
                int* __restrict__ alignment_nOps,
                bool* __restrict__ alignment_isValid,
                const int* __restrict__ d_anchorIndicesOfCandidates,
                const int* __restrict__ numAnchorsPtr,
                const int* __restrict__ numCandidatesPtr,
                const bool* __restrict__ anchorContainsN,
                bool removeAmbiguousAnchors,
                const bool* __restrict__ candidateContainsN,
                bool removeAmbiguousCandidates,
                size_t encodedSequencePitchInInts2BitHiLo,
                int min_overlap,
                float maxErrorRate,
                float min_overlap_ratio,
                float estimatedNucleotideErrorRate){

        static_assert(maxValidIntsPerSequence % 2 == 0, ""); //2bithilo has even number of ints


        const int n_subjects = *numAnchorsPtr;
        const int n_candidates = *numCandidatesPtr;

        auto popcount = [](auto i){return __popc(i);};

        auto hammingdistanceHiLoReg = [&](
                            const auto& lhi,
                            const auto& llo,
                            const auto& rhi,
                            const auto& rlo,
                            int lhi_bitcount,
                            int rhi_bitcount,
                            int max_errors){

            constexpr int N = maxValidIntsPerSequence / 2;

            const int overlap_bitcount = std::min(lhi_bitcount, rhi_bitcount);

            if(overlap_bitcount == 0)
                return max_errors+1;

            const int partitions = SDIV(overlap_bitcount, (8 * sizeof(unsigned int)));
            const int remaining_bitcount = partitions * sizeof(unsigned int) * 8 - overlap_bitcount;

            int result = 0;

            #pragma unroll 
            for(int i = 0; i < N - 1; i++){
                if(i < partitions - 1 && result < max_errors){
                    const unsigned int hixor = lhi[i] ^ rhi[i];
                    const unsigned int loxor = llo[i] ^ rlo[i];
                    const unsigned int bits = hixor | loxor;
                    result += popcount(bits);
                }
            }

            if(result >= max_errors)
                return result;

            // i == partitions - 1

            #pragma unroll 
            for(int i = N-1; i >= 0; i--){
                if(partitions - 1 == i){
                    const unsigned int mask = remaining_bitcount == 0 ? 0xFFFFFFFF : 0xFFFFFFFF << (remaining_bitcount);
                    const unsigned int hixor = lhi[i] ^ rhi[i];
                    const unsigned int loxor = llo[i] ^ rlo[i];
                    const unsigned int bits = hixor | loxor;
                    result += popcount(bits & mask);
                }
            }

            return result;
        };

        auto maskBitArray = [](auto& uintarrayHi, auto& uintarrayLo, int keeplength){
            //only keep the first keeplength bits, set remaining bits to 0
            constexpr int N = maxValidIntsPerSequence / 2;

            const int unusedInts = N - SDIV(keeplength, 32);
            if(unusedInts > 0){
                #pragma unroll
                for(int i = 0; i < N; ++i){
                    if(i >= N-unusedInts){
                        uintarrayHi[i] = 0;
                        uintarrayLo[i] = 0;
                    }
                }
            }

            const int unusedBitsInt = SDIV(keeplength, 32) * 32 - keeplength;

            if(unusedBitsInt != 0){
                #pragma unroll
                for(int i = 0; i < N - 1; ++i){
                    if(i == N-unusedInts-1){
                        unsigned int mask = ~((1u << unusedBitsInt)-1);
                        uintarrayHi[i] &= mask;
                        uintarrayLo[i] &= mask;
                        break;
                    }
                }
            }
        };

        auto shiftBitArrayLeftBy1 = [](auto& uintarray){
            constexpr int shift = 1;
            static_assert(shift < 32, "");

            constexpr int N = maxValidIntsPerSequence / 2;    
            #pragma unroll
            for(int i = 0; i < N - 1; i += 1) {
                const unsigned int a = uintarray[i];
                const unsigned int b = uintarray[i+1];
    
                uintarray[i] = (a << shift) | (b >> (8 * sizeof(unsigned int) - shift));
            }
    
            uintarray[N-1] <<= shift;
        };

        auto hammingDistanceWithShift = [&](bool doShift, int overlapsize, int max_errors,
                                    auto& shiftptr_hi, auto& shiftptr_lo,
                                    const auto& otherptr_hi, const auto& otherptr_lo
                                    ){

            if(doShift){
                shiftBitArrayLeftBy1(shiftptr_hi);
                shiftBitArrayLeftBy1(shiftptr_lo);
            }

            const int score = hammingdistanceHiLoReg(shiftptr_hi,
                                                shiftptr_lo,
                                                otherptr_hi,
                                                otherptr_lo,
                                                overlapsize,
                                                overlapsize,
                                                max_errors);

            return score;
        };

        auto alignmentComparator = [&] (int fwd_alignment_overlap,
            int revc_alignment_overlap,
            int fwd_alignment_nops,
            int revc_alignment_nops,
            bool fwd_alignment_isvalid,
            bool revc_alignment_isvalid,
            int subjectlength,
            int querylength)->BestAlignment_t{

            return choose_best_alignment(
                fwd_alignment_overlap,
                revc_alignment_overlap,
                fwd_alignment_nops,
                revc_alignment_nops,
                fwd_alignment_isvalid,
                revc_alignment_isvalid,
                subjectlength,
                querylength,
                min_overlap_ratio,
                min_overlap,
                estimatedNucleotideErrorRate * 4.0f
            );
        };


        unsigned int subjectBackupHi[maxValidIntsPerSequence / 2];
        unsigned int subjectBackupLo[maxValidIntsPerSequence / 2];
        unsigned int queryBackupHi[maxValidIntsPerSequence / 2];
        unsigned int queryBackupLo[maxValidIntsPerSequence / 2];
        unsigned int mySequenceHi[maxValidIntsPerSequence / 2];
        unsigned int mySequenceLo[maxValidIntsPerSequence / 2];

        auto reverseComplementQuery = [&](int querylength, int validInts){

            constexpr int N = maxValidIntsPerSequence / 2;

            #pragma unroll
            for(int i = 0; i < N/2; ++i){
                const unsigned int hifront = SequenceHelpers::reverseComplementInt2BitHiLoHalf(queryBackupHi[i]);
                const unsigned int hiback = SequenceHelpers::reverseComplementInt2BitHiLoHalf(queryBackupHi[N - 1 - i]);
                queryBackupHi[i] = hiback;
                queryBackupHi[N - 1 - i] = hifront;
    
                const unsigned int lofront = SequenceHelpers::reverseComplementInt2BitHiLoHalf(queryBackupLo[i]);
                const unsigned int loback = SequenceHelpers::reverseComplementInt2BitHiLoHalf(queryBackupLo[N - 1 - i]);
                queryBackupLo[i] = loback;
                queryBackupLo[N - 1 - i] = lofront;
            }

            if(N % 2 == 1){
                constexpr int middleindex = N/2;
                queryBackupHi[middleindex] = SequenceHelpers::reverseComplementInt2BitHiLoHalf(queryBackupHi[middleindex]);
                queryBackupLo[middleindex] = SequenceHelpers::reverseComplementInt2BitHiLoHalf(queryBackupLo[middleindex]);
            }

            //fix unused data

            const int unusedInts = N - SequenceHelpers::getEncodedNumInts2BitHiLo(querylength) / 2;
            if(unusedInts > 0){
                for(int iter = 0; iter < unusedInts; iter++){
                    #pragma unroll
                    for(int i = 0; i < N-1; ++i){
                        queryBackupHi[i] = queryBackupHi[i+1];
                        queryBackupLo[i] = queryBackupLo[i+1];
                    }
                }
            }

            const int unusedBitsInt = SDIV(querylength, 8 * sizeof(unsigned int)) * 8 * sizeof(unsigned int) - querylength;

            if(unusedBitsInt != 0){
                #pragma unroll
                for(int i = 0; i < N - 1; ++i){
                    queryBackupHi[i] = (queryBackupHi[i] << unusedBitsInt) | (queryBackupHi[i+1] >> (8 * sizeof(unsigned int) - unusedBitsInt));
                    queryBackupLo[i] = (queryBackupLo[i] << unusedBitsInt) | (queryBackupLo[i+1] >> (8 * sizeof(unsigned int) - unusedBitsInt));
                }
    
                queryBackupHi[N-1] <<= unusedBitsInt;
                queryBackupLo[N-1] <<= unusedBitsInt;
            }
        };

        for(int candidateIndex = threadIdx.x + blocksize * blockIdx.x; candidateIndex < n_candidates; candidateIndex += blocksize * gridDim.x){

            if(!(removeAmbiguousCandidates && candidateContainsN[candidateIndex])){

                const int subjectIndex = d_anchorIndicesOfCandidates[candidateIndex];  

                if(!(removeAmbiguousAnchors && anchorContainsN[subjectIndex])){

                    const int subjectbases = subjectSequencesLength[subjectIndex];
                    const int querybases = candidateSequencesLength[candidateIndex];

                    const unsigned int* subjectptr = subjectDataHiLoTransposed + std::size_t(subjectIndex);

                    #pragma unroll 
                    for(int i = 0; i < maxValidIntsPerSequence / 2; i++){
                        subjectBackupHi[i] = subjectptr[(i) * n_subjects];
                        subjectBackupLo[i] = subjectptr[(i + maxValidIntsPerSequence / 2) * n_subjects];
                    }

                    maskBitArray(subjectBackupHi, subjectBackupLo, subjectbases);

                    const unsigned int* candidateptr = candidateDataHiLoTransposed + std::size_t(candidateIndex);

                    //save query in reg

                    #pragma unroll 
                    for(int i = 0; i < maxValidIntsPerSequence / 2; i++){
                        queryBackupHi[i] = candidateptr[i * n_candidates];
                        queryBackupLo[i] = candidateptr[(i + maxValidIntsPerSequence / 2) * n_candidates];
                    }

                    maskBitArray(queryBackupHi, queryBackupLo, querybases);

                    //begin SHD algorithm

                    const int subjectints = SequenceHelpers::getEncodedNumInts2BitHiLo(subjectbases);
                    const int queryints = SequenceHelpers::getEncodedNumInts2BitHiLo(querybases);
                    const int totalbases = subjectbases + querybases;
                    const int minoverlap = max(min_overlap, int(float(subjectbases) * min_overlap_ratio));

                    int bestScore[2];
                    int bestShift[2];
                    int overlapsize[2];
                    int opnr[2];

                    #pragma unroll
                    for(int orientation = 0; orientation < 2; orientation++){
                        const bool isReverseComplement = orientation == 1;

                        if(isReverseComplement){
                            reverseComplementQuery(querybases, queryints);
                        }

                        bestScore[orientation] = totalbases;     // score is number of mismatches
                        bestShift[orientation] = -querybases;    // shift of query relative to subject. shift < 0 if query begins before subject

                        auto handle_shift = [&](int shift, int overlapsize,
                                                auto& shiftptr_hi, auto& shiftptr_lo,
                                                const auto& otherptr_hi, const auto& otherptr_lo){

                            //const int max_errors = int(float(overlapsize) * maxErrorRate);
                            const int max_errors_excl = min(int(float(overlapsize) * maxErrorRate),
                                                            bestScore[orientation] - totalbases + 2*overlapsize);

                            if(max_errors_excl > 0){

                                int score = hammingDistanceWithShift(shift != 0, overlapsize, max_errors_excl,
                                                    shiftptr_hi, shiftptr_lo,
                                                    otherptr_hi, otherptr_lo);

                                
                                // printf("%d, %d %d %d --- ", queryIndex, shift, overlapsize, score);

                                // printf("%d %d %d %d | %d %d %d %d --- ", 
                                //     shiftptr_hi[0], shiftptr_hi[1], shiftptr_hi[2], shiftptr_hi[3],
                                //     shiftptr_lo[0], shiftptr_lo[1], shiftptr_lo[2], shiftptr_lo[3]);

                                // printf("%d %d %d %d | %d %d %d %d\n", 
                                //     otherptr_hi[0], otherptr_hi[1], otherptr_hi[2], otherptr_hi[3],
                                //     otherptr_lo[0], otherptr_lo[1], otherptr_lo[2], otherptr_lo[3]);

                                score = (score < max_errors_excl ?
                                        score + totalbases - 2*overlapsize // non-overlapping regions count as mismatches
                                        : std::numeric_limits<int>::max()); // too many errors, discard

                                if(score < bestScore[orientation]){
                                    bestScore[orientation] = score;
                                    bestShift[orientation] = shift;
                                }

                                return true;
                            }else{
                                //printf("%d, %d %d %d max_errors_excl\n", queryIndex, shift, overlapsize, max_errors_excl);

                                return false;
                            }
                        };

                        #pragma unroll 
                        for(int i = 0; i < maxValidIntsPerSequence / 2; i++){
                            mySequenceHi[i] = subjectBackupHi[i];
                            mySequenceLo[i] = subjectBackupLo[i];
                        }

                        for(int shift = 0; shift < subjectbases - minoverlap + 1; shift += 1) {
                            const int overlapsize = min(subjectbases - shift, querybases);

                            bool b = handle_shift(
                                shift, overlapsize,
                                mySequenceHi, mySequenceLo,
                                queryBackupHi, queryBackupLo
                            );
                            if(!b){
                                break;
                            }
                        }

                        const int queryoverlapbegin_incl = max(-bestShift[orientation], 0);
                        const int queryoverlapend_excl = min(querybases, subjectbases - bestShift[orientation]);
                        overlapsize[orientation] = queryoverlapend_excl - queryoverlapbegin_incl;
                        opnr[orientation] = bestScore[orientation] - totalbases + 2*overlapsize[orientation];
                    }

                    // if(candidateIndex == 8){
                    //     printf("(%d, %d, %d, %d) (%d, %d, %d, %d)", 
                    //         overlapsize[0], bestShift[0], opnr[0], bestShift[0] != -querybases,
                    //         overlapsize[1], bestShift[1], opnr[1], bestShift[1] != -querybases);
                    // }

                    const BestAlignment_t flag = alignmentComparator(
                        overlapsize[0],
                        overlapsize[1],
                        opnr[0],
                        opnr[1],
                        bestShift[0] != -querybases,
                        bestShift[1] != -querybases,
                        subjectbases,
                        querybases
                    );

                    bestAlignmentFlags[candidateIndex] = flag;
                    alignment_overlaps[candidateIndex] = flag == BestAlignment_t::Forward ? overlapsize[0] : overlapsize[1];
                    alignment_shifts[candidateIndex] = flag == BestAlignment_t::Forward ? bestShift[0] : bestShift[1];
                    alignment_nOps[candidateIndex] = flag == BestAlignment_t::Forward ? opnr[0] : opnr[1];
                    alignment_isValid[candidateIndex] = flag == BestAlignment_t::Forward ? bestShift[0] != -querybases : bestShift[1] != -querybases;
                }else{
                    bestAlignmentFlags[candidateIndex] = BestAlignment_t::None;
                    alignment_isValid[candidateIndex] = false;
                }
            }else{
                bestAlignmentFlags[candidateIndex] = BestAlignment_t::None;
                alignment_isValid[candidateIndex] = false;
            }
        }
    }





    template<int BLOCKSIZE>
    __global__
    void cuda_filter_alignments_by_mismatchratio_kernel(
                BestAlignment_t* __restrict__ bestAlignmentFlags,
                const int* __restrict__ nOps,
                const int* __restrict__ overlaps,
                const int* __restrict__ d_candidates_per_subject_prefixsum,
                const int* __restrict__ d_numAnchors,
                const int* __restrict__ d_numCandidates,
                float mismatchratioBaseFactor,
                float goodAlignmentsCountThreshold){

        using BlockReduceInt = hipcub::BlockReduce<int, BLOCKSIZE>;

        __shared__ union {
            typename BlockReduceInt::TempStorage intreduce;
            int broadcast[3];
        } temp_storage;

        const int n_subjects = *d_numAnchors;
        //const int n_candidates = *d_numCandidates;


        for(int subjectindex = blockIdx.x; subjectindex < n_subjects; subjectindex += gridDim.x) {

            const int candidatesForSubject = d_candidates_per_subject_prefixsum[subjectindex+1]
                                            - d_candidates_per_subject_prefixsum[subjectindex];

            const int firstIndex = d_candidates_per_subject_prefixsum[subjectindex];

            //printf("subjectindex %d\n", subjectindex);

            int counts[3]{0,0,0};

            //if(threadIdx.x == 0){
            //    printf("my_n_indices %d\n", my_n_indices);
            //}

            for(int index = threadIdx.x; index < candidatesForSubject; index += blockDim.x) {

                const int candidate_index = firstIndex + index;
                if(bestAlignmentFlags[candidate_index] != BestAlignment_t::None) {

                    const int alignment_overlap = overlaps[candidate_index];
                    const int alignment_nops = nOps[candidate_index];

                    const float mismatchratio = float(alignment_nops) / alignment_overlap;

                    if(mismatchratio >= 4 * mismatchratioBaseFactor) {
                        bestAlignmentFlags[candidate_index] = BestAlignment_t::None;
                    }else{

                            #pragma unroll
                        for(int i = 2; i <= 4; i++) {
                            counts[i-2] += (mismatchratio < i * mismatchratioBaseFactor);
                        }
                    }

                }
            }

            //accumulate counts over block
                #pragma unroll
            for(int i = 0; i < 3; i++) {
                counts[i] = BlockReduceInt(temp_storage.intreduce).Sum(counts[i]);
                __syncthreads();
            }

            //broadcast accumulated counts to block
            if(threadIdx.x == 0) {
                #pragma unroll
                for(int i = 0; i < 3; i++) {
                    temp_storage.broadcast[i] = counts[i];
                    //printf("count[%d] = %d\n", i, counts[i]);
                }
                //printf("mismatchratioBaseFactor %f, goodAlignmentsCountThreshold %f\n", mismatchratioBaseFactor, goodAlignmentsCountThreshold);
            }

            __syncthreads();

            #pragma unroll
            for(int i = 0; i < 3; i++) {
                counts[i] = temp_storage.broadcast[i];
            }

            float mismatchratioThreshold = 0;
            if (counts[0] >= goodAlignmentsCountThreshold) {
                mismatchratioThreshold = 2 * mismatchratioBaseFactor;
            } else if (counts[1] >= goodAlignmentsCountThreshold) {
                mismatchratioThreshold = 3 * mismatchratioBaseFactor;
            } else if (counts[2] >= goodAlignmentsCountThreshold) {
                mismatchratioThreshold = 4 * mismatchratioBaseFactor;
            } else {
                mismatchratioThreshold = -1.0f;                         //this will invalidate all alignments for subject
                //mismatchratioThreshold = 4 * mismatchratioBaseFactor; //use alignments from every bin
                //mismatchratioThreshold = 1.1f;
            }

            // Invalidate all alignments for subject with mismatchratio >= mismatchratioThreshold
            for(int index = threadIdx.x; index < candidatesForSubject; index += blockDim.x) {
                const int candidate_index = firstIndex + index;
                if(bestAlignmentFlags[candidate_index] != BestAlignment_t::None) {

                    const int alignment_overlap = overlaps[candidate_index];
                    const int alignment_nops = nOps[candidate_index];

                    const float mismatchratio = float(alignment_nops) / alignment_overlap;

                    const bool doRemove = mismatchratio >= mismatchratioThreshold;
                    if(doRemove){
                        bestAlignmentFlags[candidate_index] = BestAlignment_t::None;
                    }
                }
            }
        }
    }







    //####################   KERNEL DISPATCH   ####################

    template<int maxValidIntsPerSequence>
    void call_popcount_shifted_hamming_distance_reg_kernel_async(
        void* d_tempstorage,
        size_t& tempstoragebytes,
        int* d_alignment_overlaps,
        int* d_alignment_shifts,
        int* d_alignment_nOps,
        bool* d_alignment_isValid,
        BestAlignment_t* d_alignment_best_alignment_flags,
        const unsigned int* d_subjectSequencesData,
        const unsigned int* d_candidateSequencesData,
        const int* d_subjectSequencesLength,
        const int* d_candidateSequencesLength,
        const int* d_candidates_per_subject_prefixsum,
        const int* d_candidates_per_subject,
        const int* d_anchorIndicesOfCandidates,
        const int* d_numAnchors,
        const int* d_numCandidates,
        const bool* d_anchorContainsN,
        bool removeAmbiguousAnchors,
        const bool* d_candidateContainsN,
        bool removeAmbiguousCandidates,
        int maxNumAnchors,
        int maxNumCandidates,
        int maximumSequenceLength,
        int encodedSequencePitchInInts2Bit,
        int min_overlap,
        float maxErrorRate,
        float min_overlap_ratio,
        float estimatedNucleotideErrorRate,
        hipStream_t stream,
        KernelLaunchHandle& handle){

        const int intsPerSequence2BitHiLo = getEncodedNumInts2BitHiLo(maximumSequenceLength);
        
        
        const std::size_t d_candidateDataHiLoTransposedBytes = SDIV(sizeof(unsigned int) * intsPerSequence2BitHiLo * maxNumCandidates, 512) * 512;
        const std::size_t d_subjectDataHiLoTransposedBytes = SDIV(sizeof(unsigned int) * intsPerSequence2BitHiLo * maxNumAnchors, 512) * 512;
        
        {
            
            const std::size_t requiredTempBytes 
                = d_candidateDataHiLoTransposedBytes
                    + d_subjectDataHiLoTransposedBytes;
            
            if(d_tempstorage == 0){
                tempstoragebytes = requiredTempBytes;
                return;
            }else{
                assert(tempstoragebytes >= requiredTempBytes);
            }
            
        }
        
        //Alias temp storage 
        unsigned int* const d_subjectDataHiLoTransposed = (unsigned int*)d_tempstorage;
        unsigned int* const d_candidateDataHiLoTransposed 
            = (unsigned int*)(((char*)d_subjectDataHiLoTransposed) 
            + d_subjectDataHiLoTransposedBytes);
       

        callConversionKernel2BitTo2BitHiLoNT(
            d_candidateSequencesData,
            encodedSequencePitchInInts2Bit,
            d_candidateDataHiLoTransposed,
            intsPerSequence2BitHiLo,
            d_candidateSequencesLength,
            d_numCandidates,
            maxNumCandidates,
            stream,
            handle
        );

        callConversionKernel2BitTo2BitHiLoNT(
            d_subjectSequencesData,
            encodedSequencePitchInInts2Bit,
            d_subjectDataHiLoTransposed,
            intsPerSequence2BitHiLo,
            d_subjectSequencesLength,
            d_numAnchors,
            maxNumAnchors,
            stream,
            handle
        );
        

        constexpr int blocksize = 128;
        int max_blocks_per_device = 1;

        KernelLaunchConfig kernelLaunchConfig;
        kernelLaunchConfig.threads_per_block = blocksize;
        kernelLaunchConfig.smem = 0;

        auto iter = handle.kernelPropertiesMap.find(KernelId::PopcountSHDReg);
        if(iter == handle.kernelPropertiesMap.end()) {

            std::map<KernelLaunchConfig, KernelProperties> mymap;

            KernelProperties kernelProperties;
            hipOccupancyMaxActiveBlocksPerMultiprocessor(
                &kernelProperties.max_blocks_per_SM,
                popcount_shifted_hamming_distance_reg_kernel<blocksize, maxValidIntsPerSequence>,
                kernelLaunchConfig.threads_per_block, 
                kernelLaunchConfig.smem
            ); CUERR;

            mymap[kernelLaunchConfig] = kernelProperties;
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

            handle.kernelPropertiesMap[KernelId::PopcountSHDReg] = std::move(mymap);
        }else{
            std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
            const KernelProperties& kernelProperties = map[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
        }

        dim3 block(blocksize, 1, 1);
        //const int numBlocks = SDIV(maxNumCandidates, blocksize);
        //dim3 grid(std::min(numBlocks, max_blocks_per_device), 1, 1);
        dim3 grid(max_blocks_per_device, 1, 1);

        popcount_shifted_hamming_distance_reg_kernel<blocksize, maxValidIntsPerSequence>
            <<<grid, block, 0, stream>>>(
                d_subjectDataHiLoTransposed,
                d_candidateDataHiLoTransposed,
                d_subjectSequencesLength,
                d_candidateSequencesLength,
                d_alignment_best_alignment_flags,
                d_alignment_overlaps,
                d_alignment_shifts,
                d_alignment_nOps,
                d_alignment_isValid,
                d_anchorIndicesOfCandidates,
                d_numAnchors,
                d_numCandidates,
                d_anchorContainsN,
                removeAmbiguousAnchors,
                d_candidateContainsN,
                removeAmbiguousCandidates,
                intsPerSequence2BitHiLo, 
                min_overlap,
                maxErrorRate,
                min_overlap_ratio,
                estimatedNucleotideErrorRate
        ); CUERR;

    }

    template<int maxValidIntsPerSequence>
    void call_popcount_rightshifted_hamming_distance_reg_kernel_async(
        void* d_tempstorage,
        size_t& tempstoragebytes,
        int* d_alignment_overlaps,
        int* d_alignment_shifts,
        int* d_alignment_nOps,
        bool* d_alignment_isValid,
        BestAlignment_t* d_alignment_best_alignment_flags,
        const unsigned int* d_subjectSequencesData,
        const unsigned int* d_candidateSequencesData,
        const int* d_subjectSequencesLength,
        const int* d_candidateSequencesLength,
        const int* d_candidates_per_subject_prefixsum,
        const int* d_candidates_per_subject,
        const int* d_anchorIndicesOfCandidates,
        const int* d_numAnchors,
        const int* d_numCandidates,
        const bool* d_anchorContainsN,
        bool removeAmbiguousAnchors,
        const bool* d_candidateContainsN,
        bool removeAmbiguousCandidates,
        int maxNumAnchors,
        int maxNumCandidates,
        int maximumSequenceLength,
        int encodedSequencePitchInInts2Bit,
        int min_overlap,
        float maxErrorRate,
        float min_overlap_ratio,
        float estimatedNucleotideErrorRate,
        hipStream_t stream,
        KernelLaunchHandle& handle){

        const int intsPerSequence2BitHiLo = SequenceHelpers::getEncodedNumInts2BitHiLo(maximumSequenceLength);
        
        
        const std::size_t d_candidateDataHiLoTransposedBytes = SDIV(sizeof(unsigned int) * intsPerSequence2BitHiLo * maxNumCandidates, 512) * 512;
        const std::size_t d_subjectDataHiLoTransposedBytes = SDIV(sizeof(unsigned int) * intsPerSequence2BitHiLo * maxNumAnchors, 512) * 512;
        
        {
            
            const std::size_t requiredTempBytes 
                = d_candidateDataHiLoTransposedBytes
                    + d_subjectDataHiLoTransposedBytes;
            
            if(d_tempstorage == 0){
                tempstoragebytes = requiredTempBytes;
                return;
            }else{
                assert(tempstoragebytes >= requiredTempBytes);
            }
            
        }
        
        //Alias temp storage 
        unsigned int* const d_subjectDataHiLoTransposed = (unsigned int*)d_tempstorage;
        unsigned int* const d_candidateDataHiLoTransposed 
            = (unsigned int*)(((char*)d_subjectDataHiLoTransposed) 
            + d_subjectDataHiLoTransposedBytes);
       

        callConversionKernel2BitTo2BitHiLoNT(
            d_candidateSequencesData,
            encodedSequencePitchInInts2Bit,
            d_candidateDataHiLoTransposed,
            intsPerSequence2BitHiLo,
            d_candidateSequencesLength,
            d_numCandidates,
            maxNumCandidates,
            stream,
            handle
        );

        callConversionKernel2BitTo2BitHiLoNT(
            d_subjectSequencesData,
            encodedSequencePitchInInts2Bit,
            d_subjectDataHiLoTransposed,
            intsPerSequence2BitHiLo,
            d_subjectSequencesLength,
            d_numAnchors,
            maxNumAnchors,
            stream,
            handle
        );
        

        constexpr int blocksize = 128;
        int max_blocks_per_device = 1;

        KernelLaunchConfig kernelLaunchConfig;
        kernelLaunchConfig.threads_per_block = blocksize;
        kernelLaunchConfig.smem = 0;

        auto iter = handle.kernelPropertiesMap.find(KernelId::PopcountRightSHDReg);
        if(iter == handle.kernelPropertiesMap.end()) {

            std::map<KernelLaunchConfig, KernelProperties> mymap;

            KernelProperties kernelProperties;
            hipOccupancyMaxActiveBlocksPerMultiprocessor(
                &kernelProperties.max_blocks_per_SM,
                popcount_rightshifted_hamming_distance_reg_kernel<blocksize, maxValidIntsPerSequence>,
                kernelLaunchConfig.threads_per_block, 
                kernelLaunchConfig.smem
            ); CUERR;

            mymap[kernelLaunchConfig] = kernelProperties;
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

            handle.kernelPropertiesMap[KernelId::PopcountRightSHDReg] = std::move(mymap);
        }else{
            std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
            const KernelProperties& kernelProperties = map[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
        }

        dim3 block(blocksize, 1, 1);
        //const int numBlocks = SDIV(maxNumCandidates, blocksize);
        //dim3 grid(std::min(numBlocks, max_blocks_per_device), 1, 1);
        dim3 grid(max_blocks_per_device, 1, 1);

        popcount_rightshifted_hamming_distance_reg_kernel<blocksize, maxValidIntsPerSequence>
            <<<grid, block, 0, stream>>>(
                d_subjectDataHiLoTransposed,
                d_candidateDataHiLoTransposed,
                d_subjectSequencesLength,
                d_candidateSequencesLength,
                d_alignment_best_alignment_flags,
                d_alignment_overlaps,
                d_alignment_shifts,
                d_alignment_nOps,
                d_alignment_isValid,
                d_anchorIndicesOfCandidates,
                d_numAnchors,
                d_numCandidates,
                d_anchorContainsN,
                removeAmbiguousAnchors,
                d_candidateContainsN,
                removeAmbiguousCandidates,
                intsPerSequence2BitHiLo, 
                min_overlap,
                maxErrorRate,
                min_overlap_ratio,
                estimatedNucleotideErrorRate
        ); CUERR;

    }


    void call_popcount_shifted_hamming_distance_smem_kernel_async(
            void* d_tempstorage,
            size_t& tempstoragebytes,
            int* d_alignment_overlaps,
            int* d_alignment_shifts,
            int* d_alignment_nOps,
            bool* d_alignment_isValid,
            BestAlignment_t* d_alignment_best_alignment_flags,
            const unsigned int* d_subjectSequencesData,
            const unsigned int* d_candidateSequencesData,
            const int* d_subjectSequencesLength,
            const int* d_candidateSequencesLength,
            const int* d_candidates_per_subject_prefixsum,
            const int* d_candidates_per_subject,
            const int* d_anchorIndicesOfCandidates,
            const int* d_numAnchors,
            const int* d_numCandidates,
            const bool* d_anchorContainsN,
            bool removeAmbiguousAnchors,
            const bool* d_candidateContainsN,
            bool removeAmbiguousCandidates,
            int maxNumAnchors,
            int maxNumCandidates,
            int maximumSequenceLength,
            int encodedSequencePitchInInts2Bit,
            int min_overlap,
            float maxErrorRate,
            float min_overlap_ratio,
            float estimatedNucleotideErrorRate,
            hipStream_t stream,
            KernelLaunchHandle& handle){
        
        constexpr int tilesize = 16;
        
        auto getTilesPerSubject = [=] __device__ (int candidates_for_subject){
            return SDIV(candidates_for_subject, tilesize);
        };
        
        hipcub::TransformInputIterator<int,decltype(getTilesPerSubject), const int*>
            d_tiles_per_subject(d_candidates_per_subject,
                            getTilesPerSubject);

        const int intsPerSequence2BitHiLo = getEncodedNumInts2BitHiLo(maximumSequenceLength);
        const int bytesPerSequence2BitHilo = intsPerSequence2BitHiLo * sizeof(unsigned int);
        
        const std::size_t d_candidateDataHiLoTransposedBytes = SDIV(sizeof(unsigned int) * intsPerSequence2BitHiLo * maxNumCandidates, 512) * 512;
        const std::size_t d_subjectDataHiLoBytes = SDIV(sizeof(unsigned int) * intsPerSequence2BitHiLo * maxNumAnchors, 512) * 512;
        const std::size_t d_tiles_per_subject_prefixsumBytes = SDIV(sizeof(int) * (maxNumAnchors+1), 512) * 512;
        std::size_t cubBytes = 0;
        
        // hipcub::DeviceScan::InclusiveSum(
        //     nullptr,
        //     cubBytes,
        //     d_tiles_per_subject,
        //     (int*) nullptr,
        //     maxNumAnchors,
        //     stream
        // );
        
        {

            const std::size_t requiredTempBytes 
                = d_candidateDataHiLoTransposedBytes
                    + d_subjectDataHiLoBytes
                    + d_tiles_per_subject_prefixsumBytes
                    + cubBytes;
            
            if(d_tempstorage == 0){
                tempstoragebytes = requiredTempBytes;
                return;
            }else{
                assert(tempstoragebytes >= requiredTempBytes);
            }
        
        }
        
        //Alias temp storage 
        unsigned int* const d_candidateDataHiLoTransposed = (unsigned int*)d_tempstorage;
        unsigned int* const d_subjectDataHiLo 
            = (unsigned int*)(((char*)d_candidateDataHiLoTransposed) 
                + d_candidateDataHiLoTransposedBytes);
        int* const d_tiles_per_subject_prefixsum
            = (int*)(((char*)d_subjectDataHiLo) 
                + d_subjectDataHiLoBytes);

        callConversionKernel2BitTo2BitHiLoNT(
            d_candidateSequencesData,
            encodedSequencePitchInInts2Bit,
            d_candidateDataHiLoTransposed,
            intsPerSequence2BitHiLo,
            d_candidateSequencesLength,
            d_numCandidates,
            maxNumCandidates,
            stream,
            handle
        );

        callConversionKernel2BitTo2BitHiLoNN(
            d_subjectSequencesData,
            encodedSequencePitchInInts2Bit,
            d_subjectDataHiLo,
            intsPerSequence2BitHiLo,
            d_subjectSequencesLength,
            d_numAnchors,
            maxNumAnchors,
            stream,
            handle
        );

        //calculate d_tiles_per_subject_prefixsum
        helpers::lambda_kernel<<<1, 256, 0, stream>>>([=]__device__(){
            using BlockScan = hipcub::BlockScan<int, 256>;

            __shared__ typename BlockScan::TempStorage temp_storage;

            const int numItems = *d_numAnchors;

            constexpr int ITEMS_PER_THREAD = 4;

            int aggregate = 0;

            const int iters = SDIV(numItems, 256 * ITEMS_PER_THREAD);

            const int threadoffset = ITEMS_PER_THREAD * threadIdx.x;

            if(threadIdx.x == 0){
                d_tiles_per_subject_prefixsum[0] = 0;
            }

            for(int iter = 0; iter < iters; iter++){
                int thread_data[ITEMS_PER_THREAD];

                const int iteroffset = 256 * ITEMS_PER_THREAD * iter;

                #pragma unroll
                for(int k = 0; k < ITEMS_PER_THREAD; k++){
                    if(iteroffset + threadoffset + k < numItems){
                        thread_data[k] = d_tiles_per_subject[iteroffset + threadoffset + k];
                    }else{
                        thread_data[k] = 0;
                    }
                }

                int block_aggregate = 0;
                BlockScan(temp_storage).InclusiveSum(thread_data, thread_data, block_aggregate);

                #pragma unroll
                for(int k = 0; k < ITEMS_PER_THREAD; k++){
                    if(iteroffset + threadoffset + k < numItems){
                        d_tiles_per_subject_prefixsum[1+iteroffset + threadoffset + k] = aggregate + thread_data[k];
                    }
                }

                aggregate += block_aggregate;

                __syncthreads();
            }

            

            // hipcub::LoadDirectBlocked(
            //     threadIdx.x,
            //     d_tiles_per_subject,
            //     thread_data,
            //     numItems,
            //     0
            // )	

            // BlockScan(temp_storage).InclusiveSum(thread_data, thread_data, T &block_aggregate)

            // hipcub::StoreDirectBlocked(
            //     	int 	linear_tid,
            //     OutputIteratorT 	block_itr,
            //     T(&) 	items[ITEMS_PER_THREAD],
            //     int 	valid_items 
            //     )	
        }); CUERR;


        constexpr int blocksize = 128;
        constexpr int tilesPerBlock = blocksize / tilesize;

        const std::size_t smem = sizeof(char) * (bytesPerSequence2BitHilo * tilesPerBlock + bytesPerSequence2BitHilo * blocksize * 2);

        int max_blocks_per_device = 1;

        KernelLaunchConfig kernelLaunchConfig;
        kernelLaunchConfig.threads_per_block = blocksize;
        kernelLaunchConfig.smem = smem;

        auto iter = handle.kernelPropertiesMap.find(KernelId::PopcountRightSHDReg);
        if(iter == handle.kernelPropertiesMap.end()) {

            std::map<KernelLaunchConfig, KernelProperties> mymap;

            KernelProperties kernelProperties;
            hipOccupancyMaxActiveBlocksPerMultiprocessor(
                &kernelProperties.max_blocks_per_SM,
                popcount_rightshifted_hamming_distance_reg_kernel<blocksize, maxValidIntsPerSequence>,
                kernelLaunchConfig.threads_per_block, 
                kernelLaunchConfig.smem
            ); CUERR;

            mymap[kernelLaunchConfig] = kernelProperties;
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

            handle.kernelPropertiesMap[KernelId::PopcountRightSHDReg] = std::move(mymap);
        }else{
            std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
            const KernelProperties& kernelProperties = map[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
        }

        dim3 block(blocksize, 1, 1);
        //const int numBlocks = SDIV(maxNumCandidates, blocksize);
        //dim3 grid(std::min(numBlocks, max_blocks_per_device), 1, 1);
        dim3 grid(max_blocks_per_device, 1, 1);

        popcount_rightshifted_hamming_distance_reg_kernel<blocksize, maxValidIntsPerSequence>
            <<<grid, block, 0, stream>>>(
                d_subjectDataHiLoTransposed,
                d_candidateDataHiLoTransposed,
                d_subjectSequencesLength,
                d_candidateSequencesLength,
                d_alignment_best_alignment_flags,
                d_alignment_overlaps,
                d_alignment_shifts,
                d_alignment_nOps,
                d_alignment_isValid,
                d_anchorIndicesOfCandidates,
                d_numAnchors,
                d_numCandidates,
                d_anchorContainsN,
                removeAmbiguousAnchors,
                d_candidateContainsN,
                removeAmbiguousCandidates,
                intsPerSequence2BitHiLo, 
                min_overlap,
                maxErrorRate,
                min_overlap_ratio,
                estimatedNucleotideErrorRate
        ); CUERR;

    }


    void call_popcount_shifted_hamming_distance_smem_kernel_async(
            void* d_tempstorage,
            size_t& tempstoragebytes,
            int* d_alignment_overlaps,
            int* d_alignment_shifts,
            int* d_alignment_nOps,
            bool* d_alignment_isValid,
            BestAlignment_t* d_alignment_best_alignment_flags,
            const unsigned int* d_subjectSequencesData,
            const unsigned int* d_candidateSequencesData,
            const int* d_subjectSequencesLength,
            const int* d_candidateSequencesLength,
            const int* d_candidates_per_subject_prefixsum,
            const int* d_candidates_per_subject,
            const int* d_anchorIndicesOfCandidates,
            const int* d_numAnchors,
            const int* d_numCandidates,
            const bool* d_anchorContainsN,
            bool removeAmbiguousAnchors,
            const bool* d_candidateContainsN,
            bool removeAmbiguousCandidates,
            int maxNumAnchors,
            int maxNumCandidates,
            int maximumSequenceLength,
            int encodedSequencePitchInInts2Bit,
            int min_overlap,
            float maxErrorRate,
            float min_overlap_ratio,
            float estimatedNucleotideErrorRate,
            hipStream_t stream,
            KernelLaunchHandle& handle){
        
        constexpr int tilesize = 16;
        
        auto getTilesPerSubject = [=] __device__ (int candidates_for_subject){
            return SDIV(candidates_for_subject, tilesize);
        };
        
        hipcub::TransformInputIterator<int,decltype(getTilesPerSubject), const int*>
            d_tiles_per_subject(d_candidates_per_subject,
                            getTilesPerSubject);

        const int intsPerSequence2BitHiLo = SequenceHelpers::getEncodedNumInts2BitHiLo(maximumSequenceLength);
        const int bytesPerSequence2BitHilo = intsPerSequence2BitHiLo * sizeof(unsigned int);
        
        const std::size_t d_candidateDataHiLoTransposedBytes = SDIV(sizeof(unsigned int) * intsPerSequence2BitHiLo * maxNumCandidates, 512) * 512;
        const std::size_t d_subjectDataHiLoBytes = SDIV(sizeof(unsigned int) * intsPerSequence2BitHiLo * maxNumAnchors, 512) * 512;
        const std::size_t d_tiles_per_subject_prefixsumBytes = SDIV(sizeof(int) * (maxNumAnchors+1), 512) * 512;
        std::size_t cubBytes = 0;
        
        // hipcub::DeviceScan::InclusiveSum(
        //     nullptr,
        //     cubBytes,
        //     d_tiles_per_subject,
        //     (int*) nullptr,
        //     maxNumAnchors,
        //     stream
        // );
        
        {

            const std::size_t requiredTempBytes 
                = d_candidateDataHiLoTransposedBytes
                    + d_subjectDataHiLoBytes
                    + d_tiles_per_subject_prefixsumBytes
                    + cubBytes;
            
            if(d_tempstorage == 0){
                tempstoragebytes = requiredTempBytes;
                return;
            }else{
                assert(tempstoragebytes >= requiredTempBytes);
            }
        
        }
        
        //Alias temp storage 
        unsigned int* const d_candidateDataHiLoTransposed = (unsigned int*)d_tempstorage;
        unsigned int* const d_subjectDataHiLo 
            = (unsigned int*)(((char*)d_candidateDataHiLoTransposed) 
                + d_candidateDataHiLoTransposedBytes);
        int* const d_tiles_per_subject_prefixsum
            = (int*)(((char*)d_subjectDataHiLo) 
                + d_subjectDataHiLoBytes);

        callConversionKernel2BitTo2BitHiLoNT(
            d_candidateSequencesData,
            encodedSequencePitchInInts2Bit,
            d_candidateDataHiLoTransposed,
            intsPerSequence2BitHiLo,
            d_candidateSequencesLength,
            d_numCandidates,
            maxNumCandidates,
            stream,
            handle
        );

        callConversionKernel2BitTo2BitHiLoNN(
            d_subjectSequencesData,
            encodedSequencePitchInInts2Bit,
            d_subjectDataHiLo,
            intsPerSequence2BitHiLo,
            d_subjectSequencesLength,
            d_numAnchors,
            maxNumAnchors,
            stream,
            handle
        );

        //calculate d_tiles_per_subject_prefixsum
        helpers::lambda_kernel<<<1, 256, 0, stream>>>([=]__device__(){
            using BlockScan = hipcub::BlockScan<int, 256>;

            __shared__ typename BlockScan::TempStorage temp_storage;

            const int numItems = *d_numAnchors;

            constexpr int ITEMS_PER_THREAD = 4;

            int aggregate = 0;

            const int iters = SDIV(numItems, 256 * ITEMS_PER_THREAD);

            const int threadoffset = ITEMS_PER_THREAD * threadIdx.x;

            if(threadIdx.x == 0){
                d_tiles_per_subject_prefixsum[0] = 0;
            }

            for(int iter = 0; iter < iters; iter++){
                int thread_data[ITEMS_PER_THREAD];

                const int iteroffset = 256 * ITEMS_PER_THREAD * iter;

                #pragma unroll
                for(int k = 0; k < ITEMS_PER_THREAD; k++){
                    if(iteroffset + threadoffset + k < numItems){
                        thread_data[k] = d_tiles_per_subject[iteroffset + threadoffset + k];
                    }else{
                        thread_data[k] = 0;
                    }
                }

                int block_aggregate = 0;
                BlockScan(temp_storage).InclusiveSum(thread_data, thread_data, block_aggregate);

                #pragma unroll
                for(int k = 0; k < ITEMS_PER_THREAD; k++){
                    if(iteroffset + threadoffset + k < numItems){
                        d_tiles_per_subject_prefixsum[1+iteroffset + threadoffset + k] = aggregate + thread_data[k];
                    }
                }

                aggregate += block_aggregate;

                __syncthreads();
            }

            

            // hipcub::LoadDirectBlocked(
            //     threadIdx.x,
            //     d_tiles_per_subject,
            //     thread_data,
            //     numItems,
            //     0
            // )    

            // BlockScan(temp_storage).InclusiveSum(thread_data, thread_data, T &block_aggregate)

            // hipcub::StoreDirectBlocked(
            //         int     linear_tid,
            //     OutputIteratorT     block_itr,
            //     T(&)     items[ITEMS_PER_THREAD],
            //     int     valid_items 
            //     )    
        }); CUERR;


        constexpr int blocksize = 128;
        constexpr int tilesPerBlock = blocksize / tilesize;

        const std::size_t smem = sizeof(char) * (bytesPerSequence2BitHilo * tilesPerBlock + bytesPerSequence2BitHilo * blocksize * 2);

        int max_blocks_per_device = 1;

        KernelLaunchConfig kernelLaunchConfig;
        kernelLaunchConfig.threads_per_block = blocksize;
        kernelLaunchConfig.smem = smem;

        auto iter = handle.kernelPropertiesMap.find(KernelId::PopcountSHDSmem);
        if(iter == handle.kernelPropertiesMap.end()) {

            std::map<KernelLaunchConfig, KernelProperties> mymap;

            #define getProp(blocksize, tilesize) { \
                    KernelLaunchConfig kernelLaunchConfig; \
                    kernelLaunchConfig.threads_per_block = (blocksize); \
                    kernelLaunchConfig.smem = sizeof(char) * (bytesPerSequence2BitHilo * tilesPerBlock + bytesPerSequence2BitHilo * blocksize * 2); \
                    KernelProperties kernelProperties; \
                    hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                        popcount_shifted_hamming_distance_smem_kernel<tilesize>, \
                                kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
                    mymap[kernelLaunchConfig] = kernelProperties; \
            }
            getProp(1, tilesize);
            getProp(32, tilesize);
            getProp(64, tilesize);
            getProp(96, tilesize);
            getProp(128, tilesize);
            getProp(160, tilesize);
            getProp(192, tilesize);
            getProp(224, tilesize);
            getProp(256, tilesize);

            const auto& kernelProperties = mymap[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

            handle.kernelPropertiesMap[KernelId::PopcountSHDSmem] = std::move(mymap);

            #undef getProp
        }else{
            std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
            const KernelProperties& kernelProperties = map[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
        }

        #define mycall popcount_shifted_hamming_distance_smem_kernel<tilesize> \
                                            <<<grid, block, smem, stream>>>( \
                                            d_subjectDataHiLo, \
                                            d_candidateDataHiLoTransposed, \
                                            d_alignment_overlaps, \
                                            d_alignment_shifts, \
                                            d_alignment_nOps, \
                                            d_alignment_isValid, \
                                            d_alignment_best_alignment_flags, \
                                            d_subjectSequencesLength, \
                                            d_candidateSequencesLength, \
                                            d_candidates_per_subject_prefixsum, \
                                            d_tiles_per_subject_prefixsum, \
                                            d_numAnchors, \
                                            d_numCandidates, \
                                            d_anchorContainsN, \
                                            removeAmbiguousAnchors, \
                                            d_candidateContainsN, \
                                            removeAmbiguousCandidates, \
                                            intsPerSequence2BitHiLo, \
                                            min_overlap, \
                                            maxErrorRate, \
                                            min_overlap_ratio, \
                                            estimatedNucleotideErrorRate); CUERR;

        dim3 block(blocksize, 1, 1);
        //dim3 grid(std::min(requiredBlocks, max_blocks_per_device), 1, 1);
        dim3 grid(max_blocks_per_device, 1, 1);

        mycall;

        #undef mycall

    }

    void call_popcount_rightshifted_hamming_distance_smem_kernel_async(
            void* d_tempstorage,
            size_t& tempstoragebytes,
            int* d_alignment_overlaps,
            int* d_alignment_shifts,
            int* d_alignment_nOps,
            bool* d_alignment_isValid,
            BestAlignment_t* d_alignment_best_alignment_flags,
            const unsigned int* d_subjectSequencesData,
            const unsigned int* d_candidateSequencesData,
            const int* d_subjectSequencesLength,
            const int* d_candidateSequencesLength,
            const int* d_candidates_per_subject_prefixsum,
            const int* d_candidates_per_subject,
            const int* d_anchorIndicesOfCandidates,
            const int* d_numAnchors,
            const int* d_numCandidates,
            const bool* d_anchorContainsN,
            bool removeAmbiguousAnchors,
            const bool* d_candidateContainsN,
            bool removeAmbiguousCandidates,
            int maxNumAnchors,
            int maxNumCandidates,
            int maximumSequenceLength,
            int encodedSequencePitchInInts2Bit,
            int min_overlap,
            float maxErrorRate,
            float min_overlap_ratio,
            float estimatedNucleotideErrorRate,
            hipStream_t stream,
            KernelLaunchHandle& handle){
        
        constexpr int tilesize = 16;
        
        auto getTilesPerSubject = [=] __device__ (int candidates_for_subject){
            return SDIV(candidates_for_subject, tilesize);
        };
        
        hipcub::TransformInputIterator<int,decltype(getTilesPerSubject), const int*>
            d_tiles_per_subject(d_candidates_per_subject,
                            getTilesPerSubject);

        const int intsPerSequence2BitHiLo = SequenceHelpers::getEncodedNumInts2BitHiLo(maximumSequenceLength);
        const int bytesPerSequence2BitHilo = intsPerSequence2BitHiLo * sizeof(unsigned int);
        
        const std::size_t d_candidateDataHiLoTransposedBytes = SDIV(sizeof(unsigned int) * intsPerSequence2BitHiLo * maxNumCandidates, 512) * 512;
        const std::size_t d_subjectDataHiLoBytes = SDIV(sizeof(unsigned int) * intsPerSequence2BitHiLo * maxNumAnchors, 512) * 512;
        const std::size_t d_tiles_per_subject_prefixsumBytes = SDIV(sizeof(int) * (maxNumAnchors+1), 512) * 512;
        std::size_t cubBytes = 0;
        
        // hipcub::DeviceScan::InclusiveSum(
        //     nullptr,
        //     cubBytes,
        //     d_tiles_per_subject,
        //     (int*) nullptr,
        //     maxNumAnchors,
        //     stream
        // );
        
        {

            const std::size_t requiredTempBytes 
                = d_candidateDataHiLoTransposedBytes
                    + d_subjectDataHiLoBytes
                    + d_tiles_per_subject_prefixsumBytes
                    + cubBytes;
            
            if(d_tempstorage == 0){
                tempstoragebytes = requiredTempBytes;
                return;
            }else{
                assert(tempstoragebytes >= requiredTempBytes);
            }
        
        }
        
        //Alias temp storage 
        unsigned int* const d_candidateDataHiLoTransposed = (unsigned int*)d_tempstorage;
        unsigned int* const d_subjectDataHiLo 
            = (unsigned int*)(((char*)d_candidateDataHiLoTransposed) 
                + d_candidateDataHiLoTransposedBytes);
        int* const d_tiles_per_subject_prefixsum
            = (int*)(((char*)d_subjectDataHiLo) 
                + d_subjectDataHiLoBytes);

        callConversionKernel2BitTo2BitHiLoNT(
            d_candidateSequencesData,
            encodedSequencePitchInInts2Bit,
            d_candidateDataHiLoTransposed,
            intsPerSequence2BitHiLo,
            d_candidateSequencesLength,
            d_numCandidates,
            maxNumCandidates,
            stream,
            handle
        );

        callConversionKernel2BitTo2BitHiLoNN(
            d_subjectSequencesData,
            encodedSequencePitchInInts2Bit,
            d_subjectDataHiLo,
            intsPerSequence2BitHiLo,
            d_subjectSequencesLength,
            d_numAnchors,
            maxNumAnchors,
            stream,
            handle
        );

        //calculate d_tiles_per_subject_prefixsum
        helpers::lambda_kernel<<<1, 256, 0, stream>>>([=]__device__(){
            using BlockScan = hipcub::BlockScan<int, 256>;

            __shared__ typename BlockScan::TempStorage temp_storage;

            const int numItems = *d_numAnchors;

            constexpr int ITEMS_PER_THREAD = 4;

            int aggregate = 0;

            const int iters = SDIV(numItems, 256 * ITEMS_PER_THREAD);

            const int threadoffset = ITEMS_PER_THREAD * threadIdx.x;

            if(threadIdx.x == 0){
                d_tiles_per_subject_prefixsum[0] = 0;
            }

            for(int iter = 0; iter < iters; iter++){
                int thread_data[ITEMS_PER_THREAD];

                const int iteroffset = 256 * ITEMS_PER_THREAD * iter;

                #pragma unroll
                for(int k = 0; k < ITEMS_PER_THREAD; k++){
                    if(iteroffset + threadoffset + k < numItems){
                        thread_data[k] = d_tiles_per_subject[iteroffset + threadoffset + k];
                    }else{
                        thread_data[k] = 0;
                    }
                }

                int block_aggregate = 0;
                BlockScan(temp_storage).InclusiveSum(thread_data, thread_data, block_aggregate);

                #pragma unroll
                for(int k = 0; k < ITEMS_PER_THREAD; k++){
                    if(iteroffset + threadoffset + k < numItems){
                        d_tiles_per_subject_prefixsum[1+iteroffset + threadoffset + k] = aggregate + thread_data[k];
                    }
                }

                aggregate += block_aggregate;

                __syncthreads();
            }

            

            // hipcub::LoadDirectBlocked(
            //     threadIdx.x,
            //     d_tiles_per_subject,
            //     thread_data,
            //     numItems,
            //     0
            // )    

            // BlockScan(temp_storage).InclusiveSum(thread_data, thread_data, T &block_aggregate)

            // hipcub::StoreDirectBlocked(
            //         int     linear_tid,
            //     OutputIteratorT     block_itr,
            //     T(&)     items[ITEMS_PER_THREAD],
            //     int     valid_items 
            //     )    
        }); CUERR;


        constexpr int blocksize = 128;
        constexpr int tilesPerBlock = blocksize / tilesize;

        const std::size_t smem = sizeof(char) * (bytesPerSequence2BitHilo * tilesPerBlock + bytesPerSequence2BitHilo * blocksize * 2);

        int max_blocks_per_device = 1;

        KernelLaunchConfig kernelLaunchConfig;
        kernelLaunchConfig.threads_per_block = blocksize;
        kernelLaunchConfig.smem = smem;

        auto iter = handle.kernelPropertiesMap.find(KernelId::PopcountRightSHDSmem);
        if(iter == handle.kernelPropertiesMap.end()) {

            std::map<KernelLaunchConfig, KernelProperties> mymap;

            #define getProp(blocksize, tilesize) { \
                    KernelLaunchConfig kernelLaunchConfig; \
                    kernelLaunchConfig.threads_per_block = (blocksize); \
                    kernelLaunchConfig.smem = sizeof(char) * (bytesPerSequence2BitHilo * tilesPerBlock + bytesPerSequence2BitHilo * blocksize * 2); \
                    KernelProperties kernelProperties; \
                    hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                        popcount_rightshifted_hamming_distance_smem_kernel<tilesize>, \
                                kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
                    mymap[kernelLaunchConfig] = kernelProperties; \
            }
            getProp(1, tilesize);
            getProp(32, tilesize);
            getProp(64, tilesize);
            getProp(96, tilesize);
            getProp(128, tilesize);
            getProp(160, tilesize);
            getProp(192, tilesize);
            getProp(224, tilesize);
            getProp(256, tilesize);

            const auto& kernelProperties = mymap[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

            handle.kernelPropertiesMap[KernelId::PopcountSHDSmem] = std::move(mymap);

            #undef getProp
        }else{
            std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
            const KernelProperties& kernelProperties = map[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
        }

        #define mycall popcount_rightshifted_hamming_distance_smem_kernel<tilesize> \
                                            <<<grid, block, smem, stream>>>( \
                                            d_subjectDataHiLo, \
                                            d_candidateDataHiLoTransposed, \
                                            d_alignment_overlaps, \
                                            d_alignment_shifts, \
                                            d_alignment_nOps, \
                                            d_alignment_isValid, \
                                            d_alignment_best_alignment_flags, \
                                            d_subjectSequencesLength, \
                                            d_candidateSequencesLength, \
                                            d_candidates_per_subject_prefixsum, \
                                            d_tiles_per_subject_prefixsum, \
                                            d_numAnchors, \
                                            d_numCandidates, \
                                            d_anchorContainsN, \
                                            removeAmbiguousAnchors, \
                                            d_candidateContainsN, \
                                            removeAmbiguousCandidates, \
                                            intsPerSequence2BitHiLo, \
                                            min_overlap, \
                                            maxErrorRate, \
                                            min_overlap_ratio, \
                                            estimatedNucleotideErrorRate); CUERR;

        dim3 block(blocksize, 1, 1);
        //dim3 grid(std::min(requiredBlocks, max_blocks_per_device), 1, 1);
        dim3 grid(max_blocks_per_device, 1, 1);

        mycall;

        #undef mycall

    }


    void call_popcount_shifted_hamming_distance_kernel_async(
            void* d_tempstorage,
            size_t& tempstoragebytes,
            int* d_alignment_overlaps,
            int* d_alignment_shifts,
            int* d_alignment_nOps,
            bool* d_alignment_isValid,
            BestAlignment_t* d_alignment_best_alignment_flags,
            const unsigned int* d_subjectSequencesData,
            const unsigned int* d_candidateSequencesData,
            const int* d_subjectSequencesLength,
            const int* d_candidateSequencesLength,
            const int* d_candidates_per_subject_prefixsum,
            const int* d_candidates_per_subject,
            const int* d_anchorIndicesOfCandidates,
            const int* d_numAnchors,
            const int* d_numCandidates,
            const bool* d_anchorContainsN,
            bool removeAmbiguousAnchors,
            const bool* d_candidateContainsN,
            bool removeAmbiguousCandidates,
            int maxNumAnchors,
            int maxNumCandidates,
            int maximumSequenceLength,
            int encodedSequencePitchInInts2Bit,
            int min_overlap,
            float maxErrorRate,
            float min_overlap_ratio,
            float estimatedNucleotideErrorRate,
            hipStream_t stream,
            KernelLaunchHandle& handle){

            #define regKernel(intsPerSequence){ \
                call_popcount_shifted_hamming_distance_reg_kernel_async<intsPerSequence>( \
                    d_tempstorage, \
                    tempstoragebytes, \
                    d_alignment_overlaps, \
                    d_alignment_shifts, \
                    d_alignment_nOps, \
                    d_alignment_isValid, \
                    d_alignment_best_alignment_flags, \
                    d_subjectSequencesData, \
                    d_candidateSequencesData, \
                    d_subjectSequencesLength, \
                    d_candidateSequencesLength, \
                    d_candidates_per_subject_prefixsum, \
                    d_candidates_per_subject, \
                    d_anchorIndicesOfCandidates, \
                    d_numAnchors, \
                    d_numCandidates, \
                    d_anchorContainsN, \
                    removeAmbiguousAnchors, \
                    d_candidateContainsN, \
                    removeAmbiguousCandidates, \
                    maxNumAnchors, \
                    maxNumCandidates, \
                    maximumSequenceLength, \
                    encodedSequencePitchInInts2Bit, \
                    min_overlap, \
                    maxErrorRate, \
                    min_overlap_ratio, \
                    estimatedNucleotideErrorRate, \
                    stream, \
                    handle \
                ); \
            };
            
            auto run = [&](){
                if(1 <= maximumSequenceLength && maximumSequenceLength <= 32){
                    
                    constexpr int maxValidIntsPerSequence = 2;
                    regKernel(maxValidIntsPerSequence);
                    
                }else if(33 <= maximumSequenceLength && maximumSequenceLength <= 64){
                    
                    constexpr int maxValidIntsPerSequence = 4;
                    regKernel(maxValidIntsPerSequence);
                    
                }else if(65 <= maximumSequenceLength && maximumSequenceLength <= 96){
                    
                    constexpr int maxValidIntsPerSequence = 6;
                    regKernel(maxValidIntsPerSequence);
                    
                }else if(97 <= maximumSequenceLength && maximumSequenceLength <= 128){
                    
                    constexpr int maxValidIntsPerSequence = 8;
                    regKernel(maxValidIntsPerSequence);
                    
                }else if(129 <= maximumSequenceLength && maximumSequenceLength <= 160){
                    
                    constexpr int maxValidIntsPerSequence = 10;
                    regKernel(maxValidIntsPerSequence);
                    
                }else if(161 <= maximumSequenceLength && maximumSequenceLength <= 192){
                    
                    constexpr int maxValidIntsPerSequence = 12;
                    regKernel(maxValidIntsPerSequence);
                    
                }else if(193 <= maximumSequenceLength && maximumSequenceLength <= 224){
                    
                    constexpr int maxValidIntsPerSequence = 14;
                    regKernel(maxValidIntsPerSequence);
                    
                }else if(225 <= maximumSequenceLength && maximumSequenceLength <= 256){
                    
                    constexpr int maxValidIntsPerSequence = 16;
                    regKernel(maxValidIntsPerSequence);
                    
                }else{
                    
                    call_popcount_shifted_hamming_distance_smem_kernel_async(
                        d_tempstorage,
                        tempstoragebytes,
                        d_alignment_overlaps,
                        d_alignment_shifts,
                        d_alignment_nOps,
                        d_alignment_isValid,
                        d_alignment_best_alignment_flags,
                        d_subjectSequencesData,
                        d_candidateSequencesData,
                        d_subjectSequencesLength,
                        d_candidateSequencesLength,
                        d_candidates_per_subject_prefixsum,
                        d_candidates_per_subject,
                        d_anchorIndicesOfCandidates,
                        d_numAnchors,
                        d_numCandidates,
                        d_anchorContainsN,
                        removeAmbiguousAnchors,
                        d_candidateContainsN,
                        removeAmbiguousCandidates,
                        maxNumAnchors,
                        maxNumCandidates,
                        maximumSequenceLength,
                        encodedSequencePitchInInts2Bit,
                        min_overlap,
                        maxErrorRate,
                        min_overlap_ratio,
                        estimatedNucleotideErrorRate,
                        stream,
                        handle
                    );
                }
            };
            
            if(d_tempstorage == nullptr){
                tempstoragebytes = 0;
                
                run();
                
                return;
            }

            
            run();

        #undef regKernel 
    }



    void call_popcount_rightshifted_hamming_distance_kernel_async(
            void* d_tempstorage,
            size_t& tempstoragebytes,
            int* d_alignment_overlaps,
            int* d_alignment_shifts,
            int* d_alignment_nOps,
            bool* d_alignment_isValid,
            BestAlignment_t* d_alignment_best_alignment_flags,
            const unsigned int* d_subjectSequencesData,
            const unsigned int* d_candidateSequencesData,
            const int* d_subjectSequencesLength,
            const int* d_candidateSequencesLength,
            const int* d_candidates_per_subject_prefixsum,
            const int* d_candidates_per_subject,
            const int* d_anchorIndicesOfCandidates,
            const int* d_numAnchors,
            const int* d_numCandidates,
            const bool* d_anchorContainsN,
            bool removeAmbiguousAnchors,
            const bool* d_candidateContainsN,
            bool removeAmbiguousCandidates,
            int maxNumAnchors,
            int maxNumCandidates,
            int maximumSequenceLength,
            int encodedSequencePitchInInts2Bit,
            int min_overlap,
            float maxErrorRate,
            float min_overlap_ratio,
            float estimatedNucleotideErrorRate,
            hipStream_t stream,
            KernelLaunchHandle& handle){

            #define regKernel(intsPerSequence){ \
                call_popcount_rightshifted_hamming_distance_reg_kernel_async<intsPerSequence>( \
                    d_tempstorage, \
                    tempstoragebytes, \
                    d_alignment_overlaps, \
                    d_alignment_shifts, \
                    d_alignment_nOps, \
                    d_alignment_isValid, \
                    d_alignment_best_alignment_flags, \
                    d_subjectSequencesData, \
                    d_candidateSequencesData, \
                    d_subjectSequencesLength, \
                    d_candidateSequencesLength, \
                    d_candidates_per_subject_prefixsum, \
                    d_candidates_per_subject, \
                    d_anchorIndicesOfCandidates, \
                    d_numAnchors, \
                    d_numCandidates, \
                    d_anchorContainsN, \
                    removeAmbiguousAnchors, \
                    d_candidateContainsN, \
                    removeAmbiguousCandidates, \
                    maxNumAnchors, \
                    maxNumCandidates, \
                    maximumSequenceLength, \
                    encodedSequencePitchInInts2Bit, \
                    min_overlap, \
                    maxErrorRate, \
                    min_overlap_ratio, \
                    estimatedNucleotideErrorRate, \
                    stream, \
                    handle \
                ); \
            };
            
            auto run = [&](){
                if(1 <= maximumSequenceLength && maximumSequenceLength <= 32){
                    
                    constexpr int maxValidIntsPerSequence = 2;
                    regKernel(maxValidIntsPerSequence);
                    
                }else if(33 <= maximumSequenceLength && maximumSequenceLength <= 64){
                    
                    constexpr int maxValidIntsPerSequence = 4;
                    regKernel(maxValidIntsPerSequence);
                    
                }else if(65 <= maximumSequenceLength && maximumSequenceLength <= 96){
                    
                    constexpr int maxValidIntsPerSequence = 6;
                    regKernel(maxValidIntsPerSequence);
                    
                }else if(97 <= maximumSequenceLength && maximumSequenceLength <= 128){
                    
                    constexpr int maxValidIntsPerSequence = 8;
                    regKernel(maxValidIntsPerSequence);
                    
                }else if(129 <= maximumSequenceLength && maximumSequenceLength <= 160){
                    
                    constexpr int maxValidIntsPerSequence = 10;
                    regKernel(maxValidIntsPerSequence);
                    
                }else if(161 <= maximumSequenceLength && maximumSequenceLength <= 192){
                    
                    constexpr int maxValidIntsPerSequence = 12;
                    regKernel(maxValidIntsPerSequence);
                    
                }else if(193 <= maximumSequenceLength && maximumSequenceLength <= 224){
                    
                    constexpr int maxValidIntsPerSequence = 14;
                    regKernel(maxValidIntsPerSequence);
                    
                }else if(225 <= maximumSequenceLength && maximumSequenceLength <= 256){
                    
                    constexpr int maxValidIntsPerSequence = 16;
                    regKernel(maxValidIntsPerSequence);
                    
                }else{
                    
                    call_popcount_rightshifted_hamming_distance_smem_kernel_async(
                        d_tempstorage,
                        tempstoragebytes,
                        d_alignment_overlaps,
                        d_alignment_shifts,
                        d_alignment_nOps,
                        d_alignment_isValid,
                        d_alignment_best_alignment_flags,
                        d_subjectSequencesData,
                        d_candidateSequencesData,
                        d_subjectSequencesLength,
                        d_candidateSequencesLength,
                        d_candidates_per_subject_prefixsum,
                        d_candidates_per_subject,
                        d_anchorIndicesOfCandidates,
                        d_numAnchors,
                        d_numCandidates,
                        d_anchorContainsN,
                        removeAmbiguousAnchors,
                        d_candidateContainsN,
                        removeAmbiguousCandidates,
                        maxNumAnchors,
                        maxNumCandidates,
                        maximumSequenceLength,
                        encodedSequencePitchInInts2Bit,
                        min_overlap,
                        maxErrorRate,
                        min_overlap_ratio,
                        estimatedNucleotideErrorRate,
                        stream,
                        handle
                    );
                }
            };
            
            if(d_tempstorage == nullptr){
                tempstoragebytes = 0;
                
                run();
                
                return;
            }

            
            run();

        #undef regKernel 
    }


    void call_cuda_filter_alignments_by_mismatchratio_kernel_async(
                BestAlignment_t* d_bestAlignmentFlags,
                const int* d_nOps,
                const int* d_overlaps,
                const int* d_candidates_per_subject_prefixsum,
                const int* d_numAnchors,
                const int* d_numCandidates,
                int maxNumAnchors,
                int maxNumCandidates,
                float mismatchratioBaseFactor,
                float goodAlignmentsCountThreshold,
                hipStream_t stream,
                KernelLaunchHandle& handle){

        constexpr int requestedBlocksize = 128;
        const std::size_t smem = 0;

        int max_blocks_per_device = 1;


        auto iter = handle.kernelPropertiesMap.find(KernelId::FilterAlignmentsByMismatchRatio);
        if(iter == handle.kernelPropertiesMap.end()) {

            std::map<KernelLaunchConfig, KernelProperties> mymap;

            #define getProp(blocksize) { \
                    KernelLaunchConfig klc; \
                    klc.threads_per_block = (blocksize); \
                    klc.smem = 0; \
                    KernelProperties kernelProperties; \
                    hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                                cuda_filter_alignments_by_mismatchratio_kernel<(blocksize)>, \
                                klc.threads_per_block, klc.smem); CUERR; \
                    mymap[klc] = kernelProperties; \
            }

            getProp(32);
            getProp(64);
            getProp(96);
            getProp(128);
            getProp(160);
            getProp(192);
            getProp(224);
            getProp(256);
            
            KernelLaunchConfig kernelLaunchConfig;
            kernelLaunchConfig.threads_per_block = requestedBlocksize;
            kernelLaunchConfig.smem = smem;    
            
            const auto& kernelProperties = mymap[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

            handle.kernelPropertiesMap[KernelId::FilterAlignmentsByMismatchRatio] = std::move(mymap);

            #undef getProp
        }else{
            KernelLaunchConfig kernelLaunchConfig;
            kernelLaunchConfig.threads_per_block = requestedBlocksize;
            kernelLaunchConfig.smem = smem;   

            std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
            const KernelProperties& kernelProperties = map[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
        }

        dim3 block(requestedBlocksize, 1, 1);
        //dim3 grid(std::min(max_blocks_per_device, maxNumAnchors));
        dim3 grid(max_blocks_per_device, 1, 1);

        #define mycall(blocksize) cuda_filter_alignments_by_mismatchratio_kernel<(blocksize)> \
                <<<grid, block, smem, stream>>>( \
            d_bestAlignmentFlags, \
            d_nOps, \
            d_overlaps, \
            d_candidates_per_subject_prefixsum, \
            d_numAnchors, \
            d_numCandidates, \
            mismatchratioBaseFactor, \
            goodAlignmentsCountThreshold); CUERR;

        switch(requestedBlocksize) {
        case 32: mycall(32); break;
        case 64: mycall(64); break;
        case 96: mycall(96); break;
        case 128: mycall(128); break;
        case 160: mycall(160); break;
        case 192: mycall(192); break;
        case 224: mycall(224); break;
        case 256: mycall(256); break;
        default: mycall(256); break;
        }

            #undef mycall
    }


    void callSelectIndicesOfGoodCandidatesKernelAsync(
            int* d_indicesOfGoodCandidates,
            int* d_numIndicesPerAnchor,
            int* d_totalNumIndices,
            const BestAlignment_t* d_alignmentFlags,
            const int* d_candidates_per_subject,
            const int* d_candidates_per_subject_prefixsum,
            const int* d_anchorIndicesOfCandidates,
            const int* d_numAnchors,
            const int* d_numCandidates,
            int maxNumAnchors,
            int maxNumCandidates,
            hipStream_t stream,
            KernelLaunchHandle& handle){

        constexpr int blocksize = 128;
        constexpr int tilesize = 32;

        const std::size_t smem = 0;

        int max_blocks_per_device = 1;

        KernelLaunchConfig kernelLaunchConfig;
        kernelLaunchConfig.threads_per_block = blocksize;
        kernelLaunchConfig.smem = smem;

        auto iter = handle.kernelPropertiesMap.find(KernelId::SelectIndicesOfGoodCandidates);
        if(iter == handle.kernelPropertiesMap.end()){

            std::map<KernelLaunchConfig, KernelProperties> mymap;

            #define getProp(blocksize) { \
                KernelLaunchConfig kernelLaunchConfig; \
                kernelLaunchConfig.threads_per_block = (blocksize); \
                kernelLaunchConfig.smem = 0; \
                KernelProperties kernelProperties; \
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&kernelProperties.max_blocks_per_SM, \
                    selectIndicesOfGoodCandidatesKernel<(blocksize), tilesize>, \
                                                                kernelLaunchConfig.threads_per_block, kernelLaunchConfig.smem); CUERR; \
                mymap[kernelLaunchConfig] = kernelProperties; \
            }

            getProp(32);
            getProp(64);
            getProp(96);
            getProp(128);
            getProp(160);
            getProp(192);
            getProp(224);
            getProp(256);

            const auto& kernelProperties = mymap[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;

            handle.kernelPropertiesMap[KernelId::SelectIndicesOfGoodCandidates] = std::move(mymap);

            #undef getProp
        }else{
            std::map<KernelLaunchConfig, KernelProperties>& map = iter->second;
            const KernelProperties& kernelProperties = map[kernelLaunchConfig];
            max_blocks_per_device = handle.deviceProperties.multiProcessorCount * kernelProperties.max_blocks_per_SM;
        }

        // hipMemsetAsync(d_numIndicesPerAnchor, 0, maxNumAnchors * sizeof(int), stream); CUERR;
        // hipMemsetAsync(d_totalNumIndices, 0, sizeof(int), stream); CUERR;
        helpers::lambda_kernel<<<4, 256, 0, stream>>>([=] __device__(){
            const int tid = threadIdx.x + blockIdx.x * blockDim.x;
            const int stride = blockDim.x * gridDim.x;

            for(int i = tid; i < maxNumAnchors; i += stride){
                d_numIndicesPerAnchor[i] = 0;
            }

            if(tid == 0){
                *d_totalNumIndices = 0;
            }
        }); CUERR;

        dim3 block(blocksize, 1, 1);
        //dim3 grid(std::min(SDIV(maxNumCandidates, blocksize), max_blocks_per_device));
        dim3 grid(max_blocks_per_device, 1, 1);

        selectIndicesOfGoodCandidatesKernel<blocksize, tilesize><<<grid, block, 0, stream>>>(
            d_indicesOfGoodCandidates,
            d_numIndicesPerAnchor,
            d_totalNumIndices,
            d_alignmentFlags,
            d_candidates_per_subject,
            d_candidates_per_subject_prefixsum,
            d_anchorIndicesOfCandidates,
            d_numAnchors,
            d_numCandidates
        ); CUERR;
    }


}
}
