#include "hip/hip_runtime.h"
#if 1


#include <gpu/distributedreadstorage.hpp>
#include <gpu/distributedarray.hpp>

#include <config.hpp>
#include <sequence.hpp>
#include <sequencefileio.hpp>

#include <fstream>
#include <omp.h>
#include <algorithm>
#include <iterator>

#ifdef __NVCC__

namespace care{
namespace gpu{

DistributedReadStorage::DistributedReadStorage(const std::vector<int>& deviceIds_, read_number nReads, bool b, int maximum_sequence_length){

    init(deviceIds_, nReads, b, maximum_sequence_length);
}

void DistributedReadStorage::init(const std::vector<int>& deviceIds_, read_number nReads, bool b, int maximum_sequence_length){
    deviceIds = deviceIds_;
    numberOfReads = nReads;
    sequenceLengthLimit = maximum_sequence_length;
    useQualityScores = b;

    int numGpus = deviceIds.size();

    if(numberOfReads > 0 && sequenceLengthLimit > 0){
        std::vector<size_t> freeMemPerGpu(numGpus, 0);
        std::vector<size_t> totalMemPerGpu(numGpus, 0);
        std::vector<size_t> maximumUsableBytesPerGpu(numGpus, 0);

        auto getGpuMemoryInfo = [&](){
            for(int gpu = 0; gpu < numGpus; gpu++){
                hipSetDevice(deviceIds[gpu]); CUERR;

                hipMemGetInfo(&freeMemPerGpu[gpu], &totalMemPerGpu[gpu]); CUERR;
            }
        };

        auto updateMemoryLimits = [&](){
            getGpuMemoryInfo();

            for(int gpu = 0; gpu < numGpus; gpu++){
                const size_t usableMem = freeMemPerGpu[gpu] > gpuReadStorageHeadroomPerGPU ? freeMemPerGpu[gpu] - gpuReadStorageHeadroomPerGPU : 0;
                maximumUsableBytesPerGpu[gpu] = usableMem;
            }
            std::cerr << "Usable memory per gpu : ";
            std::copy(maximumUsableBytesPerGpu.begin(), maximumUsableBytesPerGpu.end(), std::ostream_iterator<size_t>(std::cerr, " "));
            std::cerr << "\n";
        };

        int oldId; hipGetDevice(&oldId); CUERR;


        const int intsPerSequence = getEncodedNumInts2BitHiLo(sequenceLengthLimit);

        updateMemoryLimits();
        distributedSequenceData2 = std::move(DistributedArray<unsigned int, read_number>(deviceIds, maximumUsableBytesPerGpu, numberOfReads, intsPerSequence));

        updateMemoryLimits();
        distributedSequenceLengths2 = std::move(DistributedArray<Length_t, read_number>(deviceIds, maximumUsableBytesPerGpu, numberOfReads, 1));

        if(useQualityScores){
            updateMemoryLimits();
            distributedQualities2 = std::move(DistributedArray<char, read_number>(deviceIds, maximumUsableBytesPerGpu, numberOfReads, sequenceLengthLimit));
        }

        getGpuMemoryInfo();

        std::cerr << "Free memory per gpu after construction of distributed readstorage: ";
        std::copy(freeMemPerGpu.begin(), freeMemPerGpu.end(), std::ostream_iterator<size_t>(std::cerr, " "));
        std::cerr << "\n";

        hipSetDevice(oldId); CUERR;
    }
}

DistributedReadStorage::DistributedReadStorage(DistributedReadStorage&& other){
    *this = std::move(other);
}

DistributedReadStorage& DistributedReadStorage::operator=(DistributedReadStorage&& rhs){
    deviceIds = std::move(rhs.deviceIds);
    numberOfReads = std::move(rhs.numberOfReads);
    sequenceLengthLimit = std::move(rhs.sequenceLengthLimit);
    useQualityScores = std::move(rhs.useQualityScores);
    readIdsOfReadsWithUndeterminedBase = std::move(rhs.readIdsOfReadsWithUndeterminedBase);
    distributedSequenceData2 = std::move(rhs.distributedSequenceData2);
    distributedSequenceLengths2 = std::move(rhs.distributedSequenceLengths2);
    distributedQualities2 = std::move(rhs.distributedQualities2);
    statistics = std::move(rhs.statistics);
    hasMoved = rhs.hasMoved;
    rhs.hasMoved = true;

    return *this;
}

DistributedReadStorage::MemoryInfo DistributedReadStorage::getMemoryInfo() const{
    MemoryInfo info;
    info.deviceSizeInBytes.resize(deviceIds.size(),0);
    info.deviceIds = deviceIds;

    auto handlearray = [&](const auto& array){
        const auto partitions = array.getPartitions();

        for(int location = 0; location < distributedSequenceData2.numLocations; location++){
            size_t bytes = partitions[location] * array.sizeOfElement;
            std::cerr << "location " << location << " " << bytes << "\n";
            if(location == array.hostLocation){
                info.hostSizeInBytes += bytes;
            }else{
                info.deviceSizeInBytes[location] += bytes;
            }
        }
    };

    handlearray(distributedSequenceData2);
    handlearray(distributedSequenceLengths2);
    handlearray(distributedQualities2);

    return info;
}


DistributedReadStorage::Statistics DistributedReadStorage::getStatistics() const{
    return statistics;
}

void DistributedReadStorage::destroy(){
    numberOfReads = 0;
    sequenceLengthLimit = 0;
    std::vector<size_t> fractions(deviceIds.size(), 0);
    distributedSequenceData2 = std::move(DistributedArray<unsigned int, read_number>(deviceIds, fractions, 0, 0));
    distributedSequenceLengths2 = std::move(DistributedArray<Length_t, read_number>(deviceIds, fractions, 0, 0));
    distributedQualities2 = std::move(DistributedArray<char, read_number>(deviceIds, fractions, 0, 0));
    statistics = Statistics{};
}

read_number DistributedReadStorage::getNumberOfReads() const{
    return numberOfReads;
}

bool DistributedReadStorage::canUseQualityScores() const{
    return useQualityScores;
}

int DistributedReadStorage::getSequenceLengthLimit() const{
    return sequenceLengthLimit;
}

std::vector<int> DistributedReadStorage::getDeviceIds() const{
    return deviceIds;
}


void DistributedReadStorage::setReads(read_number firstIndex, read_number lastIndex_excl, const std::vector<Read>& reads, int numThreads){
    std::vector<read_number> indices(lastIndex_excl-firstIndex);
    std::iota(indices.begin(), indices.end(), firstIndex);

    setReads(indices, reads, numThreads);
}

void DistributedReadStorage::setReads(const std::vector<read_number>& indices, const std::vector<Read>& reads, int numThreads){

    assert(indices.size() > 0);
    assert(reads.size() == indices.size());
    assert(std::all_of(indices.begin(), indices.end(), [&](auto i){ return i < getNumberOfReads();}));
    assert(std::all_of(reads.begin(), reads.end(), [&](const auto& r){ return Length_t(r.sequence.length()) <= getSequenceLengthLimit();}));
    assert(std::all_of(reads.begin(), reads.end(), [&](const auto& r){ return r.sequence.length() == r.quality.length();}));

    auto minmax = std::minmax_element(reads.begin(), reads.end(), [](const auto& r1, const auto& r2){
        return r1.sequence.length() < r2.sequence.length();
    });

    statistics.minimumSequenceLength = std::min(statistics.minimumSequenceLength, int(minmax.first->sequence.length()));
    statistics.maximumSequenceLength = std::max(statistics.maximumSequenceLength, int(minmax.second->sequence.length()));

    std::vector<char> sequenceData;
    std::vector<Length_t> sequenceLengths;
    std::vector<char> qualityData;

    const size_t numReads = indices.size();

    const size_t encodedSequencePitch = getEncodedNumInts2BitHiLo(getSequenceLengthLimit()) * sizeof(int);
    const size_t qualityPitch = getSequenceLengthLimit();

    sequenceData.resize(encodedSequencePitch * numReads, 0);
    sequenceLengths.resize(numReads, 0);
    if(canUseQualityScores()){
        qualityData.resize(getSequenceLengthLimit() * numReads, 0);
    }

    int oldNumOMPThreads = 1;
    #pragma omp parallel
    {
        #pragma omp single
        oldNumOMPThreads = omp_get_num_threads();
    }

    omp_set_num_threads(numThreads);

    #pragma omp parallel for
    for(size_t i = 0; i < numReads; i++){
        const Read& r = reads[i];

        unsigned int* dest = (unsigned int*)&sequenceData[std::size_t(i) * encodedSequencePitch];
        encodeSequence2BitHiLo(dest,
                                r.sequence.c_str(),
                                r.sequence.length());
        sequenceLengths[i] = Length_t(r.sequence.length());
        if(canUseQualityScores()){
            std::copy(r.quality.begin(), r.quality.end(), qualityData.begin() + i * qualityPitch);
        }
    }

    omp_set_num_threads(oldNumOMPThreads);

    setSequences(indices, sequenceData.data());
    setSequenceLengths(indices, sequenceLengths.data());
    if(canUseQualityScores()){
        setQualities(indices, qualityData.data());
    }
}

void DistributedReadStorage::setReadContainsN(read_number readId, bool contains){

    auto pos = std::lower_bound(readIdsOfReadsWithUndeterminedBase.begin(),
                                        readIdsOfReadsWithUndeterminedBase.end(),
                                        readId);


    if(contains){
        if(pos != readIdsOfReadsWithUndeterminedBase.end()){
            ; //already marked
        }else{
            readIdsOfReadsWithUndeterminedBase.insert(pos, readId);
        }
    }else{
        if(pos != readIdsOfReadsWithUndeterminedBase.end()){
            //remove mark
            readIdsOfReadsWithUndeterminedBase.erase(pos);
        }else{
            ; //already unmarked
        }
    }
}

bool DistributedReadStorage::readContainsN(read_number readId) const{

    auto pos = std::lower_bound(readIdsOfReadsWithUndeterminedBase.begin(),
                                        readIdsOfReadsWithUndeterminedBase.end(),
                                        readId);
    bool b2 = readIdsOfReadsWithUndeterminedBase.end() != pos && *pos == readId;

    return b2;
}

void DistributedReadStorage::setSequences(read_number firstIndex, read_number lastIndex_excl, const char* data){
    distributedSequenceData2.setSafe(firstIndex, lastIndex_excl, reinterpret_cast<const unsigned int*>(data));
}

void DistributedReadStorage::setSequences(const std::vector<read_number>& indices, const char* data){
    distributedSequenceData2.setSafe(indices, reinterpret_cast<const unsigned int*>(data));
}

void DistributedReadStorage::setSequenceLengths(read_number firstIndex, read_number lastIndex_excl, const Length_t* data){
    distributedSequenceLengths2.setSafe(firstIndex, lastIndex_excl, data);
}

void DistributedReadStorage::setSequenceLengths(const std::vector<read_number>& indices, const Length_t* data){
    distributedSequenceLengths2.setSafe(indices, data);
}

void DistributedReadStorage::setQualities(read_number firstIndex, read_number lastIndex_excl, const char* data){
    distributedQualities2.setSafe(firstIndex, lastIndex_excl, data);
}

void DistributedReadStorage::setQualities(const std::vector<read_number>& indices, const char* data){
    distributedQualities2.setSafe(indices, data);
}

DistributedReadStorage::GatherHandleSequences DistributedReadStorage::makeGatherHandleSequences() const{
    return distributedSequenceData2.makeGatherHandle();
}

DistributedReadStorage::GatherHandleLengths DistributedReadStorage::makeGatherHandleLengths() const{
    return distributedSequenceLengths2.makeGatherHandle();
}

DistributedReadStorage::GatherHandleQualities DistributedReadStorage::makeGatherHandleQualities() const{
    return distributedQualities2.makeGatherHandle();
}

void DistributedReadStorage::gatherSequenceDataToGpuBufferAsync(
                            const DistributedReadStorage::GatherHandleSequences& handle,
                            char* d_sequence_data,
                            size_t out_sequence_pitch,
                            const read_number* h_readIds,
                            const read_number* d_readIds,
                            int nReadIds,
                            int deviceId,
                            hipStream_t stream,
                            int numCpuThreads) const{

    distributedSequenceData2.gatherElementsInGpuMemAsync(handle,
                                                        h_readIds,
                                                        d_readIds,
                                                        nReadIds,
                                                        deviceId,
                                                        (unsigned int*)d_sequence_data,
                                                        out_sequence_pitch,
                                                        stream,
                                                        numCpuThreads);

}




void DistributedReadStorage::gatherSequenceLengthsToGpuBufferAsync(
                            const GatherHandleLengths& handle,
                            int* d_lengths,
                            const read_number* h_readIds,
                            const read_number* d_readIds,
                            int nReadIds,
                            int deviceId,
                            hipStream_t stream,
                            int numCpuThreads) const{

    distributedSequenceLengths2.gatherElementsInGpuMemAsync(handle,
                                                        h_readIds,
                                                        d_readIds,
                                                        nReadIds,
                                                        deviceId,
                                                        d_lengths,
                                                        sizeof(int),
                                                        stream,
                                                        numCpuThreads);

}

void DistributedReadStorage::gatherQualitiesToGpuBufferAsync(
                            const GatherHandleQualities& handle,
                            char* d_quality_data,
                            size_t out_quality_pitch,
                            const read_number* h_readIds,
                            const read_number* d_readIds,
                            int nReadIds,
                            int deviceId,
                            hipStream_t stream,
                            int numCpuThreads) const{

    distributedQualities2.gatherElementsInGpuMemAsync(handle,
                                                        h_readIds,
                                                        d_readIds,
                                                        nReadIds,
                                                        deviceId,
                                                        d_quality_data,
                                                        out_quality_pitch,
                                                        stream,
                                                        numCpuThreads);

}


std::future<void> DistributedReadStorage::gatherSequenceDataToHostBufferAsync(
                            const GatherHandleSequences& handle,
                            char* h_sequence_data,
                            size_t out_sequence_pitch,
                            const read_number* h_readIds,
                            int nReadIds,
                            int numCpuThreads) const{

    return distributedSequenceData2.gatherElementsInHostMemAsync(handle,
                                                        h_readIds,
                                                        nReadIds,
                                                        (unsigned int*)h_sequence_data,
                                                        out_sequence_pitch);
}

std::future<void> DistributedReadStorage::gatherSequenceLengthsToHostBufferAsync(
                            const GatherHandleLengths& handle,
                            int* h_lengths,
                            const read_number* h_readIds,
                            int nReadIds,
                            int numCpuThreads) const{

    return distributedSequenceLengths2.gatherElementsInHostMemAsync(handle,
                                                        h_readIds,
                                                        nReadIds,
                                                        h_lengths,
                                                        sizeof(int));

}

std::future<void> DistributedReadStorage::gatherQualitiesToHostBufferAsync(
                            const GatherHandleQualities& handle,
                            char* h_quality_data,
                            size_t out_quality_pitch,
                            const read_number* h_readIds,
                            int nReadIds,
                            int numCpuThreads) const{

    return distributedQualities2.gatherElementsInHostMemAsync(handle,
                                                        h_readIds,
                                                        nReadIds,
                                                        h_quality_data,
                                                        out_quality_pitch);
}


void DistributedReadStorage::gatherSequenceDataToHostBuffer(
                            const GatherHandleSequences& handle,
                            char* h_sequence_data,
                            size_t out_sequence_pitch,
                            const read_number* h_readIds,
                            int nReadIds,
                            int numCpuThreads) const{

    return distributedSequenceData2.gatherElementsInHostMem(handle,
                                                        h_readIds,
                                                        nReadIds,
                                                        (unsigned int*)h_sequence_data,
                                                        out_sequence_pitch);
}

void DistributedReadStorage::gatherSequenceLengthsToHostBuffer(
                            const GatherHandleLengths& handle,
                            int* h_lengths,
                            const read_number* h_readIds,
                            int nReadIds,
                            int numCpuThreads) const{

    return distributedSequenceLengths2.gatherElementsInHostMem(handle,
                                                        h_readIds,
                                                        nReadIds,
                                                        h_lengths,
                                                        sizeof(int));

}

void DistributedReadStorage::gatherQualitiesToHostBuffer(
                            const GatherHandleQualities& handle,
                            char* h_quality_data,
                            size_t out_quality_pitch,
                            const read_number* h_readIds,
                            int nReadIds,
                            int numCpuThreads) const{

    return distributedQualities2.gatherElementsInHostMem(handle,
                                                        h_readIds,
                                                        nReadIds,
                                                        h_quality_data,
                                                        out_quality_pitch);
}


void DistributedReadStorage::saveToFile(const std::string& filename) const{
    std::ofstream stream(filename, std::ios::binary);

    //int ser_id = serialization_id;
    std::size_t lengthsize = sizeof(Length_t);
    stream.write(reinterpret_cast<const char*>(&lengthsize), sizeof(std::size_t));

    stream.write(reinterpret_cast<const char*>(&numberOfReads), sizeof(read_number));
    stream.write(reinterpret_cast<const char*>(&sequenceLengthLimit), sizeof(int));
    stream.write(reinterpret_cast<const char*>(&useQualityScores), sizeof(bool));
    stream.write(reinterpret_cast<const char*>(&statistics), sizeof(Statistics));

    constexpr read_number batchsize = 10000000;
    int numBatches = SDIV(numberOfReads, batchsize);

    {
        auto sequencehandle = makeGatherHandleSequences();
        size_t outputpitch = getEncodedNumInts2BitHiLo(sequenceLengthLimit) * sizeof(int);

        size_t totalSequenceMemory = outputpitch * numberOfReads;
        stream.write(reinterpret_cast<const char*>(&totalSequenceMemory), sizeof(size_t));

        for(int batch = 0; batch < numBatches; batch++){
            read_number begin = batch * batchsize;
            read_number end = std::min((batch+1) * batchsize, getNumberOfReads());

            std::vector<read_number> indices(end-begin);
            std::iota(indices.begin(), indices.end(), begin);

            size_t databytes = outputpitch * indices.size();
            std::vector<char> data(databytes, 0);

            auto future = gatherSequenceDataToHostBufferAsync(
                                        sequencehandle,
                                        data.data(),
                                        outputpitch,
                                        indices.data(),
                                        indices.size(),
                                        1);

            future.wait();

            stream.write(reinterpret_cast<const char*>(&data[0]), databytes);
        }
    }

    {
        auto lengthhandle = makeGatherHandleLengths();
        size_t outputpitch = sizeof(Length_t);

        size_t totalLengthMemory = outputpitch * numberOfReads;
        stream.write(reinterpret_cast<const char*>(&totalLengthMemory), sizeof(size_t));

        for(int batch = 0; batch < numBatches; batch++){
            read_number begin = batch * batchsize;
            read_number end = std::min((batch+1) * batchsize, getNumberOfReads());

            std::vector<read_number> indices(end-begin);
            std::iota(indices.begin(), indices.end(), begin);

            size_t databytes = outputpitch * indices.size();
            std::vector<Length_t> data(indices.size(), 0);

            auto future = gatherSequenceLengthsToHostBufferAsync(
                                        lengthhandle,
                                        data.data(),
                                        indices.data(),
                                        indices.size(),
                                        1);

            future.wait();

            stream.write(reinterpret_cast<const char*>(&data[0]), databytes);
        }
    }

    if(useQualityScores){
        auto qualityhandle = makeGatherHandleQualities();
        size_t outputpitch = sequenceLengthLimit;

        size_t totalqualityMemory = outputpitch * numberOfReads;
        stream.write(reinterpret_cast<const char*>(&totalqualityMemory), sizeof(size_t));

        for(int batch = 0; batch < numBatches; batch++){
            read_number begin = batch * batchsize;
            read_number end = std::min((batch+1) * batchsize, getNumberOfReads());

            std::vector<read_number> indices(end-begin);
            std::iota(indices.begin(), indices.end(), begin);

            size_t databytes = outputpitch * indices.size();
            std::vector<char> data(databytes, 0);

            auto future = gatherQualitiesToHostBufferAsync(
                                        qualityhandle,
                                        data.data(),
                                        outputpitch,
                                        indices.data(),
                                        indices.size(),
                                        1);

            future.wait();

            stream.write(reinterpret_cast<const char*>(&data[0]), databytes);
        }
    }

    //read ids with N
    std::size_t numUndeterminedReads = readIdsOfReadsWithUndeterminedBase.size();
    stream.write(reinterpret_cast<const char*>(&numUndeterminedReads), sizeof(size_t));
    stream.write(reinterpret_cast<const char*>(readIdsOfReadsWithUndeterminedBase.data()), numUndeterminedReads * sizeof(read_number));
}

void DistributedReadStorage::loadFromFile(const std::string& filename){
    loadFromFile(filename, deviceIds);
}

void DistributedReadStorage::loadFromFile(const std::string& filename, const std::vector<int>& deviceIds_){
    std::ifstream stream(filename, std::ios::binary);
    if(!stream)
        throw std::runtime_error("Cannot open file " + filename);

    destroy();

    std::size_t lengthsize = sizeof(Length_t);
    std::size_t loaded_lengthsize;
    stream.read(reinterpret_cast<char*>(&loaded_lengthsize), sizeof(std::size_t));

    if(loaded_lengthsize != lengthsize)
        throw std::runtime_error("Wrong size of length type!");


    read_number loaded_numberOfReads;
    int loaded_sequenceLengthLimit;
    bool loaded_useQualityScores;

    stream.read(reinterpret_cast<char*>(&loaded_numberOfReads), sizeof(read_number));
    stream.read(reinterpret_cast<char*>(&loaded_sequenceLengthLimit), sizeof(int));
    stream.read(reinterpret_cast<char*>(&loaded_useQualityScores), sizeof(bool));

    init(deviceIds_, loaded_numberOfReads, loaded_useQualityScores, loaded_sequenceLengthLimit);

    stream.read(reinterpret_cast<char*>(&statistics), sizeof(Statistics));

    constexpr read_number batchsize = 10000000;
    int numBatches = SDIV(numberOfReads, batchsize);

    {
        size_t seqpitch = getEncodedNumInts2BitHiLo(sequenceLengthLimit) * sizeof(int);

        size_t totalSequenceMemory = 1;
        stream.read(reinterpret_cast<char*>(&totalSequenceMemory), sizeof(size_t));

        size_t totalMemoryRead = 0;

        for(int batch = 0; batch < numBatches; batch++){
            read_number begin = batch * batchsize;
            read_number end = std::min((batch+1) * batchsize, getNumberOfReads());

            size_t databytes = seqpitch * (end-begin);
            std::vector<char> data(databytes, 0);

            stream.read(reinterpret_cast<char*>(&data[0]), databytes);
            totalMemoryRead += stream.gcount();

            assert(totalMemoryRead <= totalSequenceMemory);

            setSequences(begin, end, data.data());
        }

        assert(totalMemoryRead == totalSequenceMemory);
    }

    {
        size_t lengthpitch = sizeof(Length_t);

        size_t totalLengthMemory = 1;
        stream.read(reinterpret_cast<char*>(&totalLengthMemory), sizeof(size_t));

        size_t totalMemoryRead = 0;

        for(int batch = 0; batch < numBatches; batch++){
            read_number begin = batch * batchsize;
            read_number end = std::min((batch+1) * batchsize, getNumberOfReads());

            std::vector<Length_t> data((end-begin), 0);

            size_t databytes = lengthpitch * (end-begin);
            stream.read(reinterpret_cast<char*>(&data[0]), databytes);
            totalMemoryRead += stream.gcount();

            assert(totalMemoryRead <= totalLengthMemory);

            setSequenceLengths(begin, end, data.data());

            auto minmax = std::minmax_element(data.begin(), data.end(), [](const auto& l1, const auto& l2){
                return l1 < l2;
            });

            statistics.minimumSequenceLength = std::min(statistics.minimumSequenceLength, int(*minmax.first));
            statistics.maximumSequenceLength = std::max(statistics.maximumSequenceLength, int(*minmax.second));
        }

        assert(totalMemoryRead == totalLengthMemory);
    }

    if(useQualityScores){
        size_t qualitypitch = sequenceLengthLimit;

        size_t totalqualityMemory = 1;
        stream.read(reinterpret_cast<char*>(&totalqualityMemory), sizeof(size_t));

        size_t totalMemoryRead = 0;

        for(int batch = 0; batch < numBatches; batch++){
            read_number begin = batch * batchsize;
            read_number end = std::min((batch+1) * batchsize, getNumberOfReads());

            size_t databytes = qualitypitch * (end-begin);
            std::vector<char> data(databytes, 0);

            stream.read(reinterpret_cast<char*>(&data[0]), databytes);
            totalMemoryRead += stream.gcount();

            assert(totalMemoryRead <= totalqualityMemory);

            setQualities(begin, end, data.data());
        }

        assert(totalMemoryRead == totalqualityMemory);
    }

    //read ids with N
    std::size_t numUndeterminedReads = 0;
    stream.read(reinterpret_cast<char*>(&numUndeterminedReads), sizeof(std::size_t));
    readIdsOfReadsWithUndeterminedBase.resize(numUndeterminedReads);
    stream.read(reinterpret_cast<char*>(readIdsOfReadsWithUndeterminedBase.data()), numUndeterminedReads * sizeof(read_number));

}






}
}

#endif

#endif
