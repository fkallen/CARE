#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of BlockAdjacentDifference utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <cub/block/block_adjacent_difference.cuh>
#include <cub/util_allocator.cuh>

#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/mismatch.h>
#include <thrust/random.h>
#include <thrust/sequence.h>
#include <thrust/shuffle.h>
#include <thrust/sort.h>
#include <thrust/tabulate.h>

#include <limits>
#include <memory>
#include <typeinfo>

#include "test_util.h"


using namespace hipcub;


/**
 * \brief Generates integer sequence \f$S_n=i(i-1)/2\f$.
 *
 * The adjacent difference of this sequence produce consecutive numbers:
 * \f[
 *   p = \frac{i(i - 1)}{2} \\
 *   n = \frac{(i + 1) i}{2} \\
 *   n - p = i \\
 *   \frac{(i + 1) i}{2} - \frac{i (i - 1)}{2} = i \\
 *   (i + 1) i - i (i - 1) = 2 i \\
 *   (i + 1) - (i - 1) = 2 \\
 *   2 = 2
 * \f]
 */
template <typename DestT>
struct TestSequenceGenerator
{
  std::size_t offset;

  TestSequenceGenerator(std::size_t offset = 0)
      : offset(offset)
  {}

  template <typename SourceT>
  __device__ __host__ DestT operator()(SourceT index) const
  {
    index += static_cast<SourceT>(offset);
    return static_cast<DestT>(index * (index - 1) / SourceT(2));
  }
};



struct CustomType
{
  unsigned int key;
  unsigned int value;

  __device__ __host__ CustomType()
    : key(0)
    , value(0)
  {}

  __device__ __host__ CustomType(unsigned int key, unsigned int value)
    : key(key)
    , value(value)
  {}
};


__device__ __host__ bool operator==(const CustomType& lhs,
                                    const CustomType& rhs)
{
  return lhs.key == rhs.key && lhs.value == rhs.value;
}

__device__ __host__ bool operator!=(const CustomType& lhs,
                                    const CustomType& rhs)
{
  return !(lhs == rhs);
}

__device__ __host__ CustomType operator-(const CustomType& lhs,
                                         const CustomType& rhs)
{
  return CustomType{lhs.key - rhs.key, lhs.value - rhs.value};
}

struct CustomDifference
{
  template <typename DataType>
  __device__ DataType operator()(DataType &lhs, DataType &rhs)
  {
    return lhs - rhs;
  }
};


template <typename DataType,
          unsigned int ThreadsInBlock,
          unsigned int ItemsPerThread,
          bool ReadLeft = false>
__global__ void LastTileTestKernel(const DataType *input,
                                   DataType *output,
                                   unsigned int valid_items)
{
  using BlockAdjacentDifferenceT =
    hipcub::BlockAdjacentDifference<DataType, ThreadsInBlock>;

  __shared__ typename BlockAdjacentDifferenceT::TempStorage temp_storage;

  DataType thread_data[ItemsPerThread];
  DataType thread_result[ItemsPerThread];

  const unsigned int thread_offset = threadIdx.x * ItemsPerThread;

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    thread_data[item] = input[thread_offset + item];
  }
  __syncthreads();

  if (ReadLeft)
  {
    BlockAdjacentDifferenceT(temp_storage).SubtractLeftPartialTile(
      thread_data,
      thread_result,
      CustomDifference(),
      valid_items);
  }
  else
  {
    BlockAdjacentDifferenceT(temp_storage).SubtractRightPartialTile(
      thread_data,
      thread_result,
      CustomDifference(),
      valid_items);
  }

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    output[thread_offset + item] = thread_result[item];
  }
}


template <typename DataType,
          unsigned int ThreadsInBlock,
          unsigned int ItemsPerThread,
          bool ReadLeft = false>
__global__ void MiddleTileTestKernel(const DataType *input,
                                     DataType *output,
                                     DataType neighbour_tile_value)
{
  using BlockAdjacentDifferenceT =
    hipcub::BlockAdjacentDifference<DataType, ThreadsInBlock>;

  __shared__ typename BlockAdjacentDifferenceT::TempStorage temp_storage;

  DataType thread_data[ItemsPerThread];
  DataType thread_result[ItemsPerThread];

  const unsigned int thread_offset = threadIdx.x * ItemsPerThread;

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    thread_data[item] = input[thread_offset + item];
  }
  __syncthreads();

  if (ReadLeft)
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractLeft(thread_data,
                    thread_result,
                    CustomDifference(),
                    neighbour_tile_value);
  }
  else
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractRight(thread_data,
                     thread_result,
                     CustomDifference(),
                     neighbour_tile_value);
  }

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    output[thread_offset + item] = thread_result[item];
  }
}


template <typename DataType,
          unsigned int ThreadsInBlock,
          unsigned int ItemsPerThread,
          bool ReadLeft = false>
__global__ void MiddleTileInplaceTestKernel(const DataType *input,
                                            DataType *output,
                                            DataType neighbour_tile_value)
{
  using BlockAdjacentDifferenceT =
    hipcub::BlockAdjacentDifference<DataType, ThreadsInBlock>;

  __shared__ typename BlockAdjacentDifferenceT::TempStorage temp_storage;

  DataType thread_data[ItemsPerThread];

  const unsigned int thread_offset = threadIdx.x * ItemsPerThread;

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    thread_data[item] = input[thread_offset + item];
  }
  __syncthreads();

  if (ReadLeft)
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractLeft(thread_data,
                    thread_data,
                    CustomDifference(),
                    neighbour_tile_value);
  }
  else
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractRight(thread_data,
                     thread_data,
                     CustomDifference(),
                     neighbour_tile_value);
  }

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    output[thread_offset + item] = thread_data[item];
  }
}


template <typename DataType,
          unsigned int ThreadsInBlock,
          unsigned int ItemsPerThread,
          bool ReadLeft = false>
__global__ void TestKernel(DataType *data)
{
  using BlockAdjacentDifferenceT =
    hipcub::BlockAdjacentDifference<DataType, ThreadsInBlock>;

  __shared__ typename BlockAdjacentDifferenceT::TempStorage temp_storage;

  DataType thread_data[ItemsPerThread];
  DataType thread_result[ItemsPerThread];

  const unsigned int thread_offset = threadIdx.x * ItemsPerThread;

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    thread_data[item] = data[thread_offset + item];
  }
  __syncthreads();

  if (ReadLeft)
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractLeft(thread_data, thread_result, CustomDifference());
  }
  else
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractRight(thread_data, thread_result, CustomDifference());
  }

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    data[thread_offset + item] = thread_result[item];
  }
}


template <typename DataType,
          unsigned int ThreadsInBlock,
          unsigned int ItemsPerThread,
          bool ReadLeft = false>
__global__ void LastTileTestInplaceKernel(const DataType *input,
                                          DataType *output,
                                          unsigned int valid_items)
{
  using BlockAdjacentDifferenceT =
    hipcub::BlockAdjacentDifference<DataType, ThreadsInBlock>;

  __shared__ typename BlockAdjacentDifferenceT::TempStorage temp_storage;

  DataType thread_data[ItemsPerThread];

  const unsigned int thread_offset = threadIdx.x * ItemsPerThread;

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    thread_data[item] = input[thread_offset + item];
  }
  __syncthreads();

  if (ReadLeft)
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractLeftPartialTile(thread_data,
                               thread_data,
                               CustomDifference(),
                               valid_items);
  }
  else
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractRightPartialTile(thread_data,
                                thread_data,
                                CustomDifference(),
                                valid_items);
  }

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    output[thread_offset + item] = thread_data[item];
  }
}

template <typename DataType,
          unsigned int ThreadsInBlock,
          unsigned int ItemsPerThread,
          bool ReadLeft = false>
__global__ void TestInplaceKernel(DataType *data)
{
  using BlockAdjacentDifferenceT =
    hipcub::BlockAdjacentDifference<DataType, ThreadsInBlock>;

  __shared__ typename BlockAdjacentDifferenceT::TempStorage temp_storage;

  DataType thread_data[ItemsPerThread];

  const unsigned int thread_offset = threadIdx.x * ItemsPerThread;

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    thread_data[item] = data[thread_offset + item];
  }
  __syncthreads();

  if (ReadLeft)
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractLeft(thread_data,
                    thread_data,
                    CustomDifference());
  }
  else
  {
    BlockAdjacentDifferenceT(temp_storage)
      .SubtractRight(thread_data,
                     thread_data,
                     CustomDifference());
  }

  for (unsigned int item = 0; item < ItemsPerThread; item++)
  {
    data[thread_offset + item] = thread_data[item];
  }
}

template <typename DataType,
          unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock,
          bool ReadLeft = false>
void LastTileTest(const DataType *input,
                  DataType *output,
                  unsigned int valid_items)
{
  LastTileTestKernel<DataType, ThreadsInBlock, ItemsPerThread, ReadLeft>
    <<<1, ThreadsInBlock>>>(input, output, valid_items);

  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());
}


template <typename DataType,
          unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock,
          bool ReadLeft = false>
void Test(DataType *data)
{
  TestKernel<DataType, ThreadsInBlock, ItemsPerThread, ReadLeft>
    <<<1, ThreadsInBlock>>>(data);

  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());
}


template <typename DataType,
          unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock,
          bool ReadLeft = false>
void MiddleTileTest(const DataType *input,
                    DataType *output,
                    DataType neighbour_tile_value)
{
  MiddleTileTestKernel<DataType, ThreadsInBlock, ItemsPerThread, ReadLeft>
    <<<1, ThreadsInBlock>>>(input, output, neighbour_tile_value);

  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());
}


template <typename DataType,
          unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock,
          bool ReadLeft = false>
void LastTileInplaceTest(const DataType *input,
                         DataType *output,
                         unsigned int valid_items)
{
  LastTileTestInplaceKernel<DataType, ThreadsInBlock, ItemsPerThread, ReadLeft>
    <<<1, ThreadsInBlock>>>(input, output, valid_items);

  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());
}


template <typename DataType,
          unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock,
          bool ReadLeft = false>
void InplaceTest(DataType *data)
{
  TestInplaceKernel<DataType, ThreadsInBlock, ItemsPerThread, ReadLeft>
    <<<1, ThreadsInBlock>>>(data);

  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());
}


template <typename DataType,
          unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock,
          bool ReadLeft = false>
void MiddleTileInplaceTest(const DataType *input,
                           DataType *output,
                           DataType neighbour_tile_value)
{
  MiddleTileInplaceTestKernel<DataType, ThreadsInBlock, ItemsPerThread, ReadLeft>
    <<<1, ThreadsInBlock>>>(input, output, neighbour_tile_value);

  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());
}


template <typename FirstIteratorT,
          typename SecondOperatorT>
bool CheckResult(FirstIteratorT first_begin,
                 FirstIteratorT first_end,
                 SecondOperatorT second_begin)
{
  auto err = thrust::mismatch(first_begin, first_end, second_begin);

  if (err.first != first_end)
  {
    return false;
  }

  return true;
}


template <typename DataType,
          unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock>
void TestLastTile(bool inplace,
                  unsigned int num_items,
                  thrust::device_vector<DataType> &d_input)
{
  thrust::tabulate(d_input.begin(),
                   d_input.end(),
                   TestSequenceGenerator<DataType>{});
  thrust::device_vector<DataType> d_output(d_input.size());

  constexpr bool read_left = true;
  constexpr bool read_right = false;

  DataType *d_input_ptr = thrust::raw_pointer_cast(d_input.data());
  DataType *d_output_ptr = thrust::raw_pointer_cast(d_output.data());

  if (inplace)
  {
    LastTileInplaceTest<DataType, ItemsPerThread, ThreadsInBlock, read_left>(
      d_input_ptr,
      d_output_ptr,
      num_items);
  }
  else
  {
    LastTileTest<DataType, ItemsPerThread, ThreadsInBlock, read_left>(
      d_input_ptr,
      d_output_ptr,
      num_items);
  }

  {
    using CountingIteratorT =
      typename thrust::counting_iterator<DataType,
        thrust::use_default,
        std::size_t,
        std::size_t>;

    AssertEquals(d_output.front(), d_input.front());
    AssertTrue(CheckResult(d_output.begin() + 1,
                           d_output.begin() + num_items,
                           CountingIteratorT(DataType{0})));
    AssertTrue(CheckResult(d_output.begin() + num_items,
                           d_output.end(),
                           d_input.begin() + num_items));
  }


  thrust::tabulate(d_input.begin(),
                   d_input.end(),
                   TestSequenceGenerator<DataType>{});

  if (inplace)
  {
    LastTileInplaceTest<DataType, ItemsPerThread, ThreadsInBlock, read_right>(
      d_input_ptr,
      d_output_ptr,
      num_items);
  }
  else
  {
    LastTileTest<DataType, ItemsPerThread, ThreadsInBlock, read_right>(
      d_input_ptr,
      d_output_ptr,
      num_items);
  }

  {
    thrust::device_vector<DataType> reference(num_items);
    thrust::sequence(reference.begin(),
                     reference.end(),
                     static_cast<DataType>(0),
                     static_cast<DataType>(-1));

    AssertTrue(CheckResult(d_output.begin(),
                           d_output.begin() + num_items - 1,
                           reference.begin()));
    AssertTrue(CheckResult(d_output.begin() + num_items - 1,
                           d_output.end(),
                           d_input.begin() + num_items - 1));
  }
}


template <typename DataType,
          unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock>
void TestMiddleTile(bool inplace,
                    thrust::device_vector<DataType> &d_input)
{
  thrust::tabulate(d_input.begin(),
                   d_input.end(),
                   TestSequenceGenerator<DataType>{std::size_t{1}});
  thrust::device_vector<DataType> d_output(d_input.size());

  constexpr bool read_left  = true;
  constexpr bool read_right = false;

  DataType *d_input_ptr  = thrust::raw_pointer_cast(d_input.data());
  DataType *d_output_ptr = thrust::raw_pointer_cast(d_output.data());

  const DataType left_tile_last_value{0};
  const DataType right_tile_first_value{
    TestSequenceGenerator<DataType>{}(d_input.size())
  };

  if (inplace)
  {
    MiddleTileInplaceTest<DataType, ItemsPerThread, ThreadsInBlock, read_left>(
      d_input_ptr,
      d_output_ptr,
      left_tile_last_value);
  }
  else
  {
    MiddleTileTest<DataType, ItemsPerThread, ThreadsInBlock, read_left>(
      d_input_ptr,
      d_output_ptr,
      left_tile_last_value);
  }

  {
    using CountingIteratorT =
      typename thrust::counting_iterator<DataType,
                                         thrust::use_default,
                                         std::size_t,
                                         std::size_t>;

    AssertTrue(CheckResult(d_output.begin(),
                           d_output.end(),
                           CountingIteratorT(DataType{0})));
  }

  thrust::tabulate(d_input.begin(),
                   d_input.end(),
                   TestSequenceGenerator<DataType>{});

  if (inplace)
  {
    MiddleTileInplaceTest<DataType, ItemsPerThread, ThreadsInBlock, read_right>(
      d_input_ptr,
      d_output_ptr,
      right_tile_first_value);
  }
  else
  {
    MiddleTileTest<DataType, ItemsPerThread, ThreadsInBlock, read_right>(
      d_input_ptr,
      d_output_ptr,
      right_tile_first_value);
  }

  {
    thrust::device_vector<DataType> reference(d_input.size());
    thrust::sequence(reference.begin(),
                     reference.end(),
                     static_cast<DataType>(0),
                     static_cast<DataType>(-1));

    AssertTrue(CheckResult(d_output.begin(),
                           d_output.end(),
                           reference.begin()));
  }
}


struct IntToCustomType
{
  unsigned int offset;

  IntToCustomType()
      : offset(0)
  {}

  explicit IntToCustomType(unsigned int offset)
      : offset(offset)
  {}

  __device__ __host__ CustomType operator()(unsigned int idx) const
  {
    return { idx + offset, idx + offset };
  }
};


template <typename DataType,
          unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock>
void TestFullTile(bool inplace,
                  thrust::device_vector<DataType> &d_data)
{
  thrust::tabulate(d_data.begin(),
                   d_data.end(),
                   TestSequenceGenerator<DataType>{});

  constexpr bool read_left  = true;
  constexpr bool read_right = false;

  DataType *d_data_ptr = thrust::raw_pointer_cast(d_data.data());

  if (inplace)
  {
    InplaceTest<DataType, ItemsPerThread, ThreadsInBlock, read_left>(
      d_data_ptr);
  }
  else
  {
    Test<DataType, ItemsPerThread, ThreadsInBlock, read_left>(d_data_ptr);
  }

  {
    using CountingIteratorT =
    typename thrust::counting_iterator<DataType,
      thrust::use_default,
      std::size_t,
      std::size_t>;

    AssertEquals(d_data.front(), TestSequenceGenerator<DataType>{}(0));
    AssertTrue(CheckResult(d_data.begin() + 1,
                           d_data.end(),
                           CountingIteratorT(DataType{0})));
  }

  thrust::tabulate(d_data.begin(),
                   d_data.end(),
                   TestSequenceGenerator<DataType>{});

  if (inplace)
  {
    InplaceTest<DataType, ItemsPerThread, ThreadsInBlock, read_right>(
      d_data_ptr);
  }
  else
  {
    Test<DataType, ItemsPerThread, ThreadsInBlock, read_right>(d_data_ptr);
  }

  {
    thrust::device_vector<DataType> reference(d_data.size());
    thrust::sequence(reference.begin(),
                     reference.end(),
                     static_cast<DataType>(0),
                     static_cast<DataType>(-1));

    AssertTrue(CheckResult(d_data.begin(),
                           d_data.end() - 1,
                           reference.begin()));
    AssertEquals(d_data.back(),
                 TestSequenceGenerator<DataType>{}(d_data.size() - 1));
  }
}


template <unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock>
void TestCustomType(bool inplace,
                    thrust::device_vector<CustomType> &d_data)
{
  thrust::tabulate(d_data.begin(), d_data.end(), IntToCustomType{1});
  CustomType *d_data_ptr = thrust::raw_pointer_cast(d_data.data());

  constexpr bool read_left  = true;
  constexpr bool read_right = false;

  if (inplace)
  {
    InplaceTest<CustomType, ItemsPerThread, ThreadsInBlock, read_left>(
      d_data_ptr);
  }
  else
  {
    Test<CustomType, ItemsPerThread, ThreadsInBlock, read_left>(d_data_ptr);
  }

  {
    const std::size_t expected_count = d_data.size();
    const std::size_t actual_count =
      thrust::count(d_data.begin(), d_data.end(), CustomType{1, 1});

    AssertEquals(expected_count, actual_count);
  }

  thrust::tabulate(d_data.begin(), d_data.end(), IntToCustomType{});

  if (inplace)
  {
    InplaceTest<CustomType, ItemsPerThread, ThreadsInBlock, read_right>(
      d_data_ptr);
  }
  else
  {
    Test<CustomType, ItemsPerThread, ThreadsInBlock, read_right>(d_data_ptr);
  }

  {
    const auto unsigned_minus_one = static_cast<unsigned int>(-1);

    const std::size_t expected_count = d_data.size() - 1;
    const std::size_t actual_count =
      thrust::count(d_data.begin(),
                    d_data.end() - 1,
                    CustomType{unsigned_minus_one, unsigned_minus_one});

    AssertEquals(expected_count, actual_count);
  }
}


template <
  typename ValueType,
  unsigned int ItemsPerThread,
  unsigned int ThreadsInBlock>
void Test(bool inplace)
{
  constexpr int tile_size = ItemsPerThread * ThreadsInBlock;
  thrust::device_vector<ValueType> d_values(tile_size);

  for (unsigned int num_items = tile_size; num_items > 1; num_items /= 2)
  {
    TestLastTile<ValueType, ItemsPerThread, ThreadsInBlock>(inplace,
                                                            num_items,
                                                            d_values);
  }

  TestFullTile<ValueType, ItemsPerThread, ThreadsInBlock>(inplace, d_values);
  TestMiddleTile<ValueType, ItemsPerThread, ThreadsInBlock>(inplace, d_values);
}


template <unsigned int ItemsPerThread,
          unsigned int ThreadsInBlock>
void TestCustomType(bool inplace)
{
  constexpr int tile_size = ItemsPerThread * ThreadsInBlock;
  thrust::device_vector<CustomType> d_values(tile_size);
  TestCustomType<ItemsPerThread, ThreadsInBlock>(inplace, d_values);
}


template <unsigned int ItemsPerThread, unsigned int ThreadsPerBlock>
void Test(bool inplace)
{
  Test<std::uint8_t,  ItemsPerThread, ThreadsPerBlock>(inplace);
  Test<std::uint16_t, ItemsPerThread, ThreadsPerBlock>(inplace);
  Test<std::uint32_t, ItemsPerThread, ThreadsPerBlock>(inplace);
  Test<std::uint64_t, ItemsPerThread, ThreadsPerBlock>(inplace);
}


template <unsigned int ItemsPerThread>
void Test(bool inplace)
{
  Test<ItemsPerThread, 32>(inplace);
  Test<ItemsPerThread, 256>(inplace);
}


template <unsigned int ItemsPerThread>
void Test()
{
  Test<ItemsPerThread>(false);
  Test<ItemsPerThread>(true);
}


int main(int argc, char** argv)
{
  CommandLineArgs args(argc, argv);

  // Initialize device
  CubDebugExit(args.DeviceInit());

  Test<1>();
  Test<2>();
  Test<10>();
  Test<15>();

  // More of a compilation check
  TestCustomType<5, 256>(true);

  return 0;
}
