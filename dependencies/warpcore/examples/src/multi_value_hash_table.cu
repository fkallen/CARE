#include <iostream>
#include <algorithm>
#include <random>
#include <warpcore/multi_value_hash_table.cuh>
#include <helpers/timers.cuh>

int main ()
{
    using namespace warpcore;

    // key/value types
    using key_t = std::uint32_t;
    using value_t = std::uint32_t;

    // configure the hash table
    using hash_table_t = MultiValueHashTable<
        key_t,
        value_t,
        defaults::empty_key<key_t>(), // empty sentinel
        defaults::tombstone_key<key_t>(), // tombstone sentinel
        defaults::probing_scheme_t<key_t, 8>>; // the cooperative probing scheme

    // this type represents the current status (errors/warnings) of the table
    using status_t = typename hash_table_t::status_type;
    // we want to catch the status per-query -> configure status handle
    using status_handler_t = typename status_handlers::ReturnStatus;

    // number of unique keys
    const index_t size_unique_keys = 1UL << 22;
    // number of values per unique key
    const index_t size_values_per_key = 8;
    // the actual number of input key/value pairs
    const index_t size = size_unique_keys * size_values_per_key;
    // target load factor of the hash table
    const float load_factor = 0.8;

    helpers::GpuTimer init_table_timer("init_table");
    // initialize the hash table (ctor)
    hash_table_t hash_table(size / load_factor);
    init_table_timer.print();
    hipDeviceSynchronize(); CUERR

    helpers::GpuTimer init_data_timer("init_data");
    key_t * keys_unique_h = nullptr;
    hipHostMalloc(&keys_unique_h, sizeof(key_t) * size_unique_keys); CUERR
    key_t * keys_unique_d = nullptr;
    hipMalloc(&keys_unique_d, sizeof(key_t) * size_unique_keys); CUERR
    key_t * keys_in_h = nullptr;
    hipHostMalloc(&keys_in_h, sizeof(key_t) * size); CUERR
    key_t * keys_in_d = nullptr;
    hipMalloc(&keys_in_d, sizeof(key_t) * size); CUERR

    value_t * values_in_h = nullptr;
    hipHostMalloc(&values_in_h, sizeof(value_t) * size); CUERR
    value_t * values_in_d = nullptr;
    hipMalloc(&values_in_d, sizeof(value_t) * size); CUERR

    index_t * offsets_out_d = nullptr;
    hipMalloc(&offsets_out_d, sizeof(index_t) * (size_unique_keys+1)); CUERR

    value_t * values_out_h = nullptr;
    hipHostMalloc(&values_out_h, sizeof(value_t) * size); CUERR
    value_t * values_out_d = nullptr;
    hipMalloc(&values_out_d, sizeof(value_t) * size); CUERR

    status_t * status_h = nullptr;
    hipHostMalloc(&status_h, sizeof(status_t) * size); CUERR
    status_t * status_d = nullptr;
    hipMalloc(&status_d, sizeof(status_t) * size); CUERR

    #pragma omp parallel for
    for(index_t i = 0; i < size_unique_keys; ++i)
    {
        keys_unique_h[i] = i + 1;

        for(index_t j = 0; j < size_values_per_key; ++j)
        {
            keys_in_h[i * size_values_per_key + j] = i + 1;
        }
    }

    std::random_device rd;
    std::mt19937 g(rd());

    std::shuffle(keys_in_h, keys_in_h + size, g);

    #pragma omp parallel for
    for(index_t i= 0; i < size; ++i)
    {
        values_in_h[i] = keys_in_h[i];
        status_h[i] = status_t::none();
    }

    hipMemcpy(keys_unique_d, keys_unique_h, sizeof(key_t)*size_unique_keys, H2D); CUERR
    hipMemcpy(keys_in_d, keys_in_h, sizeof(key_t)*size, H2D); CUERR
    hipMemcpy(values_in_d, values_in_h, sizeof(value_t)*size, H2D); CUERR
    hipMemset(values_out_d, 0, sizeof(value_t)*size); CUERR
    hipMemset(offsets_out_d, 0, sizeof(index_t)*(size_unique_keys+1)); CUERR
    hipMemcpy(status_d, status_h, sizeof(status_t)*size, H2D); CUERR
    init_data_timer.print();

    helpers::GpuTimer insert_timer("insert");
    hash_table.insert<status_handler_t>(
        keys_in_d,
        values_in_d,
        size,
        0,
        defaults::probing_length(),
        status_d);
    insert_timer.print_throughput((sizeof(key_t)+sizeof(value_t)), size);
    hipDeviceSynchronize(); CUERR

    hipMemcpy(status_h, status_d, sizeof(status_t)*size, D2H); CUERR

    std::cout << "table errors " << hash_table.peek_status().get_errors() << std::endl;
    index_t errors = 0;
    for(index_t i = 0; i < size; ++i)
    {

        if(status_h[i].has_any_errors())
        {
            if(errors++ < 10)
                std::cout <<
                    "STATUS: i " << i <<
                    " key " << keys_in_h[i] <<
                    " status " << status_h[i] << std::endl;
        }
    }
    if(errors >= 10)
    {
        std::cout << "...\n" << "total errors " << errors << std::endl;
    }

    std::cout << "num pairs " << size << std::endl;
    std::cout << "table size " << hash_table.size() << std::endl;
    std::cout << "capacity " << hash_table.capacity() << std::endl;
    std::cout << "load factor " << hash_table.load_factor() << std::endl;
    std::cout << "expected unique keys " << size_unique_keys << std::endl;
    std::cout << "actual unique keys " << hash_table.num_keys() << std::endl;
    std::cout << "values per key " << size_values_per_key << std::endl;
    std::cout << "total values " << size << std::endl;

    #pragma omp parallel for
    for(index_t i= 0; i < size; ++i)
    {
        status_h[i] = status_t::none();
    }

    hipMemcpy(status_d, status_h, sizeof(status_t)*size, H2D); CUERR

    index_t value_size = 0;

    helpers::GpuTimer retrieve_dummy_timer("retrieve_dummy");
    hash_table.retrieve<status_handler_t>(
        keys_unique_d,
        size_unique_keys,
        offsets_out_d,
        offsets_out_d+1,
        values_out_d,
        value_size,
        0,
        defaults::probing_length(),
        status_d);
    retrieve_dummy_timer.print_throughput((sizeof(key_t)+sizeof(value_t)), size);

    hipDeviceSynchronize(); CUERR

    helpers::GpuTimer retrieve_timer("retrieve");
    hash_table.retrieve<status_handler_t>(
        keys_unique_d,
        size_unique_keys,
        offsets_out_d,
        offsets_out_d+1,
        values_out_d,
        value_size,
        0,
        defaults::probing_length(),
        status_d);
    retrieve_timer.print_throughput((sizeof(key_t)+sizeof(value_t)), size);

    hipDeviceSynchronize(); CUERR

    std::cout << "retrieved values " << value_size << std::endl;

    helpers::lambda_kernel<<<SDIV(size_unique_keys, 1024), 1024>>>([=] DEVICEQUALIFIER
    {
        const index_t tid = blockDim.x * blockIdx.x + threadIdx.x;

        if(tid < size_unique_keys)
        {
            const auto key = keys_unique_d[tid];
            const auto lower = offsets_out_d[tid];
            const auto upper = offsets_out_d[tid + 1];

            if(upper - lower != size_values_per_key)
            {
                printf("ERROR size values %llu\n", upper - lower);
            }

            for (index_t i = lower; i < upper; i++)
            {
                if(values_out_d[i] != key)
                {
                    printf("ERROR expected %u got %u\n", key, values_out_d[i]);
                }
            }
        }
    });

    hipDeviceSynchronize(); CUERR

    hipMemcpy(status_h, status_d, sizeof(status_t)*size, D2H); CUERR

    std::cout << "table status " << hash_table.peek_status() << std::endl;

    errors = 0;
    for(index_t i = 0; i < size; ++i)
    {

        if(status_h[i].has_any())
        {
            if(errors++ < 10)
                std::cout <<
                    "STATUS: i " << i <<
                    " key " << keys_in_h[i] <<
                    " status " << status_h[i] << std::endl;
        }
    }
    if(errors >= 10)
    {
        std::cout << "...\n" << "total errors " << errors << std::endl;
    }

    hipHostFree(keys_unique_h);
    hipHostFree(keys_in_h);
    hipHostFree(values_in_h);
    hipHostFree(values_out_h);
    hipHostFree(status_h);
    hipFree(keys_unique_d);
    hipFree(keys_in_d);
    hipFree(values_in_d);
    hipFree(offsets_out_d);
    hipFree(values_out_d);
    hipFree(status_d);

    hipDeviceSynchronize(); CUERR
}
