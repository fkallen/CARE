#include <iostream>
#include <algorithm>
#include <random>
#include <warpcore/multi_bucket_hash_table.cuh>
#include <helpers/timers.cuh>

int main ()
{
    using namespace warpcore;

    // key/value types
    using key_t = std::uint32_t;
    using value_t = std::uint64_t;

    // configure the hash table
    using hash_table_t = MultiBucketHashTable<
        key_t,
        value_t,
        defaults::empty_key<key_t>(),
        defaults::tombstone_key<key_t>(),
        defaults::empty_key<value_t>(),
        defaults::probing_scheme_t<key_t, 8>,
        storage::key_value::AoSStore<key_t, ArrayBucket<value_t,4>>>;
    using status_t = typename hash_table_t::status_type;
    using status_handler_t = typename status_handlers::ReturnStatus;

    const index_t size_unique_keys = 1UL << 5;
    const index_t size_values_per_key = 1UL << 15;
    const index_t max_values_per_key = 21;
    const index_t size = size_unique_keys * size_values_per_key;
    const index_t max_buckets = size_unique_keys * SDIV(max_values_per_key,4);
    const float load_factor = 0.95;

    helpers::GpuTimer init_table_timer("init_table");
    hash_table_t hash_table(max_buckets / load_factor, defaults::seed<key_t>(), max_values_per_key);
    init_table_timer.print();
    hipDeviceSynchronize(); CUERR

    helpers::GpuTimer init_data_timer("init_data");
    key_t * keys_unique_h = nullptr;
    hipHostMalloc(&keys_unique_h, sizeof(key_t) * size_unique_keys); CUERR
    key_t * keys_unique_d = nullptr;
    hipMalloc(&keys_unique_d, sizeof(key_t) * size_unique_keys); CUERR

    key_t * keys_in_h = nullptr;
    hipHostMalloc(&keys_in_h, sizeof(key_t) * size); CUERR
    key_t * keys_in_d = nullptr;
    hipMalloc(&keys_in_d, sizeof(key_t) * size); CUERR

    value_t * values_in_h = nullptr;
    hipHostMalloc(&values_in_h, sizeof(value_t) * size); CUERR
    value_t * values_in_d = nullptr;
    hipMalloc(&values_in_d, sizeof(value_t) * size); CUERR

    index_t * offsets_out_d = nullptr;
    hipMalloc(&offsets_out_d, sizeof(index_t) * (size_unique_keys+1)); CUERR

    value_t * values_out_h = nullptr;
    hipHostMalloc(&values_out_h, sizeof(value_t) * size); CUERR
    value_t * values_out_d = nullptr;
    hipMalloc(&values_out_d, sizeof(value_t) * size); CUERR

    status_t * status_h = nullptr;
    hipHostMalloc(&status_h, sizeof(status_t) * size); CUERR
    status_t * status_d = nullptr;
    hipMalloc(&status_d, sizeof(status_t) * size); CUERR

    #pragma omp parallel for
    for(index_t i = 0; i < size_unique_keys; ++i)
    {
        keys_unique_h[i] = i + 1;

        for(index_t j = 0; j < size_values_per_key; ++j)
        {
            keys_in_h[i * size_values_per_key + j] = i + 1;
        }
    }

    std::random_device rd;
    std::mt19937 g(rd());

    std::shuffle(keys_in_h, keys_in_h + size, g);

    #pragma omp parallel for
    for(index_t i= 0; i < size; ++i)
    {
        values_in_h[i] = keys_in_h[i];
        status_h[i] = status_t::none();
    }

    hipMemcpy(keys_in_d, keys_in_h, sizeof(key_t)*size, H2D); CUERR
    hipMemcpy(values_in_d, values_in_h, sizeof(value_t)*size, H2D); CUERR
    hipMemcpy(status_d, status_h, sizeof(status_t)*size, H2D); CUERR

    hipMemset(values_out_d, 0, sizeof(value_t)*size); CUERR
    hipMemset(offsets_out_d, 0, sizeof(index_t)*(size_unique_keys+1)); CUERR
    init_data_timer.print();

    const size_t batch_size = 1UL << 15;
    const size_t full_batches = size / batch_size;
    const size_t last_batch_size = size % batch_size;

    helpers::GpuTimer insert_timer("insert");
    for(size_t b=0; b<full_batches; ++b)
    {
        hash_table.insert<status_handler_t>(
            keys_in_d + b*batch_size,
            values_in_d + b*batch_size,
            batch_size,
            0,
            defaults::probing_length(),
            status_d + b*batch_size);
    }
    if(last_batch_size)
    {
        hash_table.insert<status_handler_t>(
            keys_in_d + full_batches*batch_size,
            values_in_d + full_batches*batch_size,
            last_batch_size,
            0,
            defaults::probing_length(),
            status_d + full_batches*batch_size);
    }
    insert_timer.print_throughput((sizeof(key_t)+sizeof(value_t)), size);
    hipDeviceSynchronize(); CUERR

    hipMemcpy(status_h, status_d, sizeof(status_t)*size, D2H); CUERR

    std::cout << "table errors " << hash_table.peek_status().get_errors() << std::endl;
    index_t errors = 0;
    for(index_t i = 0; i < size; ++i)
    {
        status_h[i] -= status_t::duplicate_key();

        if(status_h[i].has_any_errors())
        {
            if(errors++ < 10)
                std::cout <<
                    "STATUS: i " << i <<
                    " key " << keys_in_h[i] <<
                    " status " << status_h[i] << std::endl;
        }
    }
    if(errors >= 10)
    {
        std::cout << "...\n" << "total errors " << errors << std::endl;
    }

    std::cout << "num pairs " << size << std::endl;
    std::cout << "table size " << hash_table.size() << std::endl;
    std::cout << "key capacity " << hash_table.capacity() << std::endl;
    std::cout << "load factor " << hash_table.key_load_factor() << std::endl;
    std::cout << "expected unique keys " << size_unique_keys << std::endl;
    std::cout << "actual unique keys " << hash_table.num_keys() << std::endl;
    std::cout << "values per key " << size_values_per_key << std::endl;
    std::cout << "total values " << size << std::endl;
    std::cout << "capped values " << size_unique_keys*max_values_per_key << std::endl;

    #pragma omp parallel for
    for(index_t i= 0; i < size; ++i)
    {
        status_h[i] = status_t::none();
    }

    hipMemcpy(status_d, status_h, sizeof(status_t)*size, H2D); CUERR

    index_t value_size = 0;

    index_t count_unique_keys = 0;
    hash_table.retrieve_all_keys(
        keys_unique_d,
        count_unique_keys);
    hipDeviceSynchronize(); CUERR

    std::cout << "retrieved keys " << count_unique_keys << std::endl;

    helpers::GpuTimer retrieve_timer("retrieve");
    hash_table.retrieve<status_handler_t>(
        keys_unique_d,
        count_unique_keys,
        offsets_out_d,
        offsets_out_d+1,
        values_out_d,
        value_size,
        0,
        defaults::probing_length(),
        status_d);
    retrieve_timer.print_throughput((sizeof(key_t)+sizeof(value_t)), size);

    hipDeviceSynchronize(); CUERR

    std::cout << "retrieved values " << value_size << std::endl;

    helpers::lambda_kernel<<<SDIV(size_unique_keys, 1024), 1024>>>([=] DEVICEQUALIFIER
    {
        const index_t tid = blockDim.x * blockIdx.x + threadIdx.x;

        if(tid < size_unique_keys)
        {
            const auto key = keys_unique_d[tid];
            const auto lower = offsets_out_d[tid];
            const auto upper = offsets_out_d[tid + 1];

            if(upper - lower != max_values_per_key)
            {
                printf("ERROR size values %llu (expected %llu)\n", upper - lower, max_values_per_key);
            }

            for (index_t i = lower; i < upper; i++)
            {
                if(values_out_d[i] != key)
                {
                    printf("ERROR expected %u got %llu\n", key, values_out_d[i]);
                }
            }
        }
    });

    hipDeviceSynchronize(); CUERR

    hipMemcpy(status_h, status_d, sizeof(status_t)*size, D2H); CUERR

    std::cout << "table errors " << hash_table.peek_status().get_errors() << std::endl;

    errors = 0;
    for(index_t i = 0; i < size; ++i)
    {

        if(status_h[i].has_any_errors())
        {
            if(errors++ < 10)
                std::cout <<
                "STATUS: i " << i <<
                " key " << keys_in_h[i] <<
                " status " << status_h[i] << std::endl;
        }
    }
    if(errors >= 10)
    {
        std::cout << "...\n" << "total errors " << errors << std::endl;
    }

    hipHostFree(keys_unique_h);
    hipHostFree(keys_in_h);
    hipHostFree(values_in_h);
    hipHostFree(values_out_h);
    hipHostFree(status_h);
    hipFree(keys_unique_d);
    hipFree(keys_in_d);
    hipFree(values_in_d);
    hipFree(offsets_out_d);
    hipFree(values_out_d);
    hipFree(status_d);

    hipDeviceSynchronize(); CUERR
}
