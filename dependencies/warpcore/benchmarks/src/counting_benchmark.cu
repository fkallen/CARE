#include <warpcore/counting_hash_table.cuh>
#include "../include/benchmark_common.cuh"

template<class HashTable>
HOSTQUALIFIER INLINEQUALIFIER
void counting_benchmark(
    const typename HashTable::key_type * keys_d,
    const uint64_t max_keys,
    std::vector<uint64_t> input_sizes,
    std::vector<float> load_factors,
    bool print_headers = true,
    uint8_t iters = 5,
    std::chrono::milliseconds thermal_backoff = std::chrono::milliseconds(100))
{
    using key_t = typename HashTable::key_type;
    using count_t = typename HashTable::value_type;

    count_t* counts_d = nullptr;
    hipMalloc(&counts_d, sizeof(count_t)*max_keys); CUERR

    const auto max_input_size =
        *std::max_element(input_sizes.begin(), input_sizes.end());
    const auto min_load_factor =
        *std::min_element(load_factors.begin(), load_factors.end());

    if(max_input_size > max_keys)
    {
        std::cerr << "Maximum input size exceeded." << std::endl;
        exit(1);
    }

    const uint64_t max_unique_size = num_unique(keys_d, max_input_size);

    if(!sufficient_memory_oa<HashTable>(max_unique_size / min_load_factor))
    {
        std::cerr << "Not enough GPU memory." << std::endl;
        exit(1);
    }

    for(auto size : input_sizes)
    {
        for(auto load : load_factors)
        {
            const uint64_t unique_size = num_unique(keys_d, size);
            const uint64_t capacity = unique_size/load;

            HashTable hash_table(capacity);

            Output<key_t> output;
            output.sample_size = size;
            output.key_capacity = hash_table.capacity();

            output.insert_ms = benchmark_insert(
                hash_table, keys_d, size,
                iters, thermal_backoff);

            output.query_ms = benchmark_query(
                hash_table, keys_d, counts_d, size,
                iters, thermal_backoff);

            output.key_load_factor = hash_table.load_factor();
            output.density = output.key_load_factor;
            output.status = hash_table.pop_status();

            if(print_headers)
                output.print_with_headers();
            else
                output.print_without_headers();
        }
    }

    hipFree(counts_d); CUERR
}

int main(int argc, char* argv[])
{
    using key_t = uint32_t;
    using count_t = uint32_t;

    const uint64_t max_keys = 1UL << 28;

    const bool print_headers = true;

    uint64_t dev_id = 0;
    if(argc > 2) dev_id = std::atoi(argv[2]);
    hipSetDevice(dev_id); CUERR

    key_t * keys_d = nullptr;
    if(argc > 1)
        keys_d = load_keys<key_t>(argv[1], max_keys);
    else
        keys_d = generate_keys<key_t>(max_keys, 8);

    using hash_table_t = warpcore::CountingHashTable<key_t, count_t>;

    counting_benchmark<hash_table_t>(
        keys_d, max_keys, {max_keys}, {0.9}, print_headers);

    hipFree(keys_d); CUERR
}