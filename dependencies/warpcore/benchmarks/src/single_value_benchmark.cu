#include <warpcore/single_value_hash_table.cuh>
#include "../include/benchmark_common.cuh"

template<class HashTable>
HOSTQUALIFIER INLINEQUALIFIER
void single_value_benchmark(
    const typename HashTable::key_type * keys_d,
    const uint64_t max_keys,
    std::vector<uint64_t> input_sizes,
    std::vector<float> load_factors,
    bool print_headers = true,
    uint8_t iters = 5,
    std::chrono::milliseconds thermal_backoff = std::chrono::milliseconds(100))
{
    using key_t = typename HashTable::key_type;
    using value_t = typename HashTable::value_type;

    value_t* values_d = nullptr;
    hipMalloc(&values_d, sizeof(value_t)*max_keys); CUERR

    const auto max_input_size =
        *std::max_element(input_sizes.begin(), input_sizes.end());
    const auto min_load_factor =
        *std::min_element(load_factors.begin(), load_factors.end());

    if(max_input_size > max_keys)
    {
        std::cerr << "Maximum input size exceeded." << std::endl;
        exit(1);
    }

    if(!sufficient_memory_oa<HashTable>(max_input_size / min_load_factor))
    {
        std::cerr << "Not enough GPU memory." << std::endl;
        exit(1);
    }

    for(auto size : input_sizes)
    {
        for(auto load : load_factors)
        {
            const std::uint64_t capacity = size / load;

            HashTable hash_table(capacity);

            Output<key_t,value_t> output;
            output.sample_size = size;
            output.key_capacity = hash_table.capacity();

            output.insert_ms = benchmark_insert(
                hash_table, keys_d, values_d, size,
                iters, thermal_backoff);

            output.query_ms = benchmark_query(
                hash_table, keys_d, values_d, size,
                iters, thermal_backoff);

            output.key_load_factor = hash_table.load_factor();
            output.density = output.key_load_factor;
            output.status = hash_table.pop_status();

            if(print_headers)
                output.print_with_headers();
            else
                output.print_without_headers();
        }
    }

    hipFree(values_d); CUERR
}

int main(int argc, char* argv[])
{
    using namespace warpcore;

    using key_t = std::uint32_t;
    using value_t = std::uint32_t;

    const uint64_t max_keys = 1UL << 27;

    const bool print_headers = true;

    uint64_t dev_id = 0;
    if(argc > 2) dev_id = std::atoi(argv[2]);
    hipSetDevice(dev_id); CUERR

    key_t * keys_d = nullptr;
    if(argc > 1)
        keys_d = load_keys<key_t>(argv[1], max_keys);
    else
        keys_d = generate_keys<key_t>(max_keys, 1);

    using hash_table_t = SingleValueHashTable<
        key_t,
        value_t,
        defaults::empty_key<key_t>(),
        defaults::tombstone_key<key_t>(),
        defaults::probing_scheme_t<key_t, 8>,
        storage::key_value::AoSStore<key_t, value_t>>;

    single_value_benchmark<hash_table_t>(
        keys_d, max_keys, {max_keys}, {0.8}, print_headers);

    hipFree(keys_d); CUERR
}
