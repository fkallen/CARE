#include "hip/hip_runtime.h"
#include "kernels.hpp"
#include "msa.hpp"
#include "bestalignment.hpp"
#include "../hpc_helpers.cuh"

#include <cassert>

#ifdef __NVCC__
#include <hipcub/hipcub.hpp>
#endif

namespace care{
namespace gpu{

#ifdef __NVCC__

	void call_cuda_filter_alignments_by_mismatchratio_kernel_async(
                                        BestAlignment_t* d_alignment_best_alignment_flags,
                                        const int* d_alignment_overlaps,
                                        const int* d_alignment_nOps,
                                        const int* d_indices,
										const int* d_indices_per_subject,
										const int* d_indices_per_subject_prefixsum,
                                        int n_subjects,
										int n_candidates,
										const int* d_num_indices,
                                        double mismatchratioBaseFactor,
										double goodAlignmentsCountThreshold,
										hipStream_t stream){

		const int blocksize = 128;

        dim3 block(blocksize, 1, 1);
        dim3 grid(n_subjects);

        #define mycall(blocksize) cuda_filter_alignments_by_mismatchratio_kernel<(blocksize)> \
                                <<<grid, block, 0, stream>>>( \
                                    d_alignment_best_alignment_flags, \
                                    d_alignment_overlaps, \
                                    d_alignment_nOps, \
                                    d_indices, \
                                    d_indices_per_subject, \
                                    d_indices_per_subject_prefixsum, \
                                    n_subjects, \
                                    n_candidates, \
                                    d_num_indices, \
                                    mismatchratioBaseFactor, \
                                    goodAlignmentsCountThreshold); CUERR;

        switch(blocksize){
            case 32: mycall(32); break;
            case 64: mycall(64); break;
            case 96: mycall(96); break;
            case 128: mycall(128); break;
            case 160: mycall(160); break;
            case 192: mycall(192); break;
            case 224: mycall(224); break;
            case 256: mycall(256); break;
            default: mycall(256); break;
        }

        //hipDeviceSynchronize(); CUERR;

		#undef mycall
    }


    __global__
    void msa_find_consensus_kernel(
                            char* __restrict__ d_consensus,
                            float* __restrict__ d_support,
                            int* __restrict__ d_coverage,
                            float* __restrict__ d_origWeights,
                            int* __restrict__ d_origCoverages,
                            const char* __restrict__ d_multiple_sequence_alignments,
                            const float* __restrict__ d_multiple_sequence_alignment_weights,
                            const MSAColumnProperties* __restrict__ d_msa_column_properties,
                            const int* __restrict__ d_candidates_per_subject_prefixsum,
                            const int* __restrict__ d_indices_per_subject,
                            const int* __restrict__ d_indices_per_subject_prefixsum,
                            int n_subjects,
                            int n_queries,
                            const int* __restrict__ d_num_indices,
                            size_t msa_pitch,
                            size_t msa_weights_pitch,
                            int msa_max_column_count,
                            int blocks_per_msa){

        const size_t msa_weights_pitch_floats = msa_weights_pitch / sizeof(float);

        const int localBlockId = blockIdx.x % blocks_per_msa;
		//const int n_indices = *d_num_indices;

        //process multiple sequence alignment of each subject
        //for each column in msa, find consensus and support
        for(unsigned subjectIndex = blockIdx.x / blocks_per_msa; subjectIndex < n_subjects; subjectIndex += gridDim.x / blocks_per_msa){
            const int subjectColumnsBegin_incl = d_msa_column_properties[subjectIndex].subjectColumnsBegin_incl;
            const int subjectColumnsEnd_excl = d_msa_column_properties[subjectIndex].subjectColumnsEnd_excl;
			const int columnsToCheck = d_msa_column_properties[subjectIndex].columnsToCheck;

            //number of rows in multiple sequence alignment for subject[subjectIndex]
            const int msa_rows = 1 + d_indices_per_subject[subjectIndex];

            const unsigned offset1 = msa_pitch * (subjectIndex + d_indices_per_subject_prefixsum[subjectIndex]);
            const unsigned offset2 = msa_weights_pitch_floats * (subjectIndex + d_indices_per_subject_prefixsum[subjectIndex]);

            const char* const my_multiple_sequence_alignment = d_multiple_sequence_alignments + offset1;
            const float* const my_multiple_sequence_alignment_weight = d_multiple_sequence_alignment_weights + offset2;

            char* const my_consensus = d_consensus + subjectIndex * msa_pitch;
            float* const my_support = d_support + subjectIndex * msa_weights_pitch_floats;
            int* const my_coverage = d_coverage + subjectIndex * msa_weights_pitch_floats;

            float* const my_orig_weights = d_origWeights + subjectIndex * msa_weights_pitch_floats;
            int* const my_orig_coverage = d_origCoverages + subjectIndex * msa_weights_pitch_floats;

            for(int column = localBlockId * blockDim.x + threadIdx.x; column < columnsToCheck; column += blocks_per_msa * blockDim.x){

                float Aw = 0.0f;
                float Cw = 0.0f;
                float Gw = 0.0f;
                float Tw = 0.0f;

                int As = 0;
                int Cs = 0;
                int Gs = 0;
                int Ts = 0;

                int columnCoverage = 0;

                for(int row = 0; row < msa_rows; ++row){
                    const char base = my_multiple_sequence_alignment[row * msa_pitch + column];
                    const float weight = my_multiple_sequence_alignment_weight[row * msa_weights_pitch_floats + column];

					//if(!(base == 'A' || base == 'C' || base == 'G' || base == 'T')){
					//	assert(base == 'A' || base == 'C' || base == 'G' || base == 'T');
					//}

                    Aw += (base == 'A' ? weight : 0);
                    Cw += (base == 'C' ? weight : 0);
                    Gw += (base == 'G' ? weight : 0);
                    Tw += (base == 'T' ? weight : 0);

                    As += (base == 'A');
                    Cs += (base == 'C');
                    Gs += (base == 'G');
                    Ts += (base == 'T');

                    columnCoverage += (base == 'A' || base == 'C' || base == 'G' || base == 'T');
                }

				if(columnCoverage <= 0){
					//assert(columnCoverage > 0);
				}

                const float columnWeight = Aw + Cw + Gw + Tw;
                float consWeight = Aw;
                char cons = 'A';

                cons = Cw > consWeight ? 'C' : cons;
                consWeight = Cw > consWeight ? Cw : consWeight;

                cons = Gw > consWeight ? 'G' : cons;
                consWeight = Gw > consWeight ? Gw : consWeight;

                cons = Tw > consWeight ? 'T' : cons;
                consWeight = Tw > consWeight ? Tw : consWeight;

                my_consensus[column] = cons;
                my_support[column] = consWeight / columnWeight;
				//printf("subject %d, column %d, support %f\n", subjectIndex, column, my_support[column]);
                my_coverage[column] = columnCoverage;

                if(subjectColumnsBegin_incl <= column && column < subjectColumnsEnd_excl){
                    const char subjectbase = my_multiple_sequence_alignment[column];
                    if(subjectbase == 'A'){
                        my_orig_weights[column] = Aw;
                        my_orig_coverage[column] = As;
                    }else if(subjectbase == 'C'){
                        my_orig_weights[column] = Cw;
                        my_orig_coverage[column] = Cs;
                    }else if(subjectbase == 'G'){
                        my_orig_weights[column] = Gw;
                        my_orig_coverage[column] = Gs;
                    }else if(subjectbase == 'T'){
                        my_orig_weights[column] = Tw;
                        my_orig_coverage[column] = Ts;
                    }
                }
            }
        }
    }

    void call_msa_find_consensus_kernel_async(
                            char* d_consensus,
                            float* d_support,
                            int* d_coverage,
                            float* d_origWeights,
                            int* d_origCoverages,
                            const char* d_multiple_sequence_alignments,
                            const float* d_multiple_sequence_alignment_weights,
                            const MSAColumnProperties* d_msa_column_properties,
                            const int* d_candidates_per_subject_prefixsum,
                            const int* d_indices_per_subject,
                            const int* d_indices_per_subject_prefixsum,
                            int n_subjects,
                            int n_queries,
                            const int* d_num_indices,
                            size_t msa_pitch,
                            size_t msa_weights_pitch,
                            int msa_max_column_count,
                            hipStream_t stream){

        const int blocksize = 128;
        const int blocks_per_msa = 2; //SDIV(msa_max_column_count, blocksize);

        dim3 block(blocksize, 1, 1);
        dim3 grid(n_subjects * blocks_per_msa, 1, 1);

        msa_find_consensus_kernel<<<grid, block, 0, stream>>>(d_consensus,
                                                            d_support,
                                                            d_coverage,
                                                            d_origWeights,
                                                            d_origCoverages,
                                                            d_multiple_sequence_alignments,
                                                            d_multiple_sequence_alignment_weights,
                                                            d_msa_column_properties,
                                                            d_candidates_per_subject_prefixsum,
                                                            d_indices_per_subject,
                                                            d_indices_per_subject_prefixsum,
                                                            n_subjects,
                                                            n_queries,
                                                            d_num_indices,
                                                            msa_pitch,
                                                            msa_weights_pitch,
                                                            msa_max_column_count,
                                                            blocks_per_msa); CUERR;

    }


    template<int BLOCKSIZE>
    __global__
    void msa_correct_subject_kernel(
                            const char* __restrict__ d_consensus,
                            const float* __restrict__ d_support,
                            const int* __restrict__ d_coverage,
                            const int* __restrict__ d_origCoverages,
                            const char* __restrict__ d_multiple_sequence_alignments,
                            const MSAColumnProperties* __restrict__ d_msa_column_properties,
                            const int* __restrict__ d_indices_per_subject_prefixsum,
                            bool* __restrict__ d_is_high_quality_subject,
                            char* __restrict__ d_corrected_subjects,
							bool* __restrict__ d_subject_is_corrected,
                            int n_subjects,
                            int n_queries,
                            const int* __restrict__ d_num_indices,
                            size_t sequence_pitch,
                            size_t msa_pitch,
                            size_t msa_weights_pitch,
                            double estimatedErrorrate,
                            double avg_support_threshold,
                            double min_support_threshold,
                            double min_coverage_threshold,
                            int k_region){

        using BlockReduceBool = hipcub::BlockReduce<bool, BLOCKSIZE>;
        using BlockReduceInt = hipcub::BlockReduce<int, BLOCKSIZE>;
        using BlockReduceFloat = hipcub::BlockReduce<float, BLOCKSIZE>;

        __shared__ union {
            typename BlockReduceBool::TempStorage boolreduce;
            typename BlockReduceInt::TempStorage intreduce;
            typename BlockReduceFloat::TempStorage floatreduce;
        } temp_storage;

        __shared__ bool broadcastbuffer;

        auto isGoodAvgSupport = [&](double avgsupport){
            return avgsupport >= avg_support_threshold;
        };
        auto isGoodMinSupport = [&](double minsupport){
            return minsupport >= min_support_threshold;
        };
        auto isGoodMinCoverage = [&](double mincoverage){
            return mincoverage >= min_coverage_threshold;
        };

        const size_t msa_weights_pitch_floats = msa_weights_pitch / sizeof(float);
		//const int n_indices = *d_num_indices;

        for(unsigned subjectIndex = blockIdx.x; subjectIndex < n_subjects; subjectIndex += gridDim.x){
            const float* const my_support = d_support + msa_weights_pitch_floats * subjectIndex;
            const int* const my_coverage = d_coverage + msa_weights_pitch_floats * subjectIndex;
            const int* const my_orig_coverage = d_origCoverages + msa_weights_pitch_floats * subjectIndex;
            const char* const my_consensus = d_consensus + msa_pitch  * subjectIndex;
            char* const my_corrected_subject = d_corrected_subjects + subjectIndex * sequence_pitch;

            const MSAColumnProperties properties = d_msa_column_properties[subjectIndex];
            const int subjectColumnsBegin_incl = properties.subjectColumnsBegin_incl;
            const int subjectColumnsEnd_excl = properties.subjectColumnsEnd_excl;

            float avg_support = 0;
            float min_support = 1.0f;
            //int max_coverage = 0;
            int min_coverage = std::numeric_limits<int>::max();

            for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                assert(i < properties.columnsToCheck);

                avg_support += my_support[i];
                min_support = min(my_support[i], min_support);
                //max_coverage = max(my_coverage[i], max_coverage);
                min_coverage = min(my_coverage[i], min_coverage);
            }

            avg_support = BlockReduceFloat(temp_storage.floatreduce).Sum(avg_support);
			__syncthreads();

            min_support = BlockReduceFloat(temp_storage.floatreduce).Reduce(min_support, hipcub::Min());
			__syncthreads();

            //max_coverage = BlockReduceInt(temp_storage.intreduce).Reduce(max_coverage, hipcub::Max());

            min_coverage = BlockReduceInt(temp_storage.intreduce).Reduce(min_coverage, hipcub::Min());
			__syncthreads();

            avg_support /= (subjectColumnsEnd_excl - subjectColumnsBegin_incl);

            bool isHQ = isGoodAvgSupport(avg_support) && isGoodMinSupport(min_support) && isGoodMinCoverage(min_coverage);

            if(threadIdx.x == 0){
                broadcastbuffer = isHQ;
                d_is_high_quality_subject[subjectIndex] = isHQ;
            }
            __syncthreads();

            isHQ = broadcastbuffer;

            if(isHQ){
                for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                    my_corrected_subject[i - subjectColumnsBegin_incl] = my_consensus[i];
                }
                if(threadIdx.x == 0){
                    d_subject_is_corrected[subjectIndex] = true;
                }
            }else{
                const unsigned offset1 = msa_pitch * (subjectIndex + d_indices_per_subject_prefixsum[subjectIndex]);
                const char* const my_multiple_sequence_alignment = d_multiple_sequence_alignments + offset1;

                //copy orignal sequence, which is in first row of msa, to corrected sequences
                for(int i = subjectColumnsBegin_incl + threadIdx.x; i < subjectColumnsEnd_excl; i += BLOCKSIZE){
                    my_corrected_subject[i - subjectColumnsBegin_incl] = my_multiple_sequence_alignment[i];
                }

                const int subjectLength = subjectColumnsEnd_excl - subjectColumnsBegin_incl;

                bool foundAColumn = false;
                for(int i = threadIdx.x; i < subjectLength; i += BLOCKSIZE){
                    const int globalIndex = subjectColumnsBegin_incl + i;

                    if(my_support[globalIndex] > 0.5 && my_orig_coverage[globalIndex] <= min_coverage_threshold){
                        double avgsupportkregion = 0;
                        int c = 0;
                        bool kregioncoverageisgood = true;

                        for(int j = i - k_region/2; j <= i + k_region/2 && kregioncoverageisgood; j++){
                            if(j != i && j >= 0 && j < subjectLength){
                                avgsupportkregion += my_support[subjectColumnsBegin_incl + j];
                                kregioncoverageisgood &= (my_coverage[subjectColumnsBegin_incl + j] >= min_coverage_threshold);
                                c++;
                            }
                        }

                        avgsupportkregion /= c;

						//if(i == 33 || i == 34){
						//	printf("%d %f\n", i, avgsupportkregion);
						//}
                        if(kregioncoverageisgood && avgsupportkregion >= 1.0-estimatedErrorrate){
                            my_corrected_subject[i] = my_consensus[globalIndex];
                            foundAColumn = true;
                        }
                    }
                }
                //perform block wide or-reduction on foundAColumn
                foundAColumn = BlockReduceBool(temp_storage.boolreduce).Reduce(foundAColumn, [](bool a, bool b){return a || b;});
				__syncthreads();

                if(threadIdx.x == 0){
                    d_subject_is_corrected[subjectIndex] = foundAColumn;
                }
            }
        }
    }

    void call_msa_correct_subject_kernel_async(
                            const char* d_consensus,
                            const float* d_support,
                            const int* d_coverage,
                            const int* d_origCoverages,
                            const char* d_multiple_sequence_alignments,
                            const MSAColumnProperties* d_msa_column_properties,
                            const int* d_indices_per_subject_prefixsum,
                            bool* d_is_high_quality_subject,
                            char* d_corrected_subjects,
							bool* d_subject_is_corrected,
                            int n_subjects,
                            int n_queries,
                            const int* d_num_indices,
                            size_t sequence_pitch,
                            size_t msa_pitch,
                            size_t msa_weights_pitch,
                            double estimatedErrorrate,
                            double avg_support_threshold,
                            double min_support_threshold,
                            double min_coverage_threshold,
                            int k_region,
                            int maximum_sequence_length,
                            hipStream_t stream){

        const int max_block_size = 256;
        const int blocksize = std::min(max_block_size, SDIV(maximum_sequence_length, 32) * 32);

        dim3 block(blocksize, 1, 1);
        dim3 grid(n_subjects);

        #define mycall(blocksize) msa_correct_subject_kernel<(blocksize)> \
                                <<<grid, block, 0, stream>>>( \
                                    d_consensus, \
                                    d_support, \
                                    d_coverage, \
                                    d_origCoverages, \
                                    d_multiple_sequence_alignments, \
                                    d_msa_column_properties, \
                                    d_indices_per_subject_prefixsum, \
                                    d_is_high_quality_subject, \
                                    d_corrected_subjects, \
                                    d_subject_is_corrected, \
                                    n_subjects, \
                                    n_queries, \
                                    d_num_indices, \
                                    sequence_pitch, \
                                    msa_pitch, \
                                    msa_weights_pitch, \
                                    estimatedErrorrate, \
                                    avg_support_threshold, \
                                    min_support_threshold, \
                                    min_coverage_threshold, \
                                    k_region); CUERR;

        assert(blocksize > 0 && blocksize <= max_block_size);

        switch(blocksize){
            case 32: mycall(32); break;
            case 64: mycall(64); break;
            case 96: mycall(96); break;
            case 128: mycall(128); break;
            case 160: mycall(160); break;
            case 192: mycall(192); break;
            case 224: mycall(224); break;
            case 256: mycall(256); break;
            default: mycall(256); break;
        }
		#undef mycall
    }





#endif

}
}
